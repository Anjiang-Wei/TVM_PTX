
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[4];
  __shared__ float placeholder_d_shared[4];
  __shared__ float placeholder_shared[800];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 4) {
      placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((k_outer_outer * 4) + ((int)threadIdx.x)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 50))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 50) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 100))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 12800))];
    placeholder_shared[((((int)threadIdx.x) + 150))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 150) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 200))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 25600))];
    placeholder_shared[((((int)threadIdx.x) + 250))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 250) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 300))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 38400))];
    placeholder_shared[((((int)threadIdx.x) + 350))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 350) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 400))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 51200))];
    placeholder_shared[((((int)threadIdx.x) + 450))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 450) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 500))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 64000))];
    placeholder_shared[((((int)threadIdx.x) + 550))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 550) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 600))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 76800))];
    placeholder_shared[((((int)threadIdx.x) + 650))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 650) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 700))] = placeholder1[((((((((int)blockIdx.x) * 102400) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 89600))];
    placeholder_shared[((((int)threadIdx.x) + 750))] = placeholder1[(((((((int)blockIdx.x) * 102400) + (((((int)threadIdx.x) + 750) >> 2) * 512)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    __syncthreads();
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 4))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 200))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 400))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 600))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 1))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 201))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 401))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 601))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 2))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 202))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 402))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 602))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 3))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 203))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 403))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 4) + 603))]));
  }
  T_add[(((((int)blockIdx.x) * 200) + ((int)threadIdx.x)))] = (T_dense[(0)] + placeholder2[(((((int)blockIdx.x) * 200) + ((int)threadIdx.x)))]);
  T_add[((((((int)blockIdx.x) * 200) + ((int)threadIdx.x)) + 50))] = (T_dense[(1)] + placeholder2[((((((int)blockIdx.x) * 200) + ((int)threadIdx.x)) + 50))]);
  T_add[((((((int)blockIdx.x) * 200) + ((int)threadIdx.x)) + 100))] = (T_dense[(2)] + placeholder2[((((((int)blockIdx.x) * 200) + ((int)threadIdx.x)) + 100))]);
  T_add[((((((int)blockIdx.x) * 200) + ((int)threadIdx.x)) + 150))] = (T_dense[(3)] + placeholder2[((((((int)blockIdx.x) * 200) + ((int)threadIdx.x)) + 150))]);
}


