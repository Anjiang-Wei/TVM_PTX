
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[6144];
  __shared__ float placeholder_shared[96];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 12; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 96; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) / 24) * 288) + (rc_outer_outer * 24)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) % 24)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 6) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) < 24) {
        if (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 96) {
          if (((int)threadIdx.x) < 4) {
            placeholder_shared[(((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[((((((rc_outer_outer * 3072) + (((int)threadIdx.x) * 768)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2) * 128)) + (((int)blockIdx.x) * 4)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s & 3)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 24; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 48) + rc_outer_inner))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 48) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 48) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 48) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3072))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3072))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3072))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3072))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 24))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 24))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 24))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 24))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3096))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3096))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3096))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) * 48) + rc_outer_inner) + 3096))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    T_relu[((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)))] = max((Conv2dOutput[(ax2_inner)] + placeholder2[((((int)blockIdx.x) * 4))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 1))] = max((Conv2dOutput[((ax2_inner + 2))] + placeholder2[(((((int)blockIdx.x) * 4) + 1))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 2))] = max((Conv2dOutput[((ax2_inner + 4))] + placeholder2[(((((int)blockIdx.x) * 4) + 2))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 3))] = max((Conv2dOutput[((ax2_inner + 6))] + placeholder2[(((((int)blockIdx.x) * 4) + 3))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 16384))] = max((Conv2dOutput[((ax2_inner + 8))] + placeholder2[((((int)blockIdx.x) * 4))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 16385))] = max((Conv2dOutput[((ax2_inner + 10))] + placeholder2[(((((int)blockIdx.x) * 4) + 1))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 16386))] = max((Conv2dOutput[((ax2_inner + 12))] + placeholder2[(((((int)blockIdx.x) * 4) + 2))]), 0.000000e+00f);
    T_relu[(((((((int)threadIdx.x) * 256) + (ax2_inner * 128)) + (((int)blockIdx.x) * 4)) + 16387))] = max((Conv2dOutput[((ax2_inner + 14))] + placeholder2[(((((int)blockIdx.x) * 4) + 3))]), 0.000000e+00f);
  }
}


