
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ compute, float* __restrict__ placeholder) {
  T_multiply[(((int)blockIdx.x))] = ((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * (5.000000e-01f + (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * -2.726142e-10f) + 2.770681e-08f)) + -2.101024e-06f)) + -5.692506e-05f)) + -7.349906e-04f)) + -2.954600e-03f)) + -1.609603e-02f)) / (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * -1.456607e-05f) + -2.133740e-04f)) + -1.682827e-03f)) + -7.373329e-03f)) + -1.426474e-02f)) * 5.000000e-01f)));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[128];
  __shared__ float placeholder_d_shared[8192];
  __shared__ float placeholder_shared[1024];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 16; ++j_c_outer_inner_init) {
      compute_local[(((b_c_outer_inner_init * 64) + (j_c_outer_inner_init * 2)))] = 0.000000e+00f;
      compute_local[((((b_c_outer_inner_init * 64) + (j_c_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      compute_local[((((b_c_outer_inner_init * 64) + (j_c_outer_inner_init * 2)) + 32))] = 0.000000e+00f;
      compute_local[((((b_c_outer_inner_init * 64) + (j_c_outer_inner_init * 2)) + 33))] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 128; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_d_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer_outer * 2) + (((int)threadIdx.x) >> 5)) >> 7) * 262144) + ((((int)blockIdx.x) >> 7) * 131072)) + ((((ax0_ax1_fused_ax2_fused_outer_outer * 2) + (((int)threadIdx.x) >> 5)) & 127) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 2048))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 4096))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 6144))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 8192))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 10240))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 12288))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 14336))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 16384))];
    placeholder_shared[((((int)threadIdx.x) + 576))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 18432))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 20480))];
    placeholder_shared[((((int)threadIdx.x) + 704))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 22528))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 24576))];
    placeholder_shared[((((int)threadIdx.x) + 832))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 26624))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 28672))];
    placeholder_shared[((((int)threadIdx.x) + 960))] = placeholder1[(((((((((int)blockIdx.x) & 127) * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + (k_outer_outer * 32)) + (((int)threadIdx.x) & 31)) + 30720))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
        for (int j_c_outer_inner = 0; j_c_outer_inner < 16; ++j_c_outer_inner) {
          compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] = (compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] + (placeholder_d_shared[((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)))] * placeholder_shared[(((j_c_outer_inner * 64) + (k_outer_inner * 4)))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] + (placeholder_d_shared[((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 32))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 32))] * placeholder_shared[(((j_c_outer_inner * 64) + (k_outer_inner * 4)))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 32))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 32))]));
          compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] = (compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 1))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 1))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 1))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 33))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 33))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 1))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 33))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 33))]));
          compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] = (compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 2))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 2))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 2))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 34))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 34))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 2))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 34))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 34))]));
          compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] = (compute_local[(((b_c_outer_inner * 64) + (j_c_outer_inner * 2)))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 3))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 3))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 1))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 3))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 35))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 32))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 35))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 3))]));
          compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] = (compute_local[((((b_c_outer_inner * 64) + (j_c_outer_inner * 2)) + 33))] + (placeholder_d_shared[(((((b_c_outer_inner * 4096) + (((int)threadIdx.x) * 64)) + (k_outer_inner * 4)) + 35))] * placeholder_shared[((((j_c_outer_inner * 64) + (k_outer_inner * 4)) + 35))]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      for (int j_inner = 0; j_inner < 32; ++j_inner) {
        compute[(((((((b_inner * 1048576) + ((((int)blockIdx.x) >> 7) * 524288)) + (((int)threadIdx.x) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 127) * 32)) + j_inner))] = compute_local[((((b_inner * 64) + (i_inner * 32)) + j_inner))];
      }
    }
  }
}


