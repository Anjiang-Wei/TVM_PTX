
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[1344];
  __shared__ float PaddedInput_shared[1792];
  __shared__ float placeholder_shared[24];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 14; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
          for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
            for (int ff_inner_init = 0; ff_inner_init < 3; ++ff_inner_init) {
              Conv2dOutput[(((((((nn_inner_init * 336) + (yy_outer_inner_init * 24)) + (yy_inner_init * 12)) + (xx_outer_inner_init * 6)) + (xx_inner_init * 3)) + ff_inner_init))] = 0.000000e+00f;
              Conv2dOutput[((((((((nn_inner_init * 336) + (yy_outer_inner_init * 24)) + (yy_inner_init * 12)) + (xx_outer_inner_init * 6)) + (xx_inner_init * 3)) + ff_inner_init) + 672))] = 0.000000e+00f;
            }
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 96; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 56; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 43008) + ((((int)threadIdx.x) >> 2) * 5376)) + (((int)blockIdx.x) * 384)) + ((((int)threadIdx.x) & 3) * 96)) + rc_outer_outer))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 24) {
        if (((int)threadIdx.x) < 4) {
          placeholder_shared[(((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[((((rc_outer_outer * 24) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))];
        }
      }
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 14; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
        for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
              for (int ff_inner = 0; ff_inner < 3; ++ff_inner) {
                Conv2dOutput[(((((((nn_inner * 336) + (yy_outer_inner * 24)) + (yy_inner * 12)) + (xx_outer_inner * 6)) + (xx_inner * 3)) + ff_inner))] = (Conv2dOutput[(((((((nn_inner * 336) + (yy_outer_inner * 24)) + (yy_inner * 12)) + (xx_outer_inner * 6)) + (xx_inner * 3)) + ff_inner))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 3) * 448) + (nn_inner * 224)) + (((((int)threadIdx.x) & 7) >> 2) * 112)) + (yy_outer_inner * 8)) + (yy_inner * 4)) + (xx_outer_inner * 2)) + xx_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 3) + ff_inner))]));
                Conv2dOutput[((((((((nn_inner * 336) + (yy_outer_inner * 24)) + (yy_inner * 12)) + (xx_outer_inner * 6)) + (xx_inner * 3)) + ff_inner) + 672))] = (Conv2dOutput[((((((((nn_inner * 336) + (yy_outer_inner * 24)) + (yy_inner * 12)) + (xx_outer_inner * 6)) + (xx_inner * 3)) + ff_inner) + 672))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 3) * 448) + (nn_inner * 224)) + (((((int)threadIdx.x) & 7) >> 2) * 112)) + (yy_outer_inner * 8)) + (yy_inner * 4)) + (xx_outer_inner * 2)) + xx_inner))] * placeholder_shared[(((((((int)threadIdx.x) & 3) * 3) + ff_inner) + 12))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 28; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 3; ++ax3_inner) {
          T_add[((((((((((((int)threadIdx.x) >> 3) * 150528) + (ax0_inner * 75264)) + (((((int)threadIdx.x) & 7) >> 2) * 37632)) + (ax1_inner * 1344)) + (((int)blockIdx.x) * 96)) + (ax2_inner * 24)) + ((((int)threadIdx.x) & 3) * 3)) + ax3_inner))] = (Conv2dOutput[(((((ax0_inner * 336) + (ax1_inner * 12)) + (ax2_inner * 3)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 3) * 3) + ax3_inner))]);
          T_add[(((((((((((((int)threadIdx.x) >> 3) * 150528) + (ax0_inner * 75264)) + (((((int)threadIdx.x) & 7) >> 2) * 37632)) + (ax1_inner * 1344)) + (((int)blockIdx.x) * 96)) + (ax2_inner * 24)) + ((((int)threadIdx.x) & 3) * 3)) + ax3_inner) + 12))] = (Conv2dOutput[((((((ax0_inner * 336) + (ax1_inner * 12)) + (ax2_inner * 3)) + ax3_inner) + 672))] + placeholder2[(((((((int)threadIdx.x) & 3) * 3) + ax3_inner) + 12))]);
        }
      }
    }
  }
}


