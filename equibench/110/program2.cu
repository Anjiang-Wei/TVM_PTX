
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[384];
  __shared__ float placeholder_shared[96];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    ((float2*)(PaddedInput_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder + ((((((((int)blockIdx.x) >> 3) * 24576) + ((((int)threadIdx.x) / 3) * 768)) + (rc_outer_outer * 6)) + ((((int)threadIdx.x) % 3) * 2)))))[0];
    int2 _1;
      int2 _2;
        int2 _3;
          int2 _4 = make_int2((((((((int)threadIdx.x) * 2) + 128) / 192) * 196608) + ((((int)blockIdx.x) >> 3) * 24576)), (((((((int)threadIdx.x) * 2) + 128) / 192) * 196608) + ((((int)blockIdx.x) >> 3) * 24576)));
          int2 _5;
            int2 _6;
              int2 _7 = (make_int2)(((((((int)threadIdx.x) * 2) + 128) % 192))+(1*0), ((((((int)threadIdx.x) * 2) + 128) % 192))+(1*1));
              int2 _8 = make_int2(6, 6);
              _6.x = (_7.x%_8.x);
              _6.y = (_7.y%_8.y);
            int2 _9;
              int2 _10 = (make_int2)(((((((int)threadIdx.x) * 2) + 128) % 192))+(1*0), ((((((int)threadIdx.x) * 2) + 128) % 192))+(1*1));
              int2 _11 = make_int2(6, 6);
              _9.x = (_10.x/_11.x);
              _9.y = (_10.y/_11.y);
            int2 _12;
            ushort2 _13;
              ushort2 _14;
                ushort2 _15;
                  int2 _16 = make_int2(6, 6);
                  int2 _17 = make_int2(0, 0);
                  _15.x = (_16.x>=_17.x);
                  _15.y = (_16.y>=_17.y);
                ushort2 _18;
                  int2 _19 = make_int2(0, 0);
                  _18.x = (_6.x>=_19.x);
                  _18.y = (_6.y>=_19.y);
                _14.x = (_15.x&&_18.x);
                _14.y = (_15.y&&_18.y);
              ushort2 _20;
                ushort2 _21;
                  int2 _22 = make_int2(6, 6);
                  int2 _23 = make_int2(0, 0);
                  _21.x = (_22.x<_23.x);
                  _21.y = (_22.y<_23.y);
                ushort2 _24;
                  int2 _25 = make_int2(0, 0);
                  _24.x = (_6.x<=_25.x);
                  _24.y = (_6.y<=_25.y);
                _20.x = (_21.x&&_24.x);
                _20.y = (_21.y&&_24.y);
              _13.x = (_14.x||_20.x);
              _13.y = (_14.y||_20.y);
            int2 _26;
              int2 _27 = make_int2(1, 1);
              _26.x = (_9.x-_27.x);
              _26.y = (_9.y-_27.y);
            _12.x = (bool(_13.x)?_9.x:_26.x);
            _12.y = (bool(_13.y)?_9.y:_26.y);
            int2 _28 = make_int2(768, 768);
            _5.x = (_12.x*_28.x);
            _5.y = (_12.y*_28.y);
          _3.x = (_4.x+_5.x);
          _3.y = (_4.y+_5.y);
        int2 _29 = make_int2((rc_outer_outer * 6), (rc_outer_outer * 6));
        _2.x = (_3.x+_29.x);
        _2.y = (_3.y+_29.y);
      int2 _30 = (make_int2)(((((((int)threadIdx.x) * 2) + 2) % 6))+(1*0), ((((((int)threadIdx.x) * 2) + 2) % 6))+(1*1));
      _1.x = (_2.x+_30.x);
      _1.y = (_2.y+_30.y);
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = make_float2(placeholder[_1.x],placeholder[_1.y]);
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(placeholder + (((((((((((int)threadIdx.x) * 2) + 256) / 192) * 196608) + ((((int)blockIdx.x) >> 3) * 24576)) + ((((((int)threadIdx.x) * 2) + 64) / 6) * 768)) + (rc_outer_outer * 6)) + (((((int)threadIdx.x) * 2) + 4) % 6)))))[0];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[((((int)threadIdx.x) * 3))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) * 3) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) * 3) & 15)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder1[(((((rc_outer_outer * 768) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder1[(((((rc_outer_outer * 768) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)))];
    }
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
      for (int rc_inner = 0; rc_inner < 6; ++rc_inner) {
        Conv2dOutput[(xx_outer_inner)] = (Conv2dOutput[(xx_outer_inner)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 24) + (xx_outer_inner * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 16) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[((xx_outer_inner + 8))] = (Conv2dOutput[((xx_outer_inner + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 24) + (xx_outer_inner * 6)) + rc_inner) + 96))] * placeholder_shared[(((rc_inner * 16) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[((xx_outer_inner + 4))] = (Conv2dOutput[((xx_outer_inner + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 24) + (xx_outer_inner * 6)) + rc_inner) + 192))] * placeholder_shared[(((rc_inner * 16) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[((xx_outer_inner + 12))] = (Conv2dOutput[((xx_outer_inner + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 24) + (xx_outer_inner * 6)) + rc_inner) + 288))] * placeholder_shared[(((rc_inner * 16) + (((int)threadIdx.x) & 15)))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      T_relu[(((((((ax0_inner * 32768) + ((((int)blockIdx.x) >> 3) * 4096)) + ((((int)threadIdx.x) >> 4) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((int)threadIdx.x) & 15)))] = max((Conv2dOutput[(((ax0_inner * 4) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + (((int)threadIdx.x) & 15)))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 32768) + ((((int)blockIdx.x) >> 3) * 4096)) + ((((int)threadIdx.x) >> 4) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((int)threadIdx.x) & 15)) + 2048))] = max((Conv2dOutput[((((ax0_inner * 4) + ax2_inner) + 8))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + (((int)threadIdx.x) & 15)))]), 0.000000e+00f);
    }
  }
}


