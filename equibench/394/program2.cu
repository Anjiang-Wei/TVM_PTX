
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[1568];
  __shared__ float placeholder_shared[160];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 25; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 2) * 39200) + ((((int)threadIdx.x) >> 3) * 200)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 490))] = placeholder[((((((((int)blockIdx.x) >> 2) * 39200) + (((((int)threadIdx.x) + 490) >> 3) * 200)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 980))] = placeholder[((((((((int)blockIdx.x) >> 2) * 39200) + (((((int)threadIdx.x) + 980) >> 3) * 200)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    if (((int)threadIdx.x) < 98) {
      PaddedInput_shared[((((int)threadIdx.x) + 1470))] = placeholder[((((((((int)blockIdx.x) >> 2) * 39200) + (((((int)threadIdx.x) + 1470) >> 3) * 200)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 160) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 640) + ((((int)threadIdx.x) / 20) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 20)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)))] * placeholder_shared[((((int)threadIdx.x) % 10))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 10))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 56))] * placeholder_shared[((((int)threadIdx.x) % 10))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 56))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 10))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 20))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 30))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 57))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 20))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 57))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 30))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 2))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 40))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 2))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 50))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 58))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 40))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 58))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 50))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 3))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 60))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 3))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 70))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 59))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 60))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 59))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 70))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 112))] * placeholder_shared[((((int)threadIdx.x) % 10))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 112))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 10))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 168))] * placeholder_shared[((((int)threadIdx.x) % 10))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 168))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 10))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 113))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 20))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 113))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 30))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 169))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 20))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 169))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 30))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 114))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 40))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 114))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 50))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 170))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 40))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 170))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 50))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 115))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 60))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 115))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 70))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 171))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 60))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 171))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 70))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 80))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 90))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 60))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 80))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 60))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 90))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 100))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 110))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 61))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 100))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 61))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 110))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 6))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 120))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 6))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 130))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 62))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 120))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 62))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 130))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 7))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 140))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 7))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 150))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 63))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 140))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 63))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 150))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 116))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 80))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 116))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 90))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 172))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 80))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 172))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 90))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 117))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 100))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 117))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 110))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 173))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 100))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 173))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 110))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 118))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 120))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 118))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 130))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 174))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 120))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 174))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 130))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 119))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 140))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 119))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 150))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 175))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 140))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 70) * 224) + (((((int)threadIdx.x) % 70) / 10) * 8)) + 175))] * placeholder_shared[(((((int)threadIdx.x) % 10) + 150))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    T_add[((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)))] = ((Conv2dOutput[(ax1_inner)] + placeholder2[((((((int)blockIdx.x) & 3) * 20) + (((int)threadIdx.x) % 10)))]) + placeholder3[((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)))]);
    T_add[(((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)) + 10))] = ((Conv2dOutput[((ax1_inner + 2))] + placeholder2[(((((((int)blockIdx.x) & 3) * 20) + (((int)threadIdx.x) % 10)) + 10))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)) + 10))]);
    T_add[(((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)) + 560))] = ((Conv2dOutput[((ax1_inner + 4))] + placeholder2[((((((int)blockIdx.x) & 3) * 20) + (((int)threadIdx.x) % 10)))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)) + 560))]);
    T_add[(((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)) + 570))] = ((Conv2dOutput[((ax1_inner + 6))] + placeholder2[(((((((int)blockIdx.x) & 3) * 20) + (((int)threadIdx.x) % 10)) + 10))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 15680) + ((((int)threadIdx.x) / 70) * 2240)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 70) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((int)threadIdx.x) % 10)) + 570))]);
  }
}


