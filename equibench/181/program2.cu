
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[256];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[256];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 8; ++xx_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
      Conv2dOutput[(((nn_inner_init * 16) + xx_outer_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 32))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 64))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 96))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 128))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 160))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 192))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 224))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 40))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 72))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 104))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 136))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 168))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 200))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 16) + xx_outer_inner_init) + 232))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 368; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) * 11776) + ((((int)threadIdx.x) >> 1) * 736)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[((((((((int)blockIdx.x) * 11776) + ((((int)threadIdx.x) >> 1) * 736)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 47104))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((int)blockIdx.x) * 11776) + ((((int)threadIdx.x) >> 1) * 736)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 94208))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[((((((((int)blockIdx.x) * 11776) + ((((int)threadIdx.x) >> 1) * 736)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 141312))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 256) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 32))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 64))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 96))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 160))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 192))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 224))];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 8; ++xx_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
          Conv2dOutput[(((nn_inner * 16) + xx_outer_inner))] = (Conv2dOutput[(((nn_inner * 16) + xx_outer_inner))] + (PaddedInput_shared[((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((rc_inner * 128) + ((int)threadIdx.x)))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 32))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 32))] + (PaddedInput_shared[((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 32))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 64))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 64))] + (PaddedInput_shared[((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 64))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 96))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 96))] + (PaddedInput_shared[((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 96))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 128))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 128))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 64))] * placeholder_shared[(((rc_inner * 128) + ((int)threadIdx.x)))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 160))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 160))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 64))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 32))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 192))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 192))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 64))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 64))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 224))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 224))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 64))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 96))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 8))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 8))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 16))] * placeholder_shared[(((rc_inner * 128) + ((int)threadIdx.x)))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 40))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 40))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 16))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 32))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 72))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 72))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 16))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 64))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 104))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 104))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 16))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 96))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 136))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 136))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 80))] * placeholder_shared[(((rc_inner * 128) + ((int)threadIdx.x)))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 168))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 168))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 80))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 32))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 200))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 200))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 80))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 64))]));
          Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 232))] = (Conv2dOutput[((((nn_inner * 16) + xx_outer_inner) + 232))] + (PaddedInput_shared[(((((nn_inner * 32) + (xx_outer_inner * 2)) + rc_inner) + 80))] * placeholder_shared[((((rc_inner * 128) + ((int)threadIdx.x)) + 96))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
        T_relu[((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)))] = max((Conv2dOutput[((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner))] + placeholder2[(((int)threadIdx.x))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 32))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 32))] + placeholder2[((((int)threadIdx.x) + 32))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 64))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 64))] + placeholder2[((((int)threadIdx.x) + 64))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 96))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 96))] + placeholder2[((((int)threadIdx.x) + 96))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 16384))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 128))] + placeholder2[(((int)threadIdx.x))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 16416))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 160))] + placeholder2[((((int)threadIdx.x) + 32))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 16448))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 192))] + placeholder2[((((int)threadIdx.x) + 64))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 8192) + (((int)blockIdx.x) * 2048)) + (ax1_inner * 1024)) + (ax2_inner * 128)) + ((int)threadIdx.x)) + 16480))] = max((Conv2dOutput[(((((ax0_inner * 16) + (ax1_inner * 8)) + ax2_inner) + 224))] + placeholder2[((((int)threadIdx.x) + 96))]), 0.000000e+00f);
      }
    }
  }
}


