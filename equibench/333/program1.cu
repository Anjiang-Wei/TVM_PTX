
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ tensor, float* __restrict__ tensor1) {
  tensor[(((int)blockIdx.x))] = (tensor1[(((int)blockIdx.x))] * 1.020408e-02f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((((int)blockIdx.x) * 4) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 14; ++rv0) {
    for (int rv1 = 0; rv1 < 7; ++rv1) {
      tensor[(((((int)blockIdx.x) * 4) + ((int)threadIdx.x)))] = (tensor[(((((int)blockIdx.x) * 4) + ((int)threadIdx.x)))] + placeholder[((((((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) >> 9) * 50176) + (rv0 * 3584)) + (rv1 * 512)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 511)))]);
    }
  }
}


