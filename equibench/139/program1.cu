
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[1536];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[192];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 8; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 4; ++xx_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 48; ++ff_outer_inner_init) {
        Conv2dOutput[((((yy_outer_inner_init * 192) + (xx_outer_inner_init * 48)) + ff_outer_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      ((float4*)(PaddedInput_shared + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4))))[0] = ((float4*)(placeholder + (((((((((int)blockIdx.x) >> 4) * 131072) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 2) * 16384)) + (((((int)blockIdx.x) & 15) >> 3) * 8192)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 3) * 2048)) + (rc_outer_outer * 4)))))[0];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 192; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1)] = placeholder1[(((((rc_outer_outer * 1536) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 / 48) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 % 48)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 8; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 48; ++ff_outer_inner) {
          for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
            Conv2dOutput[((((yy_outer_inner * 192) + (xx_outer_inner * 48)) + ff_outer_inner))] = (Conv2dOutput[((((yy_outer_inner * 192) + (xx_outer_inner * 48)) + ff_outer_inner))] + (PaddedInput_shared[((((yy_outer_inner * 16) + (xx_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((rc_inner * 48) + ff_outer_inner))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 48; ++ax3_inner) {
        T_relu[((((((((((int)blockIdx.x) >> 4) * 24576) + (ax1_inner * 3072)) + (((((int)blockIdx.x) & 15) >> 3) * 1536)) + (ax2_inner * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 192) + (ax2_inner * 48)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) & 7) * 48) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


