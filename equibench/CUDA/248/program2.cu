
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[448];
  __shared__ float PaddedInput_shared[224];
  __shared__ float placeholder_shared[4096];
  for (int yy_inner_init = 0; yy_inner_init < 7; ++yy_inner_init) {
    Conv2dOutput[((yy_inner_init * 64))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 22))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 23))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 25))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 26))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 27))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 29))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 30))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 31))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 32))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 33))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 34))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 35))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 36))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 37))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 38))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 39))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 40))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 41))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 42))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 43))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 44))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 45))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 46))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 47))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 48))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 49))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 50))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 51))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 52))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 53))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 54))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 55))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 56))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 57))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 58))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 59))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 60))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 61))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 62))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 64) + 63))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 22; ++rc_outer_outer) {
    __syncthreads();
    ((float2*)(PaddedInput_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder + ((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 4))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 4))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 8))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 8))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 12))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 12))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 16))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 16))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 20))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 20))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 24))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 24))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 28))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 28))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 32))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 4928))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 36))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 36) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 4)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 40))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 40) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 8)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 44))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 44) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 12)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 48))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 48) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 16)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 52))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 52) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 20)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 56))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 56) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 24)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 60))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 60) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 28)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 9856))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 68))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 68) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 4)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 72))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 72) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 8)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 76))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 76) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 12)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 80))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 80) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 16)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 84))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 84) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 20)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 88))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 88) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 24)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 92))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 92) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 28)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 96))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 14784))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 100))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 100) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 4)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 104))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 104) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 8)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 108))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 108) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 12)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 112))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 112) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 16)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 116))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 116) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 20)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 120))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 120) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 24)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 124))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 124) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 28)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 19712))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 132))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 132) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 4)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 136))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 136) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 8)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 140))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 140) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 12)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 144))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 144) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 16)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 148))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 148) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 20)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 152))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 152) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 24)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 156))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 156) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 28)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 160))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 24640))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 164))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 164) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 4)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 168))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 168) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 8)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 172))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 172) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 12)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 176))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 176) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 16)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 180))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 180) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 20)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 184))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 184) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 24)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 188))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 188) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 28)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 192))))[0] = ((float2*)(placeholder + (((((((int)blockIdx.x) * 704) + (rc_outer_outer * 32)) + (((int)threadIdx.x) * 2)) + 29568))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 196))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 196) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 4)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 200))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 200) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 8)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 204))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 204) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 12)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 208))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 208) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 16)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 212))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 212) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 20)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 216))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 216) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 24)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 220))))[0] = ((float2*)(placeholder + ((((((((((int)threadIdx.x) * 2) + 220) >> 5) * 4928) + (((int)blockIdx.x) * 704)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) * 2) + 28)))))[0];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 512; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      ((float4*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) * 4)))))[0] = ((float4*)(placeholder1 + ((((rc_outer_outer * 4096) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8)) + (((int)threadIdx.x) * 4)))))[0];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 32; ++rc_inner) {
      for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
        Conv2dOutput[((yy_inner * 64))] = (Conv2dOutput[((yy_inner * 64))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) * 64)))]));
        Conv2dOutput[(((yy_inner * 64) + 1))] = (Conv2dOutput[(((yy_inner * 64) + 1))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 1))]));
        Conv2dOutput[(((yy_inner * 64) + 2))] = (Conv2dOutput[(((yy_inner * 64) + 2))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 2))]));
        Conv2dOutput[(((yy_inner * 64) + 3))] = (Conv2dOutput[(((yy_inner * 64) + 3))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 3))]));
        Conv2dOutput[(((yy_inner * 64) + 4))] = (Conv2dOutput[(((yy_inner * 64) + 4))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 4))]));
        Conv2dOutput[(((yy_inner * 64) + 5))] = (Conv2dOutput[(((yy_inner * 64) + 5))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 5))]));
        Conv2dOutput[(((yy_inner * 64) + 6))] = (Conv2dOutput[(((yy_inner * 64) + 6))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 6))]));
        Conv2dOutput[(((yy_inner * 64) + 7))] = (Conv2dOutput[(((yy_inner * 64) + 7))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 7))]));
        Conv2dOutput[(((yy_inner * 64) + 8))] = (Conv2dOutput[(((yy_inner * 64) + 8))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 8))]));
        Conv2dOutput[(((yy_inner * 64) + 9))] = (Conv2dOutput[(((yy_inner * 64) + 9))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 9))]));
        Conv2dOutput[(((yy_inner * 64) + 10))] = (Conv2dOutput[(((yy_inner * 64) + 10))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 10))]));
        Conv2dOutput[(((yy_inner * 64) + 11))] = (Conv2dOutput[(((yy_inner * 64) + 11))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 11))]));
        Conv2dOutput[(((yy_inner * 64) + 12))] = (Conv2dOutput[(((yy_inner * 64) + 12))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 12))]));
        Conv2dOutput[(((yy_inner * 64) + 13))] = (Conv2dOutput[(((yy_inner * 64) + 13))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 13))]));
        Conv2dOutput[(((yy_inner * 64) + 14))] = (Conv2dOutput[(((yy_inner * 64) + 14))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 14))]));
        Conv2dOutput[(((yy_inner * 64) + 15))] = (Conv2dOutput[(((yy_inner * 64) + 15))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 15))]));
        Conv2dOutput[(((yy_inner * 64) + 16))] = (Conv2dOutput[(((yy_inner * 64) + 16))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 16))]));
        Conv2dOutput[(((yy_inner * 64) + 17))] = (Conv2dOutput[(((yy_inner * 64) + 17))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 17))]));
        Conv2dOutput[(((yy_inner * 64) + 18))] = (Conv2dOutput[(((yy_inner * 64) + 18))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 18))]));
        Conv2dOutput[(((yy_inner * 64) + 19))] = (Conv2dOutput[(((yy_inner * 64) + 19))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 19))]));
        Conv2dOutput[(((yy_inner * 64) + 20))] = (Conv2dOutput[(((yy_inner * 64) + 20))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 20))]));
        Conv2dOutput[(((yy_inner * 64) + 21))] = (Conv2dOutput[(((yy_inner * 64) + 21))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 21))]));
        Conv2dOutput[(((yy_inner * 64) + 22))] = (Conv2dOutput[(((yy_inner * 64) + 22))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 22))]));
        Conv2dOutput[(((yy_inner * 64) + 23))] = (Conv2dOutput[(((yy_inner * 64) + 23))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 23))]));
        Conv2dOutput[(((yy_inner * 64) + 24))] = (Conv2dOutput[(((yy_inner * 64) + 24))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 24))]));
        Conv2dOutput[(((yy_inner * 64) + 25))] = (Conv2dOutput[(((yy_inner * 64) + 25))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 25))]));
        Conv2dOutput[(((yy_inner * 64) + 26))] = (Conv2dOutput[(((yy_inner * 64) + 26))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 26))]));
        Conv2dOutput[(((yy_inner * 64) + 27))] = (Conv2dOutput[(((yy_inner * 64) + 27))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 27))]));
        Conv2dOutput[(((yy_inner * 64) + 28))] = (Conv2dOutput[(((yy_inner * 64) + 28))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 28))]));
        Conv2dOutput[(((yy_inner * 64) + 29))] = (Conv2dOutput[(((yy_inner * 64) + 29))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 29))]));
        Conv2dOutput[(((yy_inner * 64) + 30))] = (Conv2dOutput[(((yy_inner * 64) + 30))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 30))]));
        Conv2dOutput[(((yy_inner * 64) + 31))] = (Conv2dOutput[(((yy_inner * 64) + 31))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 31))]));
        Conv2dOutput[(((yy_inner * 64) + 32))] = (Conv2dOutput[(((yy_inner * 64) + 32))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 32))]));
        Conv2dOutput[(((yy_inner * 64) + 33))] = (Conv2dOutput[(((yy_inner * 64) + 33))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 33))]));
        Conv2dOutput[(((yy_inner * 64) + 34))] = (Conv2dOutput[(((yy_inner * 64) + 34))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 34))]));
        Conv2dOutput[(((yy_inner * 64) + 35))] = (Conv2dOutput[(((yy_inner * 64) + 35))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 35))]));
        Conv2dOutput[(((yy_inner * 64) + 36))] = (Conv2dOutput[(((yy_inner * 64) + 36))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 36))]));
        Conv2dOutput[(((yy_inner * 64) + 37))] = (Conv2dOutput[(((yy_inner * 64) + 37))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 37))]));
        Conv2dOutput[(((yy_inner * 64) + 38))] = (Conv2dOutput[(((yy_inner * 64) + 38))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 38))]));
        Conv2dOutput[(((yy_inner * 64) + 39))] = (Conv2dOutput[(((yy_inner * 64) + 39))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 39))]));
        Conv2dOutput[(((yy_inner * 64) + 40))] = (Conv2dOutput[(((yy_inner * 64) + 40))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 40))]));
        Conv2dOutput[(((yy_inner * 64) + 41))] = (Conv2dOutput[(((yy_inner * 64) + 41))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 41))]));
        Conv2dOutput[(((yy_inner * 64) + 42))] = (Conv2dOutput[(((yy_inner * 64) + 42))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 42))]));
        Conv2dOutput[(((yy_inner * 64) + 43))] = (Conv2dOutput[(((yy_inner * 64) + 43))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 43))]));
        Conv2dOutput[(((yy_inner * 64) + 44))] = (Conv2dOutput[(((yy_inner * 64) + 44))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 44))]));
        Conv2dOutput[(((yy_inner * 64) + 45))] = (Conv2dOutput[(((yy_inner * 64) + 45))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 45))]));
        Conv2dOutput[(((yy_inner * 64) + 46))] = (Conv2dOutput[(((yy_inner * 64) + 46))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 46))]));
        Conv2dOutput[(((yy_inner * 64) + 47))] = (Conv2dOutput[(((yy_inner * 64) + 47))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 47))]));
        Conv2dOutput[(((yy_inner * 64) + 48))] = (Conv2dOutput[(((yy_inner * 64) + 48))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 48))]));
        Conv2dOutput[(((yy_inner * 64) + 49))] = (Conv2dOutput[(((yy_inner * 64) + 49))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 49))]));
        Conv2dOutput[(((yy_inner * 64) + 50))] = (Conv2dOutput[(((yy_inner * 64) + 50))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 50))]));
        Conv2dOutput[(((yy_inner * 64) + 51))] = (Conv2dOutput[(((yy_inner * 64) + 51))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 51))]));
        Conv2dOutput[(((yy_inner * 64) + 52))] = (Conv2dOutput[(((yy_inner * 64) + 52))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 52))]));
        Conv2dOutput[(((yy_inner * 64) + 53))] = (Conv2dOutput[(((yy_inner * 64) + 53))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 53))]));
        Conv2dOutput[(((yy_inner * 64) + 54))] = (Conv2dOutput[(((yy_inner * 64) + 54))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 54))]));
        Conv2dOutput[(((yy_inner * 64) + 55))] = (Conv2dOutput[(((yy_inner * 64) + 55))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 55))]));
        Conv2dOutput[(((yy_inner * 64) + 56))] = (Conv2dOutput[(((yy_inner * 64) + 56))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 56))]));
        Conv2dOutput[(((yy_inner * 64) + 57))] = (Conv2dOutput[(((yy_inner * 64) + 57))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 57))]));
        Conv2dOutput[(((yy_inner * 64) + 58))] = (Conv2dOutput[(((yy_inner * 64) + 58))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 58))]));
        Conv2dOutput[(((yy_inner * 64) + 59))] = (Conv2dOutput[(((yy_inner * 64) + 59))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 59))]));
        Conv2dOutput[(((yy_inner * 64) + 60))] = (Conv2dOutput[(((yy_inner * 64) + 60))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 60))]));
        Conv2dOutput[(((yy_inner * 64) + 61))] = (Conv2dOutput[(((yy_inner * 64) + 61))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 61))]));
        Conv2dOutput[(((yy_inner * 64) + 62))] = (Conv2dOutput[(((yy_inner * 64) + 62))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 62))]));
        Conv2dOutput[(((yy_inner * 64) + 63))] = (Conv2dOutput[(((yy_inner * 64) + 63))] + (PaddedInput_shared[(((yy_inner * 32) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 64)) + 63))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 64; ++ax3_inner) {
      T_relu[(((((ax1_inner * 896) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) * 64)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 64) + ax3_inner))] + placeholder2[(((((int)threadIdx.x) * 64) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


