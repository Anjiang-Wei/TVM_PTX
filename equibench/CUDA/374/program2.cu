
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[2048];
  __shared__ float placeholder_shared[12];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 4; ++xx_inner_init) {
        Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 4)) + xx_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 4)) + xx_inner_init) + 16))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 24; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 22; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 2048) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) < 1024) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 8) * 256) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 5)) & 7) * 32)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 31)))] = placeholder[((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 8) * 301056) + ((((int)blockIdx.x) / 56) * 43008)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 5)) & 7) * 5376)) + (((((int)blockIdx.x) % 56) >> 3) * 768)) + (((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 31) >> 2) * 96)) + (rc_outer_outer * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)))];
          }
        }
      }
    }
    if (((int)threadIdx.x) < 12) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 96) + ((((int)threadIdx.x) / 3) * 24)) + ((((int)blockIdx.x) & 7) * 3)) + (((int)threadIdx.x) % 3)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
            Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 4)) + xx_inner))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 4)) + xx_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 6) * 256) + (((((int)threadIdx.x) % 6) / 3) * 64)) + (yy_outer_inner * 32)) + (xx_outer_inner * 16)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[(((rc_inner * 3) + (((int)threadIdx.x) % 3)))]));
            Conv2dOutput[(((((yy_outer_inner * 8) + (xx_outer_inner * 4)) + xx_inner) + 16))] = (Conv2dOutput[(((((yy_outer_inner * 8) + (xx_outer_inner * 4)) + xx_inner) + 16))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 6) * 256) + (((((int)threadIdx.x) % 6) / 3) * 64)) + (yy_outer_inner * 32)) + (xx_outer_inner * 16)) + (xx_inner * 4)) + rc_inner) + 128))] * placeholder_shared[(((rc_inner * 3) + (((int)threadIdx.x) % 3)))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
      T_add[((((((((((((int)threadIdx.x) / 6) * 75264) + ((((int)blockIdx.x) / 56) * 10752)) + (((((int)threadIdx.x) % 6) / 3) * 2688)) + (ax1_inner * 1344)) + (((((int)blockIdx.x) % 56) >> 3) * 192)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 7) * 3)) + (((int)threadIdx.x) % 3)))] = (Conv2dOutput[(((ax1_inner * 8) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 7) * 3) + (((int)threadIdx.x) % 3)))]);
      T_add[(((((((((((((int)threadIdx.x) / 6) * 75264) + ((((int)blockIdx.x) / 56) * 10752)) + (((((int)threadIdx.x) % 6) / 3) * 2688)) + (ax1_inner * 1344)) + (((((int)blockIdx.x) % 56) >> 3) * 192)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 7) * 3)) + (((int)threadIdx.x) % 3)) + 5376))] = (Conv2dOutput[((((ax1_inner * 8) + ax2_inner) + 16))] + placeholder2[((((((int)blockIdx.x) & 7) * 3) + (((int)threadIdx.x) % 3)))]);
    }
  }
}


