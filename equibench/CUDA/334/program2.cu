
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
  float T_dense[200];
  __shared__ float placeholder_d_shared[8];
  __shared__ float placeholder_shared[3200];
  for (int j_outer_inner_init = 0; j_outer_inner_init < 25; ++j_outer_inner_init) {
    T_dense[((j_outer_inner_init * 2))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 100))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 101))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 50))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 150))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 51))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 151))] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 50; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[((((int)threadIdx.x) * 4))] = placeholder[(((((int)threadIdx.x) * 200) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder[((((((int)threadIdx.x) * 200) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder[((((((int)threadIdx.x) * 200) + (k_outer_outer * 2)) + 100))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder[((((((int)threadIdx.x) * 200) + (k_outer_outer * 2)) + 101))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 3200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 6400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 192))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 9600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 12800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 320))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 16000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 384))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 19200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 448))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 22400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 512))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 25600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 576))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 28800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 640))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 32000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 704))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 35200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 768))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 38400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 832))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 41600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 896))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 44800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 960))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 48000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1024))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 51200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1088))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 54400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1152))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 57600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1216))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 60800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1280))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 64000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1344))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 67200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1408))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 70400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1472))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 73600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1536))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 76800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1600))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 80000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1664))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 83200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1728))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 86400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1792))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 89600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1856))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 92800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1920))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 96000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 1984))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 99200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2048))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 102400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2112))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 105600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2176))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 108800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2240))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 112000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2304))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 115200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2368))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 118400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2432))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 121600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2496))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 124800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2560))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 128000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2624))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 131200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2688))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 134400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2752))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 137600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2816))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 140800))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2880))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 144000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 2944))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 147200))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 3008))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 150400))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 3072))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 153600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 3136))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 160000) + (((int)threadIdx.x) * 100)) + (k_outer_outer * 2)) + 156800))))[0];
    __syncthreads();
    for (int j_outer_inner = 0; j_outer_inner < 25; ++j_outer_inner) {
      T_dense[((j_outer_inner * 2))] = (T_dense[((j_outer_inner * 2))] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 100) + (j_outer_inner * 4)))]));
      T_dense[(((j_outer_inner * 2) + 100))] = (T_dense[(((j_outer_inner * 2) + 100))] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 100) + (j_outer_inner * 4)))]));
      T_dense[(((j_outer_inner * 2) + 1))] = (T_dense[(((j_outer_inner * 2) + 1))] + (placeholder_d_shared[(0)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 2))]));
      T_dense[(((j_outer_inner * 2) + 101))] = (T_dense[(((j_outer_inner * 2) + 101))] + (placeholder_d_shared[(4)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 2))]));
      T_dense[(((j_outer_inner * 2) + 50))] = (T_dense[(((j_outer_inner * 2) + 50))] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 100) + (j_outer_inner * 4)))]));
      T_dense[(((j_outer_inner * 2) + 150))] = (T_dense[(((j_outer_inner * 2) + 150))] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 100) + (j_outer_inner * 4)))]));
      T_dense[(((j_outer_inner * 2) + 51))] = (T_dense[(((j_outer_inner * 2) + 51))] + (placeholder_d_shared[(2)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 2))]));
      T_dense[(((j_outer_inner * 2) + 151))] = (T_dense[(((j_outer_inner * 2) + 151))] + (placeholder_d_shared[(6)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 2))]));
      T_dense[((j_outer_inner * 2))] = (T_dense[((j_outer_inner * 2))] + (placeholder_d_shared[(1)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 100))] = (T_dense[(((j_outer_inner * 2) + 100))] + (placeholder_d_shared[(5)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 1))] = (T_dense[(((j_outer_inner * 2) + 1))] + (placeholder_d_shared[(1)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 3))]));
      T_dense[(((j_outer_inner * 2) + 101))] = (T_dense[(((j_outer_inner * 2) + 101))] + (placeholder_d_shared[(5)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 3))]));
      T_dense[(((j_outer_inner * 2) + 50))] = (T_dense[(((j_outer_inner * 2) + 50))] + (placeholder_d_shared[(3)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 150))] = (T_dense[(((j_outer_inner * 2) + 150))] + (placeholder_d_shared[(7)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 51))] = (T_dense[(((j_outer_inner * 2) + 51))] + (placeholder_d_shared[(3)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 3))]));
      T_dense[(((j_outer_inner * 2) + 151))] = (T_dense[(((j_outer_inner * 2) + 151))] + (placeholder_d_shared[(7)] * placeholder_shared[((((((int)threadIdx.x) * 100) + (j_outer_inner * 4)) + 3))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 50; ++ax1_inner) {
      T_relu[(((((ax0_inner * 25600) + (((int)blockIdx.x) * 1600)) + (((int)threadIdx.x) * 50)) + ax1_inner))] = max(T_dense[(((ax0_inner * 50) + ax1_inner))], 0.000000e+00f);
      T_relu[((((((ax0_inner * 25600) + (((int)blockIdx.x) * 1600)) + (((int)threadIdx.x) * 50)) + ax1_inner) + 51200))] = max(T_dense[((((ax0_inner * 50) + ax1_inner) + 100))], 0.000000e+00f);
    }
  }
}


