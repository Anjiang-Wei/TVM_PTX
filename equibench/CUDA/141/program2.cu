
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[56];
  __shared__ float PaddedInput_shared[3136];
  __shared__ float placeholder_shared[256];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
      Conv2dOutput[(((nn_outer_inner_init * 2) + yy_outer_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 2) + yy_outer_inner_init) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 2) + yy_outer_inner_init) + 16))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 2) + yy_outer_inner_init) + 24))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 2) + yy_outer_inner_init) + 32))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 2) + yy_outer_inner_init) + 40))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 2) + yy_outer_inner_init) + 48))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 14; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 14) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 3136) {
        if (((int)threadIdx.x) < 224) {
          PaddedInput_shared[(((((int)threadIdx.x) * 14) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[(((((((((int)threadIdx.x) * 14) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 2) * 256) + (rc_outer_outer * 4)) + (((((int)threadIdx.x) * 14) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)))];
        }
      }
    }
    if (((int)threadIdx.x) < 256) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)))];
    }
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          Conv2dOutput[(((nn_outer_inner * 2) + yy_outer_inner))] = (Conv2dOutput[(((nn_outer_inner * 2) + yy_outer_inner))] + (PaddedInput_shared[(((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 8))] = (Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 8))] + (PaddedInput_shared[((((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner) + 112))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 16))] = (Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 16))] + (PaddedInput_shared[((((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner) + 224))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 24))] = (Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 24))] + (PaddedInput_shared[((((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner) + 336))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 32))] = (Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 32))] + (PaddedInput_shared[((((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner) + 448))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 40))] = (Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 40))] + (PaddedInput_shared[((((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner) + 560))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 48))] = (Conv2dOutput[((((nn_outer_inner * 2) + yy_outer_inner) + 48))] + (PaddedInput_shared[((((((nn_outer_inner * 784) + (yy_outer_inner * 56)) + ((((int)threadIdx.x) >> 6) * 4)) + rc_inner) + 672))] * placeholder_shared[(((rc_inner * 64) + (((int)threadIdx.x) & 63)))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      T_relu[((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)))] = max((Conv2dOutput[(((ax0_inner * 2) + ax1_inner))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 3584))] = max((Conv2dOutput[((((ax0_inner * 2) + ax1_inner) + 8))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 7168))] = max((Conv2dOutput[((((ax0_inner * 2) + ax1_inner) + 16))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 10752))] = max((Conv2dOutput[((((ax0_inner * 2) + ax1_inner) + 24))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 14336))] = max((Conv2dOutput[((((ax0_inner * 2) + ax1_inner) + 32))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 17920))] = max((Conv2dOutput[((((ax0_inner * 2) + ax1_inner) + 40))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 21504))] = max((Conv2dOutput[((((ax0_inner * 2) + ax1_inner) + 48))] + placeholder2[(((((int)blockIdx.x) * 64) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
    }
  }
}


