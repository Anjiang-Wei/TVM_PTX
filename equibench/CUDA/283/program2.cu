
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[5];
  __shared__ float placeholder_d_shared[32];
  __shared__ float placeholder_shared[1280];
  for (int j_outer_inner_init = 0; j_outer_inner_init < 5; ++j_outer_inner_init) {
    T_dense[(j_outer_inner_init)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 40; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 4; ++ax0_ax1_fused_outer_outer) {
      placeholder_d_shared[(((ax0_ax1_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder[((((k_outer_outer * 32) + (ax0_ax1_fused_outer_outer * 8)) + ((int)threadIdx.x)))];
    }
    for (int ax0_ax1_fused_outer_outer1 = 0; ax0_ax1_fused_outer_outer1 < 160; ++ax0_ax1_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_outer_outer1 * 8) + ((int)threadIdx.x)))] = placeholder1[(((((((int)blockIdx.x) * 51200) + ((((ax0_ax1_fused_outer_outer1 * 8) + ((int)threadIdx.x)) >> 5) * 1280)) + (k_outer_outer * 32)) + (((ax0_ax1_fused_outer_outer1 * 8) + ((int)threadIdx.x)) & 31)))];
    }
    __syncthreads();
    for (int j_outer_inner = 0; j_outer_inner < 5; ++j_outer_inner) {
      for (int k_inner = 0; k_inner < 32; ++k_inner) {
        T_dense[(j_outer_inner)] = (T_dense[(j_outer_inner)] + (placeholder_d_shared[(k_inner)] * placeholder_shared[((((((int)threadIdx.x) * 160) + (j_outer_inner * 32)) + k_inner))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    T_add[((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 5)) + ax1_inner))] = (T_dense[(ax1_inner)] + placeholder2[((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 5)) + ax1_inner))]);
  }
}


