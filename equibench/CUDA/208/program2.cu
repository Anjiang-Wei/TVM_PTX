
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[28];
  __shared__ float PaddedInput_shared[1120];
  __shared__ float placeholder_shared[5120];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 12; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 5))] = placeholder[(((((((int)blockIdx.x) * 13440) + ((((int)threadIdx.x) >> 3) * 480)) + (rc_outer_outer * 40)) + ((((int)threadIdx.x) & 7) * 5)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 5) + 1))] = placeholder[(((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 1) / 40) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 1) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 5) + 2))] = placeholder[(((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 2) / 40) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 2) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 5) + 3))] = placeholder[(((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 3) / 40) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 3) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 5) + 4))] = placeholder[(((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 4) / 40) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 4) % 40)))];
    if (((int)threadIdx.x) < 96) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 5) + 640) / 560) * 560) + (((((int)threadIdx.x) >> 3) + 2) * 40)) + ((((int)threadIdx.x) & 7) * 5)))] = placeholder[((((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 640) / 560) * 6720)) + (((((int)threadIdx.x) >> 3) + 2) * 480)) + (rc_outer_outer * 40)) + ((((int)threadIdx.x) & 7) * 5)))];
    }
    if (((int)threadIdx.x) < 96) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 5) + 641) / 560) * 560) + (((((((int)threadIdx.x) * 5) + 1) / 40) + 2) * 40)) + (((((int)threadIdx.x) * 5) + 1) % 40)))] = placeholder[((((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 641) / 560) * 6720)) + (((((((int)threadIdx.x) * 5) + 1) / 40) + 2) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 1) % 40)))];
    }
    if (((int)threadIdx.x) < 96) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 5) + 642) / 560) * 560) + (((((((int)threadIdx.x) * 5) + 2) / 40) + 2) * 40)) + (((((int)threadIdx.x) * 5) + 2) % 40)))] = placeholder[((((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 642) / 560) * 6720)) + (((((((int)threadIdx.x) * 5) + 2) / 40) + 2) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 2) % 40)))];
    }
    if (((int)threadIdx.x) < 96) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 5) + 643) / 560) * 560) + (((((((int)threadIdx.x) * 5) + 3) / 40) + 2) * 40)) + (((((int)threadIdx.x) * 5) + 3) % 40)))] = placeholder[((((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 643) / 560) * 6720)) + (((((((int)threadIdx.x) * 5) + 3) / 40) + 2) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 3) % 40)))];
    }
    if (((int)threadIdx.x) < 96) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 5) + 644) / 560) * 560) + (((((((int)threadIdx.x) * 5) + 4) / 40) + 2) * 40)) + (((((int)threadIdx.x) * 5) + 4) % 40)))] = placeholder[((((((((int)blockIdx.x) * 13440) + ((((((int)threadIdx.x) * 5) + 644) / 560) * 6720)) + (((((((int)threadIdx.x) * 5) + 4) / 40) + 2) * 480)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 5) + 4) % 40)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 5120) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 640))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 1024))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 1152))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1152))];
    placeholder_shared[((((int)threadIdx.x) + 1280))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1280))];
    placeholder_shared[((((int)threadIdx.x) + 1408))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1408))];
    placeholder_shared[((((int)threadIdx.x) + 1536))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1536))];
    placeholder_shared[((((int)threadIdx.x) + 1664))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1664))];
    placeholder_shared[((((int)threadIdx.x) + 1792))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1792))];
    placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 1920))];
    placeholder_shared[((((int)threadIdx.x) + 2048))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2048))];
    placeholder_shared[((((int)threadIdx.x) + 2176))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2176))];
    placeholder_shared[((((int)threadIdx.x) + 2304))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2304))];
    placeholder_shared[((((int)threadIdx.x) + 2432))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2432))];
    placeholder_shared[((((int)threadIdx.x) + 2560))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2560))];
    placeholder_shared[((((int)threadIdx.x) + 2688))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2688))];
    placeholder_shared[((((int)threadIdx.x) + 2816))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2816))];
    placeholder_shared[((((int)threadIdx.x) + 2944))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 2944))];
    placeholder_shared[((((int)threadIdx.x) + 3072))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3072))];
    placeholder_shared[((((int)threadIdx.x) + 3200))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3200))];
    placeholder_shared[((((int)threadIdx.x) + 3328))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3328))];
    placeholder_shared[((((int)threadIdx.x) + 3456))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3456))];
    placeholder_shared[((((int)threadIdx.x) + 3584))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3584))];
    placeholder_shared[((((int)threadIdx.x) + 3712))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3712))];
    placeholder_shared[((((int)threadIdx.x) + 3840))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3840))];
    placeholder_shared[((((int)threadIdx.x) + 3968))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 3968))];
    placeholder_shared[((((int)threadIdx.x) + 4096))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4096))];
    placeholder_shared[((((int)threadIdx.x) + 4224))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4224))];
    placeholder_shared[((((int)threadIdx.x) + 4352))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4352))];
    placeholder_shared[((((int)threadIdx.x) + 4480))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4480))];
    placeholder_shared[((((int)threadIdx.x) + 4608))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4608))];
    placeholder_shared[((((int)threadIdx.x) + 4736))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4736))];
    placeholder_shared[((((int)threadIdx.x) + 4864))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4864))];
    placeholder_shared[((((int)threadIdx.x) + 4992))] = placeholder1[((((rc_outer_outer * 5120) + ((int)threadIdx.x)) + 4992))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 20; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((rc_outer_inner * 2))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 40))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 80))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 120))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 160))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 200))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 240))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 280))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 320))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 360))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 400))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 440))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 480))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 520))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 41))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 81))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 121))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 161))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 201))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 241))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 281))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 321))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 361))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 401))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 441))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 481))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 521))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 560))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 600))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 640))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 680))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 720))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 760))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 800))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 840))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 880))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 920))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 960))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1000))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1040))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1080))] * placeholder_shared[(((rc_outer_inner * 256) + ((int)threadIdx.x)))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 561))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 601))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 641))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 681))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 721))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 761))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 801))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 841))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 881))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 921))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 961))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1001))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1041))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((rc_outer_inner * 2) + 1081))] * placeholder_shared[((((rc_outer_inner * 256) + ((int)threadIdx.x)) + 128))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
      T_relu[(((((((int)blockIdx.x) * 3584) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((int)threadIdx.x)))] = max((Conv2dOutput[(((ax1_inner * 14) + ax2_inner))] + placeholder2[(((int)threadIdx.x))]), 0.000000e+00f);
    }
  }
}


