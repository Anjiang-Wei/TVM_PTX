
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[14];
  __shared__ float PaddedInput_shared[360];
  __shared__ float placeholder_shared[60];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(7)] = 0.000000e+00f;
  DepthwiseConv2d[(8)] = 0.000000e+00f;
  DepthwiseConv2d[(9)] = 0.000000e+00f;
  DepthwiseConv2d[(10)] = 0.000000e+00f;
  DepthwiseConv2d[(11)] = 0.000000e+00f;
  DepthwiseConv2d[(12)] = 0.000000e+00f;
  DepthwiseConv2d[(13)] = 0.000000e+00f;
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) * 2)) / 180) * 180) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 10)) % 9) * 20)) + ((((int)threadIdx.x) % 10) * 2)))] = (((((1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 10)) % 9)) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 10)) % 9) < 8)) && (1 <= (((((int)blockIdx.x) % 336) / 48) + dj_outer_outer))) && ((((((int)blockIdx.x) % 336) / 48) + dj_outer_outer) < 8)) ? placeholder[(((((((((((int)blockIdx.x) / 336) * 94080) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) * 2)) / 180) * 47040)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 10)) % 9) * 6720)) + (dj_outer_outer * 960)) + ((((int)blockIdx.x) % 336) * 20)) + ((((int)threadIdx.x) % 10) * 2)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) * 2)) + 1) / 180) * 180) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + 1) / 20)) % 9) * 20)) + (((((int)threadIdx.x) * 2) + 1) % 20)))] = (((((1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + 1) / 20)) % 9)) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + 1) / 20)) % 9) < 8)) && (1 <= (((((int)blockIdx.x) % 336) / 48) + dj_outer_outer))) && ((((((int)blockIdx.x) % 336) / 48) + dj_outer_outer) < 8)) ? placeholder[(((((((((((int)blockIdx.x) / 336) * 94080) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) * 2)) + 1) / 180) * 47040)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + 1) / 20)) % 9) * 6720)) + (dj_outer_outer * 960)) + ((((int)blockIdx.x) % 336) * 20)) + (((((int)threadIdx.x) * 2) + 1) % 20)) - 7680))] : 0.000000e+00f);
    }
    int3 _1;
      int3 _2;
        int3 _3;
          int3 _4;
            int3 _5;
              int3 _6 = (make_int3)(((((int)threadIdx.x) * 3))+(1*0), ((((int)threadIdx.x) * 3))+(1*1), ((((int)threadIdx.x) * 3))+(1*2));
              int3 _7 = make_int3(20, 20, 20);
              _5.x = (_6.x%_7.x);
              _5.y = (_6.y%_7.y);
              _5.z = (_6.z%_7.z);
            int3 _8;
              int3 _9 = (make_int3)(((((int)threadIdx.x) * 3))+(1*0), ((((int)threadIdx.x) * 3))+(1*1), ((((int)threadIdx.x) * 3))+(1*2));
              int3 _10 = make_int3(20, 20, 20);
              _8.x = (_9.x/_10.x);
              _8.y = (_9.y/_10.y);
              _8.z = (_9.z/_10.z);
            int3 _11;
            ushort3 _12;
              ushort3 _13;
                ushort3 _14;
                  int3 _15 = make_int3(20, 20, 20);
                  int3 _16 = make_int3(0, 0, 0);
                  _14.x = (_15.x>=_16.x);
                  _14.y = (_15.y>=_16.y);
                  _14.z = (_15.z>=_16.z);
                ushort3 _17;
                  int3 _18 = make_int3(0, 0, 0);
                  _17.x = (_5.x>=_18.x);
                  _17.y = (_5.y>=_18.y);
                  _17.z = (_5.z>=_18.z);
                _13.x = (_14.x&&_17.x);
                _13.y = (_14.y&&_17.y);
                _13.z = (_14.z&&_17.z);
              ushort3 _19;
                ushort3 _20;
                  int3 _21 = make_int3(20, 20, 20);
                  int3 _22 = make_int3(0, 0, 0);
                  _20.x = (_21.x<_22.x);
                  _20.y = (_21.y<_22.y);
                  _20.z = (_21.z<_22.z);
                ushort3 _23;
                  int3 _24 = make_int3(0, 0, 0);
                  _23.x = (_5.x<=_24.x);
                  _23.y = (_5.y<=_24.y);
                  _23.z = (_5.z<=_24.z);
                _19.x = (_20.x&&_23.x);
                _19.y = (_20.y&&_23.y);
                _19.z = (_20.z&&_23.z);
              _12.x = (_13.x||_19.x);
              _12.y = (_13.y||_19.y);
              _12.z = (_13.z||_19.z);
            int3 _25;
              int3 _26 = make_int3(1, 1, 1);
              _25.x = (_8.x-_26.x);
              _25.y = (_8.y-_26.y);
              _25.z = (_8.z-_26.z);
            _11.x = (bool(_12.x)?_8.x:_25.x);
            _11.y = (bool(_12.y)?_8.y:_25.y);
            _11.z = (bool(_12.z)?_8.z:_25.z);
            int3 _27 = make_int3(2880, 2880, 2880);
            _4.x = (_11.x*_27.x);
            _4.y = (_11.y*_27.y);
            _4.z = (_11.z*_27.z);
          int3 _28 = make_int3((dj_outer_outer * 960), (dj_outer_outer * 960), (dj_outer_outer * 960));
          _3.x = (_4.x+_28.x);
          _3.y = (_4.y+_28.y);
          _3.z = (_4.z+_28.z);
        int3 _29 = make_int3(((((int)blockIdx.x) % 48) * 20), ((((int)blockIdx.x) % 48) * 20), ((((int)blockIdx.x) % 48) * 20));
        _2.x = (_3.x+_29.x);
        _2.y = (_3.y+_29.y);
        _2.z = (_3.z+_29.z);
      int3 _30;
        int3 _31 = (make_int3)(((((int)threadIdx.x) * 3))+(1*0), ((((int)threadIdx.x) * 3))+(1*1), ((((int)threadIdx.x) * 3))+(1*2));
        int3 _32 = make_int3(20, 20, 20);
        _30.x = (_31.x%_32.x);
        _30.y = (_31.y%_32.y);
        _30.z = (_31.z%_32.z);
      int3 _33;
      ushort3 _34;
        ushort3 _35;
          ushort3 _36;
            int3 _37 = make_int3(20, 20, 20);
            int3 _38 = make_int3(0, 0, 0);
            _36.x = (_37.x>=_38.x);
            _36.y = (_37.y>=_38.y);
            _36.z = (_37.z>=_38.z);
          ushort3 _39;
            int3 _40 = make_int3(0, 0, 0);
            _39.x = (_30.x>=_40.x);
            _39.y = (_30.y>=_40.y);
            _39.z = (_30.z>=_40.z);
          _35.x = (_36.x&&_39.x);
          _35.y = (_36.y&&_39.y);
          _35.z = (_36.z&&_39.z);
        ushort3 _41;
          ushort3 _42;
            int3 _43 = make_int3(20, 20, 20);
            int3 _44 = make_int3(0, 0, 0);
            _42.x = (_43.x<_44.x);
            _42.y = (_43.y<_44.y);
            _42.z = (_43.z<_44.z);
          ushort3 _45;
            int3 _46 = make_int3(0, 0, 0);
            _45.x = (_30.x<=_46.x);
            _45.y = (_30.y<=_46.y);
            _45.z = (_30.z<=_46.z);
          _41.x = (_42.x&&_45.x);
          _41.y = (_42.y&&_45.y);
          _41.z = (_42.z&&_45.z);
        _34.x = (_35.x||_41.x);
        _34.y = (_35.y||_41.y);
        _34.z = (_35.z||_41.z);
      int3 _47;
        int3 _48 = make_int3(20, 20, 20);
        _47.x = (_30.x+_48.x);
        _47.y = (_30.y+_48.y);
        _47.z = (_30.z+_48.z);
      _33.x = (bool(_34.x)?_30.x:_47.x);
      _33.y = (bool(_34.y)?_30.y:_47.y);
      _33.z = (bool(_34.z)?_30.z:_47.z);
      _1.x = (_2.x+_33.x);
      _1.y = (_2.y+_33.y);
      _1.z = (_2.z+_33.z);
    ((float3*)(placeholder_shared + ((((int)threadIdx.x) * 3))))[0] = make_float3(placeholder1[_1.x],placeholder1[_1.y],placeholder1[_1.z]);
    __syncthreads();
    for (int di_inner = 0; di_inner < 3; ++di_inner) {
      DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared[(((di_inner * 20) + ((int)threadIdx.x)))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 20))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 40))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 60))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 80))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 100))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 120))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 180))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(8)] = (DepthwiseConv2d[(8)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 200))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(9)] = (DepthwiseConv2d[(9)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 220))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(10)] = (DepthwiseConv2d[(10)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 240))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(11)] = (DepthwiseConv2d[(11)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 260))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(12)] = (DepthwiseConv2d[(12)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 280))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
      DepthwiseConv2d[(13)] = (DepthwiseConv2d[(13)] + (PaddedInput_shared[((((di_inner * 20) + ((int)threadIdx.x)) + 300))] * placeholder_shared[(((di_inner * 20) + ((int)threadIdx.x)))]));
    }
  }
  for (int i0_inner = 0; i0_inner < 2; ++i0_inner) {
    for (int i1_inner = 0; i1_inner < 7; ++i1_inner) {
      compute[(((((((((int)blockIdx.x) / 336) * 94080) + (i0_inner * 47040)) + (i1_inner * 6720)) + ((((int)blockIdx.x) % 336) * 20)) + ((int)threadIdx.x)))] = max(min((DepthwiseConv2d[(((i0_inner * 7) + i1_inner))] + placeholder2[((((((int)blockIdx.x) % 48) * 20) + ((int)threadIdx.x)))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


