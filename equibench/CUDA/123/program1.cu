
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[280];
  __shared__ float PaddedInput_shared[7350];
  __shared__ float placeholder_shared[96];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 140))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 141))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 144))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 145))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 148))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 149))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 152))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 153))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 156))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 157))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 142))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 143))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 146))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 147))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 150))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 151))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 154))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 155))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 158))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 159))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 96; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[((((((((int)threadIdx.x) + 280) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 560))] = placeholder[((((((((int)threadIdx.x) + 560) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 840))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 80640))];
    PaddedInput_shared[((((int)threadIdx.x) + 1120))] = placeholder[((((((((int)threadIdx.x) + 1120) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1400))] = placeholder[((((((((int)threadIdx.x) + 1400) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1680))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 161280))];
    PaddedInput_shared[((((int)threadIdx.x) + 1960))] = placeholder[((((((((int)threadIdx.x) + 1960) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2240))] = placeholder[((((((((int)threadIdx.x) + 2240) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2520))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 241920))];
    PaddedInput_shared[((((int)threadIdx.x) + 2800))] = placeholder[((((((((int)threadIdx.x) + 2800) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3080))] = placeholder[((((((((int)threadIdx.x) + 3080) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3360))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 322560))];
    PaddedInput_shared[((((int)threadIdx.x) + 3640))] = placeholder[((((((((int)threadIdx.x) + 3640) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3920))] = placeholder[((((((((int)threadIdx.x) + 3920) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4200))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 403200))];
    PaddedInput_shared[((((int)threadIdx.x) + 4480))] = placeholder[((((((((int)threadIdx.x) + 4480) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4760))] = placeholder[((((((((int)threadIdx.x) + 4760) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5040))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 483840))];
    PaddedInput_shared[((((int)threadIdx.x) + 5320))] = placeholder[((((((((int)threadIdx.x) + 5320) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5600))] = placeholder[((((((((int)threadIdx.x) + 5600) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5880))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 564480))];
    PaddedInput_shared[((((int)threadIdx.x) + 6160))] = placeholder[((((((((int)threadIdx.x) + 6160) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6440))] = placeholder[((((((((int)threadIdx.x) + 6440) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6720))] = placeholder[((((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 645120))];
    PaddedInput_shared[((((int)threadIdx.x) + 7000))] = placeholder[((((((((int)threadIdx.x) + 7000) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    if (((int)threadIdx.x) < 70) {
      PaddedInput_shared[((((int)threadIdx.x) + 7280))] = placeholder[((((((((int)threadIdx.x) + 7280) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[((((int)threadIdx.x) * 9))] = placeholder1[(((((rc_outer_outer * 192) + (((((int)threadIdx.x) * 9) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) * 9) & 31)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 1))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 1) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 1) & 31)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 2))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 2) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 2) & 31)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 3))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 3) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 3) & 31)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 4))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 4) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 4) & 31)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 5))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 5) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 5) & 31)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 6))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 6) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 6) & 31)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 7))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 7) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 7) & 31)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 9) + 8))] = placeholder1[(((((rc_outer_outer * 192) + ((((((int)threadIdx.x) * 9) + 8) >> 5) * 64)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) * 9) + 8) & 31)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
        Conv2dOutput[((yy_outer_inner * 20))] = (Conv2dOutput[((yy_outer_inner * 20))] + (PaddedInput_shared[((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 1))] = (Conv2dOutput[(((yy_outer_inner * 20) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 4))] = (Conv2dOutput[(((yy_outer_inner * 20) + 4))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 105))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 5))] = (Conv2dOutput[(((yy_outer_inner * 20) + 5))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 105))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 8))] = (Conv2dOutput[(((yy_outer_inner * 20) + 8))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 210))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 9))] = (Conv2dOutput[(((yy_outer_inner * 20) + 9))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 210))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 12))] = (Conv2dOutput[(((yy_outer_inner * 20) + 12))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 315))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 13))] = (Conv2dOutput[(((yy_outer_inner * 20) + 13))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 315))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 16))] = (Conv2dOutput[(((yy_outer_inner * 20) + 16))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 420))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 17))] = (Conv2dOutput[(((yy_outer_inner * 20) + 17))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 140))] = (Conv2dOutput[(((yy_outer_inner * 20) + 140))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3675))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 141))] = (Conv2dOutput[(((yy_outer_inner * 20) + 141))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3675))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 144))] = (Conv2dOutput[(((yy_outer_inner * 20) + 144))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3780))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 145))] = (Conv2dOutput[(((yy_outer_inner * 20) + 145))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3780))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 148))] = (Conv2dOutput[(((yy_outer_inner * 20) + 148))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3885))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 149))] = (Conv2dOutput[(((yy_outer_inner * 20) + 149))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3885))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 152))] = (Conv2dOutput[(((yy_outer_inner * 20) + 152))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3990))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 153))] = (Conv2dOutput[(((yy_outer_inner * 20) + 153))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3990))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 156))] = (Conv2dOutput[(((yy_outer_inner * 20) + 156))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 4095))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 157))] = (Conv2dOutput[(((yy_outer_inner * 20) + 157))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 4095))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 2))] = (Conv2dOutput[(((yy_outer_inner * 20) + 2))] + (PaddedInput_shared[((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 3))] = (Conv2dOutput[(((yy_outer_inner * 20) + 3))] + (PaddedInput_shared[((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 6))] = (Conv2dOutput[(((yy_outer_inner * 20) + 6))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 105))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 7))] = (Conv2dOutput[(((yy_outer_inner * 20) + 7))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 105))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 10))] = (Conv2dOutput[(((yy_outer_inner * 20) + 10))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 210))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 11))] = (Conv2dOutput[(((yy_outer_inner * 20) + 11))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 210))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 14))] = (Conv2dOutput[(((yy_outer_inner * 20) + 14))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 315))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 15))] = (Conv2dOutput[(((yy_outer_inner * 20) + 15))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 315))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 18))] = (Conv2dOutput[(((yy_outer_inner * 20) + 18))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 19))] = (Conv2dOutput[(((yy_outer_inner * 20) + 19))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 142))] = (Conv2dOutput[(((yy_outer_inner * 20) + 142))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3675))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 143))] = (Conv2dOutput[(((yy_outer_inner * 20) + 143))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3675))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 146))] = (Conv2dOutput[(((yy_outer_inner * 20) + 146))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3780))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 147))] = (Conv2dOutput[(((yy_outer_inner * 20) + 147))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3780))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 150))] = (Conv2dOutput[(((yy_outer_inner * 20) + 150))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3885))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 151))] = (Conv2dOutput[(((yy_outer_inner * 20) + 151))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3885))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 154))] = (Conv2dOutput[(((yy_outer_inner * 20) + 154))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3990))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 155))] = (Conv2dOutput[(((yy_outer_inner * 20) + 155))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 3990))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 158))] = (Conv2dOutput[(((yy_outer_inner * 20) + 158))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 4095))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 20) + 159))] = (Conv2dOutput[(((yy_outer_inner * 20) + 159))] + (PaddedInput_shared[(((((yy_outer_inner * 525) + ((((int)threadIdx.x) >> 3) * 3)) + rc_outer_inner) + 4095))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 35; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[(((((((ax0_inner * 78400) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 140) + (ax1_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


