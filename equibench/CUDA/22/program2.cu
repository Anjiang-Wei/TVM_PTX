
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[32];
  __shared__ float PaddedInput_shared[3328];
  __shared__ float placeholder_shared[144];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 2; ++i_outer_inner_init) {
    for (int j_outer_inner_init = 0; j_outer_inner_init < 2; ++j_outer_inner_init) {
      DepthwiseConv2d[(((i_outer_inner_init * 2) + j_outer_inner_init))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 4))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 8))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 12))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 16))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 20))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 24))] = 0.000000e+00f;
      DepthwiseConv2d[((((i_outer_inner_init * 2) + j_outer_inner_init) + 28))] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 72) + ((int)threadIdx.x)) < 208) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1152) + (((int)threadIdx.x) * 16)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 3328) {
          PaddedInput_shared[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + (((int)threadIdx.x) >> 3)) % 26) * 128) + ((((int)threadIdx.x) & 7) * 16)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = (((((1 <= ((((((int)blockIdx.x) % 200) / 40) * 24) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + (((int)threadIdx.x) >> 3)) % 26))) && (((((((int)blockIdx.x) % 200) / 40) * 24) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + (((int)threadIdx.x) >> 3)) % 26)) < 121)) && (1 <= ((((((int)blockIdx.x) % 40) >> 1) * 6) + (((int)threadIdx.x) & 7)))) && (((((((int)blockIdx.x) % 40) >> 1) * 6) + (((int)threadIdx.x) & 7)) < 121)) ? placeholder[(((((((((((int)blockIdx.x) / 40) * 92160) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + (((int)threadIdx.x) >> 3)) % 26) * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + ((((int)threadIdx.x) & 7) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) - 3872))] : 0.000000e+00f);
        }
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
    if (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 144) {
      if (((int)threadIdx.x) < 36) {
        placeholder_shared[(((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[(((((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) >> 4) * 32) + ((((int)blockIdx.x) & 1) * 16)) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) & 15)))];
      }
    }
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
      for (int j_outer_inner = 0; j_outer_inner < 2; ++j_outer_inner) {
        for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
          DepthwiseConv2d[(((i_outer_inner * 2) + j_outer_inner))] = (DepthwiseConv2d[(((i_outer_inner * 2) + j_outer_inner))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)))] * placeholder_shared[((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 4))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 4))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 2))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 2))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 8))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 8))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 4))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 4))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 12))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 12))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 6))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 6))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 16))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 16))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 8))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 8))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 20))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 20))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 10))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 10))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 24))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 24))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 12))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 12))]));
          DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 28))] = (DepthwiseConv2d[((((i_outer_inner * 2) + j_outer_inner) + 28))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 6) * 256) + (i_outer_inner * 128)) + (di_outer_inner * 128)) + (((((int)threadIdx.x) % 6) >> 1) * 32)) + (j_outer_inner * 16)) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 14))] * placeholder_shared[(((((di_outer_inner * 48) + (dj_inner * 16)) + (((int)threadIdx.x) & 1)) + 14))]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)))] = max(min((DepthwiseConv2d[(((i1_inner * 2) + i2_inner))] + placeholder2[((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 2))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 4))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 2))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 4))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 8))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 4))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 6))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 12))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 6))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 8))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 8))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 10))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 20))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 10))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 12))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 24))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 12))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((((int)blockIdx.x) / 40) * 92160) + ((((int)threadIdx.x) / 6) * 7680)) + (i1_inner * 3840)) + (((((int)blockIdx.x) % 40) >> 1) * 192)) + (((((int)threadIdx.x) % 6) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((int)threadIdx.x) & 1)) + 14))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i2_inner) + 28))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + (((int)threadIdx.x) & 1)) + 14))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


