
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float conv2d_transpose_nhwc[48];
  __shared__ float PadInput_shared[896];
  __shared__ float placeholder_shared[512];
  for (int w_inner_init = 0; w_inner_init < 24; ++w_inner_init) {
    conv2d_transpose_nhwc[(w_inner_init)] = 0.000000e+00f;
    conv2d_transpose_nhwc[((w_inner_init + 24))] = 0.000000e+00f;
  }
  for (int rh_outer_outer = 0; rh_outer_outer < 2; ++rh_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
      __syncthreads();
      PadInput_shared[(((int)threadIdx.x))] = (((((1 <= ((((((int)blockIdx.x) >> 4) * 3) + (((int)threadIdx.x) / 56)) + rh_outer_outer)) && (((((((int)blockIdx.x) >> 4) * 3) + (((int)threadIdx.x) / 56)) + rh_outer_outer) < 13)) && (4 <= (((int)threadIdx.x) % 56))) && ((((int)threadIdx.x) % 56) < 52)) ? placeholder[(((((((((((int)blockIdx.x) >> 4) * 18432) + ((((int)threadIdx.x) / 56) * 6144)) + (rh_outer_outer * 6144)) + (((((int)threadIdx.x) % 56) >> 2) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 6656))] : 0.000000e+00f);
      PadInput_shared[((((int)threadIdx.x) + 192))] = (((((1 <= ((((((int)blockIdx.x) >> 4) * 3) + ((((((int)threadIdx.x) >> 2) + 48) % 56) / 14)) + rh_outer_outer)) && (((((((int)blockIdx.x) >> 4) * 3) + ((((((int)threadIdx.x) >> 2) + 48) % 56) / 14)) + rh_outer_outer) < 13)) && (1 <= (((((int)threadIdx.x) >> 2) + 6) % 14))) && ((((((int)threadIdx.x) >> 2) + 6) % 14) < 13)) ? placeholder[(((((((((((((int)threadIdx.x) + 192) / 224) * 73728) + ((((int)blockIdx.x) >> 4) * 18432)) + (((((((int)threadIdx.x) >> 2) + 48) % 56) / 14) * 6144)) + (rh_outer_outer * 6144)) + ((((((int)threadIdx.x) >> 2) + 6) % 14) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 6656))] : 0.000000e+00f);
      PadInput_shared[((((int)threadIdx.x) + 384))] = (((((1 <= ((((((int)blockIdx.x) >> 4) * 3) + ((((((int)threadIdx.x) >> 2) + 40) % 56) / 14)) + rh_outer_outer)) && (((((((int)blockIdx.x) >> 4) * 3) + ((((((int)threadIdx.x) >> 2) + 40) % 56) / 14)) + rh_outer_outer) < 13)) && (1 <= (((((int)threadIdx.x) >> 2) + 12) % 14))) && ((((((int)threadIdx.x) >> 2) + 12) % 14) < 13)) ? placeholder[(((((((((((((int)threadIdx.x) + 384) / 224) * 73728) + ((((int)blockIdx.x) >> 4) * 18432)) + (((((((int)threadIdx.x) >> 2) + 40) % 56) / 14) * 6144)) + (rh_outer_outer * 6144)) + ((((((int)threadIdx.x) >> 2) + 12) % 14) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 6656))] : 0.000000e+00f);
      PadInput_shared[((((int)threadIdx.x) + 576))] = (((((1 <= ((((((int)blockIdx.x) >> 4) * 3) + ((((((int)threadIdx.x) >> 2) + 32) % 56) / 14)) + rh_outer_outer)) && (((((((int)blockIdx.x) >> 4) * 3) + ((((((int)threadIdx.x) >> 2) + 32) % 56) / 14)) + rh_outer_outer) < 13)) && (1 <= (((((int)threadIdx.x) >> 2) + 4) % 14))) && ((((((int)threadIdx.x) >> 2) + 4) % 14) < 13)) ? placeholder[(((((((((((((int)threadIdx.x) + 576) / 224) * 73728) + ((((int)blockIdx.x) >> 4) * 18432)) + (((((((int)threadIdx.x) >> 2) + 32) % 56) / 14) * 6144)) + (rh_outer_outer * 6144)) + ((((((int)threadIdx.x) >> 2) + 4) % 14) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 6656))] : 0.000000e+00f);
      if (((int)threadIdx.x) < 128) {
        PadInput_shared[((((int)threadIdx.x) + 768))] = ((((((((((int)blockIdx.x) >> 4) * 3) + (((((int)threadIdx.x) >> 2) + 24) / 14)) + rh_outer_outer) < 13) && (1 <= (((((int)threadIdx.x) >> 2) + 10) % 14))) && ((((((int)threadIdx.x) >> 2) + 10) % 14) < 13)) ? placeholder[(((((((((((((int)threadIdx.x) + 768) / 224) * 73728) + ((((int)blockIdx.x) >> 4) * 18432)) + ((((((int)threadIdx.x) >> 2) + 24) / 14) * 6144)) + (rh_outer_outer * 6144)) + ((((((int)threadIdx.x) >> 2) + 10) % 14) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 6656))] : 0.000000e+00f);
      }
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((((((int)threadIdx.x) >> 6) * 131072) + (rc_outer_outer * 1024)) + (((((int)threadIdx.x) & 63) >> 4) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 1048576) - (rh_outer_outer * 1048576)))];
      placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((((((((int)threadIdx.x) >> 6) * 131072) + (rc_outer_outer * 1024)) + (((((int)threadIdx.x) & 63) >> 4) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 1441792) - (rh_outer_outer * 1048576)))];
      if (((int)threadIdx.x) < 128) {
        placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[(((((((((((int)threadIdx.x) >> 6) * 131072) + (rc_outer_outer * 1024)) + (((((int)threadIdx.x) & 63) >> 4) * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15)) + 1835008) - (rh_outer_outer * 1048576)))];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
        for (int rh_inner = 0; rh_inner < 2; ++rh_inner) {
          for (int rw_inner = 0; rw_inner < 4; ++rw_inner) {
            for (int w_inner = 0; w_inner < 24; ++w_inner) {
              conv2d_transpose_nhwc[(w_inner)] = (conv2d_transpose_nhwc[(w_inner)] + ((((((((((int)threadIdx.x) % 48) >> 3) + rh_inner) % 2) == 0) && (((w_inner + rw_inner) % 2) == 0)) ? PadInput_shared[((((((((int)threadIdx.x) / 48) * 224) + (((((((int)threadIdx.x) % 48) >> 3) + rh_inner) >> 1) * 56)) + (((w_inner + rw_inner) >> 1) * 4)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[((((((rc_outer_inner * 16) + (((int)threadIdx.x) & 7)) + 448) - (rw_inner * 64)) - (rh_inner * 256)))]));
              conv2d_transpose_nhwc[((w_inner + 24))] = (conv2d_transpose_nhwc[((w_inner + 24))] + ((((((((((int)threadIdx.x) % 48) >> 3) + rh_inner) % 2) == 0) && (((w_inner + rw_inner) % 2) == 0)) ? PadInput_shared[((((((((int)threadIdx.x) / 48) * 224) + (((((((int)threadIdx.x) % 48) >> 3) + rh_inner) >> 1) * 56)) + (((w_inner + rw_inner) >> 1) * 4)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[((((((rc_outer_inner * 16) + (((int)threadIdx.x) & 7)) + 456) - (rw_inner * 64)) - (rh_inner * 256)))]));
            }
          }
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 24; ++ax2_inner) {
    T_relu[((((((((((int)threadIdx.x) / 48) * 147456) + ((((int)blockIdx.x) >> 4) * 36864)) + (((((int)threadIdx.x) % 48) >> 3) * 6144)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 7)))] = max(((conv2d_transpose_nhwc[(ax2_inner)] * placeholder2[((((((int)blockIdx.x) & 15) * 16) + (((int)threadIdx.x) & 7)))]) + placeholder3[((((((int)blockIdx.x) & 15) * 16) + (((int)threadIdx.x) & 7)))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 147456) + ((((int)blockIdx.x) >> 4) * 36864)) + (((((int)threadIdx.x) % 48) >> 3) * 6144)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 7)) + 8))] = max(((conv2d_transpose_nhwc[((ax2_inner + 24))] * placeholder2[(((((((int)blockIdx.x) & 15) * 16) + (((int)threadIdx.x) & 7)) + 8))]) + placeholder3[(((((((int)blockIdx.x) & 15) * 16) + (((int)threadIdx.x) & 7)) + 8))]), 0.000000e+00f);
  }
}


