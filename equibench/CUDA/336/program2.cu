
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
  float T_dense[1024];
  __shared__ float placeholder_d_shared[8];
  __shared__ float placeholder_shared[768];
  for (int j_outer_inner_init = 0; j_outer_inner_init < 64; ++j_outer_inner_init) {
    T_dense[((j_outer_inner_init * 2))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 128))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 129))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 256))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 257))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 384))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 385))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 512))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 513))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 640))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 641))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 768))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 769))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 896))] = 0.000000e+00f;
    T_dense[(((j_outer_inner_init * 2) + 897))] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 100; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[((((int)threadIdx.x) * 8))] = placeholder[(((((int)threadIdx.x) * 800) + k_outer_outer))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 100))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 200))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 300))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 400))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 500))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 600))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder[((((((int)threadIdx.x) * 800) + k_outer_outer) + 700))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer))];
    placeholder_shared[((((int)threadIdx.x) + 6))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 600))];
    placeholder_shared[((((int)threadIdx.x) + 12))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 1200))];
    placeholder_shared[((((int)threadIdx.x) + 18))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 1800))];
    placeholder_shared[((((int)threadIdx.x) + 24))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 2400))];
    placeholder_shared[((((int)threadIdx.x) + 30))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 3000))];
    placeholder_shared[((((int)threadIdx.x) + 36))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 3600))];
    placeholder_shared[((((int)threadIdx.x) + 42))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 4200))];
    placeholder_shared[((((int)threadIdx.x) + 48))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 4800))];
    placeholder_shared[((((int)threadIdx.x) + 54))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 5400))];
    placeholder_shared[((((int)threadIdx.x) + 60))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 6000))];
    placeholder_shared[((((int)threadIdx.x) + 66))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 6600))];
    placeholder_shared[((((int)threadIdx.x) + 72))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 7200))];
    placeholder_shared[((((int)threadIdx.x) + 78))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 7800))];
    placeholder_shared[((((int)threadIdx.x) + 84))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 8400))];
    placeholder_shared[((((int)threadIdx.x) + 90))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 9000))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 9600))];
    placeholder_shared[((((int)threadIdx.x) + 102))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 10200))];
    placeholder_shared[((((int)threadIdx.x) + 108))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 10800))];
    placeholder_shared[((((int)threadIdx.x) + 114))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 11400))];
    placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 12000))];
    placeholder_shared[((((int)threadIdx.x) + 126))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 12600))];
    placeholder_shared[((((int)threadIdx.x) + 132))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 13200))];
    placeholder_shared[((((int)threadIdx.x) + 138))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 13800))];
    placeholder_shared[((((int)threadIdx.x) + 144))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 14400))];
    placeholder_shared[((((int)threadIdx.x) + 150))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 15000))];
    placeholder_shared[((((int)threadIdx.x) + 156))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 15600))];
    placeholder_shared[((((int)threadIdx.x) + 162))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 16200))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 16800))];
    placeholder_shared[((((int)threadIdx.x) + 174))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 17400))];
    placeholder_shared[((((int)threadIdx.x) + 180))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 18000))];
    placeholder_shared[((((int)threadIdx.x) + 186))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 18600))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 19200))];
    placeholder_shared[((((int)threadIdx.x) + 198))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 19800))];
    placeholder_shared[((((int)threadIdx.x) + 204))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 20400))];
    placeholder_shared[((((int)threadIdx.x) + 210))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 21000))];
    placeholder_shared[((((int)threadIdx.x) + 216))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 21600))];
    placeholder_shared[((((int)threadIdx.x) + 222))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 22200))];
    placeholder_shared[((((int)threadIdx.x) + 228))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 22800))];
    placeholder_shared[((((int)threadIdx.x) + 234))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 23400))];
    placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 24000))];
    placeholder_shared[((((int)threadIdx.x) + 246))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 24600))];
    placeholder_shared[((((int)threadIdx.x) + 252))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 25200))];
    placeholder_shared[((((int)threadIdx.x) + 258))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 25800))];
    placeholder_shared[((((int)threadIdx.x) + 264))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 26400))];
    placeholder_shared[((((int)threadIdx.x) + 270))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 27000))];
    placeholder_shared[((((int)threadIdx.x) + 276))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 27600))];
    placeholder_shared[((((int)threadIdx.x) + 282))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 28200))];
    placeholder_shared[((((int)threadIdx.x) + 288))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 28800))];
    placeholder_shared[((((int)threadIdx.x) + 294))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 29400))];
    placeholder_shared[((((int)threadIdx.x) + 300))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 30000))];
    placeholder_shared[((((int)threadIdx.x) + 306))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 30600))];
    placeholder_shared[((((int)threadIdx.x) + 312))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 31200))];
    placeholder_shared[((((int)threadIdx.x) + 318))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 31800))];
    placeholder_shared[((((int)threadIdx.x) + 324))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 32400))];
    placeholder_shared[((((int)threadIdx.x) + 330))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 33000))];
    placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 33600))];
    placeholder_shared[((((int)threadIdx.x) + 342))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 34200))];
    placeholder_shared[((((int)threadIdx.x) + 348))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 34800))];
    placeholder_shared[((((int)threadIdx.x) + 354))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 35400))];
    placeholder_shared[((((int)threadIdx.x) + 360))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 36000))];
    placeholder_shared[((((int)threadIdx.x) + 366))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 36600))];
    placeholder_shared[((((int)threadIdx.x) + 372))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 37200))];
    placeholder_shared[((((int)threadIdx.x) + 378))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 37800))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 38400))];
    placeholder_shared[((((int)threadIdx.x) + 390))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 39000))];
    placeholder_shared[((((int)threadIdx.x) + 396))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 39600))];
    placeholder_shared[((((int)threadIdx.x) + 402))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 40200))];
    placeholder_shared[((((int)threadIdx.x) + 408))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 40800))];
    placeholder_shared[((((int)threadIdx.x) + 414))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 41400))];
    placeholder_shared[((((int)threadIdx.x) + 420))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 42000))];
    placeholder_shared[((((int)threadIdx.x) + 426))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 42600))];
    placeholder_shared[((((int)threadIdx.x) + 432))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 43200))];
    placeholder_shared[((((int)threadIdx.x) + 438))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 43800))];
    placeholder_shared[((((int)threadIdx.x) + 444))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 44400))];
    placeholder_shared[((((int)threadIdx.x) + 450))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 45000))];
    placeholder_shared[((((int)threadIdx.x) + 456))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 45600))];
    placeholder_shared[((((int)threadIdx.x) + 462))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 46200))];
    placeholder_shared[((((int)threadIdx.x) + 468))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 46800))];
    placeholder_shared[((((int)threadIdx.x) + 474))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 47400))];
    placeholder_shared[((((int)threadIdx.x) + 480))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 48000))];
    placeholder_shared[((((int)threadIdx.x) + 486))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 48600))];
    placeholder_shared[((((int)threadIdx.x) + 492))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 49200))];
    placeholder_shared[((((int)threadIdx.x) + 498))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 49800))];
    placeholder_shared[((((int)threadIdx.x) + 504))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 50400))];
    placeholder_shared[((((int)threadIdx.x) + 510))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 51000))];
    placeholder_shared[((((int)threadIdx.x) + 516))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 51600))];
    placeholder_shared[((((int)threadIdx.x) + 522))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 52200))];
    placeholder_shared[((((int)threadIdx.x) + 528))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 52800))];
    placeholder_shared[((((int)threadIdx.x) + 534))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 53400))];
    placeholder_shared[((((int)threadIdx.x) + 540))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 54000))];
    placeholder_shared[((((int)threadIdx.x) + 546))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 54600))];
    placeholder_shared[((((int)threadIdx.x) + 552))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 55200))];
    placeholder_shared[((((int)threadIdx.x) + 558))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 55800))];
    placeholder_shared[((((int)threadIdx.x) + 564))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 56400))];
    placeholder_shared[((((int)threadIdx.x) + 570))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 57000))];
    placeholder_shared[((((int)threadIdx.x) + 576))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 57600))];
    placeholder_shared[((((int)threadIdx.x) + 582))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 58200))];
    placeholder_shared[((((int)threadIdx.x) + 588))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 58800))];
    placeholder_shared[((((int)threadIdx.x) + 594))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 59400))];
    placeholder_shared[((((int)threadIdx.x) + 600))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 60000))];
    placeholder_shared[((((int)threadIdx.x) + 606))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 60600))];
    placeholder_shared[((((int)threadIdx.x) + 612))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 61200))];
    placeholder_shared[((((int)threadIdx.x) + 618))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 61800))];
    placeholder_shared[((((int)threadIdx.x) + 624))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 62400))];
    placeholder_shared[((((int)threadIdx.x) + 630))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 63000))];
    placeholder_shared[((((int)threadIdx.x) + 636))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 63600))];
    placeholder_shared[((((int)threadIdx.x) + 642))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 64200))];
    placeholder_shared[((((int)threadIdx.x) + 648))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 64800))];
    placeholder_shared[((((int)threadIdx.x) + 654))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 65400))];
    placeholder_shared[((((int)threadIdx.x) + 660))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 66000))];
    placeholder_shared[((((int)threadIdx.x) + 666))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 66600))];
    placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 67200))];
    placeholder_shared[((((int)threadIdx.x) + 678))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 67800))];
    placeholder_shared[((((int)threadIdx.x) + 684))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 68400))];
    placeholder_shared[((((int)threadIdx.x) + 690))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 69000))];
    placeholder_shared[((((int)threadIdx.x) + 696))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 69600))];
    placeholder_shared[((((int)threadIdx.x) + 702))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 70200))];
    placeholder_shared[((((int)threadIdx.x) + 708))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 70800))];
    placeholder_shared[((((int)threadIdx.x) + 714))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 71400))];
    placeholder_shared[((((int)threadIdx.x) + 720))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 72000))];
    placeholder_shared[((((int)threadIdx.x) + 726))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 72600))];
    placeholder_shared[((((int)threadIdx.x) + 732))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 73200))];
    placeholder_shared[((((int)threadIdx.x) + 738))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 73800))];
    placeholder_shared[((((int)threadIdx.x) + 744))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 74400))];
    placeholder_shared[((((int)threadIdx.x) + 750))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 75000))];
    placeholder_shared[((((int)threadIdx.x) + 756))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 75600))];
    placeholder_shared[((((int)threadIdx.x) + 762))] = placeholder1[(((((((int)blockIdx.x) * 76800) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 76200))];
    __syncthreads();
    for (int j_outer_inner = 0; j_outer_inner < 64; ++j_outer_inner) {
      T_dense[((j_outer_inner * 2))] = (T_dense[((j_outer_inner * 2))] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 1))] = (T_dense[(((j_outer_inner * 2) + 1))] + (placeholder_d_shared[(0)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 128))] = (T_dense[(((j_outer_inner * 2) + 128))] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 129))] = (T_dense[(((j_outer_inner * 2) + 129))] + (placeholder_d_shared[(1)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 256))] = (T_dense[(((j_outer_inner * 2) + 256))] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 257))] = (T_dense[(((j_outer_inner * 2) + 257))] + (placeholder_d_shared[(2)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 384))] = (T_dense[(((j_outer_inner * 2) + 384))] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 385))] = (T_dense[(((j_outer_inner * 2) + 385))] + (placeholder_d_shared[(3)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 512))] = (T_dense[(((j_outer_inner * 2) + 512))] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 513))] = (T_dense[(((j_outer_inner * 2) + 513))] + (placeholder_d_shared[(4)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 640))] = (T_dense[(((j_outer_inner * 2) + 640))] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 641))] = (T_dense[(((j_outer_inner * 2) + 641))] + (placeholder_d_shared[(5)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 768))] = (T_dense[(((j_outer_inner * 2) + 768))] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 769))] = (T_dense[(((j_outer_inner * 2) + 769))] + (placeholder_d_shared[(6)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
      T_dense[(((j_outer_inner * 2) + 896))] = (T_dense[(((j_outer_inner * 2) + 896))] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 128) + (j_outer_inner * 2)))]));
      T_dense[(((j_outer_inner * 2) + 897))] = (T_dense[(((j_outer_inner * 2) + 897))] + (placeholder_d_shared[(7)] * placeholder_shared[((((((int)threadIdx.x) * 128) + (j_outer_inner * 2)) + 1))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 8; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 128; ++ax1_inner) {
      T_relu[(((((ax0_inner * 36864) + (((int)blockIdx.x) * 768)) + (((int)threadIdx.x) * 128)) + ax1_inner))] = max(T_dense[(((ax0_inner * 128) + ax1_inner))], 0.000000e+00f);
    }
  }
}


