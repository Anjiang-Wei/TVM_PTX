
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ Conv2dOutput, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] = ((Conv2dOutput[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 960))]) * (max(min(((Conv2dOutput[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 960))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[56];
  __shared__ float PaddedInput_shared[280];
  __shared__ float placeholder_shared[3840];
  Conv2dOutput_local[(0)] = 0.000000e+00f;
  Conv2dOutput_local[(28)] = 0.000000e+00f;
  Conv2dOutput_local[(4)] = 0.000000e+00f;
  Conv2dOutput_local[(32)] = 0.000000e+00f;
  Conv2dOutput_local[(8)] = 0.000000e+00f;
  Conv2dOutput_local[(36)] = 0.000000e+00f;
  Conv2dOutput_local[(12)] = 0.000000e+00f;
  Conv2dOutput_local[(40)] = 0.000000e+00f;
  Conv2dOutput_local[(16)] = 0.000000e+00f;
  Conv2dOutput_local[(44)] = 0.000000e+00f;
  Conv2dOutput_local[(20)] = 0.000000e+00f;
  Conv2dOutput_local[(48)] = 0.000000e+00f;
  Conv2dOutput_local[(24)] = 0.000000e+00f;
  Conv2dOutput_local[(52)] = 0.000000e+00f;
  Conv2dOutput_local[(1)] = 0.000000e+00f;
  Conv2dOutput_local[(29)] = 0.000000e+00f;
  Conv2dOutput_local[(5)] = 0.000000e+00f;
  Conv2dOutput_local[(33)] = 0.000000e+00f;
  Conv2dOutput_local[(9)] = 0.000000e+00f;
  Conv2dOutput_local[(37)] = 0.000000e+00f;
  Conv2dOutput_local[(13)] = 0.000000e+00f;
  Conv2dOutput_local[(41)] = 0.000000e+00f;
  Conv2dOutput_local[(17)] = 0.000000e+00f;
  Conv2dOutput_local[(45)] = 0.000000e+00f;
  Conv2dOutput_local[(21)] = 0.000000e+00f;
  Conv2dOutput_local[(49)] = 0.000000e+00f;
  Conv2dOutput_local[(25)] = 0.000000e+00f;
  Conv2dOutput_local[(53)] = 0.000000e+00f;
  Conv2dOutput_local[(2)] = 0.000000e+00f;
  Conv2dOutput_local[(30)] = 0.000000e+00f;
  Conv2dOutput_local[(6)] = 0.000000e+00f;
  Conv2dOutput_local[(34)] = 0.000000e+00f;
  Conv2dOutput_local[(10)] = 0.000000e+00f;
  Conv2dOutput_local[(38)] = 0.000000e+00f;
  Conv2dOutput_local[(14)] = 0.000000e+00f;
  Conv2dOutput_local[(42)] = 0.000000e+00f;
  Conv2dOutput_local[(18)] = 0.000000e+00f;
  Conv2dOutput_local[(46)] = 0.000000e+00f;
  Conv2dOutput_local[(22)] = 0.000000e+00f;
  Conv2dOutput_local[(50)] = 0.000000e+00f;
  Conv2dOutput_local[(26)] = 0.000000e+00f;
  Conv2dOutput_local[(54)] = 0.000000e+00f;
  Conv2dOutput_local[(3)] = 0.000000e+00f;
  Conv2dOutput_local[(31)] = 0.000000e+00f;
  Conv2dOutput_local[(7)] = 0.000000e+00f;
  Conv2dOutput_local[(35)] = 0.000000e+00f;
  Conv2dOutput_local[(11)] = 0.000000e+00f;
  Conv2dOutput_local[(39)] = 0.000000e+00f;
  Conv2dOutput_local[(15)] = 0.000000e+00f;
  Conv2dOutput_local[(43)] = 0.000000e+00f;
  Conv2dOutput_local[(19)] = 0.000000e+00f;
  Conv2dOutput_local[(47)] = 0.000000e+00f;
  Conv2dOutput_local[(23)] = 0.000000e+00f;
  Conv2dOutput_local[(51)] = 0.000000e+00f;
  Conv2dOutput_local[(27)] = 0.000000e+00f;
  Conv2dOutput_local[(55)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 4))] = placeholder[((((((((int)threadIdx.x) / 10) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + ((((int)threadIdx.x) % 10) * 4)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder[((((((((((int)threadIdx.x) * 4) + 1) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 1) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder[((((((((((int)threadIdx.x) * 4) + 2) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 2) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder[((((((((((int)threadIdx.x) * 4) + 3) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 3) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 48))] = placeholder[((((((((((int)threadIdx.x) * 4) + 48) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 8) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 49))] = placeholder[((((((((((int)threadIdx.x) * 4) + 49) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 9) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 50))] = placeholder[((((((((((int)threadIdx.x) * 4) + 50) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 10) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 51))] = placeholder[((((((((((int)threadIdx.x) * 4) + 51) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 11) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 96))] = placeholder[((((((((((int)threadIdx.x) * 4) + 96) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 16) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 97))] = placeholder[((((((((((int)threadIdx.x) * 4) + 97) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 17) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 98))] = placeholder[((((((((((int)threadIdx.x) * 4) + 98) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 18) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 99))] = placeholder[((((((((((int)threadIdx.x) * 4) + 99) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 19) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 144))] = placeholder[((((((((((int)threadIdx.x) * 4) + 144) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 24) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 145))] = placeholder[((((((((((int)threadIdx.x) * 4) + 145) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 25) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 146))] = placeholder[((((((((((int)threadIdx.x) * 4) + 146) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 26) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 147))] = placeholder[((((((((((int)threadIdx.x) * 4) + 147) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 27) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 192))] = placeholder[((((((((((int)threadIdx.x) * 4) + 192) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 32) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 193))] = placeholder[((((((((((int)threadIdx.x) * 4) + 193) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 33) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 194))] = placeholder[((((((((((int)threadIdx.x) * 4) + 194) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 34) % 40)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 195))] = placeholder[((((((((((int)threadIdx.x) * 4) + 195) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + (((((int)threadIdx.x) * 4) + 35) % 40)))];
    if (((int)threadIdx.x) < 10) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 240))] = placeholder[((((((((int)blockIdx.x) / 10) * 160) + (rc_outer_outer * 40)) + (((int)threadIdx.x) * 4)) + 6720))];
    }
    if (((int)threadIdx.x) < 10) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 241))] = placeholder[((((((((((int)threadIdx.x) * 4) + 241) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + ((((int)threadIdx.x) * 4) + 1)))];
    }
    if (((int)threadIdx.x) < 10) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 242))] = placeholder[((((((((((int)threadIdx.x) * 4) + 242) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + ((((int)threadIdx.x) * 4) + 2)))];
    }
    if (((int)threadIdx.x) < 10) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 243))] = placeholder[((((((((((int)threadIdx.x) * 4) + 243) / 40) * 1120) + ((((int)blockIdx.x) / 10) * 160)) + (rc_outer_outer * 40)) + ((((int)threadIdx.x) * 4) + 3)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 320; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 38400) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) / 96) * 960)) + ((((int)blockIdx.x) % 10) * 96)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) % 96)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 40; ++rc_outer_inner) {
      Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(28)] = (Conv2dOutput_local[(28)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(32)] = (Conv2dOutput_local[(32)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(36)] = (Conv2dOutput_local[(36)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(40)] = (Conv2dOutput_local[(40)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(16)] = (Conv2dOutput_local[(16)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(44)] = (Conv2dOutput_local[(44)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(20)] = (Conv2dOutput_local[(20)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(48)] = (Conv2dOutput_local[(48)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(24)] = (Conv2dOutput_local[(24)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[(((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)))]));
      Conv2dOutput_local[(52)] = (Conv2dOutput_local[(52)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 48))]));
      Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(29)] = (Conv2dOutput_local[(29)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(33)] = (Conv2dOutput_local[(33)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(37)] = (Conv2dOutput_local[(37)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(41)] = (Conv2dOutput_local[(41)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(17)] = (Conv2dOutput_local[(17)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(45)] = (Conv2dOutput_local[(45)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(21)] = (Conv2dOutput_local[(21)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(49)] = (Conv2dOutput_local[(49)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(25)] = (Conv2dOutput_local[(25)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 1))]));
      Conv2dOutput_local[(53)] = (Conv2dOutput_local[(53)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 49))]));
      Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(30)] = (Conv2dOutput_local[(30)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(34)] = (Conv2dOutput_local[(34)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(38)] = (Conv2dOutput_local[(38)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(42)] = (Conv2dOutput_local[(42)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(18)] = (Conv2dOutput_local[(18)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(46)] = (Conv2dOutput_local[(46)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(22)] = (Conv2dOutput_local[(22)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(50)] = (Conv2dOutput_local[(50)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(26)] = (Conv2dOutput_local[(26)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 2))]));
      Conv2dOutput_local[(54)] = (Conv2dOutput_local[(54)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 50))]));
      Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(31)] = (Conv2dOutput_local[(31)] + (PaddedInput_shared[(rc_outer_inner)] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
      Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(35)] = (Conv2dOutput_local[(35)] + (PaddedInput_shared[((rc_outer_inner + 40))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
      Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(39)] = (Conv2dOutput_local[(39)] + (PaddedInput_shared[((rc_outer_inner + 80))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
      Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(43)] = (Conv2dOutput_local[(43)] + (PaddedInput_shared[((rc_outer_inner + 120))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
      Conv2dOutput_local[(19)] = (Conv2dOutput_local[(19)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(47)] = (Conv2dOutput_local[(47)] + (PaddedInput_shared[((rc_outer_inner + 160))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
      Conv2dOutput_local[(23)] = (Conv2dOutput_local[(23)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(51)] = (Conv2dOutput_local[(51)] + (PaddedInput_shared[((rc_outer_inner + 200))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
      Conv2dOutput_local[(27)] = (Conv2dOutput_local[(27)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 3))]));
      Conv2dOutput_local[(55)] = (Conv2dOutput_local[(55)] + (PaddedInput_shared[((rc_outer_inner + 240))] * placeholder_shared[((((rc_outer_inner * 96) + (((int)threadIdx.x) * 4)) + 51))]));
    }
  }
  for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
    for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
      Conv2dOutput[(((((yy_inner * 6720) + (((int)blockIdx.x) * 96)) + (((int)threadIdx.x) * 4)) + ff_inner))] = Conv2dOutput_local[(((yy_inner * 4) + ff_inner))];
      Conv2dOutput[((((((yy_inner * 6720) + (((int)blockIdx.x) * 96)) + (((int)threadIdx.x) * 4)) + ff_inner) + 48))] = Conv2dOutput_local[((((yy_inner * 4) + ff_inner) + 28))];
    }
  }
}


