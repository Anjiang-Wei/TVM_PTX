
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ Conv2dOutput, float* __restrict__ placeholder) {
  T_multiply[(((int)blockIdx.x))] = ((Conv2dOutput[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 184))]) * (max(min(((Conv2dOutput[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 184))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[16];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[1472];
  Conv2dOutput_local[(0)] = 0.000000e+00f;
  Conv2dOutput_local[(2)] = 0.000000e+00f;
  Conv2dOutput_local[(4)] = 0.000000e+00f;
  Conv2dOutput_local[(6)] = 0.000000e+00f;
  Conv2dOutput_local[(8)] = 0.000000e+00f;
  Conv2dOutput_local[(10)] = 0.000000e+00f;
  Conv2dOutput_local[(12)] = 0.000000e+00f;
  Conv2dOutput_local[(14)] = 0.000000e+00f;
  Conv2dOutput_local[(1)] = 0.000000e+00f;
  Conv2dOutput_local[(3)] = 0.000000e+00f;
  Conv2dOutput_local[(5)] = 0.000000e+00f;
  Conv2dOutput_local[(7)] = 0.000000e+00f;
  Conv2dOutput_local[(9)] = 0.000000e+00f;
  Conv2dOutput_local[(11)] = 0.000000e+00f;
  Conv2dOutput_local[(13)] = 0.000000e+00f;
  Conv2dOutput_local[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 10; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 128) {
      PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 3) * 10240) + ((((int)threadIdx.x) >> 4) * 1280)) + ((((int)blockIdx.x) & 7) * 160)) + (((((int)threadIdx.x) & 15) >> 3) * 80)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 1472) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 184))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 184))];
    placeholder_shared[((((int)threadIdx.x) + 368))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 368))];
    placeholder_shared[((((int)threadIdx.x) + 552))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 552))];
    placeholder_shared[((((int)threadIdx.x) + 736))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 736))];
    placeholder_shared[((((int)threadIdx.x) + 920))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 920))];
    placeholder_shared[((((int)threadIdx.x) + 1104))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 1104))];
    placeholder_shared[((((int)threadIdx.x) + 1288))] = placeholder1[((((rc_outer_outer * 1472) + ((int)threadIdx.x)) + 1288))];
    __syncthreads();
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) / 92) * 32))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) / 92) * 32))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 8))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 8))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 64))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 64))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 72))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 72))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 16))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 16))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 24))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 24))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 80))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 80))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 88))] * placeholder_shared[((((int)threadIdx.x) % 92))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 88))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 92))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 1))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 1))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 9))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 9))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 65))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 65))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 73))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 73))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 17))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 17))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 25))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 25))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 81))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 81))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 89))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 184))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 89))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 276))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 2))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 2))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 10))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 10))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 66))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 66))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 74))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 74))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 18))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 18))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 26))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 26))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 82))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 82))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 90))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 368))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 90))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 460))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 3))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 3))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 11))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 11))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 67))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 67))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 75))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 75))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 19))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 19))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 27))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 27))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 83))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 83))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 91))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 552))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 91))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 644))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 12))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 12))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 68))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 68))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 76))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 76))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 20))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 20))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 28))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 28))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 84))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 84))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 92))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 736))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 92))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 828))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 5))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 5))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 13))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 13))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 69))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 69))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 77))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 77))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 21))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 21))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 29))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 29))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 85))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 85))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 93))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 920))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 93))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1012))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 6))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 6))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 14))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 14))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 70))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 70))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 78))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 78))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 22))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 22))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 30))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 30))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 86))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 86))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 94))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1104))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 94))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1196))]));
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 7))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 7))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 15))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 15))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 71))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 71))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 79))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 79))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 23))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 23))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 31))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 31))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 87))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 87))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 95))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1288))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + 95))] * placeholder_shared[(((((int)threadIdx.x) % 92) + 1380))]));
  }
  for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
    Conv2dOutput[(((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)))] = Conv2dOutput_local[(yy_inner)];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 92))] = Conv2dOutput_local[((yy_inner + 2))];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 184))] = Conv2dOutput_local[((yy_inner + 4))];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 276))] = Conv2dOutput_local[((yy_inner + 6))];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 11776))] = Conv2dOutput_local[((yy_inner + 8))];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 11868))] = Conv2dOutput_local[((yy_inner + 10))];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 11960))] = Conv2dOutput_local[((yy_inner + 12))];
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 23552) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + ((((int)blockIdx.x) & 7) * 368)) + (((int)threadIdx.x) % 92)) + 12052))] = Conv2dOutput_local[((yy_inner + 14))];
  }
}


