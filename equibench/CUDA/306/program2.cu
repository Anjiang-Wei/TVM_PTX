
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[256];
  __shared__ float PaddedInput_shared[840];
  __shared__ float placeholder_shared[512];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
        Conv2dOutput[((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 64))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 128))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 192))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 65))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 129))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 193))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 66))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 130))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 194))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 67))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 131))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_outer_inner_init * 32) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 4)) + 195))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 105; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 32768) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 15) * 4096)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 15) * 256)) + (rc_outer_outer * 8)) + ((int)threadIdx.x)))];
    }
    ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + ((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 32))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 32))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 64))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 512))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 96))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 96) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 128))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 1024))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 160))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 160) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 192))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 1536))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 224))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 224) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 256))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 2048))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 288))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 288) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 320))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 2560))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 352))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 352) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 384))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 3072))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 416))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 416) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 448))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 4)) + 3584))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 480))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 4) + 480) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) * 4) + 32)))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
            Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 64))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 64))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 32))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 128))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 128))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 192))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 192))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 32))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 1))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 1))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 65))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 65))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 33))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 129))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 129))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 193))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 193))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 33))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 2))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 2))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 66))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 66))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 34))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 130))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 130))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 194))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 194))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 34))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 3))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 3))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 3))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 67))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 67))] + (PaddedInput_shared[(((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 35))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 131))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 131))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 3))]));
            Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 195))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (xx_outer_inner * 16)) + (ff_outer_inner * 4)) + 195))] + (PaddedInput_shared[((((((yy_outer_inner * 240) + ((((int)threadIdx.x) >> 1) * 32)) + (xx_outer_inner * 16)) + rc_outer_inner) + 480))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 35))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
        T_add[(((((((((((int)blockIdx.x) >> 3) * 16384) + (ax1_inner * 4096)) + ((((int)threadIdx.x) >> 1) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))] = (Conv2dOutput[((((ax1_inner * 32) + (ax2_inner * 16)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))]);
        T_add[((((((((((((int)blockIdx.x) >> 3) * 16384) + (ax1_inner * 4096)) + ((((int)threadIdx.x) >> 1) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 32))] = (Conv2dOutput[(((((ax1_inner * 32) + (ax2_inner * 16)) + ax3_inner) + 64))] + placeholder2[((((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 32))]);
        T_add[((((((((((((int)blockIdx.x) >> 3) * 16384) + (ax1_inner * 4096)) + ((((int)threadIdx.x) >> 1) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 8192))] = (Conv2dOutput[(((((ax1_inner * 32) + (ax2_inner * 16)) + ax3_inner) + 128))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))]);
        T_add[((((((((((((int)blockIdx.x) >> 3) * 16384) + (ax1_inner * 4096)) + ((((int)threadIdx.x) >> 1) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 8224))] = (Conv2dOutput[(((((ax1_inner * 32) + (ax2_inner * 16)) + ax3_inner) + 192))] + placeholder2[((((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 32))]);
      }
    }
  }
}


