
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[2048];
  __shared__ float placeholder_shared[1024];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
      for (int yy_inner_init = 0; yy_inner_init < 4; ++yy_inner_init) {
        Conv2dOutput[((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 256))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 257))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 2))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 258))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 3))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 259))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 4))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 260))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 5))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 261))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 6))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 262))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 7))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 263))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 8))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 264))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 9))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 265))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 10))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 266))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 11))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 267))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 12))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 268))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 13))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 269))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 14))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 270))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 15))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + 271))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 6) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 6) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 32768))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 6) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 524288))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((((((int)threadIdx.x) + 768) >> 9) * 524288) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) >> 6) + 4) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 6) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 1048576))];
    PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[((((((((((((int)threadIdx.x) + 1280) >> 9) * 524288) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) >> 6) + 4) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 6) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 1572864))];
    PaddedInput_shared[((((int)threadIdx.x) + 1792))] = placeholder[((((((((((((int)threadIdx.x) + 1792) >> 9) * 524288) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) >> 6) + 4) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 63) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + (((rc_outer_outer * 1024) + (((int)threadIdx.x) * 4)))))[0];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
          for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
            Conv2dOutput[((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)))] = (Conv2dOutput[((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)))] + (PaddedInput_shared[((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 256))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 256))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1024))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 1))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 1))] + (PaddedInput_shared[((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 257))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 257))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1024))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 2))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 2))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 4))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 258))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 258))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1028))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 3))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 3))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 259))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 259))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1028))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 4))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 4))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 8))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 260))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 260))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1032))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 5))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 5))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 8))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 261))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 261))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1032))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 6))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 6))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 12))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 262))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 262))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1036))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 7))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 7))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 12))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 263))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 263))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1036))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 8))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 8))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 16))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 264))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 264))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1040))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 9))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 9))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 16))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 265))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 265))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1040))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 10))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 10))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 20))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 266))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 266))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1044))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 11))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 11))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 20))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 267))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 267))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1044))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 12))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 12))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 24))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 268))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 268))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1048))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 13))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 13))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 24))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 269))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 269))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1048))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 14))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 14))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 28))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 270))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 270))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1052))] * placeholder_shared[(((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 15))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 15))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 28))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
            Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 271))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_inner * 32)) + (xx_outer_inner * 16)) + 271))] + (PaddedInput_shared[(((((((nn_inner * 512) + ((((int)threadIdx.x) >> 7) * 256)) + (yy_inner * 64)) + (xx_outer_inner * 32)) + rc_inner) + 1052))] * placeholder_shared[((((rc_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 16; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
          T_relu[(((((((((ax0_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 131072)) + ((((int)threadIdx.x) >> 7) * 65536)) + (ax1_inner * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner))] = max(((Conv2dOutput[(((((ax0_inner * 128) + (ax1_inner * 32)) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 127) * 2) + ax3_inner))]) + placeholder3[(((((((((ax0_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 131072)) + ((((int)threadIdx.x) >> 7) * 65536)) + (ax1_inner * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((((ax0_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 131072)) + ((((int)threadIdx.x) >> 7) * 65536)) + (ax1_inner * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner) + 2097152))] = max(((Conv2dOutput[((((((ax0_inner * 128) + (ax1_inner * 32)) + (ax2_inner * 2)) + ax3_inner) + 256))] + placeholder2[((((((int)threadIdx.x) & 127) * 2) + ax3_inner))]) + placeholder3[((((((((((ax0_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 131072)) + ((((int)threadIdx.x) >> 7) * 65536)) + (ax1_inner * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner) + 2097152))]), 0.000000e+00f);
        }
      }
    }
  }
}


