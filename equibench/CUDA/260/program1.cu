
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[16384];
  __shared__ float PaddedInput_shared[2048];
  __shared__ float placeholder_shared[256];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 32; ++yy_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 64; ++ff_outer_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
          Conv2dOutput[(((((nn_outer_inner_init * 4096) + (yy_outer_inner_init * 128)) + (xx_inner_init * 64)) + ff_outer_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 4096) + (yy_outer_inner_init * 128)) + (xx_inner_init * 64)) + ff_outer_inner_init) + 8192))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 256; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 1)) >> 8) * 524288) + ((((int)blockIdx.x) >> 4) * 262144)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 1)) & 255) >> 3) * 8192)) + (((((int)blockIdx.x) & 15) >> 1) * 1024)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 1)) & 7) * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 512) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) >> 7) * 256)) + ((((int)blockIdx.x) & 1) * 128)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) & 127)))];
    }
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 32; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 64; ++ff_outer_inner) {
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
            for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
              Conv2dOutput[(((((nn_outer_inner * 4096) + (yy_outer_inner * 128)) + (xx_inner * 64)) + ff_outer_inner))] = (Conv2dOutput[(((((nn_outer_inner * 4096) + (yy_outer_inner * 128)) + (xx_inner * 64)) + ff_outer_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 4)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + ff_outer_inner))]));
              Conv2dOutput[((((((nn_outer_inner * 4096) + (yy_outer_inner * 128)) + (xx_inner * 64)) + ff_outer_inner) + 8192))] = (Conv2dOutput[((((((nn_outer_inner * 4096) + (yy_outer_inner * 128)) + (xx_inner * 64)) + ff_outer_inner) + 8192))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 2) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 4)) + (xx_inner * 2)) + rc_inner) + 8))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + ff_outer_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 32; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 64; ++ax3_inner) {
          T_relu[((((((((((((((int)threadIdx.x) >> 2) * 2097152) + (ax0_inner * 1048576)) + ((((int)blockIdx.x) >> 4) * 524288)) + (ax1_inner * 16384)) + (((((int)blockIdx.x) & 15) >> 1) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner))] = max(((Conv2dOutput[(((((ax0_inner * 4096) + (ax1_inner * 128)) + (ax2_inner * 64)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 128) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner))]) + placeholder3[((((((((((((((int)threadIdx.x) >> 2) * 2097152) + (ax0_inner * 1048576)) + ((((int)blockIdx.x) >> 4) * 524288)) + (ax1_inner * 16384)) + (((((int)blockIdx.x) & 15) >> 1) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((((((int)threadIdx.x) >> 2) * 2097152) + (ax0_inner * 1048576)) + ((((int)blockIdx.x) >> 4) * 524288)) + (ax1_inner * 16384)) + (((((int)blockIdx.x) & 15) >> 1) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner) + 1024))] = max(((Conv2dOutput[((((((ax0_inner * 4096) + (ax1_inner * 128)) + (ax2_inner * 64)) + ax3_inner) + 8192))] + placeholder2[(((((((int)blockIdx.x) & 1) * 128) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner))]) + placeholder3[(((((((((((((((int)threadIdx.x) >> 2) * 2097152) + (ax0_inner * 1048576)) + ((((int)blockIdx.x) >> 4) * 524288)) + (ax1_inner * 16384)) + (((((int)blockIdx.x) & 15) >> 1) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 1) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner) + 1024))]), 0.000000e+00f);
        }
      }
    }
  }
}


