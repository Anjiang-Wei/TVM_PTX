
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[196];
  __shared__ float PaddedInput_shared[98];
  __shared__ float placeholder_shared[16];
  for (int yy_inner_init = 0; yy_inner_init < 14; ++yy_inner_init) {
    Conv2dOutput[(yy_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 14))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 28))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 42))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 56))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 70))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 84))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 98))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 112))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 126))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 140))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 154))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 168))] = 0.000000e+00f;
    Conv2dOutput[((yy_inner_init + 182))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 992; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)threadIdx.x) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + ((((int)threadIdx.x) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 8))] = placeholder[(((((((((int)threadIdx.x) + 8) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 1) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 16))] = placeholder[(((((((((int)threadIdx.x) + 16) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 2) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 24))] = placeholder[(((((((((int)threadIdx.x) + 24) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 3) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[(((((((((int)threadIdx.x) + 32) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 4) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 40))] = placeholder[(((((((((int)threadIdx.x) + 40) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 5) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 48))] = placeholder[(((((((((int)threadIdx.x) + 48) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 6) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 56))] = placeholder[(((((((((int)threadIdx.x) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + ((((int)threadIdx.x) % 7) * 992)) + rc_outer_outer) + 111104))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[(((((((((int)threadIdx.x) + 64) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 1) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 72))] = placeholder[(((((((((int)threadIdx.x) + 72) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 2) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 80))] = placeholder[(((((((((int)threadIdx.x) + 80) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 3) % 7) * 992)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 88))] = placeholder[(((((((((int)threadIdx.x) + 88) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + (((((int)threadIdx.x) + 4) % 7) * 992)) + rc_outer_outer))];
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[(((((((((int)threadIdx.x) + 96) / 7) * 13888) + ((((int)blockIdx.x) >> 3) * 6944)) + ((((int)threadIdx.x) + 5) * 992)) + rc_outer_outer))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 128) + ((((int)blockIdx.x) & 7) * 16)) + (((int)threadIdx.x) * 2)))))[0];
    __syncthreads();
    for (int yy_inner = 0; yy_inner < 14; ++yy_inner) {
      Conv2dOutput[(yy_inner)] = (Conv2dOutput[(yy_inner)] + (PaddedInput_shared[((yy_inner * 7))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 14))] = (Conv2dOutput[((yy_inner + 14))] + (PaddedInput_shared[((yy_inner * 7))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
      Conv2dOutput[((yy_inner + 28))] = (Conv2dOutput[((yy_inner + 28))] + (PaddedInput_shared[(((yy_inner * 7) + 1))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 42))] = (Conv2dOutput[((yy_inner + 42))] + (PaddedInput_shared[(((yy_inner * 7) + 1))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
      Conv2dOutput[((yy_inner + 56))] = (Conv2dOutput[((yy_inner + 56))] + (PaddedInput_shared[(((yy_inner * 7) + 2))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 70))] = (Conv2dOutput[((yy_inner + 70))] + (PaddedInput_shared[(((yy_inner * 7) + 2))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
      Conv2dOutput[((yy_inner + 84))] = (Conv2dOutput[((yy_inner + 84))] + (PaddedInput_shared[(((yy_inner * 7) + 3))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 98))] = (Conv2dOutput[((yy_inner + 98))] + (PaddedInput_shared[(((yy_inner * 7) + 3))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
      Conv2dOutput[((yy_inner + 112))] = (Conv2dOutput[((yy_inner + 112))] + (PaddedInput_shared[(((yy_inner * 7) + 4))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 126))] = (Conv2dOutput[((yy_inner + 126))] + (PaddedInput_shared[(((yy_inner * 7) + 4))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
      Conv2dOutput[((yy_inner + 140))] = (Conv2dOutput[((yy_inner + 140))] + (PaddedInput_shared[(((yy_inner * 7) + 5))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 154))] = (Conv2dOutput[((yy_inner + 154))] + (PaddedInput_shared[(((yy_inner * 7) + 5))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
      Conv2dOutput[((yy_inner + 168))] = (Conv2dOutput[((yy_inner + 168))] + (PaddedInput_shared[(((yy_inner * 7) + 6))] * placeholder_shared[(((int)threadIdx.x))]));
      Conv2dOutput[((yy_inner + 182))] = (Conv2dOutput[((yy_inner + 182))] + (PaddedInput_shared[(((yy_inner * 7) + 6))] * placeholder_shared[((((int)threadIdx.x) + 8))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 14; ++ax1_inner) {
    T_relu[(((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)))] = max((Conv2dOutput[(ax1_inner)] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 8))] = max((Conv2dOutput[((ax1_inner + 14))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 128))] = max((Conv2dOutput[((ax1_inner + 28))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 136))] = max((Conv2dOutput[((ax1_inner + 42))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 256))] = max((Conv2dOutput[((ax1_inner + 56))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 264))] = max((Conv2dOutput[((ax1_inner + 70))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 384))] = max((Conv2dOutput[((ax1_inner + 84))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 392))] = max((Conv2dOutput[((ax1_inner + 98))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 512))] = max((Conv2dOutput[((ax1_inner + 112))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 520))] = max((Conv2dOutput[((ax1_inner + 126))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 640))] = max((Conv2dOutput[((ax1_inner + 140))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 648))] = max((Conv2dOutput[((ax1_inner + 154))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 768))] = max((Conv2dOutput[((ax1_inner + 168))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)))]), 0.000000e+00f);
    T_relu[((((((ax1_inner * 1792) + ((((int)blockIdx.x) >> 3) * 896)) + ((((int)blockIdx.x) & 7) * 16)) + ((int)threadIdx.x)) + 776))] = max((Conv2dOutput[((ax1_inner + 182))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
  }
}


