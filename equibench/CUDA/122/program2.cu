
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[40];
  __shared__ float PaddedInput_shared[700];
  __shared__ float placeholder_shared[64];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 5; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 4) + ff_outer_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + ff_outer_inner_init) + 20))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 144; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) / 70)) < 10) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 140) + (((int)threadIdx.x) >> 1)) < 350) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 280) + ((int)threadIdx.x)) < 700) {
            PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 280) + ((int)threadIdx.x)))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) / 70)) / 5) * 352800) + ((((int)blockIdx.x) >> 1) * 50400)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) / 70)) % 5) * 10080)) + (((((int)threadIdx.x) % 70) >> 1) * 288)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 64) {
        if (((int)threadIdx.x) < 16) {
          placeholder_shared[(((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 128) + ((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 31)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 5; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
          Conv2dOutput[(((yy_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 350) + (yy_outer_inner * 70)) + (((((int)threadIdx.x) % 140) >> 2) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 3) * 4)) + ff_outer_inner))]));
          Conv2dOutput[((((yy_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((yy_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 350) + (yy_outer_inner * 70)) + (((((int)threadIdx.x) % 140) >> 2) * 2)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 3) * 4)) + ff_outer_inner) + 16))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[(((((((((((int)threadIdx.x) / 140) * 78400) + ((((int)blockIdx.x) >> 1) * 11200)) + (ax1_inner * 2240)) + (((((int)threadIdx.x) % 140) >> 2) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 4) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) / 140) * 78400) + ((((int)blockIdx.x) >> 1) * 11200)) + (ax1_inner * 2240)) + (((((int)threadIdx.x) % 140) >> 2) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 20))] + placeholder2[((((((((int)blockIdx.x) & 1) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
    }
  }
}


