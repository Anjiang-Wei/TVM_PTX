
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[90];
  __shared__ float PaddedInput_shared[900];
  __shared__ float placeholder_shared[8];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 3; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
          Conv2dOutput[(((((nn_inner_init * 45) + (yy_outer_inner_init * 15)) + (xx_outer_inner_init * 3)) + xx_inner_init))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 200; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)) < 900) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16000) + (((int)threadIdx.x) * 200)) + rc_outer_outer))];
      }
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 80) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 3; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
        for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
          for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
            Conv2dOutput[(((((nn_inner * 45) + (yy_outer_inner * 15)) + (xx_outer_inner * 3)) + xx_inner))] = (Conv2dOutput[(((((nn_inner * 45) + (yy_outer_inner * 15)) + (xx_outer_inner * 3)) + xx_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 450) + (nn_inner * 225)) + (((((int)threadIdx.x) % 40) >> 3) * 45)) + (yy_outer_inner * 15)) + (xx_outer_inner * 3)) + xx_inner))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
        T_add[(((((((((((int)threadIdx.x) / 40) * 36000) + (ax0_inner * 18000)) + (((((int)threadIdx.x) % 40) >> 3) * 3600)) + (ax1_inner * 1200)) + (ax2_inner * 80)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)))] = ((Conv2dOutput[((((ax0_inner * 45) + (ax1_inner * 15)) + ax2_inner))] + placeholder2[(((((int)blockIdx.x) * 8) + (((int)threadIdx.x) & 7)))]) + placeholder3[(((((((((((int)threadIdx.x) / 40) * 36000) + (ax0_inner * 18000)) + (((((int)threadIdx.x) % 40) >> 3) * 3600)) + (ax1_inner * 1200)) + (ax2_inner * 80)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 7)))]);
      }
    }
  }
}


