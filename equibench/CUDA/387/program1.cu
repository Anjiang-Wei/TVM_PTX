
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[64];
  __shared__ float placeholder_shared[128];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 240; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 16) {
      ((float4*)(PaddedInput_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder + ((((((((int)threadIdx.x) >> 1) * 7680) + ((((int)blockIdx.x) / 5) * 1920)) + ((((int)threadIdx.x) & 1) * 960)) + (rc_outer_outer * 4)))))[0];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 640) + ((((int)blockIdx.x) % 5) * 32)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[(((((rc_outer_outer * 640) + ((((int)blockIdx.x) % 5) * 32)) + ((int)threadIdx.x)) + 160))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((rc_outer_outer * 640) + ((((int)blockIdx.x) % 5) * 32)) + ((int)threadIdx.x)) + 320))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((rc_outer_outer * 640) + ((((int)blockIdx.x) % 5) * 32)) + ((int)threadIdx.x)) + 480))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 32))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 32))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 4))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 4))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 36))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 36))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 8))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 8))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 40))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 12))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 12))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 44))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 16) + rc_outer_inner) + 44))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      T_add[((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)))] = (placeholder2[((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)))] + (Conv2dOutput[(((ax1_inner * 2) + ax2_inner))] + placeholder3[((((((int)blockIdx.x) % 5) * 32) + (((int)threadIdx.x) & 15)))]));
      T_add[(((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)) + 16))] = (placeholder2[(((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)) + 16))] + (Conv2dOutput[((((ax1_inner * 2) + ax2_inner) + 4))] + placeholder3[(((((((int)blockIdx.x) % 5) * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      T_add[(((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)) + 5120))] = (placeholder2[(((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)) + 5120))] + (Conv2dOutput[((((ax1_inner * 2) + ax2_inner) + 8))] + placeholder3[((((((int)blockIdx.x) % 5) * 32) + (((int)threadIdx.x) & 15)))]));
      T_add[(((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)) + 5136))] = (placeholder2[(((((((((((int)threadIdx.x) >> 4) * 2560) + (ax1_inner * 1280)) + ((((int)blockIdx.x) / 5) * 320)) + (ax2_inner * 160)) + ((((int)blockIdx.x) % 5) * 32)) + (((int)threadIdx.x) & 15)) + 5136))] + (Conv2dOutput[((((ax1_inner * 2) + ax2_inner) + 12))] + placeholder3[(((((((int)blockIdx.x) % 5) * 32) + (((int)threadIdx.x) & 15)) + 16))]));
    }
  }
}


