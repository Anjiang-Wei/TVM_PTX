
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_divide, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[32];
  __shared__ float placeholder_shared[128];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((int)threadIdx.x))];
  PaddedInput_shared[((((int)threadIdx.x) + 2))] = placeholder[((((int)threadIdx.x) + 2))];
  PaddedInput_shared[((((int)threadIdx.x) + 4))] = placeholder[((((int)threadIdx.x) + 4))];
  PaddedInput_shared[((((int)threadIdx.x) + 6))] = placeholder[((((int)threadIdx.x) + 6))];
  PaddedInput_shared[((((int)threadIdx.x) + 8))] = placeholder[((((int)threadIdx.x) + 8))];
  PaddedInput_shared[((((int)threadIdx.x) + 10))] = placeholder[((((int)threadIdx.x) + 10))];
  PaddedInput_shared[((((int)threadIdx.x) + 12))] = placeholder[((((int)threadIdx.x) + 12))];
  PaddedInput_shared[((((int)threadIdx.x) + 14))] = placeholder[((((int)threadIdx.x) + 14))];
  PaddedInput_shared[((((int)threadIdx.x) + 16))] = placeholder[((((int)threadIdx.x) + 16))];
  PaddedInput_shared[((((int)threadIdx.x) + 18))] = placeholder[((((int)threadIdx.x) + 18))];
  PaddedInput_shared[((((int)threadIdx.x) + 20))] = placeholder[((((int)threadIdx.x) + 20))];
  PaddedInput_shared[((((int)threadIdx.x) + 22))] = placeholder[((((int)threadIdx.x) + 22))];
  PaddedInput_shared[((((int)threadIdx.x) + 24))] = placeholder[((((int)threadIdx.x) + 24))];
  PaddedInput_shared[((((int)threadIdx.x) + 26))] = placeholder[((((int)threadIdx.x) + 26))];
  PaddedInput_shared[((((int)threadIdx.x) + 28))] = placeholder[((((int)threadIdx.x) + 28))];
  PaddedInput_shared[((((int)threadIdx.x) + 30))] = placeholder[((((int)threadIdx.x) + 30))];
  ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 4))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 120))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 8))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 240))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 12))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 360))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 16))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 480))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 20))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 600))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 24))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 720))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 28))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 840))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 32))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 960))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 36))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1080))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 40))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1200))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 44))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1320))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 48))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1440))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 52))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1560))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 56))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1680))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 60))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1800))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 1920))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 68))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2040))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 72))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2160))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 76))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2280))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 80))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2400))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 84))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2520))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 88))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2640))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 92))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2760))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 96))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 2880))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 100))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3000))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 104))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3120))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 108))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3240))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 112))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3360))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 116))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3480))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 120))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3600))))[0];
  ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 124))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + 3720))))[0];
  __syncthreads();
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 4))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 8))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 12))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 5))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 9))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 13))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 16))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 20))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 24))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 28))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 17))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 21))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 25))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 29))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 32))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 36))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 40))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 44))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 33))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 37))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 41))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 45))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 48))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 52))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 56))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 60))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 49))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 53))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 57))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 61))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(16)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(17)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 68))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(18)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 72))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(19)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 76))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(16)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(17)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 69))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(18)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 73))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(19)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 77))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(20)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 80))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(21)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 84))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(22)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 88))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(23)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 92))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(20)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 81))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(21)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 85))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(22)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 89))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(23)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 93))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(24)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 96))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(25)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 100))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(26)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 104))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(27)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 108))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(24)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 97))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(25)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 101))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(26)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 105))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(27)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 109))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(28)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 112))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(29)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 116))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(30)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 120))]));
  Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(31)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 124))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(28)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 113))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(29)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 117))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(30)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 121))]));
  Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(31)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 125))]));
  for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
    T_divide[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + ax3_inner))] = (max(min(((Conv2dOutput[(ax3_inner)] + placeholder2[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + ax3_inner))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f);
  }
}


