
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[192];
  __shared__ float PaddedInput_shared[2400];
  __shared__ float placeholder_shared[256];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 6; ++yy_outer_inner_init) {
      Conv2dOutput[(((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 48))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 49))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 50))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 51))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 52))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 5))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 53))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 54))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 7))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 96) + (yy_outer_inner_init * 8)) + 55))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) / 20) * 7680) + (((int)blockIdx.x) * 2560)) + (((((int)threadIdx.x) % 20) >> 1) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 800))] = placeholder[((((((((((int)threadIdx.x) / 20) * 7680) + (((int)blockIdx.x) * 2560)) + (((((int)threadIdx.x) % 20) >> 1) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 307200))];
    PaddedInput_shared[((((int)threadIdx.x) + 1600))] = placeholder[((((((((((int)threadIdx.x) / 20) * 7680) + (((int)blockIdx.x) * 2560)) + (((((int)threadIdx.x) % 20) >> 1) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 614400))];
    if (((int)threadIdx.x) < 256) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 256) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 6; ++yy_outer_inner) {
          Conv2dOutput[(((nn_outer_inner * 96) + (yy_outer_inner * 8)))] = (Conv2dOutput[(((nn_outer_inner * 96) + (yy_outer_inner * 8)))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 48))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 48))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 1))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 1))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 1))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 49))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 49))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 1))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 2))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 2))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 2))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 50))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 50))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 2))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 3))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 3))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 3))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 51))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 51))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 3))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 4))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 4))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 4))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 52))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 52))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 4))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 5))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 5))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 5))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 53))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 53))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 5))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 6))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 6))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 6))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 54))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 54))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 6))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 7))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 7))] + (PaddedInput_shared[((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 7))]));
          Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 55))] = (Conv2dOutput[((((nn_outer_inner * 96) + (yy_outer_inner * 8)) + 55))] + (PaddedInput_shared[(((((((nn_outer_inner * 1200) + ((((int)threadIdx.x) / 160) * 120)) + (yy_outer_inner * 20)) + (((((int)threadIdx.x) % 160) >> 4) * 2)) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 7))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 6; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
        T_relu[(((((((ax0_inner * 115200) + ((((int)threadIdx.x) / 160) * 23040)) + (ax1_inner * 3840)) + (((int)blockIdx.x) * 1280)) + ((((int)threadIdx.x) % 160) * 8)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 48) + (ax1_inner * 8)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 15) * 8) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


