
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[768];
  __shared__ float PaddedInput_shared[256];
  __shared__ float placeholder_shared[96];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 4; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 24; ++ff_outer_inner_init) {
      Conv2dOutput[(((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 384))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 385))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 192))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 576))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 193))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 48) + (ff_outer_inner_init * 2)) + 577))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 576; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 73728) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 73728) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer) + 36864))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 73728) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer) + 147456))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[(((((((((((int)threadIdx.x) + 96) >> 6) * 147456) + ((((int)blockIdx.x) >> 1) * 73728)) + (((((int)threadIdx.x) >> 3) + 4) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 73728) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer) + 294912))];
    PaddedInput_shared[((((int)threadIdx.x) + 160))] = placeholder[(((((((((((int)threadIdx.x) + 160) >> 6) * 147456) + ((((int)blockIdx.x) >> 1) * 73728)) + (((((int)threadIdx.x) >> 3) + 4) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 73728) + ((((int)threadIdx.x) >> 3) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer) + 442368))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[(((((((((((int)threadIdx.x) + 224) >> 6) * 147456) + ((((int)blockIdx.x) >> 1) * 73728)) + (((((int)threadIdx.x) >> 3) + 4) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((int)threadIdx.x) & 7) * 576)) + rc_outer_outer))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 96) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((rc_outer_outer * 96) + ((int)threadIdx.x)) + 32))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((rc_outer_outer * 96) + ((int)threadIdx.x)) + 64))];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 24; ++ff_outer_inner) {
        Conv2dOutput[(((xx_outer_inner * 48) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((xx_outer_inner * 48) + (ff_outer_inner * 2)))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner))] * placeholder_shared[((ff_outer_inner * 2))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 384))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 384))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner))] * placeholder_shared[(((ff_outer_inner * 2) + 48))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner))] * placeholder_shared[(((ff_outer_inner * 2) + 1))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 385))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 385))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner))] * placeholder_shared[(((ff_outer_inner * 2) + 49))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 192))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner) + 8))] * placeholder_shared[((ff_outer_inner * 2))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 576))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 576))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner) + 8))] * placeholder_shared[(((ff_outer_inner * 2) + 48))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 193))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner) + 8))] * placeholder_shared[(((ff_outer_inner * 2) + 1))]));
        Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 577))] = (Conv2dOutput[((((xx_outer_inner * 48) + (ff_outer_inner * 2)) + 577))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 16) + ((((int)threadIdx.x) & 1) * 4)) + xx_outer_inner) + 8))] * placeholder_shared[(((ff_outer_inner * 2) + 49))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 48; ++ax3_inner) {
        T_add[((((((((((((int)threadIdx.x) >> 3) * 24576) + ((((int)blockIdx.x) >> 1) * 12288)) + (((((int)threadIdx.x) & 7) >> 1) * 3072)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + ((((int)threadIdx.x) & 1) * 384)) + (ax2_inner * 96)) + ax3_inner))] = (placeholder2[((((((((((((int)threadIdx.x) >> 3) * 24576) + ((((int)blockIdx.x) >> 1) * 12288)) + (((((int)threadIdx.x) & 7) >> 1) * 3072)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + ((((int)threadIdx.x) & 1) * 384)) + (ax2_inner * 96)) + ax3_inner))] + (Conv2dOutput[((((ax1_inner * 192) + (ax2_inner * 48)) + ax3_inner))] + placeholder3[(ax3_inner)]));
        T_add[(((((((((((((int)threadIdx.x) >> 3) * 24576) + ((((int)blockIdx.x) >> 1) * 12288)) + (((((int)threadIdx.x) & 7) >> 1) * 3072)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + ((((int)threadIdx.x) & 1) * 384)) + (ax2_inner * 96)) + ax3_inner) + 48))] = (placeholder2[(((((((((((((int)threadIdx.x) >> 3) * 24576) + ((((int)blockIdx.x) >> 1) * 12288)) + (((((int)threadIdx.x) & 7) >> 1) * 3072)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + ((((int)threadIdx.x) & 1) * 384)) + (ax2_inner * 96)) + ax3_inner) + 48))] + (Conv2dOutput[(((((ax1_inner * 192) + (ax2_inner * 48)) + ax3_inner) + 384))] + placeholder3[((ax3_inner + 48))]));
      }
    }
  }
}


