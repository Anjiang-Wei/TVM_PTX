
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[192];
  __shared__ float placeholder_shared[24];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 8) + (yy_outer_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 8) + (yy_outer_inner_init * 2)) + ff_inner_init) + 16))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 96; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) / 48) * 147456) + ((((int)blockIdx.x) / 192) * 36864)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 48) / 12) * 9216)) + (((((int)blockIdx.x) % 192) / 24) * 1152)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 12) / 6) * 576)) + (rc_outer_outer * 6)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 6)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 4) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 576) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 96)) + ((((int)blockIdx.x) % 24) * 4)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 6; ++rc_inner) {
          for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
            Conv2dOutput[((((nn_outer_inner * 8) + (yy_outer_inner * 2)) + ff_inner))] = (Conv2dOutput[((((nn_outer_inner * 8) + (yy_outer_inner * 2)) + ff_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 96) + (nn_outer_inner * 48)) + (yy_outer_inner * 12)) + rc_inner))] * placeholder_shared[((((rc_inner * 4) + ((((int)threadIdx.x) & 1) * 2)) + ff_inner))]));
            Conv2dOutput[(((((nn_outer_inner * 8) + (yy_outer_inner * 2)) + ff_inner) + 16))] = (Conv2dOutput[(((((nn_outer_inner * 8) + (yy_outer_inner * 2)) + ff_inner) + 16))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 96) + (nn_outer_inner * 48)) + (yy_outer_inner * 12)) + rc_inner) + 6))] * placeholder_shared[((((rc_inner * 4) + ((((int)threadIdx.x) & 1) * 2)) + ff_inner))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[((((((((((((int)threadIdx.x) >> 1) * 49152) + (ax0_inner * 24576)) + ((((int)blockIdx.x) / 192) * 6144)) + (ax1_inner * 1536)) + (((((int)blockIdx.x) % 192) / 24) * 192)) + ((((int)blockIdx.x) % 24) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))] = (placeholder2[((((((((((((int)threadIdx.x) >> 1) * 49152) + (ax0_inner * 24576)) + ((((int)blockIdx.x) / 192) * 6144)) + (ax1_inner * 1536)) + (((((int)blockIdx.x) % 192) / 24) * 192)) + ((((int)blockIdx.x) % 24) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))] + (Conv2dOutput[((((ax0_inner * 8) + (ax1_inner * 2)) + ax3_inner))] + placeholder3[(((((((int)blockIdx.x) % 24) * 4) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))]));
        T_add[(((((((((((((int)threadIdx.x) >> 1) * 49152) + (ax0_inner * 24576)) + ((((int)blockIdx.x) / 192) * 6144)) + (ax1_inner * 1536)) + (((((int)blockIdx.x) % 192) / 24) * 192)) + ((((int)blockIdx.x) % 24) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 96))] = (placeholder2[(((((((((((((int)threadIdx.x) >> 1) * 49152) + (ax0_inner * 24576)) + ((((int)blockIdx.x) / 192) * 6144)) + (ax1_inner * 1536)) + (((((int)blockIdx.x) % 192) / 24) * 192)) + ((((int)blockIdx.x) % 24) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 96))] + (Conv2dOutput[(((((ax0_inner * 8) + (ax1_inner * 2)) + ax3_inner) + 16))] + placeholder3[(((((((int)blockIdx.x) % 24) * 4) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))]));
      }
    }
  }
}


