
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[72];
  __shared__ float PaddedInput_shared[180];
  __shared__ float placeholder_shared[512];
  for (int yy_inner_init = 0; yy_inner_init < 3; ++yy_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
      Conv2dOutput[(((yy_inner_init * 3) + xx_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 9))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 18))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 27))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 36))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 45))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 54))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 63))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 1)) < 90) {
        if (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 180) {
          if (((int)threadIdx.x) < 30) {
            PaddedInput_shared[(((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 46080) + ((((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 1)) / 15) * 7680)) + (((((int)blockIdx.x) & 3) >> 1) * 3840)) + ((((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 1)) % 15) * 256)) + (rc_outer_outer * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s & 1)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + ((int)threadIdx.x)) < 512) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 1024) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + ((int)threadIdx.x)) >> 8) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + ((int)threadIdx.x)) & 255)))];
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int yy_inner = 0; yy_inner < 3; ++yy_inner) {
        for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
          Conv2dOutput[(((yy_inner * 3) + xx_inner))] = (Conv2dOutput[(((yy_inner * 3) + xx_inner))] + (PaddedInput_shared[(((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[(((rc_inner * 256) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 9))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 9))] + (PaddedInput_shared[(((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 256) + (((int)threadIdx.x) & 63)) + 64))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 18))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 18))] + (PaddedInput_shared[(((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 256) + (((int)threadIdx.x) & 63)) + 128))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 27))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 27))] + (PaddedInput_shared[(((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 256) + (((int)threadIdx.x) & 63)) + 192))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 36))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 36))] + (PaddedInput_shared[((((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner) + 90))] * placeholder_shared[(((rc_inner * 256) + (((int)threadIdx.x) & 63)))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 45))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 45))] + (PaddedInput_shared[((((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner) + 90))] * placeholder_shared[((((rc_inner * 256) + (((int)threadIdx.x) & 63)) + 64))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 54))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 54))] + (PaddedInput_shared[((((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner) + 90))] * placeholder_shared[((((rc_inner * 256) + (((int)threadIdx.x) & 63)) + 128))]));
          Conv2dOutput[((((yy_inner * 3) + xx_inner) + 63))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 63))] + (PaddedInput_shared[((((((yy_inner * 30) + ((((int)threadIdx.x) >> 6) * 6)) + (xx_inner * 2)) + rc_inner) + 90))] * placeholder_shared[((((rc_inner * 256) + (((int)threadIdx.x) & 63)) + 192))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)))] = max(((Conv2dOutput[(((ax1_inner * 3) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 64))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 9))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)) + 64))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 64))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 128))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 18))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)) + 128))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 128))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 192))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 27))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)) + 192))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 192))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46080))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 36))] + placeholder2[((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46080))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46144))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 45))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)) + 64))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46144))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46208))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 54))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)) + 128))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46208))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46272))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 63))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) & 63)) + 192))]) + placeholder3[((((((((((((int)blockIdx.x) >> 2) * 92160) + (ax1_inner * 15360)) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + ((((int)threadIdx.x) >> 6) * 1536)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 63)) + 46272))]), 0.000000e+00f);
    }
  }
}


