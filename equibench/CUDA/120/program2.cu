
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[10240];
  __shared__ float placeholder_shared[320];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int yy_inner_init = 0; yy_inner_init < 4; ++yy_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
          Conv2dOutput[(((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 128))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 256))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 384))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 1))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 129))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 257))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 385))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 2))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 130))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 258))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 386))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 3))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 131))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 259))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 64) + (yy_inner_init * 16)) + (xx_outer_inner_init * 8)) + (xx_inner_init * 4)) + 387))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 160; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) / 5120) * 327680) + ((((int)blockIdx.x) >> 2) * 163840)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) % 5120) / 10) * 320)) + (rc_outer_outer * 10)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) % 10)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((((rc_outer_outer * 1280) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((((rc_outer_outer * 1280) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((((rc_outer_outer * 1280) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((((rc_outer_outer * 1280) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 1024))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 5; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
            for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
              for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
                Conv2dOutput[(((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)))] = (Conv2dOutput[(((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 128))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 128))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 256))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 256))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5120))] * placeholder_shared[((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 384))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 384))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5280))] * placeholder_shared[((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 1))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 1))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 129))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 129))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 160))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 257))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 257))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 385))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 385))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5280))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 2))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 2))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 130))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 130))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 160))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 258))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 258))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 386))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 386))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5280))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 3))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 3))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 131))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 131))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 160))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 259))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 259))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
                Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 387))] = (Conv2dOutput[((((((yy_outer_inner * 64) + (yy_inner * 16)) + (xx_outer_inner * 8)) + (xx_inner * 4)) + 387))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2560) + (yy_outer_inner * 1280)) + (yy_inner * 320)) + (((((int)threadIdx.x) & 31) >> 3) * 40)) + (xx_outer_inner * 20)) + (xx_inner * 10)) + (rc_outer_inner * 2)) + rc_inner) + 5280))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[((((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 5) * 32768)) + (ax1_inner * 4096)) + (((((int)threadIdx.x) & 31) >> 3) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 16) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 5) * 32768)) + (ax1_inner * 4096)) + (((((int)threadIdx.x) & 31) >> 3) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 2048))] = max((Conv2dOutput[(((((ax1_inner * 16) + (ax2_inner * 4)) + ax3_inner) + 128))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 5) * 32768)) + (ax1_inner * 4096)) + (((((int)threadIdx.x) & 31) >> 3) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 131072))] = max((Conv2dOutput[(((((ax1_inner * 16) + (ax2_inner * 4)) + ax3_inner) + 256))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 5) * 32768)) + (ax1_inner * 4096)) + (((((int)threadIdx.x) & 31) >> 3) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 133120))] = max((Conv2dOutput[(((((ax1_inner * 16) + (ax2_inner * 4)) + ax3_inner) + 384))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


