
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[768];
  __shared__ float placeholder_shared[384];
  for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
    Conv2dOutput[((yy_inner_init * 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 32))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 64))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 96))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 33))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 65))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 97))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 34))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 66))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 98))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 35))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 67))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 99))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 36))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 68))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 100))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 37))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 69))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 101))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 38))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 70))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 102))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 39))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 71))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 103))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 40))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 72))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 104))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 41))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 73))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 105))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 42))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 74))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 106))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 43))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 75))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 107))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 44))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 76))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 108))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 45))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 77))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 109))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 46))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 78))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 110))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 47))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 79))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 16) + 111))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 96; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) / 3) * 288) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((int)threadIdx.x) + 256) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((int)threadIdx.x) + 512) / 3) * 288) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 384) + ((int)threadIdx.x)))];
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((rc_outer_outer * 384) + ((int)threadIdx.x)) + 256))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
      for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
        Conv2dOutput[((yy_inner * 16))] = (Conv2dOutput[((yy_inner * 16))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 32))] = (Conv2dOutput[(((yy_inner * 16) + 32))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 64))] = (Conv2dOutput[(((yy_inner * 16) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 384))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 96))] = (Conv2dOutput[(((yy_inner * 16) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 384))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 1))] = (Conv2dOutput[(((yy_inner * 16) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 3))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 33))] = (Conv2dOutput[(((yy_inner * 16) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 3))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 65))] = (Conv2dOutput[(((yy_inner * 16) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 387))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 97))] = (Conv2dOutput[(((yy_inner * 16) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 387))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 2))] = (Conv2dOutput[(((yy_inner * 16) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 6))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 34))] = (Conv2dOutput[(((yy_inner * 16) + 34))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 6))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 66))] = (Conv2dOutput[(((yy_inner * 16) + 66))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 390))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 98))] = (Conv2dOutput[(((yy_inner * 16) + 98))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 390))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 3))] = (Conv2dOutput[(((yy_inner * 16) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 9))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 35))] = (Conv2dOutput[(((yy_inner * 16) + 35))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 9))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 67))] = (Conv2dOutput[(((yy_inner * 16) + 67))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 393))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 99))] = (Conv2dOutput[(((yy_inner * 16) + 99))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 393))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 4))] = (Conv2dOutput[(((yy_inner * 16) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 12))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 36))] = (Conv2dOutput[(((yy_inner * 16) + 36))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 12))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 68))] = (Conv2dOutput[(((yy_inner * 16) + 68))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 396))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 100))] = (Conv2dOutput[(((yy_inner * 16) + 100))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 396))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 5))] = (Conv2dOutput[(((yy_inner * 16) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 15))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 37))] = (Conv2dOutput[(((yy_inner * 16) + 37))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 15))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 69))] = (Conv2dOutput[(((yy_inner * 16) + 69))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 399))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 101))] = (Conv2dOutput[(((yy_inner * 16) + 101))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 399))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 6))] = (Conv2dOutput[(((yy_inner * 16) + 6))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 18))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 38))] = (Conv2dOutput[(((yy_inner * 16) + 38))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 18))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 70))] = (Conv2dOutput[(((yy_inner * 16) + 70))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 402))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 102))] = (Conv2dOutput[(((yy_inner * 16) + 102))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 402))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 7))] = (Conv2dOutput[(((yy_inner * 16) + 7))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 21))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 39))] = (Conv2dOutput[(((yy_inner * 16) + 39))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 21))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 71))] = (Conv2dOutput[(((yy_inner * 16) + 71))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 405))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 103))] = (Conv2dOutput[(((yy_inner * 16) + 103))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 405))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 8))] = (Conv2dOutput[(((yy_inner * 16) + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 24))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 40))] = (Conv2dOutput[(((yy_inner * 16) + 40))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 24))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 72))] = (Conv2dOutput[(((yy_inner * 16) + 72))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 408))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 104))] = (Conv2dOutput[(((yy_inner * 16) + 104))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 408))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 9))] = (Conv2dOutput[(((yy_inner * 16) + 9))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 27))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 41))] = (Conv2dOutput[(((yy_inner * 16) + 41))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 27))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 73))] = (Conv2dOutput[(((yy_inner * 16) + 73))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 411))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 105))] = (Conv2dOutput[(((yy_inner * 16) + 105))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 411))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 10))] = (Conv2dOutput[(((yy_inner * 16) + 10))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 30))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 42))] = (Conv2dOutput[(((yy_inner * 16) + 42))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 30))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 74))] = (Conv2dOutput[(((yy_inner * 16) + 74))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 414))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 106))] = (Conv2dOutput[(((yy_inner * 16) + 106))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 414))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 11))] = (Conv2dOutput[(((yy_inner * 16) + 11))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 33))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 43))] = (Conv2dOutput[(((yy_inner * 16) + 43))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 33))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 75))] = (Conv2dOutput[(((yy_inner * 16) + 75))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 417))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 107))] = (Conv2dOutput[(((yy_inner * 16) + 107))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 417))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 12))] = (Conv2dOutput[(((yy_inner * 16) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 36))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 44))] = (Conv2dOutput[(((yy_inner * 16) + 44))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 36))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 76))] = (Conv2dOutput[(((yy_inner * 16) + 76))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 420))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 108))] = (Conv2dOutput[(((yy_inner * 16) + 108))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 420))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 13))] = (Conv2dOutput[(((yy_inner * 16) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 39))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 45))] = (Conv2dOutput[(((yy_inner * 16) + 45))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 39))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 77))] = (Conv2dOutput[(((yy_inner * 16) + 77))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 423))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 109))] = (Conv2dOutput[(((yy_inner * 16) + 109))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 423))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 14))] = (Conv2dOutput[(((yy_inner * 16) + 14))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 42))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 46))] = (Conv2dOutput[(((yy_inner * 16) + 46))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 42))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 78))] = (Conv2dOutput[(((yy_inner * 16) + 78))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 426))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 110))] = (Conv2dOutput[(((yy_inner * 16) + 110))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 426))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 15))] = (Conv2dOutput[(((yy_inner * 16) + 15))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 45))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 47))] = (Conv2dOutput[(((yy_inner * 16) + 47))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 45))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
        Conv2dOutput[(((yy_inner * 16) + 79))] = (Conv2dOutput[(((yy_inner * 16) + 79))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 429))] * placeholder_shared[(((rc_inner * 128) + (((int)threadIdx.x) & 63)))]));
        Conv2dOutput[(((yy_inner * 16) + 111))] = (Conv2dOutput[(((yy_inner * 16) + 111))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 96) + (yy_inner * 48)) + rc_inner) + 429))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) & 63)) + 64))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 16; ++ax2_inner) {
      T_relu[((((((((int)threadIdx.x) >> 6) * 4096) + (ax1_inner * 2048)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 63)))] = max((Conv2dOutput[(((ax1_inner * 16) + ax2_inner))] + placeholder2[((((int)threadIdx.x) & 63))]), 0.000000e+00f);
      T_relu[(((((((((int)threadIdx.x) >> 6) * 4096) + (ax1_inner * 2048)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 63)) + 64))] = max((Conv2dOutput[((((ax1_inner * 16) + ax2_inner) + 32))] + placeholder2[(((((int)threadIdx.x) & 63) + 64))]), 0.000000e+00f);
      T_relu[(((((((((int)threadIdx.x) >> 6) * 4096) + (ax1_inner * 2048)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 63)) + 16384))] = max((Conv2dOutput[((((ax1_inner * 16) + ax2_inner) + 64))] + placeholder2[((((int)threadIdx.x) & 63))]), 0.000000e+00f);
      T_relu[(((((((((int)threadIdx.x) >> 6) * 4096) + (ax1_inner * 2048)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 63)) + 16448))] = max((Conv2dOutput[((((ax1_inner * 16) + ax2_inner) + 96))] + placeholder2[(((((int)threadIdx.x) & 63) + 64))]), 0.000000e+00f);
    }
  }
}


