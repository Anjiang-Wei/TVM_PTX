
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[64];
  __shared__ float PaddedInput_shared[2176];
  __shared__ float placeholder_shared[1088];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + (((((int)threadIdx.x) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + (((int)threadIdx.x) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)threadIdx.x) + 128) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 60) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 9) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((((int)threadIdx.x) + 256) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 52) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 1) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((((int)threadIdx.x) + 384) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 44) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 10) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((((int)threadIdx.x) + 512) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 36) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 2) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[((((((((((int)threadIdx.x) + 640) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 28) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 11) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((((int)threadIdx.x) + 768) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 20) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 3) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 896))] = placeholder[((((((((((int)threadIdx.x) + 896) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 12) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 12) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[((((((((((int)threadIdx.x) + 1024) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 4) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 4) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1152))] = placeholder[((((((((((int)threadIdx.x) + 1152) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 64) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 13) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[((((((((((int)threadIdx.x) + 1280) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 56) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 5) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1408))] = placeholder[((((((((((int)threadIdx.x) + 1408) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 48) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 14) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((((int)threadIdx.x) + 1536) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 40) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 6) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1664))] = placeholder[((((((((((int)threadIdx.x) + 1664) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 32) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 15) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1792))] = placeholder[((((((((((int)threadIdx.x) + 1792) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 24) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 7) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[((((((((((int)threadIdx.x) + 1920) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 16) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 16) % 17)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[((((((((((int)threadIdx.x) + 2048) / 68) * 4352) + ((((int)blockIdx.x) >> 1) * 2176)) + ((((((int)threadIdx.x) + 8) % 68) / 17) * 544)) + (rc_outer_outer * 17)) + ((((int)threadIdx.x) + 8) % 17)))];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[((((int)threadIdx.x) * 34))] = placeholder1[(((((rc_outer_outer * 2176) + (((((int)threadIdx.x) * 34) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) * 34) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 1))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 1) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 1) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 2))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 2) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 2) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 3))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 3) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 3) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 4))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 4) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 4) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 5))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 5) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 5) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 6))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 6) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 6) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 7))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 7) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 7) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 8))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 8) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 8) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 9))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 9) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 9) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 10))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 10) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 10) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 11))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 11) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 11) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 12))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 12) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 12) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 13))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 13) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 13) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 14))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 14) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 14) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 15))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 15) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 15) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 16))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 16) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 16) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 17))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 17) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 17) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 18))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 18) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 18) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 19))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 19) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 19) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 20))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 20) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 20) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 21))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 21) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 21) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 22))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 22) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 22) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 23))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 23) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 23) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 24))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 24) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 24) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 25))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 25) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 25) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 26))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 26) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 26) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 27))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 27) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 27) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 28))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 28) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 28) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 29))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 29) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 29) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 30))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 30) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 30) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 31))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 31) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 31) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 32))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 32) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 32) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 34) + 33))] = placeholder1[(((((rc_outer_outer * 2176) + ((((((int)threadIdx.x) * 34) + 33) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 34) + 33) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 17; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 17))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 17))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 17))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 17))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 68))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 68))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 68))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 68))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 85))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 85))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 85))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 85))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 136))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 136))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 136))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 136))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 153))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 153))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 153))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 153))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 204))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 204))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 204))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 204))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 221))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 221))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 221))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 221))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 544))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 544))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 544))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 544))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 561))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 561))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 561))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 561))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 612))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 612))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 612))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 612))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 629))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 629))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 629))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 629))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 680))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 697))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 697))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 697))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 697))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 748))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 748))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 748))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 748))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 765))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 765))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 1))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 765))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 2))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 1088) + (((((int)threadIdx.x) & 63) >> 5) * 272)) + (((((int)threadIdx.x) & 31) >> 4) * 34)) + rc_outer_inner) + 765))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 4)) + 3))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
          T_relu[((((((((((((((int)threadIdx.x) >> 6) * 16384) + (ax0_inner * 8192)) + (((((int)threadIdx.x) & 63) >> 5) * 4096)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 31) >> 4) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 32) + (ax1_inner * 8)) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


