
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[64];
  __shared__ float PaddedInput_shared[3072];
  __shared__ float placeholder_shared[96];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    Conv2dOutput[((nn_outer_inner_init * 4))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 32))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 48))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 33))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 49))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 34))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 50))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 35))] = 0.000000e+00f;
    Conv2dOutput[(((nn_outer_inner_init * 4) + 51))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 2) * 24576) + ((((int)threadIdx.x) / 3) * 96)) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((((int)threadIdx.x) + 512) / 768) * 393216) + ((((int)blockIdx.x) >> 2) * 24576)) + ((((((int)threadIdx.x) + 512) % 768) / 3) * 96)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[((((((((((int)threadIdx.x) + 1024) / 768) * 393216) + ((((int)blockIdx.x) >> 2) * 24576)) + (((((int)threadIdx.x) + 256) / 3) * 96)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 24576) + ((((int)threadIdx.x) / 3) * 96)) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 786432))];
    PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[((((((((((int)threadIdx.x) + 2048) / 768) * 393216) + ((((int)blockIdx.x) >> 2) * 24576)) + ((((((int)threadIdx.x) + 512) % 768) / 3) * 96)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[((((((((((int)threadIdx.x) + 2560) / 768) * 393216) + ((((int)blockIdx.x) >> 2) * 24576)) + (((((int)threadIdx.x) + 256) / 3) * 96)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[((((int)threadIdx.x) * 3))] = placeholder1[(((((rc_outer_outer * 384) + (((((int)threadIdx.x) * 3) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) * 3) & 31)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder1[(((((rc_outer_outer * 384) + ((((((int)threadIdx.x) * 3) + 1) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 3) + 1) & 31)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder1[(((((rc_outer_outer * 384) + ((((((int)threadIdx.x) * 3) + 2) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 3) + 2) & 31)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
        Conv2dOutput[((nn_outer_inner * 4))] = (Conv2dOutput[((nn_outer_inner * 4))] + (PaddedInput_shared[((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 16))] = (Conv2dOutput[(((nn_outer_inner * 4) + 16))] + (PaddedInput_shared[((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 32))] = (Conv2dOutput[(((nn_outer_inner * 4) + 32))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 384))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 48))] = (Conv2dOutput[(((nn_outer_inner * 4) + 48))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 384))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 1))] = (Conv2dOutput[(((nn_outer_inner * 4) + 1))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 3))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 17))] = (Conv2dOutput[(((nn_outer_inner * 4) + 17))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 3))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 33))] = (Conv2dOutput[(((nn_outer_inner * 4) + 33))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 387))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 49))] = (Conv2dOutput[(((nn_outer_inner * 4) + 49))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 387))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 2))] = (Conv2dOutput[(((nn_outer_inner * 4) + 2))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 6))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 18))] = (Conv2dOutput[(((nn_outer_inner * 4) + 18))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 6))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 34))] = (Conv2dOutput[(((nn_outer_inner * 4) + 34))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 390))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 50))] = (Conv2dOutput[(((nn_outer_inner * 4) + 50))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 390))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 3))] = (Conv2dOutput[(((nn_outer_inner * 4) + 3))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 9))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 19))] = (Conv2dOutput[(((nn_outer_inner * 4) + 19))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 9))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 35))] = (Conv2dOutput[(((nn_outer_inner * 4) + 35))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 393))] * placeholder_shared[(((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[(((nn_outer_inner * 4) + 51))] = (Conv2dOutput[(((nn_outer_inner * 4) + 51))] + (PaddedInput_shared[(((((nn_outer_inner * 768) + ((((int)threadIdx.x) >> 4) * 12)) + rc_outer_inner) + 393))] * placeholder_shared[((((rc_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      T_relu[(((((((ax0_inner * 524288) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 4) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 15)))] = max((Conv2dOutput[(((ax0_inner * 4) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 15)))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 524288) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 4) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 15)) + 16))] = max((Conv2dOutput[((((ax0_inner * 4) + ax2_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 15)) + 16))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 524288) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 4) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 15)) + 16384))] = max((Conv2dOutput[((((ax0_inner * 4) + ax2_inner) + 32))] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 15)))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 524288) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 4) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 15)) + 16400))] = max((Conv2dOutput[((((ax0_inner * 4) + ax2_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 15)) + 16))]), 0.000000e+00f);
    }
  }
}


