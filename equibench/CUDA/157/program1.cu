
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[64];
  __shared__ float PaddedInput_shared[4096];
  __shared__ float placeholder_shared[32];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 8; ++xx_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 16) + (xx_outer_inner_init * 8)) + xx_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 16) + (xx_outer_inner_init * 8)) + xx_inner_init) + 32))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 52; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 128; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 212992) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1664)) + ((((int)threadIdx.x) >> 3) * 416)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 32) {
        if (((int)threadIdx.x) < 16) {
          placeholder_shared[(((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
          for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
            for (int xx_inner = 0; xx_inner < 8; ++xx_inner) {
              Conv2dOutput[((((nn_outer_inner * 16) + (xx_outer_inner * 8)) + xx_inner))] = (Conv2dOutput[((((nn_outer_inner * 16) + (xx_outer_inner * 8)) + xx_inner))] + (PaddedInput_shared[(((((((nn_outer_inner * 2048) + ((((int)threadIdx.x) >> 2) * 128)) + (xx_outer_inner * 64)) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 8) + (rc_inner * 4)) + (((int)threadIdx.x) & 3)))]));
              Conv2dOutput[(((((nn_outer_inner * 16) + (xx_outer_inner * 8)) + xx_inner) + 32))] = (Conv2dOutput[(((((nn_outer_inner * 16) + (xx_outer_inner * 8)) + xx_inner) + 32))] + (PaddedInput_shared[((((((((nn_outer_inner * 2048) + ((((int)threadIdx.x) >> 2) * 128)) + (xx_outer_inner * 64)) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 1024))] * placeholder_shared[((((rc_outer_inner * 8) + (rc_inner * 4)) + (((int)threadIdx.x) & 3)))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 16; ++ax2_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 5) * 65536) + (ax0_inner * 32768)) + ((((int)threadIdx.x) >> 2) * 2048)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)))] = max((Conv2dOutput[(((ax0_inner * 16) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 31) * 4) + (((int)threadIdx.x) & 3)))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 5) * 65536) + (ax0_inner * 32768)) + ((((int)threadIdx.x) >> 2) * 2048)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)) + 16384))] = max((Conv2dOutput[((((ax0_inner * 16) + ax2_inner) + 32))] + placeholder2[((((((int)blockIdx.x) & 31) * 4) + (((int)threadIdx.x) & 3)))]), 0.000000e+00f);
    }
  }
}


