
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[384];
  __shared__ float placeholder_shared[768];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 64; ++ff_outer_inner_init) {
    Conv2dOutput[(ff_outer_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 64))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    ((float3*)(PaddedInput_shared + ((((int)threadIdx.x) * 3))))[0] = ((float3*)(placeholder + ((((((((((int)blockIdx.x) >> 2) * 24576) + ((((int)threadIdx.x) >> 5) * 6144)) + ((((int)blockIdx.x) & 3) * 1536)) + (((((int)threadIdx.x) & 31) >> 1) * 96)) + (rc_outer_outer * 6)) + ((((int)threadIdx.x) & 1) * 3)))))[0];
    ((float3*)(PaddedInput_shared + (((((int)threadIdx.x) * 3) + 192))))[0] = ((float3*)(placeholder + (((((((((((int)blockIdx.x) >> 2) * 24576) + ((((int)threadIdx.x) >> 5) * 6144)) + ((((int)blockIdx.x) & 3) * 1536)) + (((((int)threadIdx.x) & 31) >> 1) * 96)) + (rc_outer_outer * 6)) + ((((int)threadIdx.x) & 1) * 3)) + 12288))))[0];
    placeholder_shared[((((int)threadIdx.x) * 8))] = placeholder1[(((rc_outer_outer * 768) + (((int)threadIdx.x) * 8)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 1)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 2)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 3)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 4)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 5)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 6)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder1[(((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 7)))];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 512))] = placeholder1[((((rc_outer_outer * 768) + (((int)threadIdx.x) * 8)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 513))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 1)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 514))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 2)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 515))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 3)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 516))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 4)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 517))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 5)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 518))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 6)) + 512))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 519))] = placeholder1[((((rc_outer_outer * 768) + ((((int)threadIdx.x) * 8) + 7)) + 512))];
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 64; ++ff_outer_inner) {
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((int)threadIdx.x) * 6))] * placeholder_shared[(ff_outer_inner)]));
      Conv2dOutput[((ff_outer_inner + 64))] = (Conv2dOutput[((ff_outer_inner + 64))] + (PaddedInput_shared[((((int)threadIdx.x) * 6))] * placeholder_shared[((ff_outer_inner + 64))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 1))] * placeholder_shared[((ff_outer_inner + 128))]));
      Conv2dOutput[((ff_outer_inner + 64))] = (Conv2dOutput[((ff_outer_inner + 64))] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 1))] * placeholder_shared[((ff_outer_inner + 192))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 2))] * placeholder_shared[((ff_outer_inner + 256))]));
      Conv2dOutput[((ff_outer_inner + 64))] = (Conv2dOutput[((ff_outer_inner + 64))] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 2))] * placeholder_shared[((ff_outer_inner + 320))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 3))] * placeholder_shared[((ff_outer_inner + 384))]));
      Conv2dOutput[((ff_outer_inner + 64))] = (Conv2dOutput[((ff_outer_inner + 64))] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 3))] * placeholder_shared[((ff_outer_inner + 448))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 4))] * placeholder_shared[((ff_outer_inner + 512))]));
      Conv2dOutput[((ff_outer_inner + 64))] = (Conv2dOutput[((ff_outer_inner + 64))] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 4))] * placeholder_shared[((ff_outer_inner + 576))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 5))] * placeholder_shared[((ff_outer_inner + 640))]));
      Conv2dOutput[((ff_outer_inner + 64))] = (Conv2dOutput[((ff_outer_inner + 64))] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + 5))] * placeholder_shared[((ff_outer_inner + 704))]));
    }
  }
  for (int ax3_inner = 0; ax3_inner < 64; ++ax3_inner) {
    T_relu[(((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[(ax3_inner)]), 0.000000e+00f);
    T_relu[((((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) & 15) * 128)) + ax3_inner) + 64))] = max((Conv2dOutput[((ax3_inner + 64))] + placeholder2[((ax3_inner + 64))]), 0.000000e+00f);
  }
}


