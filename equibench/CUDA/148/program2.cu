
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[240];
  __shared__ float PaddedInput_shared[90];
  __shared__ float placeholder_shared[128];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 5; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
        for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
          Conv2dOutput[(((((yy_inner_init * 12) + (xx_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 12) + (xx_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 60))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 12) + (xx_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 120))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 12) + (xx_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 180))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 10; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 5) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 1)) < 45) {
        if (((((int)threadIdx.x) * 10) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 90) {
          if (((int)threadIdx.x) < 9) {
            PaddedInput_shared[(((((int)threadIdx.x) * 10) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((int)blockIdx.x) / 20) * 230400) + ((((((int)threadIdx.x) * 5) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 1)) / 3) * 15360)) + (((((int)blockIdx.x) % 20) >> 2) * 3072)) + ((((((int)threadIdx.x) * 5) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 1)) % 3) * 1024)) + (rc_outer_outer * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s & 1)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 128) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 32) {
            placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[(((((rc_outer_outer * 512) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) & 63)))];
          }
        }
      }
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 5; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
            for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
              Conv2dOutput[(((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner))] = (Conv2dOutput[(((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 30) + (yy_inner * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 60))] = (Conv2dOutput[((((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 60))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 30) + (yy_inner * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 16))]));
              Conv2dOutput[((((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 120))] = (Conv2dOutput[((((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 120))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 30) + (yy_inner * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 32))]));
              Conv2dOutput[((((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 180))] = (Conv2dOutput[((((((yy_inner * 12) + (xx_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 180))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 30) + (yy_inner * 6)) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 48))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) >> 2) * 19200)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) % 20) >> 2) * 768)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 12) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) >> 2) * 19200)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) % 20) >> 2) * 768)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))] = max((Conv2dOutput[(((((ax1_inner * 12) + (ax2_inner * 4)) + ax3_inner) + 60))] + placeholder2[((((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) >> 2) * 19200)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) % 20) >> 2) * 768)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 32))] = max((Conv2dOutput[(((((ax1_inner * 12) + (ax2_inner * 4)) + ax3_inner) + 120))] + placeholder2[((((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 32))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) >> 2) * 19200)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) % 20) >> 2) * 768)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 48))] = max((Conv2dOutput[(((((ax1_inner * 12) + (ax2_inner * 4)) + ax3_inner) + 180))] + placeholder2[((((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 48))]), 0.000000e+00f);
      }
    }
  }
}


