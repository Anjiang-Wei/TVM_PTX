
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[512];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 5) * 16384) + ((((int)threadIdx.x) >> 5) * 4096)) + (((((int)blockIdx.x) & 31) >> 1) * 256)) + (((((int)threadIdx.x) & 31) >> 3) * 64)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((((rc_outer_outer * 1024) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((((rc_outer_outer * 1024) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((((rc_outer_outer * 1024) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 768))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 32))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 33))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 32))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 33))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 96))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 97))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 64))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 96))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 8) + (rc_outer_inner * 2)) + 97))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 31) * 2)) + 65))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 5) * 32768) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 31) >> 1) * 512)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


