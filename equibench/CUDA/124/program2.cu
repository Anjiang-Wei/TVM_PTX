
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[896];
  __shared__ float PaddedInput_shared[100352];
  __shared__ float placeholder_shared[1024];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 8; ++ff_outer_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 14; ++yy_inner_init) {
          Conv2dOutput[(((((nn_outer_inner_init * 448) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + ff_outer_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 448) + (yy_inner_init * 32)) + (xx_outer_inner_init * 16)) + ff_outer_inner_init) + 8))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 224; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + ((int)threadIdx.x)))] = placeholder[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1792) + ((((int)threadIdx.x) >> 4) * 64)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 896))];
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 6) * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) & 63)) + 1792))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 8; ++ff_outer_inner) {
            for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              for (int yy_inner = 0; yy_inner < 14; ++yy_inner) {
                Conv2dOutput[(((((nn_outer_inner * 448) + (yy_inner * 32)) + (xx_outer_inner * 16)) + ff_outer_inner))] = (Conv2dOutput[(((((nn_outer_inner * 448) + (yy_inner * 32)) + (xx_outer_inner * 16)) + ff_outer_inner))] + (PaddedInput_shared[((((((((nn_outer_inner * 50176) + ((((int)threadIdx.x) / 112) * 12544)) + (yy_inner * 896)) + (((((int)threadIdx.x) % 112) >> 3) * 64)) + (xx_outer_inner * 32)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ff_outer_inner))]));
                Conv2dOutput[((((((nn_outer_inner * 448) + (yy_inner * 32)) + (xx_outer_inner * 16)) + ff_outer_inner) + 8))] = (Conv2dOutput[((((((nn_outer_inner * 448) + (yy_inner * 32)) + (xx_outer_inner * 16)) + ff_outer_inner) + 8))] + (PaddedInput_shared[(((((((((nn_outer_inner * 50176) + ((((int)threadIdx.x) / 112) * 12544)) + (yy_inner * 896)) + (((((int)threadIdx.x) % 112) >> 3) * 64)) + (xx_outer_inner * 32)) + (rc_outer_inner * 2)) + rc_inner) + 16))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ff_outer_inner))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 14; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
          T_relu[(((((((((ax0_inner * 401408) + ((((int)threadIdx.x) / 112) * 100352)) + (ax1_inner * 7168)) + (((((int)threadIdx.x) % 112) >> 3) * 512)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 448) + (ax1_inner * 32)) + (ax2_inner * 8)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 64) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


