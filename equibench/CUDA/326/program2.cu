
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 672))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 672))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[16];
  __shared__ float PaddedInput_shared[5880];
  __shared__ float placeholder_shared[280];
  DepthwiseConv2d_local[(0)] = 0.000000e+00f;
  DepthwiseConv2d_local[(8)] = 0.000000e+00f;
  DepthwiseConv2d_local[(1)] = 0.000000e+00f;
  DepthwiseConv2d_local[(9)] = 0.000000e+00f;
  DepthwiseConv2d_local[(2)] = 0.000000e+00f;
  DepthwiseConv2d_local[(10)] = 0.000000e+00f;
  DepthwiseConv2d_local[(3)] = 0.000000e+00f;
  DepthwiseConv2d_local[(11)] = 0.000000e+00f;
  DepthwiseConv2d_local[(4)] = 0.000000e+00f;
  DepthwiseConv2d_local[(12)] = 0.000000e+00f;
  DepthwiseConv2d_local[(5)] = 0.000000e+00f;
  DepthwiseConv2d_local[(13)] = 0.000000e+00f;
  DepthwiseConv2d_local[(6)] = 0.000000e+00f;
  DepthwiseConv2d_local[(14)] = 0.000000e+00f;
  DepthwiseConv2d_local[(7)] = 0.000000e+00f;
  DepthwiseConv2d_local[(15)] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 5; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 105; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = (((((2 <= ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 7) + di_outer_outer)) && (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 7) + di_outer_outer) < 17)) && (2 <= (((((int)blockIdx.x) / 12) * 4) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 7)))) && ((((((int)blockIdx.x) / 12) * 4) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 7)) < 17)) ? placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 7) * 10080) + (di_outer_outer * 10080)) + ((((int)blockIdx.x) / 12) * 2688)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 7) * 672)) + ((((int)blockIdx.x) % 12) * 56)) + ((int)threadIdx.x)) - 21504))] : 0.000000e+00f);
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((di_outer_outer * 3360) + ((((int)blockIdx.x) % 12) * 56)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((di_outer_outer * 3360) + ((((int)blockIdx.x) % 12) * 56)) + ((int)threadIdx.x)) + 672))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((di_outer_outer * 3360) + ((((int)blockIdx.x) % 12) * 56)) + ((int)threadIdx.x)) + 1344))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[(((((di_outer_outer * 3360) + ((((int)blockIdx.x) % 12) * 56)) + ((int)threadIdx.x)) + 2016))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[(((((di_outer_outer * 3360) + ((((int)blockIdx.x) % 12) * 56)) + ((int)threadIdx.x)) + 2688))];
    __syncthreads();
    for (int dj_outer_inner = 0; dj_outer_inner < 5; ++dj_outer_inner) {
      DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)))] * placeholder_shared[(((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)))]));
      DepthwiseConv2d_local[(8)] = (DepthwiseConv2d_local[(8)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3136))] * placeholder_shared[(((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)))]));
      DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 1))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 1))]));
      DepthwiseConv2d_local[(9)] = (DepthwiseConv2d_local[(9)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3137))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 1))]));
      DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 2))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 2))]));
      DepthwiseConv2d_local[(10)] = (DepthwiseConv2d_local[(10)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3138))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 2))]));
      DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 3))]));
      DepthwiseConv2d_local[(11)] = (DepthwiseConv2d_local[(11)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3139))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 3))]));
      DepthwiseConv2d_local[(4)] = (DepthwiseConv2d_local[(4)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 784))] * placeholder_shared[(((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)))]));
      DepthwiseConv2d_local[(12)] = (DepthwiseConv2d_local[(12)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3920))] * placeholder_shared[(((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)))]));
      DepthwiseConv2d_local[(5)] = (DepthwiseConv2d_local[(5)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 785))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 1))]));
      DepthwiseConv2d_local[(13)] = (DepthwiseConv2d_local[(13)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3921))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 1))]));
      DepthwiseConv2d_local[(6)] = (DepthwiseConv2d_local[(6)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 786))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 2))]));
      DepthwiseConv2d_local[(14)] = (DepthwiseConv2d_local[(14)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3922))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 2))]));
      DepthwiseConv2d_local[(7)] = (DepthwiseConv2d_local[(7)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 787))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 3))]));
      DepthwiseConv2d_local[(15)] = (DepthwiseConv2d_local[(15)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 1568) + (((((int)threadIdx.x) % 28) / 14) * 112)) + (dj_outer_inner * 56)) + ((((int)threadIdx.x) % 14) * 4)) + 3923))] * placeholder_shared[((((dj_outer_inner * 56) + ((((int)threadIdx.x) % 14) * 4)) + 3))]));
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int c_inner = 0; c_inner < 4; ++c_inner) {
      DepthwiseConv2d[(((((((((((int)threadIdx.x) / 28) * 10752) + (i_inner * 5376)) + ((((int)blockIdx.x) / 12) * 1344)) + (((((int)threadIdx.x) % 28) / 14) * 672)) + ((((int)blockIdx.x) % 12) * 56)) + ((((int)threadIdx.x) % 14) * 4)) + c_inner))] = DepthwiseConv2d_local[(((i_inner * 4) + c_inner))];
      DepthwiseConv2d[((((((((((((int)threadIdx.x) / 28) * 10752) + (i_inner * 5376)) + ((((int)blockIdx.x) / 12) * 1344)) + (((((int)threadIdx.x) % 28) / 14) * 672)) + ((((int)blockIdx.x) % 12) * 56)) + ((((int)threadIdx.x) % 14) * 4)) + c_inner) + 21504))] = DepthwiseConv2d_local[((((i_inner * 4) + c_inner) + 8))];
    }
  }
}


