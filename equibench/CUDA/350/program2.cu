
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float placeholder_d_shared[2048];
  __shared__ float placeholder_shared[4096];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 16; ++k_outer_outer) {
    __syncthreads();
    placeholder_d_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 8) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (((((int)threadIdx.x) & 255) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    placeholder_d_shared[((((int)threadIdx.x) + 1024))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 131072) + ((((int)threadIdx.x) >> 8) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (((((int)threadIdx.x) & 255) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 65536))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    placeholder_shared[((((int)threadIdx.x) + 1024))] = placeholder1[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 16384))];
    placeholder_shared[((((int)threadIdx.x) + 2048))] = placeholder1[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 32768))];
    placeholder_shared[((((int)threadIdx.x) + 3072))] = placeholder1[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 49152))];
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int k_inner = 0; k_inner < 8; ++k_inner) {
        compute_local[((i_c_outer_inner * 4))] = (compute_local[((i_c_outer_inner * 4))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 128) + (i_c_outer_inner * 32)) + k_inner))] * placeholder_shared[(((((((int)threadIdx.x) >> 7) * 512) + ((((int)threadIdx.x) & 63) * 8)) + k_inner))]));
        compute_local[(((i_c_outer_inner * 4) + 1))] = (compute_local[(((i_c_outer_inner * 4) + 1))] + (placeholder_d_shared[((((((((int)threadIdx.x) >> 6) * 128) + (i_c_outer_inner * 32)) + k_inner) + 8))] * placeholder_shared[(((((((int)threadIdx.x) >> 7) * 512) + ((((int)threadIdx.x) & 63) * 8)) + k_inner))]));
        compute_local[(((i_c_outer_inner * 4) + 2))] = (compute_local[(((i_c_outer_inner * 4) + 2))] + (placeholder_d_shared[((((((((int)threadIdx.x) >> 6) * 128) + (i_c_outer_inner * 32)) + k_inner) + 16))] * placeholder_shared[(((((((int)threadIdx.x) >> 7) * 512) + ((((int)threadIdx.x) & 63) * 8)) + k_inner))]));
        compute_local[(((i_c_outer_inner * 4) + 3))] = (compute_local[(((i_c_outer_inner * 4) + 3))] + (placeholder_d_shared[((((((((int)threadIdx.x) >> 6) * 128) + (i_c_outer_inner * 32)) + k_inner) + 24))] * placeholder_shared[(((((((int)threadIdx.x) >> 7) * 512) + ((((int)threadIdx.x) & 63) * 8)) + k_inner))]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    compute[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 7) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) & 127) >> 6) * 1024)) + (i_inner * 64)) + (((int)threadIdx.x) & 63)))] = compute_local[(i_inner)];
  }
}


