
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[224];
  __shared__ float PaddedInput_shared[896];
  __shared__ float placeholder_shared[896];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 4; ++nn_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 14; ++ff_inner_init) {
        Conv2dOutput[((((nn_inner_init * 28) + (ff_outer_inner_init * 14)) + ff_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 28) + (ff_outer_inner_init * 14)) + ff_inner_init) + 112))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 84; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 56))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 56) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 112))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 65856))];
    PaddedInput_shared[((((int)threadIdx.x) + 168))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 168) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 131712))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 280) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 336))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 197568))];
    PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 392) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 263424))];
    PaddedInput_shared[((((int)threadIdx.x) + 504))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 504) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 560))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 329280))];
    PaddedInput_shared[((((int)threadIdx.x) + 616))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 616) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 672))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 395136))];
    PaddedInput_shared[((((int)threadIdx.x) + 728))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 728) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 784))] = placeholder[(((((((((((int)blockIdx.x) / 7) * 526848) + ((((int)threadIdx.x) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + (((((int)threadIdx.x) & 15) >> 3) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 460992))];
    PaddedInput_shared[((((int)threadIdx.x) + 840))] = placeholder[((((((((((int)blockIdx.x) / 7) * 526848) + (((((int)threadIdx.x) + 840) >> 4) * 9408)) + ((((int)blockIdx.x) % 7) * 1344)) + ((((((int)threadIdx.x) >> 3) + 1) & 1) * 672)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 896) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 56))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 112))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 168))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 224))];
    placeholder_shared[((((int)threadIdx.x) + 280))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 280))];
    placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 336))];
    placeholder_shared[((((int)threadIdx.x) + 392))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 392))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 448))];
    placeholder_shared[((((int)threadIdx.x) + 504))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 504))];
    placeholder_shared[((((int)threadIdx.x) + 560))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 560))];
    placeholder_shared[((((int)threadIdx.x) + 616))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 616))];
    placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 672))];
    placeholder_shared[((((int)threadIdx.x) + 728))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 728))];
    placeholder_shared[((((int)threadIdx.x) + 784))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 784))];
    placeholder_shared[((((int)threadIdx.x) + 840))] = placeholder1[((((rc_outer_outer * 896) + ((int)threadIdx.x)) + 840))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int nn_inner = 0; nn_inner < 4; ++nn_inner) {
          for (int ff_inner = 0; ff_inner < 14; ++ff_inner) {
            Conv2dOutput[((((nn_inner * 28) + (ff_outer_inner * 14)) + ff_inner))] = (Conv2dOutput[((((nn_inner * 28) + (ff_outer_inner * 14)) + ff_inner))] + (PaddedInput_shared[((((nn_inner * 224) + ((((int)threadIdx.x) >> 2) * 16)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 112) + ((((int)threadIdx.x) & 3) * 28)) + (ff_outer_inner * 14)) + ff_inner))]));
            Conv2dOutput[(((((nn_inner * 28) + (ff_outer_inner * 14)) + ff_inner) + 112))] = (Conv2dOutput[(((((nn_inner * 28) + (ff_outer_inner * 14)) + ff_inner) + 112))] + (PaddedInput_shared[(((((nn_inner * 224) + ((((int)threadIdx.x) >> 2) * 16)) + rc_outer_inner) + 8))] * placeholder_shared[(((((rc_outer_inner * 112) + ((((int)threadIdx.x) & 3) * 28)) + (ff_outer_inner * 14)) + ff_inner))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax3_inner = 0; ax3_inner < 28; ++ax3_inner) {
      T_add[((((((((((int)blockIdx.x) / 7) * 87808) + (ax0_inner * 21952)) + ((((int)threadIdx.x) >> 2) * 1568)) + ((((int)blockIdx.x) % 7) * 224)) + ((((int)threadIdx.x) & 3) * 28)) + ax3_inner))] = ((Conv2dOutput[(((ax0_inner * 28) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 3) * 28) + ax3_inner))]) + placeholder3[((((((((((int)blockIdx.x) / 7) * 87808) + (ax0_inner * 21952)) + ((((int)threadIdx.x) >> 2) * 1568)) + ((((int)blockIdx.x) % 7) * 224)) + ((((int)threadIdx.x) & 3) * 28)) + ax3_inner))]);
      T_add[(((((((((((int)blockIdx.x) / 7) * 87808) + (ax0_inner * 21952)) + ((((int)threadIdx.x) >> 2) * 1568)) + ((((int)blockIdx.x) % 7) * 224)) + ((((int)threadIdx.x) & 3) * 28)) + ax3_inner) + 112))] = ((Conv2dOutput[((((ax0_inner * 28) + ax3_inner) + 112))] + placeholder2[((((((int)threadIdx.x) & 3) * 28) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) / 7) * 87808) + (ax0_inner * 21952)) + ((((int)threadIdx.x) >> 2) * 1568)) + ((((int)blockIdx.x) % 7) * 224)) + ((((int)threadIdx.x) & 3) * 28)) + ax3_inner) + 112))]);
    }
  }
}


