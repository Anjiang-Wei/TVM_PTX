
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[16];
  __shared__ float placeholder_d_shared[1024];
  __shared__ float placeholder_shared[4096];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  T_dense[(4)] = 0.000000e+00f;
  T_dense[(5)] = 0.000000e+00f;
  T_dense[(6)] = 0.000000e+00f;
  T_dense[(7)] = 0.000000e+00f;
  T_dense[(8)] = 0.000000e+00f;
  T_dense[(9)] = 0.000000e+00f;
  T_dense[(10)] = 0.000000e+00f;
  T_dense[(11)] = 0.000000e+00f;
  T_dense[(12)] = 0.000000e+00f;
  T_dense[(13)] = 0.000000e+00f;
  T_dense[(14)] = 0.000000e+00f;
  T_dense[(15)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 1024; ++ax0_ax1_fused_outer_outer) {
      placeholder_d_shared[(ax0_ax1_fused_outer_outer)] = placeholder[(((((ax0_ax1_fused_outer_outer >> 9) * 1024) + (k_outer_outer * 512)) + (ax0_ax1_fused_outer_outer & 511)))];
    }
    for (int ax0_ax1_fused_outer_outer1 = 0; ax0_ax1_fused_outer_outer1 < 1024; ++ax0_ax1_fused_outer_outer1) {
      ((float4*)(placeholder_shared + ((ax0_ax1_fused_outer_outer1 * 4))))[0] = ((float4*)(placeholder1 + (((((((int)blockIdx.x) * 8192) + ((ax0_ax1_fused_outer_outer1 >> 7) * 1024)) + (k_outer_outer * 512)) + ((ax0_ax1_fused_outer_outer1 & 127) * 4)))))[0];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 64; ++k_outer_inner) {
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[((k_outer_inner * 8))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 512))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 1))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 513))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 2))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 514))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 3))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 515))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 4))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 516))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 5))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 517))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 6))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 518))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 7))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 519))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 1024))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 1536))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 1025))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 1537))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 1026))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 1538))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 1027))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 1539))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 1028))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 1540))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 1029))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 1541))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 1030))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 1542))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 1031))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 1543))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 2048))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 2560))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 2049))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 2561))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 2050))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 2562))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 2051))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 2563))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 2052))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 2564))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 2053))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 2565))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 2054))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 2566))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 2055))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 2567))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 3072))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[((k_outer_inner * 8))] * placeholder_shared[(((k_outer_inner * 8) + 3584))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 3073))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 1))] * placeholder_shared[(((k_outer_inner * 8) + 3585))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 3074))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 2))] * placeholder_shared[(((k_outer_inner * 8) + 3586))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 3075))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 3))] * placeholder_shared[(((k_outer_inner * 8) + 3587))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 3076))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 4))] * placeholder_shared[(((k_outer_inner * 8) + 3588))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 3077))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 5))] * placeholder_shared[(((k_outer_inner * 8) + 3589))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 3078))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 6))] * placeholder_shared[(((k_outer_inner * 8) + 3590))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 3079))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((k_outer_inner * 8) + 7))] * placeholder_shared[(((k_outer_inner * 8) + 3591))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[((k_outer_inner * 8))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 512))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 1))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 513))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 2))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 514))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 3))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 515))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 4))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 516))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 5))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 517))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 6))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 518))]));
      T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 7))]));
      T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 519))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 1024))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 1536))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 1025))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 1537))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 1026))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 1538))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 1027))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 1539))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 1028))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 1540))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 1029))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 1541))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 1030))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 1542))]));
      T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 1031))]));
      T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 1543))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 2048))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 2560))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 2049))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 2561))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 2050))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 2562))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 2051))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 2563))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 2052))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 2564))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 2053))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 2565))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 2054))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 2566))]));
      T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 2055))]));
      T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 2567))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 3072))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 512))] * placeholder_shared[(((k_outer_inner * 8) + 3584))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 3073))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 513))] * placeholder_shared[(((k_outer_inner * 8) + 3585))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 3074))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 514))] * placeholder_shared[(((k_outer_inner * 8) + 3586))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 3075))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 515))] * placeholder_shared[(((k_outer_inner * 8) + 3587))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 3076))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 516))] * placeholder_shared[(((k_outer_inner * 8) + 3588))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 3077))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 517))] * placeholder_shared[(((k_outer_inner * 8) + 3589))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 3078))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 518))] * placeholder_shared[(((k_outer_inner * 8) + 3590))]));
      T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 3079))]));
      T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(((k_outer_inner * 8) + 519))] * placeholder_shared[(((k_outer_inner * 8) + 3591))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      T_add[((((ax0_inner * 1000) + (((int)blockIdx.x) * 8)) + ax1_inner))] = (T_dense[(((ax0_inner * 8) + ax1_inner))] + placeholder2[(((((int)blockIdx.x) * 8) + ax1_inner))]);
    }
  }
}


