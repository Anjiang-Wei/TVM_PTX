
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[64];
  __shared__ float placeholder_d_shared[49152];
  __shared__ float placeholder_shared[12288];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(32)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(33)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(34)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(35)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(36)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(37)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(38)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(39)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(40)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(41)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(42)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(43)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(44)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(45)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(46)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(47)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(48)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(49)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(50)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(51)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(52)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(53)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(54)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(55)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(56)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(57)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(58)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(59)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(60)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(61)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(62)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  compute_local[(63)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 256; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_d_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 192) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 3) * 196608) + ((((ax0_ax1_fused_ax2_fused_outer_outer * 3) + (((int)threadIdx.x) >> 6)) >> 6) * 16384)) + (((((int)blockIdx.x) & 7) >> 2) * 8192)) + ((((ax0_ax1_fused_ax2_fused_outer_outer * 3) + (((int)threadIdx.x) >> 6)) & 63) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 576))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1152))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 1536))];
    placeholder_shared[((((int)threadIdx.x) + 960))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 960) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 15) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 1152))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 1152) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 2) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 1344))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 1344) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 5) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 1536))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 1536) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 8) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 1728))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 1728) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 11) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 1920) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 14) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 2112))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 2112) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 1) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 2304))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 2304) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 4) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 2496))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 2496) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 7) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 2688))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 2688) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 10) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 2880))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 2880) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 13) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 3072))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 24576))];
    placeholder_shared[((((int)threadIdx.x) + 3264))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 3264) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 3) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 3456))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 3456) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 3648))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 3648) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 9) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 3840))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 3840) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 12) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 4032))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 4032) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 15) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 4224))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 4224) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 2) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 4416))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 4416) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 5) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 4608))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 4608) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 8) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 4800))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 4800) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 11) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 4992))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 4992) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 14) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 5184))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 5184) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 1) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 5376))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 5376) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 4) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 5568))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 5568) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 7) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 5760))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 5760) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 10) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 5952))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 5952) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 13) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 6144))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 49152))];
    placeholder_shared[((((int)threadIdx.x) + 6336))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 6336) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 3) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 6528))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 6528) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 6720))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 6720) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 9) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 6912))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 6912) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 12) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 7104))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 7104) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 15) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 7296))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 7296) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 2) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 7488))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 7488) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 5) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 7680))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 7680) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 8) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 7872))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 7872) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 11) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 8064))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 8064) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 14) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 8256))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 8256) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 1) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 8448))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 8448) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 4) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 8640))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 8640) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 7) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 8832))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 8832) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 10) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 9024))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 9024) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 13) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 9216))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)blockIdx.x) & 3) * 2048)) + ((((int)threadIdx.x) >> 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)) + 73728))];
    placeholder_shared[((((int)threadIdx.x) + 9408))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 9408) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 3) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 9600))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 9600) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 6) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 9792))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 9792) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 9) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 9984))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 9984) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 12) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 10176))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 10176) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 15) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 10368))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 10368) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 2) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 10560))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 10560) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 5) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 10752))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 10752) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 8) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 10944))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 10944) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 11) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 11136))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 11136) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + ((((((int)threadIdx.x) >> 6) + 14) & 15) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 11328))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 11328) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 1) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 11520))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 11520) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 4) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 11712))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 11712) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 7) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 11904))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 11904) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 10) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 12096))] = placeholder1[((((((((((int)blockIdx.x) >> 3) * 98304) + (((((int)threadIdx.x) + 12096) >> 10) * 8192)) + ((((int)blockIdx.x) & 3) * 2048)) + (((((int)threadIdx.x) >> 6) + 13) * 128)) + (k_outer_outer * 64)) + (((int)threadIdx.x) & 63)))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
        for (int i_c_outer_inner = 0; i_c_outer_inner < 8; ++i_c_outer_inner) {
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[(((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)))] * placeholder_shared[((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2048))] * placeholder_shared[((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 1))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 1))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2049))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 1))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 2))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2050))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 2))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 3))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 3))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2051))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 3))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 4))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 4))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2052))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 4))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 5))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 5))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2053))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 5))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 6))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 6))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2054))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 6))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 7))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 7))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2055))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 7))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 8))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 8))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2056))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 8))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 9))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 9))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2057))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 9))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 10))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 10))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2058))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 10))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 11))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 11))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2059))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 11))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 12))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 12))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2060))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 12))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 13))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 13))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2061))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 13))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 14))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 14))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2062))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 14))]));
          compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] = (compute_local[(((b_c_outer_inner * 8) + i_c_outer_inner))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 15))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 15))]));
          compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] = (compute_local[((((b_c_outer_inner * 8) + i_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((((int)threadIdx.x) >> 6) * 16384) + (b_c_outer_inner * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_c_outer_inner * 64)) + (k_outer_inner * 16)) + 2063))] * placeholder_shared[(((((((((int)threadIdx.x) >> 6) * 4096) + (b_c_outer_inner * 1024)) + ((((int)threadIdx.x) & 15) * 64)) + (k_outer_inner * 16)) + 15))]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 8; ++i_inner) {
      compute[((((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)threadIdx.x) >> 6) * 32768)) + (b_inner * 8192)) + (((((int)blockIdx.x) & 7) >> 2) * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)))] = compute_local[(((b_inner * 8) + i_inner))];
      compute[(((((((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)threadIdx.x) >> 6) * 32768)) + (b_inner * 8192)) + (((((int)blockIdx.x) & 7) >> 2) * 4096)) + (((((int)threadIdx.x) & 63) >> 4) * 512)) + (i_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)) + 2048))] = compute_local[((((b_inner * 8) + i_inner) + 32))];
    }
  }
}


