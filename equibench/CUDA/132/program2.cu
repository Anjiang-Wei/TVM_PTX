
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[112];
  __shared__ float PaddedInput_shared[672];
  __shared__ float placeholder_shared[3072];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
    Conv2dOutput[(xx_outer_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 14))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 28))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 42))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 56))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 70))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 84))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 98))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 7))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 21))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 35))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 49))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 63))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 77))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 91))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 105))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 14; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 84; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) / 336) * 32928) + ((((int)blockIdx.x) >> 1) * 4704)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) % 336) / 48) * 672)) + (rc_outer_outer * 48)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) % 48)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 384; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 6144) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 24; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          Conv2dOutput[(xx_outer_inner)] = (Conv2dOutput[(xx_outer_inner)] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)))]));
          Conv2dOutput[((xx_outer_inner + 14))] = (Conv2dOutput[((xx_outer_inner + 14))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 8))]));
          Conv2dOutput[((xx_outer_inner + 28))] = (Conv2dOutput[((xx_outer_inner + 28))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 16))]));
          Conv2dOutput[((xx_outer_inner + 42))] = (Conv2dOutput[((xx_outer_inner + 42))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 24))]));
          Conv2dOutput[((xx_outer_inner + 56))] = (Conv2dOutput[((xx_outer_inner + 56))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 32))]));
          Conv2dOutput[((xx_outer_inner + 70))] = (Conv2dOutput[((xx_outer_inner + 70))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 40))]));
          Conv2dOutput[((xx_outer_inner + 84))] = (Conv2dOutput[((xx_outer_inner + 84))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 48))]));
          Conv2dOutput[((xx_outer_inner + 98))] = (Conv2dOutput[((xx_outer_inner + 98))] + (PaddedInput_shared[((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 56))]));
          Conv2dOutput[((xx_outer_inner + 7))] = (Conv2dOutput[((xx_outer_inner + 7))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)))]));
          Conv2dOutput[((xx_outer_inner + 21))] = (Conv2dOutput[((xx_outer_inner + 21))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 8))]));
          Conv2dOutput[((xx_outer_inner + 35))] = (Conv2dOutput[((xx_outer_inner + 35))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 16))]));
          Conv2dOutput[((xx_outer_inner + 49))] = (Conv2dOutput[((xx_outer_inner + 49))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 24))]));
          Conv2dOutput[((xx_outer_inner + 63))] = (Conv2dOutput[((xx_outer_inner + 63))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 32))]));
          Conv2dOutput[((xx_outer_inner + 77))] = (Conv2dOutput[((xx_outer_inner + 77))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 40))]));
          Conv2dOutput[((xx_outer_inner + 91))] = (Conv2dOutput[((xx_outer_inner + 91))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 48))]));
          Conv2dOutput[((xx_outer_inner + 105))] = (Conv2dOutput[((xx_outer_inner + 105))] + (PaddedInput_shared[(((((xx_outer_inner * 48) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((int)threadIdx.x)) + 56))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
      T_relu[((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))] = max((Conv2dOutput[(((ax0_inner * 7) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 8))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 14))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 8))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 16))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 28))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 16))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 24))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 42))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 24))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 32))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 56))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 32))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 40))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 70))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 40))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 48))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 84))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 48))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 6272) + ((((int)blockIdx.x) >> 1) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 56))] = max((Conv2dOutput[((((ax0_inner * 7) + ax2_inner) + 98))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((int)threadIdx.x)) + 56))]), 0.000000e+00f);
    }
  }
}


