
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[64];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 48; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[((((int)threadIdx.x) * 16))] = placeholder[((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 1))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 2))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 96))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 3))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 97))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 4))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 6144))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 5))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 6145))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 6))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 6240))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 7))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 6241))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 8))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 12288))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 9))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 12289))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 10))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 12384))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 11))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 12385))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 12))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 18432))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 13))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 18433))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 14))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 18528))];
    }
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[(((((int)threadIdx.x) * 16) + 15))] = placeholder[(((((((((int)blockIdx.x) >> 7) * 196608) + (((int)threadIdx.x) * 24576)) + (((((int)blockIdx.x) & 127) >> 2) * 192)) + (rc_outer_outer * 2)) + 18529))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[((((int)threadIdx.x) * 16))] = placeholder1[(((((rc_outer_outer * 256) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 16)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 1))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 1) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 1) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 2))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 2) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 2) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 3))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 3) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 3) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 4))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 4) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 4) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 5))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 5) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 5) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 6))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 6) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 6) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 7))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 7) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 7) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 8))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 8) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 8) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 9))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 9) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 9) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 10))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 10) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 10) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 11))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 11) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 11) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 12))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 12) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 12) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 13))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 13) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 13) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 14))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 14) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 14) & 31)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 15))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 16) + 15) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 16) + 15) & 31)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 8) + (((((int)threadIdx.x) & 7) >> 2) * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[((((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 127) >> 2) * 256)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 4) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((((int)blockIdx.x) >> 7) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 127) >> 2) * 256)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 8))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
    }
  }
}


