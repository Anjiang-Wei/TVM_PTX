
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[12];
  __shared__ float PaddedInput_shared[3072];
  __shared__ float placeholder_shared[288];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(7)] = 0.000000e+00f;
  DepthwiseConv2d[(8)] = 0.000000e+00f;
  DepthwiseConv2d[(9)] = 0.000000e+00f;
  DepthwiseConv2d[(10)] = 0.000000e+00f;
  DepthwiseConv2d[(11)] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = (((1 <= (((((int)blockIdx.x) >> 2) * 2) + di_outer_outer)) && (96 <= ((int)threadIdx.x))) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 10752) + (di_outer_outer * 5376)) + ((((int)threadIdx.x) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((int)threadIdx.x) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = ((1 <= (((((int)blockIdx.x) >> 2) * 2) + di_outer_outer)) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 10752) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 224) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 32) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = ((1 <= (((((int)blockIdx.x) >> 2) * 2) + di_outer_outer)) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 10752) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 448) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 64) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 672))] = ((1 <= (((((int)blockIdx.x) >> 2) * 2) + di_outer_outer)) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 10752) + (di_outer_outer * 5376)) + ((((int)threadIdx.x) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((int)threadIdx.x) % 96)) - 3072))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 896))] = ((1 <= (((((int)blockIdx.x) >> 2) * 2) + di_outer_outer)) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 10752) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 896) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 32) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 1120))] = ((1 <= (((((int)blockIdx.x) >> 2) * 2) + di_outer_outer)) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 10752) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 1120) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 64) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 1344))] = (((((1 <= ((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 1344) / 1536)) + di_outer_outer)) && (((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 1344) / 1536)) + di_outer_outer) < 15)) && (96 <= ((((int)threadIdx.x) + 1344) % 1536))) && (((((int)threadIdx.x) + 1344) % 1536) < 1440)) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 1344) / 1536) * 5376)) + (di_outer_outer * 5376)) + ((((((int)threadIdx.x) + 1344) % 1536) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((int)threadIdx.x) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 1568))] = (((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 1568) / 1536)) + di_outer_outer) < 15) && (64 <= ((int)threadIdx.x))) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 1568) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 32) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 32) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 1792))] = ((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 1792) / 1536)) + di_outer_outer) < 15) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 1792) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 256) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 64) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 2016))] = ((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 2016) / 1536)) + di_outer_outer) < 15) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 2016) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 480) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((int)threadIdx.x) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 2240))] = ((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 2240) / 1536)) + di_outer_outer) < 15) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 2240) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 704) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 32) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 2464))] = ((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 2464) / 1536)) + di_outer_outer) < 15) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 2464) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 928) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) + 64) % 96)) - 5760))] : 0.000000e+00f);
    PaddedInput_shared[((((int)threadIdx.x) + 2688))] = ((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 2688) / 1536)) + di_outer_outer) < 15) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 2688) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 1152) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((int)threadIdx.x) % 96)) - 5760))] : 0.000000e+00f);
    if (((int)threadIdx.x) < 160) {
      PaddedInput_shared[((((int)threadIdx.x) + 2912))] = (((((((((int)blockIdx.x) >> 2) * 2) + ((((int)threadIdx.x) + 2912) / 1536)) + di_outer_outer) < 15) && (((int)threadIdx.x) < 64)) ? placeholder[(((((((((((int)blockIdx.x) >> 2) * 10752) + (((((int)threadIdx.x) + 2912) / 1536) * 5376)) + (di_outer_outer * 5376)) + (((((int)threadIdx.x) + 1376) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((int)threadIdx.x) + 32)) - 5760))] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 288) {
        if (((int)threadIdx.x) < 16) {
          placeholder_shared[(((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((di_outer_outer * 1152) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 96) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 96)))];
        }
      }
    }
    __syncthreads();
    for (int c_outer_inner = 0; c_outer_inner < 4; ++c_outer_inner) {
      DepthwiseConv2d[((c_outer_inner * 3))] = (DepthwiseConv2d[((c_outer_inner * 3))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)))]));
      DepthwiseConv2d[(((c_outer_inner * 3) + 1))] = (DepthwiseConv2d[(((c_outer_inner * 3) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 1))]));
      DepthwiseConv2d[(((c_outer_inner * 3) + 2))] = (DepthwiseConv2d[(((c_outer_inner * 3) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 2))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 2))]));
      DepthwiseConv2d[((c_outer_inner * 3))] = (DepthwiseConv2d[((c_outer_inner * 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 96))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 96))]));
      DepthwiseConv2d[(((c_outer_inner * 3) + 1))] = (DepthwiseConv2d[(((c_outer_inner * 3) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 97))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 97))]));
      DepthwiseConv2d[(((c_outer_inner * 3) + 2))] = (DepthwiseConv2d[(((c_outer_inner * 3) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 98))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 98))]));
      DepthwiseConv2d[((c_outer_inner * 3))] = (DepthwiseConv2d[((c_outer_inner * 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 192))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 192))]));
      DepthwiseConv2d[(((c_outer_inner * 3) + 1))] = (DepthwiseConv2d[(((c_outer_inner * 3) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 193))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 193))]));
      DepthwiseConv2d[(((c_outer_inner * 3) + 2))] = (DepthwiseConv2d[(((c_outer_inner * 3) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 112) * 1536) + ((((int)threadIdx.x) % 112) * 12)) + (c_outer_inner * 3)) + 194))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 12) + (c_outer_inner * 3)) + 194))]));
    }
  }
  for (int i3_inner = 0; i3_inner < 12; ++i3_inner) {
    compute[(((((((((int)blockIdx.x) >> 2) * 10752) + ((((int)threadIdx.x) >> 3) * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) & 7) * 12)) + i3_inner))] = max(min((DepthwiseConv2d[(i3_inner)] + placeholder2[(((((((int)blockIdx.x) & 3) * 96) + ((((int)threadIdx.x) & 7) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
  }
}


