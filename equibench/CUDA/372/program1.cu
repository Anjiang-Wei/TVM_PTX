
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[48];
  __shared__ float PaddedInput_shared[6144];
  __shared__ float placeholder_shared[64];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        Conv2dOutput[((((nn_inner_init * 4) + (yy_outer_inner_init * 2)) + ff_outer_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 4) + (yy_outer_inner_init * 2)) + ff_outer_inner_init) + 8))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 4) + (yy_outer_inner_init * 2)) + ff_outer_inner_init) + 16))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 4) + (yy_outer_inner_init * 2)) + ff_outer_inner_init) + 24))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 4) + (yy_outer_inner_init * 2)) + ff_outer_inner_init) + 32))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 4) + (yy_outer_inner_init * 2)) + ff_outer_inner_init) + 40))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 6)) / 12) * 460800) + ((((int)blockIdx.x) / 30) * 46080)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 6)) % 12) * 3840)) + (((((int)blockIdx.x) % 30) >> 1) * 256)) + (((((int)threadIdx.x) & 63) >> 3) * 32)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 64) {
        if (((int)threadIdx.x) < 32) {
          placeholder_shared[(((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 128) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 3) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 7)))];
        }
      }
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
            Conv2dOutput[((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner))] = (Conv2dOutput[((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 6) * 1536) + (nn_inner * 768)) + (((((int)threadIdx.x) & 63) >> 5) * 128)) + (yy_outer_inner * 64)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 8))] = (Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 8))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 6) * 1536) + (nn_inner * 768)) + (((((int)threadIdx.x) & 63) >> 5) * 128)) + (yy_outer_inner * 64)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + rc_inner) + 256))] * placeholder_shared[((((rc_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 16))] = (Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 16))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 6) * 1536) + (nn_inner * 768)) + (((((int)threadIdx.x) & 63) >> 5) * 128)) + (yy_outer_inner * 64)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + rc_inner) + 512))] * placeholder_shared[((((rc_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 24))] = (Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 24))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 6) * 1536) + (nn_inner * 768)) + (((((int)threadIdx.x) & 63) >> 5) * 128)) + (yy_outer_inner * 64)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + rc_inner) + 3072))] * placeholder_shared[((((rc_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 32))] = (Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 32))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 6) * 1536) + (nn_inner * 768)) + (((((int)threadIdx.x) & 63) >> 5) * 128)) + (yy_outer_inner * 64)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + rc_inner) + 3328))] * placeholder_shared[((((rc_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 40))] = (Conv2dOutput[(((((nn_inner * 4) + (yy_outer_inner * 2)) + ff_outer_inner) + 40))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 6) * 1536) + (nn_inner * 768)) + (((((int)threadIdx.x) & 63) >> 5) * 128)) + (yy_outer_inner * 64)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + rc_inner) + 3584))] * placeholder_shared[((((rc_inner * 8) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[((((((((((((((int)threadIdx.x) >> 6) * 460800) + (ax0_inner * 230400)) + ((((int)blockIdx.x) / 30) * 23040)) + (((((int)threadIdx.x) & 63) >> 5) * 3840)) + (ax1_inner * 1920)) + (((((int)blockIdx.x) % 30) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))] = (Conv2dOutput[((((ax0_inner * 4) + (ax1_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]);
        T_add[(((((((((((((((int)threadIdx.x) >> 6) * 460800) + (ax0_inner * 230400)) + ((((int)blockIdx.x) / 30) * 23040)) + (((((int)threadIdx.x) & 63) >> 5) * 3840)) + (ax1_inner * 1920)) + (((((int)blockIdx.x) % 30) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 7680))] = (Conv2dOutput[(((((ax0_inner * 4) + (ax1_inner * 2)) + ax3_inner) + 8))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]);
        T_add[(((((((((((((((int)threadIdx.x) >> 6) * 460800) + (ax0_inner * 230400)) + ((((int)blockIdx.x) / 30) * 23040)) + (((((int)threadIdx.x) & 63) >> 5) * 3840)) + (ax1_inner * 1920)) + (((((int)blockIdx.x) % 30) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 15360))] = (Conv2dOutput[(((((ax0_inner * 4) + (ax1_inner * 2)) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]);
        T_add[(((((((((((((((int)threadIdx.x) >> 6) * 460800) + (ax0_inner * 230400)) + ((((int)blockIdx.x) / 30) * 23040)) + (((((int)threadIdx.x) & 63) >> 5) * 3840)) + (ax1_inner * 1920)) + (((((int)blockIdx.x) % 30) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 921600))] = (Conv2dOutput[(((((ax0_inner * 4) + (ax1_inner * 2)) + ax3_inner) + 24))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]);
        T_add[(((((((((((((((int)threadIdx.x) >> 6) * 460800) + (ax0_inner * 230400)) + ((((int)blockIdx.x) / 30) * 23040)) + (((((int)threadIdx.x) & 63) >> 5) * 3840)) + (ax1_inner * 1920)) + (((((int)blockIdx.x) % 30) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 929280))] = (Conv2dOutput[(((((ax0_inner * 4) + (ax1_inner * 2)) + ax3_inner) + 32))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]);
        T_add[(((((((((((((((int)threadIdx.x) >> 6) * 460800) + (ax0_inner * 230400)) + ((((int)blockIdx.x) / 30) * 23040)) + (((((int)threadIdx.x) & 63) >> 5) * 3840)) + (ax1_inner * 1920)) + (((((int)blockIdx.x) % 30) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 936960))] = (Conv2dOutput[(((((ax0_inner * 4) + (ax1_inner * 2)) + ax3_inner) + 40))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]);
      }
    }
  }
}


