
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[150];
  __shared__ float placeholder_shared[128];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[((((int)threadIdx.x) * 24))] = placeholder[(((((((((int)threadIdx.x) * 12) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + (((((int)threadIdx.x) * 12) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 1))] = placeholder[((((((((((int)threadIdx.x) * 12) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + (((((int)threadIdx.x) * 12) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 2))] = placeholder[((((((((((int)threadIdx.x) * 12) + 1) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 1) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 3))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 1) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 1) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 4))] = placeholder[((((((((((int)threadIdx.x) * 12) + 2) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 2) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 5))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 2) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 2) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 6))] = placeholder[((((((((((int)threadIdx.x) * 12) + 3) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 3) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 7))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 3) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 3) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 8))] = placeholder[((((((((((int)threadIdx.x) * 12) + 4) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 4) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 9))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 4) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 4) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 10))] = placeholder[((((((((((int)threadIdx.x) * 12) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + (((((int)threadIdx.x) * 12) % 5) * 256)) + (rc_outer_outer * 2)) + 3840))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 11))] = placeholder[((((((((((int)threadIdx.x) * 12) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + (((((int)threadIdx.x) * 12) % 5) * 256)) + (rc_outer_outer * 2)) + 3841))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 12))] = placeholder[((((((((((int)threadIdx.x) * 12) + 6) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 1) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 13))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 6) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 1) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 14))] = placeholder[((((((((((int)threadIdx.x) * 12) + 7) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 2) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 15))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 7) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 2) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 16))] = placeholder[((((((((((int)threadIdx.x) * 12) + 8) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 3) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 17))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 8) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 3) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 18))] = placeholder[((((((((((int)threadIdx.x) * 12) + 9) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 4) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 19))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 9) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 4) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 20))] = placeholder[((((((((((int)threadIdx.x) * 12) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + (((((int)threadIdx.x) * 12) % 5) * 256)) + (rc_outer_outer * 2)) + 7680))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 21))] = placeholder[((((((((((int)threadIdx.x) * 12) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + (((((int)threadIdx.x) * 12) % 5) * 256)) + (rc_outer_outer * 2)) + 7681))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 22))] = placeholder[((((((((((int)threadIdx.x) * 12) + 11) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 1) % 5) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 24) + 23))] = placeholder[(((((((((((int)threadIdx.x) * 12) + 11) / 5) * 3840) + ((((int)blockIdx.x) >> 4) * 1280)) + ((((((int)threadIdx.x) * 12) + 1) % 5) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[((((int)threadIdx.x) * 4))] = placeholder1[(((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 4) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 4)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder1[(((((rc_outer_outer * 2048) + ((((((int)threadIdx.x) * 4) + 1) >> 6) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((((int)threadIdx.x) * 4) + 1) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder1[(((((rc_outer_outer * 2048) + ((((((int)threadIdx.x) * 4) + 2) >> 6) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((((int)threadIdx.x) * 4) + 2) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder1[(((((rc_outer_outer * 2048) + ((((((int)threadIdx.x) * 4) + 3) >> 6) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((((int)threadIdx.x) * 4) + 3) & 63)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 8))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 16))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 32))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 48))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 1))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 17))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 33))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 49))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 64))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 80))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 96))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 112))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 65))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 81))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 97))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 113))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 2))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 18))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 34))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 50))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 3))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 19))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 35))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 51))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 66))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 82))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 98))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 114))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 67))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 83))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 99))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 115))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 4))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 20))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 36))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 52))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 5))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 21))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 37))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 53))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 68))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 84))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 100))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 116))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 69))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 85))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 101))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 117))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 6))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 22))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 38))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 54))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 7))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 23))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 39))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 55))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 70))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 86))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 102))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 118))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 71))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 87))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 103))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + 119))]));
  }
  for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
    T_relu[((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))] = max(((Conv2dOutput[(ax3_inner)] + placeholder2[(((((((int)blockIdx.x) & 15) * 64) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]) + placeholder3[((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 16))] = max(((Conv2dOutput[((ax3_inner + 8))] + placeholder2[((((((((int)blockIdx.x) & 15) * 64) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 16))]) + placeholder3[(((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 16))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 32))] = max(((Conv2dOutput[((ax3_inner + 16))] + placeholder2[((((((((int)blockIdx.x) & 15) * 64) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 32))]) + placeholder3[(((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 32))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 48))] = max(((Conv2dOutput[((ax3_inner + 24))] + placeholder2[((((((((int)blockIdx.x) & 15) * 64) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 48))]) + placeholder3[(((((((((((int)threadIdx.x) / 10) * 15360) + ((((int)blockIdx.x) >> 4) * 5120)) + (((((int)threadIdx.x) % 10) >> 1) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 48))]), 0.000000e+00f);
  }
}


