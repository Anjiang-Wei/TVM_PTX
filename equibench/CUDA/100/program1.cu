
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[7840];
  __shared__ float placeholder_shared[2560];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  Conv2dOutput[(64)] = 0.000000e+00f;
  Conv2dOutput[(65)] = 0.000000e+00f;
  Conv2dOutput[(66)] = 0.000000e+00f;
  Conv2dOutput[(67)] = 0.000000e+00f;
  Conv2dOutput[(68)] = 0.000000e+00f;
  Conv2dOutput[(69)] = 0.000000e+00f;
  Conv2dOutput[(70)] = 0.000000e+00f;
  Conv2dOutput[(71)] = 0.000000e+00f;
  Conv2dOutput[(72)] = 0.000000e+00f;
  Conv2dOutput[(73)] = 0.000000e+00f;
  Conv2dOutput[(74)] = 0.000000e+00f;
  Conv2dOutput[(75)] = 0.000000e+00f;
  Conv2dOutput[(76)] = 0.000000e+00f;
  Conv2dOutput[(77)] = 0.000000e+00f;
  Conv2dOutput[(78)] = 0.000000e+00f;
  Conv2dOutput[(79)] = 0.000000e+00f;
  Conv2dOutput[(80)] = 0.000000e+00f;
  Conv2dOutput[(81)] = 0.000000e+00f;
  Conv2dOutput[(82)] = 0.000000e+00f;
  Conv2dOutput[(83)] = 0.000000e+00f;
  Conv2dOutput[(84)] = 0.000000e+00f;
  Conv2dOutput[(85)] = 0.000000e+00f;
  Conv2dOutput[(86)] = 0.000000e+00f;
  Conv2dOutput[(87)] = 0.000000e+00f;
  Conv2dOutput[(88)] = 0.000000e+00f;
  Conv2dOutput[(89)] = 0.000000e+00f;
  Conv2dOutput[(90)] = 0.000000e+00f;
  Conv2dOutput[(91)] = 0.000000e+00f;
  Conv2dOutput[(92)] = 0.000000e+00f;
  Conv2dOutput[(93)] = 0.000000e+00f;
  Conv2dOutput[(94)] = 0.000000e+00f;
  Conv2dOutput[(95)] = 0.000000e+00f;
  Conv2dOutput[(96)] = 0.000000e+00f;
  Conv2dOutput[(97)] = 0.000000e+00f;
  Conv2dOutput[(98)] = 0.000000e+00f;
  Conv2dOutput[(99)] = 0.000000e+00f;
  Conv2dOutput[(100)] = 0.000000e+00f;
  Conv2dOutput[(101)] = 0.000000e+00f;
  Conv2dOutput[(102)] = 0.000000e+00f;
  Conv2dOutput[(103)] = 0.000000e+00f;
  Conv2dOutput[(104)] = 0.000000e+00f;
  Conv2dOutput[(105)] = 0.000000e+00f;
  Conv2dOutput[(106)] = 0.000000e+00f;
  Conv2dOutput[(107)] = 0.000000e+00f;
  Conv2dOutput[(108)] = 0.000000e+00f;
  Conv2dOutput[(109)] = 0.000000e+00f;
  Conv2dOutput[(110)] = 0.000000e+00f;
  Conv2dOutput[(111)] = 0.000000e+00f;
  Conv2dOutput[(112)] = 0.000000e+00f;
  Conv2dOutput[(113)] = 0.000000e+00f;
  Conv2dOutput[(114)] = 0.000000e+00f;
  Conv2dOutput[(115)] = 0.000000e+00f;
  Conv2dOutput[(116)] = 0.000000e+00f;
  Conv2dOutput[(117)] = 0.000000e+00f;
  Conv2dOutput[(118)] = 0.000000e+00f;
  Conv2dOutput[(119)] = 0.000000e+00f;
  Conv2dOutput[(120)] = 0.000000e+00f;
  Conv2dOutput[(121)] = 0.000000e+00f;
  Conv2dOutput[(122)] = 0.000000e+00f;
  Conv2dOutput[(123)] = 0.000000e+00f;
  Conv2dOutput[(124)] = 0.000000e+00f;
  Conv2dOutput[(125)] = 0.000000e+00f;
  Conv2dOutput[(126)] = 0.000000e+00f;
  Conv2dOutput[(127)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[((((int)threadIdx.x) * 40))] = placeholder[(((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 1))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 1))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 2))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 2))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 3))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 3))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 4))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 4))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 5))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 5))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 6))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 6))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 7))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 7))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 8))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 8))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 9))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 9))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 10))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 10))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 11))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 11))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 12))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 12))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 13))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 13))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 14))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 14))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 15))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 15))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 16))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 16))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 17))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 17))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 18))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 18))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 19))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 19))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 20))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 320))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 21))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 321))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 22))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 322))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 23))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 323))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 24))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 324))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 25))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 325))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 26))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 326))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 27))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 327))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 28))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 328))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 29))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 329))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 30))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 330))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 31))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 331))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 32))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 332))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 33))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 333))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 34))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 334))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 35))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 335))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 36))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 336))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 37))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 337))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 38))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 338))];
    }
    if (((int)threadIdx.x) < 196) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 39))] = placeholder[((((((int)threadIdx.x) * 640) + (rc_outer_outer * 20)) + 339))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 2560) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 392))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 392))];
    placeholder_shared[((((int)threadIdx.x) + 784))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 784))];
    placeholder_shared[((((int)threadIdx.x) + 1176))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1176))];
    placeholder_shared[((((int)threadIdx.x) + 1568))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1568))];
    placeholder_shared[((((int)threadIdx.x) + 1960))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1960))];
    if (((int)threadIdx.x) < 208) {
      placeholder_shared[((((int)threadIdx.x) + 2352))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 2352))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[(((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 1))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 2))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 3))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 4))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 5))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 6))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 7))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 8))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 9))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 10))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 11))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 12))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 13))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 14))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 15))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 16))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 17))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 18))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 19))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 20))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 21))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 22))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 23))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 24))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 25))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 26))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 27))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 28))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 29))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 30))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 31))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 32))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 33))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 34))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 35))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 36))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 37))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 38))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 39))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 40))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 41))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 42))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 43))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 44))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 45))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 46))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 47))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 48))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 49))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 50))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 51))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 52))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 53))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 54))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 55))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 56))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 57))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 58))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 59))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 60))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 61))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 62))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 63))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[(((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 1))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 2))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 3))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 4))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 5))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 6))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 7))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 8))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 9))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 10))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 11))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 12))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 13))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 14))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 15))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 16))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 17))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 18))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 19))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 20))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 21))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 22))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 23))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 24))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 25))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 26))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 27))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 28))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 29))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 30))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 31))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 32))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 33))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 34))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 35))]));
      Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 36))]));
      Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 37))]));
      Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 38))]));
      Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 39))]));
      Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 40))]));
      Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 41))]));
      Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 42))]));
      Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 43))]));
      Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 44))]));
      Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 45))]));
      Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 46))]));
      Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 47))]));
      Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 48))]));
      Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 49))]));
      Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 50))]));
      Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 51))]));
      Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 52))]));
      Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 53))]));
      Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 54))]));
      Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 55))]));
      Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 56))]));
      Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 57))]));
      Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 58))]));
      Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 59))]));
      Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 60))]));
      Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 61))]));
      Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 62))]));
      Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 20))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 63))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 128))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 129))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 130))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 131))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 132))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 133))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 134))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 135))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 136))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 137))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 138))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 139))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 140))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 141))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 142))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 143))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 144))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 145))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 146))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 147))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 148))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 149))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 150))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 151))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 152))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 153))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 154))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 155))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 156))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 157))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 158))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 159))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 160))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 161))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 162))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 163))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 164))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 165))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 166))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 167))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 168))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 169))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 170))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 171))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 172))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 173))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 174))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 175))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 176))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 177))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 178))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 179))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 180))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 181))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 182))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 183))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 184))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 185))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 186))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 187))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 188))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 189))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 190))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 1))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 191))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 128))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 129))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 130))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 131))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 132))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 133))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 134))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 135))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 136))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 137))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 138))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 139))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 140))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 141))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 142))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 143))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 144))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 145))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 146))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 147))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 148))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 149))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 150))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 151))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 152))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 153))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 154))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 155))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 156))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 157))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 158))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 159))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 160))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 161))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 162))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 163))]));
      Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 164))]));
      Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 165))]));
      Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 166))]));
      Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 167))]));
      Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 168))]));
      Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 169))]));
      Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 170))]));
      Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 171))]));
      Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 172))]));
      Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 173))]));
      Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 174))]));
      Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 175))]));
      Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 176))]));
      Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 177))]));
      Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 178))]));
      Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 179))]));
      Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 180))]));
      Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 181))]));
      Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 182))]));
      Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 183))]));
      Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 184))]));
      Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 185))]));
      Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 186))]));
      Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 187))]));
      Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 188))]));
      Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 189))]));
      Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 190))]));
      Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 21))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 191))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 256))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 257))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 258))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 259))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 260))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 261))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 262))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 263))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 264))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 265))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 266))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 267))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 268))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 269))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 270))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 271))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 272))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 273))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 274))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 275))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 276))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 277))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 278))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 279))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 280))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 281))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 282))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 283))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 284))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 285))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 286))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 287))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 288))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 289))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 290))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 291))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 292))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 293))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 294))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 295))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 296))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 297))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 298))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 299))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 300))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 301))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 302))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 303))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 304))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 305))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 306))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 307))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 308))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 309))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 310))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 311))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 312))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 313))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 314))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 315))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 316))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 317))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 318))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 2))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 319))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 256))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 257))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 258))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 259))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 260))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 261))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 262))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 263))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 264))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 265))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 266))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 267))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 268))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 269))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 270))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 271))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 272))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 273))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 274))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 275))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 276))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 277))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 278))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 279))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 280))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 281))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 282))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 283))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 284))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 285))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 286))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 287))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 288))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 289))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 290))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 291))]));
      Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 292))]));
      Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 293))]));
      Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 294))]));
      Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 295))]));
      Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 296))]));
      Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 297))]));
      Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 298))]));
      Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 299))]));
      Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 300))]));
      Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 301))]));
      Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 302))]));
      Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 303))]));
      Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 304))]));
      Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 305))]));
      Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 306))]));
      Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 307))]));
      Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 308))]));
      Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 309))]));
      Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 310))]));
      Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 311))]));
      Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 312))]));
      Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 313))]));
      Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 314))]));
      Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 315))]));
      Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 316))]));
      Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 317))]));
      Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 318))]));
      Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 22))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 319))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 384))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 385))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 386))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 387))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 388))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 389))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 390))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 391))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 392))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 393))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 394))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 395))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 396))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 397))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 398))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 399))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 400))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 401))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 402))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 403))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 404))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 405))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 406))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 407))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 408))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 409))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 410))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 411))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 412))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 413))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 414))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 415))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 416))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 417))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 418))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 419))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 420))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 421))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 422))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 423))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 424))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 425))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 426))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 427))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 428))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 429))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 430))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 431))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 432))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 433))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 434))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 435))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 436))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 437))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 438))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 439))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 440))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 441))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 442))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 443))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 444))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 445))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 446))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 3))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 447))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 384))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 385))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 386))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 387))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 388))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 389))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 390))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 391))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 392))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 393))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 394))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 395))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 396))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 397))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 398))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 399))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 400))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 401))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 402))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 403))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 404))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 405))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 406))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 407))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 408))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 409))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 410))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 411))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 412))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 413))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 414))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 415))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 416))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 417))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 418))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 419))]));
      Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 420))]));
      Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 421))]));
      Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 422))]));
      Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 423))]));
      Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 424))]));
      Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 425))]));
      Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 426))]));
      Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 427))]));
      Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 428))]));
      Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 429))]));
      Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 430))]));
      Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 431))]));
      Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 432))]));
      Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 433))]));
      Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 434))]));
      Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 435))]));
      Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 436))]));
      Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 437))]));
      Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 438))]));
      Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 439))]));
      Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 440))]));
      Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 441))]));
      Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 442))]));
      Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 443))]));
      Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 444))]));
      Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 445))]));
      Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 446))]));
      Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 23))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 447))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 512))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 513))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 514))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 515))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 516))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 517))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 518))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 519))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 520))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 521))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 522))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 523))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 524))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 525))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 526))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 527))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 528))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 529))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 530))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 531))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 532))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 533))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 534))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 535))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 536))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 537))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 538))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 539))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 540))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 541))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 542))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 543))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 544))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 545))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 546))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 547))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 548))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 549))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 550))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 551))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 552))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 553))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 554))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 555))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 556))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 557))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 558))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 559))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 560))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 561))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 562))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 563))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 564))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 565))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 566))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 567))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 568))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 569))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 570))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 571))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 572))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 573))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 574))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 4))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 575))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 512))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 513))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 514))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 515))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 516))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 517))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 518))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 519))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 520))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 521))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 522))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 523))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 524))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 525))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 526))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 527))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 528))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 529))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 530))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 531))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 532))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 533))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 534))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 535))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 536))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 537))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 538))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 539))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 540))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 541))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 542))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 543))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 544))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 545))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 546))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 547))]));
      Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 548))]));
      Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 549))]));
      Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 550))]));
      Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 551))]));
      Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 552))]));
      Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 553))]));
      Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 554))]));
      Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 555))]));
      Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 556))]));
      Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 557))]));
      Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 558))]));
      Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 559))]));
      Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 560))]));
      Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 561))]));
      Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 562))]));
      Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 563))]));
      Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 564))]));
      Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 565))]));
      Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 566))]));
      Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 567))]));
      Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 568))]));
      Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 569))]));
      Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 570))]));
      Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 571))]));
      Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 572))]));
      Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 573))]));
      Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 574))]));
      Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 40) + (rc_outer_inner * 5)) + 24))] * placeholder_shared[((((rc_outer_inner * 640) + ((((int)threadIdx.x) & 1) * 64)) + 575))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 64; ++ax3_inner) {
      T_relu[((((((((int)threadIdx.x) >> 1) * 256) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 64) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 64) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


