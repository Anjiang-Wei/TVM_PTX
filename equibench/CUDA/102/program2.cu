
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[896];
  __shared__ float PaddedInput_shared[4704];
  __shared__ float placeholder_shared[1536];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 8; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 7; ++yy_inner_init) {
      Conv2dOutput[(((yy_inner_init * 64) + (ff_outer_inner_init * 8)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 448))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 449))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 450))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 451))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 452))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 5))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 453))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 454))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 7))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 64) + (ff_outer_inner_init * 8)) + 455))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 84; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 12) * 384) + (rc_outer_outer * 12)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 12)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1536) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)))] = placeholder1[((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1535) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 1))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 1))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1534) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 2))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 2))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1533) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 3))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 3))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1532) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 4))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 4))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1531) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 5))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 5))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1530) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 6))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 6))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1529) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 7))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 7))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1528) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 8))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 8))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1527) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 9))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 9))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1526) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 10))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 10))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) < 1525) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 128) {
          placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + (((int)threadIdx.x) * 12)) + 11))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672)) + (((int)threadIdx.x) * 12)) + 11))];
        }
      }
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 8; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 12; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
          Conv2dOutput[(((yy_inner * 64) + (ff_outer_inner * 8)))] = (Conv2dOutput[(((yy_inner * 64) + (ff_outer_inner * 8)))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 448))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 448))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 1))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 1))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 449))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 449))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 1))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 2))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 2))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 450))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 450))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 2))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 3))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 3))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 451))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 451))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 3))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 4))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 4))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 452))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 452))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 4))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 5))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 5))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 453))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 453))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 5))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 6))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 6))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 6))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 454))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 454))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 6))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 7))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 7))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 7))]));
          Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 455))] = (Conv2dOutput[((((yy_inner * 64) + (ff_outer_inner * 8)) + 455))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 28) * 2352) + (yy_inner * 168)) + (((((int)threadIdx.x) % 28) >> 1) * 12)) + rc_inner) + 1176))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 1) * 64)) + (ff_outer_inner * 8)) + 7))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 64; ++ax3_inner) {
      T_relu[((((((((int)threadIdx.x) / 28) * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) % 28) * 64)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 64) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 64) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)threadIdx.x) / 28) * 25088) + (ax1_inner * 1792)) + ((((int)threadIdx.x) % 28) * 64)) + ax3_inner) + 12544))] = max((Conv2dOutput[((((ax1_inner * 64) + ax3_inner) + 448))] + placeholder2[((((((int)threadIdx.x) & 1) * 64) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


