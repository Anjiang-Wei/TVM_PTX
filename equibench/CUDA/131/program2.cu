
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[168];
  __shared__ float placeholder_shared[1536];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 56; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 168) {
      PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)threadIdx.x) / 12) * 4704) + (((int)blockIdx.x) * 672)) + (rc_outer_outer * 12)) + (((int)threadIdx.x) % 12)))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[((((int)threadIdx.x) * 8))] = placeholder1[(((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 1))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 2))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 3))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 4))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 5))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 6))];
    }
    if (((int)threadIdx.x) < 192) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder1[((((rc_outer_outer * 1536) + (((int)threadIdx.x) * 8)) + 7))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 6) * 12))] * placeholder_shared[(((((int)threadIdx.x) & 63) * 2))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 128))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 256))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 384))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 6) * 12))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 129))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 257))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 385))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 512))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 640))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 768))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 896))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 513))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 641))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 769))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 897))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1024))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1152))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1280))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1408))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1025))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1153))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1281))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 12) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 2) + 1409))]));
  }
  for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
    T_relu[((((((((int)threadIdx.x) >> 6) * 896) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[((((((int)threadIdx.x) & 63) * 2) + ax3_inner))]), 0.000000e+00f);
  }
}


