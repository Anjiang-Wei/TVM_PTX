
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[56];
  __shared__ float PaddedInput_shared[4704];
  __shared__ float placeholder_shared[288];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
        Conv2dOutput[((((nn_inner_init * 14) + (xx_outer_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 14) + (xx_outer_inner_init * 2)) + ff_inner_init) + 28))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 12; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 168) + ((int)threadIdx.x)) < 392) {
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
          if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2016) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 4704) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 168) + ((int)threadIdx.x)) / 196) * 2352) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 6) + (((int)threadIdx.x) / 28)) % 7) * 336)) + ((((int)threadIdx.x) % 28) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((((int)blockIdx.x) >> 4) * 903168) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 168) + ((int)threadIdx.x)) / 196) * 451584)) + (((((int)blockIdx.x) & 15) >> 1) * 56448)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 6) + (((int)threadIdx.x) / 28)) % 7) * 8064)) + ((((int)blockIdx.x) & 1) * 4032)) + ((((int)threadIdx.x) % 28) * 144)) + (rc_outer_outer * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) / 24)) < 12) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 168) + ((int)threadIdx.x)) < 288) {
          placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 168) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 288) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 168)) + ((int)threadIdx.x)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
            for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
              Conv2dOutput[((((nn_inner * 14) + (xx_outer_inner * 2)) + ff_inner))] = (Conv2dOutput[((((nn_inner * 14) + (xx_outer_inner * 2)) + ff_inner))] + (PaddedInput_shared[((((((nn_inner * 2352) + ((((int)threadIdx.x) / 6) * 84)) + (xx_outer_inner * 12)) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 96) + (rc_inner * 24)) + ((((int)threadIdx.x) % 6) * 2)) + ff_inner))]));
              Conv2dOutput[(((((nn_inner * 14) + (xx_outer_inner * 2)) + ff_inner) + 28))] = (Conv2dOutput[(((((nn_inner * 14) + (xx_outer_inner * 2)) + ff_inner) + 28))] + (PaddedInput_shared[((((((nn_inner * 2352) + ((((int)threadIdx.x) / 6) * 84)) + (xx_outer_inner * 12)) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 96) + (rc_inner * 24)) + ((((int)threadIdx.x) % 6) * 2)) + ff_inner) + 12))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[(((((((((((((int)blockIdx.x) >> 4) * 150528) + (ax0_inner * 75264)) + (((((int)blockIdx.x) & 15) >> 1) * 9408)) + ((((int)threadIdx.x) / 24) * 1344)) + ((((int)blockIdx.x) & 1) * 672)) + (((((int)threadIdx.x) % 24) / 6) * 168)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 6) * 2)) + ax3_inner))] = (placeholder2[(((((((((((((int)blockIdx.x) >> 4) * 150528) + (ax0_inner * 75264)) + (((((int)blockIdx.x) & 15) >> 1) * 9408)) + ((((int)threadIdx.x) / 24) * 1344)) + ((((int)blockIdx.x) & 1) * 672)) + (((((int)threadIdx.x) % 24) / 6) * 168)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 6) * 2)) + ax3_inner))] + (Conv2dOutput[((((ax0_inner * 14) + (ax2_inner * 2)) + ax3_inner))] + placeholder3[((((((int)threadIdx.x) % 6) * 2) + ax3_inner))]));
        T_add[((((((((((((((int)blockIdx.x) >> 4) * 150528) + (ax0_inner * 75264)) + (((((int)blockIdx.x) & 15) >> 1) * 9408)) + ((((int)threadIdx.x) / 24) * 1344)) + ((((int)blockIdx.x) & 1) * 672)) + (((((int)threadIdx.x) % 24) / 6) * 168)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 6) * 2)) + ax3_inner) + 12))] = (placeholder2[((((((((((((((int)blockIdx.x) >> 4) * 150528) + (ax0_inner * 75264)) + (((((int)blockIdx.x) & 15) >> 1) * 9408)) + ((((int)threadIdx.x) / 24) * 1344)) + ((((int)blockIdx.x) & 1) * 672)) + (((((int)threadIdx.x) % 24) / 6) * 168)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 6) * 2)) + ax3_inner) + 12))] + (Conv2dOutput[(((((ax0_inner * 14) + (ax2_inner * 2)) + ax3_inner) + 28))] + placeholder3[(((((((int)threadIdx.x) % 6) * 2) + ax3_inner) + 12))]));
      }
    }
  }
}


