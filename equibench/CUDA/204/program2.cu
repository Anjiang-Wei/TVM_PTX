
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[28];
  __shared__ float pad_temp_shared[18560];
  __shared__ float placeholder_shared[18432];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
    compute[((xx_outer_inner_init * 2))] = 0.000000e+00f;
    compute[(((xx_outer_inner_init * 2) + 14))] = 0.000000e+00f;
    compute[(((xx_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    compute[(((xx_outer_inner_init * 2) + 15))] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 145; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    pad_temp_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)))] = (((1 <= (((((int)blockIdx.x) >> 2) * 4) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 29))) && (1 <= (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 29))) ? placeholder[((((((((((int)blockIdx.x) >> 2) * 57344) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 29) * 14336)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 29) * 512)) + ((((int)blockIdx.x) & 3) * 128)) + ((int)threadIdx.x)) - 14848))] : 0.000000e+00f);
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 144; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)))] = placeholder1[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((((int)blockIdx.x) & 3) * 128)) + ((int)threadIdx.x)))];
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
    for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          compute[(((xx_outer_inner * 2) + ff_outer_inner))] = (compute[(((xx_outer_inner * 2) + ff_outer_inner))] + (pad_temp_shared[(((((((((int)threadIdx.x) >> 6) * 7424) + (ry_inner * 3712)) + (xx_outer_inner * 256)) + (((((((int)threadIdx.x) & 63) * 2) + ff_outer_inner) >> 4) * 16)) + rc_outer_inner))] * placeholder_shared[(((((ry_inner * 6144) + (rc_outer_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ff_outer_inner))]));
          compute[((((xx_outer_inner * 2) + ff_outer_inner) + 14))] = (compute[((((xx_outer_inner * 2) + ff_outer_inner) + 14))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 6) * 7424) + (ry_inner * 3712)) + (xx_outer_inner * 256)) + (((((((int)threadIdx.x) & 63) * 2) + ff_outer_inner) >> 4) * 16)) + rc_outer_inner) + 1792))] * placeholder_shared[(((((ry_inner * 6144) + (rc_outer_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ff_outer_inner))]));
          compute[(((xx_outer_inner * 2) + ff_outer_inner))] = (compute[(((xx_outer_inner * 2) + ff_outer_inner))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 6) * 7424) + (ry_inner * 3712)) + (xx_outer_inner * 256)) + (((((((int)threadIdx.x) & 63) * 2) + ff_outer_inner) >> 4) * 16)) + rc_outer_inner) + 128))] * placeholder_shared[((((((ry_inner * 6144) + (rc_outer_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ff_outer_inner) + 2048))]));
          compute[((((xx_outer_inner * 2) + ff_outer_inner) + 14))] = (compute[((((xx_outer_inner * 2) + ff_outer_inner) + 14))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 6) * 7424) + (ry_inner * 3712)) + (xx_outer_inner * 256)) + (((((((int)threadIdx.x) & 63) * 2) + ff_outer_inner) >> 4) * 16)) + rc_outer_inner) + 1920))] * placeholder_shared[((((((ry_inner * 6144) + (rc_outer_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ff_outer_inner) + 2048))]));
          compute[(((xx_outer_inner * 2) + ff_outer_inner))] = (compute[(((xx_outer_inner * 2) + ff_outer_inner))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 6) * 7424) + (ry_inner * 3712)) + (xx_outer_inner * 256)) + (((((((int)threadIdx.x) & 63) * 2) + ff_outer_inner) >> 4) * 16)) + rc_outer_inner) + 256))] * placeholder_shared[((((((ry_inner * 6144) + (rc_outer_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ff_outer_inner) + 4096))]));
          compute[((((xx_outer_inner * 2) + ff_outer_inner) + 14))] = (compute[((((xx_outer_inner * 2) + ff_outer_inner) + 14))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 6) * 7424) + (ry_inner * 3712)) + (xx_outer_inner * 256)) + (((((((int)threadIdx.x) & 63) * 2) + ff_outer_inner) >> 4) * 16)) + rc_outer_inner) + 2048))] * placeholder_shared[((((((ry_inner * 6144) + (rc_outer_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ff_outer_inner) + 4096))]));
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 2) * 14336) + ((((int)threadIdx.x) >> 6) * 7168)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 3) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ax3_inner))] = max((compute[(((ax2_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 128) + ((((int)threadIdx.x) & 63) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 14336) + ((((int)threadIdx.x) >> 6) * 7168)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 3) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ax3_inner) + 3584))] = max((compute[((((ax2_inner * 2) + ax3_inner) + 14))] + placeholder2[(((((((int)blockIdx.x) & 3) * 128) + ((((int)threadIdx.x) & 63) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


