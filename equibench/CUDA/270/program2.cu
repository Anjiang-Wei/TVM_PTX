
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[8];
  __shared__ float pad_temp_shared[512];
  __shared__ float placeholder_shared[288];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    pad_temp_shared[(((int)threadIdx.x))] = (((64 <= (((int)blockIdx.x) % 896)) && (1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((int)threadIdx.x) >> 2)))) ? placeholder[((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 56))] = ((((1 <= ((((((int)blockIdx.x) % 896) >> 6) * 2) + ((((int)threadIdx.x) + 56) >> 6))) && (1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 14) & 15)))) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 14) & 15)) < 29)) ? placeholder[(((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (((((int)threadIdx.x) + 56) >> 6) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((((int)threadIdx.x) >> 2) + 14) & 15) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 112))] = (((1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 12) & 15))) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 12) & 15)) < 29)) ? placeholder[(((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (((((int)threadIdx.x) + 112) >> 6) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((((int)threadIdx.x) >> 2) + 12) & 15) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 168))] = ((((((((((int)blockIdx.x) % 896) >> 6) * 2) + ((((int)threadIdx.x) + 168) >> 6)) < 29) && (1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 10) & 15)))) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 10) & 15)) < 29)) ? placeholder[(((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (((((int)threadIdx.x) + 168) >> 6) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((((int)threadIdx.x) >> 2) + 10) & 15) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 224))] = (((((1 <= ((((((int)blockIdx.x) % 896) >> 6) * 2) + ((((((int)threadIdx.x) >> 2) + 56) & 63) >> 4))) && (((((((int)blockIdx.x) % 896) >> 6) * 2) + ((((((int)threadIdx.x) >> 2) + 56) & 63) >> 4)) < 29)) && (1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 8) & 15)))) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 8) & 15)) < 29)) ? placeholder[((((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)threadIdx.x) + 224) >> 8) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (((((((int)threadIdx.x) >> 2) + 56) & 63) >> 4) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((((int)threadIdx.x) >> 2) + 8) & 15) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 280))] = ((((1 <= ((((((int)blockIdx.x) % 896) >> 6) * 2) + (((((int)threadIdx.x) >> 2) + 6) >> 4))) && (1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 6) & 15)))) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 6) & 15)) < 29)) ? placeholder[((((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)threadIdx.x) + 280) >> 8) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + ((((((int)threadIdx.x) >> 2) + 6) >> 4) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((((int)threadIdx.x) >> 2) + 6) & 15) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 336))] = (((1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 4) & 15))) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + (((((int)threadIdx.x) >> 2) + 4) & 15)) < 29)) ? placeholder[((((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)threadIdx.x) + 336) >> 8) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + ((((((int)threadIdx.x) >> 2) + 20) >> 4) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((((int)threadIdx.x) >> 2) + 4) & 15) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 392))] = ((((((((int)blockIdx.x) & 63) >> 5) * 14) + ((((int)threadIdx.x) >> 2) + 2)) < 29) ? placeholder[((((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)threadIdx.x) + 392) >> 8) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + ((((((int)threadIdx.x) >> 2) + 34) >> 4) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + (((((int)threadIdx.x) >> 2) + 2) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    pad_temp_shared[((((int)threadIdx.x) + 448))] = (((((((((int)blockIdx.x) % 896) >> 6) * 2) + (((((int)threadIdx.x) >> 2) + 48) >> 4)) < 29) && (1 <= ((((((int)blockIdx.x) & 63) >> 5) * 14) + (((int)threadIdx.x) >> 2)))) ? placeholder[((((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)threadIdx.x) + 448) >> 8) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + ((((((int)threadIdx.x) >> 2) + 48) >> 4) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + ((((int)threadIdx.x) >> 2) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    if (((int)threadIdx.x) < 8) {
      pad_temp_shared[((((int)threadIdx.x) + 504))] = (((((((((int)blockIdx.x) % 896) >> 6) * 2) + (((((int)threadIdx.x) >> 2) + 62) >> 4)) < 29) && (((((((int)blockIdx.x) & 63) >> 5) * 14) + ((((int)threadIdx.x) >> 2) + 14)) < 29)) ? placeholder[((((((((((((((int)blockIdx.x) / 896) * 401408) + (((((int)threadIdx.x) + 504) >> 8) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + ((((((int)threadIdx.x) >> 2) + 62) >> 4) * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + (((((int)threadIdx.x) >> 2) + 14) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 7424))] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[((((int)threadIdx.x) * 12))] = placeholder1[((((((((((int)threadIdx.x) * 12) >> 5) * 2048) + (rc_outer_outer * 1024)) + ((((((int)threadIdx.x) * 12) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + ((((int)threadIdx.x) * 12) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 1))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 1) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 1) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 2))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 2) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 2) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 3))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 3) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 3) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 4))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 4) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 4) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 5))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 5) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 5) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 6))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 6) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 6) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 7))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 7) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 7) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 8))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 8) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 8) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + ((((int)threadIdx.x) * 12) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 9))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 9) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 9) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 10))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 10) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 10) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[(((((int)threadIdx.x) * 12) + 11))] = placeholder1[(((((((((((int)threadIdx.x) * 12) + 11) >> 5) * 2048) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 12) + 11) & 31) >> 3) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((((int)threadIdx.x) * 12) + 3) & 7)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        compute[(0)] = (compute[(0)] + (pad_temp_shared[((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)))] * placeholder_shared[((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 2))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 4))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 6))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 2))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 4))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 6))]));
        compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 8))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 10))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 12))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 14))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 8))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 10))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 12))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 14))]));
        compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 4))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 32))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 4))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 34))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 4))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 36))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 4))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 38))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 68))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 32))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 68))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 34))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 68))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 36))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 68))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 38))]));
        compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 5))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 40))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 5))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 42))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 5))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 44))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 5))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 46))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 69))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 40))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 69))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 42))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 69))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 44))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 69))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 46))]));
        compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 8))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 64))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 8))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 66))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 8))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 68))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 8))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 70))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 72))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 64))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 72))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 66))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 72))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 68))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 72))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 70))]));
        compute[(0)] = (compute[(0)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 9))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 72))]));
        compute[(2)] = (compute[(2)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 9))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 74))]));
        compute[(4)] = (compute[(4)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 9))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 76))]));
        compute[(6)] = (compute[(6)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 9))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 78))]));
        compute[(1)] = (compute[(1)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 73))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 72))]));
        compute[(3)] = (compute[(3)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 73))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 74))]));
        compute[(5)] = (compute[(5)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 73))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 76))]));
        compute[(7)] = (compute[(7)] + (pad_temp_shared[(((((((((int)threadIdx.x) / 28) * 256) + (ry_inner * 64)) + (((((int)threadIdx.x) % 28) >> 1) * 4)) + (rc_outer_inner * 2)) + 73))] * placeholder_shared[(((((ry_inner * 96) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 1)) + 78))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    T_relu[((((((((((((int)blockIdx.x) / 896) * 401408) + ((((int)threadIdx.x) / 28) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (ax1_inner * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + (((((int)threadIdx.x) % 28) >> 1) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((int)threadIdx.x) & 1)))] = max((compute[(ax1_inner)] + placeholder2[((((((int)blockIdx.x) & 31) * 8) + (((int)threadIdx.x) & 1)))]), 0.000000e+00f);
    T_relu[(((((((((((((int)blockIdx.x) / 896) * 401408) + ((((int)threadIdx.x) / 28) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (ax1_inner * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + (((((int)threadIdx.x) % 28) >> 1) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((int)threadIdx.x) & 1)) + 2))] = max((compute[((ax1_inner + 2))] + placeholder2[(((((((int)blockIdx.x) & 31) * 8) + (((int)threadIdx.x) & 1)) + 2))]), 0.000000e+00f);
    T_relu[(((((((((((((int)blockIdx.x) / 896) * 401408) + ((((int)threadIdx.x) / 28) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (ax1_inner * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + (((((int)threadIdx.x) % 28) >> 1) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((int)threadIdx.x) & 1)) + 4))] = max((compute[((ax1_inner + 4))] + placeholder2[(((((((int)blockIdx.x) & 31) * 8) + (((int)threadIdx.x) & 1)) + 4))]), 0.000000e+00f);
    T_relu[(((((((((((((int)blockIdx.x) / 896) * 401408) + ((((int)threadIdx.x) / 28) * 200704)) + (((((int)blockIdx.x) % 896) >> 6) * 14336)) + (ax1_inner * 7168)) + (((((int)blockIdx.x) & 63) >> 5) * 3584)) + (((((int)threadIdx.x) % 28) >> 1) * 256)) + ((((int)blockIdx.x) & 31) * 8)) + (((int)threadIdx.x) & 1)) + 6))] = max((compute[((ax1_inner + 6))] + placeholder2[(((((((int)blockIdx.x) & 31) * 8) + (((int)threadIdx.x) & 1)) + 6))]), 0.000000e+00f);
  }
}


