
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[256];
  __shared__ float pad_temp_shared[43008];
  __shared__ float placeholder_shared[768];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
      for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
        for (int nn_inner_init = 0; nn_inner_init < 4; ++nn_inner_init) {
          compute[(((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 128))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 129))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 130))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
          compute[((((((nn_outer_inner_init * 64) + (nn_inner_init * 16)) + (yy_outer_inner_init * 8)) + (xx_outer_inner_init * 4)) + 131))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int ry_outer_outer = 0; ry_outer_outer < 3; ++ry_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 384; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      pad_temp_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)))] = (((((1 <= (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) % 5376) / 192) + ry_outer_outer)) && ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) % 5376) / 192) + ry_outer_outer) < 29)) && (1 <= (((((int)blockIdx.x) >> 3) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) % 192) >> 5)))) && ((((((int)blockIdx.x) >> 3) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) % 192) >> 5)) < 29)) ? placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) / 192) * 7168) + (ry_outer_outer * 7168)) + ((((int)blockIdx.x) >> 3) * 1024)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) % 192) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) & 31)) - 7424))] : 0.000000e+00f);
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((ry_outer_outer * 6144) + ((((int)threadIdx.x) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((ry_outer_outer * 6144) + (((((int)threadIdx.x) + 112) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((((ry_outer_outer * 6144) + ((((int)threadIdx.x) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 1792))];
    placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[(((((ry_outer_outer * 6144) + (((((int)threadIdx.x) + 336) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((((ry_outer_outer * 6144) + ((((int)threadIdx.x) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 3584))];
    placeholder_shared[((((int)threadIdx.x) + 560))] = placeholder1[(((((ry_outer_outer * 6144) + (((((int)threadIdx.x) + 560) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    if (((int)threadIdx.x) < 96) {
      placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[((((((ry_outer_outer * 6144) + ((((int)threadIdx.x) >> 5) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) & 31)) + 5376))];
    }
    __syncthreads();
    for (int rx_outer_inner = 0; rx_outer_inner < 3; ++rx_outer_inner) {
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
          for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
            for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
              for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
                for (int nn_inner = 0; nn_inner < 4; ++nn_inner) {
                  compute[(((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)))] = (compute[(((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)))] + (pad_temp_shared[((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 128))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 128))] + (pad_temp_shared[(((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + (rc_outer_inner * 4)) + rc_inner) + 2688))] * placeholder_shared[(((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 1))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 1))] + (pad_temp_shared[((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((((int)threadIdx.x) & 15) * 2) + 1) >> 3) * 8)) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[((((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 129))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 129))] + (pad_temp_shared[(((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((((int)threadIdx.x) & 15) * 2) + 1) >> 3) * 8)) + (rc_outer_inner * 4)) + rc_inner) + 2688))] * placeholder_shared[((((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 2))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 2))] + (pad_temp_shared[(((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + (rc_outer_inner * 4)) + rc_inner) + 32))] * placeholder_shared[(((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 130))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 130))] + (pad_temp_shared[(((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + (rc_outer_inner * 4)) + rc_inner) + 2720))] * placeholder_shared[(((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 3))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 3))] + (pad_temp_shared[(((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((((int)threadIdx.x) & 15) * 2) + 1) >> 3) * 8)) + (rc_outer_inner * 4)) + rc_inner) + 32))] * placeholder_shared[((((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
                  compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 131))] = (compute[((((((nn_outer_inner * 64) + (nn_inner * 16)) + (yy_outer_inner * 8)) + (xx_outer_inner * 4)) + 131))] + (pad_temp_shared[(((((((((((nn_outer_inner * 21504) + (nn_inner * 5376)) + ((((int)threadIdx.x) >> 4) * 384)) + (yy_outer_inner * 192)) + (xx_outer_inner * 64)) + (rx_outer_inner * 32)) + (((((((int)threadIdx.x) & 15) * 2) + 1) >> 3) * 8)) + (rc_outer_inner * 4)) + rc_inner) + 2720))] * placeholder_shared[((((((rx_outer_inner * 256) + (rc_outer_inner * 128)) + (rc_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 8; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
          T_relu[(((((((((ax0_inner * 200704) + ((((int)threadIdx.x) >> 4) * 14336)) + (ax1_inner * 7168)) + ((((int)blockIdx.x) >> 3) * 1024)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = max((compute[(((((ax0_inner * 16) + (ax1_inner * 8)) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((((ax0_inner * 200704) + ((((int)threadIdx.x) >> 4) * 14336)) + (ax1_inner * 7168)) + ((((int)blockIdx.x) >> 3) * 1024)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 100352))] = max((compute[((((((ax0_inner * 16) + (ax1_inner * 8)) + (ax2_inner * 2)) + ax3_inner) + 128))] + placeholder2[(((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


