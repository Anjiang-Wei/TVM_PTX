
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[4608];
  __shared__ float placeholder_shared[432];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 3) * 73728) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + ((int)threadIdx.x)) / 576) * 9216)) + (((((int)blockIdx.x) & 7) >> 1) * 2304)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + ((int)threadIdx.x)) % 576) / 36) * 144)) + (rc_outer_outer * 36)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + ((int)threadIdx.x)) % 36)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 864) + ((((int)threadIdx.x) / 12) * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[((((((rc_outer_outer * 864) + ((((int)threadIdx.x) / 12) * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 192))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((((rc_outer_outer * 864) + ((((int)threadIdx.x) / 12) * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 288))] = placeholder1[((((((rc_outer_outer * 864) + ((((int)threadIdx.x) / 12) * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 576))];
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((((rc_outer_outer * 864) + ((((int)threadIdx.x) / 12) * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 768))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 36; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 144))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 288))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 432))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 36))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 180))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 324))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 468))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 72))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 216))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 360))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 504))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 108))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 252))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 396))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 12) * 576) + rc_outer_inner) + 540))] * placeholder_shared[(((rc_outer_inner * 12) + (((int)threadIdx.x) % 12)))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
    T_add[((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)))] = (placeholder2[((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)))] + (Conv2dOutput[(ax2_inner)] + placeholder3[((((((int)blockIdx.x) & 1) * 12) + (((int)threadIdx.x) % 12)))]));
    T_add[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 96))] = (placeholder2[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 96))] + (Conv2dOutput[((ax2_inner + 4))] + placeholder3[((((((int)blockIdx.x) & 1) * 12) + (((int)threadIdx.x) % 12)))]));
    T_add[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 192))] = (placeholder2[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 192))] + (Conv2dOutput[((ax2_inner + 8))] + placeholder3[((((((int)blockIdx.x) & 1) * 12) + (((int)threadIdx.x) % 12)))]));
    T_add[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 288))] = (placeholder2[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 1536)) + (((((int)blockIdx.x) & 7) >> 1) * 384)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 1) * 12)) + (((int)threadIdx.x) % 12)) + 288))] + (Conv2dOutput[((ax2_inner + 12))] + placeholder3[((((((int)blockIdx.x) & 1) * 12) + (((int)threadIdx.x) % 12)))]));
  }
}


