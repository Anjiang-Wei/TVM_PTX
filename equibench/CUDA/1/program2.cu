
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[56];
  __shared__ float PaddedInput_shared[50176];
  __shared__ float placeholder_shared[8192];
  for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
        Conv2dOutput[((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init) + 8))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init) + 16))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init) + 24))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init) + 32))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init) + 40))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_inner_init) + 48))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 112; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + ((int)threadIdx.x)) / 1792) * 7168) + ((((int)blockIdx.x) >> 3) * 3584)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + ((int)threadIdx.x)) % 1792) >> 7) * 256)) + (rc_outer_outer * 128)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + ((int)threadIdx.x)) & 127)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 19; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((int)threadIdx.x) >> 6)) < 128) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 448) + ((int)threadIdx.x)) < 8192) {
          placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 448) + ((int)threadIdx.x)))] = placeholder1[((((((rc_outer_outer * 65536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 3584)) + ((((int)threadIdx.x) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) & 63)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 64; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
            for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
              Conv2dOutput[((((yy_inner * 4) + (xx_inner * 2)) + ff_inner))] = (Conv2dOutput[((((yy_inner * 4) + (xx_inner * 2)) + ff_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 8))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 8))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner) + 7168))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 16))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 16))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner) + 14336))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 24))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 24))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner) + 21504))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 32))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 32))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner) + 28672))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 40))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 40))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner) + 35840))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 48))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_inner) + 48))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 224) * 3584) + (yy_inner * 1792)) + (((((int)threadIdx.x) % 224) >> 5) * 256)) + (xx_inner * 128)) + (rc_outer_inner * 2)) + rc_inner) + 43008))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ff_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))] = max(((Conv2dOutput[((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 57344))] = max(((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 8))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 57344))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 114688))] = max(((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 114688))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 172032))] = max(((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 24))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 172032))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 229376))] = max(((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 32))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 229376))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 286720))] = max(((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 40))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 286720))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 344064))] = max(((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 224) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 3) * 7168)) + (((((int)threadIdx.x) % 224) >> 5) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 344064))]), 0.000000e+00f);
      }
    }
  }
}


