
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[256];
  __shared__ float PaddedInput_shared[3072];
  __shared__ float placeholder_shared[48];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 32))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 64))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 96))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 128))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 160))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 192))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 224))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 33))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 65))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 97))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 129))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 161))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 193))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 225))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 34))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 66))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 98))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 130))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 162))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 194))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 226))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 35))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 67))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 99))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 131))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 163))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 195))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 227))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 36))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 68))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 100))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 132))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 164))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 196))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 228))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 37))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 69))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 101))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 133))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 165))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 197))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 229))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 38))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 70))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 102))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 134))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 166))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 198))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 230))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 39))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 71))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 103))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 135))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 167))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 199))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 8) + 231))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((int)threadIdx.x) + 64) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((int)threadIdx.x) + 128) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 12288))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((int)threadIdx.x) + 256) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((int)threadIdx.x) + 320) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 24576))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[((((((((int)threadIdx.x) + 448) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((int)threadIdx.x) + 512) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 576))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 36864))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[((((((((int)threadIdx.x) + 640) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 704))] = placeholder[((((((((int)threadIdx.x) + 704) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 49152))];
    PaddedInput_shared[((((int)threadIdx.x) + 832))] = placeholder[((((((((int)threadIdx.x) + 832) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 896))] = placeholder[((((((((int)threadIdx.x) + 896) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 960))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 61440))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[((((((((int)threadIdx.x) + 1024) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1088))] = placeholder[((((((((int)threadIdx.x) + 1088) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1152))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 73728))];
    PaddedInput_shared[((((int)threadIdx.x) + 1216))] = placeholder[((((((((int)threadIdx.x) + 1216) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[((((((((int)threadIdx.x) + 1280) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1344))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 86016))];
    PaddedInput_shared[((((int)threadIdx.x) + 1408))] = placeholder[((((((((int)threadIdx.x) + 1408) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1472))] = placeholder[((((((((int)threadIdx.x) + 1472) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 98304))];
    PaddedInput_shared[((((int)threadIdx.x) + 1600))] = placeholder[((((((((int)threadIdx.x) + 1600) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1664))] = placeholder[((((((((int)threadIdx.x) + 1664) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1728))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 110592))];
    PaddedInput_shared[((((int)threadIdx.x) + 1792))] = placeholder[((((((((int)threadIdx.x) + 1792) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1856))] = placeholder[((((((((int)threadIdx.x) + 1856) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 122880))];
    PaddedInput_shared[((((int)threadIdx.x) + 1984))] = placeholder[((((((((int)threadIdx.x) + 1984) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[((((((((int)threadIdx.x) + 2048) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2112))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 135168))];
    PaddedInput_shared[((((int)threadIdx.x) + 2176))] = placeholder[((((((((int)threadIdx.x) + 2176) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2240))] = placeholder[((((((((int)threadIdx.x) + 2240) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2304))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 147456))];
    PaddedInput_shared[((((int)threadIdx.x) + 2368))] = placeholder[((((((((int)threadIdx.x) + 2368) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2432))] = placeholder[((((((((int)threadIdx.x) + 2432) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2496))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 159744))];
    PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[((((((((int)threadIdx.x) + 2560) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2624))] = placeholder[((((((((int)threadIdx.x) + 2624) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2688))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 172032))];
    PaddedInput_shared[((((int)threadIdx.x) + 2752))] = placeholder[((((((((int)threadIdx.x) + 2752) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2816))] = placeholder[((((((((int)threadIdx.x) + 2816) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2880))] = placeholder[((((((((int)threadIdx.x) / 3) * 192) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 184320))];
    PaddedInput_shared[((((int)threadIdx.x) + 2944))] = placeholder[((((((((int)threadIdx.x) + 2944) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3008))] = placeholder[((((((((int)threadIdx.x) + 3008) / 3) * 192) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 384) + ((((int)threadIdx.x) >> 4) * 128)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 15)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        Conv2dOutput[((yy_outer_inner * 8))] = (Conv2dOutput[((yy_outer_inner * 8))] + (PaddedInput_shared[((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 32))] = (Conv2dOutput[(((yy_outer_inner * 8) + 32))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 48))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 64))] = (Conv2dOutput[(((yy_outer_inner * 8) + 64))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 768))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 96))] = (Conv2dOutput[(((yy_outer_inner * 8) + 96))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 816))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 128))] = (Conv2dOutput[(((yy_outer_inner * 8) + 128))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1536))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 160))] = (Conv2dOutput[(((yy_outer_inner * 8) + 160))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1584))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 192))] = (Conv2dOutput[(((yy_outer_inner * 8) + 192))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2304))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 224))] = (Conv2dOutput[(((yy_outer_inner * 8) + 224))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2352))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 1))] = (Conv2dOutput[(((yy_outer_inner * 8) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 33))] = (Conv2dOutput[(((yy_outer_inner * 8) + 33))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 48))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 65))] = (Conv2dOutput[(((yy_outer_inner * 8) + 65))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 768))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 97))] = (Conv2dOutput[(((yy_outer_inner * 8) + 97))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 816))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 129))] = (Conv2dOutput[(((yy_outer_inner * 8) + 129))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1536))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 161))] = (Conv2dOutput[(((yy_outer_inner * 8) + 161))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1584))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 193))] = (Conv2dOutput[(((yy_outer_inner * 8) + 193))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2304))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 225))] = (Conv2dOutput[(((yy_outer_inner * 8) + 225))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2352))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 2))] = (Conv2dOutput[(((yy_outer_inner * 8) + 2))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 3))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 34))] = (Conv2dOutput[(((yy_outer_inner * 8) + 34))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 51))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 66))] = (Conv2dOutput[(((yy_outer_inner * 8) + 66))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 771))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 98))] = (Conv2dOutput[(((yy_outer_inner * 8) + 98))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 819))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 130))] = (Conv2dOutput[(((yy_outer_inner * 8) + 130))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1539))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 162))] = (Conv2dOutput[(((yy_outer_inner * 8) + 162))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1587))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 194))] = (Conv2dOutput[(((yy_outer_inner * 8) + 194))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2307))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 226))] = (Conv2dOutput[(((yy_outer_inner * 8) + 226))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2355))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 3))] = (Conv2dOutput[(((yy_outer_inner * 8) + 3))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 3))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 35))] = (Conv2dOutput[(((yy_outer_inner * 8) + 35))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 51))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 67))] = (Conv2dOutput[(((yy_outer_inner * 8) + 67))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 771))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 99))] = (Conv2dOutput[(((yy_outer_inner * 8) + 99))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 819))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 131))] = (Conv2dOutput[(((yy_outer_inner * 8) + 131))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1539))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 163))] = (Conv2dOutput[(((yy_outer_inner * 8) + 163))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1587))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 195))] = (Conv2dOutput[(((yy_outer_inner * 8) + 195))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2307))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 227))] = (Conv2dOutput[(((yy_outer_inner * 8) + 227))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2355))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 4))] = (Conv2dOutput[(((yy_outer_inner * 8) + 4))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 96))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 36))] = (Conv2dOutput[(((yy_outer_inner * 8) + 36))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 144))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 68))] = (Conv2dOutput[(((yy_outer_inner * 8) + 68))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 864))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 100))] = (Conv2dOutput[(((yy_outer_inner * 8) + 100))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 912))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 132))] = (Conv2dOutput[(((yy_outer_inner * 8) + 132))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1632))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 164))] = (Conv2dOutput[(((yy_outer_inner * 8) + 164))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1680))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 196))] = (Conv2dOutput[(((yy_outer_inner * 8) + 196))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2400))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 228))] = (Conv2dOutput[(((yy_outer_inner * 8) + 228))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2448))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 5))] = (Conv2dOutput[(((yy_outer_inner * 8) + 5))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 96))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 37))] = (Conv2dOutput[(((yy_outer_inner * 8) + 37))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 69))] = (Conv2dOutput[(((yy_outer_inner * 8) + 69))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 864))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 101))] = (Conv2dOutput[(((yy_outer_inner * 8) + 101))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 912))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 133))] = (Conv2dOutput[(((yy_outer_inner * 8) + 133))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1632))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 165))] = (Conv2dOutput[(((yy_outer_inner * 8) + 165))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1680))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 197))] = (Conv2dOutput[(((yy_outer_inner * 8) + 197))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2400))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 229))] = (Conv2dOutput[(((yy_outer_inner * 8) + 229))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2448))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 6))] = (Conv2dOutput[(((yy_outer_inner * 8) + 6))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 99))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 38))] = (Conv2dOutput[(((yy_outer_inner * 8) + 38))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 147))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 70))] = (Conv2dOutput[(((yy_outer_inner * 8) + 70))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 867))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 102))] = (Conv2dOutput[(((yy_outer_inner * 8) + 102))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 915))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 134))] = (Conv2dOutput[(((yy_outer_inner * 8) + 134))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1635))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 166))] = (Conv2dOutput[(((yy_outer_inner * 8) + 166))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1683))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 198))] = (Conv2dOutput[(((yy_outer_inner * 8) + 198))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2403))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 230))] = (Conv2dOutput[(((yy_outer_inner * 8) + 230))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2451))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 7))] = (Conv2dOutput[(((yy_outer_inner * 8) + 7))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 99))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 39))] = (Conv2dOutput[(((yy_outer_inner * 8) + 39))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 147))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 71))] = (Conv2dOutput[(((yy_outer_inner * 8) + 71))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 867))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 103))] = (Conv2dOutput[(((yy_outer_inner * 8) + 103))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 915))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 135))] = (Conv2dOutput[(((yy_outer_inner * 8) + 135))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1635))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 167))] = (Conv2dOutput[(((yy_outer_inner * 8) + 167))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 1683))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 199))] = (Conv2dOutput[(((yy_outer_inner * 8) + 199))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2403))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 8) + 231))] = (Conv2dOutput[(((yy_outer_inner * 8) + 231))] + (PaddedInput_shared[(((((yy_outer_inner * 192) + ((((int)threadIdx.x) >> 3) * 6)) + rc_outer_inner) + 2451))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 2048))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 32))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 32768))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 64))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 34816))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 96))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 65536))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 128))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 67584))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 160))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 98304))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 192))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 4096) + ((((int)threadIdx.x) >> 3) * 256)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 100352))] = max((Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 224))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


