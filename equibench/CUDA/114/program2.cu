
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[896];
  __shared__ float PaddedInput_shared[7840];
  __shared__ float placeholder_shared[640];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 28; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 32))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 22))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 23))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 25))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 26))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 27))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 29))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 30))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 31))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 48; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 56))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 56) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 112))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 112) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 168))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 168) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 224) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 13440))];
    PaddedInput_shared[((((int)threadIdx.x) + 336))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 336) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 392) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 448) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 504))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 504) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 560))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 26880))];
    PaddedInput_shared[((((int)threadIdx.x) + 616))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 616) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 672))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 672) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 728))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 728) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 784))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 784) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 840))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 40320))];
    PaddedInput_shared[((((int)threadIdx.x) + 896))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 896) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 952))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 952) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1008))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1008) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1064))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1064) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1120))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 53760))];
    PaddedInput_shared[((((int)threadIdx.x) + 1176))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1176) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1232))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1232) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1288))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1288) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1344))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1344) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1400))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 67200))];
    PaddedInput_shared[((((int)threadIdx.x) + 1456))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1456) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1512))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1512) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1568))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1568) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1624))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1624) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1680))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 80640))];
    PaddedInput_shared[((((int)threadIdx.x) + 1736))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1736) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1792))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1792) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1848))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1848) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1904))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 1904) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1960))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 94080))];
    PaddedInput_shared[((((int)threadIdx.x) + 2016))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2016) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2072))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2072) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2128))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2128) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2184))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2184) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2240))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 107520))];
    PaddedInput_shared[((((int)threadIdx.x) + 2296))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2296) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2352))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2352) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2408))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2408) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2464))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2464) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2520))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 120960))];
    PaddedInput_shared[((((int)threadIdx.x) + 2576))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2576) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2632))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2632) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2688))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2688) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2744))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2744) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2800))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 134400))];
    PaddedInput_shared[((((int)threadIdx.x) + 2856))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2856) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2912))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2912) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2968))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 2968) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3024))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3024) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3080))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 147840))];
    PaddedInput_shared[((((int)threadIdx.x) + 3136))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3136) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3192))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3192) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3248))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3248) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3304))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3304) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3360))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 161280))];
    PaddedInput_shared[((((int)threadIdx.x) + 3416))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3416) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3472))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3472) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3528))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3528) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3584))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3584) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3640))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 174720))];
    PaddedInput_shared[((((int)threadIdx.x) + 3696))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3696) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3752))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3752) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3808))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3808) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3864))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3864) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3920))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 188160))];
    PaddedInput_shared[((((int)threadIdx.x) + 3976))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 3976) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4032))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4032) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4088))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4088) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4144))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4144) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4200))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 201600))];
    PaddedInput_shared[((((int)threadIdx.x) + 4256))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4256) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4312))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4312) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4368))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4368) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4424))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4424) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4480))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 215040))];
    PaddedInput_shared[((((int)threadIdx.x) + 4536))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4536) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4592))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4592) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4648))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4648) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4704))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4704) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4760))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 228480))];
    PaddedInput_shared[((((int)threadIdx.x) + 4816))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4816) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4872))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4872) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4928))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4928) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4984))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 4984) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5040))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 241920))];
    PaddedInput_shared[((((int)threadIdx.x) + 5096))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5096) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5152))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5152) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5208))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5208) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5264))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5264) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5320))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 255360))];
    PaddedInput_shared[((((int)threadIdx.x) + 5376))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5376) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5432))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5432) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5488))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5488) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5544))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5544) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5600))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 268800))];
    PaddedInput_shared[((((int)threadIdx.x) + 5656))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5656) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5712))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5712) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5768))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5768) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5824))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5824) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5880))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 282240))];
    PaddedInput_shared[((((int)threadIdx.x) + 5936))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5936) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 5992))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 5992) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6048))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6048) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6104))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6104) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6160))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 295680))];
    PaddedInput_shared[((((int)threadIdx.x) + 6216))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6216) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6272))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6272) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6328))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6328) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6384))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6384) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6440))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 309120))];
    PaddedInput_shared[((((int)threadIdx.x) + 6496))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6496) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6552))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6552) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6608))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6608) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6664))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6664) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6720))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 322560))];
    PaddedInput_shared[((((int)threadIdx.x) + 6776))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6776) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6832))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6832) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6888))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6888) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 6944))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 6944) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7000))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 336000))];
    PaddedInput_shared[((((int)threadIdx.x) + 7056))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7056) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7112))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7112) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7168))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7168) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7224))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7224) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7280))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 349440))];
    PaddedInput_shared[((((int)threadIdx.x) + 7336))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7336) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7392))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7392) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7448))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7448) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7504))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7504) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7560))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 376320) + ((((int)threadIdx.x) / 10) * 480)) + (rc_outer_outer * 10)) + (((int)threadIdx.x) % 10)) + 362880))];
    PaddedInput_shared[((((int)threadIdx.x) + 7616))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7616) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 6) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7672))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7672) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 2) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7728))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7728) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 8) % 10)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7784))] = placeholder[((((((((int)blockIdx.x) >> 1) * 376320) + (((((int)threadIdx.x) + 7784) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) + 4) % 10)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 56) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 112) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 168) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 224) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 280))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 280) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 24) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 336) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 16) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 392))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 392) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) + 8)))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 504))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 504) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 560))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 560) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    if (((int)threadIdx.x) < 24) {
      placeholder_shared[((((int)threadIdx.x) + 616))] = placeholder1[(((((rc_outer_outer * 1280) + (((((int)threadIdx.x) + 616) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) + 40)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 10; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 28; ++yy_outer_inner) {
        Conv2dOutput[((yy_outer_inner * 32))] = (Conv2dOutput[((yy_outer_inner * 32))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 1))] = (Conv2dOutput[(((yy_outer_inner * 32) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 2))] = (Conv2dOutput[(((yy_outer_inner * 32) + 2))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 3))] = (Conv2dOutput[(((yy_outer_inner * 32) + 3))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 4))] = (Conv2dOutput[(((yy_outer_inner * 32) + 4))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 4))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 5))] = (Conv2dOutput[(((yy_outer_inner * 32) + 5))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 5))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 6))] = (Conv2dOutput[(((yy_outer_inner * 32) + 6))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 6))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 7))] = (Conv2dOutput[(((yy_outer_inner * 32) + 7))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 7))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 16))] = (Conv2dOutput[(((yy_outer_inner * 32) + 16))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 17))] = (Conv2dOutput[(((yy_outer_inner * 32) + 17))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 18))] = (Conv2dOutput[(((yy_outer_inner * 32) + 18))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 19))] = (Conv2dOutput[(((yy_outer_inner * 32) + 19))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 20))] = (Conv2dOutput[(((yy_outer_inner * 32) + 20))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 4))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 21))] = (Conv2dOutput[(((yy_outer_inner * 32) + 21))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 5))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 22))] = (Conv2dOutput[(((yy_outer_inner * 32) + 22))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 6))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 23))] = (Conv2dOutput[(((yy_outer_inner * 32) + 23))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 7))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 8))] = (Conv2dOutput[(((yy_outer_inner * 32) + 8))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 8))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 9))] = (Conv2dOutput[(((yy_outer_inner * 32) + 9))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 9))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 10))] = (Conv2dOutput[(((yy_outer_inner * 32) + 10))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 10))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 11))] = (Conv2dOutput[(((yy_outer_inner * 32) + 11))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 11))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 12))] = (Conv2dOutput[(((yy_outer_inner * 32) + 12))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 12))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 13))] = (Conv2dOutput[(((yy_outer_inner * 32) + 13))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 13))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 14))] = (Conv2dOutput[(((yy_outer_inner * 32) + 14))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 14))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 15))] = (Conv2dOutput[(((yy_outer_inner * 32) + 15))] + (PaddedInput_shared[((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 15))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 24))] = (Conv2dOutput[(((yy_outer_inner * 32) + 24))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 8))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 25))] = (Conv2dOutput[(((yy_outer_inner * 32) + 25))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 9))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 26))] = (Conv2dOutput[(((yy_outer_inner * 32) + 26))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 10))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 27))] = (Conv2dOutput[(((yy_outer_inner * 32) + 27))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 11))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 28))] = (Conv2dOutput[(((yy_outer_inner * 32) + 28))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 12))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 29))] = (Conv2dOutput[(((yy_outer_inner * 32) + 29))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 13))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 30))] = (Conv2dOutput[(((yy_outer_inner * 32) + 30))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 14))]));
        Conv2dOutput[(((yy_outer_inner * 32) + 31))] = (Conv2dOutput[(((yy_outer_inner * 32) + 31))] + (PaddedInput_shared[(((((yy_outer_inner * 280) + ((((int)threadIdx.x) >> 2) * 20)) + rc_outer_inner) + 10))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 16)) + 15))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 28; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
        T_relu[(((((((((((int)blockIdx.x) >> 1) * 100352) + (ax1_inner * 3584)) + ((((int)threadIdx.x) >> 2) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 16)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 32) + (ax2_inner * 16)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 16)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


