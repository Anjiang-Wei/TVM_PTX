
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[64];
  __shared__ float PaddedInput_shared[112];
  __shared__ float placeholder_shared[128];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 240; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[((((int)threadIdx.x) * 32))] = placeholder[(((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 1))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 2))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 480))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 3))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 481))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 4))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 960))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 5))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 961))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 6))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 1440))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 7))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 1441))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 8))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 1920))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 9))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 1921))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 10))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 2400))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 11))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 2401))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 12))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 2880))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 13))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 2881))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 14))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 3360))];
    }
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 15))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 3361))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 16))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 3840))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 17))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 3841))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 18))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 4320))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 19))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 4321))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 20))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 4800))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 21))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 4801))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 22))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 5280))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 23))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 5281))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 24))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 5760))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 25))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 5761))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 26))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 6240))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 27))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 6241))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 28))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 6720))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 29))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 6721))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 30))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 7200))];
    }
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[(((((int)threadIdx.x) * 32) + 31))] = placeholder[((((((((int)blockIdx.x) >> 1) * 26880) + (((int)threadIdx.x) * 7680)) + (rc_outer_outer * 2)) + 7201))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 256) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((rc_outer_outer * 256) + (((((int)threadIdx.x) + 56) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((rc_outer_outer * 256) + (((((int)threadIdx.x) + 112) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) + 48)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 8))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 1))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 2))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 3))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 64))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 65))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 66))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 67))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 4))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 5))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 6))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 7))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 68))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 69))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 70))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 2) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 8) + 71))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 1) * 7168) + (ax1_inner * 3584)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 8) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 1) * 7168) + (ax1_inner * 3584)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner) + 896))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 1) * 7168) + (ax1_inner * 3584)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner) + 1792))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 32))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 1) * 7168) + (ax1_inner * 3584)) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner) + 2688))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 8)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


