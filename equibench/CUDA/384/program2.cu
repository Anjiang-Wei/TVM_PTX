
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[48];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 8; ++xx_inner_init) {
        for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
          Conv2dOutput[(((((yy_outer_inner_init * 32) + (yy_inner_init * 16)) + (xx_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_outer_inner_init * 32) + (yy_inner_init * 16)) + (xx_inner_init * 2)) + ff_inner_init) + 64))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 72; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + (((int)threadIdx.x) >> 1)) < 256) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) < 512) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 73728) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + (((int)threadIdx.x) >> 1)) >> 5) * 9216)) + ((((int)blockIdx.x) & 1) * 4608)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + (((int)threadIdx.x) >> 1)) & 31) * 144)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 48) {
        if (((int)threadIdx.x) < 24) {
          placeholder_shared[(((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[((((rc_outer_outer * 48) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 8; ++xx_inner) {
            for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
              Conv2dOutput[(((((yy_outer_inner * 32) + (yy_inner * 16)) + (xx_inner * 2)) + ff_inner))] = (Conv2dOutput[(((((yy_outer_inner * 32) + (yy_inner * 16)) + (xx_inner * 2)) + ff_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 24) * 256) + (yy_outer_inner * 128)) + (yy_inner * 64)) + (((((int)threadIdx.x) % 24) / 12) * 16)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 24) + ((((int)threadIdx.x) % 12) * 2)) + ff_inner))]));
              Conv2dOutput[((((((yy_outer_inner * 32) + (yy_inner * 16)) + (xx_inner * 2)) + ff_inner) + 64))] = (Conv2dOutput[((((((yy_outer_inner * 32) + (yy_inner * 16)) + (xx_inner * 2)) + ff_inner) + 64))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 24) * 256) + (yy_outer_inner * 128)) + (yy_inner * 64)) + (((((int)threadIdx.x) % 24) / 12) * 16)) + (xx_inner * 2)) + rc_outer_inner) + 32))] * placeholder_shared[((((rc_outer_inner * 24) + ((((int)threadIdx.x) % 12) * 2)) + ff_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[((((((((((((int)blockIdx.x) >> 1) * 12288) + ((((int)threadIdx.x) / 24) * 6144)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + (((((int)threadIdx.x) % 24) / 12) * 192)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 12) * 2)) + ax3_inner))] = (placeholder2[((((((((((((int)blockIdx.x) >> 1) * 12288) + ((((int)threadIdx.x) / 24) * 6144)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + (((((int)threadIdx.x) % 24) / 12) * 192)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 12) * 2)) + ax3_inner))] + (Conv2dOutput[((((ax1_inner * 16) + (ax2_inner * 2)) + ax3_inner))] + placeholder3[((((((int)threadIdx.x) % 12) * 2) + ax3_inner))]));
        T_add[(((((((((((((int)blockIdx.x) >> 1) * 12288) + ((((int)threadIdx.x) / 24) * 6144)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + (((((int)threadIdx.x) % 24) / 12) * 192)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 12) * 2)) + ax3_inner) + 384))] = (placeholder2[(((((((((((((int)blockIdx.x) >> 1) * 12288) + ((((int)threadIdx.x) / 24) * 6144)) + (ax1_inner * 1536)) + ((((int)blockIdx.x) & 1) * 768)) + (((((int)threadIdx.x) % 24) / 12) * 192)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 12) * 2)) + ax3_inner) + 384))] + (Conv2dOutput[(((((ax1_inner * 16) + (ax2_inner * 2)) + ax3_inner) + 64))] + placeholder3[((((((int)threadIdx.x) % 12) * 2) + ax3_inner))]));
      }
    }
  }
}


