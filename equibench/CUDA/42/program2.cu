
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[672];
  __shared__ float placeholder_shared[768];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 2))] = placeholder[((((((((((int)blockIdx.x) / 7) * 150528) + ((((int)threadIdx.x) / 168) * 75264)) + ((((int)blockIdx.x) % 7) * 10752)) + (((((int)threadIdx.x) % 168) / 6) * 384)) + (rc_outer_outer * 12)) + ((((int)threadIdx.x) % 6) * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder[((((((((((int)blockIdx.x) / 7) * 150528) + ((((((int)threadIdx.x) * 2) + 1) / 336) * 75264)) + ((((int)blockIdx.x) % 7) * 10752)) + (((((((int)threadIdx.x) * 2) + 1) % 336) / 12) * 384)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 2) + 1) % 12)))];
    if (((int)threadIdx.x) < 112) {
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 448))] = placeholder[((((((((((int)blockIdx.x) / 7) * 150528) + ((((((int)threadIdx.x) * 2) + 448) / 336) * 75264)) + ((((int)blockIdx.x) % 7) * 10752)) + ((((((int)threadIdx.x) * 2) + 112) / 12) * 384)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 2) + 4) % 12)))];
    }
    if (((int)threadIdx.x) < 112) {
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 449))] = placeholder[((((((((((int)blockIdx.x) / 7) * 150528) + ((((((int)threadIdx.x) * 2) + 449) / 336) * 75264)) + ((((int)blockIdx.x) % 7) * 10752)) + ((((((int)threadIdx.x) * 2) + 113) / 12) * 384)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 2) + 5) % 12)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 768) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 768) + ((int)threadIdx.x)) + 224))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 768) + ((int)threadIdx.x)) + 448))];
    if (((int)threadIdx.x) < 96) {
      placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[((((rc_outer_outer * 768) + ((int)threadIdx.x)) + 672))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 12))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 12))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 336))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 336))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 168))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 168))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 504))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 504))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 337))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 337))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 169))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 169))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 505))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 505))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 338))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 338))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 170))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 170))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 506))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 506))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 339))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 339))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 171))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 171))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 507))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 507))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 340))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 340))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 172))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 172))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 508))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 508))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 341))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 341))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 173))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 173))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 509))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 509))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 342))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 342))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 174))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 174))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 510))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 510))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 343))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 343))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 175))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 175))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 511))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 511))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 512))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 544))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 344))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 512))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 344))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 544))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 176))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 512))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 176))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 544))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 512))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 512))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 512))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 544))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 576))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 608))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 345))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 576))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 345))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 608))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 177))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 576))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 177))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 608))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 513))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 576))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 513))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 608))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 640))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 672))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 346))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 640))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 346))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 672))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 178))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 640))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 178))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 672))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 514))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 640))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 514))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 672))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 704))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 736))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 347))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 704))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 347))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 736))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 179))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 704))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 179))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 736))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 515))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 704))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 515))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 736))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 12))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 12))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 336))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 336))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 168))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 168))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 504))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 504))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 337))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 337))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 169))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 169))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 505))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 505))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 338))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 338))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 170))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 170))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 506))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 506))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 339))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 339))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 171))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 171))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 507))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 507))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 340))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 340))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 172))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 172))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 508))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 508))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 341))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 341))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 173))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 173))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 509))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 509))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 342))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 342))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 174))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 174))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 510))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 510))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 343))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 343))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 175))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 175))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 511))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 511))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 513))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 545))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 344))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 513))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 344))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 545))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 176))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 513))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 176))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 545))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 512))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 513))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 512))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 545))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 577))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 609))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 345))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 577))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 345))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 609))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 177))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 577))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 177))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 609))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 513))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 577))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 513))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 609))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 641))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 673))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 346))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 641))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 346))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 673))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 178))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 641))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 178))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 673))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 514))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 641))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 514))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 673))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 705))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 737))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 347))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 705))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 347))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 737))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 179))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 705))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 179))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 737))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 515))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 705))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 12) + 515))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 737))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_add[((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = (placeholder2[((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] + (Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder3[((((((int)threadIdx.x) & 15) * 2) + ax3_inner))]));
      T_add[(((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))] = (placeholder2[(((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))] + (Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 4))] + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax3_inner) + 32))]));
      T_add[(((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 12544))] = (placeholder2[(((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 12544))] + (Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 8))] + placeholder3[((((((int)threadIdx.x) & 15) * 2) + ax3_inner))]));
      T_add[(((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 12576))] = (placeholder2[(((((((((((int)blockIdx.x) / 7) * 25088) + ((((int)blockIdx.x) % 7) * 1792)) + (ax1_inner * 896)) + ((((int)threadIdx.x) >> 4) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 12576))] + (Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 12))] + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax3_inner) + 32))]));
    }
  }
}


