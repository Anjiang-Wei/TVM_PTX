
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[720];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float placeholder_shared[128];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 3; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
        for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
          Conv2dOutput[(((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 72))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 144))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 216))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 288))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 360))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 432))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 504))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 576))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 24) + (xx_outer_inner_init * 12)) + (xx_inner_init * 4)) + ff_inner_init) + 648))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 40; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 400) + (((int)threadIdx.x) * 10)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) < 900) {
          if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1600) + (((int)threadIdx.x) * 40)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 3600) {
            if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 90) {
              PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1600) + (((int)threadIdx.x) * 40)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 102400) + (((int)threadIdx.x) * 2560)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2) * 256)) + (rc_outer_outer * 4)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s & 3)))];
            }
          }
        }
      }
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 5) * 512)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 40))] = placeholder1[(((((rc_outer_outer * 2048) + (((((int)threadIdx.x) + 40) >> 5) * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 8) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[(((((rc_outer_outer * 2048) + (((((int)threadIdx.x) + 80) >> 5) * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[(((((rc_outer_outer * 2048) + (((((int)threadIdx.x) + 120) >> 5) * 512)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 24)))];
    }
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 3; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
            for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
              Conv2dOutput[(((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner))] = (Conv2dOutput[(((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner))] + (PaddedInput_shared[((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 72))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 72))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 360))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 144))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 144))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 720))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 216))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 216))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 1080))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 288))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 288))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 1440))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 360))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 360))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 1800))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 432))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 432))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 2160))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 504))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 504))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 2520))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 576))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 576))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 2880))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
              Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 648))] = (Conv2dOutput[((((((yy_inner * 24) + (xx_outer_inner * 12)) + (xx_inner * 4)) + ff_inner) + 648))] + (PaddedInput_shared[(((((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 24)) + (xx_outer_inner * 12)) + (xx_inner * 4)) + rc_inner) + 3240))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 6; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[(((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max(((Conv2dOutput[((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[(((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 46080))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 72))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 46080))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 92160))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 144))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 92160))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 138240))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 216))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 138240))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 184320))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 288))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 184320))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 230400))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 360))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 230400))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 276480))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 432))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 276480))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 322560))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 504))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 322560))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 368640))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 576))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 368640))]), 0.000000e+00f);
        T_relu[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 414720))] = max(((Conv2dOutput[(((((ax1_inner * 24) + (ax2_inner * 4)) + ax3_inner) + 648))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]) + placeholder3[((((((((ax1_inner * 15360) + ((((int)threadIdx.x) >> 3) * 3072)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 414720))]), 0.000000e+00f);
      }
    }
  }
}


