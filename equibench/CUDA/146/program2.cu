
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[240];
  __shared__ float PaddedInput_shared[1440];
  __shared__ float placeholder_shared[8192];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 15; ++yy_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 4)) + ff_outer_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 60; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 1440) {
        if (((int)threadIdx.x) < 24) {
          PaddedInput_shared[(((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[(((((((((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 24) * 15360) + (((int)blockIdx.x) * 3072)) + (((((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 24) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 7)))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
      if (((((int)threadIdx.x) * 32) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 8192) {
        if (((int)threadIdx.x) < 256) {
          placeholder_shared[(((((int)threadIdx.x) * 32) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[((((rc_outer_outer * 8192) + (((int)threadIdx.x) * 32)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 15; ++yy_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
            for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              Conv2dOutput[((((nn_outer_inner * 60) + (yy_outer_inner * 4)) + ff_outer_inner))] = (Conv2dOutput[((((nn_outer_inner * 60) + (yy_outer_inner * 4)) + ff_outer_inner))] + (PaddedInput_shared[((((((nn_outer_inner * 360) + (yy_outer_inner * 24)) + ((((int)threadIdx.x) >> 8) * 8)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 2048) + (rc_inner * 1024)) + ((((int)threadIdx.x) & 255) * 4)) + ff_outer_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 15; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[((((((ax0_inner * 230400) + (ax1_inner * 15360)) + (((int)blockIdx.x) * 3072)) + (((int)threadIdx.x) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 60) + (ax1_inner * 4)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 255) * 4) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


