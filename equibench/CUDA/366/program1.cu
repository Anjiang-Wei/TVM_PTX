
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[2048];
  __shared__ float PaddedInput_shared[8192];
  __shared__ float placeholder_shared[640];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 4; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 4; ++yy_inner_init) {
          Conv2dOutput[(((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 512))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1024))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1536))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 513))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1025))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1537))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 2))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 514))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1026))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1538))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 3))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 515))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1027))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1539))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 4))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 516))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1028))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1540))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 5))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 517))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1029))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1541))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 6))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 518))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1030))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1542))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 7))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 519))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1031))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 256) + (yy_inner_init * 64)) + (xx_outer_inner_init * 16)) + (ff_outer_inner_init * 8)) + 1543))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 30; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8192) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8191) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 1))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 1))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8190) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 2))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 2))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8189) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 3))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 3))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8188) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 4))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 4))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8187) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 5))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 5))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8186) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 6))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 6))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8185) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 7))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((int)threadIdx.x) >> 3)) & 15) * 3840)) + ((((int)threadIdx.x) & 7) * 480)) + (rc_outer_outer * 8)) + 7))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8184) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8183) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 1))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 1))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8182) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 2))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 2))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8181) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 3))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 3))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8180) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 4))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 4))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8179) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 5))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 5))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8178) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 6))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 6))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) < 1023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 640) + (((int)threadIdx.x) * 16)) < 8177) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 512) {
            PaddedInput_shared[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 128)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 8)) + 7))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + 1) >> 8) * 61440) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 5) + (((((int)threadIdx.x) * 2) + 1) >> 4)) & 15) * 3840)) + ((((((int)threadIdx.x) * 2) + 1) & 15) * 240)) + (rc_outer_outer * 8)) + 7))];
          }
        }
      }
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 640) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 40))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 40))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 80))];
    placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 120))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 160))];
    placeholder_shared[((((int)threadIdx.x) + 200))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 200))];
    placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 240))];
    placeholder_shared[((((int)threadIdx.x) + 280))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 280))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 320))];
    placeholder_shared[((((int)threadIdx.x) + 360))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 360))];
    placeholder_shared[((((int)threadIdx.x) + 400))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 400))];
    placeholder_shared[((((int)threadIdx.x) + 440))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 440))];
    placeholder_shared[((((int)threadIdx.x) + 480))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 480))];
    placeholder_shared[((((int)threadIdx.x) + 520))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 520))];
    placeholder_shared[((((int)threadIdx.x) + 560))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 560))];
    placeholder_shared[((((int)threadIdx.x) + 600))] = placeholder1[((((rc_outer_outer * 640) + ((int)threadIdx.x)) + 600))];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
            for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
              Conv2dOutput[(((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)))] = (Conv2dOutput[(((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 512))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 512))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1024))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1024))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1536))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1536))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 1))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 513))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 513))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 1))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1025))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1025))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 1))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1537))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1537))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 1))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 2))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 2))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 2))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 514))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 514))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 2))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1026))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1026))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 2))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1538))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1538))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 2))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 3))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 3))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 3))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 515))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 515))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 3))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1027))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1027))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 3))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1539))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1539))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 3))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 4))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 4))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 4))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 516))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 516))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 4))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1028))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1028))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 4))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1540))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1540))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 4))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 5))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 5))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 5))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 517))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 517))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 5))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1029))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1029))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 5))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1541))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1541))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 5))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 6))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 6))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 6))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 518))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 518))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 6))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1030))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1030))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 6))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1542))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1542))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 6))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 7))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 7))] + (PaddedInput_shared[(((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 7))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 519))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 519))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 1024))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 7))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1031))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1031))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 4096))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 7))]));
              Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1543))] = (Conv2dOutput[((((((nn_inner * 256) + (yy_inner * 64)) + (xx_outer_inner * 16)) + (ff_outer_inner * 8)) + 1543))] + (PaddedInput_shared[((((((((nn_inner * 2048) + ((((int)threadIdx.x) / 20) * 512)) + (yy_inner * 128)) + (((((int)threadIdx.x) % 20) / 5) * 32)) + (xx_outer_inner * 8)) + rc_inner) + 5120))] * placeholder_shared[(((((rc_inner * 80) + ((((int)threadIdx.x) % 5) * 16)) + (ff_outer_inner * 8)) + 7))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
          T_add[((((((((ax0_inner * 20480) + ((((int)threadIdx.x) / 20) * 5120)) + (ax1_inner * 1280)) + (((((int)threadIdx.x) % 20) / 5) * 320)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 5) * 16)) + ax3_inner))] = (Conv2dOutput[(((((ax0_inner * 256) + (ax1_inner * 64)) + (ax2_inner * 16)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) % 5) * 16) + ax3_inner))]);
          T_add[(((((((((ax0_inner * 20480) + ((((int)threadIdx.x) / 20) * 5120)) + (ax1_inner * 1280)) + (((((int)threadIdx.x) % 20) / 5) * 320)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 5) * 16)) + ax3_inner) + 10240))] = (Conv2dOutput[((((((ax0_inner * 256) + (ax1_inner * 64)) + (ax2_inner * 16)) + ax3_inner) + 512))] + placeholder2[((((((int)threadIdx.x) % 5) * 16) + ax3_inner))]);
          T_add[(((((((((ax0_inner * 20480) + ((((int)threadIdx.x) / 20) * 5120)) + (ax1_inner * 1280)) + (((((int)threadIdx.x) % 20) / 5) * 320)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 5) * 16)) + ax3_inner) + 40960))] = (Conv2dOutput[((((((ax0_inner * 256) + (ax1_inner * 64)) + (ax2_inner * 16)) + ax3_inner) + 1024))] + placeholder2[((((((int)threadIdx.x) % 5) * 16) + ax3_inner))]);
          T_add[(((((((((ax0_inner * 20480) + ((((int)threadIdx.x) / 20) * 5120)) + (ax1_inner * 1280)) + (((((int)threadIdx.x) % 20) / 5) * 320)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 5) * 16)) + ax3_inner) + 51200))] = (Conv2dOutput[((((((ax0_inner * 256) + (ax1_inner * 64)) + (ax2_inner * 16)) + ax3_inner) + 1536))] + placeholder2[((((((int)threadIdx.x) % 5) * 16) + ax3_inner))]);
        }
      }
    }
  }
}


