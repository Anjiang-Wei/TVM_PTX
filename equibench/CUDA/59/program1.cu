
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[24];
  __shared__ float PaddedInput_shared[48];
  __shared__ float placeholder_shared[288];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 384; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 48) {
      PaddedInput_shared[(((int)threadIdx.x))] = (((1 <= (((((int)blockIdx.x) >> 2) * 4) + (((int)threadIdx.x) >> 3))) && ((((((int)blockIdx.x) >> 2) * 4) + (((int)threadIdx.x) >> 3)) < 9)) ? placeholder[((((((((int)blockIdx.x) >> 2) * 12288) + (((int)threadIdx.x) * 384)) + rc_outer_outer) - 3072))] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[((((int)threadIdx.x) * 27))] = placeholder1[(((((((((int)threadIdx.x) * 27) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) * 27) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 1))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 1) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 1) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 2))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 2) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 2) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 3))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 3) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 3) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 4))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 4) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 4) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 5))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 5) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 5) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 6))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 6) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 6) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 7))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 7) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 7) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 8))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 8) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 8) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 9))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 9) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 9) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 10))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 10) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 10) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 11))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 11) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 11) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 12))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 12) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 12) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 13))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 13) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 13) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 14))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 14) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 14) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 15))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 15) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 15) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 16))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 16) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 16) % 96)))];
    }
    if (((int)threadIdx.x) < 11) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 17))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 17) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 17) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 18))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 18) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 18) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 19))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 19) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 19) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 20))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 20) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 20) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 21))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 21) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 21) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 22))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 22) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 22) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 23))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 23) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 23) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 24))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 24) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 24) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 25))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 25) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 25) % 96)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.x) * 27) + 26))] = placeholder1[((((((((((int)threadIdx.x) * 27) + 26) / 96) * 147456) + (rc_outer_outer * 384)) + ((((int)blockIdx.x) & 3) * 96)) + (((((int)threadIdx.x) * 27) + 26) % 96)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 3))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 1))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 2))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 96))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 97))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 98))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 20))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 20))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 20))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 192))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 193))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 3) + 194))]));
  }
  for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 3; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 2) * 12288) + ((((int)threadIdx.x) >> 5) * 3072)) + (ax2_inner * 384)) + ((((int)blockIdx.x) & 3) * 96)) + ((((int)threadIdx.x) & 31) * 3)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 3) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 96) + ((((int)threadIdx.x) & 31) * 3)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


