
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 240))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 240))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[32];
  __shared__ float PaddedInput_shared[10200];
  __shared__ float placeholder_shared[120];
  for (int c_c_outer_inner_init = 0; c_c_outer_inner_init < 2; ++c_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
      DepthwiseConv2d_local[(((i_c_inner_init * 2) + c_c_outer_inner_init))] = 0.000000e+00f;
      DepthwiseConv2d_local[((((i_c_inner_init * 2) + c_c_outer_inner_init) + 16))] = 0.000000e+00f;
    }
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 128; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 40)) < 255) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)) < 10200) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)))] = (((1 <= ((((((int)blockIdx.x) / 12) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 40)) / 17)) + di_outer_outer)) && (1 <= ((((((int)blockIdx.x) % 12) / 6) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 40)) % 17)))) ? placeholder[((((((((((((int)blockIdx.x) / 12) * 122880) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 40)) / 17) * 7680)) + (di_outer_outer * 7680)) + (((((int)blockIdx.x) % 12) / 6) * 3840)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) / 40)) % 17) * 240)) + ((((int)blockIdx.x) % 6) * 40)) + (((int)threadIdx.x) % 40)) - 7920))] : 0.000000e+00f);
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + (((int)threadIdx.x) / 40)) < 3) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 80) + ((int)threadIdx.x)) < 120) {
          placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 80) + ((int)threadIdx.x)))] = placeholder1[((((((di_outer_outer * 720) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 480)) + ((((int)threadIdx.x) / 40) * 240)) + ((((int)blockIdx.x) % 6) * 40)) + (((int)threadIdx.x) % 40)))];
        }
      }
    }
    __syncthreads();
    for (int c_c_outer_inner = 0; c_c_outer_inner < 2; ++c_c_outer_inner) {
      for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
        for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
          DepthwiseConv2d_local[(((i_c_inner * 2) + c_c_outer_inner))] = (DepthwiseConv2d_local[(((i_c_inner * 2) + c_c_outer_inner))] + (PaddedInput_shared[((((((i_c_inner * 1360) + ((((int)threadIdx.x) / 10) * 80)) + (dj_inner * 40)) + ((((int)threadIdx.x) % 10) * 2)) + c_c_outer_inner))] * placeholder_shared[((((dj_inner * 40) + ((((int)threadIdx.x) % 10) * 2)) + c_c_outer_inner))]));
          DepthwiseConv2d_local[((((i_c_inner * 2) + c_c_outer_inner) + 16))] = (DepthwiseConv2d_local[((((i_c_inner * 2) + c_c_outer_inner) + 16))] + (PaddedInput_shared[(((((((i_c_inner * 1360) + ((((int)threadIdx.x) / 10) * 80)) + (dj_inner * 40)) + ((((int)threadIdx.x) % 10) * 2)) + c_c_outer_inner) + 20))] * placeholder_shared[(((((dj_inner * 40) + ((((int)threadIdx.x) % 10) * 2)) + c_c_outer_inner) + 20))]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int c_inner = 0; c_inner < 2; ++c_inner) {
      DepthwiseConv2d[(((((((((((int)blockIdx.x) / 12) * 30720) + (i_inner * 3840)) + (((((int)blockIdx.x) % 12) / 6) * 1920)) + ((((int)threadIdx.x) / 10) * 240)) + ((((int)blockIdx.x) % 6) * 40)) + ((((int)threadIdx.x) % 10) * 2)) + c_inner))] = DepthwiseConv2d_local[(((i_inner * 2) + c_inner))];
      DepthwiseConv2d[((((((((((((int)blockIdx.x) / 12) * 30720) + (i_inner * 3840)) + (((((int)blockIdx.x) % 12) / 6) * 1920)) + ((((int)threadIdx.x) / 10) * 240)) + ((((int)blockIdx.x) % 6) * 40)) + ((((int)threadIdx.x) % 10) * 2)) + c_inner) + 20))] = DepthwiseConv2d_local[((((i_inner * 2) + c_inner) + 16))];
    }
  }
}


