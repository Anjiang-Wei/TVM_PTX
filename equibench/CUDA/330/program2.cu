
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((int)blockIdx.x))] = ((DepthwiseConv2d[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 240))]) * (max(min(((DepthwiseConv2d[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 240))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[80];
  __shared__ float PaddedInput_shared[10230];
  __shared__ float placeholder_shared[30];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 8; ++j_c_outer_inner_init) {
    for (int c_c_inner_init = 0; c_c_inner_init < 5; ++c_c_inner_init) {
      DepthwiseConv2d_local[(((j_c_outer_inner_init * 5) + c_c_inner_init))] = 0.000000e+00f;
      DepthwiseConv2d_local[((((j_c_outer_inner_init * 5) + c_c_inner_init) + 40))] = 0.000000e+00f;
    }
  }
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 10; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 35; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1120) + (((int)threadIdx.x) * 35)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 10230) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) < 293) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1120) + (((int)threadIdx.x) * 35)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 310) * 310) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + (((((int)threadIdx.x) * 35) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 10)) % 31) * 10)) + (((((int)threadIdx.x) * 35) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 10)))] = (((310 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1120) + (((int)threadIdx.x) * 35)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s)) && (1 <= (dj_outer_outer + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + (((((int)threadIdx.x) * 35) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 10)) % 31)))) ? placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1120) + (((int)threadIdx.x) * 35)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 310) * 7680) + (dj_outer_outer * 240)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + (((((int)threadIdx.x) * 35) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 10)) % 31) * 240)) + (((int)blockIdx.x) * 10)) + (((((int)threadIdx.x) * 35) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 10)) - 7920))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((int)threadIdx.x) < 30) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((int)threadIdx.x) / 10) * 720) + (dj_outer_outer * 240)) + (((int)blockIdx.x) * 10)) + (((int)threadIdx.x) % 10)))];
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 8; ++j_c_outer_inner) {
      for (int di_inner = 0; di_inner < 3; ++di_inner) {
        for (int c_c_inner = 0; c_c_inner < 5; ++c_c_inner) {
          DepthwiseConv2d_local[(((j_c_outer_inner * 5) + c_c_inner))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 5) + c_c_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 620) + (di_inner * 310)) + ((((int)threadIdx.x) & 1) * 160)) + (j_c_outer_inner * 20)) + c_c_inner))] * placeholder_shared[(((di_inner * 10) + c_c_inner))]));
          DepthwiseConv2d_local[((((j_c_outer_inner * 5) + c_c_inner) + 40))] = (DepthwiseConv2d_local[((((j_c_outer_inner * 5) + c_c_inner) + 40))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 1) * 620) + (di_inner * 310)) + ((((int)threadIdx.x) & 1) * 160)) + (j_c_outer_inner * 20)) + c_c_inner) + 5))] * placeholder_shared[((((di_inner * 10) + c_c_inner) + 5))]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 8; ++j_inner) {
    for (int c_inner = 0; c_inner < 5; ++c_inner) {
      DepthwiseConv2d[(((((((int)threadIdx.x) * 1920) + (j_inner * 240)) + (((int)blockIdx.x) * 10)) + c_inner))] = DepthwiseConv2d_local[(((j_inner * 5) + c_inner))];
      DepthwiseConv2d[((((((((int)threadIdx.x) * 1920) + (j_inner * 240)) + (((int)blockIdx.x) * 10)) + c_inner) + 5))] = DepthwiseConv2d_local[((((j_inner * 5) + c_inner) + 40))];
    }
  }
}


