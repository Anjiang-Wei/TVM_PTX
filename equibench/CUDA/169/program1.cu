
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[2048];
  __shared__ float placeholder_shared[32];
  for (int xx_inner_init = 0; xx_inner_init < 8; ++xx_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 16; ++ff_inner_init) {
      Conv2dOutput[(((xx_inner_init * 16) + ff_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + ff_inner_init) + 128))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + ff_inner_init) + 256))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + ff_inner_init) + 384))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 262144) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4096)) + (((((int)blockIdx.x) & 15) >> 3) * 2048)) + ((((int)threadIdx.x) >> 1) * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 32) {
        if (((int)threadIdx.x) < 16) {
          placeholder_shared[(((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 256) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15)))];
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int xx_inner = 0; xx_inner < 8; ++xx_inner) {
        for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
          Conv2dOutput[(((xx_inner * 16) + ff_inner))] = (Conv2dOutput[(((xx_inner * 16) + ff_inner))] + (PaddedInput_shared[((((((int)threadIdx.x) * 16) + (xx_inner * 2)) + rc_inner))] * placeholder_shared[(((rc_inner * 16) + ff_inner))]));
          Conv2dOutput[((((xx_inner * 16) + ff_inner) + 128))] = (Conv2dOutput[((((xx_inner * 16) + ff_inner) + 128))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 16) + (xx_inner * 2)) + rc_inner) + 512))] * placeholder_shared[(((rc_inner * 16) + ff_inner))]));
          Conv2dOutput[((((xx_inner * 16) + ff_inner) + 256))] = (Conv2dOutput[((((xx_inner * 16) + ff_inner) + 256))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 16) + (xx_inner * 2)) + rc_inner) + 1024))] * placeholder_shared[(((rc_inner * 16) + ff_inner))]));
          Conv2dOutput[((((xx_inner * 16) + ff_inner) + 384))] = (Conv2dOutput[((((xx_inner * 16) + ff_inner) + 384))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 16) + (xx_inner * 2)) + rc_inner) + 1536))] * placeholder_shared[(((rc_inner * 16) + ff_inner))]));
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 3) * 2048)) + ((((int)threadIdx.x) & 1) * 1024)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 16) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 3) * 2048)) + ((((int)threadIdx.x) & 1) * 1024)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ax3_inner) + 65536))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 128))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 3) * 2048)) + ((((int)threadIdx.x) & 1) * 1024)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ax3_inner) + 131072))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 256))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 262144) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 3) * 2048)) + ((((int)threadIdx.x) & 1) * 1024)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ax3_inner) + 196608))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 384))] + placeholder2[((((((int)blockIdx.x) & 7) * 16) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


