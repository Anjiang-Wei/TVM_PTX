
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[112];
  __shared__ float placeholder_shared[1024];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[((((int)threadIdx.x) * 4))] = placeholder[((((((((int)threadIdx.x) >> 1) * 3584) + (((int)blockIdx.x) * 512)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder[((((((((((int)threadIdx.x) * 4) + 1) >> 3) * 3584) + (((int)blockIdx.x) * 512)) + (rc_outer_outer * 8)) + (((((int)threadIdx.x) * 4) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder[((((((((((int)threadIdx.x) * 4) + 2) >> 3) * 3584) + (((int)blockIdx.x) * 512)) + (rc_outer_outer * 8)) + (((((int)threadIdx.x) * 4) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder[((((((((((int)threadIdx.x) * 4) + 3) >> 3) * 3584) + (((int)blockIdx.x) * 512)) + (rc_outer_outer * 8)) + (((((int)threadIdx.x) * 4) + 3) & 7)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 1024) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 1024) + ((int)threadIdx.x)) + 224))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 1024) + ((int)threadIdx.x)) + 448))];
    placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[((((rc_outer_outer * 1024) + ((int)threadIdx.x)) + 672))];
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((rc_outer_outer * 1024) + ((int)threadIdx.x)) + 896))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 4))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 56))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 4))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 2))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 3))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 3))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 128))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 128))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 129))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 129))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 130))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 130))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 131))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 131))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 256))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 58))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 256))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 257))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 58))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 257))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 258))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 58))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 258))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 259))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 58))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 259))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 384))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 59))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 384))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 385))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 59))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 385))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 386))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 59))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 386))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 387))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 59))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 387))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 512))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 60))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 512))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 513))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 60))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 513))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 514))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 60))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 514))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 515))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 60))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 515))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 640))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 61))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 640))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 641))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 61))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 641))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 642))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 61))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 642))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 643))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 61))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 643))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 768))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 62))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 768))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 769))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 62))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 769))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 770))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 62))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 770))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 771))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 62))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 771))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 896))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 63))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 896))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 897))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 63))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 897))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 898))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 63))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 898))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 899))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 8) + 63))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 899))]));
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[((((((ax0_inner * 6272) + ((((int)threadIdx.x) >> 5) * 896)) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax0_inner * 4) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 31) * 4) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


