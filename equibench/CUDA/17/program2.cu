
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[120];
  __shared__ float PaddedInput_shared[10736];
  __shared__ float placeholder_shared[18];
  for (int i_inner_init = 0; i_inner_init < 10; ++i_inner_init) {
    DepthwiseConv2d[((i_inner_init * 6))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 60))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 1))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 61))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 2))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 62))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 3))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 63))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 4))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 64))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 5))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_inner_init * 6) + 65))] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 135; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) < 5368) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)) < 10736) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)))] = (((((1 <= ((((((int)blockIdx.x) % 96) >> 4) * 20) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) % 2684) / 122))) && (((((((int)blockIdx.x) % 96) >> 4) * 20) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) % 2684) / 122)) < 121)) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) % 122))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) % 122) < 121)) ? placeholder[((((((((((((int)blockIdx.x) / 96) * 921600) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) / 2684) * 460800)) + (((((int)blockIdx.x) % 96) >> 4) * 76800)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) % 2684) / 122) * 3840)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) >> 1)) % 122) * 32)) + ((((int)blockIdx.x) & 15) * 2)) + (((int)threadIdx.x) & 1)) - 3872))] : 0.000000e+00f);
      }
    }
  }
  if (((int)threadIdx.x) < 6) {
    placeholder_shared[((((int)threadIdx.x) * 3))] = placeholder1[((((((((int)threadIdx.x) * 3) >> 1) * 32) + ((((int)blockIdx.x) & 15) * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
  }
  if (((int)threadIdx.x) < 6) {
    placeholder_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder1[(((((((((int)threadIdx.x) * 3) + 1) >> 1) * 32) + ((((int)blockIdx.x) & 15) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
  }
  if (((int)threadIdx.x) < 6) {
    placeholder_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder1[(((((((((int)threadIdx.x) * 3) >> 1) * 32) + ((((int)blockIdx.x) & 15) * 2)) + ((((int)threadIdx.x) * 3) & 1)) + 32))];
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int dj_outer_inner = 0; dj_outer_inner < 3; ++dj_outer_inner) {
      for (int i_inner = 0; i_inner < 10; ++i_inner) {
        DepthwiseConv2d[((i_inner * 6))] = (DepthwiseConv2d[((i_inner * 6))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)))] * placeholder_shared[(((di_outer_inner * 6) + (dj_outer_inner * 2)))]));
        DepthwiseConv2d[(((i_inner * 6) + 60))] = (DepthwiseConv2d[(((i_inner * 6) + 60))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2440))] * placeholder_shared[(((di_outer_inner * 6) + (dj_outer_inner * 2)))]));
        DepthwiseConv2d[(((i_inner * 6) + 1))] = (DepthwiseConv2d[(((i_inner * 6) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 1))] * placeholder_shared[((((di_outer_inner * 6) + (dj_outer_inner * 2)) + 1))]));
        DepthwiseConv2d[(((i_inner * 6) + 61))] = (DepthwiseConv2d[(((i_inner * 6) + 61))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2441))] * placeholder_shared[((((di_outer_inner * 6) + (dj_outer_inner * 2)) + 1))]));
        DepthwiseConv2d[(((i_inner * 6) + 2))] = (DepthwiseConv2d[(((i_inner * 6) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2))] * placeholder_shared[(((di_outer_inner * 6) + (dj_outer_inner * 2)))]));
        DepthwiseConv2d[(((i_inner * 6) + 62))] = (DepthwiseConv2d[(((i_inner * 6) + 62))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2442))] * placeholder_shared[(((di_outer_inner * 6) + (dj_outer_inner * 2)))]));
        DepthwiseConv2d[(((i_inner * 6) + 3))] = (DepthwiseConv2d[(((i_inner * 6) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 3))] * placeholder_shared[((((di_outer_inner * 6) + (dj_outer_inner * 2)) + 1))]));
        DepthwiseConv2d[(((i_inner * 6) + 63))] = (DepthwiseConv2d[(((i_inner * 6) + 63))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2443))] * placeholder_shared[((((di_outer_inner * 6) + (dj_outer_inner * 2)) + 1))]));
        DepthwiseConv2d[(((i_inner * 6) + 4))] = (DepthwiseConv2d[(((i_inner * 6) + 4))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 4))] * placeholder_shared[(((di_outer_inner * 6) + (dj_outer_inner * 2)))]));
        DepthwiseConv2d[(((i_inner * 6) + 64))] = (DepthwiseConv2d[(((i_inner * 6) + 64))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2444))] * placeholder_shared[(((di_outer_inner * 6) + (dj_outer_inner * 2)))]));
        DepthwiseConv2d[(((i_inner * 6) + 5))] = (DepthwiseConv2d[(((i_inner * 6) + 5))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 5))] * placeholder_shared[((((di_outer_inner * 6) + (dj_outer_inner * 2)) + 1))]));
        DepthwiseConv2d[(((i_inner * 6) + 65))] = (DepthwiseConv2d[(((i_inner * 6) + 65))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 40) * 5368) + (i_inner * 244)) + (di_outer_inner * 244)) + ((((int)threadIdx.x) % 40) * 6)) + (dj_outer_inner * 2)) + 2445))] * placeholder_shared[((((di_outer_inner * 6) + (dj_outer_inner * 2)) + 1))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 10; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 3; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
        compute[((((((((((((int)blockIdx.x) / 96) * 921600) + ((((int)threadIdx.x) / 40) * 460800)) + (((((int)blockIdx.x) % 96) >> 4) * 76800)) + (i1_inner * 3840)) + ((((int)threadIdx.x) % 40) * 96)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 15) * 2)) + i3_inner))] = max(min((DepthwiseConv2d[((((i1_inner * 6) + (i2_inner * 2)) + i3_inner))] + placeholder2[((((((int)blockIdx.x) & 15) * 2) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
        compute[(((((((((((((int)blockIdx.x) / 96) * 921600) + ((((int)threadIdx.x) / 40) * 460800)) + (((((int)blockIdx.x) % 96) >> 4) * 76800)) + (i1_inner * 3840)) + ((((int)threadIdx.x) % 40) * 96)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 15) * 2)) + i3_inner) + 38400))] = max(min((DepthwiseConv2d[(((((i1_inner * 6) + (i2_inner * 2)) + i3_inner) + 60))] + placeholder2[((((((int)blockIdx.x) & 15) * 2) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      }
    }
  }
}


