
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[4800];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float placeholder_shared[768];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 15; ++yy_outer_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 20; ++xx_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 320) + (xx_inner_init * 16)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 5))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 7))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 9))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 10))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 11))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 12))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 13))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 14))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 320) + (xx_inner_init * 16)) + 15))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 225; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) / 1800) * 345600) + ((((int)blockIdx.x) / 3) * 86400)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 1800) / 120) * 5760)) + ((((int)blockIdx.x) % 3) * 1920)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 120) / 6) * 96)) + (rc_outer_outer * 6)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 6)))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + (((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 32))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 32))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 64))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 96))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 96))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 128))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 160))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 160))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 192))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 192))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 224))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 224))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 256))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 288))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 288))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 320))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 320))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 352))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 352))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 384))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 384))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 416))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 416))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 448))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 448))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 480))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 480))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 512))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 512))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 544))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 544))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 576))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 576))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 608))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 608))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 640))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 640))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 672))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 672))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 704))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 704))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 736))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 768) + (((int)threadIdx.x) * 2)) + 736))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 15; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int xx_inner = 0; xx_inner < 20; ++xx_inner) {
            Conv2dOutput[(((yy_outer_inner * 320) + (xx_inner * 16)))] = (Conv2dOutput[(((yy_outer_inner * 320) + (xx_inner * 16)))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 1))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 1))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 2))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 2))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 2))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 3))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 3))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 3))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 4))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 4))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 4))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 5))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 5))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 5))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 6))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 6))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 6))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 7))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 7))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 7))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 8))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 8))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 9))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 9))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 9))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 10))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 10))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 10))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 11))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 11))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 11))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 12))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 12))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 12))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 13))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 13))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 13))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 14))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 14))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 14))]));
            Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 15))] = (Conv2dOutput[((((yy_outer_inner * 320) + (xx_inner * 16)) + 15))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 1800) + (yy_outer_inner * 120)) + (xx_inner * 6)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + 15))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 15; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 20; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
        T_relu[(((((((((((int)threadIdx.x) >> 3) * 460800) + ((((int)blockIdx.x) / 3) * 115200)) + (ax1_inner * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 320) + (ax2_inner * 16)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


