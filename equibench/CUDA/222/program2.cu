
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2880];
  __shared__ float PaddedInput_shared[2700];
  __shared__ float placeholder_shared[1536];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 32; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 3; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 15; ++xx_inner_init) {
        Conv2dOutput[((((yy_inner_init * 480) + (xx_inner_init * 32)) + ff_outer_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 480) + (xx_inner_init * 32)) + ff_outer_inner_init) + 1440))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 36; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 30) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s / 12)) < 225) {
          if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 360) + (((int)threadIdx.x) * 36)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 2700) {
            if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 10) + ((int)threadIdx.x)) < 75) {
              PaddedInput_shared[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s / 12)) / 15)) % 15) * 180) + ((((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s / 12)) % 15) * 12)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s % 12)))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s / 12)) / 15)) % 15) * 11520) + ((((((int)threadIdx.x) * 3) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s / 12)) % 15) * 768)) + (rc_outer_outer * 12)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s % 12)))];
            }
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 52; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 30) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 1536) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 10) + ((int)threadIdx.x)) < 512) {
            placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 30) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[(((((rc_outer_outer * 1536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 30)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 6; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 32; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 3; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 15; ++xx_inner) {
              Conv2dOutput[((((yy_inner * 480) + (xx_inner * 32)) + ff_outer_inner))] = (Conv2dOutput[((((yy_inner * 480) + (xx_inner * 32)) + ff_outer_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 540) + (yy_inner * 180)) + (xx_inner * 12)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 1) * 32)) + ff_outer_inner))]));
              Conv2dOutput[(((((yy_inner * 480) + (xx_inner * 32)) + ff_outer_inner) + 1440))] = (Conv2dOutput[(((((yy_inner * 480) + (xx_inner * 32)) + ff_outer_inner) + 1440))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 540) + (yy_inner * 180)) + (xx_inner * 12)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 1) * 32)) + ff_outer_inner) + 64))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
        T_relu[(((((((((int)threadIdx.x) >> 1) * 5760) + (ax1_inner * 1920)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 1) * 32)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 480) + (ax2_inner * 32)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 32) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((((int)threadIdx.x) >> 1) * 5760) + (ax1_inner * 1920)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 1) * 32)) + ax3_inner) + 64))] = max((Conv2dOutput[(((((ax1_inner * 480) + (ax2_inner * 32)) + ax3_inner) + 1440))] + placeholder2[(((((((int)threadIdx.x) & 1) * 32) + ax3_inner) + 64))]), 0.000000e+00f);
      }
    }
  }
}


