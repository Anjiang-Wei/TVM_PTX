
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[224];
  __shared__ float PaddedInput_shared[1792];
  __shared__ float placeholder_shared[512];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 112))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 113))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 116))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 117))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 120))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 121))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 124))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 125))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 114))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 115))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 118))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 119))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 122))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 123))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 126))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 16) + 127))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 112; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 1) * 14336) + ((((int)blockIdx.x) >> 3) * 2048)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1) * 1024)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 32))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 256))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 512))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 96))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 768))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 1024))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 160))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 1280))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 192))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 1536))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 224))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 1792))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 2048))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 288))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 2304))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 320))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 2560))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 352))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 2816))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 384))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 3072))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 416))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 3328))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 448))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 3584))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 480))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 4096) + ((((int)blockIdx.x) & 7) * 32)) + (((int)threadIdx.x) * 2)) + 3840))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
        Conv2dOutput[((yy_outer_inner * 16))] = (Conv2dOutput[((yy_outer_inner * 16))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 112))] = (Conv2dOutput[(((yy_outer_inner * 16) + 112))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 896))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 1))] = (Conv2dOutput[(((yy_outer_inner * 16) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 113))] = (Conv2dOutput[(((yy_outer_inner * 16) + 113))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 896))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 4))] = (Conv2dOutput[(((yy_outer_inner * 16) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 16))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 116))] = (Conv2dOutput[(((yy_outer_inner * 16) + 116))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 912))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 5))] = (Conv2dOutput[(((yy_outer_inner * 16) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 16))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 117))] = (Conv2dOutput[(((yy_outer_inner * 16) + 117))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 912))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 8))] = (Conv2dOutput[(((yy_outer_inner * 16) + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 32))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 120))] = (Conv2dOutput[(((yy_outer_inner * 16) + 120))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 928))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 9))] = (Conv2dOutput[(((yy_outer_inner * 16) + 9))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 32))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 121))] = (Conv2dOutput[(((yy_outer_inner * 16) + 121))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 928))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 12))] = (Conv2dOutput[(((yy_outer_inner * 16) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 48))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 124))] = (Conv2dOutput[(((yy_outer_inner * 16) + 124))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 944))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 13))] = (Conv2dOutput[(((yy_outer_inner * 16) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 48))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 125))] = (Conv2dOutput[(((yy_outer_inner * 16) + 125))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 944))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 2))] = (Conv2dOutput[(((yy_outer_inner * 16) + 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 114))] = (Conv2dOutput[(((yy_outer_inner * 16) + 114))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 896))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 3))] = (Conv2dOutput[(((yy_outer_inner * 16) + 3))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 115))] = (Conv2dOutput[(((yy_outer_inner * 16) + 115))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 896))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 6))] = (Conv2dOutput[(((yy_outer_inner * 16) + 6))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 16))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 118))] = (Conv2dOutput[(((yy_outer_inner * 16) + 118))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 912))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 7))] = (Conv2dOutput[(((yy_outer_inner * 16) + 7))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 16))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 119))] = (Conv2dOutput[(((yy_outer_inner * 16) + 119))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 912))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 10))] = (Conv2dOutput[(((yy_outer_inner * 16) + 10))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 32))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 122))] = (Conv2dOutput[(((yy_outer_inner * 16) + 122))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 928))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 11))] = (Conv2dOutput[(((yy_outer_inner * 16) + 11))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 32))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 123))] = (Conv2dOutput[(((yy_outer_inner * 16) + 123))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 928))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 14))] = (Conv2dOutput[(((yy_outer_inner * 16) + 14))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 48))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 126))] = (Conv2dOutput[(((yy_outer_inner * 16) + 126))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 944))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 15))] = (Conv2dOutput[(((yy_outer_inner * 16) + 15))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 48))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_outer_inner * 16) + 127))] = (Conv2dOutput[(((yy_outer_inner * 16) + 127))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 448) + (yy_outer_inner * 64)) + rc_outer_inner) + 944))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 14; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[(((((((((((int)threadIdx.x) >> 3) * 50176) + (ax1_inner * 3584)) + ((((int)blockIdx.x) >> 3) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 8) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((((((int)threadIdx.x) >> 3) * 50176) + (ax1_inner * 3584)) + ((((int)blockIdx.x) >> 3) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 100352))] = max((Conv2dOutput[(((((ax1_inner * 8) + (ax2_inner * 4)) + ax3_inner) + 112))] + placeholder2[(((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


