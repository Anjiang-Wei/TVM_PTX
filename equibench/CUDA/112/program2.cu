
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[4];
  __shared__ float PaddedInput_shared[448];
  __shared__ float placeholder_shared[16];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 7))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)) + 7))];
    PaddedInput_shared[((((int)threadIdx.x) + 14))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 14) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 14) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 21))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 21) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 5)))];
    PaddedInput_shared[((((int)threadIdx.x) + 28))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 28) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 12) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 35))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 35) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 42))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 42) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 10) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 49))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 49) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 56))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 56) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 63))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 63) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 15) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 70))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 70) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 6)))];
    PaddedInput_shared[((((int)threadIdx.x) + 77))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 77) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 13) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 84))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 84) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 4)))];
    PaddedInput_shared[((((int)threadIdx.x) + 91))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 91) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 11) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 98))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 98) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 2)))];
    PaddedInput_shared[((((int)threadIdx.x) + 105))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 105) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 9)))];
    PaddedInput_shared[((((int)threadIdx.x) + 112))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)) + 25088))];
    PaddedInput_shared[((((int)threadIdx.x) + 119))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 119) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 126))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 126) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 14) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 133))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 133) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 5)))];
    PaddedInput_shared[((((int)threadIdx.x) + 140))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 140) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 12) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 147))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 147) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 154))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 154) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 10) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 161))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 161) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 168))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 168) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 175))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 175) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 15) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 182))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 182) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 6)))];
    PaddedInput_shared[((((int)threadIdx.x) + 189))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 189) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 13) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 196))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 196) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 4)))];
    PaddedInput_shared[((((int)threadIdx.x) + 203))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 203) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 11) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 210))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 210) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 2)))];
    PaddedInput_shared[((((int)threadIdx.x) + 217))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)threadIdx.x) + 217) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 9)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[(((((((((int)blockIdx.x) / 3584) * 50176) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)) + 100352))];
    PaddedInput_shared[((((int)threadIdx.x) + 231))] = placeholder[((((((((((int)threadIdx.x) + 231) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 238))] = placeholder[(((((((((((int)threadIdx.x) + 238) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 14) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 14) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 245))] = placeholder[(((((((((((int)threadIdx.x) + 245) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 21) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 5)))];
    PaddedInput_shared[((((int)threadIdx.x) + 252))] = placeholder[(((((((((((int)threadIdx.x) + 252) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 28) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 12) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 259))] = placeholder[(((((((((((int)threadIdx.x) + 259) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 35) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 266))] = placeholder[(((((((((((int)threadIdx.x) + 266) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 42) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 10) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 273))] = placeholder[(((((((((((int)threadIdx.x) + 273) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 49) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[(((((((((((int)threadIdx.x) + 280) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 56) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 287))] = placeholder[(((((((((((int)threadIdx.x) + 287) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 63) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 15) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 294))] = placeholder[(((((((((((int)threadIdx.x) + 294) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 70) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 6)))];
    PaddedInput_shared[((((int)threadIdx.x) + 301))] = placeholder[(((((((((((int)threadIdx.x) + 301) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 77) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 13) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 308))] = placeholder[(((((((((((int)threadIdx.x) + 308) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 84) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 4)))];
    PaddedInput_shared[((((int)threadIdx.x) + 315))] = placeholder[(((((((((((int)threadIdx.x) + 315) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 91) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 11) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 322))] = placeholder[(((((((((((int)threadIdx.x) + 322) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 98) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 2)))];
    PaddedInput_shared[((((int)threadIdx.x) + 329))] = placeholder[(((((((((((int)threadIdx.x) + 329) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 105) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 9)))];
    PaddedInput_shared[((((int)threadIdx.x) + 336))] = placeholder[(((((((((((int)threadIdx.x) + 336) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 112) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 343))] = placeholder[(((((((((((int)threadIdx.x) + 343) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 119) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 350))] = placeholder[(((((((((((int)threadIdx.x) + 350) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 126) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 14) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 357))] = placeholder[(((((((((((int)threadIdx.x) + 357) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 133) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 5)))];
    PaddedInput_shared[((((int)threadIdx.x) + 364))] = placeholder[(((((((((((int)threadIdx.x) + 364) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 140) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 12) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 371))] = placeholder[(((((((((((int)threadIdx.x) + 371) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 147) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 378))] = placeholder[(((((((((((int)threadIdx.x) + 378) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 154) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 10) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 385))] = placeholder[(((((((((((int)threadIdx.x) + 385) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 161) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[(((((((((((int)threadIdx.x) + 392) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 168) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 399))] = placeholder[(((((((((((int)threadIdx.x) + 399) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 175) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 15) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 406))] = placeholder[(((((((((((int)threadIdx.x) + 406) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 182) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 6)))];
    PaddedInput_shared[((((int)threadIdx.x) + 413))] = placeholder[(((((((((((int)threadIdx.x) + 413) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 189) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 13) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 420))] = placeholder[(((((((((((int)threadIdx.x) + 420) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 196) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 4)))];
    PaddedInput_shared[((((int)threadIdx.x) + 427))] = placeholder[(((((((((((int)threadIdx.x) + 427) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 203) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) + 11) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 434))] = placeholder[(((((((((((int)threadIdx.x) + 434) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 210) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 2)))];
    PaddedInput_shared[((((int)threadIdx.x) + 441))] = placeholder[(((((((((((int)threadIdx.x) + 441) / 224) * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((((int)threadIdx.x) + 217) >> 4) * 3584)) + (((((int)blockIdx.x) % 3584) >> 7) * 128)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) + 9)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 2048) + (((int)threadIdx.x) * 128)) + (((int)blockIdx.x) & 127)))];
    placeholder_shared[((((int)threadIdx.x) + 7))] = placeholder1[(((((rc_outer_outer * 2048) + (((int)threadIdx.x) * 128)) + (((int)blockIdx.x) & 127)) + 896))];
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[((((int)threadIdx.x) + 14))] = placeholder1[(((((rc_outer_outer * 2048) + (((int)threadIdx.x) * 128)) + (((int)blockIdx.x) & 127)) + 1792))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((int)threadIdx.x) * 32))] * placeholder_shared[(0)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 224))] * placeholder_shared[(0)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 1))] * placeholder_shared[(1)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 225))] * placeholder_shared[(1)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 2))] * placeholder_shared[(2)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 226))] * placeholder_shared[(2)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 3))] * placeholder_shared[(3)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 227))] * placeholder_shared[(3)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 4))] * placeholder_shared[(4)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 228))] * placeholder_shared[(4)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 5))] * placeholder_shared[(5)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 229))] * placeholder_shared[(5)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 6))] * placeholder_shared[(6)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 230))] * placeholder_shared[(6)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 7))] * placeholder_shared[(7)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 231))] * placeholder_shared[(7)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 8))] * placeholder_shared[(8)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 232))] * placeholder_shared[(8)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 9))] * placeholder_shared[(9)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 233))] * placeholder_shared[(9)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 10))] * placeholder_shared[(10)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 234))] * placeholder_shared[(10)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 11))] * placeholder_shared[(11)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 235))] * placeholder_shared[(11)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 12))] * placeholder_shared[(12)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 236))] * placeholder_shared[(12)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 13))] * placeholder_shared[(13)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 237))] * placeholder_shared[(13)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 14))] * placeholder_shared[(14)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 238))] * placeholder_shared[(14)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 15))] * placeholder_shared[(15)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 239))] * placeholder_shared[(15)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 16))] * placeholder_shared[(0)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 240))] * placeholder_shared[(0)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 17))] * placeholder_shared[(1)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 241))] * placeholder_shared[(1)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 18))] * placeholder_shared[(2)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 242))] * placeholder_shared[(2)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 19))] * placeholder_shared[(3)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 243))] * placeholder_shared[(3)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 20))] * placeholder_shared[(4)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 244))] * placeholder_shared[(4)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 21))] * placeholder_shared[(5)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 245))] * placeholder_shared[(5)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 22))] * placeholder_shared[(6)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 246))] * placeholder_shared[(6)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 23))] * placeholder_shared[(7)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 247))] * placeholder_shared[(7)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 24))] * placeholder_shared[(8)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 248))] * placeholder_shared[(8)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 25))] * placeholder_shared[(9)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 249))] * placeholder_shared[(9)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 26))] * placeholder_shared[(10)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 250))] * placeholder_shared[(10)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 27))] * placeholder_shared[(11)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 251))] * placeholder_shared[(11)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 28))] * placeholder_shared[(12)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 252))] * placeholder_shared[(12)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 29))] * placeholder_shared[(13)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 253))] * placeholder_shared[(13)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 30))] * placeholder_shared[(14)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 254))] * placeholder_shared[(14)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 31))] * placeholder_shared[(15)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 32) + 255))] * placeholder_shared[(15)]));
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      T_relu[((((((ax0_inner * 100352) + ((((int)blockIdx.x) / 3584) * 50176)) + (((int)threadIdx.x) * 7168)) + (ax1_inner * 3584)) + (((int)blockIdx.x) % 3584)))] = max((Conv2dOutput[(((ax0_inner * 2) + ax1_inner))] + placeholder2[((((int)blockIdx.x) & 127))]), 0.000000e+00f);
    }
  }
}


