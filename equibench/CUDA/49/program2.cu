
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[1800];
  __shared__ float PaddedInput_shared[10800];
  __shared__ float placeholder_shared[36];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 15; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 5; ++xx_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 450) + (yy_inner_init * 30)) + (xx_inner_init * 6)))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 450) + (yy_inner_init * 30)) + (xx_inner_init * 6)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 450) + (yy_inner_init * 30)) + (xx_inner_init * 6)) + 2))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 450) + (yy_inner_init * 30)) + (xx_inner_init * 6)) + 3))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 450) + (yy_inner_init * 30)) + (xx_inner_init * 6)) + 4))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 450) + (yy_inner_init * 30)) + (xx_inner_init * 6)) + 5))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 24; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 1800; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 6) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 450) * 518400) + ((((int)blockIdx.x) >> 3) * 129600)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 450) / 30) * 8640)) + (((((int)blockIdx.x) & 7) >> 2) * 4320)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 30) * 144)) + (rc_outer_outer * 6)) + ((int)threadIdx.x)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 144) + ((((int)blockIdx.x) & 3) * 6)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 6))] = placeholder1[(((((rc_outer_outer * 144) + ((((int)blockIdx.x) & 3) * 6)) + ((int)threadIdx.x)) + 24))];
    placeholder_shared[((((int)threadIdx.x) + 12))] = placeholder1[(((((rc_outer_outer * 144) + ((((int)blockIdx.x) & 3) * 6)) + ((int)threadIdx.x)) + 48))];
    placeholder_shared[((((int)threadIdx.x) + 18))] = placeholder1[(((((rc_outer_outer * 144) + ((((int)blockIdx.x) & 3) * 6)) + ((int)threadIdx.x)) + 72))];
    placeholder_shared[((((int)threadIdx.x) + 24))] = placeholder1[(((((rc_outer_outer * 144) + ((((int)blockIdx.x) & 3) * 6)) + ((int)threadIdx.x)) + 96))];
    placeholder_shared[((((int)threadIdx.x) + 30))] = placeholder1[(((((rc_outer_outer * 144) + ((((int)blockIdx.x) & 3) * 6)) + ((int)threadIdx.x)) + 120))];
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
      for (int rc_inner = 0; rc_inner < 6; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 15; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 5; ++xx_inner) {
            Conv2dOutput[((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)))] = (Conv2dOutput[((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)))] + (PaddedInput_shared[((((((nn_outer_inner * 2700) + (yy_inner * 180)) + (((int)threadIdx.x) * 30)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[((rc_inner * 6))]));
            Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 1))] = (Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 1))] + (PaddedInput_shared[((((((nn_outer_inner * 2700) + (yy_inner * 180)) + (((int)threadIdx.x) * 30)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 6) + 1))]));
            Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 2))] = (Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 2))] + (PaddedInput_shared[((((((nn_outer_inner * 2700) + (yy_inner * 180)) + (((int)threadIdx.x) * 30)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 6) + 2))]));
            Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 3))] = (Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 3))] + (PaddedInput_shared[((((((nn_outer_inner * 2700) + (yy_inner * 180)) + (((int)threadIdx.x) * 30)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 6) + 3))]));
            Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 4))] = (Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 4))] + (PaddedInput_shared[((((((nn_outer_inner * 2700) + (yy_inner * 180)) + (((int)threadIdx.x) * 30)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 6) + 4))]));
            Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 5))] = (Conv2dOutput[(((((nn_outer_inner * 450) + (yy_inner * 30)) + (xx_inner * 6)) + 5))] + (PaddedInput_shared[((((((nn_outer_inner * 2700) + (yy_inner * 180)) + (((int)threadIdx.x) * 30)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 6) + 5))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 15; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 6; ++ax3_inner) {
          T_add[(((((((((ax0_inner * 86400) + ((((int)blockIdx.x) >> 3) * 21600)) + (ax1_inner * 1440)) + (((((int)blockIdx.x) & 7) >> 2) * 720)) + (((int)threadIdx.x) * 120)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 3) * 6)) + ax3_inner))] = (placeholder2[(((((((((ax0_inner * 86400) + ((((int)blockIdx.x) >> 3) * 21600)) + (ax1_inner * 1440)) + (((((int)blockIdx.x) & 7) >> 2) * 720)) + (((int)threadIdx.x) * 120)) + (ax2_inner * 24)) + ((((int)blockIdx.x) & 3) * 6)) + ax3_inner))] + (Conv2dOutput[(((((ax0_inner * 450) + (ax1_inner * 30)) + (ax2_inner * 6)) + ax3_inner))] + placeholder3[((((((int)blockIdx.x) & 3) * 6) + ax3_inner))]));
        }
      }
    }
  }
}


