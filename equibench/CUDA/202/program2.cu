
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
  float Conv3dOutput[4096];
  __shared__ float PaddedInput_shared[3456];
  __shared__ float placeholder_shared[4608];
  for (int cc_outer_inner_init = 0; cc_outer_inner_init < 32; ++cc_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
      for (int dd_inner_init = 0; dd_inner_init < 8; ++dd_inner_init) {
        Conv3dOutput[((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 1024))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 2048))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 3072))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 32))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 1056))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 2080))] = 0.000000e+00f;
        Conv3dOutput[(((((nn_inner_init * 512) + (dd_inner_init * 64)) + cc_outer_inner_init) + 3104))] = 0.000000e+00f;
      }
    }
  }
  for (int rh_outer_outer = 0; rh_outer_outer < 3; ++rh_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 108; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)))] = (((((((1 <= (((((int)blockIdx.x) >> 1) * 16) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 864) / 48))) && ((((((int)blockIdx.x) >> 1) * 16) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 864) / 48)) < 33)) && (1 <= (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 48) / 3) + rh_outer_outer))) && ((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 48) / 3) + rh_outer_outer) < 17)) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 3) + (((int)blockIdx.x) & 1)))) && (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 3) + (((int)blockIdx.x) & 1)) < 3)) ? placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) / 864) * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 864) / 3) * 512)) + (rh_outer_outer * 512)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 1)) % 3) * 256)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) - 8960))] : 0.000000e+00f);
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 < 144; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)))] = placeholder1[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)) / 1536) * 589824) + (rh_outer_outer * 196608)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)) % 1536) >> 9) * 65536)) + (rc_outer_outer * 512)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)) & 511)))];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int cc_outer_inner = 0; cc_outer_inner < 32; ++cc_outer_inner) {
          for (int rd_inner = 0; rd_inner < 3; ++rd_inner) {
            for (int rw_inner = 0; rw_inner < 3; ++rw_inner) {
              for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
                for (int dd_inner = 0; dd_inner < 8; ++dd_inner) {
                  Conv3dOutput[((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner))] = (Conv3dOutput[((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner))] + (PaddedInput_shared[((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 1024))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 1024))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 24))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 2048))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 2048))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 48))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 3072))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 3072))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 72))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 32))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 32))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 6))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 1056))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 1056))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 30))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 2080))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 2080))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 54))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                  Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 3104))] = (Conv3dOutput[(((((nn_inner * 512) + (dd_inner * 64)) + cc_outer_inner) + 3104))] + (PaddedInput_shared[(((((((((nn_inner * 1728) + ((((int)threadIdx.x) >> 4) * 768)) + (dd_inner * 96)) + (rd_inner * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 12)) + (rw_inner * 2)) + rc_outer_inner) + 78))] * placeholder_shared[((((((rd_inner * 1536) + (rw_inner * 512)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 7) * 32)) + cc_outer_inner))]));
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax4_inner = 0; ax4_inner < 32; ++ax4_inner) {
          T_relu[((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner))] = max((((Conv3dOutput[(((((ax0_inner * 512) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax4_inner))] * placeholder2[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder4[((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner) + 2048))] = max((((Conv3dOutput[((((((ax0_inner * 512) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax4_inner) + 1024))] * placeholder2[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder4[(((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner) + 2048))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner) + 4096))] = max((((Conv3dOutput[((((((ax0_inner * 512) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax4_inner) + 2048))] * placeholder2[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder4[(((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner) + 4096))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner) + 6144))] = max((((Conv3dOutput[((((((ax0_inner * 512) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax4_inner) + 3072))] * placeholder2[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 32) + ax4_inner))]) + placeholder4[(((((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 1) * 131072)) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax1_inner * 8192)) + (((((int)threadIdx.x) & 15) >> 3) * 1024)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax4_inner) + 6144))]), 0.000000e+00f);
        }
      }
    }
  }
}


