
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
  float Conv3dOutput[256];
  __shared__ float PaddedInput_shared[640];
  __shared__ float placeholder_shared[96];
  for (int hh_outer_inner_init = 0; hh_outer_inner_init < 16; ++hh_outer_inner_init) {
    Conv3dOutput[((hh_outer_inner_init * 8))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 128))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 1))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 129))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 2))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 130))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 3))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 131))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 4))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 132))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 5))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 133))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 6))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 134))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 7))] = 0.000000e+00f;
    Conv3dOutput[(((hh_outer_inner_init * 8) + 135))] = 0.000000e+00f;
  }
  for (int rd_outer_outer = 0; rd_outer_outer < 3; ++rd_outer_outer) {
    for (int rh_outer_outer = 0; rh_outer_outer < 3; ++rh_outer_outer) {
      for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) {
          if (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) < 640) {
            if (((int)threadIdx.x) < 36) {
              PaddedInput_shared[(((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s))] = (((((((1 <= ((((((int)blockIdx.x) >> 3) * 4) + (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) / 160)) + rd_outer_outer)) && (((((((int)blockIdx.x) >> 3) * 4) + (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) / 160)) + rd_outer_outer) < 129)) && (1 <= (((((((int)blockIdx.x) & 7) >> 1) * 16) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) % 160) / 10)) + rh_outer_outer))) && ((((((((int)blockIdx.x) & 7) >> 1) * 16) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) % 160) / 10)) + rh_outer_outer) < 65)) && (1 <= (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) % 10))) && ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) % 10) < 9)) ? placeholder[(((((((((((((int)blockIdx.x) >> 3) * 131072) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) / 160) * 32768)) + (rd_outer_outer * 32768)) + (((((int)blockIdx.x) & 7) >> 1) * 8192)) + (((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) % 160) / 10) * 512)) + (rh_outer_outer * 512)) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) % 10) * 64)) + rc_outer_outer) - 33344))] : 0.000000e+00f);
            }
          }
        }
        placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((rd_outer_outer * 36864) + (rh_outer_outer * 12288)) + ((((int)threadIdx.x) >> 5) * 4096)) + (rc_outer_outer * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)))];
        if (((int)threadIdx.x) < 32) {
          placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((((rd_outer_outer * 36864) + (rh_outer_outer * 12288)) + (rc_outer_outer * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 8192))];
        }
        __syncthreads();
        for (int rw_outer_inner = 0; rw_outer_inner < 3; ++rw_outer_inner) {
          for (int hh_outer_inner = 0; hh_outer_inner < 16; ++hh_outer_inner) {
            Conv3dOutput[((hh_outer_inner * 8))] = (Conv3dOutput[((hh_outer_inner * 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 128))] = (Conv3dOutput[(((hh_outer_inner * 8) + 128))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 1))] = (Conv3dOutput[(((hh_outer_inner * 8) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 1))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 129))] = (Conv3dOutput[(((hh_outer_inner * 8) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 1))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 2))] = (Conv3dOutput[(((hh_outer_inner * 8) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 2))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 130))] = (Conv3dOutput[(((hh_outer_inner * 8) + 130))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 2))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 3))] = (Conv3dOutput[(((hh_outer_inner * 8) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 3))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 131))] = (Conv3dOutput[(((hh_outer_inner * 8) + 131))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 3))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 4))] = (Conv3dOutput[(((hh_outer_inner * 8) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 4))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 132))] = (Conv3dOutput[(((hh_outer_inner * 8) + 132))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 4))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 5))] = (Conv3dOutput[(((hh_outer_inner * 8) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 5))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 133))] = (Conv3dOutput[(((hh_outer_inner * 8) + 133))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 5))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 6))] = (Conv3dOutput[(((hh_outer_inner * 8) + 6))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 6))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 134))] = (Conv3dOutput[(((hh_outer_inner * 8) + 134))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 6))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 7))] = (Conv3dOutput[(((hh_outer_inner * 8) + 7))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 7))] * placeholder_shared[(((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)))]));
            Conv3dOutput[(((hh_outer_inner * 8) + 135))] = (Conv3dOutput[(((hh_outer_inner * 8) + 135))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 160) + (hh_outer_inner * 10)) + rw_outer_inner) + 7))] * placeholder_shared[((((rw_outer_inner * 32) + (((int)threadIdx.x) & 15)) + 16))]));
          }
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 16; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 3) * 131072) + ((((int)threadIdx.x) >> 4) * 32768)) + (((((int)blockIdx.x) & 7) >> 1) * 8192)) + (ax2_inner * 512)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 15)))] = max((((Conv3dOutput[(((ax2_inner * 8) + ax3_inner))] * placeholder2[((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 15)))]) + placeholder3[((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 15)))]) + placeholder4[(((((((((((int)blockIdx.x) >> 3) * 131072) + ((((int)threadIdx.x) >> 4) * 32768)) + (((((int)blockIdx.x) & 7) >> 1) * 8192)) + (ax2_inner * 512)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 15)))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 3) * 131072) + ((((int)threadIdx.x) >> 4) * 32768)) + (((((int)blockIdx.x) & 7) >> 1) * 8192)) + (ax2_inner * 512)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 15)) + 16))] = max((((Conv3dOutput[((((ax2_inner * 8) + ax3_inner) + 128))] * placeholder2[(((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 15)) + 16))]) + placeholder3[(((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 15)) + 16))]) + placeholder4[((((((((((((int)blockIdx.x) >> 3) * 131072) + ((((int)threadIdx.x) >> 4) * 32768)) + (((((int)blockIdx.x) & 7) >> 1) * 8192)) + (ax2_inner * 512)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 15)) + 16))]), 0.000000e+00f);
    }
  }
}


