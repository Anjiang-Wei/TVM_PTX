
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[224];
  __shared__ float PaddedInput_shared[23328];
  __shared__ float placeholder_shared[512];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 8; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 7; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
        Conv2dOutput[((((yy_inner_init * 16) + (xx_inner_init * 8)) + ff_outer_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 16) + (xx_inner_init * 8)) + ff_outer_inner_init) + 112))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 417; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 23328) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 5) * 802816) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 11664) * 401408)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 11664) / 432) * 14336)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 432) >> 4) * 512)) + (rc_outer_outer * 16)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) & 15)))];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 10; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 512) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 16384) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) >> 5) * 1024)) + ((((int)blockIdx.x) & 31) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) & 31)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 8; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
              Conv2dOutput[((((yy_inner * 16) + (xx_inner * 8)) + ff_outer_inner))] = (Conv2dOutput[((((yy_inner * 16) + (xx_inner * 8)) + ff_outer_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 28) * 6048) + (yy_inner * 864)) + (((((int)threadIdx.x) % 28) >> 2) * 64)) + (xx_inner * 32)) + (rc_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ff_outer_inner))]));
              Conv2dOutput[(((((yy_inner * 16) + (xx_inner * 8)) + ff_outer_inner) + 112))] = (Conv2dOutput[(((((yy_inner * 16) + (xx_inner * 8)) + ff_outer_inner) + 112))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 28) * 6048) + (yy_inner * 864)) + (((((int)threadIdx.x) % 28) >> 2) * 64)) + (xx_inner * 32)) + (rc_outer_inner * 8)) + rc_inner) + 11664))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ff_outer_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
        T_add[((((((((((((int)blockIdx.x) >> 5) * 401408) + ((((int)threadIdx.x) / 28) * 100352)) + (ax1_inner * 14336)) + (((((int)threadIdx.x) % 28) >> 2) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 31) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))] = (Conv2dOutput[((((ax1_inner * 16) + (ax2_inner * 8)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 31) * 32) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]);
        T_add[(((((((((((((int)blockIdx.x) >> 5) * 401408) + ((((int)threadIdx.x) / 28) * 100352)) + (ax1_inner * 14336)) + (((((int)threadIdx.x) % 28) >> 2) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 31) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner) + 200704))] = (Conv2dOutput[(((((ax1_inner * 16) + (ax2_inner * 8)) + ax3_inner) + 112))] + placeholder2[(((((((int)blockIdx.x) & 31) * 32) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]);
      }
    }
  }
}


