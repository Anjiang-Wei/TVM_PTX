
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[1568];
  __shared__ float placeholder_shared[16];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 56; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 196))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 10976))];
    PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 21952))];
    PaddedInput_shared[((((int)threadIdx.x) + 588))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 32928))];
    PaddedInput_shared[((((int)threadIdx.x) + 784))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 43904))];
    PaddedInput_shared[((((int)threadIdx.x) + 980))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 54880))];
    PaddedInput_shared[((((int)threadIdx.x) + 1176))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 65856))];
    PaddedInput_shared[((((int)threadIdx.x) + 1372))] = placeholder[(((((((((int)blockIdx.x) >> 5) * 87808) + ((((int)threadIdx.x) >> 2) * 224)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 76832))];
    if (((int)threadIdx.x) < 4) {
      ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + ((((rc_outer_outer * 512) + (((int)threadIdx.x) * 128)) + ((((int)blockIdx.x) & 31) * 4)))))[0];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 4))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 8))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 12))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 1))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 5))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 9))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 13))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 112))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 116))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 120))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 124))] * placeholder_shared[((((int)threadIdx.x) & 3))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 113))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 117))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 121))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 125))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 4))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 2))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 6))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 10))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 14))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 3))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 7))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 11))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 15))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 114))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 118))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 122))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 126))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 8))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 115))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 119))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 123))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 28) * 224) + (((((int)threadIdx.x) % 28) >> 2) * 16)) + 127))] * placeholder_shared[(((((int)threadIdx.x) & 3) + 12))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 5) * 50176) + ((((int)threadIdx.x) / 28) * 7168)) + (ax1_inner * 3584)) + (((((int)threadIdx.x) % 28) >> 2) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)))] = max((Conv2dOutput[(((ax1_inner * 4) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 31) * 4) + (((int)threadIdx.x) & 3)))]), 0.000000e+00f);
    }
  }
}


