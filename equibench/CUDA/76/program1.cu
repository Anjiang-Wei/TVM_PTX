
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 480))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 480))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[64];
  __shared__ float PaddedInput_shared[1280];
  __shared__ float placeholder_shared[24];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    DepthwiseConv2d_local[((j_c_outer_inner_init * 8))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 1))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 2))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 3))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 4))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 5))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 6))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 7))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 32))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 33))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 34))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 35))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 36))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 37))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 38))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((j_c_outer_inner_init * 8) + 39))] = 0.000000e+00f;
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 80; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = (((((1 <= ((((((int)blockIdx.x) / 120) * 2) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) % 20) / 10)) + di_outer_outer)) && (((((((int)blockIdx.x) / 120) * 2) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) % 20) / 10)) + di_outer_outer) < 17)) && (1 <= ((((((int)blockIdx.x) % 120) / 60) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) % 10)))) && (((((((int)blockIdx.x) % 120) / 60) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) % 10)) < 17)) ? placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) / 20) * 122880) + ((((int)blockIdx.x) / 120) * 15360)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) % 20) / 10) * 7680)) + (di_outer_outer * 7680)) + (((((int)blockIdx.x) % 120) / 60) * 3840)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 3)) % 10) * 480)) + ((((int)blockIdx.x) % 60) * 8)) + (((int)threadIdx.x) & 7)) - 8160))] : 0.000000e+00f);
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((di_outer_outer * 1440) + ((((int)threadIdx.x) >> 3) * 480)) + ((((int)blockIdx.x) % 60) * 8)) + (((int)threadIdx.x) & 7)))];
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[((((int)threadIdx.x) + 16))] = placeholder1[(((((di_outer_outer * 1440) + ((((int)blockIdx.x) % 60) * 8)) + ((int)threadIdx.x)) + 960))];
    }
    __syncthreads();
    for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
      for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
        DepthwiseConv2d_local[((j_c_outer_inner * 8))] = (DepthwiseConv2d_local[((j_c_outer_inner * 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)))] * placeholder_shared[(((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 1))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 1))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 1))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 2))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 2))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 2))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 3))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 3))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 3))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 4))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 4))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 8))] * placeholder_shared[(((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 5))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 5))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 9))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 1))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 6))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 6))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 10))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 2))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 7))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 7))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 11))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 3))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 32))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 32))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 160))] * placeholder_shared[(((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 33))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 33))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 161))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 1))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 34))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 34))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 162))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 2))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 35))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 35))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 163))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 3))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 36))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 36))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 168))] * placeholder_shared[(((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 37))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 37))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 169))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 1))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 38))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 38))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 170))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 2))]));
        DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 39))] = (DepthwiseConv2d_local[(((j_c_outer_inner * 8) + 39))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 320) + (((((int)threadIdx.x) & 3) >> 1) * 80)) + (j_c_outer_inner * 16)) + (dj_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 171))] * placeholder_shared[((((dj_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + 3))]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      for (int c_inner = 0; c_inner < 4; ++c_inner) {
        DepthwiseConv2d[(((((((((((((int)threadIdx.x) >> 2) * 245760) + (b_inner * 122880)) + ((((int)blockIdx.x) / 120) * 15360)) + (((((int)threadIdx.x) & 3) >> 1) * 7680)) + (((((int)blockIdx.x) % 120) / 60) * 3840)) + (j_inner * 480)) + ((((int)blockIdx.x) % 60) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + c_inner))] = DepthwiseConv2d_local[((((b_inner * 32) + (j_inner * 4)) + c_inner))];
      }
    }
  }
}


