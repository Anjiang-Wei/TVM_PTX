
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[60];
  __shared__ float PaddedInput_shared[2925];
  __shared__ float placeholder_shared[208];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 4))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 22))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 23))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 40))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 41))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 42))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 43))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 49; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 60) + ((int)threadIdx.x)) < 2925) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 60) + ((int)threadIdx.x)))] = placeholder[((((((((int)blockIdx.x) >> 3) * 187200) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 60) + ((int)threadIdx.x)) / 13) * 832)) + (rc_outer_outer * 13)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 60) + ((int)threadIdx.x)) % 13)))];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 52; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 52) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 208) {
        if (((int)threadIdx.x) < 4) {
          placeholder_shared[(((((int)threadIdx.x) * 52) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 1664) + ((((((int)threadIdx.x) * 52) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 52) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15)))];
        }
      }
    }
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
      for (int rc_inner = 0; rc_inner < 13; ++rc_inner) {
        Conv2dOutput[((xx_outer_inner * 4))] = (Conv2dOutput[((xx_outer_inner * 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner))] * placeholder_shared[(((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 1))] = (Conv2dOutput[(((xx_outer_inner * 4) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 2))] = (Conv2dOutput[(((xx_outer_inner * 4) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 3))] = (Conv2dOutput[(((xx_outer_inner * 4) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 3))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 20))] = (Conv2dOutput[(((xx_outer_inner * 4) + 20))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 195))] * placeholder_shared[(((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 21))] = (Conv2dOutput[(((xx_outer_inner * 4) + 21))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 195))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 22))] = (Conv2dOutput[(((xx_outer_inner * 4) + 22))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 195))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 23))] = (Conv2dOutput[(((xx_outer_inner * 4) + 23))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 195))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 3))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 40))] = (Conv2dOutput[(((xx_outer_inner * 4) + 40))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 390))] * placeholder_shared[(((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 41))] = (Conv2dOutput[(((xx_outer_inner * 4) + 41))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 390))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 42))] = (Conv2dOutput[(((xx_outer_inner * 4) + 42))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 390))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 4) + 43))] = (Conv2dOutput[(((xx_outer_inner * 4) + 43))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 585) + (((((int)threadIdx.x) % 12) >> 2) * 65)) + (xx_outer_inner * 13)) + rc_inner) + 390))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 3) * 4)) + 3))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[((((((((((((int)blockIdx.x) >> 3) * 28800) + ((((int)threadIdx.x) / 12) * 5760)) + (ax1_inner * 1920)) + (((((int)threadIdx.x) % 12) >> 2) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 20) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


