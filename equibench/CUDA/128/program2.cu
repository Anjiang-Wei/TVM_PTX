
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[1024];
  __shared__ float placeholder_shared[1024];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 2) + yy_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + yy_inner_init) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + yy_inner_init) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + yy_inner_init) + 12))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 3) * 262144) + ((((int)blockIdx.x) >> 7) * 32768)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 7) * 4096)) + (((((int)blockIdx.x) & 127) >> 2) * 128)) + ((((int)threadIdx.x) >> 5) * 64)) + (rc_outer_outer * 32)) + (((int)threadIdx.x) & 31)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + ((int)threadIdx.x)))] = placeholder1[((((((rc_outer_outer * 4096) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 256)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      for (int rc_inner = 0; rc_inner < 32; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
          Conv2dOutput[(((yy_outer_inner * 2) + yy_inner))] = (Conv2dOutput[(((yy_outer_inner * 2) + yy_inner))] + (PaddedInput_shared[(((((yy_outer_inner * 128) + (yy_inner * 64)) + ((((int)threadIdx.x) >> 5) * 32)) + rc_inner))] * placeholder_shared[(((rc_inner * 32) + (((int)threadIdx.x) & 31)))]));
          Conv2dOutput[((((yy_outer_inner * 2) + yy_inner) + 4))] = (Conv2dOutput[((((yy_outer_inner * 2) + yy_inner) + 4))] + (PaddedInput_shared[((((((yy_outer_inner * 128) + (yy_inner * 64)) + ((((int)threadIdx.x) >> 5) * 32)) + rc_inner) + 256))] * placeholder_shared[(((rc_inner * 32) + (((int)threadIdx.x) & 31)))]));
          Conv2dOutput[((((yy_outer_inner * 2) + yy_inner) + 8))] = (Conv2dOutput[((((yy_outer_inner * 2) + yy_inner) + 8))] + (PaddedInput_shared[((((((yy_outer_inner * 128) + (yy_inner * 64)) + ((((int)threadIdx.x) >> 5) * 32)) + rc_inner) + 512))] * placeholder_shared[(((rc_inner * 32) + (((int)threadIdx.x) & 31)))]));
          Conv2dOutput[((((yy_outer_inner * 2) + yy_inner) + 12))] = (Conv2dOutput[((((yy_outer_inner * 2) + yy_inner) + 12))] + (PaddedInput_shared[((((((yy_outer_inner * 128) + (yy_inner * 64)) + ((((int)threadIdx.x) >> 5) * 32)) + rc_inner) + 768))] * placeholder_shared[(((rc_inner * 32) + (((int)threadIdx.x) & 31)))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    T_relu[((((((((((int)blockIdx.x) >> 7) * 65536) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 127) >> 2) * 256)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))] = max((Conv2dOutput[(ax1_inner)] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 31)))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) >> 7) * 65536) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 127) >> 2) * 256)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 32768))] = max((Conv2dOutput[((ax1_inner + 4))] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 31)))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) >> 7) * 65536) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 127) >> 2) * 256)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 524288))] = max((Conv2dOutput[((ax1_inner + 8))] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 31)))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) >> 7) * 65536) + (ax1_inner * 8192)) + (((((int)blockIdx.x) & 127) >> 2) * 256)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)) + 557056))] = max((Conv2dOutput[((ax1_inner + 12))] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 31)))]), 0.000000e+00f);
  }
}


