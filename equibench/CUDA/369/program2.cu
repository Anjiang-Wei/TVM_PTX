
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[20];
  __shared__ float PaddedInput_shared[84];
  __shared__ float placeholder_shared[1920];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 5; ++ff_inner_init) {
      Conv2dOutput[(((ff_outer_inner_init * 5) + ff_inner_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 56; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 84) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = placeholder[(((((((int)blockIdx.x) * 4704) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 12) * 672)) + (rc_outer_outer * 12)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 12)))];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 35; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 1920) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 1920) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56)) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 6; ++rc_inner) {
          for (int ff_inner = 0; ff_inner < 5; ++ff_inner) {
            Conv2dOutput[(((ff_outer_inner * 5) + ff_inner))] = (Conv2dOutput[(((ff_outer_inner * 5) + ff_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 12) + (rc_outer_inner * 6)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 960) + (rc_inner * 160)) + ((((int)threadIdx.x) & 7) * 20)) + (ff_outer_inner * 5)) + ff_inner))]));
          }
        }
      }
    }
  }
  for (int ax3_inner = 0; ax3_inner < 20; ++ax3_inner) {
    T_add[((((((int)blockIdx.x) * 1120) + (((int)threadIdx.x) * 20)) + ax3_inner))] = (Conv2dOutput[(ax3_inner)] + placeholder2[((((((int)threadIdx.x) & 7) * 20) + ax3_inner))]);
  }
}


