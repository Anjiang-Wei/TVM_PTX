
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[64];
  __shared__ float PaddedInput_shared[7424];
  __shared__ float placeholder_shared[36];
  for (int j_outer_inner_init = 0; j_outer_inner_init < 4; ++j_outer_inner_init) {
    for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
      for (int c_inner_init = 0; c_inner_init < 2; ++c_inner_init) {
        DepthwiseConv2d[((((i_inner_init * 8) + (j_outer_inner_init * 2)) + c_inner_init))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_inner_init * 8) + (j_outer_inner_init * 2)) + c_inner_init) + 16))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_inner_init * 8) + (j_outer_inner_init * 2)) + c_inner_init) + 32))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_inner_init * 8) + (j_outer_inner_init * 2)) + c_inner_init) + 48))] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 34; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 1856) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 7424) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 224) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = (((((1 <= (((((int)blockIdx.x) / 18) * 2) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 232) / 58))) && ((((((int)blockIdx.x) / 18) * 2) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 232) / 58)) < 57)) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 58))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 58) < 57)) ? placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 232) * 225792) + ((((int)blockIdx.x) / 18) * 8064)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 232) / 58) * 4032)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 58) * 72)) + ((((int)blockIdx.x) % 18) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) - 4104))] : 0.000000e+00f);
        }
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
    if (((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 36) {
      if (((int)threadIdx.x) < 4) {
        placeholder_shared[(((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[(((((((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) >> 2) * 72) + ((((int)blockIdx.x) % 18) * 4)) + (((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) & 3)))];
      }
    }
  }
  __syncthreads();
  for (int j_outer_inner = 0; j_outer_inner < 4; ++j_outer_inner) {
    for (int di_inner = 0; di_inner < 3; ++di_inner) {
      for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
        for (int i_inner = 0; i_inner < 2; ++i_inner) {
          for (int c_inner = 0; c_inner < 2; ++c_inner) {
            DepthwiseConv2d[((((i_inner * 8) + (j_outer_inner * 2)) + c_inner))] = (DepthwiseConv2d[((((i_inner * 8) + (j_outer_inner * 2)) + c_inner))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 14) * 928) + (i_inner * 232)) + (di_inner * 232)) + ((((int)threadIdx.x) % 14) * 16)) + (j_outer_inner * 4)) + (dj_inner * 4)) + c_inner))] * placeholder_shared[((((di_inner * 12) + (dj_inner * 4)) + c_inner))]));
            DepthwiseConv2d[(((((i_inner * 8) + (j_outer_inner * 2)) + c_inner) + 16))] = (DepthwiseConv2d[(((((i_inner * 8) + (j_outer_inner * 2)) + c_inner) + 16))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 14) * 928) + (i_inner * 232)) + (di_inner * 232)) + ((((int)threadIdx.x) % 14) * 16)) + (j_outer_inner * 4)) + (dj_inner * 4)) + c_inner) + 2))] * placeholder_shared[(((((di_inner * 12) + (dj_inner * 4)) + c_inner) + 2))]));
            DepthwiseConv2d[(((((i_inner * 8) + (j_outer_inner * 2)) + c_inner) + 32))] = (DepthwiseConv2d[(((((i_inner * 8) + (j_outer_inner * 2)) + c_inner) + 32))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 14) * 928) + (i_inner * 232)) + (di_inner * 232)) + ((((int)threadIdx.x) % 14) * 16)) + (j_outer_inner * 4)) + (dj_inner * 4)) + c_inner) + 3712))] * placeholder_shared[((((di_inner * 12) + (dj_inner * 4)) + c_inner))]));
            DepthwiseConv2d[(((((i_inner * 8) + (j_outer_inner * 2)) + c_inner) + 48))] = (DepthwiseConv2d[(((((i_inner * 8) + (j_outer_inner * 2)) + c_inner) + 48))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 14) * 928) + (i_inner * 232)) + (di_inner * 232)) + ((((int)threadIdx.x) % 14) * 16)) + (j_outer_inner * 4)) + (dj_inner * 4)) + c_inner) + 3714))] * placeholder_shared[(((((di_inner * 12) + (dj_inner * 4)) + c_inner) + 2))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((((((int)threadIdx.x) / 14) * 225792) + ((((int)blockIdx.x) / 18) * 8064)) + (ax1_inner * 4032)) + ((((int)threadIdx.x) % 14) * 288)) + (ax2_inner * 72)) + ((((int)blockIdx.x) % 18) * 4)) + ax3_inner))] = max((DepthwiseConv2d[((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) % 18) * 4) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((((((int)threadIdx.x) / 14) * 225792) + ((((int)blockIdx.x) / 18) * 8064)) + (ax1_inner * 4032)) + ((((int)threadIdx.x) % 14) * 288)) + (ax2_inner * 72)) + ((((int)blockIdx.x) % 18) * 4)) + ax3_inner) + 2))] = max((DepthwiseConv2d[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) % 18) * 4) + ax3_inner) + 2))]), 0.000000e+00f);
        T_relu[((((((((((((int)threadIdx.x) / 14) * 225792) + ((((int)blockIdx.x) / 18) * 8064)) + (ax1_inner * 4032)) + ((((int)threadIdx.x) % 14) * 288)) + (ax2_inner * 72)) + ((((int)blockIdx.x) % 18) * 4)) + ax3_inner) + 903168))] = max((DepthwiseConv2d[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 32))] + placeholder2[((((((int)blockIdx.x) % 18) * 4) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((((((int)threadIdx.x) / 14) * 225792) + ((((int)blockIdx.x) / 18) * 8064)) + (ax1_inner * 4032)) + ((((int)threadIdx.x) % 14) * 288)) + (ax2_inner * 72)) + ((((int)blockIdx.x) % 18) * 4)) + ax3_inner) + 903170))] = max((DepthwiseConv2d[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) % 18) * 4) + ax3_inner) + 2))]), 0.000000e+00f);
      }
    }
  }
}


