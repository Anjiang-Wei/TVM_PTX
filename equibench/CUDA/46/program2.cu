
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[40];
  __shared__ float PaddedInput_shared[2400];
  __shared__ float placeholder_shared[16];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 96; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 8))] = placeholder[(((((((((int)threadIdx.x) / 75) * 172800) + ((((((int)threadIdx.x) % 75) * 4) / 10) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + (((((int)threadIdx.x) * 4) % 10) * 192)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder[((((((((((int)threadIdx.x) / 75) * 172800) + ((((((int)threadIdx.x) % 75) * 4) / 10) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + (((((int)threadIdx.x) * 4) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder[((((((((((int)threadIdx.x) * 4) + 1) / 10) * 5760) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 1) % 10) * 192)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 1) / 10) * 5760) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 1) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder[((((((((((int)threadIdx.x) * 4) + 2) / 10) * 5760) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 2) % 10) * 192)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 2) / 10) * 5760) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 2) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder[((((((((((int)threadIdx.x) * 4) + 3) / 10) * 5760) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 3) % 10) * 192)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 3) / 10) * 5760) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 3) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 4) + 960) / 300) * 600) + ((((((int)threadIdx.x) * 4) / 10) + 6) * 20)) + (((((int)threadIdx.x) * 4) % 10) * 2)))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 960) / 300) * 172800) + ((((((int)threadIdx.x) * 4) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + (((((int)threadIdx.x) * 4) % 10) * 192)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[((((((((((int)threadIdx.x) * 4) + 960) / 300) * 600) + ((((((int)threadIdx.x) * 4) / 10) + 6) * 20)) + (((((int)threadIdx.x) * 4) % 10) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.x) * 4) + 960) / 300) * 172800) + ((((((int)threadIdx.x) * 4) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + (((((int)threadIdx.x) * 4) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 4) + 961) / 300) * 600) + (((((((int)threadIdx.x) * 4) + 1) / 10) + 6) * 20)) + ((((((int)threadIdx.x) * 4) + 1) % 10) * 2)))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 961) / 300) * 172800) + (((((((int)threadIdx.x) * 4) + 1) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 1) % 10) * 192)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[((((((((((int)threadIdx.x) * 4) + 961) / 300) * 600) + (((((((int)threadIdx.x) * 4) + 1) / 10) + 6) * 20)) + ((((((int)threadIdx.x) * 4) + 1) % 10) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.x) * 4) + 961) / 300) * 172800) + (((((((int)threadIdx.x) * 4) + 1) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 1) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 4) + 962) / 300) * 600) + (((((((int)threadIdx.x) * 4) + 2) / 10) + 6) * 20)) + ((((((int)threadIdx.x) * 4) + 2) % 10) * 2)))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 962) / 300) * 172800) + (((((((int)threadIdx.x) * 4) + 2) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 2) % 10) * 192)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[((((((((((int)threadIdx.x) * 4) + 962) / 300) * 600) + (((((((int)threadIdx.x) * 4) + 2) / 10) + 6) * 20)) + ((((((int)threadIdx.x) * 4) + 2) % 10) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.x) * 4) + 962) / 300) * 172800) + (((((((int)threadIdx.x) * 4) + 2) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 2) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 4) + 963) / 300) * 600) + (((((((int)threadIdx.x) * 4) + 3) / 10) + 6) * 20)) + ((((((int)threadIdx.x) * 4) + 3) % 10) * 2)))] = placeholder[(((((((((((int)threadIdx.x) * 4) + 963) / 300) * 172800) + (((((((int)threadIdx.x) * 4) + 3) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 3) % 10) * 192)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[((((((((((int)threadIdx.x) * 4) + 963) / 300) * 600) + (((((((int)threadIdx.x) * 4) + 3) / 10) + 6) * 20)) + ((((((int)threadIdx.x) * 4) + 3) % 10) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.x) * 4) + 963) / 300) * 172800) + (((((((int)threadIdx.x) * 4) + 3) / 10) + 6) * 5760)) + ((((int)blockIdx.x) >> 2) * 1920)) + ((((((int)threadIdx.x) * 4) + 3) % 10) * 192)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 3) {
      placeholder_shared[((((int)threadIdx.x) * 6))] = placeholder1[(((((rc_outer_outer * 64) + (((((int)threadIdx.x) * 6) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + ((((int)threadIdx.x) * 6) & 7)))];
    }
    if (((int)threadIdx.x) < 3) {
      placeholder_shared[(((((int)threadIdx.x) * 6) + 1))] = placeholder1[(((((rc_outer_outer * 64) + ((((((int)threadIdx.x) * 6) + 1) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((((int)threadIdx.x) * 6) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 3) {
      placeholder_shared[(((((int)threadIdx.x) * 6) + 2))] = placeholder1[(((((rc_outer_outer * 64) + ((((((int)threadIdx.x) * 6) + 2) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((((int)threadIdx.x) * 6) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 3) {
      placeholder_shared[(((((int)threadIdx.x) * 6) + 3))] = placeholder1[(((((rc_outer_outer * 64) + ((((((int)threadIdx.x) * 6) + 3) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((((int)threadIdx.x) * 6) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 6) + 4))] = placeholder1[(((((rc_outer_outer * 64) + ((((((int)threadIdx.x) * 6) + 4) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((((int)threadIdx.x) * 6) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 6) + 5))] = placeholder1[(((((rc_outer_outer * 64) + ((((((int)threadIdx.x) * 6) + 5) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((((int)threadIdx.x) * 6) + 5) & 7)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      Conv2dOutput[((yy_outer_inner * 5))] = (Conv2dOutput[((yy_outer_inner * 5))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 1))] = (Conv2dOutput[(((yy_outer_inner * 5) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 20))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 2))] = (Conv2dOutput[(((yy_outer_inner * 5) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 40))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 3))] = (Conv2dOutput[(((yy_outer_inner * 5) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 60))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 4))] = (Conv2dOutput[(((yy_outer_inner * 5) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 80))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 10))] = (Conv2dOutput[(((yy_outer_inner * 5) + 10))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 600))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 11))] = (Conv2dOutput[(((yy_outer_inner * 5) + 11))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 620))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 12))] = (Conv2dOutput[(((yy_outer_inner * 5) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 640))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 13))] = (Conv2dOutput[(((yy_outer_inner * 5) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 660))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 14))] = (Conv2dOutput[(((yy_outer_inner * 5) + 14))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 680))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 20))] = (Conv2dOutput[(((yy_outer_inner * 5) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1200))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 21))] = (Conv2dOutput[(((yy_outer_inner * 5) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1220))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 22))] = (Conv2dOutput[(((yy_outer_inner * 5) + 22))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1240))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 23))] = (Conv2dOutput[(((yy_outer_inner * 5) + 23))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1260))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 24))] = (Conv2dOutput[(((yy_outer_inner * 5) + 24))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1280))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 30))] = (Conv2dOutput[(((yy_outer_inner * 5) + 30))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1800))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 31))] = (Conv2dOutput[(((yy_outer_inner * 5) + 31))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1820))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 32))] = (Conv2dOutput[(((yy_outer_inner * 5) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1840))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 33))] = (Conv2dOutput[(((yy_outer_inner * 5) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1860))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 34))] = (Conv2dOutput[(((yy_outer_inner * 5) + 34))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1880))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
      Conv2dOutput[((yy_outer_inner * 5))] = (Conv2dOutput[((yy_outer_inner * 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 1))] = (Conv2dOutput[(((yy_outer_inner * 5) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 21))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 2))] = (Conv2dOutput[(((yy_outer_inner * 5) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 41))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 3))] = (Conv2dOutput[(((yy_outer_inner * 5) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 61))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 4))] = (Conv2dOutput[(((yy_outer_inner * 5) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 81))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 10))] = (Conv2dOutput[(((yy_outer_inner * 5) + 10))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 601))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 11))] = (Conv2dOutput[(((yy_outer_inner * 5) + 11))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 621))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 12))] = (Conv2dOutput[(((yy_outer_inner * 5) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 641))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 13))] = (Conv2dOutput[(((yy_outer_inner * 5) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 661))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 14))] = (Conv2dOutput[(((yy_outer_inner * 5) + 14))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 681))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 20))] = (Conv2dOutput[(((yy_outer_inner * 5) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1201))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 21))] = (Conv2dOutput[(((yy_outer_inner * 5) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1221))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 22))] = (Conv2dOutput[(((yy_outer_inner * 5) + 22))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1241))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 23))] = (Conv2dOutput[(((yy_outer_inner * 5) + 23))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1261))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 24))] = (Conv2dOutput[(((yy_outer_inner * 5) + 24))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1281))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 30))] = (Conv2dOutput[(((yy_outer_inner * 5) + 30))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1801))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 31))] = (Conv2dOutput[(((yy_outer_inner * 5) + 31))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1821))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 32))] = (Conv2dOutput[(((yy_outer_inner * 5) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1841))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 33))] = (Conv2dOutput[(((yy_outer_inner * 5) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1861))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 5) + 34))] = (Conv2dOutput[(((yy_outer_inner * 5) + 34))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 80) * 200) + (yy_outer_inner * 100)) + (((((int)threadIdx.x) % 80) >> 3) * 2)) + 1881))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 10; ++ax1_inner) {
      T_add[((((((((ax0_inner * 28800) + ((((int)threadIdx.x) / 80) * 9600)) + (ax1_inner * 960)) + ((((int)blockIdx.x) >> 2) * 320)) + (((((int)threadIdx.x) % 80) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((int)threadIdx.x) & 7)))] = (placeholder2[((((((((ax0_inner * 28800) + ((((int)threadIdx.x) / 80) * 9600)) + (ax1_inner * 960)) + ((((int)blockIdx.x) >> 2) * 320)) + (((((int)threadIdx.x) % 80) >> 3) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + (((int)threadIdx.x) & 7)))] + (Conv2dOutput[(((ax0_inner * 10) + ax1_inner))] + placeholder3[((((((int)blockIdx.x) & 3) * 8) + (((int)threadIdx.x) & 7)))]));
    }
  }
}


