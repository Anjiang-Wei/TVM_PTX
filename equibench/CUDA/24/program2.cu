
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[100];
  __shared__ float PaddedInput_shared[3136];
  __shared__ float placeholder_shared[288];
  for (int b_outer_inner_init = 0; b_outer_inner_init < 2; ++b_outer_inner_init) {
    for (int j_outer_inner_init = 0; j_outer_inner_init < 5; ++j_outer_inner_init) {
      DepthwiseConv2d[(((b_outer_inner_init * 50) + (j_outer_inner_init * 2)))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 10))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 20))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 30))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 40))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 11))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 21))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 31))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 50) + (j_outer_inner_init * 2)) + 41))] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 42; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 672) + (((int)threadIdx.x) * 42)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 3136) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) < 75) {
          PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 672) + (((int)threadIdx.x) * 42)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 1568) * 1568) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3) + (((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 224)) % 7) * 224)) + (((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224)))] = (((((1 <= ((((((int)blockIdx.x) % 108) / 36) * 5) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3) + (((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 224)) % 7))) && (((((((int)blockIdx.x) % 108) / 36) * 5) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3) + (((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 224)) % 7)) < 16)) && (1 <= ((((((int)blockIdx.x) % 36) / 12) * 5) + ((((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224) >> 5)))) && (((((((int)blockIdx.x) % 36) / 12) * 5) + ((((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224) >> 5)) < 16)) ? placeholder[(((((((((((((int)blockIdx.x) / 108) * 172800) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 672) + (((int)threadIdx.x) * 42)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 1568) * 86400)) + (((((int)blockIdx.x) % 108) / 36) * 28800)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3) + (((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 224)) % 7) * 5760)) + (((((int)blockIdx.x) % 36) / 12) * 1920)) + (((((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224) >> 5) * 384)) + ((((int)blockIdx.x) % 12) * 32)) + (((((int)threadIdx.x) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 31)) - 6144))] : 0.000000e+00f);
        }
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) >> 5) * 384) + ((((int)blockIdx.x) % 12) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) & 31)))];
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int b_outer_inner = 0; b_outer_inner < 2; ++b_outer_inner) {
      for (int j_outer_inner = 0; j_outer_inner < 5; ++j_outer_inner) {
        for (int c_outer_inner = 0; c_outer_inner < 2; ++c_outer_inner) {
          DepthwiseConv2d[((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner))] = (DepthwiseConv2d[((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner))] + (PaddedInput_shared[((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner))] * placeholder_shared[((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 10))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 10))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 224))] * placeholder_shared[((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 20))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 20))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 448))] * placeholder_shared[((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 30))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 30))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 672))] * placeholder_shared[((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 40))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 40))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 896))] * placeholder_shared[((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner))]));
          DepthwiseConv2d[((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner))] = (DepthwiseConv2d[((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 32))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 32))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 10))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 10))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 256))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 32))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 20))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 20))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 480))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 32))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 30))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 30))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 704))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 32))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 40))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 40))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 928))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 32))]));
          DepthwiseConv2d[((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner))] = (DepthwiseConv2d[((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 64))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 64))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 10))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 10))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 288))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 64))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 20))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 20))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 512))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 64))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 30))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 30))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 736))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 64))]));
          DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 40))] = (DepthwiseConv2d[(((((b_outer_inner * 50) + (j_outer_inner * 2)) + c_outer_inner) + 40))] + (PaddedInput_shared[(((((((b_outer_inner * 1568) + (di_outer_inner * 224)) + (j_outer_inner * 32)) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 960))] * placeholder_shared[(((((di_outer_inner * 96) + (((int)threadIdx.x) * 2)) + c_outer_inner) + 64))]));
        }
      }
    }
  }
  for (int i0_inner = 0; i0_inner < 2; ++i0_inner) {
    for (int i1_inner = 0; i1_inner < 5; ++i1_inner) {
      for (int i2_inner = 0; i2_inner < 5; ++i2_inner) {
        for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
          compute[(((((((((((((int)blockIdx.x) / 108) * 172800) + (i0_inner * 86400)) + (((((int)blockIdx.x) % 108) / 36) * 28800)) + (i1_inner * 5760)) + (((((int)blockIdx.x) % 36) / 12) * 1920)) + (i2_inner * 384)) + ((((int)blockIdx.x) % 12) * 32)) + (((int)threadIdx.x) * 2)) + i3_inner))] = max(min((DepthwiseConv2d[(((((i0_inner * 50) + (i1_inner * 10)) + (i2_inner * 2)) + i3_inner))] + placeholder2[(((((((int)blockIdx.x) % 12) * 32) + (((int)threadIdx.x) * 2)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
        }
      }
    }
  }
}


