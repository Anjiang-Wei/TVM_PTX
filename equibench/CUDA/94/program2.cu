
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float compute[256];
  __shared__ float placeholder_d_shared[128];
  __shared__ float placeholder_shared[1536];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 64; ++i_outer_inner_init) {
    for (int j_outer_inner_init = 0; j_outer_inner_init < 2; ++j_outer_inner_init) {
      for (int j_inner_init = 0; j_inner_init < 2; ++j_inner_init) {
        compute[((((i_outer_inner_init * 4) + (j_outer_inner_init * 2)) + j_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_inner_s < 16; ++ax0_ax1_fused_ax2_fused_inner_s) {
      if (((((int)threadIdx.x) * 16) + ax0_ax1_fused_ax2_fused_inner_s) < 128) {
        if (((int)threadIdx.x) < 8) {
          placeholder_d_shared[(((((int)threadIdx.x) * 16) + ax0_ax1_fused_ax2_fused_inner_s))] = placeholder[((((((((int)blockIdx.x) >> 1) * 131072) + (((int)threadIdx.x) * 16384)) + (ax0_ax1_fused_ax2_fused_inner_s * 1024)) + k_outer_outer))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 768) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.x) & 1) * 1572864) + (ax0_ax1_fused_ax2_fused_outer_outer * 786432)) + (((int)threadIdx.x) * 1024)) + k_outer_outer))];
    }
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 64; ++i_outer_inner) {
      for (int j_outer_inner = 0; j_outer_inner < 2; ++j_outer_inner) {
        for (int j_inner = 0; j_inner < 2; ++j_inner) {
          compute[((((i_outer_inner * 4) + (j_outer_inner * 2)) + j_inner))] = (compute[((((i_outer_inner * 4) + (j_outer_inner * 2)) + j_inner))] + (placeholder_d_shared[((((((int)threadIdx.x) / 384) * 64) + i_outer_inner))] * placeholder_shared[(((((((int)threadIdx.x) % 384) * 4) + (j_outer_inner * 2)) + j_inner))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 64; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      T_add[((((((((((int)blockIdx.x) >> 1) * 393216) + ((((int)threadIdx.x) / 384) * 196608)) + (ax1_inner * 3072)) + ((((int)blockIdx.x) & 1) * 1536)) + ((((int)threadIdx.x) % 384) * 4)) + ax2_inner))] = (compute[(((ax1_inner * 4) + ax2_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 1536) + ((((int)threadIdx.x) % 384) * 4)) + ax2_inner))]);
    }
  }
}


