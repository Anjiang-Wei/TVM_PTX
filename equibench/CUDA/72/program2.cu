
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 184))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 184))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[4];
  __shared__ float PaddedInput_shared[4416];
  __shared__ float placeholder_shared[414];
  DepthwiseConv2d_local[(0)] = 0.000000e+00f;
  DepthwiseConv2d_local[(2)] = 0.000000e+00f;
  DepthwiseConv2d_local[(1)] = 0.000000e+00f;
  DepthwiseConv2d_local[(3)] = 0.000000e+00f;
  PaddedInput_shared[((((int)threadIdx.x) * 9))] = (((((1 <= ((((((int)threadIdx.x) * 9) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= ((((int)threadIdx.x) * 9) % 736))) && (((((int)threadIdx.x) * 9) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + (((((int)threadIdx.x) * 9) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((int)threadIdx.x) * 9) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + ((((int)threadIdx.x) * 9) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 1))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 1) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 1) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 1) % 736))) && ((((((int)threadIdx.x) * 9) + 1) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 1) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 1) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 1) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 1) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 2))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 2) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 2) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 2) % 736))) && ((((((int)threadIdx.x) * 9) + 2) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 2) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 2) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 2) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 3))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 3) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 3) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 3) % 736))) && ((((((int)threadIdx.x) * 9) + 3) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 3) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 3) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 3) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 3) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 4))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 4) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 4) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 4) % 736))) && ((((((int)threadIdx.x) * 9) + 4) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 4) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 4) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 4) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 4) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 5))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 5) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 5) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 5) % 736))) && ((((((int)threadIdx.x) * 9) + 5) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 5) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 5) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 5) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 5) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 6))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 6) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 6) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 6) % 736))) && ((((((int)threadIdx.x) * 9) + 6) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 6) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 6) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 6) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 6) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 7))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 7) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 7) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 7) % 736))) && ((((((int)threadIdx.x) * 9) + 7) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 7) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 7) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 7) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 7) % 46)) - 2760))] : 0.000000e+00f);
  PaddedInput_shared[(((((int)threadIdx.x) * 9) + 8))] = (((((1 <= (((((((int)threadIdx.x) * 9) + 8) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && ((((((((int)threadIdx.x) * 9) + 8) % 2208) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (46 <= (((((int)threadIdx.x) * 9) + 8) % 736))) && ((((((int)threadIdx.x) * 9) + 8) % 736) < 690)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 8) / 2208) * 36064)) + (((((((int)threadIdx.x) * 9) + 8) % 2208) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) + 8) % 736) / 46) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 8) % 46)) - 2760))] : 0.000000e+00f);
  if (((int)threadIdx.x) < 169) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2898) / 736) * 736) + (((((((int)threadIdx.x) * 9) / 46) + 15) & 15) * 46)) + ((((int)threadIdx.x) * 9) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 690) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 690) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= ((((((int)threadIdx.x) * 9) / 46) + 15) & 15))) && (((((((int)threadIdx.x) * 9) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2898) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 690) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + (((((((int)threadIdx.x) * 9) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + ((((int)threadIdx.x) * 9) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 169) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2899) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 1) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 1) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 691) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 691) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 1) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 1) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2899) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 691) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 1) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 1) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 169) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2900) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 2) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 2) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 692) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 692) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 2) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 2) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2900) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 692) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 2) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 2) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 169) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2901) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 3) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 3) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 693) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 693) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 3) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 3) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2901) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 693) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 3) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 3) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 169) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2902) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 4) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 4) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 694) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 694) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 4) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 4) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2902) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 694) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 4) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 4) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 169) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2903) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 5) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 5) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 695) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 695) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 5) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 5) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2903) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 695) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 5) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 5) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 168) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2904) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 6) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 6) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 696) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 696) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 6) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 6) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2904) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 696) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 6) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 6) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 168) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2905) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 7) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 7) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 697) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 697) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 7) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 7) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2905) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 697) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 7) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 7) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 168) {
    PaddedInput_shared[(((((((((int)threadIdx.x) * 9) + 2906) / 736) * 736) + ((((((((int)threadIdx.x) * 9) + 8) / 46) + 15) & 15) * 46)) + (((((int)threadIdx.x) * 9) + 8) % 46)))] = (((((1 <= ((((((int)threadIdx.x) * 9) + 698) / 736) + ((((int)blockIdx.x) % 56) >> 2))) && (((((((int)threadIdx.x) * 9) + 698) / 736) + ((((int)blockIdx.x) % 56) >> 2)) < 15)) && (1 <= (((((((int)threadIdx.x) * 9) + 8) / 46) + 15) & 15))) && ((((((((int)threadIdx.x) * 9) + 8) / 46) + 15) & 15) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 56) * 72128) + ((((((int)threadIdx.x) * 9) + 2906) / 2208) * 36064)) + ((((((int)threadIdx.x) * 9) + 698) / 736) * 2576)) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((((((int)threadIdx.x) * 9) + 8) / 46) + 15) & 15) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 8) % 46)) - 2760))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[((((int)threadIdx.x) * 9))] = placeholder1[((((((((int)threadIdx.x) * 9) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + ((((int)threadIdx.x) * 9) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 1))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 1) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 1) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 2))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 2) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 2) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 3))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 3) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 3) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 4))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 4) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 4) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 5))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 5) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 5) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 6))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 6) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 6) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 7))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 7) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 7) % 46)))];
  }
  if (((int)threadIdx.x) < 46) {
    placeholder_shared[(((((int)threadIdx.x) * 9) + 8))] = placeholder1[(((((((((int)threadIdx.x) * 9) + 8) / 46) * 184) + ((((int)blockIdx.x) & 3) * 46)) + (((((int)threadIdx.x) * 9) + 8) % 46)))];
  }
  __syncthreads();
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 2))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2208))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 2))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 1))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2209))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 1))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 46))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 46))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2254))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 46))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 47))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 47))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2255))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 47))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 92))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 92))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2300))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 92))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 93))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 93))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2301))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 93))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 736))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 138))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2944))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 138))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 737))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 139))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2945))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 139))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 782))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 184))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2990))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 184))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 783))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 185))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2991))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 185))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 828))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 230))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3036))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 230))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 829))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 231))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3037))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 231))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1472))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 276))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3680))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 276))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1473))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 277))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3681))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 277))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1518))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 322))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3726))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 322))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1519))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 323))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3727))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 323))]));
  DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1564))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 368))]));
  DepthwiseConv2d_local[(2)] = (DepthwiseConv2d_local[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3772))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 368))]));
  DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1565))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 369))]));
  DepthwiseConv2d_local[(3)] = (DepthwiseConv2d_local[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3773))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 2) + 369))]));
  for (int c_inner = 0; c_inner < 2; ++c_inner) {
    DepthwiseConv2d[((((((((((int)blockIdx.x) / 56) * 72128) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((int)threadIdx.x) / 23) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + ((((int)threadIdx.x) % 23) * 2)) + c_inner))] = DepthwiseConv2d_local[(c_inner)];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 56) * 72128) + (((((int)blockIdx.x) % 56) >> 2) * 2576)) + ((((int)threadIdx.x) / 23) * 184)) + ((((int)blockIdx.x) & 3) * 46)) + ((((int)threadIdx.x) % 23) * 2)) + c_inner) + 36064))] = DepthwiseConv2d_local[((c_inner + 2))];
  }
}


