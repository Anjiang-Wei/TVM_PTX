
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ compute, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] = ((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * (5.000000e-01f + (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * -2.726142e-10f) + 2.770681e-08f)) + -2.101024e-06f)) + -5.692506e-05f)) + -7.349906e-04f)) + -2.954600e-03f)) + -1.609603e-02f)) / (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * (((max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f) * max(min(((compute[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) & 4095))]) * 7.071068e-01f), 4.000000e+00f), -4.000000e+00f)) * -1.456607e-05f) + -2.133740e-04f)) + -1.682827e-03f)) + -7.373329e-03f)) + -1.426474e-02f)) * 5.000000e-01f)));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[256];
  __shared__ float placeholder_d_shared[256];
  __shared__ float placeholder_shared[1024];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      compute_local[(((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 64))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 128))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 192))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 65))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 129))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 193))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 66))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 130))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 194))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 67))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 131))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 195))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 32))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 96))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 160))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 224))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 33))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 97))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 161))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 225))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 34))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 98))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 162))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 226))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 35))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 99))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 163))] = 0.000000e+00f;
      compute_local[((((i_c_outer_inner_init * 16) + (j_c_outer_inner_init * 4)) + 227))] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 256) {
      placeholder_d_shared[(((int)threadIdx.x))] = placeholder[((((((((int)threadIdx.x) >> 7) * 262144) + ((((int)blockIdx.x) >> 2) * 131072)) + ((((int)threadIdx.x) & 127) * 1024)) + k_outer_outer))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[((((int)threadIdx.x) * 16))] = placeholder1[(((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 1))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 1024))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 2))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 2048))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 3))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 3072))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 4))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 4096))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 5))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 5120))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 6))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 6144))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 7))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 7168))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 8))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 8192))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 9))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 9216))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 10))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 10240))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 11))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 11264))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 12))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 12288))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 13))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 13312))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 14))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 14336))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 15))] = placeholder1[((((((((int)blockIdx.x) & 3) * 1048576) + (((int)threadIdx.x) * 16384)) + k_outer_outer) + 15360))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        compute_local[(((i_c_outer_inner * 16) + (j_c_outer_inner * 4)))] = (compute_local[(((i_c_outer_inner * 16) + (j_c_outer_inner * 4)))] + (placeholder_d_shared[((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 64))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 64))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 128))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 128))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 192))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 192))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 1))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 1))] + (placeholder_d_shared[((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 65))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 65))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 32))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 129))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 129))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 64))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 193))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 193))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 96))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 2))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 2))] + (placeholder_d_shared[((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 66))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 66))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 32))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 130))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 130))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 64))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 194))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 194))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 96))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 3))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 3))] + (placeholder_d_shared[((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 67))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 67))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 32))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 131))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 131))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 64))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 195))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 195))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 96))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 32))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 32))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 96))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 96))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 160))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 160))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 224))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 224))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 33))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 33))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 128))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 97))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 97))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 160))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 161))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 161))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 192))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 225))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 225))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 224))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 1))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 34))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 34))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 128))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 98))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 98))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 160))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 162))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 162))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 192))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 226))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 226))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 224))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 2))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 35))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 35))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 128))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 99))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 99))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 160))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 163))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 163))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 192))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
        compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 227))] = (compute_local[((((i_c_outer_inner * 16) + (j_c_outer_inner * 4)) + 227))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 6) * 2) + i_c_outer_inner) + 224))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 16) + (j_c_outer_inner * 4)) + 3))]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      for (int j_inner = 0; j_inner < 16; ++j_inner) {
        compute[((((((((b_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 524288)) + ((((int)threadIdx.x) >> 6) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 16)) + j_inner))] = compute_local[((((b_inner * 32) + (i_inner * 16)) + j_inner))];
        compute[(((((((((b_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 524288)) + ((((int)threadIdx.x) >> 6) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 16)) + j_inner) + 131072))] = compute_local[(((((b_inner * 32) + (i_inner * 16)) + j_inner) + 64))];
        compute[(((((((((b_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 524288)) + ((((int)threadIdx.x) >> 6) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 16)) + j_inner) + 262144))] = compute_local[(((((b_inner * 32) + (i_inner * 16)) + j_inner) + 128))];
        compute[(((((((((b_inner * 1048576) + ((((int)blockIdx.x) >> 2) * 524288)) + ((((int)threadIdx.x) >> 6) * 8192)) + (i_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 63) * 16)) + j_inner) + 393216))] = compute_local[(((((b_inner * 32) + (i_inner * 16)) + j_inner) + 192))];
      }
    }
  }
}


