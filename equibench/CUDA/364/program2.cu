
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float conv2d_transpose_nhwc[128];
  __shared__ float PadInput_shared[864];
  __shared__ float placeholder_shared[2048];
  conv2d_transpose_nhwc[(0)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(32)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(64)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(96)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(4)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(36)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(68)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(100)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(16)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(48)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(80)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(112)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(20)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(52)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(84)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(116)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(1)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(33)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(65)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(97)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(5)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(37)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(69)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(101)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(17)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(49)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(81)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(113)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(21)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(53)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(85)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(117)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(2)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(34)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(66)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(98)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(6)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(38)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(70)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(102)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(18)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(50)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(82)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(114)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(22)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(54)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(86)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(118)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(3)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(35)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(67)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(99)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(7)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(39)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(71)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(103)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(19)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(51)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(83)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(115)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(23)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(55)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(87)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(119)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(8)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(40)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(72)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(104)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(12)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(44)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(76)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(108)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(24)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(56)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(88)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(120)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(28)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(60)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(92)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(124)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(9)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(41)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(73)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(105)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(13)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(45)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(77)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(109)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(25)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(57)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(89)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(121)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(29)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(61)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(93)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(125)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(10)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(42)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(74)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(106)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(14)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(46)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(78)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(110)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(26)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(58)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(90)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(122)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(30)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(62)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(94)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(126)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(11)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(43)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(75)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(107)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(15)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(47)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(79)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(111)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(27)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(59)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(91)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(123)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(31)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(63)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(95)] = 0.000000e+00f;
  conv2d_transpose_nhwc[(127)] = 0.000000e+00f;
  for (int rh_outer_outer = 0; rh_outer_outer < 2; ++rh_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
      __syncthreads();
      PadInput_shared[(((int)threadIdx.x))] = (((((1 <= (((((int)threadIdx.x) % 216) / 24) + rh_outer_outer)) && ((((((int)threadIdx.x) % 216) / 24) + rh_outer_outer) < 9)) && (1 <= (((((int)blockIdx.x) >> 2) * 4) + ((((int)threadIdx.x) % 24) >> 2)))) && ((((((int)blockIdx.x) >> 2) * 4) + ((((int)threadIdx.x) % 24) >> 2)) < 9)) ? placeholder[((((((((((((int)threadIdx.x) / 216) * 32768) + (((((int)threadIdx.x) % 216) / 24) * 4096)) + (rh_outer_outer * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + (((((int)threadIdx.x) % 24) >> 2) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 4608))] : 0.000000e+00f);
      PadInput_shared[((((int)threadIdx.x) + 256))] = (((((1 <= (((((((int)threadIdx.x) >> 2) + 10) % 54) / 6) + rh_outer_outer)) && ((((((((int)threadIdx.x) >> 2) + 10) % 54) / 6) + rh_outer_outer) < 9)) && (1 <= (((((int)blockIdx.x) >> 2) * 4) + (((((int)threadIdx.x) >> 2) + 4) % 6)))) && ((((((int)blockIdx.x) >> 2) * 4) + (((((int)threadIdx.x) >> 2) + 4) % 6)) < 9)) ? placeholder[(((((((((((((int)threadIdx.x) + 256) / 216) * 32768) + (((((((int)threadIdx.x) >> 2) + 10) % 54) / 6) * 4096)) + (rh_outer_outer * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + ((((((int)threadIdx.x) >> 2) + 4) % 6) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 4608))] : 0.000000e+00f);
      PadInput_shared[((((int)threadIdx.x) + 512))] = (((((1 <= (((((((int)threadIdx.x) >> 2) + 20) % 54) / 6) + rh_outer_outer)) && ((((((((int)threadIdx.x) >> 2) + 20) % 54) / 6) + rh_outer_outer) < 9)) && (1 <= (((((int)blockIdx.x) >> 2) * 4) + (((((int)threadIdx.x) >> 2) + 2) % 6)))) && ((((((int)blockIdx.x) >> 2) * 4) + (((((int)threadIdx.x) >> 2) + 2) % 6)) < 9)) ? placeholder[(((((((((((((int)threadIdx.x) + 512) / 216) * 32768) + (((((((int)threadIdx.x) >> 2) + 20) % 54) / 6) * 4096)) + (rh_outer_outer * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + ((((((int)threadIdx.x) >> 2) + 2) % 6) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 4608))] : 0.000000e+00f);
      if (((int)threadIdx.x) < 96) {
        PadInput_shared[((((int)threadIdx.x) + 768))] = ((((((((((int)threadIdx.x) >> 2) + 30) / 6) + rh_outer_outer) < 9) && (1 <= (((((int)blockIdx.x) >> 2) * 4) + ((((int)threadIdx.x) % 24) >> 2)))) && ((((((int)blockIdx.x) >> 2) * 4) + ((((int)threadIdx.x) % 24) >> 2)) < 9)) ? placeholder[(((((((((((((int)threadIdx.x) + 768) / 216) * 32768) + ((((((int)threadIdx.x) >> 2) + 30) / 6) * 4096)) + (rh_outer_outer * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + (((((int)threadIdx.x) % 24) >> 2) * 512)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 4608))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[((((int)threadIdx.x) * 32))] = placeholder1[(((((((((((int)threadIdx.x) >> 3) * 131072) + (rc_outer_outer * 1024)) + (((((int)threadIdx.x) & 7) >> 1) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 1))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 1) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 1) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 1) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 2))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 2) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 2) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 2) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 3))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 3) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 3) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 3) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 4))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 4) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 4) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 4) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 5))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 5) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 5) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 5) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 6))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 6) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 6) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 6) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 7))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 7) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 7) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 7) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 8))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 8) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 8) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 8) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 9))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 9) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 9) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 9) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 10))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 10) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 10) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 10) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 11))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 11) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 11) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 11) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 12))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 12) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 12) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 12) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 13))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 13) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 13) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 13) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 14))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 14) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 14) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 14) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 15))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 15) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 15) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 15) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 16))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 16) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 16) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 16) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 17))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 17) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 17) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 17) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 18))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 18) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 18) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 18) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 19))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 19) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 19) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 19) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 20))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 20) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 20) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 20) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 21))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 21) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 21) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 21) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 22))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 22) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 22) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 22) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 23))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 23) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 23) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 23) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 24))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 24) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 24) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 24) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 25))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 25) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 25) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 25) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 26))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 26) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 26) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 26) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 27))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 27) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 27) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 27) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 28))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 28) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 28) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 28) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 29))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 29) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 29) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 29) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 30))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 30) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 30) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 30) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      if (((int)threadIdx.x) < 64) {
        placeholder_shared[(((((int)threadIdx.x) * 32) + 31))] = placeholder1[(((((((((((((int)threadIdx.x) * 32) + 31) >> 8) * 131072) + (rc_outer_outer * 1024)) + (((((((int)threadIdx.x) * 32) + 31) & 255) >> 6) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + (((((int)threadIdx.x) * 32) + 31) & 63)) + 1048576) - (rh_outer_outer * 1048576)))];
      }
      __syncthreads();
      for (int rw_outer_inner = 0; rw_outer_inner < 4; ++rw_outer_inner) {
        for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
          conv2d_transpose_nhwc[(0)] = (conv2d_transpose_nhwc[(0)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(32)] = (conv2d_transpose_nhwc[(32)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 8))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(64)] = (conv2d_transpose_nhwc[(64)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 432))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(96)] = (conv2d_transpose_nhwc[(96)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 440))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(16)] = (conv2d_transpose_nhwc[(16)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 216))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(48)] = (conv2d_transpose_nhwc[(48)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 224))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(80)] = (conv2d_transpose_nhwc[(80)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 648))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(112)] = (conv2d_transpose_nhwc[(112)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 656))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(4)] = (conv2d_transpose_nhwc[(4)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(36)] = (conv2d_transpose_nhwc[(36)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(68)] = (conv2d_transpose_nhwc[(68)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(100)] = (conv2d_transpose_nhwc[(100)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(20)] = (conv2d_transpose_nhwc[(20)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(52)] = (conv2d_transpose_nhwc[(52)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(84)] = (conv2d_transpose_nhwc[(84)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(116)] = (conv2d_transpose_nhwc[(116)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(1)] = (conv2d_transpose_nhwc[(1)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(33)] = (conv2d_transpose_nhwc[(33)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 8))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(65)] = (conv2d_transpose_nhwc[(65)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 432))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(97)] = (conv2d_transpose_nhwc[(97)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 440))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(17)] = (conv2d_transpose_nhwc[(17)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 216))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(49)] = (conv2d_transpose_nhwc[(49)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 224))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(81)] = (conv2d_transpose_nhwc[(81)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 648))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(113)] = (conv2d_transpose_nhwc[(113)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 656))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(5)] = (conv2d_transpose_nhwc[(5)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(37)] = (conv2d_transpose_nhwc[(37)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(69)] = (conv2d_transpose_nhwc[(69)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(101)] = (conv2d_transpose_nhwc[(101)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(21)] = (conv2d_transpose_nhwc[(21)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(53)] = (conv2d_transpose_nhwc[(53)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(85)] = (conv2d_transpose_nhwc[(85)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(117)] = (conv2d_transpose_nhwc[(117)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(2)] = (conv2d_transpose_nhwc[(2)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(34)] = (conv2d_transpose_nhwc[(34)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 8))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(66)] = (conv2d_transpose_nhwc[(66)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 432))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(98)] = (conv2d_transpose_nhwc[(98)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 440))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(18)] = (conv2d_transpose_nhwc[(18)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 216))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(50)] = (conv2d_transpose_nhwc[(50)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 224))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(82)] = (conv2d_transpose_nhwc[(82)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 648))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(114)] = (conv2d_transpose_nhwc[(114)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 656))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(6)] = (conv2d_transpose_nhwc[(6)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(38)] = (conv2d_transpose_nhwc[(38)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(70)] = (conv2d_transpose_nhwc[(70)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(102)] = (conv2d_transpose_nhwc[(102)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(22)] = (conv2d_transpose_nhwc[(22)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(54)] = (conv2d_transpose_nhwc[(54)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(86)] = (conv2d_transpose_nhwc[(86)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(118)] = (conv2d_transpose_nhwc[(118)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(3)] = (conv2d_transpose_nhwc[(3)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(35)] = (conv2d_transpose_nhwc[(35)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 8))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(67)] = (conv2d_transpose_nhwc[(67)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 432))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(99)] = (conv2d_transpose_nhwc[(99)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 440))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(19)] = (conv2d_transpose_nhwc[(19)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 216))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(51)] = (conv2d_transpose_nhwc[(51)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 224))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(83)] = (conv2d_transpose_nhwc[(83)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 648))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(115)] = (conv2d_transpose_nhwc[(115)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 656))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(7)] = (conv2d_transpose_nhwc[(7)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(39)] = (conv2d_transpose_nhwc[(39)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(71)] = (conv2d_transpose_nhwc[(71)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(103)] = (conv2d_transpose_nhwc[(103)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(23)] = (conv2d_transpose_nhwc[(23)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(55)] = (conv2d_transpose_nhwc[(55)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(87)] = (conv2d_transpose_nhwc[(87)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(119)] = (conv2d_transpose_nhwc[(119)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(8)] = (conv2d_transpose_nhwc[(8)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(40)] = (conv2d_transpose_nhwc[(40)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(72)] = (conv2d_transpose_nhwc[(72)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(104)] = (conv2d_transpose_nhwc[(104)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(24)] = (conv2d_transpose_nhwc[(24)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(56)] = (conv2d_transpose_nhwc[(56)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(88)] = (conv2d_transpose_nhwc[(88)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(120)] = (conv2d_transpose_nhwc[(120)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(12)] = (conv2d_transpose_nhwc[(12)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 48))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(44)] = (conv2d_transpose_nhwc[(44)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 56))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(76)] = (conv2d_transpose_nhwc[(76)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 480))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(108)] = (conv2d_transpose_nhwc[(108)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 488))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(28)] = (conv2d_transpose_nhwc[(28)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 264))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(60)] = (conv2d_transpose_nhwc[(60)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 272))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(92)] = (conv2d_transpose_nhwc[(92)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 696))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(124)] = (conv2d_transpose_nhwc[(124)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 704))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(9)] = (conv2d_transpose_nhwc[(9)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(41)] = (conv2d_transpose_nhwc[(41)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(73)] = (conv2d_transpose_nhwc[(73)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(105)] = (conv2d_transpose_nhwc[(105)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(25)] = (conv2d_transpose_nhwc[(25)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(57)] = (conv2d_transpose_nhwc[(57)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(89)] = (conv2d_transpose_nhwc[(89)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(121)] = (conv2d_transpose_nhwc[(121)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(13)] = (conv2d_transpose_nhwc[(13)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 48))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(45)] = (conv2d_transpose_nhwc[(45)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 56))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(77)] = (conv2d_transpose_nhwc[(77)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 480))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(109)] = (conv2d_transpose_nhwc[(109)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 488))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(29)] = (conv2d_transpose_nhwc[(29)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 264))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(61)] = (conv2d_transpose_nhwc[(61)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 272))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(93)] = (conv2d_transpose_nhwc[(93)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 696))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(125)] = (conv2d_transpose_nhwc[(125)] + ((((rw_outer_inner % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + ((rw_outer_inner >> 1) * 4)) + rc_outer_inner) + 704))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(10)] = (conv2d_transpose_nhwc[(10)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(42)] = (conv2d_transpose_nhwc[(42)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(74)] = (conv2d_transpose_nhwc[(74)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(106)] = (conv2d_transpose_nhwc[(106)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(26)] = (conv2d_transpose_nhwc[(26)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(58)] = (conv2d_transpose_nhwc[(58)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(90)] = (conv2d_transpose_nhwc[(90)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(122)] = (conv2d_transpose_nhwc[(122)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1792) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(14)] = (conv2d_transpose_nhwc[(14)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 48))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(46)] = (conv2d_transpose_nhwc[(46)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 56))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(78)] = (conv2d_transpose_nhwc[(78)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 480))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(110)] = (conv2d_transpose_nhwc[(110)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 488))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(30)] = (conv2d_transpose_nhwc[(30)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 264))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(62)] = (conv2d_transpose_nhwc[(62)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 272))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(94)] = (conv2d_transpose_nhwc[(94)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 696))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(126)] = (conv2d_transpose_nhwc[(126)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 704))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 768) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(11)] = (conv2d_transpose_nhwc[(11)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 24))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(43)] = (conv2d_transpose_nhwc[(43)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 32))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(75)] = (conv2d_transpose_nhwc[(75)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 456))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(107)] = (conv2d_transpose_nhwc[(107)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 464))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(27)] = (conv2d_transpose_nhwc[(27)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 240))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(59)] = (conv2d_transpose_nhwc[(59)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 248))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(91)] = (conv2d_transpose_nhwc[(91)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 672))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(123)] = (conv2d_transpose_nhwc[(123)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 680))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1793) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(15)] = (conv2d_transpose_nhwc[(15)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 48))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(47)] = (conv2d_transpose_nhwc[(47)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 56))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(79)] = (conv2d_transpose_nhwc[(79)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 480))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(111)] = (conv2d_transpose_nhwc[(111)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 488))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(31)] = (conv2d_transpose_nhwc[(31)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 264))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(63)] = (conv2d_transpose_nhwc[(63)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 272))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(95)] = (conv2d_transpose_nhwc[(95)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 696))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
          conv2d_transpose_nhwc[(127)] = (conv2d_transpose_nhwc[(127)] + (((((rw_outer_inner + 1) % 2) == 0) ? PadInput_shared[(((((((((int)threadIdx.x) >> 6) * 48) + (((((int)threadIdx.x) & 63) >> 5) * 4)) + (((rw_outer_inner + 1) >> 1) * 4)) + rc_outer_inner) + 704))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 769) - (rw_outer_inner * 256)))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
          T_relu[((((((((((ax0_inner * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))] = max(((conv2d_transpose_nhwc[(((((ax0_inner * 16) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner))] * placeholder2[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 1024))] = max(((conv2d_transpose_nhwc[((((((ax0_inner * 16) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner) + 32))] * placeholder2[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 131072))] = max(((conv2d_transpose_nhwc[((((((ax0_inner * 16) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner) + 64))] * placeholder2[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 65536) + ((((int)threadIdx.x) >> 6) * 16384)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) >> 2) * 2048)) + (((((int)threadIdx.x) & 63) >> 5) * 512)) + (ax2_inner * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 132096))] = max(((conv2d_transpose_nhwc[((((((ax0_inner * 16) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner) + 96))] * placeholder2[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]) + placeholder3[(((((((int)blockIdx.x) & 3) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


