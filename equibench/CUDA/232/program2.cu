
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[90];
  __shared__ float PaddedInput_shared[360];
  __shared__ float placeholder_shared[1024];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 3; ++yy_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 2) + ff_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 12))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 18))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 24))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 30))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 36))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 42))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 48))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 54))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 60))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 66))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 72))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 78))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 2) + ff_inner_init) + 84))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 52; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 360) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) < 72) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 120) * 120) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 3)) % 15) * 8)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 7)))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 37440) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 320) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 120) * 12480)) + ((((int)blockIdx.x) & 1) * 6240)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 3)) % 15) * 416)) + (rc_outer_outer * 8)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 7)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      ((float4*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 256) + (((int)threadIdx.x) * 4)))))[0] = ((float4*)(placeholder1 + ((((rc_outer_outer * 1024) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 256)) + (((int)threadIdx.x) * 4)))))[0];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 3; ++yy_outer_inner) {
      for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
        for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
          Conv2dOutput[(((yy_outer_inner * 2) + ff_inner))] = (Conv2dOutput[(((yy_outer_inner * 2) + ff_inner))] + (PaddedInput_shared[(((yy_outer_inner * 120) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 6))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 6))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 8))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 12))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 12))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 16))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 18))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 18))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 24))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 24))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 24))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 32))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 30))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 30))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 40))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 36))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 36))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 48))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 42))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 42))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 56))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 48))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 48))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 64))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 54))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 54))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 72))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 60))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 60))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 80))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 66))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 66))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 88))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 72))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 72))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 96))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 78))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 78))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 104))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
          Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 84))] = (Conv2dOutput[((((yy_outer_inner * 2) + ff_inner) + 84))] + (PaddedInput_shared[((((yy_outer_inner * 120) + rc_inner) + 112))] * placeholder_shared[((((rc_inner * 128) + (((int)threadIdx.x) * 2)) + ff_inner))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 128))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 6))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 256))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 12))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 384))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 18))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 512))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 24))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 640))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 30))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 768))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 36))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 896))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 42))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1024))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 48))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1152))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 54))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1280))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 60))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1408))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 66))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1536))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 72))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1664))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 78))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 11520) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1792))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 84))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


