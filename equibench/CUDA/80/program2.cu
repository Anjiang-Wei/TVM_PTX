
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[160];
  __shared__ float PaddedInput_shared[6400];
  __shared__ float placeholder_shared[4000];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 80; ++ff_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 80) + ff_outer_inner_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 100; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((int)blockIdx.x) * 25600) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) / 50) * 200)) + (rc_outer_outer * 50)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) % 50)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 63; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + ((int)threadIdx.x)) < 4000) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 4000) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64)) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 10; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 80; ++ff_outer_inner) {
          for (int rc_inner = 0; rc_inner < 5; ++rc_inner) {
            Conv2dOutput[(((yy_outer_inner * 80) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 80) + ff_outer_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 1600) + (yy_outer_inner * 800)) + ((((int)threadIdx.x) & 15) * 50)) + (rc_outer_inner * 5)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 400) + (rc_inner * 80)) + ff_outer_inner))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 80; ++ax3_inner) {
      T_add[((((((((int)blockIdx.x) * 10240) + ((((int)threadIdx.x) >> 4) * 2560)) + (ax1_inner * 1280)) + ((((int)threadIdx.x) & 15) * 80)) + ax3_inner))] = ((Conv2dOutput[(((ax1_inner * 80) + ax3_inner))] + placeholder2[(ax3_inner)]) + placeholder3[((((((((int)blockIdx.x) * 10240) + ((((int)threadIdx.x) >> 4) * 2560)) + (ax1_inner * 1280)) + ((((int)threadIdx.x) & 15) * 80)) + ax3_inner))]);
    }
  }
}


