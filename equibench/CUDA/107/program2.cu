
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[200];
  __shared__ float PaddedInput_shared[900];
  __shared__ float placeholder_shared[384];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 100))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 110))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 120))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 30))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 130))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 40))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 140))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 50))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 150))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 60))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 160))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 70))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 170))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 80))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 180))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 90))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 190))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 101))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 111))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 121))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 31))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 131))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 41))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 141))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 51))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 151))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 61))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 161))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 71))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 171))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 81))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 181))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 91))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 191))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 144; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 18))] = placeholder[((((((((int)threadIdx.x) / 25) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((int)threadIdx.x) % 25) * 2592)) + (rc_outer_outer * 6)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 1))] = placeholder[(((((((((int)threadIdx.x) / 25) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((int)threadIdx.x) % 25) * 2592)) + (rc_outer_outer * 6)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 2))] = placeholder[(((((((((int)threadIdx.x) / 25) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((int)threadIdx.x) % 25) * 2592)) + (rc_outer_outer * 6)) + 2))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 3))] = placeholder[(((((((((int)threadIdx.x) / 25) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((int)threadIdx.x) % 25) * 2592)) + (rc_outer_outer * 6)) + 3))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 4))] = placeholder[(((((((((int)threadIdx.x) / 25) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((int)threadIdx.x) % 25) * 2592)) + (rc_outer_outer * 6)) + 4))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 5))] = placeholder[(((((((((int)threadIdx.x) / 25) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((int)threadIdx.x) % 25) * 2592)) + (rc_outer_outer * 6)) + 5))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 6))] = placeholder[((((((((((int)threadIdx.x) * 3) + 1) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 1) % 75) * 864)) + (rc_outer_outer * 6)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 7))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 1) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 1) % 75) * 864)) + (rc_outer_outer * 6)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 8))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 1) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 1) % 75) * 864)) + (rc_outer_outer * 6)) + 2))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 9))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 1) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 1) % 75) * 864)) + (rc_outer_outer * 6)) + 3))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 10))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 1) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 1) % 75) * 864)) + (rc_outer_outer * 6)) + 4))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 11))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 1) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 1) % 75) * 864)) + (rc_outer_outer * 6)) + 5))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 12))] = placeholder[((((((((((int)threadIdx.x) * 3) + 2) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 2) % 75) * 864)) + (rc_outer_outer * 6)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 13))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 2) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 2) % 75) * 864)) + (rc_outer_outer * 6)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 14))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 2) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 2) % 75) * 864)) + (rc_outer_outer * 6)) + 2))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 15))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 2) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 2) % 75) * 864)) + (rc_outer_outer * 6)) + 3))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 16))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 2) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 2) % 75) * 864)) + (rc_outer_outer * 6)) + 4))];
    PaddedInput_shared[(((((int)threadIdx.x) * 18) + 17))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 2) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + ((((((int)threadIdx.x) * 3) + 2) % 75) * 864)) + (rc_outer_outer * 6)) + 5))];
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 864))] = placeholder[((((((((((int)threadIdx.x) * 3) + 144) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 69) * 864)) + (rc_outer_outer * 6)))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 865))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 144) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 69) * 864)) + (rc_outer_outer * 6)) + 1))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 866))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 144) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 69) * 864)) + (rc_outer_outer * 6)) + 2))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 867))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 144) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 69) * 864)) + (rc_outer_outer * 6)) + 3))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 868))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 144) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 69) * 864)) + (rc_outer_outer * 6)) + 4))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 869))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 144) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 69) * 864)) + (rc_outer_outer * 6)) + 5))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 870))] = placeholder[((((((((((int)threadIdx.x) * 3) + 145) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 70) * 864)) + (rc_outer_outer * 6)))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 871))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 145) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 70) * 864)) + (rc_outer_outer * 6)) + 1))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 872))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 145) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 70) * 864)) + (rc_outer_outer * 6)) + 2))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 873))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 145) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 70) * 864)) + (rc_outer_outer * 6)) + 3))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 874))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 145) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 70) * 864)) + (rc_outer_outer * 6)) + 4))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 875))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 145) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 70) * 864)) + (rc_outer_outer * 6)) + 5))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 876))] = placeholder[((((((((((int)threadIdx.x) * 3) + 146) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 71) * 864)) + (rc_outer_outer * 6)))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 877))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 146) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 71) * 864)) + (rc_outer_outer * 6)) + 1))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 878))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 146) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 71) * 864)) + (rc_outer_outer * 6)) + 2))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 879))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 146) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 71) * 864)) + (rc_outer_outer * 6)) + 3))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 880))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 146) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 71) * 864)) + (rc_outer_outer * 6)) + 4))];
    }
    if (((int)threadIdx.x) < 2) {
      PaddedInput_shared[(((((int)threadIdx.x) * 18) + 881))] = placeholder[(((((((((((int)threadIdx.x) * 3) + 146) / 75) * 194400) + ((((int)blockIdx.x) >> 1) * 64800)) + (((((int)threadIdx.x) * 3) + 71) * 864)) + (rc_outer_outer * 6)) + 5))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 768) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 48))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) + 48) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) + 96) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 144))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) + 144) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((rc_outer_outer * 768) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) + 240) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 288))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) + 288) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[(((((rc_outer_outer * 768) + (((((int)threadIdx.x) + 336) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) + 16)))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 6; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
        Conv2dOutput[((xx_outer_inner * 2))] = (Conv2dOutput[((xx_outer_inner * 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 100))] = (Conv2dOutput[(((xx_outer_inner * 2) + 100))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 10))] = (Conv2dOutput[(((xx_outer_inner * 2) + 10))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 90))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 110))] = (Conv2dOutput[(((xx_outer_inner * 2) + 110))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 90))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 20))] = (Conv2dOutput[(((xx_outer_inner * 2) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 180))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 120))] = (Conv2dOutput[(((xx_outer_inner * 2) + 120))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 30))] = (Conv2dOutput[(((xx_outer_inner * 2) + 30))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 270))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 130))] = (Conv2dOutput[(((xx_outer_inner * 2) + 130))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 270))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 40))] = (Conv2dOutput[(((xx_outer_inner * 2) + 40))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 360))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 140))] = (Conv2dOutput[(((xx_outer_inner * 2) + 140))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 50))] = (Conv2dOutput[(((xx_outer_inner * 2) + 50))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 450))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 150))] = (Conv2dOutput[(((xx_outer_inner * 2) + 150))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 450))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 60))] = (Conv2dOutput[(((xx_outer_inner * 2) + 60))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 540))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 160))] = (Conv2dOutput[(((xx_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 70))] = (Conv2dOutput[(((xx_outer_inner * 2) + 70))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 630))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 170))] = (Conv2dOutput[(((xx_outer_inner * 2) + 170))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 630))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 80))] = (Conv2dOutput[(((xx_outer_inner * 2) + 80))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 720))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 180))] = (Conv2dOutput[(((xx_outer_inner * 2) + 180))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 720))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 90))] = (Conv2dOutput[(((xx_outer_inner * 2) + 90))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 810))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 190))] = (Conv2dOutput[(((xx_outer_inner * 2) + 190))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 810))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 1))] = (Conv2dOutput[(((xx_outer_inner * 2) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 101))] = (Conv2dOutput[(((xx_outer_inner * 2) + 101))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 11))] = (Conv2dOutput[(((xx_outer_inner * 2) + 11))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 90))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 111))] = (Conv2dOutput[(((xx_outer_inner * 2) + 111))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 90))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 21))] = (Conv2dOutput[(((xx_outer_inner * 2) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 121))] = (Conv2dOutput[(((xx_outer_inner * 2) + 121))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 31))] = (Conv2dOutput[(((xx_outer_inner * 2) + 31))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 270))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 131))] = (Conv2dOutput[(((xx_outer_inner * 2) + 131))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 270))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 41))] = (Conv2dOutput[(((xx_outer_inner * 2) + 41))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 141))] = (Conv2dOutput[(((xx_outer_inner * 2) + 141))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 51))] = (Conv2dOutput[(((xx_outer_inner * 2) + 51))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 450))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 151))] = (Conv2dOutput[(((xx_outer_inner * 2) + 151))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 450))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 61))] = (Conv2dOutput[(((xx_outer_inner * 2) + 61))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 161))] = (Conv2dOutput[(((xx_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 71))] = (Conv2dOutput[(((xx_outer_inner * 2) + 71))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 630))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 171))] = (Conv2dOutput[(((xx_outer_inner * 2) + 171))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 630))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 81))] = (Conv2dOutput[(((xx_outer_inner * 2) + 81))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 720))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 181))] = (Conv2dOutput[(((xx_outer_inner * 2) + 181))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 720))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 91))] = (Conv2dOutput[(((xx_outer_inner * 2) + 91))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 810))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 191))] = (Conv2dOutput[(((xx_outer_inner * 2) + 191))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 30) + (xx_outer_inner * 6)) + rc_outer_inner) + 810))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
          T_relu[(((((((((ax0_inner * 28800) + ((((int)blockIdx.x) >> 1) * 9600)) + (ax1_inner * 1920)) + ((((int)threadIdx.x) >> 4) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 50) + (ax1_inner * 10)) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((((ax0_inner * 28800) + ((((int)blockIdx.x) >> 1) * 9600)) + (ax1_inner * 1920)) + ((((int)threadIdx.x) >> 4) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[((((((ax0_inner * 50) + (ax1_inner * 10)) + (ax2_inner * 2)) + ax3_inner) + 100))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
        }
      }
    }
  }
}


