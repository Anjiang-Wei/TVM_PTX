
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[768];
  __shared__ float placeholder_d_shared[768];
  __shared__ float placeholder_shared[48];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 3; ++b_c_outer_inner_init) {
    for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 16; ++i_c_outer_inner_init) {
      for (int b_c_inner_init = 0; b_c_inner_init < 8; ++b_c_inner_init) {
        for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
          compute_local[(((((b_c_outer_inner_init * 256) + (b_c_inner_init * 32)) + (i_c_outer_inner_init * 2)) + j_c_inner_init))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 384; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_d_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x)))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x)) >> 5) * 16384) + ((((int)blockIdx.x) >> 5) * 4096)) + ((((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x)) & 31) * 128)) + k_outer_outer))];
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_outer_outer1 < 24; ++ax0_ax1_fused_ax2_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_outer_outer1 * 2) + ((int)threadIdx.x)))] = placeholder1[(((((ax0_ax1_fused_ax2_fused_outer_outer1 * 8192) + ((((int)blockIdx.x) & 31) * 256)) + (((int)threadIdx.x) * 128)) + k_outer_outer))];
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 3; ++b_c_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 16; ++i_c_outer_inner) {
        for (int b_c_inner = 0; b_c_inner < 8; ++b_c_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            compute_local[(((((b_c_outer_inner * 256) + (b_c_inner * 32)) + (i_c_outer_inner * 2)) + j_c_inner))] = (compute_local[(((((b_c_outer_inner * 256) + (b_c_inner * 32)) + (i_c_outer_inner * 2)) + j_c_inner))] + (placeholder_d_shared[(((((b_c_outer_inner * 256) + (b_c_inner * 32)) + (((int)threadIdx.x) * 16)) + i_c_outer_inner))] * placeholder_shared[((((b_c_outer_inner * 16) + (b_c_inner * 2)) + j_c_inner))]));
          }
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 24; ++b_inner) {
    for (int i_inner = 0; i_inner < 16; ++i_inner) {
      for (int j_inner = 0; j_inner < 2; ++j_inner) {
        compute[(((((((b_inner * 8192) + ((((int)blockIdx.x) >> 5) * 2048)) + (((int)threadIdx.x) * 1024)) + (i_inner * 64)) + ((((int)blockIdx.x) & 31) * 2)) + j_inner))] = compute_local[((((b_inner * 32) + (i_inner * 2)) + j_inner))];
      }
    }
  }
}


