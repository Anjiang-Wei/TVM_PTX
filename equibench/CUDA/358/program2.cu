
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float placeholder_d_shared[512];
  __shared__ float placeholder_shared[256];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[((((int)threadIdx.x) * 16))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((int)threadIdx.x) & 7) * 1024)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 1))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((int)threadIdx.x) & 7) * 1024)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 2))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 1) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 1) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 3))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 1) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 1) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 4))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 2) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 2) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 5))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 2) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 2) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 6))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 3) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 3) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 7))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 3) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 3) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 8))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 4) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 4) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 9))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 4) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 4) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 10))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 5) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 5) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 11))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 5) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 5) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 12))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 6) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 6) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 13))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 6) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 6) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 14))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 7) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 7) & 63) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_d_shared[(((((int)threadIdx.x) * 16) + 15))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 65536) + ((((((int)threadIdx.x) * 8) + 7) >> 6) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 8) + 7) & 63) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[((((int)threadIdx.x) * 4))] = placeholder1[(((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)blockIdx.x) & 1) * 4096)) + ((((int)threadIdx.x) & 15) * 256)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder1[((((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 4) * 8192)) + ((((int)blockIdx.x) & 1) * 4096)) + ((((int)threadIdx.x) & 15) * 256)) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder1[(((((((((int)blockIdx.x) >> 2) * 32768) + ((((((int)threadIdx.x) * 2) + 1) >> 5) * 8192)) + ((((int)blockIdx.x) & 1) * 4096)) + ((((((int)threadIdx.x) * 2) + 1) & 31) * 128)) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder1[((((((((((int)blockIdx.x) >> 2) * 32768) + ((((((int)threadIdx.x) * 2) + 1) >> 5) * 8192)) + ((((int)blockIdx.x) & 1) * 4096)) + ((((((int)threadIdx.x) * 2) + 1) & 31) * 128)) + (k_outer_outer * 2)) + 1))];
    }
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)))] * placeholder_shared[((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 64))] * placeholder_shared[((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 64))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 64))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 64))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 65))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 65))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 65))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 8) * 128) + (((((int)threadIdx.x) & 255) >> 3) * 2)) + 65))] * placeholder_shared[(((((((int)threadIdx.x) >> 8) * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    compute[(((((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 8) * 8192)) + (((((int)blockIdx.x) & 3) >> 1) * 4096)) + (((((int)threadIdx.x) & 255) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner))] = compute_local[(j_inner)];
    compute[((((((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 8) * 8192)) + (((((int)blockIdx.x) & 3) >> 1) * 4096)) + (((((int)threadIdx.x) & 255) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 16))] = compute_local[((j_inner + 2))];
    compute[((((((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 8) * 8192)) + (((((int)blockIdx.x) & 3) >> 1) * 4096)) + (((((int)threadIdx.x) & 255) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 2048))] = compute_local[((j_inner + 4))];
    compute[((((((((((((int)blockIdx.x) >> 2) * 32768) + ((((int)threadIdx.x) >> 8) * 8192)) + (((((int)blockIdx.x) & 3) >> 1) * 4096)) + (((((int)threadIdx.x) & 255) >> 3) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 7) * 2)) + j_inner) + 2064))] = compute_local[((j_inner + 6))];
  }
}


