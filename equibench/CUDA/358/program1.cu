
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float placeholder_d_shared[128];
  __shared__ float placeholder_shared[64];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    ((float2*)(placeholder_d_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder + (((((((((int)blockIdx.x) >> 8) * 65536) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0];
    ((float2*)(placeholder_d_shared + (((((int)threadIdx.x) * 2) + 16))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 512))))[0];
    ((float2*)(placeholder_d_shared + (((((int)threadIdx.x) * 2) + 32))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 16384))))[0];
    ((float2*)(placeholder_d_shared + (((((((((int)threadIdx.x) * 2) + 48) >> 5) * 32) + (((((int)threadIdx.x) >> 1) + 4) * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + ((((((int)threadIdx.x) * 2) + 48) >> 5) * 16384)) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + (((((int)threadIdx.x) >> 1) + 4) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0];
    ((float2*)(placeholder_d_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 32768))))[0];
    ((float2*)(placeholder_d_shared + (((((((((int)threadIdx.x) * 2) + 80) >> 5) * 32) + (((((int)threadIdx.x) >> 1) + 4) * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + ((((((int)threadIdx.x) * 2) + 80) >> 5) * 16384)) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + (((((int)threadIdx.x) >> 1) + 4) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0];
    ((float2*)(placeholder_d_shared + (((((int)threadIdx.x) * 2) + 96))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)) + 49152))))[0];
    ((float2*)(placeholder_d_shared + (((((((((int)threadIdx.x) * 2) + 112) >> 5) * 32) + (((((int)threadIdx.x) >> 1) + 4) * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) >> 8) * 65536) + ((((((int)threadIdx.x) * 2) + 112) >> 5) * 16384)) + (((((int)blockIdx.x) & 255) >> 4) * 1024)) + (((((int)threadIdx.x) >> 1) + 4) * 128)) + (k_outer_outer * 4)) + ((((int)threadIdx.x) & 1) * 2)))))[0];
    ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + (((((((((int)blockIdx.x) >> 8) * 32768) + ((((int)threadIdx.x) >> 2) * 8192)) + ((((int)blockIdx.x) & 15) * 512)) + ((((int)threadIdx.x) & 3) * 128)) + (k_outer_outer * 4)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 32))))[0] = ((float4*)(placeholder1 + ((((((((((int)blockIdx.x) >> 8) * 32768) + ((((int)threadIdx.x) >> 2) * 8192)) + ((((int)blockIdx.x) & 15) * 512)) + ((((int)threadIdx.x) & 3) * 128)) + (k_outer_outer * 4)) + 16384))))[0];
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[((((int)threadIdx.x) * 8))] * placeholder_shared[(((((int)threadIdx.x) >> 2) * 16))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 64))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 32))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[((((int)threadIdx.x) * 8))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 4))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 64))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 36))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 4))] * placeholder_shared[(((((int)threadIdx.x) >> 2) * 16))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 68))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 32))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 4))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 68))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 36))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 1))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 65))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 33))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 5))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 65))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 37))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 1))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 69))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 33))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 5))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 69))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 37))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[((((int)threadIdx.x) * 8))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 8))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 64))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 40))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[((((int)threadIdx.x) * 8))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 12))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 64))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 44))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 8))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 68))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 40))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 12))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 68))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 44))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 9))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 65))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 41))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 13))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 65))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 45))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 9))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 69))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 41))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 13))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 69))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 45))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 2))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 66))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 34))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 6))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 66))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 38))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 2))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 70))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 34))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 6))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 70))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 38))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 3))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 67))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 35))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 7))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 67))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 39))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 3))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 71))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 35))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 7))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 71))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 39))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 10))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 66))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 42))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 14))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 66))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 46))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 10))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 70))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 42))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 14))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 70))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 46))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 11))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 67))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 43))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 15))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 67))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 47))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 11))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 71))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 43))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 15))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[(((((int)threadIdx.x) * 8) + 71))] * placeholder_shared[((((((int)threadIdx.x) >> 2) * 16) + 47))]));
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      compute[(((((((((((int)blockIdx.x) >> 8) * 32768) + ((((int)threadIdx.x) >> 2) * 8192)) + (((((int)blockIdx.x) & 255) >> 4) * 512)) + ((((int)threadIdx.x) & 3) * 128)) + (i_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + j_inner))] = compute_local[(((i_inner * 4) + j_inner))];
      compute[((((((((((((int)blockIdx.x) >> 8) * 32768) + ((((int)threadIdx.x) >> 2) * 8192)) + (((((int)blockIdx.x) & 255) >> 4) * 512)) + ((((int)threadIdx.x) & 3) * 128)) + (i_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + j_inner) + 16384))] = compute_local[((((i_inner * 4) + j_inner) + 8))];
    }
  }
}


