
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[896];
  __shared__ float PaddedInput_shared[3136];
  __shared__ float placeholder_shared[256];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 14; ++xx_inner_init) {
      Conv2dOutput[(((xx_inner_init * 16) + (ff_outer_inner_init * 4)))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 224))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 448))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 672))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 225))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 449))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 673))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 226))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 450))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 674))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 227))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 451))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 16) + (ff_outer_inner_init * 4)) + 675))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 68; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 224; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 14) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 14) + ((int)threadIdx.x)) >> 3) * 544) + (rc_outer_outer * 8)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 14) + ((int)threadIdx.x)) & 7)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 1024) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 14))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 14))];
    placeholder_shared[((((int)threadIdx.x) + 28))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 28) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 28) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 42))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 42) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 10)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 56) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 24) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 70))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 70) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 6)))];
    placeholder_shared[((((int)threadIdx.x) + 84))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 84) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 20) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 98))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 98) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 2)))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 112) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 126))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 126) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 30) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 140))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 140) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 12)))];
    placeholder_shared[((((int)threadIdx.x) + 154))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 154) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 26) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 168) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 8)))];
    placeholder_shared[((((int)threadIdx.x) + 182))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 182) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 22) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 196))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 196) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 4)))];
    placeholder_shared[((((int)threadIdx.x) + 210))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 210) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 18)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 238))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 238) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 14)))];
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[((((int)threadIdx.x) + 252))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 252) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 28)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int xx_inner = 0; xx_inner < 14; ++xx_inner) {
            Conv2dOutput[(((xx_inner * 16) + (ff_outer_inner * 4)))] = (Conv2dOutput[(((xx_inner * 16) + (ff_outer_inner * 4)))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 224))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 224))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 784))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 448))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 448))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 1568))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 672))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 672))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 2352))] * placeholder_shared[(((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 1))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 225))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 225))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 784))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 449))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 449))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 1568))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 673))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 673))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 2352))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 2))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 226))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 226))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 784))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 450))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 450))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 1568))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 674))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 674))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 2352))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 3))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 3))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 3))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 227))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 227))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 784))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 3))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 451))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 451))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 1568))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 3))]));
            Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 675))] = (Conv2dOutput[((((xx_inner * 16) + (ff_outer_inner * 4)) + 675))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 1) * 112) + (xx_inner * 8)) + (rc_outer_inner * 2)) + rc_inner) + 2352))] * placeholder_shared[((((((rc_outer_inner * 64) + (rc_inner * 32)) + ((((int)threadIdx.x) & 1) * 16)) + (ff_outer_inner * 4)) + 3))]));
          }
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
      T_relu[(((((((((int)threadIdx.x) >> 1) * 1792) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 16) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 1) * 1792) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 12544))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 224))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 1) * 1792) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 25088))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 448))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 1) * 1792) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner) + 37632))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 672))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 1) * 16)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


