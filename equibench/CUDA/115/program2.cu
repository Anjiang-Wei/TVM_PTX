
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[2048];
  __shared__ float placeholder_shared[2048];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[((((int)threadIdx.x) * 64))] = placeholder[(((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 1))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 2))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 2))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 3))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 3))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 4))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 4))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 5))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 5))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 6))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 6))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 7))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 7))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 8))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 8))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 9))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 9))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 10))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 10))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 11))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 11))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 12))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 12))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 13))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 13))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 14))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 14))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 15))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 15))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 16))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 16))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 17))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 17))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 18))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 18))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 19))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 19))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 20))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 20))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 21))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 21))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 22))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 22))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 23))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 23))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 24))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 24))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 25))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 25))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 26))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 26))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 27))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 27))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 28))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 28))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 29))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 29))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 30))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 30))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 31))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 1) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((int)threadIdx.x) & 1) * 256)) + (rc_outer_outer * 32)) + 31))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 32))] = placeholder[(((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 33))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 1))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 34))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 2))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 35))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 3))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 36))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 4))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 37))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 5))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 38))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 6))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 39))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 7))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 40))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 8))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 41))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 9))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 42))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 10))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 43))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 11))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 44))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 12))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 45))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 13))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 46))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 14))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 47))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 15))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 48))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 16))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 49))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 17))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 50))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 18))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 51))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 19))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 52))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 20))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 53))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 21))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 54))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 22))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 55))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 23))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 56))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 24))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 57))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 25))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 58))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 26))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 59))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 27))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 60))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 28))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 61))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 29))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 62))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 30))];
    }
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 64) + 63))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + ((((((int)threadIdx.x) * 2) + 1) & 3) * 128)) + (rc_outer_outer * 32)) + 31))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 1280))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 1536))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 1792))];
    placeholder_shared[((((int)threadIdx.x) + 1024))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 2048))];
    placeholder_shared[((((int)threadIdx.x) + 1152))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 2304))];
    placeholder_shared[((((int)threadIdx.x) + 1280))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 2560))];
    placeholder_shared[((((int)threadIdx.x) + 1408))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 2816))];
    placeholder_shared[((((int)threadIdx.x) + 1536))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 3072))];
    placeholder_shared[((((int)threadIdx.x) + 1664))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 3328))];
    placeholder_shared[((((int)threadIdx.x) + 1792))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 3584))];
    placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)) + 3840))];
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 128))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 128))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 64))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1024))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1024))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1088))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1088))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 32))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 96))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1056))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1056))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1120))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1120))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 65))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 65))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1025))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1025))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1089))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1089))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 33))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 33))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 97))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 97))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1057))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1057))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1121))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 64))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1121))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 96))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 2))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 2))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 66))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 66))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1026))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1026))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1090))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1090))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 34))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 34))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 98))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 98))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1058))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1058))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1122))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 128))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1122))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 160))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 3))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 3))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 67))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 67))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1027))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1027))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1091))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1091))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 35))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 35))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 99))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 99))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1059))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1059))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1123))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 192))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1123))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 224))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 4))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 4))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 68))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 68))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1028))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1028))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1092))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1092))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 36))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 36))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 100))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 100))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1060))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1060))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1124))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 256))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1124))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 288))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 5))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 5))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 69))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 69))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1029))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1029))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1093))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1093))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 37))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 37))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 101))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 101))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1061))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1061))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1125))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 320))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1125))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 352))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 6))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 6))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 70))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 70))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1030))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1030))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1094))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1094))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 38))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 38))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 102))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 102))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1062))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1062))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1126))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 384))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1126))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 416))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 7))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 7))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 71))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 71))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1031))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1031))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1095))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1095))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 39))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 39))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 103))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 103))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1063))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1063))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1127))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 448))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1127))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 480))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 8))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 8))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 72))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 72))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1032))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1032))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1096))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1096))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 40))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 40))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 104))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 104))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1064))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1064))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1128))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 512))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1128))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 544))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 9))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 9))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 73))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 73))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1033))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1033))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1097))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1097))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 41))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 41))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 105))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 105))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1065))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1065))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1129))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 576))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1129))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 608))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 10))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 10))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 74))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 74))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1034))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1034))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1098))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1098))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 42))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 42))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 106))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 106))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1066))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1066))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1130))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 640))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1130))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 672))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 11))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 11))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 75))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 75))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1035))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1035))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1099))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1099))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 43))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 43))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 107))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 107))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1067))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1067))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1131))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 704))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1131))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 736))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 12))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 12))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 76))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 76))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1036))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1036))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1100))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1100))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 44))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 44))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 108))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 108))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1068))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1068))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1132))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 768))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1132))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 800))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 13))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 13))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 77))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 77))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1037))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1037))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1101))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1101))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 45))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 45))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 109))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 109))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1069))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1069))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1133))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 832))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1133))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 864))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 14))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 14))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 78))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 78))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1038))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1038))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1102))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1102))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 46))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 46))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 110))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 110))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1070))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1070))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1134))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 896))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1134))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 928))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 15))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 15))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 79))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 79))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1039))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1039))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1103))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1103))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 47))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 47))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 111))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 111))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1071))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1071))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1135))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 960))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1135))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 992))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 16))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 16))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 80))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 80))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1040))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1040))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1104))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1104))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 48))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 48))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 112))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 112))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1072))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1072))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1136))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1024))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1136))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1056))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 17))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 17))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 81))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 81))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1041))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1041))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1105))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1105))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 49))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 49))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 113))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 113))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1073))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1073))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1137))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1088))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1137))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1120))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 18))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 18))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 82))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 82))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1042))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1042))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1106))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1106))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 50))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 50))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 114))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 114))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1074))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1074))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1138))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1152))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1138))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1184))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 19))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 19))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 83))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 83))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1043))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1043))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1107))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1107))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 51))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 51))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 115))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 115))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1075))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1075))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1139))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1216))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1139))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1248))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 20))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 20))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 84))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 84))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1044))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1044))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1108))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1108))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 52))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 52))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 116))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 116))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1076))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1076))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1140))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1280))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1140))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1312))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 21))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 21))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 85))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 85))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1045))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1045))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1109))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1109))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 53))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 53))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 117))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 117))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1077))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1077))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1141))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1344))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1141))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1376))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 22))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 22))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 86))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 86))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1046))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1046))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1110))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1110))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 54))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 54))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 118))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 118))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1078))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1078))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1142))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1408))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1142))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1440))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 23))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 23))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 87))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 87))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1047))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1047))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1111))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1111))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 55))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 55))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 119))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 119))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1079))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1079))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1143))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1472))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1143))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1504))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 24))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 24))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 88))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 88))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1048))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1048))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1112))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1112))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 56))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 56))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 120))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 120))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1080))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1080))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1144))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1536))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1144))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1568))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 25))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 25))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 89))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 89))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1049))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1049))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1113))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1113))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 57))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 57))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 121))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 121))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1081))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1081))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1145))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1600))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1145))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1632))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 26))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 26))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 90))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 90))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1050))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1050))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1114))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1114))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 58))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 58))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 122))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 122))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1082))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1082))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1146))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1664))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1146))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1696))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 27))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 27))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 91))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 91))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1051))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1051))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1115))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1115))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 59))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 59))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 123))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 123))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1083))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1083))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1147))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1728))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1147))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1760))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 28))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 28))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 92))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 92))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1052))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1052))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1116))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1116))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 60))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 60))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 124))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 124))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1084))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1084))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1148))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1792))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1148))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1824))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 29))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 29))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 93))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 93))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1053))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1053))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1117))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1117))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 61))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 61))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 125))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 125))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1085))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1085))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1149))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1856))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1149))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1888))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 30))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 30))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 94))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 94))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1054))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1054))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1118))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1118))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 62))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 62))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 126))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 126))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1086))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1086))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1150))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1920))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1150))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1952))]));
      Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 31))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 31))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 95))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 95))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1055))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1055))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1119))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 28))] = (Conv2dOutput[((ff_outer_inner + 28))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1119))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 2))] = (Conv2dOutput[((ff_outer_inner + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 63))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 6))] = (Conv2dOutput[((ff_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 63))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 10))] = (Conv2dOutput[((ff_outer_inner + 10))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 127))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 14))] = (Conv2dOutput[((ff_outer_inner + 14))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 127))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 18))] = (Conv2dOutput[((ff_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1087))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 22))] = (Conv2dOutput[((ff_outer_inner + 22))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1087))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
      Conv2dOutput[((ff_outer_inner + 26))] = (Conv2dOutput[((ff_outer_inner + 26))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1151))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 1984))]));
      Conv2dOutput[((ff_outer_inner + 30))] = (Conv2dOutput[((ff_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 128) + 1151))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 2016))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 4))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 256))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 8))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 288))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 12))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32768))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32800))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 20))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 33024))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 24))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 15) >> 1) * 512)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 33056))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 28))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
    }
  }
}


