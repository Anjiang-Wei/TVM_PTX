
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[112];
  __shared__ float PaddedInput_shared[8400];
  __shared__ float placeholder_shared[384];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 7; ++xx_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 14) + (xx_inner_init * 2)) + ff_outer_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 14) + (xx_inner_init * 2)) + ff_outer_inner_init) + 56))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 56; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 56) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 8400) {
        if (((int)threadIdx.x) < 150) {
          PaddedInput_shared[(((((int)threadIdx.x) * 56) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[(((((((((((int)threadIdx.x) * 56) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 2100) * 235200) + (((int)blockIdx.x) * 33600)) + (((((((int)threadIdx.x) * 56) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 2100) / 12) * 192)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 56) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 12)))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
      if (((((int)threadIdx.x) * 12) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 384) {
        if (((int)threadIdx.x) < 32) {
          placeholder_shared[(((((int)threadIdx.x) * 12) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[((((rc_outer_outer * 384) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
          for (int rc_inner = 0; rc_inner < 6; ++rc_inner) {
            for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
              Conv2dOutput[((((nn_outer_inner * 14) + (xx_inner * 2)) + ff_outer_inner))] = (Conv2dOutput[((((nn_outer_inner * 14) + (xx_inner * 2)) + ff_outer_inner))] + (PaddedInput_shared[((((((nn_outer_inner * 2100) + ((((int)threadIdx.x) >> 3) * 84)) + (xx_inner * 12)) + (rc_outer_inner * 6)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 192) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + ff_outer_inner))]));
              Conv2dOutput[(((((nn_outer_inner * 14) + (xx_inner * 2)) + ff_outer_inner) + 56))] = (Conv2dOutput[(((((nn_outer_inner * 14) + (xx_inner * 2)) + ff_outer_inner) + 56))] + (PaddedInput_shared[((((((nn_outer_inner * 2100) + ((((int)threadIdx.x) >> 3) * 84)) + (xx_inner * 12)) + (rc_outer_inner * 6)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 192) + (rc_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + ff_outer_inner) + 16))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((ax0_inner * 39200) + (((int)blockIdx.x) * 5600)) + ((((int)threadIdx.x) >> 3) * 224)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 14) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 7) * 2) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax0_inner * 39200) + (((int)blockIdx.x) * 5600)) + ((((int)threadIdx.x) >> 3) * 224)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 16))] = max((Conv2dOutput[(((((ax0_inner * 14) + (ax2_inner * 2)) + ax3_inner) + 56))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 16))]), 0.000000e+00f);
      }
    }
  }
}


