
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float compute[64];
  __shared__ float placeholder_d_shared[2];
  __shared__ float placeholder_shared[256];
  for (int j_outer_inner_init = 0; j_outer_inner_init < 8; ++j_outer_inner_init) {
    for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
      for (int j_inner_init = 0; j_inner_init < 4; ++j_inner_init) {
        compute[((((i_inner_init * 32) + (j_outer_inner_init * 4)) + j_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 2048; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) >> 1) * 4096) + (((int)threadIdx.x) * 2048)) + k_outer_outer))];
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.x) & 1) * 524288) + (ax0_ax1_fused_ax2_fused_outer_outer * 16384)) + (((int)threadIdx.x) * 2048)) + k_outer_outer))];
    }
    __syncthreads();
    for (int j_outer_inner = 0; j_outer_inner < 8; ++j_outer_inner) {
      for (int i_inner = 0; i_inner < 2; ++i_inner) {
        for (int j_inner = 0; j_inner < 4; ++j_inner) {
          compute[((((i_inner * 32) + (j_outer_inner * 4)) + j_inner))] = (compute[((((i_inner * 32) + (j_outer_inner * 4)) + j_inner))] + (placeholder_d_shared[(i_inner)] * placeholder_shared[((((((int)threadIdx.x) * 32) + (j_outer_inner * 4)) + j_inner))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 32; ++ax2_inner) {
      T_add[(((((((((int)blockIdx.x) >> 1) * 1024) + (ax1_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 32)) + ax2_inner))] = ((compute[(((ax1_inner * 32) + ax2_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 32)) + ax2_inner))]) + placeholder3[(((((((((int)blockIdx.x) >> 1) * 1024) + (ax1_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 32)) + ax2_inner))]);
    }
  }
}


