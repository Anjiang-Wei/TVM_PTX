
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[300];
  __shared__ float PaddedInput_shared[2400];
  __shared__ float placeholder_shared[64];
  for (int xx_c_outer_inner_init = 0; xx_c_outer_inner_init < 5; ++xx_c_outer_inner_init) {
    for (int yy_c_inner_init = 0; yy_c_inner_init < 5; ++yy_c_inner_init) {
      for (int ff_c_inner_init = 0; ff_c_inner_init < 2; ++ff_c_inner_init) {
        Conv2dOutput_local[((((yy_c_inner_init * 10) + (xx_c_outer_inner_init * 2)) + ff_c_inner_init))] = 0.000000e+00f;
        Conv2dOutput_local[(((((yy_c_inner_init * 10) + (xx_c_outer_inner_init * 2)) + ff_c_inner_init) + 50))] = 0.000000e+00f;
        Conv2dOutput_local[(((((yy_c_inner_init * 10) + (xx_c_outer_inner_init * 2)) + ff_c_inner_init) + 100))] = 0.000000e+00f;
        Conv2dOutput_local[(((((yy_c_inner_init * 10) + (xx_c_outer_inner_init * 2)) + ff_c_inner_init) + 150))] = 0.000000e+00f;
        Conv2dOutput_local[(((((yy_c_inner_init * 10) + (xx_c_outer_inner_init * 2)) + ff_c_inner_init) + 200))] = 0.000000e+00f;
        Conv2dOutput_local[(((((yy_c_inner_init * 10) + (xx_c_outer_inner_init * 2)) + ff_c_inner_init) + 250))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 19; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) < 1200) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)) < 2400) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) / 12) * 921600) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) / 20) * 15360)) + (((((int)blockIdx.x) % 12) >> 2) * 5120)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 20) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
        }
      }
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 256) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))];
    }
    __syncthreads();
    for (int xx_c_outer_inner = 0; xx_c_outer_inner < 5; ++xx_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int yy_c_inner = 0; yy_c_inner < 5; ++yy_c_inner) {
          for (int ff_c_inner = 0; ff_c_inner < 2; ++ff_c_inner) {
            Conv2dOutput_local[((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner))] = (Conv2dOutput_local[((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 6) * 200) + (yy_c_inner * 40)) + (((((int)threadIdx.x) & 63) >> 4) * 10)) + (xx_c_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + ff_c_inner))]));
            Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 50))] = (Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 50))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (yy_c_inner * 40)) + (((((int)threadIdx.x) & 63) >> 4) * 10)) + (xx_c_outer_inner * 2)) + rc_inner) + 400))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + ff_c_inner))]));
            Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 100))] = (Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 100))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (yy_c_inner * 40)) + (((((int)threadIdx.x) & 63) >> 4) * 10)) + (xx_c_outer_inner * 2)) + rc_inner) + 800))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + ff_c_inner))]));
            Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 150))] = (Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 150))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (yy_c_inner * 40)) + (((((int)threadIdx.x) & 63) >> 4) * 10)) + (xx_c_outer_inner * 2)) + rc_inner) + 1200))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + ff_c_inner))]));
            Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 200))] = (Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 200))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (yy_c_inner * 40)) + (((((int)threadIdx.x) & 63) >> 4) * 10)) + (xx_c_outer_inner * 2)) + rc_inner) + 1600))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + ff_c_inner))]));
            Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 250))] = (Conv2dOutput_local[(((((yy_c_inner * 10) + (xx_c_outer_inner * 2)) + ff_c_inner) + 250))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (yy_c_inner * 40)) + (((((int)threadIdx.x) & 63) >> 4) * 10)) + (xx_c_outer_inner * 2)) + rc_inner) + 2000))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + ff_c_inner))]));
          }
        }
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 5; ++yy_inner) {
    for (int xx_inner = 0; xx_inner < 5; ++xx_inner) {
      for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
        Conv2dOutput[(((((((((((((int)blockIdx.x) / 12) * 460800) + ((((int)threadIdx.x) >> 6) * 38400)) + (yy_inner * 7680)) + (((((int)blockIdx.x) % 12) >> 2) * 2560)) + (((((int)threadIdx.x) & 63) >> 4) * 640)) + (xx_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ff_inner))] = Conv2dOutput_local[((((yy_inner * 10) + (xx_inner * 2)) + ff_inner))];
        Conv2dOutput[((((((((((((((int)blockIdx.x) / 12) * 460800) + ((((int)threadIdx.x) >> 6) * 38400)) + (yy_inner * 7680)) + (((((int)blockIdx.x) % 12) >> 2) * 2560)) + (((((int)threadIdx.x) & 63) >> 4) * 640)) + (xx_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ff_inner) + 76800))] = Conv2dOutput_local[(((((yy_inner * 10) + (xx_inner * 2)) + ff_inner) + 50))];
        Conv2dOutput[((((((((((((((int)blockIdx.x) / 12) * 460800) + ((((int)threadIdx.x) >> 6) * 38400)) + (yy_inner * 7680)) + (((((int)blockIdx.x) % 12) >> 2) * 2560)) + (((((int)threadIdx.x) & 63) >> 4) * 640)) + (xx_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ff_inner) + 153600))] = Conv2dOutput_local[(((((yy_inner * 10) + (xx_inner * 2)) + ff_inner) + 100))];
        Conv2dOutput[((((((((((((((int)blockIdx.x) / 12) * 460800) + ((((int)threadIdx.x) >> 6) * 38400)) + (yy_inner * 7680)) + (((((int)blockIdx.x) % 12) >> 2) * 2560)) + (((((int)threadIdx.x) & 63) >> 4) * 640)) + (xx_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ff_inner) + 230400))] = Conv2dOutput_local[(((((yy_inner * 10) + (xx_inner * 2)) + ff_inner) + 150))];
        Conv2dOutput[((((((((((((((int)blockIdx.x) / 12) * 460800) + ((((int)threadIdx.x) >> 6) * 38400)) + (yy_inner * 7680)) + (((((int)blockIdx.x) % 12) >> 2) * 2560)) + (((((int)threadIdx.x) & 63) >> 4) * 640)) + (xx_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ff_inner) + 307200))] = Conv2dOutput_local[(((((yy_inner * 10) + (xx_inner * 2)) + ff_inner) + 200))];
        Conv2dOutput[((((((((((((((int)blockIdx.x) / 12) * 460800) + ((((int)threadIdx.x) >> 6) * 38400)) + (yy_inner * 7680)) + (((((int)blockIdx.x) % 12) >> 2) * 2560)) + (((((int)threadIdx.x) & 63) >> 4) * 640)) + (xx_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ff_inner) + 384000))] = Conv2dOutput_local[(((((yy_inner * 10) + (xx_inner * 2)) + ff_inner) + 250))];
      }
    }
  }
}


