
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[48];
  __shared__ float PaddedInput_shared[5376];
  __shared__ float placeholder_shared[6];
  for (int c_outer_inner_init = 0; c_outer_inner_init < 3; ++c_outer_inner_init) {
    DepthwiseConv2d[(c_outer_inner_init)] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 12))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 24))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 36))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 3))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 15))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 27))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 39))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 6))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 18))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 30))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 42))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 9))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 21))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 33))] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 45))] = 0.000000e+00f;
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)))] = (((((1 <= ((((((int)blockIdx.x) / 168) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 1344) / 48)) + di_outer_outer)) && (((((((int)blockIdx.x) / 168) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 1344) / 48)) + di_outer_outer) < 57)) && (1 <= (((((((int)blockIdx.x) % 168) / 24) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 48) / 6)) + dj_outer_outer))) && ((((((((int)blockIdx.x) % 168) / 24) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 48) / 6)) + dj_outer_outer) < 57)) ? placeholder[((((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) / 1344) * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 1344) / 48) * 8064)) + (di_outer_outer * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 48) / 6) * 144)) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) % 6)) - 8208))] : 0.000000e+00f);
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)blockIdx.x) / 168) * 28) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 1344) / 48)) + di_outer_outer)) && (((((((int)blockIdx.x) / 168) * 28) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 1344) / 48)) + di_outer_outer) < 57)) && (1 <= (((((((int)blockIdx.x) % 168) / 24) * 8) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 48) / 6)) + dj_outer_outer))) && ((((((((int)blockIdx.x) % 168) / 24) * 8) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 48) / 6)) + dj_outer_outer) < 57)) ? placeholder[(((((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) / 1344) * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 1344) / 48) * 8064)) + (di_outer_outer * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 48) / 6) * 144)) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 1) % 6)) - 8208))] : 0.000000e+00f);
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)blockIdx.x) / 168) * 28) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 1344) / 48)) + di_outer_outer)) && (((((((int)blockIdx.x) / 168) * 28) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 1344) / 48)) + di_outer_outer) < 57)) && (1 <= (((((((int)blockIdx.x) % 168) / 24) * 8) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 48) / 6)) + dj_outer_outer))) && ((((((((int)blockIdx.x) % 168) / 24) * 8) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 48) / 6)) + dj_outer_outer) < 57)) ? placeholder[(((((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) / 1344) * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 1344) / 48) * 8064)) + (di_outer_outer * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 48) / 6) * 144)) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 2) % 6)) - 8208))] : 0.000000e+00f);
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)blockIdx.x) / 168) * 28) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 1344) / 48)) + di_outer_outer)) && (((((((int)blockIdx.x) / 168) * 28) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 1344) / 48)) + di_outer_outer) < 57)) && (1 <= (((((((int)blockIdx.x) % 168) / 24) * 8) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 48) / 6)) + dj_outer_outer))) && ((((((((int)blockIdx.x) % 168) / 24) * 8) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 48) / 6)) + dj_outer_outer) < 57)) ? placeholder[(((((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) / 1344) * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 1344) / 48) * 8064)) + (di_outer_outer * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 48) / 6) * 144)) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 448) + (((int)threadIdx.x) * 4)) + 3) % 6)) - 8208))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 2) {
        placeholder_shared[((((int)threadIdx.x) * 3))] = placeholder1[(((((di_outer_outer * 432) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + (((int)threadIdx.x) * 3)))];
      }
      if (((int)threadIdx.x) < 2) {
        placeholder_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder1[((((((di_outer_outer * 432) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + (((int)threadIdx.x) * 3)) + 1))];
      }
      if (((int)threadIdx.x) < 2) {
        placeholder_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder1[((((((di_outer_outer * 432) + (dj_outer_outer * 144)) + ((((int)blockIdx.x) % 24) * 6)) + (((int)threadIdx.x) * 3)) + 2))];
      }
      __syncthreads();
      for (int c_outer_inner = 0; c_outer_inner < 3; ++c_outer_inner) {
        DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[(((((int)threadIdx.x) * 6) + c_outer_inner))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 12))] = (DepthwiseConv2d[((c_outer_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 3))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 24))] = (DepthwiseConv2d[((c_outer_inner + 24))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 672))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 36))] = (DepthwiseConv2d[((c_outer_inner + 36))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 675))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 3))] = (DepthwiseConv2d[((c_outer_inner + 3))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 1344))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 15))] = (DepthwiseConv2d[((c_outer_inner + 15))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 1347))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 27))] = (DepthwiseConv2d[((c_outer_inner + 27))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 2016))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 39))] = (DepthwiseConv2d[((c_outer_inner + 39))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 2019))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 6))] = (DepthwiseConv2d[((c_outer_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 2688))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 18))] = (DepthwiseConv2d[((c_outer_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 2691))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 30))] = (DepthwiseConv2d[((c_outer_inner + 30))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 3360))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 42))] = (DepthwiseConv2d[((c_outer_inner + 42))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 3363))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 9))] = (DepthwiseConv2d[((c_outer_inner + 9))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 4032))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 21))] = (DepthwiseConv2d[((c_outer_inner + 21))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 4035))] * placeholder_shared[((c_outer_inner + 3))]));
        DepthwiseConv2d[((c_outer_inner + 33))] = (DepthwiseConv2d[((c_outer_inner + 33))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 4704))] * placeholder_shared[(c_outer_inner)]));
        DepthwiseConv2d[((c_outer_inner + 45))] = (DepthwiseConv2d[((c_outer_inner + 45))] + (PaddedInput_shared[((((((int)threadIdx.x) * 6) + c_outer_inner) + 4707))] * placeholder_shared[((c_outer_inner + 3))]));
      }
    }
  }
  for (int i0_inner = 0; i0_inner < 4; ++i0_inner) {
    for (int i3_inner = 0; i3_inner < 3; ++i3_inner) {
      compute[((((((((i0_inner * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((int)threadIdx.x) >> 3) * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((int)threadIdx.x) & 7) * 144)) + ((((int)blockIdx.x) % 24) * 6)) + i3_inner))] = max(min((DepthwiseConv2d[(((i0_inner * 3) + i3_inner))] + placeholder2[((((((int)blockIdx.x) % 24) * 6) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((i0_inner * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((int)threadIdx.x) >> 3) * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((int)threadIdx.x) & 7) * 144)) + ((((int)blockIdx.x) % 24) * 6)) + i3_inner) + 3))] = max(min((DepthwiseConv2d[((((i0_inner * 3) + i3_inner) + 12))] + placeholder2[(((((((int)blockIdx.x) % 24) * 6) + i3_inner) + 3))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((i0_inner * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((int)threadIdx.x) >> 3) * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((int)threadIdx.x) & 7) * 144)) + ((((int)blockIdx.x) % 24) * 6)) + i3_inner) + 112896))] = max(min((DepthwiseConv2d[((((i0_inner * 3) + i3_inner) + 24))] + placeholder2[((((((int)blockIdx.x) % 24) * 6) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((i0_inner * 451584) + ((((int)blockIdx.x) / 168) * 225792)) + ((((int)threadIdx.x) >> 3) * 8064)) + (((((int)blockIdx.x) % 168) / 24) * 1152)) + ((((int)threadIdx.x) & 7) * 144)) + ((((int)blockIdx.x) % 24) * 6)) + i3_inner) + 112899))] = max(min((DepthwiseConv2d[((((i0_inner * 3) + i3_inner) + 36))] + placeholder2[(((((((int)blockIdx.x) % 24) * 6) + i3_inner) + 3))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


