
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 3) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 3) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) % 672))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 3) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) % 672))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[98];
  __shared__ float PaddedInput_shared[7072];
  __shared__ float placeholder_shared[160];
  for (int i_c_inner_init = 0; i_c_inner_init < 7; ++i_c_inner_init) {
    DepthwiseConv2d_local[((i_c_inner_init * 7))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 49))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 1))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 50))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 2))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 51))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 3))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 52))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 4))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 53))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 5))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 54))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 6))] = 0.000000e+00f;
    DepthwiseConv2d_local[(((i_c_inner_init * 7) + 55))] = 0.000000e+00f;
  }
  for (int dj_outer_outer = 0; dj_outer_outer < 5; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 442; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = (((((832 <= ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x))) && (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) < 6656)) && (2 <= (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 416) >> 5) + dj_outer_outer))) && ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 416) >> 5) + dj_outer_outer) < 16)) ? placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) / 416) * 9408) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 416) >> 5) * 672)) + (dj_outer_outer * 672)) + (((int)blockIdx.x) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) & 31)) - 20160))] : 0.000000e+00f);
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((dj_outer_outer * 672) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 16))] = placeholder1[(((((dj_outer_outer * 672) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 16))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[(((((dj_outer_outer * 672) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 3360))];
    placeholder_shared[((((int)threadIdx.x) + 48))] = placeholder1[(((((((((int)threadIdx.x) + 48) >> 5) * 3360) + (dj_outer_outer * 672)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((dj_outer_outer * 672) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 6720))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[(((((((((int)threadIdx.x) + 80) >> 5) * 3360) + (dj_outer_outer * 672)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((dj_outer_outer * 672) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 10080))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((((((int)threadIdx.x) + 112) >> 5) * 3360) + (dj_outer_outer * 672)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[(((((dj_outer_outer * 672) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 13440))];
    placeholder_shared[((((int)threadIdx.x) + 144))] = placeholder1[(((((((((int)threadIdx.x) + 144) >> 5) * 3360) + (dj_outer_outer * 672)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 16)))];
    __syncthreads();
    for (int di_outer_inner = 0; di_outer_inner < 5; ++di_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 7; ++i_c_inner) {
        DepthwiseConv2d_local[((i_c_inner * 7))] = (DepthwiseConv2d_local[((i_c_inner * 7))] + (PaddedInput_shared[((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 49))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 49))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 16))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 1))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 1))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 64))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 50))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 50))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 80))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 2))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 2))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 128))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 51))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 51))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 144))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 3))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 3))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 192))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 52))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 52))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 208))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 4))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 4))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 256))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 53))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 53))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 272))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 5))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 5))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 320))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 54))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 54))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 336))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 6))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 6))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 384))] * placeholder_shared[(((di_outer_inner * 32) + ((int)threadIdx.x)))]));
        DepthwiseConv2d_local[(((i_c_inner * 7) + 55))] = (DepthwiseConv2d_local[(((i_c_inner * 7) + 55))] + (PaddedInput_shared[(((((i_c_inner * 832) + (di_outer_inner * 416)) + ((int)threadIdx.x)) + 400))] * placeholder_shared[((((di_outer_inner * 32) + ((int)threadIdx.x)) + 16))]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 7; ++i_inner) {
    for (int j_inner = 0; j_inner < 7; ++j_inner) {
      DepthwiseConv2d[(((((i_inner * 4704) + (j_inner * 672)) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)))] = DepthwiseConv2d_local[(((i_inner * 7) + j_inner))];
      DepthwiseConv2d[((((((i_inner * 4704) + (j_inner * 672)) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)) + 16))] = DepthwiseConv2d_local[((((i_inner * 7) + j_inner) + 49))];
    }
  }
}


