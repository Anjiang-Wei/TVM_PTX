
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[448];
  __shared__ float PaddedInput_shared[3025];
  __shared__ float placeholder_shared[64];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 14; ++xx_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 56) + (xx_inner_init * 4)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 224))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 225))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 226))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 56) + (xx_inner_init * 4)) + 227))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3025) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) / 55) * 3584) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) % 55) * 64)) + rc_outer_outer))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3024) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 1))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 1) / 55) * 3584) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 1) % 55) * 64)) + rc_outer_outer))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3023) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 2))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 2) / 55) * 3584) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 2) % 55) * 64)) + rc_outer_outer))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3022) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 3))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 3) / 55) * 3584) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 3) % 55) * 64)) + rc_outer_outer))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3021) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 4))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 4) / 55) * 3584) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 4) % 55) * 64)) + rc_outer_outer))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3020) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 5))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 5) / 55) * 3584) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 5) % 55) * 64)) + rc_outer_outer))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) < 3019) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 112) + ((int)threadIdx.x)) < 433) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 6))] = placeholder[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 6) / 55) * 3584) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 784) + (((int)threadIdx.x) * 7)) + 6) % 55) * 64)) + rc_outer_outer))];
        }
      }
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 128) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
      for (int xx_inner = 0; xx_inner < 14; ++xx_inner) {
        Conv2dOutput[(((yy_outer_inner * 56) + (xx_inner * 4)))] = (Conv2dOutput[(((yy_outer_inner * 56) + (xx_inner * 4)))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 4))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 224))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)) + 28))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 4))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 225))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 2))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 226))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 226))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 3))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 3))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 3))]));
        Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 227))] = (Conv2dOutput[((((yy_outer_inner * 56) + (xx_inner * 4)) + 227))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 440) + (yy_outer_inner * 110)) + (xx_inner * 2)) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 3))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_add[((((((((((int)threadIdx.x) >> 4) * 14336) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))] = (Conv2dOutput[((((ax1_inner * 56) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 64) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))]);
        T_add[(((((((((((int)threadIdx.x) >> 4) * 14336) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner) + 1792))] = (Conv2dOutput[(((((ax1_inner * 56) + (ax2_inner * 4)) + ax3_inner) + 224))] + placeholder2[((((((int)blockIdx.x) * 64) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))]);
      }
    }
  }
}


