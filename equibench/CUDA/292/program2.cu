
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[32];
  __shared__ float placeholder_d_shared[16];
  __shared__ float placeholder_shared[1000];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(4)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(5)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(6)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  T_dense[(7)] = 0.000000e+00f;
  T_dense[(8)] = 0.000000e+00f;
  T_dense[(12)] = 0.000000e+00f;
  T_dense[(9)] = 0.000000e+00f;
  T_dense[(13)] = 0.000000e+00f;
  T_dense[(10)] = 0.000000e+00f;
  T_dense[(14)] = 0.000000e+00f;
  T_dense[(11)] = 0.000000e+00f;
  T_dense[(15)] = 0.000000e+00f;
  T_dense[(16)] = 0.000000e+00f;
  T_dense[(20)] = 0.000000e+00f;
  T_dense[(17)] = 0.000000e+00f;
  T_dense[(21)] = 0.000000e+00f;
  T_dense[(18)] = 0.000000e+00f;
  T_dense[(22)] = 0.000000e+00f;
  T_dense[(19)] = 0.000000e+00f;
  T_dense[(23)] = 0.000000e+00f;
  T_dense[(24)] = 0.000000e+00f;
  T_dense[(28)] = 0.000000e+00f;
  T_dense[(25)] = 0.000000e+00f;
  T_dense[(29)] = 0.000000e+00f;
  T_dense[(26)] = 0.000000e+00f;
  T_dense[(30)] = 0.000000e+00f;
  T_dense[(27)] = 0.000000e+00f;
  T_dense[(31)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 1024; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 16) {
      placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) >> 1) * 2048) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + ((((((int)blockIdx.x) * 1024000) + (((int)threadIdx.x) * 2048)) + (k_outer_outer * 2)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 250))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 1024000) + (((int)threadIdx.x) * 2048)) + (k_outer_outer * 2)) + 256000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 500))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 1024000) + (((int)threadIdx.x) * 2048)) + (k_outer_outer * 2)) + 512000))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 750))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 1024000) + (((int)threadIdx.x) * 2048)) + (k_outer_outer * 2)) + 768000))))[0];
    __syncthreads();
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(2)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(4)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(6)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(16)] = (T_dense[(16)] + (placeholder_d_shared[(8)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(20)] = (T_dense[(20)] + (placeholder_d_shared[(10)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(17)] = (T_dense[(17)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(21)] = (T_dense[(21)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(18)] = (T_dense[(18)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(22)] = (T_dense[(22)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(19)] = (T_dense[(19)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(23)] = (T_dense[(23)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(24)] = (T_dense[(24)] + (placeholder_d_shared[(12)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(28)] = (T_dense[(28)] + (placeholder_d_shared[(14)] * placeholder_shared[((((int)threadIdx.x) * 8))]));
    T_dense[(25)] = (T_dense[(25)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(29)] = (T_dense[(29)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 2))]));
    T_dense[(26)] = (T_dense[(26)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(30)] = (T_dense[(30)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 4))]));
    T_dense[(27)] = (T_dense[(27)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(31)] = (T_dense[(31)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 6))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(16)] = (T_dense[(16)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(20)] = (T_dense[(20)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(17)] = (T_dense[(17)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(21)] = (T_dense[(21)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(18)] = (T_dense[(18)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(22)] = (T_dense[(22)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(19)] = (T_dense[(19)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(23)] = (T_dense[(23)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(24)] = (T_dense[(24)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(28)] = (T_dense[(28)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 1))]));
    T_dense[(25)] = (T_dense[(25)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(29)] = (T_dense[(29)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 3))]));
    T_dense[(26)] = (T_dense[(26)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(30)] = (T_dense[(30)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 5))]));
    T_dense[(27)] = (T_dense[(27)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
    T_dense[(31)] = (T_dense[(31)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 8) + 7))]));
  }
  for (int ax0_inner = 0; ax0_inner < 8; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      T_add[(((((ax0_inner * 1000) + (((int)blockIdx.x) * 500)) + (((int)threadIdx.x) * 4)) + ax1_inner))] = (T_dense[(((ax0_inner * 4) + ax1_inner))] + placeholder2[((((((int)blockIdx.x) * 500) + (((int)threadIdx.x) * 4)) + ax1_inner))]);
    }
  }
}


