
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[1530];
  __shared__ float placeholder_shared[576];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 8; ++xx_inner_init) {
        Conv2dOutput[((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 256))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 257))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 2))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 258))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 3))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 128) + (yy_outer_inner_init * 32)) + (xx_inner_init * 4)) + 259))] = 0.000000e+00f;
      }
    }
  }
  for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 96; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) < 1530) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) / 45) * 3264) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 45) / 3) * 192)) + (rx_outer_outer * 192)) + (rc_outer_outer * 3)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 3)))];
        }
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 36; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)))] = placeholder1[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) / 192) * 110592) + (rx_outer_outer * 36864)) + (rc_outer_outer * 576)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) % 192) >> 6) * 192)) + (((int)blockIdx.x) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) & 63)))];
      }
      __syncthreads();
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
          for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
            for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
              for (int xx_inner = 0; xx_inner < 8; ++xx_inner) {
                Conv2dOutput[((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)))] = (Conv2dOutput[((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)))] + (PaddedInput_shared[((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 256))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 256))] + (PaddedInput_shared[(((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner) + 360))] * placeholder_shared[((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 1))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 1))] + (PaddedInput_shared[((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)) + 1))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 257))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 257))] + (PaddedInput_shared[(((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner) + 360))] * placeholder_shared[(((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)) + 1))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 2))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 2))] + (PaddedInput_shared[((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)) + 2))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 258))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 258))] + (PaddedInput_shared[(((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner) + 360))] * placeholder_shared[(((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)) + 2))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 3))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 3))] + (PaddedInput_shared[((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner))] * placeholder_shared[(((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)) + 3))]));
                Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 259))] = (Conv2dOutput[(((((nn_inner * 128) + (yy_outer_inner * 32)) + (xx_inner * 4)) + 259))] + (PaddedInput_shared[(((((((nn_inner * 765) + (yy_outer_inner * 90)) + (ry_inner * 45)) + (xx_inner * 6)) + rc_inner) + 360))] * placeholder_shared[(((((ry_inner * 192) + (rc_inner * 64)) + (((int)threadIdx.x) * 4)) + 3))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
          T_relu[(((((((ax0_inner * 12288) + (ax1_inner * 1536)) + (ax2_inner * 192)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 4)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 128) + (ax1_inner * 32)) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) * 4)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((ax0_inner * 12288) + (ax1_inner * 1536)) + (ax2_inner * 192)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 6144))] = max((Conv2dOutput[((((((ax0_inner * 128) + (ax1_inner * 32)) + (ax2_inner * 4)) + ax3_inner) + 256))] + placeholder2[((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) * 4)) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


