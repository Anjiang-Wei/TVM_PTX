
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[336];
  __shared__ float placeholder_shared[2048];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[((((int)threadIdx.x) * 12))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((int)threadIdx.x) >> 2) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + ((((((int)threadIdx.x) & 3) * 12) >> 4) * 64)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) * 12) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 1) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 1) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 1) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 2) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 2) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 2) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 3) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 3) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 3) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 4) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 4) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 4) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 5) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 5) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 5) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 6) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 6) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 6) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 7) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 7) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 7) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 8) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 8) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 8) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 9) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 9) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 9) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 10) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 10) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 10) & 15)))];
    }
    if (((int)threadIdx.x) < 28) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] = placeholder[((((((((((int)blockIdx.x) / 14) * 28672) + ((((((int)threadIdx.x) * 12) + 11) / 48) * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + (((((((int)threadIdx.x) * 12) + 11) % 48) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 12) + 11) & 15)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 2048) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 640))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 1024))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 1152))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1152))];
    placeholder_shared[((((int)threadIdx.x) + 1280))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1280))];
    placeholder_shared[((((int)threadIdx.x) + 1408))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1408))];
    placeholder_shared[((((int)threadIdx.x) + 1536))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1536))];
    placeholder_shared[((((int)threadIdx.x) + 1664))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1664))];
    placeholder_shared[((((int)threadIdx.x) + 1792))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1792))];
    placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1920))];
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 4))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 3))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 96))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 4))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 2))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 3))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 128))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 129))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 130))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 131))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 128))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 129))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 130))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 131))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 256))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 257))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 258))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 259))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 256))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 257))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 258))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 259))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 384))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 385))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 386))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 387))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 384))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 385))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 386))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 387))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 512))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 513))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 514))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 515))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 512))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 513))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 514))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 515))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 640))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 641))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 642))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 643))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 640))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 641))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 642))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 643))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 768))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 769))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 770))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 771))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 768))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 769))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 770))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 771))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 896))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 897))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 898))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 899))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 896))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 897))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 898))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 899))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1024))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1025))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1026))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1027))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 104))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1024))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 104))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1025))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 104))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1026))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 104))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1027))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1152))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1153))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1154))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1155))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 105))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1152))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 105))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1153))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 105))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1154))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 105))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1155))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1280))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1281))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1282))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1283))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 106))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1280))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 106))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1281))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 106))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1282))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 106))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1283))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1408))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1409))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1410))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1411))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 107))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1408))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 107))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1409))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 107))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1410))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 107))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1411))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1536))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1537))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1538))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1539))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1536))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1537))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1538))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1539))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1664))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1665))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1666))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1667))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1664))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1665))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1666))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1667))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1792))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1793))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1794))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1795))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 110))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1792))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 110))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1793))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 110))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1794))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 110))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1795))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1920))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1921))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1922))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1923))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 111))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1920))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 111))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1921))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 111))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1922))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 192) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + 111))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 4) + 1923))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_add[((((((((((int)blockIdx.x) / 14) * 14336) + ((((int)threadIdx.x) >> 6) * 7168)) + (ax1_inner * 3584)) + ((((int)blockIdx.x) % 14) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner))] = (Conv2dOutput[(((ax1_inner * 4) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 31) * 4) + ax3_inner))]);
    }
  }
}


