
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float compute[48];
  __shared__ float placeholder_d_shared[64];
  __shared__ float placeholder_shared[1536];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 2; ++i_outer_inner_init) {
    for (int j_outer_inner_init = 0; j_outer_inner_init < 4; ++j_outer_inner_init) {
      for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
        compute[((((i_outer_inner_init * 8) + (i_inner_init * 4)) + j_outer_inner_init))] = 0.000000e+00f;
        compute[(((((i_outer_inner_init * 8) + (i_inner_init * 4)) + j_outer_inner_init) + 16))] = 0.000000e+00f;
        compute[(((((i_outer_inner_init * 8) + (i_inner_init * 4)) + j_outer_inner_init) + 32))] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) >> 5) * 131072) + ((((int)blockIdx.x) >> 3) * 8192)) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 12; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 128) + ((int)threadIdx.x)))] = placeholder1[(((((((((int)blockIdx.x) & 7) * 393216) + (ax0_ax1_fused_ax2_fused_outer_outer * 32768)) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
        for (int j_outer_inner = 0; j_outer_inner < 4; ++j_outer_inner) {
          for (int k_inner = 0; k_inner < 2; ++k_inner) {
            for (int i_inner = 0; i_inner < 2; ++i_inner) {
              compute[((((i_outer_inner * 8) + (i_inner * 4)) + j_outer_inner))] = (compute[((((i_outer_inner * 8) + (i_inner * 4)) + j_outer_inner))] + (placeholder_d_shared[(((((((((int)threadIdx.x) >> 5) * 16) + (i_outer_inner * 8)) + (i_inner * 4)) + (k_outer_inner * 2)) + k_inner))] * placeholder_shared[((((((((int)threadIdx.x) & 31) * 16) + (j_outer_inner * 4)) + (k_outer_inner * 2)) + k_inner))]));
              compute[(((((i_outer_inner * 8) + (i_inner * 4)) + j_outer_inner) + 16))] = (compute[(((((i_outer_inner * 8) + (i_inner * 4)) + j_outer_inner) + 16))] + (placeholder_d_shared[(((((((((int)threadIdx.x) >> 5) * 16) + (i_outer_inner * 8)) + (i_inner * 4)) + (k_outer_inner * 2)) + k_inner))] * placeholder_shared[(((((((((int)threadIdx.x) & 31) * 16) + (j_outer_inner * 4)) + (k_outer_inner * 2)) + k_inner) + 512))]));
              compute[(((((i_outer_inner * 8) + (i_inner * 4)) + j_outer_inner) + 32))] = (compute[(((((i_outer_inner * 8) + (i_inner * 4)) + j_outer_inner) + 32))] + (placeholder_d_shared[(((((((((int)threadIdx.x) >> 5) * 16) + (i_outer_inner * 8)) + (i_inner * 4)) + (k_outer_inner * 2)) + k_inner))] * placeholder_shared[(((((((((int)threadIdx.x) & 31) * 16) + (j_outer_inner * 4)) + (k_outer_inner * 2)) + k_inner) + 1024))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      T_add[(((((((((((int)threadIdx.x) >> 6) * 393216) + ((((int)blockIdx.x) >> 3) * 24576)) + (((((int)threadIdx.x) & 63) >> 5) * 12288)) + (ax1_inner * 3072)) + ((((int)blockIdx.x) & 7) * 384)) + ((((int)threadIdx.x) & 31) * 4)) + ax2_inner))] = (compute[(((ax1_inner * 4) + ax2_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 384) + ((((int)threadIdx.x) & 31) * 4)) + ax2_inner))]);
      T_add[((((((((((((int)threadIdx.x) >> 6) * 393216) + ((((int)blockIdx.x) >> 3) * 24576)) + (((((int)threadIdx.x) & 63) >> 5) * 12288)) + (ax1_inner * 3072)) + ((((int)blockIdx.x) & 7) * 384)) + ((((int)threadIdx.x) & 31) * 4)) + ax2_inner) + 128))] = (compute[((((ax1_inner * 4) + ax2_inner) + 16))] + placeholder2[((((((((int)blockIdx.x) & 7) * 384) + ((((int)threadIdx.x) & 31) * 4)) + ax2_inner) + 128))]);
      T_add[((((((((((((int)threadIdx.x) >> 6) * 393216) + ((((int)blockIdx.x) >> 3) * 24576)) + (((((int)threadIdx.x) & 63) >> 5) * 12288)) + (ax1_inner * 3072)) + ((((int)blockIdx.x) & 7) * 384)) + ((((int)threadIdx.x) & 31) * 4)) + ax2_inner) + 256))] = (compute[((((ax1_inner * 4) + ax2_inner) + 32))] + placeholder2[((((((((int)blockIdx.x) & 7) * 384) + ((((int)threadIdx.x) & 31) * 4)) + ax2_inner) + 256))]);
    }
  }
}


