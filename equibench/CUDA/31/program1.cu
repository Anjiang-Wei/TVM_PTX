
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ Conv2dOutput, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((Conv2dOutput[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 184))]) * (max(min(((Conv2dOutput[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 184))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[128];
  __shared__ float PaddedInput_shared[256];
  __shared__ float placeholder_shared[92];
  Conv2dOutput_local[(0)] = 0.000000e+00f;
  Conv2dOutput_local[(32)] = 0.000000e+00f;
  Conv2dOutput_local[(64)] = 0.000000e+00f;
  Conv2dOutput_local[(96)] = 0.000000e+00f;
  Conv2dOutput_local[(1)] = 0.000000e+00f;
  Conv2dOutput_local[(33)] = 0.000000e+00f;
  Conv2dOutput_local[(65)] = 0.000000e+00f;
  Conv2dOutput_local[(97)] = 0.000000e+00f;
  Conv2dOutput_local[(8)] = 0.000000e+00f;
  Conv2dOutput_local[(40)] = 0.000000e+00f;
  Conv2dOutput_local[(72)] = 0.000000e+00f;
  Conv2dOutput_local[(104)] = 0.000000e+00f;
  Conv2dOutput_local[(9)] = 0.000000e+00f;
  Conv2dOutput_local[(41)] = 0.000000e+00f;
  Conv2dOutput_local[(73)] = 0.000000e+00f;
  Conv2dOutput_local[(105)] = 0.000000e+00f;
  Conv2dOutput_local[(16)] = 0.000000e+00f;
  Conv2dOutput_local[(48)] = 0.000000e+00f;
  Conv2dOutput_local[(80)] = 0.000000e+00f;
  Conv2dOutput_local[(112)] = 0.000000e+00f;
  Conv2dOutput_local[(17)] = 0.000000e+00f;
  Conv2dOutput_local[(49)] = 0.000000e+00f;
  Conv2dOutput_local[(81)] = 0.000000e+00f;
  Conv2dOutput_local[(113)] = 0.000000e+00f;
  Conv2dOutput_local[(24)] = 0.000000e+00f;
  Conv2dOutput_local[(56)] = 0.000000e+00f;
  Conv2dOutput_local[(88)] = 0.000000e+00f;
  Conv2dOutput_local[(120)] = 0.000000e+00f;
  Conv2dOutput_local[(25)] = 0.000000e+00f;
  Conv2dOutput_local[(57)] = 0.000000e+00f;
  Conv2dOutput_local[(89)] = 0.000000e+00f;
  Conv2dOutput_local[(121)] = 0.000000e+00f;
  Conv2dOutput_local[(2)] = 0.000000e+00f;
  Conv2dOutput_local[(34)] = 0.000000e+00f;
  Conv2dOutput_local[(66)] = 0.000000e+00f;
  Conv2dOutput_local[(98)] = 0.000000e+00f;
  Conv2dOutput_local[(3)] = 0.000000e+00f;
  Conv2dOutput_local[(35)] = 0.000000e+00f;
  Conv2dOutput_local[(67)] = 0.000000e+00f;
  Conv2dOutput_local[(99)] = 0.000000e+00f;
  Conv2dOutput_local[(10)] = 0.000000e+00f;
  Conv2dOutput_local[(42)] = 0.000000e+00f;
  Conv2dOutput_local[(74)] = 0.000000e+00f;
  Conv2dOutput_local[(106)] = 0.000000e+00f;
  Conv2dOutput_local[(11)] = 0.000000e+00f;
  Conv2dOutput_local[(43)] = 0.000000e+00f;
  Conv2dOutput_local[(75)] = 0.000000e+00f;
  Conv2dOutput_local[(107)] = 0.000000e+00f;
  Conv2dOutput_local[(18)] = 0.000000e+00f;
  Conv2dOutput_local[(50)] = 0.000000e+00f;
  Conv2dOutput_local[(82)] = 0.000000e+00f;
  Conv2dOutput_local[(114)] = 0.000000e+00f;
  Conv2dOutput_local[(19)] = 0.000000e+00f;
  Conv2dOutput_local[(51)] = 0.000000e+00f;
  Conv2dOutput_local[(83)] = 0.000000e+00f;
  Conv2dOutput_local[(115)] = 0.000000e+00f;
  Conv2dOutput_local[(26)] = 0.000000e+00f;
  Conv2dOutput_local[(58)] = 0.000000e+00f;
  Conv2dOutput_local[(90)] = 0.000000e+00f;
  Conv2dOutput_local[(122)] = 0.000000e+00f;
  Conv2dOutput_local[(27)] = 0.000000e+00f;
  Conv2dOutput_local[(59)] = 0.000000e+00f;
  Conv2dOutput_local[(91)] = 0.000000e+00f;
  Conv2dOutput_local[(123)] = 0.000000e+00f;
  Conv2dOutput_local[(4)] = 0.000000e+00f;
  Conv2dOutput_local[(36)] = 0.000000e+00f;
  Conv2dOutput_local[(68)] = 0.000000e+00f;
  Conv2dOutput_local[(100)] = 0.000000e+00f;
  Conv2dOutput_local[(5)] = 0.000000e+00f;
  Conv2dOutput_local[(37)] = 0.000000e+00f;
  Conv2dOutput_local[(69)] = 0.000000e+00f;
  Conv2dOutput_local[(101)] = 0.000000e+00f;
  Conv2dOutput_local[(12)] = 0.000000e+00f;
  Conv2dOutput_local[(44)] = 0.000000e+00f;
  Conv2dOutput_local[(76)] = 0.000000e+00f;
  Conv2dOutput_local[(108)] = 0.000000e+00f;
  Conv2dOutput_local[(13)] = 0.000000e+00f;
  Conv2dOutput_local[(45)] = 0.000000e+00f;
  Conv2dOutput_local[(77)] = 0.000000e+00f;
  Conv2dOutput_local[(109)] = 0.000000e+00f;
  Conv2dOutput_local[(20)] = 0.000000e+00f;
  Conv2dOutput_local[(52)] = 0.000000e+00f;
  Conv2dOutput_local[(84)] = 0.000000e+00f;
  Conv2dOutput_local[(116)] = 0.000000e+00f;
  Conv2dOutput_local[(21)] = 0.000000e+00f;
  Conv2dOutput_local[(53)] = 0.000000e+00f;
  Conv2dOutput_local[(85)] = 0.000000e+00f;
  Conv2dOutput_local[(117)] = 0.000000e+00f;
  Conv2dOutput_local[(28)] = 0.000000e+00f;
  Conv2dOutput_local[(60)] = 0.000000e+00f;
  Conv2dOutput_local[(92)] = 0.000000e+00f;
  Conv2dOutput_local[(124)] = 0.000000e+00f;
  Conv2dOutput_local[(29)] = 0.000000e+00f;
  Conv2dOutput_local[(61)] = 0.000000e+00f;
  Conv2dOutput_local[(93)] = 0.000000e+00f;
  Conv2dOutput_local[(125)] = 0.000000e+00f;
  Conv2dOutput_local[(6)] = 0.000000e+00f;
  Conv2dOutput_local[(38)] = 0.000000e+00f;
  Conv2dOutput_local[(70)] = 0.000000e+00f;
  Conv2dOutput_local[(102)] = 0.000000e+00f;
  Conv2dOutput_local[(7)] = 0.000000e+00f;
  Conv2dOutput_local[(39)] = 0.000000e+00f;
  Conv2dOutput_local[(71)] = 0.000000e+00f;
  Conv2dOutput_local[(103)] = 0.000000e+00f;
  Conv2dOutput_local[(14)] = 0.000000e+00f;
  Conv2dOutput_local[(46)] = 0.000000e+00f;
  Conv2dOutput_local[(78)] = 0.000000e+00f;
  Conv2dOutput_local[(110)] = 0.000000e+00f;
  Conv2dOutput_local[(15)] = 0.000000e+00f;
  Conv2dOutput_local[(47)] = 0.000000e+00f;
  Conv2dOutput_local[(79)] = 0.000000e+00f;
  Conv2dOutput_local[(111)] = 0.000000e+00f;
  Conv2dOutput_local[(22)] = 0.000000e+00f;
  Conv2dOutput_local[(54)] = 0.000000e+00f;
  Conv2dOutput_local[(86)] = 0.000000e+00f;
  Conv2dOutput_local[(118)] = 0.000000e+00f;
  Conv2dOutput_local[(23)] = 0.000000e+00f;
  Conv2dOutput_local[(55)] = 0.000000e+00f;
  Conv2dOutput_local[(87)] = 0.000000e+00f;
  Conv2dOutput_local[(119)] = 0.000000e+00f;
  Conv2dOutput_local[(30)] = 0.000000e+00f;
  Conv2dOutput_local[(62)] = 0.000000e+00f;
  Conv2dOutput_local[(94)] = 0.000000e+00f;
  Conv2dOutput_local[(126)] = 0.000000e+00f;
  Conv2dOutput_local[(31)] = 0.000000e+00f;
  Conv2dOutput_local[(63)] = 0.000000e+00f;
  Conv2dOutput_local[(95)] = 0.000000e+00f;
  Conv2dOutput_local[(127)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 80; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      PaddedInput_shared[((((int)threadIdx.x) * 4))] = placeholder[((((((((int)threadIdx.x) >> 4) * 20480) + ((((int)blockIdx.x) >> 1) * 5120)) + ((((int)threadIdx.x) & 15) * 320)) + rc_outer_outer))];
    }
    if (((int)threadIdx.x) < 64) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder[((((((((((int)threadIdx.x) * 4) + 1) >> 6) * 20480) + ((((int)blockIdx.x) >> 1) * 5120)) + ((((((int)threadIdx.x) * 4) + 1) & 63) * 80)) + rc_outer_outer))];
    }
    if (((int)threadIdx.x) < 64) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder[((((((((((int)threadIdx.x) * 4) + 2) >> 6) * 20480) + ((((int)blockIdx.x) >> 1) * 5120)) + ((((((int)threadIdx.x) * 4) + 2) & 63) * 80)) + rc_outer_outer))];
    }
    if (((int)threadIdx.x) < 64) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder[((((((((((int)threadIdx.x) * 4) + 3) >> 6) * 20480) + ((((int)blockIdx.x) >> 1) * 5120)) + ((((((int)threadIdx.x) * 4) + 3) & 63) * 80)) + rc_outer_outer))];
    }
    if (((int)threadIdx.x) < 46) {
      placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[((((rc_outer_outer * 184) + ((((int)blockIdx.x) & 1) * 92)) + (((int)threadIdx.x) * 2)))];
    }
    if (((int)threadIdx.x) < 46) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((rc_outer_outer * 184) + ((((int)blockIdx.x) & 1) * 92)) + (((int)threadIdx.x) * 2)) + 1))];
    }
    __syncthreads();
    Conv2dOutput_local[(0)] = (Conv2dOutput_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(32)] = (Conv2dOutput_local[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(64)] = (Conv2dOutput_local[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 8))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(96)] = (Conv2dOutput_local[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 12))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(1)] = (Conv2dOutput_local[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(33)] = (Conv2dOutput_local[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(65)] = (Conv2dOutput_local[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(97)] = (Conv2dOutput_local[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(8)] = (Conv2dOutput_local[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 64))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(40)] = (Conv2dOutput_local[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 68))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(72)] = (Conv2dOutput_local[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 72))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(104)] = (Conv2dOutput_local[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 76))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(9)] = (Conv2dOutput_local[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(41)] = (Conv2dOutput_local[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(73)] = (Conv2dOutput_local[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 72))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(105)] = (Conv2dOutput_local[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 76))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(16)] = (Conv2dOutput_local[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 128))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(48)] = (Conv2dOutput_local[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 132))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(80)] = (Conv2dOutput_local[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 136))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(112)] = (Conv2dOutput_local[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 140))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(17)] = (Conv2dOutput_local[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(49)] = (Conv2dOutput_local[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(81)] = (Conv2dOutput_local[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 136))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(113)] = (Conv2dOutput_local[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 140))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(24)] = (Conv2dOutput_local[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 192))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(56)] = (Conv2dOutput_local[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 196))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(88)] = (Conv2dOutput_local[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 200))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(120)] = (Conv2dOutput_local[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 204))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(25)] = (Conv2dOutput_local[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(57)] = (Conv2dOutput_local[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(89)] = (Conv2dOutput_local[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 200))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(121)] = (Conv2dOutput_local[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 204))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(2)] = (Conv2dOutput_local[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(34)] = (Conv2dOutput_local[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(66)] = (Conv2dOutput_local[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(98)] = (Conv2dOutput_local[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(3)] = (Conv2dOutput_local[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(35)] = (Conv2dOutput_local[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(67)] = (Conv2dOutput_local[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(99)] = (Conv2dOutput_local[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(10)] = (Conv2dOutput_local[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(42)] = (Conv2dOutput_local[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(74)] = (Conv2dOutput_local[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 72))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(106)] = (Conv2dOutput_local[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 76))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(11)] = (Conv2dOutput_local[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(43)] = (Conv2dOutput_local[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(75)] = (Conv2dOutput_local[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 72))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(107)] = (Conv2dOutput_local[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 76))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(18)] = (Conv2dOutput_local[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(50)] = (Conv2dOutput_local[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(82)] = (Conv2dOutput_local[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 136))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(114)] = (Conv2dOutput_local[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 140))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(19)] = (Conv2dOutput_local[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(51)] = (Conv2dOutput_local[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(83)] = (Conv2dOutput_local[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 136))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(115)] = (Conv2dOutput_local[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 140))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(26)] = (Conv2dOutput_local[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(58)] = (Conv2dOutput_local[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(90)] = (Conv2dOutput_local[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 200))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(122)] = (Conv2dOutput_local[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 204))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(27)] = (Conv2dOutput_local[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(59)] = (Conv2dOutput_local[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(91)] = (Conv2dOutput_local[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 200))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(123)] = (Conv2dOutput_local[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 204))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(4)] = (Conv2dOutput_local[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 16))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(36)] = (Conv2dOutput_local[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 20))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(68)] = (Conv2dOutput_local[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 24))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(100)] = (Conv2dOutput_local[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 28))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(5)] = (Conv2dOutput_local[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(37)] = (Conv2dOutput_local[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(69)] = (Conv2dOutput_local[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(101)] = (Conv2dOutput_local[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(12)] = (Conv2dOutput_local[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 80))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(44)] = (Conv2dOutput_local[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 84))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(76)] = (Conv2dOutput_local[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 88))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(108)] = (Conv2dOutput_local[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 92))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(13)] = (Conv2dOutput_local[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 80))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(45)] = (Conv2dOutput_local[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 84))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(77)] = (Conv2dOutput_local[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 88))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(109)] = (Conv2dOutput_local[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 92))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(20)] = (Conv2dOutput_local[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 144))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(52)] = (Conv2dOutput_local[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 148))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(84)] = (Conv2dOutput_local[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 152))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(116)] = (Conv2dOutput_local[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 156))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(21)] = (Conv2dOutput_local[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 144))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(53)] = (Conv2dOutput_local[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 148))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(85)] = (Conv2dOutput_local[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 152))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(117)] = (Conv2dOutput_local[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 156))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(28)] = (Conv2dOutput_local[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 208))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(60)] = (Conv2dOutput_local[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 212))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(92)] = (Conv2dOutput_local[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 216))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(124)] = (Conv2dOutput_local[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 220))] * placeholder_shared[(((((int)threadIdx.x) % 23) * 4))]));
    Conv2dOutput_local[(29)] = (Conv2dOutput_local[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 208))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(61)] = (Conv2dOutput_local[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 212))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(93)] = (Conv2dOutput_local[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 216))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(125)] = (Conv2dOutput_local[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 220))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 1))]));
    Conv2dOutput_local[(6)] = (Conv2dOutput_local[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(38)] = (Conv2dOutput_local[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(70)] = (Conv2dOutput_local[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(102)] = (Conv2dOutput_local[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(7)] = (Conv2dOutput_local[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(39)] = (Conv2dOutput_local[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(71)] = (Conv2dOutput_local[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(103)] = (Conv2dOutput_local[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(14)] = (Conv2dOutput_local[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 80))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(46)] = (Conv2dOutput_local[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 84))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(78)] = (Conv2dOutput_local[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 88))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(110)] = (Conv2dOutput_local[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 92))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(15)] = (Conv2dOutput_local[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 80))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(47)] = (Conv2dOutput_local[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 84))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(79)] = (Conv2dOutput_local[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 88))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(111)] = (Conv2dOutput_local[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 92))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(22)] = (Conv2dOutput_local[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 144))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(54)] = (Conv2dOutput_local[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 148))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(86)] = (Conv2dOutput_local[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 152))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(118)] = (Conv2dOutput_local[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 156))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(23)] = (Conv2dOutput_local[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 144))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(55)] = (Conv2dOutput_local[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 148))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(87)] = (Conv2dOutput_local[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 152))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(119)] = (Conv2dOutput_local[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 156))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(30)] = (Conv2dOutput_local[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 208))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(62)] = (Conv2dOutput_local[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 212))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(94)] = (Conv2dOutput_local[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 216))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(126)] = (Conv2dOutput_local[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 220))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 2))]));
    Conv2dOutput_local[(31)] = (Conv2dOutput_local[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 208))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(63)] = (Conv2dOutput_local[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 212))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(95)] = (Conv2dOutput_local[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 216))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
    Conv2dOutput_local[(127)] = (Conv2dOutput_local[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 92) * 32) + ((((int)threadIdx.x) % 92) / 23)) + 220))] * placeholder_shared[((((((int)threadIdx.x) % 23) * 4) + 3))]));
  }
  for (int nn_inner = 0; nn_inner < 4; ++nn_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
        Conv2dOutput[(((((((((nn_inner * 47104) + ((((int)blockIdx.x) >> 1) * 11776)) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + (((((int)threadIdx.x) % 92) / 23) * 184)) + ((((int)blockIdx.x) & 1) * 92)) + ((((int)threadIdx.x) % 23) * 4)) + ff_inner))] = Conv2dOutput_local[((((nn_inner * 8) + (yy_inner * 4)) + ff_inner))];
        Conv2dOutput[((((((((((nn_inner * 47104) + ((((int)blockIdx.x) >> 1) * 11776)) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + (((((int)threadIdx.x) % 92) / 23) * 184)) + ((((int)blockIdx.x) & 1) * 92)) + ((((int)threadIdx.x) % 23) * 4)) + ff_inner) + 736))] = Conv2dOutput_local[(((((nn_inner * 8) + (yy_inner * 4)) + ff_inner) + 32))];
        Conv2dOutput[((((((((((nn_inner * 47104) + ((((int)blockIdx.x) >> 1) * 11776)) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + (((((int)threadIdx.x) % 92) / 23) * 184)) + ((((int)blockIdx.x) & 1) * 92)) + ((((int)threadIdx.x) % 23) * 4)) + ff_inner) + 1472))] = Conv2dOutput_local[(((((nn_inner * 8) + (yy_inner * 4)) + ff_inner) + 64))];
        Conv2dOutput[((((((((((nn_inner * 47104) + ((((int)blockIdx.x) >> 1) * 11776)) + ((((int)threadIdx.x) / 92) * 5888)) + (yy_inner * 2944)) + (((((int)threadIdx.x) % 92) / 23) * 184)) + ((((int)blockIdx.x) & 1) * 92)) + ((((int)threadIdx.x) % 23) * 4)) + ff_inner) + 2208))] = Conv2dOutput_local[(((((nn_inner * 8) + (yy_inner * 4)) + ff_inner) + 96))];
      }
    }
  }
}


