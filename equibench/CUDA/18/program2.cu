
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[256];
  __shared__ float PaddedInput_shared[10240];
  __shared__ float placeholder_shared[288];
  for (int b_outer_inner_init = 0; b_outer_inner_init < 2; ++b_outer_inner_init) {
    for (int c_outer_inner_init = 0; c_outer_inner_init < 4; ++c_outer_inner_init) {
      for (int j_inner_init = 0; j_inner_init < 4; ++j_inner_init) {
        for (int c_inner_init = 0; c_inner_init < 2; ++c_inner_init) {
          DepthwiseConv2d[(((((b_outer_inner_init * 32) + (j_inner_init * 8)) + (c_outer_inner_init * 2)) + c_inner_init))] = 0.000000e+00f;
          DepthwiseConv2d[((((((b_outer_inner_init * 32) + (j_inner_init * 8)) + (c_outer_inner_init * 2)) + c_inner_init) + 64))] = 0.000000e+00f;
          DepthwiseConv2d[((((((b_outer_inner_init * 32) + (j_inner_init * 8)) + (c_outer_inner_init * 2)) + c_inner_init) + 128))] = 0.000000e+00f;
          DepthwiseConv2d[((((((b_outer_inner_init * 32) + (j_inner_init * 8)) + (c_outer_inner_init * 2)) + c_inner_init) + 192))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 427; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) < 10240) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)))] = (((((1 <= (((((int)blockIdx.x) / 15) * 6) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) % 2560) / 320))) && ((((((int)blockIdx.x) / 15) * 6) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) % 2560) / 320)) < 121)) && (1 <= (((((int)blockIdx.x) % 15) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) % 320) >> 5)))) && ((((((int)blockIdx.x) % 15) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) % 320) >> 5)) < 121)) ? placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) / 2560) * 460800) + ((((int)blockIdx.x) / 15) * 23040)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) % 2560) / 320) * 3840)) + ((((int)blockIdx.x) % 15) * 256)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24) + ((int)threadIdx.x)) % 320)) - 3872))] : 0.000000e+00f);
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    ((float4*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 96) + (((int)threadIdx.x) * 4)))))[0] = ((float4*)(placeholder1 + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 96) + (((int)threadIdx.x) * 4)))))[0];
  }
  __syncthreads();
  for (int dj_outer_inner = 0; dj_outer_inner < 3; ++dj_outer_inner) {
    for (int b_outer_inner = 0; b_outer_inner < 2; ++b_outer_inner) {
      for (int c_outer_inner = 0; c_outer_inner < 4; ++c_outer_inner) {
        for (int di_inner = 0; di_inner < 3; ++di_inner) {
          for (int j_inner = 0; j_inner < 4; ++j_inner) {
            for (int c_inner = 0; c_inner < 2; ++c_inner) {
              DepthwiseConv2d[(((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner))] = (DepthwiseConv2d[(((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner))] + (PaddedInput_shared[((((((((((b_outer_inner * 2560) + ((((int)threadIdx.x) >> 2) * 320)) + (di_inner * 320)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (j_inner * 32)) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner))] * placeholder_shared[((((((di_inner * 96) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner))]));
              DepthwiseConv2d[((((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner) + 64))] = (DepthwiseConv2d[((((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner) + 64))] + (PaddedInput_shared[(((((((((((b_outer_inner * 2560) + ((((int)threadIdx.x) >> 2) * 320)) + (di_inner * 320)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (j_inner * 32)) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner) + 16))] * placeholder_shared[(((((((di_inner * 96) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner) + 16))]));
              DepthwiseConv2d[((((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner) + 128))] = (DepthwiseConv2d[((((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner) + 128))] + (PaddedInput_shared[(((((((((((b_outer_inner * 2560) + ((((int)threadIdx.x) >> 2) * 320)) + (di_inner * 320)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (j_inner * 32)) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner) + 5120))] * placeholder_shared[((((((di_inner * 96) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner))]));
              DepthwiseConv2d[((((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner) + 192))] = (DepthwiseConv2d[((((((b_outer_inner * 32) + (j_inner * 8)) + (c_outer_inner * 2)) + c_inner) + 192))] + (PaddedInput_shared[(((((((((((b_outer_inner * 2560) + ((((int)threadIdx.x) >> 2) * 320)) + (di_inner * 320)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (j_inner * 32)) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner) + 5136))] * placeholder_shared[(((((((di_inner * 96) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (c_outer_inner * 2)) + c_inner) + 16))]));
            }
          }
        }
      }
    }
  }
  for (int i0_inner = 0; i0_inner < 2; ++i0_inner) {
    for (int i2_inner = 0; i2_inner < 4; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 8; ++i3_inner) {
        compute[(((((((((i0_inner * 460800) + ((((int)blockIdx.x) / 15) * 23040)) + ((((int)threadIdx.x) >> 2) * 3840)) + ((((int)blockIdx.x) % 15) * 256)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (i2_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + i3_inner))] = max(min((DepthwiseConv2d[((((i0_inner * 32) + (i2_inner * 8)) + i3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 8) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
        compute[((((((((((i0_inner * 460800) + ((((int)blockIdx.x) / 15) * 23040)) + ((((int)threadIdx.x) >> 2) * 3840)) + ((((int)blockIdx.x) % 15) * 256)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (i2_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + i3_inner) + 16))] = max(min((DepthwiseConv2d[(((((i0_inner * 32) + (i2_inner * 8)) + i3_inner) + 64))] + placeholder2[(((((((int)threadIdx.x) & 1) * 8) + i3_inner) + 16))]), 6.000000e+00f), 0.000000e+00f);
        compute[((((((((((i0_inner * 460800) + ((((int)blockIdx.x) / 15) * 23040)) + ((((int)threadIdx.x) >> 2) * 3840)) + ((((int)blockIdx.x) % 15) * 256)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (i2_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + i3_inner) + 921600))] = max(min((DepthwiseConv2d[(((((i0_inner * 32) + (i2_inner * 8)) + i3_inner) + 128))] + placeholder2[((((((int)threadIdx.x) & 1) * 8) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
        compute[((((((((((i0_inner * 460800) + ((((int)blockIdx.x) / 15) * 23040)) + ((((int)threadIdx.x) >> 2) * 3840)) + ((((int)blockIdx.x) % 15) * 256)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (i2_inner * 32)) + ((((int)threadIdx.x) & 1) * 8)) + i3_inner) + 921616))] = max(min((DepthwiseConv2d[(((((i0_inner * 32) + (i2_inner * 8)) + i3_inner) + 192))] + placeholder2[(((((((int)threadIdx.x) & 1) * 8) + i3_inner) + 16))]), 6.000000e+00f), 0.000000e+00f);
      }
    }
  }
}


