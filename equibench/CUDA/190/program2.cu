
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[128];
  __shared__ float PaddedInput_shared[5616];
  __shared__ float placeholder_shared[672];
  for (int dd_outer_inner_init = 0; dd_outer_inner_init < 4; ++dd_outer_inner_init) {
    for (int dd_inner_init = 0; dd_inner_init < 2; ++dd_inner_init) {
      Conv3dOutput[(((dd_outer_inner_init * 16) + (dd_inner_init * 8)))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 64))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 1))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 65))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 2))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 66))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 3))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 67))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 4))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 68))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 5))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 69))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 6))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 70))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 7))] = 0.000000e+00f;
      Conv3dOutput[((((dd_outer_inner_init * 16) + (dd_inner_init * 8)) + 71))] = 0.000000e+00f;
    }
  }
  for (int rd_outer_outer = 0; rd_outer_outer < 3; ++rd_outer_outer) {
    for (int rw_outer_outer = 0; rw_outer_outer < 7; ++rw_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 59; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) < 1872) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 96) + ((int)threadIdx.x)) < 5616) {
            PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 96) + ((int)threadIdx.x)))] = (((((((1 <= (((((((int)blockIdx.x) % 432) / 144) * 48) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) / 39)) + rd_outer_outer)) && ((((((((int)blockIdx.x) % 432) / 144) * 48) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) / 39)) + rd_outer_outer) < 145)) && (3 <= ((((((int)blockIdx.x) % 144) >> 3) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) % 39) / 3)))) && (((((((int)blockIdx.x) % 144) >> 3) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) % 39) / 3)) < 147)) && (3 <= (((((((int)blockIdx.x) & 7) >> 1) * 4) + rw_outer_outer) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) % 3)))) && ((((((((int)blockIdx.x) & 7) >> 1) * 4) + rw_outer_outer) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) % 3)) < 19)) ? placeholder[((((((((((((((int)blockIdx.x) / 144) * 331776) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) / 39) * 6912)) + (rd_outer_outer * 6912)) + (((((int)blockIdx.x) % 144) >> 3) * 384)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) % 39) / 3) * 48)) + (((((int)blockIdx.x) & 7) >> 1) * 12)) + (rw_outer_outer * 3)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) / 3)) % 3) * 3)) + (((int)threadIdx.x) % 3)) - 7065))] : 0.000000e+00f);
          }
        }
      }
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)))];
      placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 1344))];
      placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 2688))];
      placeholder_shared[((((int)threadIdx.x) + 288))] = placeholder1[(((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 4032))];
      placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[(((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 5376))];
      placeholder_shared[((((int)threadIdx.x) + 480))] = placeholder1[(((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 6720))];
      placeholder_shared[((((int)threadIdx.x) + 576))] = placeholder1[(((((((rd_outer_outer * 9408) + (rw_outer_outer * 192)) + ((((int)threadIdx.x) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 8064))];
      __syncthreads();
      for (int rh_outer_inner = 0; rh_outer_inner < 7; ++rh_outer_inner) {
        for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
          for (int dd_outer_inner = 0; dd_outer_inner < 4; ++dd_outer_inner) {
            for (int dd_inner = 0; dd_inner < 2; ++dd_inner) {
              Conv3dOutput[(((dd_outer_inner * 16) + (dd_inner * 8)))] = (Conv3dOutput[(((dd_outer_inner * 16) + (dd_inner * 8)))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 64))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 64))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2808))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 1))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 6))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 65))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 65))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2814))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 2))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 18))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 66))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 66))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2826))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 3))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 24))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 67))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 67))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2832))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 4))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 4))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 36))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 68))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 68))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2844))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 5))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 5))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 42))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 69))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 69))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2850))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 6))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 6))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 54))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 70))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 70))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2862))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 7))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 7))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 60))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
              Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 71))] = (Conv3dOutput[((((dd_outer_inner * 16) + (dd_inner * 8)) + 71))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 936) + (dd_outer_inner * 234)) + (dd_inner * 117)) + (rh_outer_inner * 9)) + rc_outer_inner) + 2868))] * placeholder_shared[((((rh_outer_inner * 96) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 31)))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((((((((int)blockIdx.x) / 144) * 1769472) + ((((int)threadIdx.x) >> 5) * 294912)) + (ax1_inner * 36864)) + (((((int)blockIdx.x) % 144) >> 3) * 2048)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 7) >> 1) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)))] = max(((Conv3dOutput[((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner))] * placeholder2[((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 31)))]) + placeholder3[((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 31)))]), 0.000000e+00f);
        T_relu[((((((((((((((int)blockIdx.x) / 144) * 1769472) + ((((int)threadIdx.x) >> 5) * 294912)) + (ax1_inner * 36864)) + (((((int)blockIdx.x) % 144) >> 3) * 2048)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 7) >> 1) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) & 31)) + 884736))] = max(((Conv3dOutput[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 64))] * placeholder2[((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 31)))]) + placeholder3[((((((int)blockIdx.x) & 1) * 32) + (((int)threadIdx.x) & 31)))]), 0.000000e+00f);
      }
    }
  }
}


