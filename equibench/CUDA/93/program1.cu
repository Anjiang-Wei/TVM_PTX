
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ Conv2dOutput, float* __restrict__ placeholder) {
  T_multiply[(((int)blockIdx.x))] = ((Conv2dOutput[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 960))]) * (max(min(((Conv2dOutput[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 960))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[42];
  __shared__ float PaddedInput_shared[35];
  __shared__ float placeholder_shared[4800];
  for (int yy_c_outer_inner_init = 0; yy_c_outer_inner_init < 7; ++yy_c_outer_inner_init) {
    Conv2dOutput_local[((yy_c_outer_inner_init * 6))] = 0.000000e+00f;
    Conv2dOutput_local[(((yy_c_outer_inner_init * 6) + 1))] = 0.000000e+00f;
    Conv2dOutput_local[(((yy_c_outer_inner_init * 6) + 2))] = 0.000000e+00f;
    Conv2dOutput_local[(((yy_c_outer_inner_init * 6) + 3))] = 0.000000e+00f;
    Conv2dOutput_local[(((yy_c_outer_inner_init * 6) + 4))] = 0.000000e+00f;
    Conv2dOutput_local[(((yy_c_outer_inner_init * 6) + 5))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[((((int)threadIdx.x) * 7))] = placeholder[(((((((((int)threadIdx.x) * 7) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + ((((int)threadIdx.x) * 7) % 5)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 1))] = placeholder[((((((((((int)threadIdx.x) * 7) + 1) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + (((((int)threadIdx.x) * 7) + 1) % 5)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 2))] = placeholder[((((((((((int)threadIdx.x) * 7) + 2) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + (((((int)threadIdx.x) * 7) + 2) % 5)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 3))] = placeholder[((((((((((int)threadIdx.x) * 7) + 3) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + (((((int)threadIdx.x) * 7) + 3) % 5)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 4))] = placeholder[((((((((((int)threadIdx.x) * 7) + 4) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + (((((int)threadIdx.x) * 7) + 4) % 5)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 5))] = placeholder[((((((((((int)threadIdx.x) * 7) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + ((((int)threadIdx.x) * 7) % 5)) + 1120))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 6))] = placeholder[((((((((((int)threadIdx.x) * 7) + 6) / 5) * 1120) + (((int)blockIdx.x) * 160)) + (rc_outer_outer * 5)) + (((((int)threadIdx.x) * 7) + 1) % 5)))];
    }
    ((float3*)(placeholder_shared + ((((int)threadIdx.x) * 3))))[0] = ((float3*)(placeholder1 + (((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 480))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 480))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 960))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 960))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 1440))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 1440))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 1920))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 1920))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 2400))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 2400))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 2880))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 2880))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 3360))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 3360))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 3840))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 3840))))[0];
    ((float3*)(placeholder_shared + (((((int)threadIdx.x) * 3) + 4320))))[0] = ((float3*)(placeholder1 + ((((rc_outer_outer * 4800) + (((int)threadIdx.x) * 3)) + 4320))))[0];
    __syncthreads();
    for (int yy_c_outer_inner = 0; yy_c_outer_inner < 7; ++yy_c_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 3; ++ff_c_outer_inner) {
        Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] = (Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] + (PaddedInput_shared[((yy_c_outer_inner * 5))] * placeholder_shared[(((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)))]));
        Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] = (Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] + (PaddedInput_shared[((yy_c_outer_inner * 5))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 1))]));
        Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] = (Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 1))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 960))]));
        Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] = (Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 1))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 961))]));
        Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] = (Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 2))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 1920))]));
        Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] = (Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 2))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 1921))]));
        Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] = (Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 3))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 2880))]));
        Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] = (Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 3))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 2881))]));
        Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] = (Conv2dOutput_local[(((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 4))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 3840))]));
        Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] = (Conv2dOutput_local[((((yy_c_outer_inner * 6) + (ff_c_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((yy_c_outer_inner * 5) + 4))] * placeholder_shared[((((((int)threadIdx.x) * 6) + (ff_c_outer_inner * 2)) + 3841))]));
      }
    }
  }
  for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
    for (int ff_inner = 0; ff_inner < 6; ++ff_inner) {
      Conv2dOutput[(((((yy_inner * 6720) + (((int)blockIdx.x) * 960)) + (((int)threadIdx.x) * 6)) + ff_inner))] = Conv2dOutput_local[(((yy_inner * 6) + ff_inner))];
    }
  }
}


