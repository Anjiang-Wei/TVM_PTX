
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[384];
  __shared__ float placeholder_shared[96];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 4; ++xx_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 32))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 64))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 96))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 33))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 65))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 8) + (xx_outer_inner_init * 2)) + 97))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 48; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + (((((int)threadIdx.x) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 32) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 8) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 64) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 16) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + (((((int)threadIdx.x) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 18432))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 128) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 8) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 160))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 160) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 16) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + (((((int)threadIdx.x) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 36864))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 224) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 8) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 256) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 16) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 288))] = placeholder[(((((((((((int)blockIdx.x) >> 2) * 73728) + ((((int)threadIdx.x) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + (((((int)threadIdx.x) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3)) + 55296))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 320) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 8) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 2) % 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 352))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 73728) + (((((int)threadIdx.x) + 352) / 24) * 4608)) + ((((int)blockIdx.x) & 3) * 1152)) + ((((((int)threadIdx.x) + 16) % 24) / 3) * 144)) + (rc_outer_outer * 3)) + ((((int)threadIdx.x) + 1) % 3)))];
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[((((int)threadIdx.x) * 16))] = placeholder1[(((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 1))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 1))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 2))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 2))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 3))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 3))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 4))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 4))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 5))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 5))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 6))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 6))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 7))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 7))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 8))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 8))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 9))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 9))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 10))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 10))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 11))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 11))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 12))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 12))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 13))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 13))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 14))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 14))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 15))] = placeholder1[((((rc_outer_outer * 96) + (((int)threadIdx.x) * 16)) + 15))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
          Conv2dOutput[(((yy_outer_inner * 8) + (xx_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 8) + (xx_outer_inner * 2)))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 32))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 32))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner) + 12))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 64))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 64))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner) + 192))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 96))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 96))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner) + 204))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 33))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 33))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner) + 12))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 65))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 65))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner) + 192))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 97))] = (Conv2dOutput[((((yy_outer_inner * 8) + (xx_outer_inner * 2)) + 97))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 4) * 96) + (yy_outer_inner * 24)) + (xx_outer_inner * 3)) + rc_outer_inner) + 204))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[(((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) & 3) * 256)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = (Conv2dOutput[((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 15) * 2) + ax3_inner))]);
        T_add[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) & 3) * 256)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 128))] = (Conv2dOutput[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 32))] + placeholder2[((((((int)threadIdx.x) & 15) * 2) + ax3_inner))]);
        T_add[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) & 3) * 256)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 8192))] = (Conv2dOutput[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 64))] + placeholder2[((((((int)threadIdx.x) & 15) * 2) + ax3_inner))]);
        T_add[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) & 3) * 256)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 8320))] = (Conv2dOutput[(((((ax1_inner * 8) + (ax2_inner * 2)) + ax3_inner) + 96))] + placeholder2[((((((int)threadIdx.x) & 15) * 2) + ax3_inner))]);
      }
    }
  }
}


