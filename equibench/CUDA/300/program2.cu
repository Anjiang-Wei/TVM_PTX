
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[24];
  __shared__ float PaddedInput_shared[1920];
  __shared__ float placeholder_shared[2304];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 12))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 12) + 11))] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 96; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + ((int)threadIdx.x)))] = placeholder[((((((int)blockIdx.x) * 1920) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20)) + ((int)threadIdx.x)))];
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 116; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 2304) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)))] = placeholder1[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 48; ++rc_outer_inner) {
    for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 6; ++ff_outer_inner) {
        Conv2dOutput[(((xx_outer_inner * 12) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((xx_outer_inner * 12) + (ff_outer_inner * 2)))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 192) + (xx_outer_inner * 96)) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 48) + ((((int)threadIdx.x) & 1) * 12)) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((xx_outer_inner * 12) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((xx_outer_inner * 12) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 192) + (xx_outer_inner * 96)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((rc_outer_inner * 48) + ((((int)threadIdx.x) & 1) * 12)) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 12) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((xx_outer_inner * 12) + (ff_outer_inner * 2)))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 192) + (xx_outer_inner * 96)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((rc_outer_inner * 48) + ((((int)threadIdx.x) & 1) * 12)) + (ff_outer_inner * 2)) + 24))]));
        Conv2dOutput[((((xx_outer_inner * 12) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((xx_outer_inner * 12) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 192) + (xx_outer_inner * 96)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((rc_outer_inner * 48) + ((((int)threadIdx.x) & 1) * 12)) + (ff_outer_inner * 2)) + 25))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 12; ++ax3_inner) {
      T_add[((((((((int)blockIdx.x) * 480) + ((((int)threadIdx.x) >> 1) * 48)) + (ax2_inner * 24)) + ((((int)threadIdx.x) & 1) * 12)) + ax3_inner))] = (Conv2dOutput[(((ax2_inner * 12) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 12) + ax3_inner))]);
    }
  }
}


