
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[128];
  __shared__ float placeholder_d_shared[1024];
  __shared__ float placeholder_shared[1024];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 8; ++i_c_inner_init) {
      compute_local[(((i_c_inner_init * 4) + j_c_outer_inner_init))] = 0.000000e+00f;
      compute_local[((((i_c_inner_init * 4) + j_c_outer_inner_init) + 32))] = 0.000000e+00f;
      compute_local[((((i_c_inner_init * 4) + j_c_outer_inner_init) + 64))] = 0.000000e+00f;
      compute_local[((((i_c_inner_init * 4) + j_c_outer_inner_init) + 96))] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_d_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_outer_outer * 131072) + ((((int)threadIdx.x) >> 5) * 65536)) + ((((int)blockIdx.x) >> 2) * 4096)) + (((((int)threadIdx.x) & 31) >> 1) * 256)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_outer_outer1 < 16; ++ax0_ax1_fused_ax2_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_outer_outer1 * 64) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_outer_outer1 * 32768) + ((((int)threadIdx.x) >> 5) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + (((((int)threadIdx.x) & 31) >> 1) * 256)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 8; ++i_c_inner) {
          compute_local[(((i_c_inner * 4) + j_c_outer_inner))] = (compute_local[(((i_c_inner * 4) + j_c_outer_inner))] + (placeholder_d_shared[(((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 2)) + k_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) * 8) + (j_c_outer_inner * 2)) + k_outer_inner))]));
          compute_local[((((i_c_inner * 4) + j_c_outer_inner) + 32))] = (compute_local[((((i_c_inner * 4) + j_c_outer_inner) + 32))] + (placeholder_d_shared[((((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 2)) + k_outer_inner) + 16))] * placeholder_shared[((((((int)threadIdx.x) * 8) + (j_c_outer_inner * 2)) + k_outer_inner))]));
          compute_local[((((i_c_inner * 4) + j_c_outer_inner) + 64))] = (compute_local[((((i_c_inner * 4) + j_c_outer_inner) + 64))] + (placeholder_d_shared[((((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 2)) + k_outer_inner) + 512))] * placeholder_shared[(((((((int)threadIdx.x) * 8) + (j_c_outer_inner * 2)) + k_outer_inner) + 512))]));
          compute_local[((((i_c_inner * 4) + j_c_outer_inner) + 96))] = (compute_local[((((i_c_inner * 4) + j_c_outer_inner) + 96))] + (placeholder_d_shared[((((((((int)threadIdx.x) >> 2) * 32) + (i_c_inner * 2)) + k_outer_inner) + 528))] * placeholder_shared[(((((((int)threadIdx.x) * 8) + (j_c_outer_inner * 2)) + k_outer_inner) + 512))]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      compute[((((((((((int)threadIdx.x) >> 2) * 16384) + ((((int)blockIdx.x) >> 2) * 1024)) + (i_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner))] = compute_local[(((i_inner * 4) + j_inner))];
      compute[(((((((((((int)threadIdx.x) >> 2) * 16384) + ((((int)blockIdx.x) >> 2) * 1024)) + (i_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 512))] = compute_local[((((i_inner * 4) + j_inner) + 32))];
      compute[(((((((((((int)threadIdx.x) >> 2) * 16384) + ((((int)blockIdx.x) >> 2) * 1024)) + (i_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 262144))] = compute_local[((((i_inner * 4) + j_inner) + 64))];
      compute[(((((((((((int)threadIdx.x) >> 2) * 16384) + ((((int)blockIdx.x) >> 2) * 1024)) + (i_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + j_inner) + 262656))] = compute_local[((((i_inner * 4) + j_inner) + 96))];
    }
  }
}


