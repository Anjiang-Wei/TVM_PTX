
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float Conv2dOutput[384];
  __shared__ float PaddedInput_shared[8192];
  __shared__ float placeholder_shared[1536];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 32; ++ff_outer_inner_init) {
      Conv2dOutput[(((xx_outer_inner_init * 96) + (ff_outer_inner_init * 3)))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 96) + (ff_outer_inner_init * 3)) + 192))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 96) + (ff_outer_inner_init * 3)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 96) + (ff_outer_inner_init * 3)) + 193))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 96) + (ff_outer_inner_init * 3)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 96) + (ff_outer_inner_init * 3)) + 194))] = 0.000000e+00f;
    }
  }
  PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)))];
  PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 128))];
  PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 2048))];
  PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 384) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 4096))];
  PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 640) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 6144))];
  PaddedInput_shared[((((int)threadIdx.x) + 896))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 896) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 8192))];
  PaddedInput_shared[((((int)threadIdx.x) + 1152))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 1152) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 10240))];
  PaddedInput_shared[((((int)threadIdx.x) + 1408))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 1408) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 12288))];
  PaddedInput_shared[((((int)threadIdx.x) + 1664))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 1664) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 1792))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 14336))];
  PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 1920) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 16384))];
  PaddedInput_shared[((((int)threadIdx.x) + 2176))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 2176) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 2304))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 18432))];
  PaddedInput_shared[((((int)threadIdx.x) + 2432))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 2432) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 20480))];
  PaddedInput_shared[((((int)threadIdx.x) + 2688))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 2688) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 2816))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 22528))];
  PaddedInput_shared[((((int)threadIdx.x) + 2944))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 2944) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 3072))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 24576))];
  PaddedInput_shared[((((int)threadIdx.x) + 3200))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 3200) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 3328))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 26624))];
  PaddedInput_shared[((((int)threadIdx.x) + 3456))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 3456) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 3584))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 28672))];
  PaddedInput_shared[((((int)threadIdx.x) + 3712))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 3712) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 3840))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 30720))];
  PaddedInput_shared[((((int)threadIdx.x) + 3968))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 3968) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 4096))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 32768))];
  PaddedInput_shared[((((int)threadIdx.x) + 4224))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 4224) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 4352))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 34816))];
  PaddedInput_shared[((((int)threadIdx.x) + 4480))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 4480) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 4608))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 36864))];
  PaddedInput_shared[((((int)threadIdx.x) + 4736))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 4736) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 4864))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 38912))];
  PaddedInput_shared[((((int)threadIdx.x) + 4992))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 4992) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 5120))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 40960))];
  PaddedInput_shared[((((int)threadIdx.x) + 5248))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 5248) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 5376))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 43008))];
  PaddedInput_shared[((((int)threadIdx.x) + 5504))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 5504) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 5632))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 45056))];
  PaddedInput_shared[((((int)threadIdx.x) + 5760))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 5760) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 5888))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 47104))];
  PaddedInput_shared[((((int)threadIdx.x) + 6016))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 6016) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 6144))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 49152))];
  PaddedInput_shared[((((int)threadIdx.x) + 6272))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 6272) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 6400))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 51200))];
  PaddedInput_shared[((((int)threadIdx.x) + 6528))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 6528) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 6656))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 53248))];
  PaddedInput_shared[((((int)threadIdx.x) + 6784))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 6784) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 6912))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 55296))];
  PaddedInput_shared[((((int)threadIdx.x) + 7040))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 7040) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 7168))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 57344))];
  PaddedInput_shared[((((int)threadIdx.x) + 7296))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 7296) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 7424))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 59392))];
  PaddedInput_shared[((((int)threadIdx.x) + 7552))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 7552) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 7680))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 61440))];
  PaddedInput_shared[((((int)threadIdx.x) + 7808))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 7808) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  PaddedInput_shared[((((int)threadIdx.x) + 7936))] = placeholder[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 256)) + ((int)threadIdx.x)) + 63488))];
  PaddedInput_shared[((((int)threadIdx.x) + 8064))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 65536) + (((((int)threadIdx.x) + 8064) >> 8) * 2048)) + ((((int)blockIdx.x) & 7) * 256)) + (((((int)threadIdx.x) >> 4) + 8) * 16)) + (((int)threadIdx.x) & 15)))];
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[((((int)threadIdx.x) * 48))] = placeholder1[((((int)threadIdx.x) * 48))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 1))] = placeholder1[(((((int)threadIdx.x) * 48) + 1))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 2))] = placeholder1[(((((int)threadIdx.x) * 48) + 2))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 3))] = placeholder1[(((((int)threadIdx.x) * 48) + 3))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 4))] = placeholder1[(((((int)threadIdx.x) * 48) + 4))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 5))] = placeholder1[(((((int)threadIdx.x) * 48) + 5))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 6))] = placeholder1[(((((int)threadIdx.x) * 48) + 6))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 7))] = placeholder1[(((((int)threadIdx.x) * 48) + 7))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 8))] = placeholder1[(((((int)threadIdx.x) * 48) + 8))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 9))] = placeholder1[(((((int)threadIdx.x) * 48) + 9))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 10))] = placeholder1[(((((int)threadIdx.x) * 48) + 10))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 11))] = placeholder1[(((((int)threadIdx.x) * 48) + 11))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 12))] = placeholder1[(((((int)threadIdx.x) * 48) + 12))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 13))] = placeholder1[(((((int)threadIdx.x) * 48) + 13))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 14))] = placeholder1[(((((int)threadIdx.x) * 48) + 14))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 15))] = placeholder1[(((((int)threadIdx.x) * 48) + 15))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 16))] = placeholder1[(((((int)threadIdx.x) * 48) + 16))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 17))] = placeholder1[(((((int)threadIdx.x) * 48) + 17))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 18))] = placeholder1[(((((int)threadIdx.x) * 48) + 18))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 19))] = placeholder1[(((((int)threadIdx.x) * 48) + 19))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 20))] = placeholder1[(((((int)threadIdx.x) * 48) + 20))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 21))] = placeholder1[(((((int)threadIdx.x) * 48) + 21))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 22))] = placeholder1[(((((int)threadIdx.x) * 48) + 22))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 23))] = placeholder1[(((((int)threadIdx.x) * 48) + 23))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 24))] = placeholder1[(((((int)threadIdx.x) * 48) + 24))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 25))] = placeholder1[(((((int)threadIdx.x) * 48) + 25))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 26))] = placeholder1[(((((int)threadIdx.x) * 48) + 26))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 27))] = placeholder1[(((((int)threadIdx.x) * 48) + 27))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 28))] = placeholder1[(((((int)threadIdx.x) * 48) + 28))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 29))] = placeholder1[(((((int)threadIdx.x) * 48) + 29))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 30))] = placeholder1[(((((int)threadIdx.x) * 48) + 30))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 31))] = placeholder1[(((((int)threadIdx.x) * 48) + 31))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 32))] = placeholder1[(((((int)threadIdx.x) * 48) + 32))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 33))] = placeholder1[(((((int)threadIdx.x) * 48) + 33))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 34))] = placeholder1[(((((int)threadIdx.x) * 48) + 34))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 35))] = placeholder1[(((((int)threadIdx.x) * 48) + 35))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 36))] = placeholder1[(((((int)threadIdx.x) * 48) + 36))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 37))] = placeholder1[(((((int)threadIdx.x) * 48) + 37))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 38))] = placeholder1[(((((int)threadIdx.x) * 48) + 38))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 39))] = placeholder1[(((((int)threadIdx.x) * 48) + 39))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 40))] = placeholder1[(((((int)threadIdx.x) * 48) + 40))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 41))] = placeholder1[(((((int)threadIdx.x) * 48) + 41))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 42))] = placeholder1[(((((int)threadIdx.x) * 48) + 42))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 43))] = placeholder1[(((((int)threadIdx.x) * 48) + 43))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 44))] = placeholder1[(((((int)threadIdx.x) * 48) + 44))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 45))] = placeholder1[(((((int)threadIdx.x) * 48) + 45))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 46))] = placeholder1[(((((int)threadIdx.x) * 48) + 46))];
  }
  if (((int)threadIdx.x) < 32) {
    placeholder_shared[(((((int)threadIdx.x) * 48) + 47))] = placeholder1[(((((int)threadIdx.x) * 48) + 47))];
  }
  __syncthreads();
  for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
    for (int ff_outer_inner = 0; ff_outer_inner < 32; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        Conv2dOutput[(((xx_outer_inner * 96) + (ff_outer_inner * 3)))] = (Conv2dOutput[(((xx_outer_inner * 96) + (ff_outer_inner * 3)))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 256) + ((((int)threadIdx.x) & 3) * 32)) + (xx_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((rc_inner * 96) + (ff_outer_inner * 3)))]));
        Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 192))] = (Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 192))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 256) + ((((int)threadIdx.x) & 3) * 32)) + (xx_outer_inner * 16)) + rc_inner) + 128))] * placeholder_shared[(((rc_inner * 96) + (ff_outer_inner * 3)))]));
        Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 1))] = (Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 256) + ((((int)threadIdx.x) & 3) * 32)) + (xx_outer_inner * 16)) + rc_inner))] * placeholder_shared[((((rc_inner * 96) + (ff_outer_inner * 3)) + 1))]));
        Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 193))] = (Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 193))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 256) + ((((int)threadIdx.x) & 3) * 32)) + (xx_outer_inner * 16)) + rc_inner) + 128))] * placeholder_shared[((((rc_inner * 96) + (ff_outer_inner * 3)) + 1))]));
        Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 2))] = (Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 256) + ((((int)threadIdx.x) & 3) * 32)) + (xx_outer_inner * 16)) + rc_inner))] * placeholder_shared[((((rc_inner * 96) + (ff_outer_inner * 3)) + 2))]));
        Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 194))] = (Conv2dOutput[((((xx_outer_inner * 96) + (ff_outer_inner * 3)) + 194))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 256) + ((((int)threadIdx.x) & 3) * 32)) + (xx_outer_inner * 16)) + rc_inner) + 128))] * placeholder_shared[((((rc_inner * 96) + (ff_outer_inner * 3)) + 2))]));
      }
    }
  }
  for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
    for (int i3_inner = 0; i3_inner < 96; ++i3_inner) {
      compute[((((((((((int)blockIdx.x) >> 3) * 393216) + ((((int)threadIdx.x) >> 2) * 12288)) + ((((int)blockIdx.x) & 7) * 1536)) + ((((int)threadIdx.x) & 3) * 192)) + (i2_inner * 96)) + i3_inner))] = max(min((Conv2dOutput[(((i2_inner * 96) + i3_inner))] + placeholder2[(i3_inner)]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) >> 3) * 393216) + ((((int)threadIdx.x) >> 2) * 12288)) + ((((int)blockIdx.x) & 7) * 1536)) + ((((int)threadIdx.x) & 3) * 192)) + (i2_inner * 96)) + i3_inner) + 768))] = max(min((Conv2dOutput[((((i2_inner * 96) + i3_inner) + 192))] + placeholder2[(i3_inner)]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


