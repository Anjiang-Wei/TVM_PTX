
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[14];
  __shared__ float PaddedInput_shared[1344];
  __shared__ float placeholder_shared[24];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(7)] = 0.000000e+00f;
  DepthwiseConv2d[(8)] = 0.000000e+00f;
  DepthwiseConv2d[(9)] = 0.000000e+00f;
  DepthwiseConv2d[(10)] = 0.000000e+00f;
  DepthwiseConv2d[(11)] = 0.000000e+00f;
  DepthwiseConv2d[(12)] = 0.000000e+00f;
  DepthwiseConv2d[(13)] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
      __syncthreads();
      PaddedInput_shared[((((int)threadIdx.x) * 14))] = (((((1 <= ((((((int)threadIdx.x) % 12) * 14) / 24) + di_outer_outer)) && (((((((int)threadIdx.x) % 12) * 14) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[(((((((((((int)threadIdx.x) * 14) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + ((((int)threadIdx.x) * 14) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 1))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 1) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 1) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 1) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 1) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 2))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 2) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 2) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 2) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 2) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 3))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 3) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 3) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 3) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 3) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 4))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 4) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 4) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 4) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 4) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 5))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 5) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 5) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 5) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 5) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 6))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 6) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 6) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 6) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 6) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 7))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 7) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 7) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 7) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 7) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 8))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 8) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 8) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 8) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 8) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 9))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 9) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 9) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 9) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 9) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 10))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 10) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 10) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 10) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 10) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 11))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 11) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 11) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 11) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 11) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 12))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 12) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 12) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 12) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 12) % 24)) - 7680))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 14) + 13))] = (((((1 <= (((((((int)threadIdx.x) * 14) + 13) % 168) / 24) + di_outer_outer)) && ((((((((int)threadIdx.x) * 14) + 13) % 168) / 24) + di_outer_outer) < 8)) && (1 <= ((((int)blockIdx.x) / 40) + dj_outer_outer))) && (((((int)blockIdx.x) / 40) + dj_outer_outer) < 8)) ? placeholder[((((((((((((int)threadIdx.x) * 14) + 13) / 24) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 960)) + (((int)blockIdx.x) * 24)) + (((((int)threadIdx.x) * 14) + 13) % 24)) - 7680))] : 0.000000e+00f);
      if (((int)threadIdx.x) < 12) {
        placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[(((((di_outer_outer * 2880) + (dj_outer_outer * 960)) + ((((int)blockIdx.x) % 40) * 24)) + (((int)threadIdx.x) * 2)))];
      }
      if (((int)threadIdx.x) < 12) {
        placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[((((((di_outer_outer * 2880) + (dj_outer_outer * 960)) + ((((int)blockIdx.x) % 40) * 24)) + (((int)threadIdx.x) * 2)) + 1))];
      }
      __syncthreads();
      DepthwiseConv2d[(0)] = (DepthwiseConv2d[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(1)] = (DepthwiseConv2d[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 24))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(2)] = (DepthwiseConv2d[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 48))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(3)] = (DepthwiseConv2d[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 72))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(4)] = (DepthwiseConv2d[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 96))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(5)] = (DepthwiseConv2d[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 120))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(6)] = (DepthwiseConv2d[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 144))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(7)] = (DepthwiseConv2d[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 168))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(8)] = (DepthwiseConv2d[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 192))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(9)] = (DepthwiseConv2d[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 216))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(10)] = (DepthwiseConv2d[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 240))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(11)] = (DepthwiseConv2d[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 264))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(12)] = (DepthwiseConv2d[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 288))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
      DepthwiseConv2d[(13)] = (DepthwiseConv2d[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 24) * 336) + (((int)threadIdx.x) % 24)) + 312))] * placeholder_shared[((((int)threadIdx.x) % 24))]));
    }
  }
  for (int i0_inner = 0; i0_inner < 2; ++i0_inner) {
    for (int i1_inner = 0; i1_inner < 7; ++i1_inner) {
      compute[(((((((((int)threadIdx.x) / 24) * 94080) + (i0_inner * 47040)) + (i1_inner * 6720)) + (((int)blockIdx.x) * 24)) + (((int)threadIdx.x) % 24)))] = max(min((DepthwiseConv2d[(((i0_inner * 7) + i1_inner))] + placeholder2[((((((int)blockIdx.x) % 40) * 24) + (((int)threadIdx.x) % 24)))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


