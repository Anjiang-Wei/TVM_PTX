
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_divide, float* __restrict__ placeholder_red) {
  T_divide[(((((int)blockIdx.x) * 4) + ((int)threadIdx.x)))] = (placeholder_red[(((((int)blockIdx.x) * 4) + ((int)threadIdx.x)))] * 9.765625e-04f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder_red, float* __restrict__ placeholder) {
  placeholder_red[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int k2 = 0; k2 < 1024; ++k2) {
    placeholder_red[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = (placeholder_red[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 2048) + (((int)threadIdx.x) * 1024)) + k2))]);
  }
}


