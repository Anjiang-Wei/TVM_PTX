
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[896];
  __shared__ float placeholder_shared[160];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 10; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) / 112) * 31360) + ((((int)blockIdx.x) / 3) * 1120)) + (((((int)threadIdx.x) % 112) >> 2) * 40)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    if (((int)threadIdx.x) < 336) {
      PaddedInput_shared[((((int)threadIdx.x) + 560))] = placeholder[((((((((((int)threadIdx.x) / 112) * 31360) + ((((int)blockIdx.x) / 3) * 1120)) + (((((int)threadIdx.x) % 112) >> 2) * 40)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 156800))];
    }
    if (((int)threadIdx.x) < 160) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 480) + ((((int)threadIdx.x) / 40) * 120)) + ((((int)blockIdx.x) % 3) * 40)) + (((int)threadIdx.x) % 40)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 4))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 4))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 8))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 12))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 8))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 12))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 112))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 116))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 112))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 116))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 120))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 124))] * placeholder_shared[(((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 140) * 224) + (((((int)threadIdx.x) % 140) / 20) * 16)) + rc_outer_inner) + 124))] * placeholder_shared[((((rc_outer_inner * 40) + ((((int)threadIdx.x) % 20) * 2)) + 1))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((((int)threadIdx.x) / 140) * 188160) + (ax0_inner * 94080)) + ((((int)blockIdx.x) / 3) * 3360)) + (((((int)threadIdx.x) % 140) / 20) * 480)) + (ax2_inner * 120)) + ((((int)blockIdx.x) % 3) * 40)) + ((((int)threadIdx.x) % 20) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 8) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) % 3) * 40) + ((((int)threadIdx.x) % 20) * 2)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


