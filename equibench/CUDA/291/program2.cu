
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[8];
  __shared__ float placeholder_d_shared[128];
  __shared__ float placeholder_shared[640];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(4)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(5)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(6)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  T_dense[(7)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[((((int)threadIdx.x) * 64))] = placeholder[(((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 1))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 1))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 2))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 3))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 3))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 4))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 5))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 5))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 6))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 7))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 7))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 8))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 8))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 9))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 9))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 10))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 10))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 11))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 11))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 12))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 12))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 13))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 13))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 14))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 14))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 15))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 15))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 16))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2048))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 17))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2049))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 18))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2050))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 19))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2051))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 20))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2052))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 21))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2053))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 22))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2054))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 23))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2055))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 24))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2056))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 25))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2057))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 26))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2058))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 27))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2059))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 28))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2060))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 29))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2061))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 30))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2062))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 31))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 2063))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 32))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4096))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 33))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4097))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 34))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4098))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 35))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4099))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 36))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4100))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 37))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4101))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 38))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4102))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 39))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4103))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 40))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4104))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 41))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4105))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 42))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4106))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 43))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4107))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 44))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4108))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 45))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4109))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 46))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4110))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 47))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 4111))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 48))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6144))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 49))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6145))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 50))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6146))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 51))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6147))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 52))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6148))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 53))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6149))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 54))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6150))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 55))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6151))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 56))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6152))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 57))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6153))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 58))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6154))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 59))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6155))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 60))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6156))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 61))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6157))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 62))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6158))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_d_shared[(((((int)threadIdx.x) * 64) + 63))] = placeholder[((((((int)threadIdx.x) * 8192) + (k_outer_outer * 16)) + 6159))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 40))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 40) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 10240))];
    placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 120) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 20480))];
    placeholder_shared[((((int)threadIdx.x) + 200))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 200) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 30720))];
    placeholder_shared[((((int)threadIdx.x) + 280))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 280) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 40960))];
    placeholder_shared[((((int)threadIdx.x) + 360))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 360) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 400))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 51200))];
    placeholder_shared[((((int)threadIdx.x) + 440))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 440) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 480))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 61440))];
    placeholder_shared[((((int)threadIdx.x) + 520))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 520) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 560))] = placeholder1[((((((((int)blockIdx.x) * 81920) + ((((int)threadIdx.x) >> 4) * 2048)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 71680))];
    placeholder_shared[((((int)threadIdx.x) + 600))] = placeholder1[(((((((int)blockIdx.x) * 81920) + (((((int)threadIdx.x) + 600) >> 4) * 2048)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) + 8) & 15)))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 320))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 16))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 336))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 16))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 320))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 16))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 16))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 16))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 336))]));
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 1))]));
      T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 321))]));
      T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 17))]));
      T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 337))]));
      T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 17))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 1))]));
      T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 17))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 321))]));
      T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 17))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 17))]));
      T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(((((((int)threadIdx.x) / 10) * 32) + (k_outer_inner * 2)) + 17))] * placeholder_shared[(((((((int)threadIdx.x) % 10) * 32) + (k_outer_inner * 2)) + 337))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      T_add[(((((((((int)threadIdx.x) / 10) * 2000) + (ax0_inner * 1000)) + (((int)blockIdx.x) * 40)) + ((((int)threadIdx.x) % 10) * 2)) + ax1_inner))] = (T_dense[(((ax0_inner * 2) + ax1_inner))] + placeholder2[((((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) % 10) * 2)) + ax1_inner))]);
      T_add[((((((((((int)threadIdx.x) / 10) * 2000) + (ax0_inner * 1000)) + (((int)blockIdx.x) * 40)) + ((((int)threadIdx.x) % 10) * 2)) + ax1_inner) + 20))] = (T_dense[((((ax0_inner * 2) + ax1_inner) + 4))] + placeholder2[(((((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) % 10) * 2)) + ax1_inner) + 20))]);
    }
  }
}


