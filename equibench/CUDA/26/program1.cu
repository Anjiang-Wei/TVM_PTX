
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[40];
  __shared__ float PaddedInput_shared[680];
  __shared__ float placeholder_shared[12];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 5; ++i_outer_inner_init) {
    DepthwiseConv2d[((i_outer_inner_init * 2))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 10))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 20))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 30))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 11))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 21))] = 0.000000e+00f;
    DepthwiseConv2d[(((i_outer_inner_init * 2) + 31))] = 0.000000e+00f;
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 60; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) < 170) {
        if (((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 680) {
          if (((int)threadIdx.x) < 12) {
            PaddedInput_shared[(((((int)threadIdx.x) * 60) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = (((((1 <= (((((((int)blockIdx.x) % 432) / 144) * 5) + ((((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 85) / 17)) + di_outer_outer)) && ((((((((int)blockIdx.x) % 432) / 144) * 5) + ((((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 85) / 17)) + di_outer_outer) < 16)) && (1 <= (((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 17))) && ((((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 17) < 16)) ? placeholder[(((((((((((((int)blockIdx.x) / 432) * 259200) + ((((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) / 85) * 129600)) + (((((int)blockIdx.x) % 432) / 144) * 43200)) + (((((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 85) / 17) * 8640)) + (di_outer_outer * 8640)) + ((((((int)threadIdx.x) * 15) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 17) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s & 3)) - 9216))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[(((((di_outer_outer * 1728) + ((((int)threadIdx.x) >> 1) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + ((((int)threadIdx.x) & 1) * 2)))];
    }
    if (((int)threadIdx.x) < 6) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((di_outer_outer * 1728) + ((((((int)threadIdx.x) * 2) + 1) >> 2) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + (((((int)threadIdx.x) * 2) + 1) & 3)))];
    }
    __syncthreads();
    for (int dj_outer_inner = 0; dj_outer_inner < 3; ++dj_outer_inner) {
      for (int i_outer_inner = 0; i_outer_inner < 5; ++i_outer_inner) {
        DepthwiseConv2d[((i_outer_inner * 2))] = (DepthwiseConv2d[((i_outer_inner * 2))] + (PaddedInput_shared[((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)))] * placeholder_shared[((dj_outer_inner * 4))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 10))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 10))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 2))] * placeholder_shared[(((dj_outer_inner * 4) + 2))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 20))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 20))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 340))] * placeholder_shared[((dj_outer_inner * 4))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 30))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 30))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 342))] * placeholder_shared[(((dj_outer_inner * 4) + 2))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 1))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 1))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 1))] * placeholder_shared[(((dj_outer_inner * 4) + 1))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 11))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 11))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 3))] * placeholder_shared[(((dj_outer_inner * 4) + 3))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 21))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 21))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 341))] * placeholder_shared[(((dj_outer_inner * 4) + 1))]));
        DepthwiseConv2d[(((i_outer_inner * 2) + 31))] = (DepthwiseConv2d[(((i_outer_inner * 2) + 31))] + (PaddedInput_shared[(((((i_outer_inner * 68) + (((int)threadIdx.x) * 4)) + (dj_outer_inner * 4)) + 343))] * placeholder_shared[(((dj_outer_inner * 4) + 3))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 5; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
      compute[((((((((((int)blockIdx.x) / 432) * 259200) + (((((int)blockIdx.x) % 432) / 144) * 43200)) + (i1_inner * 8640)) + (((int)threadIdx.x) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + i3_inner))] = max(min((DepthwiseConv2d[(((i1_inner * 2) + i3_inner))] + placeholder2[((((((int)blockIdx.x) % 144) * 4) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) / 432) * 259200) + (((((int)blockIdx.x) % 432) / 144) * 43200)) + (i1_inner * 8640)) + (((int)threadIdx.x) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + i3_inner) + 2))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i3_inner) + 10))] + placeholder2[(((((((int)blockIdx.x) % 144) * 4) + i3_inner) + 2))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) / 432) * 259200) + (((((int)blockIdx.x) % 432) / 144) * 43200)) + (i1_inner * 8640)) + (((int)threadIdx.x) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + i3_inner) + 129600))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i3_inner) + 20))] + placeholder2[((((((int)blockIdx.x) % 144) * 4) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[(((((((((((int)blockIdx.x) / 432) * 259200) + (((((int)blockIdx.x) % 432) / 144) * 43200)) + (i1_inner * 8640)) + (((int)threadIdx.x) * 576)) + ((((int)blockIdx.x) % 144) * 4)) + i3_inner) + 129602))] = max(min((DepthwiseConv2d[((((i1_inner * 2) + i3_inner) + 30))] + placeholder2[(((((((int)blockIdx.x) % 144) * 4) + i3_inner) + 2))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


