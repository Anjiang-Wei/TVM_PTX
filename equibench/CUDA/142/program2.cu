
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[224];
  __shared__ float PaddedInput_shared[1792];
  __shared__ float placeholder_shared[2048];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 7; ++xx_inner_init) {
        for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
          Conv2dOutput[(((((yy_inner_init * 56) + (xx_outer_inner_init * 28)) + (xx_inner_init * 4)) + ff_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 56) + (xx_outer_inner_init * 28)) + (xx_inner_init * 4)) + ff_inner_init) + 112))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 28; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 4)) / 28) * 50176) + (((int)blockIdx.x) * 7168)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 4)) % 28) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 2048) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
              for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
                Conv2dOutput[(((((yy_inner * 56) + (xx_outer_inner * 28)) + (xx_inner * 4)) + ff_inner))] = (Conv2dOutput[(((((yy_inner * 56) + (xx_outer_inner * 28)) + (xx_inner * 4)) + ff_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 4) * 448) + (yy_inner * 224)) + (xx_outer_inner * 112)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 15) * 4)) + ff_inner))]));
                Conv2dOutput[((((((yy_inner * 56) + (xx_outer_inner * 28)) + (xx_inner * 4)) + ff_inner) + 112))] = (Conv2dOutput[((((((yy_inner * 56) + (xx_outer_inner * 28)) + (xx_inner * 4)) + ff_inner) + 112))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 4) * 448) + (yy_inner * 224)) + (xx_outer_inner * 112)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 15) * 4)) + ff_inner) + 64))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[((((((((((int)threadIdx.x) >> 4) * 25088) + (((int)blockIdx.x) * 3584)) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 56) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 15) * 4) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)threadIdx.x) >> 4) * 25088) + (((int)blockIdx.x) * 3584)) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner) + 64))] = max((Conv2dOutput[(((((ax1_inner * 56) + (ax2_inner * 4)) + ax3_inner) + 112))] + placeholder2[(((((((int)threadIdx.x) & 15) * 4) + ax3_inner) + 64))]), 0.000000e+00f);
      }
    }
  }
}


