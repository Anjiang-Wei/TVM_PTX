
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[256];
  __shared__ float placeholder_shared[768];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    Conv2dOutput[(yy_outer_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 2))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 4))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 6))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 256) {
        if (((int)threadIdx.x) < 32) {
          PaddedInput_shared[(((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 8192) + ((((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 5) * 1024)) + (((((int)blockIdx.x) & 63) >> 2) * 64)) + (((((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 31) >> 4) * 32)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15)))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 576) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 768) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + ((int)threadIdx.x)) < 128) {
            placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) / 48)) & 15) * 48) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) % 48)))] = placeholder1[(((((rc_outer_outer * 3072) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) / 48)) & 15) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) % 48)))];
          }
        }
      }
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        Conv2dOutput[(yy_outer_inner)] = (Conv2dOutput[(yy_outer_inner)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 48) * 64) + (yy_outer_inner * 32)) + rc_inner))] * placeholder_shared[(((rc_inner * 48) + (((int)threadIdx.x) % 48)))]));
        Conv2dOutput[((yy_outer_inner + 2))] = (Conv2dOutput[((yy_outer_inner + 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 48) * 64) + (yy_outer_inner * 32)) + rc_inner) + 16))] * placeholder_shared[(((rc_inner * 48) + (((int)threadIdx.x) % 48)))]));
        Conv2dOutput[((yy_outer_inner + 4))] = (Conv2dOutput[((yy_outer_inner + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 48) * 64) + (yy_outer_inner * 32)) + rc_inner) + 128))] * placeholder_shared[(((rc_inner * 48) + (((int)threadIdx.x) % 48)))]));
        Conv2dOutput[((yy_outer_inner + 6))] = (Conv2dOutput[((yy_outer_inner + 6))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 48) * 64) + (yy_outer_inner * 32)) + rc_inner) + 144))] * placeholder_shared[(((rc_inner * 48) + (((int)threadIdx.x) % 48)))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((((((int)blockIdx.x) >> 6) * 49152) + ((((int)threadIdx.x) / 48) * 12288)) + (i1_inner * 6144)) + (((((int)blockIdx.x) & 63) >> 2) * 384)) + ((((int)blockIdx.x) & 3) * 48)) + (((int)threadIdx.x) % 48)))] = max(min((Conv2dOutput[(i1_inner)] + placeholder2[((((((int)blockIdx.x) & 3) * 48) + (((int)threadIdx.x) % 48)))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)blockIdx.x) >> 6) * 49152) + ((((int)threadIdx.x) / 48) * 12288)) + (i1_inner * 6144)) + (((((int)blockIdx.x) & 63) >> 2) * 384)) + ((((int)blockIdx.x) & 3) * 48)) + (((int)threadIdx.x) % 48)) + 192))] = max(min((Conv2dOutput[((i1_inner + 2))] + placeholder2[((((((int)blockIdx.x) & 3) * 48) + (((int)threadIdx.x) % 48)))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)blockIdx.x) >> 6) * 49152) + ((((int)threadIdx.x) / 48) * 12288)) + (i1_inner * 6144)) + (((((int)blockIdx.x) & 63) >> 2) * 384)) + ((((int)blockIdx.x) & 3) * 48)) + (((int)threadIdx.x) % 48)) + 24576))] = max(min((Conv2dOutput[((i1_inner + 4))] + placeholder2[((((((int)blockIdx.x) & 3) * 48) + (((int)threadIdx.x) % 48)))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)blockIdx.x) >> 6) * 49152) + ((((int)threadIdx.x) / 48) * 12288)) + (i1_inner * 6144)) + (((((int)blockIdx.x) & 63) >> 2) * 384)) + ((((int)blockIdx.x) & 3) * 48)) + (((int)threadIdx.x) % 48)) + 24768))] = max(min((Conv2dOutput[((i1_inner + 6))] + placeholder2[((((((int)blockIdx.x) & 3) * 48) + (((int)threadIdx.x) % 48)))]), 6.000000e+00f), 0.000000e+00f);
  }
}


