
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[640];
  __shared__ float PaddedInput_shared[93600];
  __shared__ float placeholder_shared[3328];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 5; ++xx_inner_init) {
          for (int ff_inner_init = 0; ff_inner_init < 8; ++ff_inner_init) {
            Conv2dOutput[((((((nn_outer_inner_init * 160) + (nn_inner_init * 80)) + (xx_inner_init * 16)) + (ff_outer_inner_init * 8)) + ff_inner_init))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_outer_inner_init * 160) + (nn_inner_init * 80)) + (xx_inner_init * 16)) + (ff_outer_inner_init * 8)) + ff_inner_init) + 320))] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2080; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 45) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 45) + ((int)threadIdx.x)) / 104) * 416) + (rc_outer_outer * 104)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 45) + ((int)threadIdx.x)) % 104)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 74; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 45) + ((int)threadIdx.x)) < 3328) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 45) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 13312) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 45) + ((int)threadIdx.x)) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 45) + ((int)threadIdx.x)) & 31)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 104; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
          for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
            for (int xx_inner = 0; xx_inner < 5; ++xx_inner) {
              for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
                Conv2dOutput[((((((nn_outer_inner * 160) + (nn_inner * 80)) + (xx_inner * 16)) + (ff_outer_inner * 8)) + ff_inner))] = (Conv2dOutput[((((((nn_outer_inner * 160) + (nn_inner * 80)) + (xx_inner * 16)) + (ff_outer_inner * 8)) + ff_inner))] + (PaddedInput_shared[((((((nn_outer_inner * 46800) + (nn_inner * 23400)) + (((int)threadIdx.x) * 520)) + (xx_inner * 104)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 8)) + ff_inner))]));
                Conv2dOutput[(((((((nn_outer_inner * 160) + (nn_inner * 80)) + (xx_inner * 16)) + (ff_outer_inner * 8)) + ff_inner) + 320))] = (Conv2dOutput[(((((((nn_outer_inner * 160) + (nn_inner * 80)) + (xx_inner * 16)) + (ff_outer_inner * 8)) + ff_inner) + 320))] + (PaddedInput_shared[((((((nn_outer_inner * 46800) + (nn_inner * 23400)) + (((int)threadIdx.x) * 520)) + (xx_inner * 104)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 32) + (ff_outer_inner * 8)) + ff_inner) + 16))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
        T_relu[((((((ax0_inner * 28800) + (((int)threadIdx.x) * 640)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 80) + (ax2_inner * 16)) + ax3_inner))] + placeholder2[(((((int)blockIdx.x) * 32) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 28800) + (((int)threadIdx.x) * 640)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ax3_inner) + 16))] = max((Conv2dOutput[(((((ax0_inner * 80) + (ax2_inner * 16)) + ax3_inner) + 320))] + placeholder2[((((((int)blockIdx.x) * 32) + ax3_inner) + 16))]), 0.000000e+00f);
      }
    }
  }
}


