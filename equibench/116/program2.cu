
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[2048];
  __shared__ float placeholder_shared[8192];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 2))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 3))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 4))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 5))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 6))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 7))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 16))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 17))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 18))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 19))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 20))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 21))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 22))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 64) + (yy_outer_inner_init * 32)) + (ff_outer_inner_init * 8)) + 23))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) >> 10) * 131072) + ((((int)blockIdx.x) >> 5) * 65536)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) & 1023) >> 6) * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + (rc_outer_outer * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) & 63)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 256; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 8192) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 64; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
            Conv2dOutput[((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)))] = (Conv2dOutput[((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 1))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 1))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 1))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 2))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 2))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 2))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 3))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 3))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 3))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 4))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 4))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 4))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 5))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 5))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 5))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 6))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 6))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 6))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 7))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 7))] + (PaddedInput_shared[(((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 7))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 16))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 16))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 17))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 17))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 1))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 18))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 18))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 2))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 19))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 19))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 3))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 20))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 20))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 4))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 21))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 21))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 5))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 22))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 22))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 6))]));
            Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 23))] = (Conv2dOutput[(((((nn_outer_inner * 64) + (yy_outer_inner * 32)) + (ff_outer_inner * 8)) + 23))] + (PaddedInput_shared[((((((nn_outer_inner * 1024) + ((((int)threadIdx.x) >> 3) * 256)) + (yy_outer_inner * 128)) + rc_outer_inner) + 64))] * placeholder_shared[(((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (ff_outer_inner * 8)) + 7))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
        T_relu[((((((((ax0_inner * 131072) + ((((int)blockIdx.x) >> 5) * 65536)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) & 31) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 64) + (ax1_inner * 16)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


