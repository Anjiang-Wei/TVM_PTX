
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[56];
  __shared__ float PaddedInput_shared[330];
  __shared__ float placeholder_shared[128];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 14; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 29))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 2))] = placeholder[((((((((int)blockIdx.x) >> 3) * 57344) + ((((int)threadIdx.x) / 55) * 14336)) + ((((int)threadIdx.x) % 55) * 256)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 57344) + ((((int)threadIdx.x) / 55) * 14336)) + ((((int)threadIdx.x) % 55) * 256)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 128))] = placeholder[((((((((int)blockIdx.x) >> 3) * 57344) + (((((int)threadIdx.x) + 64) / 55) * 14336)) + (((((int)threadIdx.x) + 9) % 55) * 256)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 129))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 57344) + (((((int)threadIdx.x) + 64) / 55) * 14336)) + (((((int)threadIdx.x) + 9) % 55) * 256)) + (rc_outer_outer * 2)) + 1))];
    if (((int)threadIdx.x) < 37) {
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 256))] = placeholder[((((((((int)blockIdx.x) >> 3) * 57344) + (((((int)threadIdx.x) + 128) / 55) * 14336)) + ((((int)threadIdx.x) + 18) * 256)) + (rc_outer_outer * 2)))];
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 257))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 57344) + (((((int)threadIdx.x) + 128) / 55) * 14336)) + ((((int)threadIdx.x) + 18) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[((((int)threadIdx.x) * 8))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 7) * 8)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 1) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 1) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 2) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 2) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 3) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 3) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 4) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 4) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 5) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 5) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 6) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 6) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder1[(((((rc_outer_outer * 1024) + ((((((int)threadIdx.x) * 8) + 7) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + 7) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 14; ++xx_outer_inner) {
        Conv2dOutput[((xx_outer_inner * 2))] = (Conv2dOutput[((xx_outer_inner * 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 56) + (xx_outer_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 28))] = (Conv2dOutput[(((xx_outer_inner * 2) + 28))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 56) + (xx_outer_inner * 4)) + rc_outer_inner) + 220))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 1))] = (Conv2dOutput[(((xx_outer_inner * 2) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 56) + (xx_outer_inner * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 29))] = (Conv2dOutput[(((xx_outer_inner * 2) + 29))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 56) + (xx_outer_inner * 4)) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_add[((((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))] = (Conv2dOutput[(((ax2_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]);
      T_add[(((((((((((int)blockIdx.x) >> 3) * 28672) + ((((int)threadIdx.x) >> 5) * 7168)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner) + 14336))] = (Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 28))] + placeholder2[(((((((int)blockIdx.x) & 7) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]);
    }
  }
}


