
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[128];
  __shared__ float PaddedInput_shared[4896];
  __shared__ float placeholder_shared[72];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 2; ++i_outer_inner_init) {
    for (int j_outer_inner_init = 0; j_outer_inner_init < 2; ++j_outer_inner_init) {
      for (int c_outer_inner_init = 0; c_outer_inner_init < 2; ++c_outer_inner_init) {
        DepthwiseConv2d[((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 32))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 64))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 96))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 33))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 65))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 97))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 8))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 40))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 72))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 104))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 9))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 41))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 73))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 16) + (j_outer_inner_init * 4)) + (c_outer_inner_init * 2)) + 105))] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 153; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = (((((1 <= (((((int)blockIdx.x) >> 4) * 32) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 3)) / 18))) && ((((((int)blockIdx.x) >> 4) * 32) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 3)) / 18)) < 129)) && (1 <= ((((((int)blockIdx.x) & 15) >> 1) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 3)) % 18)))) && (((((((int)blockIdx.x) & 15) >> 1) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 3)) % 18)) < 129)) ? placeholder[(((((((((((int)blockIdx.x) >> 4) * 65536) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 3)) / 18) * 2048)) + (((((int)blockIdx.x) & 15) >> 1) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + (((int)threadIdx.x) >> 3)) % 18) * 16)) + ((((int)blockIdx.x) & 1) * 8)) + (((int)threadIdx.x) & 7)) - 2064))] : 0.000000e+00f);
  }
  placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[(((((((int)threadIdx.x) >> 2) * 16) + ((((int)blockIdx.x) & 1) * 8)) + ((((int)threadIdx.x) & 3) * 2)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1) >> 3) * 16) + ((((int)blockIdx.x) & 1) * 8)) + (((((int)threadIdx.x) * 2) + 1) & 7)))];
  if (((int)threadIdx.x) < 4) {
    placeholder_shared[(((((((((int)threadIdx.x) * 2) + 64) / 24) * 24) + (((int)threadIdx.x) * 2)) + 16))] = placeholder1[((((((((((int)threadIdx.x) * 2) + 64) / 24) * 48) + ((((int)blockIdx.x) & 1) * 8)) + (((int)threadIdx.x) * 2)) + 32))];
  }
  if (((int)threadIdx.x) < 4) {
    placeholder_shared[(((((((((int)threadIdx.x) * 2) + 65) / 24) * 24) + (((int)threadIdx.x) * 2)) + 17))] = placeholder1[((((((((((int)threadIdx.x) * 2) + 65) / 24) * 48) + ((((int)blockIdx.x) & 1) * 8)) + (((int)threadIdx.x) * 2)) + 33))];
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int dj_outer_inner = 0; dj_outer_inner < 3; ++dj_outer_inner) {
      for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
        for (int j_outer_inner = 0; j_outer_inner < 2; ++j_outer_inner) {
          for (int c_outer_inner = 0; c_outer_inner < 2; ++c_outer_inner) {
            DepthwiseConv2d[((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)))] = (DepthwiseConv2d[((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)))] * placeholder_shared[((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 32))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 32))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 4))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 4))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 64))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 64))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 64))] * placeholder_shared[((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 96))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 96))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 68))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 4))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 1))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 1))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 1))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 33))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 33))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 5))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 5))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 65))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 65))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 65))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 1))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 97))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 97))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 69))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 5))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 8))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 8))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 144))] * placeholder_shared[((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 40))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 40))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 148))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 4))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 72))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 72))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 208))] * placeholder_shared[((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 104))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 104))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 212))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 4))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 9))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 9))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 145))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 1))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 41))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 41))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 149))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 5))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 73))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 73))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 209))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 1))]));
            DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 105))] = (DepthwiseConv2d[(((((i_outer_inner * 16) + (j_outer_inner * 4)) + (c_outer_inner * 2)) + 105))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 2) * 576) + (i_outer_inner * 288)) + (di_outer_inner * 144)) + ((((int)threadIdx.x) & 3) * 16)) + (j_outer_inner * 8)) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 213))] * placeholder_shared[(((((di_outer_inner * 24) + (dj_outer_inner * 8)) + (c_outer_inner * 2)) + 5))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax1_inner * 2048)) + (((((int)blockIdx.x) & 15) >> 1) * 256)) + ((((int)threadIdx.x) & 3) * 32)) + (ax2_inner * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ax3_inner))] = max((DepthwiseConv2d[((((ax1_inner * 8) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) & 1) * 8) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax1_inner * 2048)) + (((((int)blockIdx.x) & 15) >> 1) * 256)) + ((((int)threadIdx.x) & 3) * 32)) + (ax2_inner * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ax3_inner) + 4))] = max((DepthwiseConv2d[(((((ax1_inner * 8) + (ax2_inner * 4)) + ax3_inner) + 32))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ax3_inner) + 4))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax1_inner * 2048)) + (((((int)blockIdx.x) & 15) >> 1) * 256)) + ((((int)threadIdx.x) & 3) * 32)) + (ax2_inner * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ax3_inner) + 128))] = max((DepthwiseConv2d[(((((ax1_inner * 8) + (ax2_inner * 4)) + ax3_inner) + 64))] + placeholder2[((((((int)blockIdx.x) & 1) * 8) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 4) * 65536) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax1_inner * 2048)) + (((((int)blockIdx.x) & 15) >> 1) * 256)) + ((((int)threadIdx.x) & 3) * 32)) + (ax2_inner * 16)) + ((((int)blockIdx.x) & 1) * 8)) + ax3_inner) + 132))] = max((DepthwiseConv2d[(((((ax1_inner * 8) + (ax2_inner * 4)) + ax3_inner) + 96))] + placeholder2[(((((((int)blockIdx.x) & 1) * 8) + ax3_inner) + 4))]), 0.000000e+00f);
      }
    }
  }
}


