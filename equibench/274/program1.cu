
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[4608];
  __shared__ float placeholder_shared[65536];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) / 14) * 57344) + ((((int)threadIdx.x) / 384) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (((((int)threadIdx.x) % 384) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((((int)blockIdx.x) / 14) * 57344) + (((((int)threadIdx.x) + 512) / 384) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((((int)threadIdx.x) >> 7) + 1) % 3) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[((((((((((((int)threadIdx.x) + 1024) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + (((((((int)threadIdx.x) >> 7) + 8) % 9) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((((int)threadIdx.x) >> 7) + 2) % 3) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((((((int)threadIdx.x) + 1536) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + ((((((int)threadIdx.x) >> 7) + 3) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (((((int)threadIdx.x) % 384) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[((((((((((((int)threadIdx.x) + 2048) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + (((((((int)threadIdx.x) >> 7) + 7) % 9) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((((int)threadIdx.x) >> 7) + 1) % 3) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[((((((((((((int)threadIdx.x) + 2560) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + ((((((int)threadIdx.x) >> 7) + 2) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((((int)threadIdx.x) >> 7) + 2) % 3) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3072))] = placeholder[((((((((((((int)threadIdx.x) + 3072) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + (((((((int)threadIdx.x) >> 7) + 6) % 9) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (((((int)threadIdx.x) % 384) >> 7) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3584))] = placeholder[((((((((((((int)threadIdx.x) + 3584) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + ((((((int)threadIdx.x) >> 7) + 1) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((((int)threadIdx.x) >> 7) + 1) % 3) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4096))] = placeholder[((((((((((((int)threadIdx.x) + 4096) / 1152) * 401408) + ((((int)blockIdx.x) / 14) * 57344)) + ((((((int)threadIdx.x) >> 7) + 5) / 3) * 14336)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((((int)threadIdx.x) >> 7) + 2) % 3) * 512)) + (rc_outer_outer * 128)) + (((int)threadIdx.x) & 127)))];
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 128; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 131072) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1024)) + ((((int)blockIdx.x) & 1) * 512)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 64; ++rc_inner) {
          Conv2dOutput[(xx_outer_inner)] = (Conv2dOutput[(xx_outer_inner)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)))]));
          Conv2dOutput[((xx_outer_inner + 4))] = (Conv2dOutput[((xx_outer_inner + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)) + 256))]));
          Conv2dOutput[((xx_outer_inner + 8))] = (Conv2dOutput[((xx_outer_inner + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner) + 768))] * placeholder_shared[((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)))]));
          Conv2dOutput[((xx_outer_inner + 12))] = (Conv2dOutput[((xx_outer_inner + 12))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner) + 768))] * placeholder_shared[(((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)) + 256))]));
          Conv2dOutput[((xx_outer_inner + 2))] = (Conv2dOutput[((xx_outer_inner + 2))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner) + 1152))] * placeholder_shared[((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)))]));
          Conv2dOutput[((xx_outer_inner + 6))] = (Conv2dOutput[((xx_outer_inner + 6))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner) + 1152))] * placeholder_shared[(((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)) + 256))]));
          Conv2dOutput[((xx_outer_inner + 10))] = (Conv2dOutput[((xx_outer_inner + 10))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner) + 1920))] * placeholder_shared[((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)))]));
          Conv2dOutput[((xx_outer_inner + 14))] = (Conv2dOutput[((xx_outer_inner + 14))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 8) * 2304) + (xx_outer_inner * 256)) + (rc_outer_inner * 64)) + rc_inner) + 1920))] * placeholder_shared[(((((rc_outer_inner * 32768) + (rc_inner * 512)) + (((int)threadIdx.x) & 255)) + 256))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      T_add[(((((((((((int)threadIdx.x) >> 8) * 401408) + (ax0_inner * 200704)) + ((((int)blockIdx.x) / 14) * 28672)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 1) * 512)) + (((int)threadIdx.x) & 255)))] = (Conv2dOutput[(((ax0_inner * 2) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 1) * 512) + (((int)threadIdx.x) & 255)))]);
      T_add[((((((((((((int)threadIdx.x) >> 8) * 401408) + (ax0_inner * 200704)) + ((((int)blockIdx.x) / 14) * 28672)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 1) * 512)) + (((int)threadIdx.x) & 255)) + 256))] = (Conv2dOutput[((((ax0_inner * 2) + ax2_inner) + 4))] + placeholder2[(((((((int)blockIdx.x) & 1) * 512) + (((int)threadIdx.x) & 255)) + 256))]);
      T_add[((((((((((((int)threadIdx.x) >> 8) * 401408) + (ax0_inner * 200704)) + ((((int)blockIdx.x) / 14) * 28672)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 1) * 512)) + (((int)threadIdx.x) & 255)) + 14336))] = (Conv2dOutput[((((ax0_inner * 2) + ax2_inner) + 8))] + placeholder2[((((((int)blockIdx.x) & 1) * 512) + (((int)threadIdx.x) & 255)))]);
      T_add[((((((((((((int)threadIdx.x) >> 8) * 401408) + (ax0_inner * 200704)) + ((((int)blockIdx.x) / 14) * 28672)) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 1) * 512)) + (((int)threadIdx.x) & 255)) + 14592))] = (Conv2dOutput[((((ax0_inner * 2) + ax2_inner) + 12))] + placeholder2[(((((((int)blockIdx.x) & 1) * 512) + (((int)threadIdx.x) & 255)) + 256))]);
    }
  }
}


