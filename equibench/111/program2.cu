
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[3136];
  __shared__ float PaddedInput_shared[6272];
  __shared__ float placeholder_shared[256];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
      for (int yy_inner_init = 0; yy_inner_init < 4; ++yy_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 7; ++xx_inner_init) {
          Conv2dOutput[(((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 392))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 784))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 1176))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 1568))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 1960))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 2352))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 196) + (yy_outer_inner_init * 28)) + (yy_inner_init * 7)) + xx_inner_init) + 2744))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 196; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1024) + ((((int)threadIdx.x) >> 2) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      ((float2*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + (((int)threadIdx.x) * 2)))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 512) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)))))[0];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
          for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
              Conv2dOutput[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner))] = (Conv2dOutput[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner))] + (PaddedInput_shared[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[(((rc_inner * 64) + ((int)threadIdx.x)))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 392))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 392))] + (PaddedInput_shared[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((int)threadIdx.x)) + 32))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 784))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 784))] + (PaddedInput_shared[(((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner) + 28))] * placeholder_shared[(((rc_inner * 64) + ((int)threadIdx.x)))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 1176))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 1176))] + (PaddedInput_shared[(((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner) + 28))] * placeholder_shared[((((rc_inner * 64) + ((int)threadIdx.x)) + 32))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 1568))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 1568))] + (PaddedInput_shared[(((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner) + 56))] * placeholder_shared[(((rc_inner * 64) + ((int)threadIdx.x)))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 1960))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 1960))] + (PaddedInput_shared[(((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner) + 56))] * placeholder_shared[((((rc_inner * 64) + ((int)threadIdx.x)) + 32))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 2352))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 2352))] + (PaddedInput_shared[(((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner) + 84))] * placeholder_shared[(((rc_inner * 64) + ((int)threadIdx.x)))]));
              Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 2744))] = (Conv2dOutput[((((((nn_inner * 196) + (yy_outer_inner * 28)) + (yy_inner * 7)) + xx_inner) + 2744))] + (PaddedInput_shared[(((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (yy_inner * 112)) + (xx_inner * 4)) + rc_inner) + 84))] * placeholder_shared[((((rc_inner * 64) + ((int)threadIdx.x)) + 32))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 28; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
        T_relu[((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)))] = max((Conv2dOutput[((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner))] + placeholder2[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 32))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 392))] + placeholder2[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) + 32))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 896))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 784))] + placeholder2[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 928))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 1176))] + placeholder2[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) + 32))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 1792))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 1568))] + placeholder2[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 1824))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 1960))] + placeholder2[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) + 32))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 2688))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 2352))] + placeholder2[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 100352) + (ax1_inner * 3584)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 64)) + ((int)threadIdx.x)) + 2720))] = max((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 7)) + ax2_inner) + 2744))] + placeholder2[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) + 32))]), 0.000000e+00f);
      }
    }
  }
}


