
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[406];
  __shared__ float placeholder_shared[464];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) >> 4) * 928) + (rc_outer_outer * 58)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 28))] = placeholder[((((((((int)blockIdx.x) >> 4) * 928) + (rc_outer_outer * 58)) + ((int)threadIdx.x)) + 28))];
    PaddedInput_shared[((((int)threadIdx.x) + 56))] = placeholder[(((((((((int)threadIdx.x) + 56) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 56) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 84))] = placeholder[(((((((((int)threadIdx.x) + 84) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 26)))];
    PaddedInput_shared[((((int)threadIdx.x) + 112))] = placeholder[(((((((((int)threadIdx.x) + 112) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 54) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 140))] = placeholder[(((((((((int)threadIdx.x) + 140) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 168))] = placeholder[(((((((((int)threadIdx.x) + 168) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 52) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 196))] = placeholder[(((((((((int)threadIdx.x) + 196) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 22)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[(((((((((int)threadIdx.x) + 224) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 50) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 252))] = placeholder[(((((((((int)threadIdx.x) + 252) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 20)))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[(((((((((int)threadIdx.x) + 280) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 48) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 308))] = placeholder[(((((((((int)threadIdx.x) + 308) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 18)))];
    PaddedInput_shared[((((int)threadIdx.x) + 336))] = placeholder[(((((((((int)threadIdx.x) + 336) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 46) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 364))] = placeholder[(((((((((int)threadIdx.x) + 364) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 16)))];
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[(((((((((int)threadIdx.x) + 392) / 58) * 6496) + ((((int)blockIdx.x) >> 4) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 44)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 58; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 58) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 464) {
        if (((int)threadIdx.x) < 8) {
          placeholder_shared[(((((int)threadIdx.x) * 58) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 7424) + ((((((int)threadIdx.x) * 58) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 3) * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((((int)threadIdx.x) * 58) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 7)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 58; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 2) * 58) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 3)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 2) * 58) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 8) + (((int)threadIdx.x) & 3)) + 4))]));
    }
  }
  T_relu[(((((((int)threadIdx.x) >> 2) * 896) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 3)))] = max((Conv2dOutput[(0)] + placeholder2[((((((int)blockIdx.x) & 15) * 8) + (((int)threadIdx.x) & 3)))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.x) >> 2) * 896) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) & 3)) + 4))] = max((Conv2dOutput[(1)] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + (((int)threadIdx.x) & 3)) + 4))]), 0.000000e+00f);
}


