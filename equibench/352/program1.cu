
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[64];
  __shared__ float placeholder_d_shared[128];
  __shared__ float placeholder_shared[1024];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 4; ++b_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 4; ++b_c_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        compute_local[((((b_c_outer_inner_init * 8) + (b_c_inner_init * 2)) + i_c_inner_init))] = 0.000000e+00f;
        compute_local[(((((b_c_outer_inner_init * 8) + (b_c_inner_init * 2)) + i_c_inner_init) + 32))] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      placeholder_d_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_outer_outer * 16384) + ((((int)threadIdx.x) >> 3) * 4096)) + ((((int)blockIdx.x) >> 1) * 256)) + (((((int)threadIdx.x) & 7) >> 1) * 64)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_outer_outer1 * 32) + ((int)threadIdx.x)))] = placeholder1[(((((((((ax0_ax1_fused_ax2_fused_outer_outer1 * 16) + (((int)threadIdx.x) >> 1)) >> 5) * 4096) + ((((int)blockIdx.x) & 1) * 2048)) + ((((ax0_ax1_fused_ax2_fused_outer_outer1 * 16) + (((int)threadIdx.x) >> 1)) & 31) * 64)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
        for (int b_c_inner = 0; b_c_inner < 4; ++b_c_inner) {
          for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
            compute_local[((((b_c_outer_inner * 8) + (b_c_inner * 2)) + i_c_inner))] = (compute_local[((((b_c_outer_inner * 8) + (b_c_inner * 2)) + i_c_inner))] + (placeholder_d_shared[(((((b_c_outer_inner * 32) + (b_c_inner * 8)) + (i_c_inner * 2)) + k_outer_inner))] * placeholder_shared[(((((b_c_outer_inner * 256) + (b_c_inner * 64)) + (((int)threadIdx.x) * 2)) + k_outer_inner))]));
            compute_local[(((((b_c_outer_inner * 8) + (b_c_inner * 2)) + i_c_inner) + 32))] = (compute_local[(((((b_c_outer_inner * 8) + (b_c_inner * 2)) + i_c_inner) + 32))] + (placeholder_d_shared[((((((b_c_outer_inner * 32) + (b_c_inner * 8)) + (i_c_inner * 2)) + k_outer_inner) + 4))] * placeholder_shared[(((((b_c_outer_inner * 256) + (b_c_inner * 64)) + (((int)threadIdx.x) * 2)) + k_outer_inner))]));
          }
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 16; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      compute[((((((b_inner * 4096) + ((((int)blockIdx.x) >> 1) * 256)) + (i_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)))] = compute_local[(((b_inner * 2) + i_inner))];
      compute[(((((((b_inner * 4096) + ((((int)blockIdx.x) >> 1) * 256)) + (i_inner * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 128))] = compute_local[((((b_inner * 2) + i_inner) + 32))];
    }
  }
}


