
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[196];
  __shared__ float PaddedInput_shared[98];
  __shared__ float placeholder_shared[256];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 28))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 56))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 84))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 112))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 140))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 168))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 29))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 57))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 85))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 113))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 141))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + 169))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 1024; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 98) {
      PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)threadIdx.x) / 7) * 14336) + (((int)blockIdx.x) * 7168)) + ((((int)threadIdx.x) % 7) * 1024)) + rc_outer_outer))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 256) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 128))];
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        Conv2dOutput[(((yy_outer_inner * 4) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 4) + (ff_outer_inner * 2)))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 28))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 28))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 56))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 56))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 84))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 84))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 112))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 112))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 140))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 140))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 168))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 168))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 29))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 29))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 57))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 57))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 2))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 85))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 85))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 3))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 113))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 113))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 4))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 141))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 141))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 5))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 169))] = (Conv2dOutput[((((yy_outer_inner * 4) + (ff_outer_inner * 2)) + 169))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 49) + (yy_outer_inner * 7)) + 6))] * placeholder_shared[(((((((int)threadIdx.x) & 63) * 4) + (ff_outer_inner * 2)) + 1))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[(((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 4) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner) + 256))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 28))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner) + 512))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 56))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner) + 768))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 84))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner) + 1024))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 112))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner) + 1280))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 140))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 6) * 25088) + (ax1_inner * 3584)) + (((int)blockIdx.x) * 1792)) + ((((int)threadIdx.x) & 63) * 4)) + ax3_inner) + 1536))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 168))] + placeholder2[((((((int)threadIdx.x) & 63) * 4) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


