
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[1024];
  __shared__ float placeholder_shared[160];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(64)] = 0.000000e+00f;
  Conv2dOutput[(65)] = 0.000000e+00f;
  Conv2dOutput[(66)] = 0.000000e+00f;
  Conv2dOutput[(67)] = 0.000000e+00f;
  Conv2dOutput[(68)] = 0.000000e+00f;
  Conv2dOutput[(69)] = 0.000000e+00f;
  Conv2dOutput[(70)] = 0.000000e+00f;
  Conv2dOutput[(71)] = 0.000000e+00f;
  Conv2dOutput[(80)] = 0.000000e+00f;
  Conv2dOutput[(81)] = 0.000000e+00f;
  Conv2dOutput[(82)] = 0.000000e+00f;
  Conv2dOutput[(83)] = 0.000000e+00f;
  Conv2dOutput[(84)] = 0.000000e+00f;
  Conv2dOutput[(85)] = 0.000000e+00f;
  Conv2dOutput[(86)] = 0.000000e+00f;
  Conv2dOutput[(87)] = 0.000000e+00f;
  Conv2dOutput[(96)] = 0.000000e+00f;
  Conv2dOutput[(97)] = 0.000000e+00f;
  Conv2dOutput[(98)] = 0.000000e+00f;
  Conv2dOutput[(99)] = 0.000000e+00f;
  Conv2dOutput[(100)] = 0.000000e+00f;
  Conv2dOutput[(101)] = 0.000000e+00f;
  Conv2dOutput[(102)] = 0.000000e+00f;
  Conv2dOutput[(103)] = 0.000000e+00f;
  Conv2dOutput[(112)] = 0.000000e+00f;
  Conv2dOutput[(113)] = 0.000000e+00f;
  Conv2dOutput[(114)] = 0.000000e+00f;
  Conv2dOutput[(115)] = 0.000000e+00f;
  Conv2dOutput[(116)] = 0.000000e+00f;
  Conv2dOutput[(117)] = 0.000000e+00f;
  Conv2dOutput[(118)] = 0.000000e+00f;
  Conv2dOutput[(119)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  Conv2dOutput[(72)] = 0.000000e+00f;
  Conv2dOutput[(73)] = 0.000000e+00f;
  Conv2dOutput[(74)] = 0.000000e+00f;
  Conv2dOutput[(75)] = 0.000000e+00f;
  Conv2dOutput[(76)] = 0.000000e+00f;
  Conv2dOutput[(77)] = 0.000000e+00f;
  Conv2dOutput[(78)] = 0.000000e+00f;
  Conv2dOutput[(79)] = 0.000000e+00f;
  Conv2dOutput[(88)] = 0.000000e+00f;
  Conv2dOutput[(89)] = 0.000000e+00f;
  Conv2dOutput[(90)] = 0.000000e+00f;
  Conv2dOutput[(91)] = 0.000000e+00f;
  Conv2dOutput[(92)] = 0.000000e+00f;
  Conv2dOutput[(93)] = 0.000000e+00f;
  Conv2dOutput[(94)] = 0.000000e+00f;
  Conv2dOutput[(95)] = 0.000000e+00f;
  Conv2dOutput[(104)] = 0.000000e+00f;
  Conv2dOutput[(105)] = 0.000000e+00f;
  Conv2dOutput[(106)] = 0.000000e+00f;
  Conv2dOutput[(107)] = 0.000000e+00f;
  Conv2dOutput[(108)] = 0.000000e+00f;
  Conv2dOutput[(109)] = 0.000000e+00f;
  Conv2dOutput[(110)] = 0.000000e+00f;
  Conv2dOutput[(111)] = 0.000000e+00f;
  Conv2dOutput[(120)] = 0.000000e+00f;
  Conv2dOutput[(121)] = 0.000000e+00f;
  Conv2dOutput[(122)] = 0.000000e+00f;
  Conv2dOutput[(123)] = 0.000000e+00f;
  Conv2dOutput[(124)] = 0.000000e+00f;
  Conv2dOutput[(125)] = 0.000000e+00f;
  Conv2dOutput[(126)] = 0.000000e+00f;
  Conv2dOutput[(127)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 50; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[((((int)threadIdx.x) * 40))] = placeholder[(((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 1))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 2))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 3))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 3))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 4))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 200))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 5))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 201))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 6))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 202))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 7))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 203))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 8))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 400))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 9))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 401))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 10))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 402))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 11))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 403))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 12))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 600))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 13))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 601))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 14))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 602))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 15))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 603))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 16))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 800))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 17))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 801))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 18))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 802))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 19))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 803))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 20))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1000))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 21))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1001))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 22))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1002))];
    }
    if (((int)threadIdx.x) < 26) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 23))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1003))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 24))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1200))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 25))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1201))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 26))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1202))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 27))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1203))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 28))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1400))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 29))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1401))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 30))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1402))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 31))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1403))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 32))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1600))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 33))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1601))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 34))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1602))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 35))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1603))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 36))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1800))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 37))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1801))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 38))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1802))];
    }
    if (((int)threadIdx.x) < 25) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 39))] = placeholder[((((((int)threadIdx.x) * 2000) + (rc_outer_outer * 4)) + 1803))];
    }
    if (((int)threadIdx.x) < 40) {
      placeholder_shared[((((int)threadIdx.x) * 4))] = placeholder1[(((((rc_outer_outer * 320) + ((((int)threadIdx.x) / 10) * 80)) + (((int)blockIdx.x) * 40)) + ((((int)threadIdx.x) % 10) * 4)))];
    }
    if (((int)threadIdx.x) < 40) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder1[(((((rc_outer_outer * 320) + ((((((int)threadIdx.x) * 4) + 1) / 40) * 80)) + (((int)blockIdx.x) * 40)) + (((((int)threadIdx.x) * 4) + 1) % 40)))];
    }
    if (((int)threadIdx.x) < 40) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder1[(((((rc_outer_outer * 320) + ((((((int)threadIdx.x) * 4) + 2) / 40) * 80)) + (((int)blockIdx.x) * 40)) + (((((int)threadIdx.x) * 4) + 2) % 40)))];
    }
    if (((int)threadIdx.x) < 40) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder1[(((((rc_outer_outer * 320) + ((((((int)threadIdx.x) * 4) + 3) / 40) * 80)) + (((int)blockIdx.x) * 40)) + (((((int)threadIdx.x) * 4) + 3) % 40)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 256))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 320))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 384))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 448))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 260))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 324))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 388))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[(((((int)threadIdx.x) % 5) * 8))]));
    Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 1))]));
    Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 2))]));
    Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 3))]));
    Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 4))]));
    Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 5))]));
    Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 6))]));
    Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 452))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 7))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 257))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 321))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 385))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 449))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 261))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 325))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 389))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 40))]));
    Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 41))]));
    Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 42))]));
    Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 43))]));
    Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 44))]));
    Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 45))]));
    Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 46))]));
    Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 453))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 47))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 258))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 322))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 386))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 450))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 262))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 326))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 390))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 80))]));
    Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 81))]));
    Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 82))]));
    Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 83))]));
    Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 84))]));
    Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 85))]));
    Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 86))]));
    Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 454))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 87))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 259))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 323))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 387))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 451))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 263))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 327))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 391))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
    Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 120))]));
    Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 121))]));
    Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 122))]));
    Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 123))]));
    Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 124))]));
    Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 125))]));
    Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 126))]));
    Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 40) * 512) + (((((int)threadIdx.x) % 40) / 5) * 8)) + 455))] * placeholder_shared[((((((int)threadIdx.x) % 5) * 8) + 127))]));
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
        T_add[(((((((((((int)threadIdx.x) / 40) * 10240) + (ax1_inner * 1280)) + (((((int)threadIdx.x) % 40) / 5) * 160)) + (ax2_inner * 80)) + (((int)blockIdx.x) * 40)) + ((((int)threadIdx.x) % 5) * 8)) + ax3_inner))] = ((Conv2dOutput[((((ax1_inner * 16) + (ax2_inner * 8)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) % 5) * 8)) + ax3_inner))]) + placeholder3[(((((((((((int)threadIdx.x) / 40) * 10240) + (ax1_inner * 1280)) + (((((int)threadIdx.x) % 40) / 5) * 160)) + (ax2_inner * 80)) + (((int)blockIdx.x) * 40)) + ((((int)threadIdx.x) % 5) * 8)) + ax3_inner))]);
      }
    }
  }
}


