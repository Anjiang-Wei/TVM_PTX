
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[96];
  __shared__ float PaddedInput_shared[8064];
  __shared__ float placeholder_shared[432];
  for (int b_outer_inner_init = 0; b_outer_inner_init < 4; ++b_outer_inner_init) {
    for (int c_outer_inner_init = 0; c_outer_inner_init < 4; ++c_outer_inner_init) {
      DepthwiseConv2d[(((b_outer_inner_init * 4) + c_outer_inner_init))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 4) + c_outer_inner_init) + 16))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 4) + c_outer_inner_init) + 32))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 4) + c_outer_inner_init) + 48))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 4) + c_outer_inner_init) + 64))] = 0.000000e+00f;
      DepthwiseConv2d[((((b_outer_inner_init * 4) + c_outer_inner_init) + 80))] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 17; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 480) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 8064) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + ((int)threadIdx.x)) < 336) {
          PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 480) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 144) * 144) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 10) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 48)) % 3) * 48)) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 48)))] = (((((1 <= (((((int)blockIdx.x) / 120) * 5) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 480) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 1008) / 144))) && ((((((int)blockIdx.x) / 120) * 5) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 480) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 1008) / 144)) < 16)) && (1 <= (((((int)blockIdx.x) % 120) >> 3) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 10) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 48)) % 3)))) && ((((((int)blockIdx.x) % 120) >> 3) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 10) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 48)) % 3)) < 16)) ? placeholder[((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 480) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 1008) * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 480) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 1008) / 144) * 5760)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 10) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 48)) % 3) * 384)) + ((((int)blockIdx.x) % 120) * 48)) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 48)) - 6144))] : 0.000000e+00f);
        }
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) < 432) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 72) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) / 48) * 384) + ((((int)blockIdx.x) & 7) * 48)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) % 48)))];
      }
    }
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) < 431) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 72) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 1))] = placeholder1[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 1) / 48) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 1) % 48)))];
      }
    }
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) < 430) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 72) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 2))] = placeholder1[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 2) / 48) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 2) % 48)))];
      }
    }
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) < 429) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 72) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 3))] = placeholder1[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 3) / 48) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 3) % 48)))];
      }
    }
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) < 428) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 72) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 4))] = placeholder1[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 4) / 48) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 4) % 48)))];
      }
    }
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) < 427) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 20) + ((int)threadIdx.x)) < 72) {
        placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 5))] = placeholder1[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 5) / 48) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 120) + (((int)threadIdx.x) * 6)) + 5) % 48)))];
      }
    }
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int dj_outer_inner = 0; dj_outer_inner < 3; ++dj_outer_inner) {
      for (int b_outer_inner = 0; b_outer_inner < 4; ++b_outer_inner) {
        for (int c_outer_inner = 0; c_outer_inner < 4; ++c_outer_inner) {
          DepthwiseConv2d[(((b_outer_inner * 4) + c_outer_inner))] = (DepthwiseConv2d[(((b_outer_inner * 4) + c_outer_inner))] + (PaddedInput_shared[(((((((b_outer_inner * 1008) + ((((int)threadIdx.x) >> 2) * 144)) + (di_outer_inner * 144)) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner))] * placeholder_shared[(((((di_outer_inner * 144) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner))]));
          DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 16))] = (DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 16))] + (PaddedInput_shared[((((((((b_outer_inner * 1008) + ((((int)threadIdx.x) >> 2) * 144)) + (di_outer_inner * 144)) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 16))] * placeholder_shared[((((((di_outer_inner * 144) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 16))]));
          DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 32))] = (DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 32))] + (PaddedInput_shared[((((((((b_outer_inner * 1008) + ((((int)threadIdx.x) >> 2) * 144)) + (di_outer_inner * 144)) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 32))] * placeholder_shared[((((((di_outer_inner * 144) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 32))]));
          DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 48))] = (DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 48))] + (PaddedInput_shared[((((((((b_outer_inner * 1008) + ((((int)threadIdx.x) >> 2) * 144)) + (di_outer_inner * 144)) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 4032))] * placeholder_shared[(((((di_outer_inner * 144) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner))]));
          DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 64))] = (DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 64))] + (PaddedInput_shared[((((((((b_outer_inner * 1008) + ((((int)threadIdx.x) >> 2) * 144)) + (di_outer_inner * 144)) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 4048))] * placeholder_shared[((((((di_outer_inner * 144) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 16))]));
          DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 80))] = (DepthwiseConv2d[((((b_outer_inner * 4) + c_outer_inner) + 80))] + (PaddedInput_shared[((((((((b_outer_inner * 1008) + ((((int)threadIdx.x) >> 2) * 144)) + (di_outer_inner * 144)) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 4064))] * placeholder_shared[((((((di_outer_inner * 144) + (dj_outer_inner * 48)) + ((((int)threadIdx.x) & 3) * 4)) + c_outer_inner) + 32))]));
        }
      }
    }
  }
  for (int i0_inner = 0; i0_inner < 4; ++i0_inner) {
    for (int i3_inner = 0; i3_inner < 4; ++i3_inner) {
      compute[(((((((i0_inner * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((int)threadIdx.x) >> 2) * 5760)) + ((((int)blockIdx.x) % 120) * 48)) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner))] = max(min((DepthwiseConv2d[(((i0_inner * 4) + i3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[((((((((i0_inner * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((int)threadIdx.x) >> 2) * 5760)) + ((((int)blockIdx.x) % 120) * 48)) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 16))] = max(min((DepthwiseConv2d[((((i0_inner * 4) + i3_inner) + 16))] + placeholder2[((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 16))]), 6.000000e+00f), 0.000000e+00f);
      compute[((((((((i0_inner * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((int)threadIdx.x) >> 2) * 5760)) + ((((int)blockIdx.x) % 120) * 48)) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 32))] = max(min((DepthwiseConv2d[((((i0_inner * 4) + i3_inner) + 32))] + placeholder2[((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 32))]), 6.000000e+00f), 0.000000e+00f);
      compute[((((((((i0_inner * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((int)threadIdx.x) >> 2) * 5760)) + ((((int)blockIdx.x) % 120) * 48)) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 345600))] = max(min((DepthwiseConv2d[((((i0_inner * 4) + i3_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[((((((((i0_inner * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((int)threadIdx.x) >> 2) * 5760)) + ((((int)blockIdx.x) % 120) * 48)) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 345616))] = max(min((DepthwiseConv2d[((((i0_inner * 4) + i3_inner) + 64))] + placeholder2[((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 16))]), 6.000000e+00f), 0.000000e+00f);
      compute[((((((((i0_inner * 86400) + ((((int)blockIdx.x) / 120) * 28800)) + ((((int)threadIdx.x) >> 2) * 5760)) + ((((int)blockIdx.x) % 120) * 48)) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 345632))] = max(min((DepthwiseConv2d[((((i0_inner * 4) + i3_inner) + 80))] + placeholder2[((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 4)) + i3_inner) + 32))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


