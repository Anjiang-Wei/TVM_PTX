
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((int)blockIdx.x))] = ((DepthwiseConv2d[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 184))]) * (max(min(((DepthwiseConv2d[(((int)blockIdx.x))] + placeholder[((((int)blockIdx.x) % 184))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[120];
  __shared__ float PaddedInput_shared[2720];
  __shared__ float placeholder_shared[12];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 15; ++i_c_inner_init) {
      for (int c_c_inner_init = 0; c_c_inner_init < 4; ++c_c_inner_init) {
        DepthwiseConv2d_local[((((b_c_outer_inner_init * 60) + (i_c_inner_init * 4)) + c_c_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 68; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 340) * 340) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 20)) % 17) * 20)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 20)))] = (((((1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 20)) % 17)) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 20)) % 17) < 16)) && (1 <= ((((((int)blockIdx.x) / 46) * 5) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 20) >> 2)) + dj_outer_outer))) && (((((((int)blockIdx.x) / 46) * 5) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 20) >> 2)) + dj_outer_outer) < 16)) ? placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 340) * 41400) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 20)) % 17) * 2760)) + ((((int)blockIdx.x) / 46) * 920)) + (((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 20) >> 2) * 184)) + (dj_outer_outer * 184)) + ((((int)blockIdx.x) % 46) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)) - 2944))] : 0.000000e+00f);
      }
    }
    if (((int)threadIdx.x) < 12) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((int)threadIdx.x) >> 2) * 552) + (dj_outer_outer * 184)) + ((((int)blockIdx.x) % 46) * 4)) + (((int)threadIdx.x) & 3)))];
    }
    __syncthreads();
    for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
      for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 15; ++i_c_inner) {
          for (int c_c_inner = 0; c_c_inner < 4; ++c_c_inner) {
            DepthwiseConv2d_local[((((b_c_outer_inner * 60) + (i_c_inner * 4)) + c_c_inner))] = (DepthwiseConv2d_local[((((b_c_outer_inner * 60) + (i_c_inner * 4)) + c_c_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 5) * 680) + (b_c_outer_inner * 340)) + (i_c_inner * 20)) + (di_outer_inner * 20)) + ((((int)threadIdx.x) % 5) * 4)) + c_c_inner))] * placeholder_shared[(((di_outer_inner * 4) + c_c_inner))]));
          }
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 15; ++i_inner) {
      for (int c_inner = 0; c_inner < 4; ++c_inner) {
        DepthwiseConv2d[(((((((((((int)threadIdx.x) / 5) * 82800) + (b_inner * 41400)) + (i_inner * 2760)) + ((((int)blockIdx.x) / 46) * 920)) + ((((int)threadIdx.x) % 5) * 184)) + ((((int)blockIdx.x) % 46) * 4)) + c_inner))] = DepthwiseConv2d_local[((((b_inner * 60) + (i_inner * 4)) + c_inner))];
      }
    }
  }
}


