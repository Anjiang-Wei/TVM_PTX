
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[64];
  __shared__ float PaddedInput_shared[196];
  __shared__ float placeholder_shared[256];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 2; ++ff_c_outer_inner_init) {
    for (int ff_c_inner_init = 0; ff_c_inner_init < 32; ++ff_c_inner_init) {
      Conv2dOutput_local[(((ff_c_outer_inner_init * 32) + ff_c_inner_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 256; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 49) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 100352) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 49) + ((int)threadIdx.x)) / 28) * 14336)) + (((((int)blockIdx.x) & 15) >> 3) * 7168)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 49) + ((int)threadIdx.x)) % 28) >> 2) * 1024)) + (rc_outer_outer * 4)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 49) + ((int)threadIdx.x)) & 3)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 256) {
        if (((int)threadIdx.x) < 32) {
          placeholder_shared[(((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 2048) + ((((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 8) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 63)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_c_outer_inner = 0; ff_c_outer_inner < 2; ++ff_c_outer_inner) {
        for (int ff_c_inner = 0; ff_c_inner < 32; ++ff_c_inner) {
          Conv2dOutput_local[(((ff_c_outer_inner * 32) + ff_c_inner))] = (Conv2dOutput_local[(((ff_c_outer_inner * 32) + ff_c_inner))] + (PaddedInput_shared[(((((int)threadIdx.x) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + (ff_c_outer_inner * 32)) + ff_c_inner))]));
        }
      }
    }
  }
  for (int ff_inner = 0; ff_inner < 64; ++ff_inner) {
    Conv2dOutput[((((((((((int)blockIdx.x) >> 4) * 50176) + ((((int)threadIdx.x) / 7) * 7168)) + (((((int)blockIdx.x) & 15) >> 3) * 3584)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner))] = Conv2dOutput_local[(ff_inner)];
  }
}


