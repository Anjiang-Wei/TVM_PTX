
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[256];
  for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
    Conv2dOutput[(ff_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((ff_inner_init + 4))] = 0.000000e+00f;
    Conv2dOutput[((ff_inner_init + 8))] = 0.000000e+00f;
    Conv2dOutput[((ff_inner_init + 12))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 18; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) >> 7) * 147456) + ((((int)blockIdx.x) >> 4) * 2304)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) & 127) >> 5) * 576)) + (rc_outer_outer * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) & 31)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 4096) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128)) + ((((int)blockIdx.x) & 15) * 8)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
        Conv2dOutput[(ff_inner)] = (Conv2dOutput[(ff_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 128) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + ff_inner))]));
        Conv2dOutput[((ff_inner + 4))] = (Conv2dOutput[((ff_inner + 4))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 128) + rc_outer_inner) + 32))] * placeholder_shared[((((rc_outer_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + ff_inner))]));
        Conv2dOutput[((ff_inner + 8))] = (Conv2dOutput[((ff_inner + 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 128) + rc_outer_inner) + 64))] * placeholder_shared[((((rc_outer_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + ff_inner))]));
        Conv2dOutput[((ff_inner + 12))] = (Conv2dOutput[((ff_inner + 12))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 128) + rc_outer_inner) + 96))] * placeholder_shared[((((rc_outer_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + ff_inner))]));
      }
    }
  }
  for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
    T_relu[(((((((((int)threadIdx.x) >> 1) * 32768) + ((((int)blockIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 15) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))]), 0.000000e+00f);
    T_relu[((((((((((int)threadIdx.x) >> 1) * 32768) + ((((int)blockIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 15) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 128))] = max((Conv2dOutput[((ax3_inner + 4))] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))]), 0.000000e+00f);
    T_relu[((((((((((int)threadIdx.x) >> 1) * 32768) + ((((int)blockIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 15) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 256))] = max((Conv2dOutput[((ax3_inner + 8))] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))]), 0.000000e+00f);
    T_relu[((((((((((int)threadIdx.x) >> 1) * 32768) + ((((int)blockIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 15) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 384))] = max((Conv2dOutput[((ax3_inner + 12))] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))]), 0.000000e+00f);
  }
}


