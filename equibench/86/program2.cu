
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_tanh) {
  float conv2d_transpose_nhwc[12];
  __shared__ float PadInput_shared[400];
  __shared__ float placeholder_shared[96];
  for (int h_outer_inner_init = 0; h_outer_inner_init < 4; ++h_outer_inner_init) {
    conv2d_transpose_nhwc[(h_outer_inner_init)] = 0.000000e+00f;
    conv2d_transpose_nhwc[((h_outer_inner_init + 4))] = 0.000000e+00f;
    conv2d_transpose_nhwc[((h_outer_inner_init + 8))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) < 200) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)) < 400) {
          PadInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)))] = (((((1 <= ((((((int)blockIdx.x) % 25) / 5) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 100) / 10))) && (((((((int)blockIdx.x) % 25) / 5) * 8) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 100) / 10)) < 41)) && (1 <= (((((int)blockIdx.x) % 5) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 10)))) && ((((((int)blockIdx.x) % 5) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 10)) < 41)) ? placeholder[(((((((((((((int)blockIdx.x) / 25) * 409600) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) / 100) * 204800)) + (((((int)blockIdx.x) % 25) / 5) * 40960)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 100) / 10) * 5120)) + ((((int)blockIdx.x) % 5) * 1024)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + (((int)threadIdx.x) >> 1)) % 10) * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) - 5248))] : 0.000000e+00f);
        }
      }
    }
    if (((int)threadIdx.x) < 96) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((int)threadIdx.x) / 6) * 384) + (rc_outer_outer * 6)) + (((int)threadIdx.x) % 6)))];
    }
    __syncthreads();
    for (int rh_outer_inner = 0; rh_outer_inner < 2; ++rh_outer_inner) {
      for (int h_outer_inner = 0; h_outer_inner < 4; ++h_outer_inner) {
        for (int rh_inner = 0; rh_inner < 2; ++rh_inner) {
          for (int rw_inner = 0; rw_inner < 4; ++rw_inner) {
            for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              conv2d_transpose_nhwc[(h_outer_inner)] = (conv2d_transpose_nhwc[(h_outer_inner)] + ((((((h_outer_inner + rh_inner) % 2) == 0) && (((rw_inner + (((int)threadIdx.x) & 15)) % 2) == 0)) ? PadInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (((((int)threadIdx.x) & 63) >> 4) * 40)) + (((h_outer_inner + rh_inner) >> 1) * 20)) + (rh_outer_inner * 20)) + (((rw_inner + (((int)threadIdx.x) & 15)) >> 1) * 2)) + rc_inner))] : 0.000000e+00f) * placeholder_shared[((((((rc_inner * 3) + 90) - (rw_inner * 6)) - (rh_inner * 24)) - (rh_outer_inner * 48)))]));
              conv2d_transpose_nhwc[((h_outer_inner + 4))] = (conv2d_transpose_nhwc[((h_outer_inner + 4))] + ((((((h_outer_inner + rh_inner) % 2) == 0) && (((rw_inner + (((int)threadIdx.x) & 15)) % 2) == 0)) ? PadInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (((((int)threadIdx.x) & 63) >> 4) * 40)) + (((h_outer_inner + rh_inner) >> 1) * 20)) + (rh_outer_inner * 20)) + (((rw_inner + (((int)threadIdx.x) & 15)) >> 1) * 2)) + rc_inner))] : 0.000000e+00f) * placeholder_shared[((((((rc_inner * 3) + 91) - (rw_inner * 6)) - (rh_inner * 24)) - (rh_outer_inner * 48)))]));
              conv2d_transpose_nhwc[((h_outer_inner + 8))] = (conv2d_transpose_nhwc[((h_outer_inner + 8))] + ((((((h_outer_inner + rh_inner) % 2) == 0) && (((rw_inner + (((int)threadIdx.x) & 15)) % 2) == 0)) ? PadInput_shared[((((((((((int)threadIdx.x) >> 6) * 200) + (((((int)threadIdx.x) & 63) >> 4) * 40)) + (((h_outer_inner + rh_inner) >> 1) * 20)) + (rh_outer_inner * 20)) + (((rw_inner + (((int)threadIdx.x) & 15)) >> 1) * 2)) + rc_inner))] : 0.000000e+00f) * placeholder_shared[((((((rc_inner * 3) + 92) - (rw_inner * 6)) - (rh_inner * 24)) - (rh_outer_inner * 48)))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    T_tanh[(((((((((((int)blockIdx.x) / 25) * 38400) + ((((int)threadIdx.x) >> 6) * 19200)) + (((((int)blockIdx.x) % 25) / 5) * 3840)) + (((((int)threadIdx.x) & 63) >> 4) * 960)) + (ax1_inner * 240)) + ((((int)blockIdx.x) % 5) * 48)) + ((((int)threadIdx.x) & 15) * 3)))] = tanhf(conv2d_transpose_nhwc[(ax1_inner)]);
    T_tanh[((((((((((((int)blockIdx.x) / 25) * 38400) + ((((int)threadIdx.x) >> 6) * 19200)) + (((((int)blockIdx.x) % 25) / 5) * 3840)) + (((((int)threadIdx.x) & 63) >> 4) * 960)) + (ax1_inner * 240)) + ((((int)blockIdx.x) % 5) * 48)) + ((((int)threadIdx.x) & 15) * 3)) + 1))] = tanhf(conv2d_transpose_nhwc[((ax1_inner + 4))]);
    T_tanh[((((((((((((int)blockIdx.x) / 25) * 38400) + ((((int)threadIdx.x) >> 6) * 19200)) + (((((int)blockIdx.x) % 25) / 5) * 3840)) + (((((int)threadIdx.x) & 63) >> 4) * 960)) + (ax1_inner * 240)) + ((((int)blockIdx.x) % 5) * 48)) + ((((int)threadIdx.x) & 15) * 3)) + 2))] = tanhf(conv2d_transpose_nhwc[((ax1_inner + 8))]);
  }
}


