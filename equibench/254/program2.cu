
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[1460];
  __shared__ float PaddedInput_shared[4672];
  __shared__ float placeholder_shared[2560];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 73; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 20) + 19))] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2336; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)) >> 6) * 4672) + ((((int)blockIdx.x) >> 1) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)) & 63)))];
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 1280; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)) / 40) * 80) + ((((int)blockIdx.x) & 1) * 40)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)) % 40)))];
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
    for (int yy_outer_inner = 0; yy_outer_inner < 73; ++yy_outer_inner) {
      Conv2dOutput[((yy_outer_inner * 20))] = (Conv2dOutput[((yy_outer_inner * 20))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[(((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 1))] = (Conv2dOutput[(((yy_outer_inner * 20) + 1))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 1))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 2))] = (Conv2dOutput[(((yy_outer_inner * 20) + 2))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 2))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 3))] = (Conv2dOutput[(((yy_outer_inner * 20) + 3))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 3))]));
      Conv2dOutput[((yy_outer_inner * 20))] = (Conv2dOutput[((yy_outer_inner * 20))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 40))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 1))] = (Conv2dOutput[(((yy_outer_inner * 20) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 41))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 2))] = (Conv2dOutput[(((yy_outer_inner * 20) + 2))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 42))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 3))] = (Conv2dOutput[(((yy_outer_inner * 20) + 3))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 43))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 4))] = (Conv2dOutput[(((yy_outer_inner * 20) + 4))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 4))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 5))] = (Conv2dOutput[(((yy_outer_inner * 20) + 5))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 5))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 6))] = (Conv2dOutput[(((yy_outer_inner * 20) + 6))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 6))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 7))] = (Conv2dOutput[(((yy_outer_inner * 20) + 7))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 7))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 4))] = (Conv2dOutput[(((yy_outer_inner * 20) + 4))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 44))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 5))] = (Conv2dOutput[(((yy_outer_inner * 20) + 5))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 45))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 6))] = (Conv2dOutput[(((yy_outer_inner * 20) + 6))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 46))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 7))] = (Conv2dOutput[(((yy_outer_inner * 20) + 7))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 47))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 8))] = (Conv2dOutput[(((yy_outer_inner * 20) + 8))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 8))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 9))] = (Conv2dOutput[(((yy_outer_inner * 20) + 9))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 9))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 10))] = (Conv2dOutput[(((yy_outer_inner * 20) + 10))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 10))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 11))] = (Conv2dOutput[(((yy_outer_inner * 20) + 11))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 11))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 8))] = (Conv2dOutput[(((yy_outer_inner * 20) + 8))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 48))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 9))] = (Conv2dOutput[(((yy_outer_inner * 20) + 9))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 49))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 10))] = (Conv2dOutput[(((yy_outer_inner * 20) + 10))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 50))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 11))] = (Conv2dOutput[(((yy_outer_inner * 20) + 11))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 51))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 12))] = (Conv2dOutput[(((yy_outer_inner * 20) + 12))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 12))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 13))] = (Conv2dOutput[(((yy_outer_inner * 20) + 13))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 13))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 14))] = (Conv2dOutput[(((yy_outer_inner * 20) + 14))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 14))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 15))] = (Conv2dOutput[(((yy_outer_inner * 20) + 15))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 15))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 12))] = (Conv2dOutput[(((yy_outer_inner * 20) + 12))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 52))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 13))] = (Conv2dOutput[(((yy_outer_inner * 20) + 13))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 53))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 14))] = (Conv2dOutput[(((yy_outer_inner * 20) + 14))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 54))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 15))] = (Conv2dOutput[(((yy_outer_inner * 20) + 15))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 55))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 16))] = (Conv2dOutput[(((yy_outer_inner * 20) + 16))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 16))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 17))] = (Conv2dOutput[(((yy_outer_inner * 20) + 17))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 17))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 18))] = (Conv2dOutput[(((yy_outer_inner * 20) + 18))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 18))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 19))] = (Conv2dOutput[(((yy_outer_inner * 20) + 19))] + (PaddedInput_shared[(((yy_outer_inner * 64) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 19))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 16))] = (Conv2dOutput[(((yy_outer_inner * 20) + 16))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 56))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 17))] = (Conv2dOutput[(((yy_outer_inner * 20) + 17))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 57))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 18))] = (Conv2dOutput[(((yy_outer_inner * 20) + 18))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 58))]));
      Conv2dOutput[(((yy_outer_inner * 20) + 19))] = (Conv2dOutput[(((yy_outer_inner * 20) + 19))] + (PaddedInput_shared[((((yy_outer_inner * 64) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 80) + (((int)threadIdx.x) * 20)) + 59))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 73; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 20; ++ax3_inner) {
      T_relu[(((((ax1_inner * 5840) + (((int)blockIdx.x) * 40)) + (((int)threadIdx.x) * 20)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 20) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) * 20)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


