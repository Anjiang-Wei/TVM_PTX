
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[10];
  __shared__ float PaddedInput_shared[1168];
  __shared__ float placeholder_shared[320];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    ((float2*)(PaddedInput_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder + ((((((((int)blockIdx.x) >> 2) * 4672) + ((((int)threadIdx.x) >> 3) * 64)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 292))))[0] = ((float2*)(placeholder + ((((((((int)blockIdx.x) >> 2) * 4672) + ((((((int)threadIdx.x) * 2) + 292) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 2) + 4) & 15)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 584))))[0] = ((float2*)(placeholder + ((((((((int)blockIdx.x) >> 2) * 4672) + ((((((int)threadIdx.x) * 2) + 584) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 2) + 8) & 15)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 876))))[0] = ((float2*)(placeholder + ((((((((int)blockIdx.x) >> 2) * 4672) + ((((((int)threadIdx.x) * 2) + 876) >> 4) * 64)) + (rc_outer_outer * 16)) + (((((int)threadIdx.x) * 2) + 12) & 15)))))[0];
    placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)threadIdx.x) / 10) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + ((((int)threadIdx.x) % 10) * 2)))];
    placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((rc_outer_outer * 1280) + ((((((int)threadIdx.x) * 2) + 1) / 20) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((((int)threadIdx.x) * 2) + 1) % 20)))];
    if (((int)threadIdx.x) < 14) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 292))] = placeholder1[(((((rc_outer_outer * 1280) + ((((((int)threadIdx.x) * 2) + 292) / 20) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((((int)threadIdx.x) * 2) + 12) % 20)))];
    }
    if (((int)threadIdx.x) < 14) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 293))] = placeholder1[(((((rc_outer_outer * 1280) + ((((((int)threadIdx.x) * 2) + 293) / 20) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + (((((int)threadIdx.x) * 2) + 13) % 20)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 2))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 4))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 8))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 12))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 16))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 20))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 24))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 28))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 32))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 36))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 40))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 44))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 48))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 52))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 56))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 60))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 64))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 68))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 72))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 76))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 1))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 5))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 9))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 13))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 1) * 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 17))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 21))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 25))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 29))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 33))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 37))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 41))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 45))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 49))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 53))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 57))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 61))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 65))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 69))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 73))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 77))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 80))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 84))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 88))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 92))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 96))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 100))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 104))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 108))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 112))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 116))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 120))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 124))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 128))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 132))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 136))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 140))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 144))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 148))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 152))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 156))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 81))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 85))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 89))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 93))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 97))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 101))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 105))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 109))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 113))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 117))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 121))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 125))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 129))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 133))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 137))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 141))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 145))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 149))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 153))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 157))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 160))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 164))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 168))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 172))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 176))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 180))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 184))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 188))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 192))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 196))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 200))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 204))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 208))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 212))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 216))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 220))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 224))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 228))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 232))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 236))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 161))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 165))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 169))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 173))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 177))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 181))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 185))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 189))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 193))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 197))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 201))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 205))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 209))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 213))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 217))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 221))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 225))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 229))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 233))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 237))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 240))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 244))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 248))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 252))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 256))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 260))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 264))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 268))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 272))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 276))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 280))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 284))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 288))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 292))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 296))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 300))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 304))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 308))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 312))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 316))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 241))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 245))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 249))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 253))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 257))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 261))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 265))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 269))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 273))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 277))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 281))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 285))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 289))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 293))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 297))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 301))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 305))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 309))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 313))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 16) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 317))]));
  }
  for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
    T_relu[(((((((((int)blockIdx.x) >> 2) * 5840) + ((((int)threadIdx.x) >> 1) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[(((((((int)blockIdx.x) & 3) * 20) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))]), 0.000000e+00f);
    T_relu[((((((((((int)blockIdx.x) >> 2) * 5840) + ((((int)threadIdx.x) >> 1) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 4))] = max((Conv2dOutput[((ax3_inner + 2))] + placeholder2[((((((((int)blockIdx.x) & 3) * 20) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 4))]), 0.000000e+00f);
    T_relu[((((((((((int)blockIdx.x) >> 2) * 5840) + ((((int)threadIdx.x) >> 1) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 8))] = max((Conv2dOutput[((ax3_inner + 4))] + placeholder2[((((((((int)blockIdx.x) & 3) * 20) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 8))]), 0.000000e+00f);
    T_relu[((((((((((int)blockIdx.x) >> 2) * 5840) + ((((int)threadIdx.x) >> 1) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 12))] = max((Conv2dOutput[((ax3_inner + 6))] + placeholder2[((((((((int)blockIdx.x) & 3) * 20) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 12))]), 0.000000e+00f);
    T_relu[((((((((((int)blockIdx.x) >> 2) * 5840) + ((((int)threadIdx.x) >> 1) * 80)) + ((((int)blockIdx.x) & 3) * 20)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 16))] = max((Conv2dOutput[((ax3_inner + 8))] + placeholder2[((((((((int)blockIdx.x) & 3) * 20) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner) + 16))]), 0.000000e+00f);
  }
}


