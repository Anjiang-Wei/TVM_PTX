
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[14];
  __shared__ float PaddedInput_shared[392];
  __shared__ float placeholder_shared[2];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 192; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 28; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 14) + ((int)threadIdx.x)))] = placeholder[(((((((((int)blockIdx.x) >> 6) * 75264) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2688)) + ((((int)threadIdx.x) >> 1) * 384)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    if (((int)threadIdx.x) < 1) {
      int2 _1 = (make_int2)(((((((int)threadIdx.x) * 24576) + (rc_outer_outer * 128)) + (((int)blockIdx.x) & 63)))+(64*0), ((((((int)threadIdx.x) * 24576) + (rc_outer_outer * 128)) + (((int)blockIdx.x) & 63)))+(64*1));
      ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = make_float2(placeholder1[_1.x],placeholder1[_1.y]);
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 28) + rc_inner))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 2))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 4))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 6))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 8))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 10))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 12))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 14))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 16))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 18))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 20))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 22))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 24))] * placeholder_shared[(rc_inner)]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) * 28) + rc_inner) + 26))] * placeholder_shared[(rc_inner)]));
    }
  }
  T_add[(((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)))] = (placeholder2[(((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)))] + (Conv2dOutput[(0)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 64))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 64))] + (Conv2dOutput[(1)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 128))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 128))] + (Conv2dOutput[(2)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 192))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 192))] + (Conv2dOutput[(3)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 256))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 256))] + (Conv2dOutput[(4)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 320))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 320))] + (Conv2dOutput[(5)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 384))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 384))] + (Conv2dOutput[(6)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 448))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 448))] + (Conv2dOutput[(7)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 512))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 512))] + (Conv2dOutput[(8)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 576))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 576))] + (Conv2dOutput[(9)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 640))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 640))] + (Conv2dOutput[(10)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 704))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 704))] + (Conv2dOutput[(11)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 768))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 768))] + (Conv2dOutput[(12)] + placeholder3[((((int)blockIdx.x) & 63))]));
  T_add[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 832))] = (placeholder2[((((((((int)blockIdx.x) >> 6) * 12544) + (((int)threadIdx.x) * 896)) + (((int)blockIdx.x) & 63)) + 832))] + (Conv2dOutput[(13)] + placeholder3[((((int)blockIdx.x) & 63))]));
}


