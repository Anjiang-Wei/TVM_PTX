
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ Conv2dOutput, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((Conv2dOutput[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 240))]) * (max(min(((Conv2dOutput[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 240))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[1440];
  __shared__ float PaddedInput_shared[1200];
  __shared__ float placeholder_shared[240];
  for (int ff_c_outer_inner_init = 0; ff_c_outer_inner_init < 6; ++ff_c_outer_inner_init) {
    for (int nn_c_inner_init = 0; nn_c_inner_init < 2; ++nn_c_inner_init) {
      for (int xx_c_inner_init = 0; xx_c_inner_init < 30; ++xx_c_inner_init) {
        Conv2dOutput_local[((((nn_c_inner_init * 720) + (xx_c_inner_init * 24)) + (ff_c_outer_inner_init * 4)))] = 0.000000e+00f;
        Conv2dOutput_local[(((((nn_c_inner_init * 720) + (xx_c_inner_init * 24)) + (ff_c_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
        Conv2dOutput_local[(((((nn_c_inner_init * 720) + (xx_c_inner_init * 24)) + (ff_c_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
        Conv2dOutput_local[(((((nn_c_inner_init * 720) + (xx_c_inner_init * 24)) + (ff_c_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 20; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 50))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 1000))];
    PaddedInput_shared[((((int)threadIdx.x) + 100))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 2000))];
    PaddedInput_shared[((((int)threadIdx.x) + 150))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 3000))];
    PaddedInput_shared[((((int)threadIdx.x) + 200))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 4000))];
    PaddedInput_shared[((((int)threadIdx.x) + 250))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 5000))];
    PaddedInput_shared[((((int)threadIdx.x) + 300))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 6000))];
    PaddedInput_shared[((((int)threadIdx.x) + 350))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 7000))];
    PaddedInput_shared[((((int)threadIdx.x) + 400))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 8000))];
    PaddedInput_shared[((((int)threadIdx.x) + 450))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 9000))];
    PaddedInput_shared[((((int)threadIdx.x) + 500))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 10000))];
    PaddedInput_shared[((((int)threadIdx.x) + 550))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 11000))];
    PaddedInput_shared[((((int)threadIdx.x) + 600))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + ((((int)threadIdx.x) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 36000))];
    PaddedInput_shared[((((int)threadIdx.x) + 650))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 650) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 25) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 700))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 700) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 50) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 750))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 750) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 75) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 800))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 800) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 100) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 850))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 850) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 125) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 900))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 900) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 150) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 950))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 950) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 175) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1000))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 1000) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 200) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1050))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 1050) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 225) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1100))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 1100) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 250) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1150))] = placeholder[((((((((((int)blockIdx.x) / 6) * 72000) + (((((int)threadIdx.x) + 1150) / 600) * 36000)) + (((((int)blockIdx.x) % 6) >> 1) * 12000)) + (((((int)threadIdx.x) >> 1) + 275) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[((((int)threadIdx.x) * 60))] = placeholder1[(((((rc_outer_outer * 480) + ((((int)threadIdx.x) >> 1) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + ((((int)threadIdx.x) & 1) * 60)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 1))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 1) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 1) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 2))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 2) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 2) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 3))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 3) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 3) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 4))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 4) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 4) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 5))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 5) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 5) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 6))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 6) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 6) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 7))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 7) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 7) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 8))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 8) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 8) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 9))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 9) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 9) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 10))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 10) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 10) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 11))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 11) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 11) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 12))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 12) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 12) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 13))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 13) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 13) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 14))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 14) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 14) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 15))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 15) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 15) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 16))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 16) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 16) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 17))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 17) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 17) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 18))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 18) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 18) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 19))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 19) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 19) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 20))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 20) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 20) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 21))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 21) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 21) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 22))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 22) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 22) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 23))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 23) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 23) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 24))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 24) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 24) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 25))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 25) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 25) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 26))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 26) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 26) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 27))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 27) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 27) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 28))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 28) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 28) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 29))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 29) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 29) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 30))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 30) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 30) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 31))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 31) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 31) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 32))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 32) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 32) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 33))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 33) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 33) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 34))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 34) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 34) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 35))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 35) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 35) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 36))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 36) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 36) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 37))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 37) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 37) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 38))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 38) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 38) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 39))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 39) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 39) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 40))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 40) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 40) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 41))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 41) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 41) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 42))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 42) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 42) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 43))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 43) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 43) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 44))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 44) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 44) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 45))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 45) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 45) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 46))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 46) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 46) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 47))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 47) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 47) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 48))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 48) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 48) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 49))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 49) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 49) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 50))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 50) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 50) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 51))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 51) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 51) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 52))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 52) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 52) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 53))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 53) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 53) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 54))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 54) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 54) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 55))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 55) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 55) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 56))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 56) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 56) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 57))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 57) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 57) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 58))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 58) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 58) % 120)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 59))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 59) / 120) * 240)) + ((((int)blockIdx.x) & 1) * 120)) + (((((int)threadIdx.x) * 60) + 59) % 120)))];
    }
    __syncthreads();
    for (int ff_c_outer_inner = 0; ff_c_outer_inner < 6; ++ff_c_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int nn_c_inner = 0; nn_c_inner < 2; ++nn_c_inner) {
          for (int xx_c_inner = 0; xx_c_inner < 30; ++xx_c_inner) {
            Conv2dOutput_local[((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)))] = (Conv2dOutput_local[((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)))] + (PaddedInput_shared[(((((nn_c_inner * 600) + ((((int)threadIdx.x) / 5) * 60)) + (xx_c_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_inner * 120) + ((((int)threadIdx.x) % 5) * 24)) + (ff_c_outer_inner * 4)))]));
            Conv2dOutput_local[(((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)) + 1))] = (Conv2dOutput_local[(((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)) + 1))] + (PaddedInput_shared[(((((nn_c_inner * 600) + ((((int)threadIdx.x) / 5) * 60)) + (xx_c_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_inner * 120) + ((((int)threadIdx.x) % 5) * 24)) + (ff_c_outer_inner * 4)) + 1))]));
            Conv2dOutput_local[(((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)) + 2))] = (Conv2dOutput_local[(((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)) + 2))] + (PaddedInput_shared[(((((nn_c_inner * 600) + ((((int)threadIdx.x) / 5) * 60)) + (xx_c_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_inner * 120) + ((((int)threadIdx.x) % 5) * 24)) + (ff_c_outer_inner * 4)) + 2))]));
            Conv2dOutput_local[(((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)) + 3))] = (Conv2dOutput_local[(((((nn_c_inner * 720) + (xx_c_inner * 24)) + (ff_c_outer_inner * 4)) + 3))] + (PaddedInput_shared[(((((nn_c_inner * 600) + ((((int)threadIdx.x) / 5) * 60)) + (xx_c_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_inner * 120) + ((((int)threadIdx.x) % 5) * 24)) + (ff_c_outer_inner * 4)) + 3))]));
          }
        }
      }
    }
  }
  for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
    for (int xx_inner = 0; xx_inner < 30; ++xx_inner) {
      for (int ff_inner = 0; ff_inner < 24; ++ff_inner) {
        Conv2dOutput[((((((((((((int)blockIdx.x) / 6) * 432000) + (nn_inner * 216000)) + (((((int)blockIdx.x) % 6) >> 1) * 72000)) + ((((int)threadIdx.x) / 5) * 7200)) + (xx_inner * 240)) + ((((int)blockIdx.x) & 1) * 120)) + ((((int)threadIdx.x) % 5) * 24)) + ff_inner))] = Conv2dOutput_local[((((nn_inner * 720) + (xx_inner * 24)) + ff_inner))];
      }
    }
  }
}


