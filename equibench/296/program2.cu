
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[5120];
  __shared__ float placeholder_shared[3200];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
    Conv2dOutput[(yy_outer_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 4))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 8))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 12))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 16))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 20))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 24))] = 0.000000e+00f;
    Conv2dOutput[((yy_outer_inner_init + 28))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 20; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1600) + (((int)threadIdx.x) * 20)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 5120) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + ((int)threadIdx.x)) < 256) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1600) + (((int)threadIdx.x) * 20)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 1280) * 1280) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((((int)threadIdx.x) * 20) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 80)) & 15) * 80)) + (((((int)threadIdx.x) * 20) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 80)))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 15360) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1600) + (((int)threadIdx.x) * 20)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 1280) * 3840)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((((int)threadIdx.x) * 20) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 80)) & 15) * 240)) + (rc_outer_outer * 80)) + (((((int)threadIdx.x) * 20) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 80)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 40; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 80) + ((int)threadIdx.x)))] = placeholder1[((((((rc_outer_outer * 6400) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 160)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 5; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          Conv2dOutput[(yy_outer_inner)] = (Conv2dOutput[(yy_outer_inner)] + (PaddedInput_shared[(((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 4))] = (Conv2dOutput[((yy_outer_inner + 4))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 8))] = (Conv2dOutput[((yy_outer_inner + 8))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 12))] = (Conv2dOutput[((yy_outer_inner + 12))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 16))] = (Conv2dOutput[((yy_outer_inner + 16))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 20))] = (Conv2dOutput[((yy_outer_inner + 20))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 800))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 24))] = (Conv2dOutput[((yy_outer_inner + 24))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 960))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
          Conv2dOutput[((yy_outer_inner + 28))] = (Conv2dOutput[((yy_outer_inner + 28))] + (PaddedInput_shared[((((((yy_outer_inner * 1280) + ((((int)threadIdx.x) / 40) * 80)) + (rc_outer_inner * 16)) + rc_inner) + 1120))] * placeholder_shared[((((rc_outer_inner * 640) + (rc_inner * 40)) + (((int)threadIdx.x) % 40)))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    T_add[(((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)))] = (Conv2dOutput[(ax1_inner)] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 160))] = (Conv2dOutput[((ax1_inner + 4))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 320))] = (Conv2dOutput[((ax1_inner + 8))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 480))] = (Conv2dOutput[((ax1_inner + 12))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 640))] = (Conv2dOutput[((ax1_inner + 16))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 800))] = (Conv2dOutput[((ax1_inner + 20))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 960))] = (Conv2dOutput[((ax1_inner + 24))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
    T_add[((((((((((int)blockIdx.x) >> 1) * 5120) + (ax1_inner * 1280)) + ((((int)threadIdx.x) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((int)threadIdx.x) % 40)) + 1120))] = (Conv2dOutput[((ax1_inner + 28))] + placeholder2[((((((int)blockIdx.x) & 1) * 40) + (((int)threadIdx.x) % 40)))]);
  }
}


