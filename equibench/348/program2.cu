
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
  float Conv3dOutput[32];
  __shared__ float PaddedInput_shared[5280];
  __shared__ float placeholder_shared[144];
  for (int ww_outer_inner_init = 0; ww_outer_inner_init < 2; ++ww_outer_inner_init) {
    Conv3dOutput[((ww_outer_inner_init * 2))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 8))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 16))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 24))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 4))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 12))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 20))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 28))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 9))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 17))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 25))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 5))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 13))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 21))] = 0.000000e+00f;
    Conv3dOutput[(((ww_outer_inner_init * 2) + 29))] = 0.000000e+00f;
  }
  for (int rw_outer_outer = 0; rw_outer_outer < 3; ++rw_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 42; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) < 660) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + (((int)threadIdx.x) >> 2)) < 1320) {
            if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) < 5280) {
              PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)))] = (((((((1 <= (((((int)blockIdx.x) >> 6) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) / 66))) && ((((((int)blockIdx.x) >> 6) * 8) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) / 66)) < 129)) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) % 66))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) % 66) < 65)) && (1 <= (((((((int)blockIdx.x) & 63) >> 4) * 2) + ((((int)threadIdx.x) & 7) >> 2)) + rw_outer_outer))) && ((((((((int)blockIdx.x) & 63) >> 4) * 2) + ((((int)threadIdx.x) & 7) >> 2)) + rw_outer_outer) < 9)) ? placeholder[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) / 66) * 32768)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 16) + (((int)threadIdx.x) >> 3)) % 66) * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (((((int)threadIdx.x) & 7) >> 2) * 64)) + (rw_outer_outer * 64)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 33344))] : 0.000000e+00f);
            }
          }
        }
      }
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((((int)threadIdx.x) >> 4) * 12288) + (rw_outer_outer * 4096)) + (rc_outer_outer * 256)) + (((((int)threadIdx.x) & 15) >> 2) * 64)) + ((((int)blockIdx.x) & 15) * 4)) + (((int)threadIdx.x) & 3)))];
      if (((int)threadIdx.x) < 16) {
        placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[(((((((rw_outer_outer * 4096) + (rc_outer_outer * 256)) + ((((int)threadIdx.x) >> 2) * 64)) + ((((int)blockIdx.x) & 15) * 4)) + (((int)threadIdx.x) & 3)) + 98304))];
      }
      __syncthreads();
      for (int ww_outer_inner = 0; ww_outer_inner < 2; ++ww_outer_inner) {
        for (int cc_outer_inner = 0; cc_outer_inner < 2; ++cc_outer_inner) {
          for (int rd_inner = 0; rd_inner < 3; ++rd_inner) {
            for (int rh_inner = 0; rh_inner < 3; ++rh_inner) {
              for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
                Conv3dOutput[(((ww_outer_inner * 2) + cc_outer_inner))] = (Conv3dOutput[(((ww_outer_inner * 2) + cc_outer_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 8))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 8))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner))] * placeholder_shared[((((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner) + 2))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 16))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 16))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner) + 256))] * placeholder_shared[(((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 24))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 24))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner) + 256))] * placeholder_shared[((((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner) + 2))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 4))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 4))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner) + 8))] * placeholder_shared[(((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 12))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 12))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner) + 8))] * placeholder_shared[((((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner) + 2))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 20))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 20))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner) + 264))] * placeholder_shared[(((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner))]));
                Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 28))] = (Conv3dOutput[((((ww_outer_inner * 2) + cc_outer_inner) + 28))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 4) * 528) + (rd_inner * 528)) + ((((int)threadIdx.x) & 15) * 16)) + (rh_inner * 8)) + (ww_outer_inner * 4)) + rc_inner) + 264))] * placeholder_shared[((((((rd_inner * 48) + (rh_inner * 16)) + (rc_inner * 4)) + cc_outer_inner) + 2))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      for (int ax4_inner = 0; ax4_inner < 2; ++ax4_inner) {
        T_relu[((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner))] = max((((Conv3dOutput[((((ax2_inner * 4) + (ax3_inner * 2)) + ax4_inner))] * placeholder2[((((((int)blockIdx.x) & 15) * 4) + ax4_inner))]) + placeholder3[((((((int)blockIdx.x) & 15) * 4) + ax4_inner))]) + placeholder4[((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner) + 2))] = max((((Conv3dOutput[(((((ax2_inner * 4) + (ax3_inner * 2)) + ax4_inner) + 8))] * placeholder2[(((((((int)blockIdx.x) & 15) * 4) + ax4_inner) + 2))]) + placeholder3[(((((((int)blockIdx.x) & 15) * 4) + ax4_inner) + 2))]) + placeholder4[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner) + 2))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner) + 16384))] = max((((Conv3dOutput[(((((ax2_inner * 4) + (ax3_inner * 2)) + ax4_inner) + 16))] * placeholder2[((((((int)blockIdx.x) & 15) * 4) + ax4_inner))]) + placeholder3[((((((int)blockIdx.x) & 15) * 4) + ax4_inner))]) + placeholder4[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner) + 16384))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner) + 16386))] = max((((Conv3dOutput[(((((ax2_inner * 4) + (ax3_inner * 2)) + ax4_inner) + 24))] * placeholder2[(((((((int)blockIdx.x) & 15) * 4) + ax4_inner) + 2))]) + placeholder3[(((((((int)blockIdx.x) & 15) * 4) + ax4_inner) + 2))]) + placeholder4[(((((((((((((int)blockIdx.x) >> 6) * 262144) + ((((int)threadIdx.x) >> 4) * 32768)) + ((((int)threadIdx.x) & 15) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 63) >> 4) * 128)) + (ax3_inner * 64)) + ((((int)blockIdx.x) & 15) * 4)) + ax4_inner) + 16386))]), 0.000000e+00f);
      }
    }
  }
}


