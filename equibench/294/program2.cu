
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[256];
  __shared__ float PaddedInput_shared[98];
  __shared__ float placeholder_shared[128];
  for (int dd_outer_inner_init = 0; dd_outer_inner_init < 4; ++dd_outer_inner_init) {
    Conv3dOutput[((dd_outer_inner_init * 64))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 1))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 2))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 3))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 4))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 5))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 6))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 7))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 8))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 9))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 10))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 11))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 12))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 13))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 14))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 15))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 16))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 17))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 18))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 19))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 20))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 21))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 22))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 23))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 24))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 25))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 26))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 27))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 28))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 29))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 30))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 31))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 32))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 33))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 34))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 35))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 36))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 37))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 38))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 39))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 40))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 41))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 42))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 43))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 44))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 45))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 46))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 47))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 48))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 49))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 50))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 51))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 52))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 53))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 54))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 55))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 56))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 57))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 58))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 59))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 60))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 61))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 62))] = 0.000000e+00f;
    Conv3dOutput[(((dd_outer_inner_init * 64) + 63))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 3))] = placeholder[(((((((((int)blockIdx.x) >> 4) * 65536) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((int)threadIdx.x) * 3) >> 1) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder[(((((((((int)blockIdx.x) >> 4) * 65536) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((int)threadIdx.x) * 3) + 1) >> 1) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((int)threadIdx.x) * 3) >> 1) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)) + 512))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 12) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 6) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 12) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 6) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 13) / 14) * 14) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 6) % 7) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 13) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 6) % 7) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 3) + 14))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((int)threadIdx.x) * 3) >> 1) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)) + 8192))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 24) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 5) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 24) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 5) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 25) / 14) * 14) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 5) % 7) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 25) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 5) % 7) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 26) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 6) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 26) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 6) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 36) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 4) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 36) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 4) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 37) / 14) * 14) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 4) % 7) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 37) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 4) % 7) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 38) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 5) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 38) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 5) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 48) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 3) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 48) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 3) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 49) / 14) * 14) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 3) % 7) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 49) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 3) % 7) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 50) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 4) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 50) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 4) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 60) / 14) * 14) + ((((((int)threadIdx.x) * 3) >> 1) + 2) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 60) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((int)threadIdx.x) * 3) >> 1) + 2) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 61) / 14) * 14) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 2) % 7) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 61) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((((int)threadIdx.x) * 3) + 1) >> 1) + 2) % 7) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 62) / 14) * 14) + (((((((int)threadIdx.x) * 3) >> 1) + 3) % 7) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 62) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) >> 1) + 3) % 7) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 72) / 14) * 14) + ((((((int)threadIdx.x) * 3) >> 1) + 1) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 72) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((int)threadIdx.x) * 3) >> 1) + 1) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 73) / 14) * 14) + (((((((int)threadIdx.x) * 3) + 1) >> 1) + 1) * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 73) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((((int)threadIdx.x) * 3) + 1) >> 1) + 1) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 74) / 14) * 14) + ((((((int)threadIdx.x) * 3) >> 1) + 2) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 74) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((int)threadIdx.x) * 3) >> 1) + 2) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 3) + 84))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((((int)threadIdx.x) * 3) >> 1) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)) + 49152))];
    PaddedInput_shared[((((((((int)threadIdx.x) * 3) + 85) / 14) * 14) + ((((int)threadIdx.x) * 3) + 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 85) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((int)threadIdx.x) * 3) + 1) >> 1) * 512)) + (rc_outer_outer * 2)) + (((((int)threadIdx.x) * 3) + 1) & 1)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 86) / 14) * 14) + ((((((int)threadIdx.x) * 3) >> 1) + 1) * 2)) + ((((int)threadIdx.x) * 3) & 1)))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 86) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + ((((((int)threadIdx.x) * 3) >> 1) + 1) * 512)) + (rc_outer_outer * 2)) + ((((int)threadIdx.x) * 3) & 1)))];
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 96) / 14) * 14) + (((int)threadIdx.x) * 3)) + 12))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 96) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((int)threadIdx.x) * 3)) + (rc_outer_outer * 2)) + 3072))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((((((int)threadIdx.x) * 3) + 97) / 14) * 14) + (((int)threadIdx.x) * 3)) + 13))] = placeholder[((((((((((int)blockIdx.x) >> 4) * 65536) + ((((((int)threadIdx.x) * 3) + 97) / 14) * 8192)) + (((((int)blockIdx.x) & 15) >> 3) * 4096)) + (((int)threadIdx.x) * 3)) + (rc_outer_outer * 2)) + 3073))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 4))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 4))];
    placeholder_shared[((((int)threadIdx.x) + 8))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 8))];
    placeholder_shared[((((int)threadIdx.x) + 12))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 12))];
    placeholder_shared[((((int)threadIdx.x) + 16))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 16))];
    placeholder_shared[((((int)threadIdx.x) + 20))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 20))];
    placeholder_shared[((((int)threadIdx.x) + 24))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 24))];
    placeholder_shared[((((int)threadIdx.x) + 28))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 28))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 32))];
    placeholder_shared[((((int)threadIdx.x) + 36))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 36))];
    placeholder_shared[((((int)threadIdx.x) + 40))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 40))];
    placeholder_shared[((((int)threadIdx.x) + 44))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 44))];
    placeholder_shared[((((int)threadIdx.x) + 48))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 48))];
    placeholder_shared[((((int)threadIdx.x) + 52))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 52))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 56))];
    placeholder_shared[((((int)threadIdx.x) + 60))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 60))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 7) * 64)) + ((int)threadIdx.x)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 68))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 68) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 4)))];
    placeholder_shared[((((int)threadIdx.x) + 72))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 72) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 8)))];
    placeholder_shared[((((int)threadIdx.x) + 76))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 76) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 12)))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 80) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 84))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 84) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 20)))];
    placeholder_shared[((((int)threadIdx.x) + 88))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 88) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 24)))];
    placeholder_shared[((((int)threadIdx.x) + 92))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 92) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 28)))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 96) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 32)))];
    placeholder_shared[((((int)threadIdx.x) + 100))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 100) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 36)))];
    placeholder_shared[((((int)threadIdx.x) + 104))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 104) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 40)))];
    placeholder_shared[((((int)threadIdx.x) + 108))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 108) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 44)))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 112) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 48)))];
    placeholder_shared[((((int)threadIdx.x) + 116))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 116) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 52)))];
    placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 120) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 56)))];
    placeholder_shared[((((int)threadIdx.x) + 124))] = placeholder1[(((((rc_outer_outer * 1024) + (((((int)threadIdx.x) + 124) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) + 60)))];
    __syncthreads();
    for (int dd_outer_inner = 0; dd_outer_inner < 4; ++dd_outer_inner) {
      for (int hh_outer_inner = 0; hh_outer_inner < 2; ++hh_outer_inner) {
        Conv3dOutput[(((dd_outer_inner * 64) + (hh_outer_inner * 32)))] = (Conv3dOutput[(((dd_outer_inner * 64) + (hh_outer_inner * 32)))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[((((int)threadIdx.x) * 16))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 1))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 1))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 2))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 2))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 3))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 3))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 4))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 4))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 5))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 5))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 6))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 6))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 7))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 7))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 8))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 8))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 9))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 9))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 10))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 10))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 11))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 11))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 12))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 12))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 13))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 13))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 14))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 14))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 15))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 15))] + (PaddedInput_shared[(((dd_outer_inner * 28) + (hh_outer_inner * 8)))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 16))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 16))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[((((int)threadIdx.x) * 16))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 17))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 17))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 18))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 18))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 19))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 19))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 20))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 20))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 21))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 21))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 22))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 22))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 23))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 23))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 24))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 24))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 25))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 25))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 26))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 26))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 27))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 27))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 28))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 28))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 29))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 29))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 30))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 30))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 31))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 31))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 4))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
        Conv3dOutput[(((dd_outer_inner * 64) + (hh_outer_inner * 32)))] = (Conv3dOutput[(((dd_outer_inner * 64) + (hh_outer_inner * 32)))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 64))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 1))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 1))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 65))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 2))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 2))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 66))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 3))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 3))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 67))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 4))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 4))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 68))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 5))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 5))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 69))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 6))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 6))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 70))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 7))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 7))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 71))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 8))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 8))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 72))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 9))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 9))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 73))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 10))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 10))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 74))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 11))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 11))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 75))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 12))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 12))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 76))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 13))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 13))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 77))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 14))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 14))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 78))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 15))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 15))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 79))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 16))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 16))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 64))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 17))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 17))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 65))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 18))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 18))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 66))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 19))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 19))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 67))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 20))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 20))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 68))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 21))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 21))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 69))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 22))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 22))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 70))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 23))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 23))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 71))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 24))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 24))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 72))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 25))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 25))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 73))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 26))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 26))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 74))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 27))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 27))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 75))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 28))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 28))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 76))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 29))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 29))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 77))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 30))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 30))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 78))]));
        Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 31))] = (Conv3dOutput[((((dd_outer_inner * 64) + (hh_outer_inner * 32)) + 31))] + (PaddedInput_shared[((((dd_outer_inner * 28) + (hh_outer_inner * 8)) + 5))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 79))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
      for (int ax4_inner = 0; ax4_inner < 16; ++ax4_inner) {
        T_add[(((((((((((int)blockIdx.x) >> 4) * 16384) + (ax1_inner * 4096)) + (((((int)blockIdx.x) & 15) >> 3) * 2048)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((int)threadIdx.x) * 16)) + ax4_inner))] = ((Conv3dOutput[((((ax1_inner * 64) + (ax2_inner * 16)) + ax4_inner))] * placeholder2[(((((((int)blockIdx.x) & 7) * 64) + (((int)threadIdx.x) * 16)) + ax4_inner))]) + placeholder3[(((((((int)blockIdx.x) & 7) * 64) + (((int)threadIdx.x) * 16)) + ax4_inner))]);
      }
    }
  }
}


