
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[320];
  __shared__ float PaddedInput_shared[1680];
  __shared__ float placeholder_shared[9216];
  for (int nn_inner_init = 0; nn_inner_init < 4; ++nn_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 8; ++ff_inner_init) {
      Conv2dOutput[(((nn_inner_init * 8) + ff_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 32))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 64))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 96))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 128))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 160))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 192))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 224))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 256))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 8) + ff_inner_init) + 288))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 192) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) < 420) {
          if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 768) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 1680) {
            if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 64) + ((int)threadIdx.x)) < 140) {
              PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 768) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = (((1 <= (((((int)blockIdx.x) / 3) * 4) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 192) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 105) / 21))) && (1 <= (((((int)blockIdx.x) % 3) * 20) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 192) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 21)))) ? placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 192) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) / 105) * 921600) + ((((int)blockIdx.x) / 3) * 61440)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 192) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 105) / 21) * 15360)) + ((((int)blockIdx.x) % 3) * 5120)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 192) + (((int)threadIdx.x) * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s >> 2)) % 21) * 256)) + (rc_outer_outer * 4)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_s & 3)) - 15616))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 72; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      ((float2*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + (((int)threadIdx.x) * 2)))))[0] = ((float2*)(placeholder1 + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + (((int)threadIdx.x) * 2)) >> 10) * 65536) + (rc_outer_outer * 1024)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + (((int)threadIdx.x) * 2)) & 1023)))))[0];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          for (int nn_inner = 0; nn_inner < 4; ++nn_inner) {
            for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
              Conv2dOutput[(((nn_inner * 8) + ff_inner))] = (Conv2dOutput[(((nn_inner * 8) + ff_inner))] + (PaddedInput_shared[((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 32))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 32))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 8))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 64))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 64))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 16))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 96))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 96))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 24))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 128))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 128))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 32))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 160))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 160))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 40))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 192))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 192))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 48))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 224))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 224))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 56))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 256))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 256))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 64))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
              Conv2dOutput[((((nn_inner * 8) + ff_inner) + 288))] = (Conv2dOutput[((((nn_inner * 8) + ff_inner) + 288))] + (PaddedInput_shared[(((((((nn_inner * 420) + ((((int)threadIdx.x) >> 5) * 168)) + (ry_inner * 84)) + (rx_inner * 4)) + rc_outer_inner) + 72))] * placeholder_shared[((((((ry_inner * 3072) + (rx_inner * 1024)) + (rc_outer_inner * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ff_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[(((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax0_inner * 8) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 256))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 32))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 512))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 64))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 768))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 96))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 1024))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 128))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 1280))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 160))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 1536))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 192))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 1792))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 224))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 2048))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 256))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((ax0_inner * 230400) + ((((int)blockIdx.x) / 3) * 15360)) + ((((int)threadIdx.x) >> 5) * 7680)) + ((((int)blockIdx.x) % 3) * 2560)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner) + 2304))] = max((Conv2dOutput[((((ax0_inner * 8) + ax3_inner) + 288))] + placeholder2[((((((int)threadIdx.x) & 31) * 8) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


