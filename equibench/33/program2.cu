
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
  float Conv3dOutput[16];
  __shared__ float PaddedInput_shared[108];
  __shared__ float placeholder_shared[144];
  Conv3dOutput[(0)] = 0.000000e+00f;
  Conv3dOutput[(8)] = 0.000000e+00f;
  Conv3dOutput[(1)] = 0.000000e+00f;
  Conv3dOutput[(9)] = 0.000000e+00f;
  Conv3dOutput[(2)] = 0.000000e+00f;
  Conv3dOutput[(10)] = 0.000000e+00f;
  Conv3dOutput[(3)] = 0.000000e+00f;
  Conv3dOutput[(11)] = 0.000000e+00f;
  Conv3dOutput[(4)] = 0.000000e+00f;
  Conv3dOutput[(12)] = 0.000000e+00f;
  Conv3dOutput[(5)] = 0.000000e+00f;
  Conv3dOutput[(13)] = 0.000000e+00f;
  Conv3dOutput[(6)] = 0.000000e+00f;
  Conv3dOutput[(14)] = 0.000000e+00f;
  Conv3dOutput[(7)] = 0.000000e+00f;
  Conv3dOutput[(15)] = 0.000000e+00f;
  for (int rh_outer_outer = 0; rh_outer_outer < 3; ++rh_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) {
        if (((((int)threadIdx.x) * 6) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s / 3)) < 36) {
          if (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) < 108) {
            if (((int)threadIdx.x) < 6) {
              PaddedInput_shared[(((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s))] = (((((((1 <= (((((int)blockIdx.x) / 224) * 7) + (((((int)threadIdx.x) * 6) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s / 3)) % 9))) && ((((((int)blockIdx.x) / 224) * 7) + (((((int)threadIdx.x) * 6) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s / 3)) % 9)) < 15)) && (1 <= (((((int)blockIdx.x) % 224) >> 5) + rh_outer_outer))) && ((((((int)blockIdx.x) % 224) >> 5) + rh_outer_outer) < 8)) && (1 <= (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s % 3))) && ((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s % 3) < 2)) ? placeholder[((((((((((((((int)threadIdx.x) * 6) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s / 3)) / 9) * 50176) + ((((int)blockIdx.x) / 224) * 25088)) + ((((((int)threadIdx.x) * 6) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s / 3)) % 9) * 3584)) + (((((int)blockIdx.x) % 224) >> 5) * 512)) + (rh_outer_outer * 512)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s % 3) * 512)) + rc_outer_outer) - 4608))] : 0.000000e+00f);
            }
          }
        }
      }
      placeholder_shared[((((int)threadIdx.x) * 3))] = placeholder1[((((((((((int)threadIdx.x) >> 4) * 2359296) + (rh_outer_outer * 786432)) + ((((((int)threadIdx.x) & 15) * 3) >> 4) * 262144)) + (rc_outer_outer * 512)) + ((((int)blockIdx.x) & 31) * 16)) + ((((int)threadIdx.x) * 3) & 15)))];
      placeholder_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder1[((((((((((((int)threadIdx.x) * 3) + 1) / 48) * 2359296) + (rh_outer_outer * 786432)) + (((((((int)threadIdx.x) * 3) + 1) % 48) >> 4) * 262144)) + (rc_outer_outer * 512)) + ((((int)blockIdx.x) & 31) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)))];
      placeholder_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder1[((((((((((((int)threadIdx.x) * 3) + 2) / 48) * 2359296) + (rh_outer_outer * 786432)) + (((((((int)threadIdx.x) * 3) + 2) % 48) >> 4) * 262144)) + (rc_outer_outer * 512)) + ((((int)blockIdx.x) & 31) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)))];
      if (((int)threadIdx.x) < 20) {
        placeholder_shared[(((((int)threadIdx.x) * 3) + 84))] = placeholder1[((((((((((((int)threadIdx.x) * 3) + 84) / 48) * 2359296) + (rh_outer_outer * 786432)) + (((((((int)threadIdx.x) * 3) + 36) % 48) >> 4) * 262144)) + (rc_outer_outer * 512)) + ((((int)blockIdx.x) & 31) * 16)) + (((((int)threadIdx.x) * 3) + 4) & 15)))];
      }
      if (((int)threadIdx.x) < 20) {
        placeholder_shared[(((((int)threadIdx.x) * 3) + 85))] = placeholder1[((((((((((((int)threadIdx.x) * 3) + 85) / 48) * 2359296) + (rh_outer_outer * 786432)) + (((((((int)threadIdx.x) * 3) + 37) % 48) >> 4) * 262144)) + (rc_outer_outer * 512)) + ((((int)blockIdx.x) & 31) * 16)) + (((((int)threadIdx.x) * 3) + 5) & 15)))];
      }
      if (((int)threadIdx.x) < 20) {
        placeholder_shared[(((((int)threadIdx.x) * 3) + 86))] = placeholder1[((((((((((((int)threadIdx.x) * 3) + 86) / 48) * 2359296) + (rh_outer_outer * 786432)) + (((((((int)threadIdx.x) * 3) + 38) % 48) >> 4) * 262144)) + (rc_outer_outer * 512)) + ((((int)blockIdx.x) & 31) * 16)) + (((((int)threadIdx.x) * 3) + 6) & 15)))];
      }
      __syncthreads();
      for (int rd_inner = 0; rd_inner < 3; ++rd_inner) {
        for (int rw_inner = 0; rw_inner < 3; ++rw_inner) {
          Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)))]));
          Conv3dOutput[(8)] = (Conv3dOutput[(8)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 8))]));
          Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 1))]));
          Conv3dOutput[(9)] = (Conv3dOutput[(9)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 9))]));
          Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 2))]));
          Conv3dOutput[(10)] = (Conv3dOutput[(10)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 10))]));
          Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 3))]));
          Conv3dOutput[(11)] = (Conv3dOutput[(11)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 11))]));
          Conv3dOutput[(4)] = (Conv3dOutput[(4)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)))]));
          Conv3dOutput[(12)] = (Conv3dOutput[(12)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 8))]));
          Conv3dOutput[(5)] = (Conv3dOutput[(5)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 1))]));
          Conv3dOutput[(13)] = (Conv3dOutput[(13)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 9))]));
          Conv3dOutput[(6)] = (Conv3dOutput[(6)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 2))]));
          Conv3dOutput[(14)] = (Conv3dOutput[(14)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 10))]));
          Conv3dOutput[(7)] = (Conv3dOutput[(7)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 3))]));
          Conv3dOutput[(15)] = (Conv3dOutput[(15)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 14) * 54) + (((((int)threadIdx.x) % 14) >> 1) * 3)) + (rd_inner * 3)) + rw_inner) + 27))] * placeholder_shared[(((((rd_inner * 48) + (rw_inner * 16)) + ((((int)threadIdx.x) & 1) * 4)) + 11))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax4_inner = 0; ax4_inner < 4; ++ax4_inner) {
      T_relu[(((((((((((int)threadIdx.x) / 14) * 100352) + (ax0_inner * 50176)) + ((((int)blockIdx.x) / 224) * 25088)) + (((((int)threadIdx.x) % 14) >> 1) * 3584)) + ((((int)blockIdx.x) % 224) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner))] = max((((Conv3dOutput[(((ax0_inner * 4) + ax4_inner))] * placeholder2[(((((((int)blockIdx.x) & 31) * 16) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner))]) + placeholder3[(((((((int)blockIdx.x) & 31) * 16) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner))]) + placeholder4[(((((((((((int)threadIdx.x) / 14) * 100352) + (ax0_inner * 50176)) + ((((int)blockIdx.x) / 224) * 25088)) + (((((int)threadIdx.x) % 14) >> 1) * 3584)) + ((((int)blockIdx.x) % 224) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) / 14) * 100352) + (ax0_inner * 50176)) + ((((int)blockIdx.x) / 224) * 25088)) + (((((int)threadIdx.x) % 14) >> 1) * 3584)) + ((((int)blockIdx.x) % 224) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner) + 8))] = max((((Conv3dOutput[((((ax0_inner * 4) + ax4_inner) + 8))] * placeholder2[((((((((int)blockIdx.x) & 31) * 16) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner) + 8))]) + placeholder3[((((((((int)blockIdx.x) & 31) * 16) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner) + 8))]) + placeholder4[((((((((((((int)threadIdx.x) / 14) * 100352) + (ax0_inner * 50176)) + ((((int)blockIdx.x) / 224) * 25088)) + (((((int)threadIdx.x) % 14) >> 1) * 3584)) + ((((int)blockIdx.x) % 224) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + ax4_inner) + 8))]), 0.000000e+00f);
    }
  }
}


