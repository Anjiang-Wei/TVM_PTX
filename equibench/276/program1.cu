
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[1200];
  __shared__ float PaddedInput_shared[2242];
  __shared__ float placeholder_shared[64];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 15; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 600))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 300))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 900))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 601))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 301))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 901))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 602))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 302))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 902))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 603))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 303))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + 903))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 71; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) < 2242) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) / 6) * 460800) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) / 1121) * 230400)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 1121) / 19) * 3840)) + (((((int)blockIdx.x) % 6) >> 1) * 1280)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 19) * 64)) + rc_outer_outer))];
      }
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[(((((rc_outer_outer * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 32))];
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 15; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
        Conv2dOutput[(((yy_outer_inner * 20) + (xx_outer_inner * 4)))] = (Conv2dOutput[(((yy_outer_inner * 20) + (xx_outer_inner * 4)))] + (PaddedInput_shared[((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 4))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 600))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 600))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 570))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 4))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 300))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 300))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1121))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 4))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 900))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 900))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1691))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 4))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 601))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 601))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 570))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 301))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 301))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1121))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 901))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 901))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1691))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 2))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 2))] + (PaddedInput_shared[((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 602))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 602))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 570))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 302))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 302))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1121))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 902))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 902))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1691))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 3))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 3))] + (PaddedInput_shared[((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 3))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 603))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 603))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 570))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 3))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 303))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 303))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1121))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 3))]));
        Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 903))] = (Conv2dOutput[((((yy_outer_inner * 20) + (xx_outer_inner * 4)) + 903))] + (PaddedInput_shared[(((((yy_outer_inner * 38) + ((((int)threadIdx.x) >> 4) * 10)) + (xx_outer_inner * 2)) + 1691))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 4) + 3))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 15; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
          T_add[(((((((((((((int)blockIdx.x) / 6) * 230400) + (ax0_inner * 115200)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) % 6) >> 1) * 1280)) + ((((int)threadIdx.x) >> 4) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))] = (Conv2dOutput[(((((ax0_inner * 300) + (ax1_inner * 20)) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))]);
          T_add[((((((((((((((int)blockIdx.x) / 6) * 230400) + (ax0_inner * 115200)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) % 6) >> 1) * 1280)) + ((((int)threadIdx.x) >> 4) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner) + 57600))] = (Conv2dOutput[((((((ax0_inner * 300) + (ax1_inner * 20)) + (ax2_inner * 4)) + ax3_inner) + 600))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 4)) + ax3_inner))]);
        }
      }
    }
  }
}


