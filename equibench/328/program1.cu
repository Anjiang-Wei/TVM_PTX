
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 240))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 240))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[2];
  __shared__ float PaddedInput_shared[729];
  __shared__ float placeholder_shared[1];
  DepthwiseConv2d_local[(0)] = 0.000000e+00f;
  DepthwiseConv2d_local[(1)] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
      __syncthreads();
      PaddedInput_shared[((((int)threadIdx.x) * 6))] = (((1 <= (((((int)threadIdx.x) * 6) / 27) + di_outer_outer)) && (1 <= (dj_outer_outer + ((((int)threadIdx.x) * 6) % 27)))) ? placeholder[(((((((((((int)threadIdx.x) * 6) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + (((((int)threadIdx.x) * 6) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 6) + 1))] = (((1 <= ((((((int)threadIdx.x) * 6) + 1) / 27) + di_outer_outer)) && (1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 1) % 27)))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 1) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 1) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 6) + 2))] = (((1 <= ((((((int)threadIdx.x) * 6) + 2) / 27) + di_outer_outer)) && (1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 2) % 27)))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 2) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 2) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 6) + 3))] = (((1 <= ((((((int)threadIdx.x) * 6) + 3) / 27) + di_outer_outer)) && (1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 3) % 27)))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 3) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 3) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 6) + 4))] = (((1 <= ((((((int)threadIdx.x) * 6) + 4) / 27) + di_outer_outer)) && (1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 4) % 27)))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 4) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 4) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      PaddedInput_shared[(((((int)threadIdx.x) * 6) + 5))] = (((1 <= ((((((int)threadIdx.x) * 6) + 5) / 27) + di_outer_outer)) && (1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 5) % 27)))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 5) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 5) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      if (((int)threadIdx.x) < 24) {
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 588))] = ((1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 21) % 27))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 588) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 21) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 24) {
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 589))] = ((1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 22) % 27))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 589) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 22) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 24) {
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 590))] = ((1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 23) % 27))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 590) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 23) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 23) {
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 591))] = ((1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 24) % 27))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 591) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 24) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 23) {
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 592))] = ((1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 25) % 27))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 592) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 25) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 23) {
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 593))] = ((1 <= (dj_outer_outer + (((((int)threadIdx.x) * 6) + 26) % 27))) ? placeholder[((((((((((((int)threadIdx.x) * 6) + 593) / 27) * 6720) + (di_outer_outer * 6720)) + (dj_outer_outer * 240)) + ((((((int)threadIdx.x) * 6) + 26) % 27) * 240)) + ((int)blockIdx.x)) - 6960))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((int)threadIdx.x) * 720) + (di_outer_outer * 720)) + (dj_outer_outer * 240)) + ((int)blockIdx.x)))];
      }
      __syncthreads();
      DepthwiseConv2d_local[(0)] = (DepthwiseConv2d_local[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 7) * 54) + ((((int)threadIdx.x) % 7) * 4)))] * placeholder_shared[(0)]));
      DepthwiseConv2d_local[(1)] = (DepthwiseConv2d_local[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 7) * 54) + ((((int)threadIdx.x) % 7) * 4)) + 2))] * placeholder_shared[(0)]));
    }
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    DepthwiseConv2d[((((((int)threadIdx.x) * 480) + (j_inner * 240)) + ((int)blockIdx.x)))] = DepthwiseConv2d_local[(j_inner)];
  }
}


