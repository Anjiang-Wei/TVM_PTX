
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ tensor, float* __restrict__ tensor1) {
  tensor[(((int)blockIdx.x))] = (tensor1[(((int)blockIdx.x))] * 9.765625e-04f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 32; ++rv0) {
    for (int rv1 = 0; rv1 < 32; ++rv1) {
      tensor[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] = (tensor[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] + placeholder[((((((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 72) * 73728) + (rv0 * 2304)) + (rv1 * 72)) + (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 72)))]);
    }
  }
}


