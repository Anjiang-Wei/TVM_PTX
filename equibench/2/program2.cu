
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[784];
  __shared__ float placeholder_shared[512];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 16; ++ff_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 16) + ff_outer_inner_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) / 28) * 7168) + ((((int)blockIdx.x) >> 2) * 1792)) + (((((int)threadIdx.x) % 28) >> 2) * 256)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    if (((int)threadIdx.x) < 512) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 7) * 512)) + ((((int)blockIdx.x) & 3) * 128)) + (((int)threadIdx.x) & 127)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 16; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
          Conv2dOutput[(((yy_outer_inner * 16) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 16) + ff_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 56) * 56) + (yy_outer_inner * 28)) + (((((int)threadIdx.x) % 56) >> 3) * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + ff_outer_inner))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
      T_relu[(((((((((((int)threadIdx.x) / 56) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 2) * 3584)) + (((((int)threadIdx.x) % 56) >> 3) * 512)) + ((((int)blockIdx.x) & 3) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax3_inner))] = max(((Conv2dOutput[(((ax1_inner * 16) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 128) + ((((int)threadIdx.x) & 7) * 16)) + ax3_inner))]) + placeholder3[(((((((((((int)threadIdx.x) / 56) * 28672) + (ax1_inner * 14336)) + ((((int)blockIdx.x) >> 2) * 3584)) + (((((int)threadIdx.x) % 56) >> 3) * 512)) + ((((int)blockIdx.x) & 3) * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


