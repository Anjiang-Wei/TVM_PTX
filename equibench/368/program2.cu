
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[576];
  __shared__ float placeholder_shared[288];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 4; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 8))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 32))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 64))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 96))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 33))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 65))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 97))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 34))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 66))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 98))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 35))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 67))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 99))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 36))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 68))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 100))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 37))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 69))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 101))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 38))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 70))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 102))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 39))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 71))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 8) + 103))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 144; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) / 144) * 147456) + ((((int)blockIdx.x) >> 4) * 9216)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 144) / 72) * 4608)) + (((((int)blockIdx.x) & 15) >> 1) * 576)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 72) / 18) * 144)) + (rc_outer_outer * 18)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4) + ((int)threadIdx.x)) % 18)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 72; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 4) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 576) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 4) + ((int)threadIdx.x)) >> 4) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 4) + ((int)threadIdx.x)) & 15)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 18; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
        Conv2dOutput[((xx_outer_inner * 8))] = (Conv2dOutput[((xx_outer_inner * 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 32))] = (Conv2dOutput[(((xx_outer_inner * 8) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 64))] = (Conv2dOutput[(((xx_outer_inner * 8) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 96))] = (Conv2dOutput[(((xx_outer_inner * 8) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[(((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 1))] = (Conv2dOutput[(((xx_outer_inner * 8) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 33))] = (Conv2dOutput[(((xx_outer_inner * 8) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 65))] = (Conv2dOutput[(((xx_outer_inner * 8) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 97))] = (Conv2dOutput[(((xx_outer_inner * 8) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 1))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 2))] = (Conv2dOutput[(((xx_outer_inner * 8) + 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 34))] = (Conv2dOutput[(((xx_outer_inner * 8) + 34))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 66))] = (Conv2dOutput[(((xx_outer_inner * 8) + 66))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 98))] = (Conv2dOutput[(((xx_outer_inner * 8) + 98))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 2))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 3))] = (Conv2dOutput[(((xx_outer_inner * 8) + 3))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 3))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 35))] = (Conv2dOutput[(((xx_outer_inner * 8) + 35))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 3))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 67))] = (Conv2dOutput[(((xx_outer_inner * 8) + 67))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 3))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 99))] = (Conv2dOutput[(((xx_outer_inner * 8) + 99))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 3))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 4))] = (Conv2dOutput[(((xx_outer_inner * 8) + 4))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 4))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 36))] = (Conv2dOutput[(((xx_outer_inner * 8) + 36))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 4))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 68))] = (Conv2dOutput[(((xx_outer_inner * 8) + 68))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 4))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 100))] = (Conv2dOutput[(((xx_outer_inner * 8) + 100))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 4))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 5))] = (Conv2dOutput[(((xx_outer_inner * 8) + 5))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 5))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 37))] = (Conv2dOutput[(((xx_outer_inner * 8) + 37))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 5))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 69))] = (Conv2dOutput[(((xx_outer_inner * 8) + 69))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 5))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 101))] = (Conv2dOutput[(((xx_outer_inner * 8) + 101))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 5))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 6))] = (Conv2dOutput[(((xx_outer_inner * 8) + 6))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 6))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 38))] = (Conv2dOutput[(((xx_outer_inner * 8) + 38))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 6))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 70))] = (Conv2dOutput[(((xx_outer_inner * 8) + 70))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 6))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 102))] = (Conv2dOutput[(((xx_outer_inner * 8) + 102))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 6))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 7))] = (Conv2dOutput[(((xx_outer_inner * 8) + 7))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 7))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 39))] = (Conv2dOutput[(((xx_outer_inner * 8) + 39))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 7))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 71))] = (Conv2dOutput[(((xx_outer_inner * 8) + 71))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 7))]));
        Conv2dOutput[(((xx_outer_inner * 8) + 103))] = (Conv2dOutput[(((xx_outer_inner * 8) + 103))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 72) + (xx_outer_inner * 18)) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + 7))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_add[(((((((((((int)blockIdx.x) >> 4) * 2048) + ((((int)threadIdx.x) >> 1) * 1024)) + (((((int)blockIdx.x) & 15) >> 1) * 128)) + (ax2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))] = (Conv2dOutput[(((ax2_inner * 8) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]);
      T_add[((((((((((((int)blockIdx.x) >> 4) * 2048) + ((((int)threadIdx.x) >> 1) * 1024)) + (((((int)blockIdx.x) & 15) >> 1) * 128)) + (ax2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 32768))] = (Conv2dOutput[((((ax2_inner * 8) + ax3_inner) + 32))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]);
      T_add[((((((((((((int)blockIdx.x) >> 4) * 2048) + ((((int)threadIdx.x) >> 1) * 1024)) + (((((int)blockIdx.x) & 15) >> 1) * 128)) + (ax2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 65536))] = (Conv2dOutput[((((ax2_inner * 8) + ax3_inner) + 64))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]);
      T_add[((((((((((((int)blockIdx.x) >> 4) * 2048) + ((((int)threadIdx.x) >> 1) * 1024)) + (((((int)blockIdx.x) & 15) >> 1) * 128)) + (ax2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 98304))] = (Conv2dOutput[((((ax2_inner * 8) + ax3_inner) + 96))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]);
    }
  }
}


