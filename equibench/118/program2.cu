
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[4];
  __shared__ float PaddedInput_shared[2560];
  __shared__ float placeholder_shared[1280];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 2560) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) < 640) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 1280) * 163840) + ((((int)blockIdx.x) >> 5) * 40960)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 1280) / 160) * 5120)) + (((((int)blockIdx.x) & 31) >> 2) * 640)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 160) / 40) * 160)) + (rc_outer_outer * 40)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2048) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 40)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((int)threadIdx.x) >> 5)) < 40) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((int)threadIdx.x)) < 1280) {
          placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 512) + ((int)threadIdx.x)))] = placeholder1[((((((rc_outer_outer * 5120) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2048)) + ((((int)threadIdx.x) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 31)))];
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 40; ++rc_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 40) + rc_inner))] * placeholder_shared[(((rc_inner * 32) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 40) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + (((int)threadIdx.x) & 7)) + 8))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 40) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + (((int)threadIdx.x) & 7)) + 16))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 40) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + (((int)threadIdx.x) & 7)) + 24))]));
    }
  }
  T_relu[(((((((((((int)threadIdx.x) >> 8) * 131072) + ((((int)blockIdx.x) >> 5) * 32768)) + (((((int)threadIdx.x) & 255) >> 5) * 4096)) + (((((int)blockIdx.x) & 31) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 7)))] = max((Conv2dOutput[(0)] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 7)))]), 0.000000e+00f);
  T_relu[((((((((((((int)threadIdx.x) >> 8) * 131072) + ((((int)blockIdx.x) >> 5) * 32768)) + (((((int)threadIdx.x) & 255) >> 5) * 4096)) + (((((int)blockIdx.x) & 31) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 7)) + 8))] = max((Conv2dOutput[(1)] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 7)) + 8))]), 0.000000e+00f);
  T_relu[((((((((((((int)threadIdx.x) >> 8) * 131072) + ((((int)blockIdx.x) >> 5) * 32768)) + (((((int)threadIdx.x) & 255) >> 5) * 4096)) + (((((int)blockIdx.x) & 31) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 7)) + 16))] = max((Conv2dOutput[(2)] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 7)) + 16))]), 0.000000e+00f);
  T_relu[((((((((((((int)threadIdx.x) >> 8) * 131072) + ((((int)blockIdx.x) >> 5) * 32768)) + (((((int)threadIdx.x) & 255) >> 5) * 4096)) + (((((int)blockIdx.x) & 31) >> 2) * 512)) + (((((int)threadIdx.x) & 31) >> 3) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) & 7)) + 24))] = max((Conv2dOutput[(3)] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + (((int)threadIdx.x) & 7)) + 24))]), 0.000000e+00f);
}


