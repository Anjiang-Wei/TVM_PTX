
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float placeholder_d_shared[512];
  __shared__ float placeholder_shared[256];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 32))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 256))];
    placeholder_d_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 512))];
    placeholder_d_shared[((((int)threadIdx.x) + 96))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 768))];
    placeholder_d_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1024))];
    placeholder_d_shared[((((int)threadIdx.x) + 160))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1280))];
    placeholder_d_shared[((((int)threadIdx.x) + 192))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1536))];
    placeholder_d_shared[((((int)threadIdx.x) + 224))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 1792))];
    placeholder_d_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 16384))];
    placeholder_d_shared[((((int)threadIdx.x) + 288))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 288) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 2) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 320) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 352))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 352) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 6) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 384) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 8) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 416))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 416) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 10) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 448))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 448) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 12) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_d_shared[((((int)threadIdx.x) + 480))] = placeholder[((((((((((int)blockIdx.x) >> 6) * 32768) + (((((int)threadIdx.x) + 480) >> 8) * 16384)) + (((((int)blockIdx.x) & 63) >> 3) * 2048)) + (((((int)threadIdx.x) >> 4) + 14) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((((int)blockIdx.x) >> 6) * 16384) + ((((int)blockIdx.x) & 7) * 1024)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + ((((int)blockIdx.x) & 7) * 1024)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + ((((int)blockIdx.x) & 7) * 1024)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + ((((int)blockIdx.x) & 7) * 1024)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + ((((int)blockIdx.x) & 7) * 1024)) + ((((int)threadIdx.x) >> 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 8192))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + (((((int)threadIdx.x) + 160) >> 7) * 8192)) + ((((int)blockIdx.x) & 7) * 1024)) + (((((int)threadIdx.x) >> 4) + 2) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + (((((int)threadIdx.x) + 192) >> 7) * 8192)) + ((((int)blockIdx.x) & 7) * 1024)) + (((((int)threadIdx.x) >> 4) + 4) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((((((((int)blockIdx.x) >> 6) * 16384) + (((((int)threadIdx.x) + 224) >> 7) * 8192)) + ((((int)blockIdx.x) & 7) * 1024)) + (((((int)threadIdx.x) >> 4) + 6) * 128)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          compute_local[(b_c_outer_inner)] = (compute_local[(b_c_outer_inner)] + (placeholder_d_shared[(((((b_c_outer_inner * 256) + ((((int)threadIdx.x) >> 3) * 16)) + (k_outer_inner * 8)) + k_inner))] * placeholder_shared[(((((b_c_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (k_outer_inner * 8)) + k_inner))]));
          compute_local[((b_c_outer_inner + 2))] = (compute_local[((b_c_outer_inner + 2))] + (placeholder_d_shared[((((((b_c_outer_inner * 256) + ((((int)threadIdx.x) >> 3) * 16)) + (k_outer_inner * 8)) + k_inner) + 64))] * placeholder_shared[(((((b_c_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (k_outer_inner * 8)) + k_inner))]));
          compute_local[((b_c_outer_inner + 4))] = (compute_local[((b_c_outer_inner + 4))] + (placeholder_d_shared[((((((b_c_outer_inner * 256) + ((((int)threadIdx.x) >> 3) * 16)) + (k_outer_inner * 8)) + k_inner) + 128))] * placeholder_shared[(((((b_c_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (k_outer_inner * 8)) + k_inner))]));
          compute_local[((b_c_outer_inner + 6))] = (compute_local[((b_c_outer_inner + 6))] + (placeholder_d_shared[((((((b_c_outer_inner * 256) + ((((int)threadIdx.x) >> 3) * 16)) + (k_outer_inner * 8)) + k_inner) + 192))] * placeholder_shared[(((((b_c_outer_inner * 128) + ((((int)threadIdx.x) & 7) * 16)) + (k_outer_inner * 8)) + k_inner))]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    compute[((((((((((int)blockIdx.x) >> 6) * 16384) + (b_inner * 8192)) + (((((int)blockIdx.x) & 63) >> 3) * 1024)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 7) * 8)) + (((int)threadIdx.x) & 7)))] = compute_local[(b_inner)];
    compute[(((((((((((int)blockIdx.x) >> 6) * 16384) + (b_inner * 8192)) + (((((int)blockIdx.x) & 63) >> 3) * 1024)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 7) * 8)) + (((int)threadIdx.x) & 7)) + 256))] = compute_local[((b_inner + 2))];
    compute[(((((((((((int)blockIdx.x) >> 6) * 16384) + (b_inner * 8192)) + (((((int)blockIdx.x) & 63) >> 3) * 1024)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 7) * 8)) + (((int)threadIdx.x) & 7)) + 512))] = compute_local[((b_inner + 4))];
    compute[(((((((((((int)blockIdx.x) >> 6) * 16384) + (b_inner * 8192)) + (((((int)blockIdx.x) & 63) >> 3) * 1024)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)blockIdx.x) & 7) * 8)) + (((int)threadIdx.x) & 7)) + 768))] = compute_local[((b_inner + 6))];
  }
}


