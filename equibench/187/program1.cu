
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[6720];
  __shared__ float PaddedInput_shared[896];
  __shared__ float placeholder_shared[240];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 6; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 28; ++yy_inner_init) {
      Conv2dOutput[(((yy_inner_init * 120) + (ff_outer_inner_init * 10)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3360))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3361))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3362))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3363))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3364))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 5))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3365))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3366))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 7))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3367))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3368))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 9))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3369))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 60))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3420))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 61))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3421))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 62))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3422))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 63))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3423))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 64))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3424))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 65))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3425))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 66))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3426))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 67))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3427))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 68))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3428))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 69))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 120) + (ff_outer_inner_init * 10)) + 3429))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 20; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 8))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 2240))];
    PaddedInput_shared[((((int)threadIdx.x) + 16))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 4480))];
    PaddedInput_shared[((((int)threadIdx.x) + 24))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 6720))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 8960))];
    PaddedInput_shared[((((int)threadIdx.x) + 40))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 11200))];
    PaddedInput_shared[((((int)threadIdx.x) + 48))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 13440))];
    PaddedInput_shared[((((int)threadIdx.x) + 56))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 15680))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 17920))];
    PaddedInput_shared[((((int)threadIdx.x) + 72))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 20160))];
    PaddedInput_shared[((((int)threadIdx.x) + 80))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 22400))];
    PaddedInput_shared[((((int)threadIdx.x) + 88))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 24640))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 26880))];
    PaddedInput_shared[((((int)threadIdx.x) + 104))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 29120))];
    PaddedInput_shared[((((int)threadIdx.x) + 112))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 31360))];
    PaddedInput_shared[((((int)threadIdx.x) + 120))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 33600))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 35840))];
    PaddedInput_shared[((((int)threadIdx.x) + 136))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 38080))];
    PaddedInput_shared[((((int)threadIdx.x) + 144))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 40320))];
    PaddedInput_shared[((((int)threadIdx.x) + 152))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 42560))];
    PaddedInput_shared[((((int)threadIdx.x) + 160))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 44800))];
    PaddedInput_shared[((((int)threadIdx.x) + 168))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 47040))];
    PaddedInput_shared[((((int)threadIdx.x) + 176))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 49280))];
    PaddedInput_shared[((((int)threadIdx.x) + 184))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 51520))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 53760))];
    PaddedInput_shared[((((int)threadIdx.x) + 200))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 56000))];
    PaddedInput_shared[((((int)threadIdx.x) + 208))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 58240))];
    PaddedInput_shared[((((int)threadIdx.x) + 216))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 60480))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 62720))];
    PaddedInput_shared[((((int)threadIdx.x) + 232))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 64960))];
    PaddedInput_shared[((((int)threadIdx.x) + 240))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 67200))];
    PaddedInput_shared[((((int)threadIdx.x) + 248))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 69440))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 71680))];
    PaddedInput_shared[((((int)threadIdx.x) + 264))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 73920))];
    PaddedInput_shared[((((int)threadIdx.x) + 272))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 76160))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 78400))];
    PaddedInput_shared[((((int)threadIdx.x) + 288))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 80640))];
    PaddedInput_shared[((((int)threadIdx.x) + 296))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 82880))];
    PaddedInput_shared[((((int)threadIdx.x) + 304))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 85120))];
    PaddedInput_shared[((((int)threadIdx.x) + 312))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 87360))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 89600))];
    PaddedInput_shared[((((int)threadIdx.x) + 328))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 91840))];
    PaddedInput_shared[((((int)threadIdx.x) + 336))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 94080))];
    PaddedInput_shared[((((int)threadIdx.x) + 344))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 96320))];
    PaddedInput_shared[((((int)threadIdx.x) + 352))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 98560))];
    PaddedInput_shared[((((int)threadIdx.x) + 360))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 100800))];
    PaddedInput_shared[((((int)threadIdx.x) + 368))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 103040))];
    PaddedInput_shared[((((int)threadIdx.x) + 376))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 105280))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 107520))];
    PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 109760))];
    PaddedInput_shared[((((int)threadIdx.x) + 400))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 112000))];
    PaddedInput_shared[((((int)threadIdx.x) + 408))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 114240))];
    PaddedInput_shared[((((int)threadIdx.x) + 416))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 116480))];
    PaddedInput_shared[((((int)threadIdx.x) + 424))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 118720))];
    PaddedInput_shared[((((int)threadIdx.x) + 432))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 120960))];
    PaddedInput_shared[((((int)threadIdx.x) + 440))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 123200))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 125440))];
    PaddedInput_shared[((((int)threadIdx.x) + 456))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 127680))];
    PaddedInput_shared[((((int)threadIdx.x) + 464))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 129920))];
    PaddedInput_shared[((((int)threadIdx.x) + 472))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 132160))];
    PaddedInput_shared[((((int)threadIdx.x) + 480))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 134400))];
    PaddedInput_shared[((((int)threadIdx.x) + 488))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 136640))];
    PaddedInput_shared[((((int)threadIdx.x) + 496))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 138880))];
    PaddedInput_shared[((((int)threadIdx.x) + 504))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 141120))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 143360))];
    PaddedInput_shared[((((int)threadIdx.x) + 520))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 145600))];
    PaddedInput_shared[((((int)threadIdx.x) + 528))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 147840))];
    PaddedInput_shared[((((int)threadIdx.x) + 536))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 150080))];
    PaddedInput_shared[((((int)threadIdx.x) + 544))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 152320))];
    PaddedInput_shared[((((int)threadIdx.x) + 552))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 154560))];
    PaddedInput_shared[((((int)threadIdx.x) + 560))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 156800))];
    PaddedInput_shared[((((int)threadIdx.x) + 568))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 159040))];
    PaddedInput_shared[((((int)threadIdx.x) + 576))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 161280))];
    PaddedInput_shared[((((int)threadIdx.x) + 584))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 163520))];
    PaddedInput_shared[((((int)threadIdx.x) + 592))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 165760))];
    PaddedInput_shared[((((int)threadIdx.x) + 600))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 168000))];
    PaddedInput_shared[((((int)threadIdx.x) + 608))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 170240))];
    PaddedInput_shared[((((int)threadIdx.x) + 616))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 172480))];
    PaddedInput_shared[((((int)threadIdx.x) + 624))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 174720))];
    PaddedInput_shared[((((int)threadIdx.x) + 632))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 176960))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 179200))];
    PaddedInput_shared[((((int)threadIdx.x) + 648))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 181440))];
    PaddedInput_shared[((((int)threadIdx.x) + 656))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 183680))];
    PaddedInput_shared[((((int)threadIdx.x) + 664))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 185920))];
    PaddedInput_shared[((((int)threadIdx.x) + 672))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 188160))];
    PaddedInput_shared[((((int)threadIdx.x) + 680))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 190400))];
    PaddedInput_shared[((((int)threadIdx.x) + 688))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 192640))];
    PaddedInput_shared[((((int)threadIdx.x) + 696))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 194880))];
    PaddedInput_shared[((((int)threadIdx.x) + 704))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 197120))];
    PaddedInput_shared[((((int)threadIdx.x) + 712))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 199360))];
    PaddedInput_shared[((((int)threadIdx.x) + 720))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 201600))];
    PaddedInput_shared[((((int)threadIdx.x) + 728))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 203840))];
    PaddedInput_shared[((((int)threadIdx.x) + 736))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 206080))];
    PaddedInput_shared[((((int)threadIdx.x) + 744))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 208320))];
    PaddedInput_shared[((((int)threadIdx.x) + 752))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 210560))];
    PaddedInput_shared[((((int)threadIdx.x) + 760))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 212800))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 215040))];
    PaddedInput_shared[((((int)threadIdx.x) + 776))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 217280))];
    PaddedInput_shared[((((int)threadIdx.x) + 784))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 219520))];
    PaddedInput_shared[((((int)threadIdx.x) + 792))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 221760))];
    PaddedInput_shared[((((int)threadIdx.x) + 800))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 224000))];
    PaddedInput_shared[((((int)threadIdx.x) + 808))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 226240))];
    PaddedInput_shared[((((int)threadIdx.x) + 816))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 228480))];
    PaddedInput_shared[((((int)threadIdx.x) + 824))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 230720))];
    PaddedInput_shared[((((int)threadIdx.x) + 832))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 232960))];
    PaddedInput_shared[((((int)threadIdx.x) + 840))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 235200))];
    PaddedInput_shared[((((int)threadIdx.x) + 848))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 237440))];
    PaddedInput_shared[((((int)threadIdx.x) + 856))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 239680))];
    PaddedInput_shared[((((int)threadIdx.x) + 864))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 241920))];
    PaddedInput_shared[((((int)threadIdx.x) + 872))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 244160))];
    PaddedInput_shared[((((int)threadIdx.x) + 880))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 246400))];
    PaddedInput_shared[((((int)threadIdx.x) + 888))] = placeholder[((((((((((int)threadIdx.x) >> 2) * 1120) + (((int)blockIdx.x) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 40)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 248640))];
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[((((int)threadIdx.x) * 60))] = placeholder1[(((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 1))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 1))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 2))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 2))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 3))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 3))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 4))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 4))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 5))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 5))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 6))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 6))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 7))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 7))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 8))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 8))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 9))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 9))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 10))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 10))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 11))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 11))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 12))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 12))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 13))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 13))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 14))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 14))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 15))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 15))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 16))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 16))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 17))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 17))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 18))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 18))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 19))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 19))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 20))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 20))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 21))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 21))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 22))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 22))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 23))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 23))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 24))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 24))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 25))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 25))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 26))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 26))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 27))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 27))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 28))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 28))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 29))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 29))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 30))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 30))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 31))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 31))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 32))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 32))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 33))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 33))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 34))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 34))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 35))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 35))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 36))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 36))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 37))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 37))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 38))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 38))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 39))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 39))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 40))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 40))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 41))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 41))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 42))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 42))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 43))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 43))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 44))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 44))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 45))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 45))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 46))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 46))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 47))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 47))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 48))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 48))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 49))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 49))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 50))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 50))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 51))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 51))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 52))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 52))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 53))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 53))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 54))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 54))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 55))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 55))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 56))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 56))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 57))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 57))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 58))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 58))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 59))] = placeholder1[((((rc_outer_outer * 240) + (((int)threadIdx.x) * 60)) + 59))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 6; ++ff_outer_inner) {
        for (int yy_inner = 0; yy_inner < 28; ++yy_inner) {
          Conv2dOutput[(((yy_inner * 120) + (ff_outer_inner * 10)))] = (Conv2dOutput[(((yy_inner * 120) + (ff_outer_inner * 10)))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3360))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3360))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 1))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 1))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3361))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3361))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 1))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 2))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 2))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3362))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3362))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 2))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 3))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3363))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3363))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 3))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 4))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 4))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 4))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3364))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3364))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 4))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 5))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 5))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 5))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3365))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3365))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 5))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 6))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 6))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 6))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3366))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3366))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 6))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 7))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 7))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 7))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3367))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3367))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 7))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 8))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 8))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3368))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3368))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 8))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 9))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 9))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 9))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3369))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3369))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 448))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 9))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 60))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 60))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3420))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3420))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 61))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 61))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 1))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3421))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3421))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 1))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 62))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 62))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 2))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3422))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3422))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 2))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 63))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 63))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 3))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3423))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3423))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 3))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 64))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 4))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3424))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3424))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 4))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 65))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 5))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3425))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3425))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 5))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 66))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 66))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 6))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3426))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3426))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 6))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 67))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 67))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 7))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3427))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3427))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 7))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 68))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 68))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 8))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3428))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3428))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 8))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 69))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 69))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 2))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 9))]));
          Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3429))] = (Conv2dOutput[((((yy_inner * 120) + (ff_outer_inner * 10)) + 3429))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 112) + (yy_inner * 4)) + rc_outer_inner) + 450))] * placeholder_shared[(((((rc_outer_inner * 120) + ((((int)threadIdx.x) & 1) * 60)) + (ff_outer_inner * 10)) + 9))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 28; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 60; ++ax3_inner) {
        T_relu[((((((((((int)threadIdx.x) >> 1) * 94080) + (ax1_inner * 3360)) + (((int)blockIdx.x) * 240)) + (ax2_inner * 120)) + ((((int)threadIdx.x) & 1) * 60)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 120) + (ax2_inner * 60)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 60) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)threadIdx.x) >> 1) * 94080) + (ax1_inner * 3360)) + (((int)blockIdx.x) * 240)) + (ax2_inner * 120)) + ((((int)threadIdx.x) & 1) * 60)) + ax3_inner) + 376320))] = max((Conv2dOutput[(((((ax1_inner * 120) + (ax2_inner * 60)) + ax3_inner) + 3360))] + placeholder2[((((((int)threadIdx.x) & 1) * 60) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


