
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[1024];
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[128];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 16; ++ff_outer_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 256))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 512))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 768))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 257))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 513))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 769))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 32))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 288))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 544))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 800))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 33))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 289))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 545))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_outer_inner_init * 128) + (yy_outer_inner_init * 64)) + (ff_outer_inner_init * 2)) + 801))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer) + 131072))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer) + 524288))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[(((((((((((int)threadIdx.x) + 192) >> 7) * 524288) + ((((int)blockIdx.x) >> 2) * 262144)) + (((((int)threadIdx.x) >> 3) + 8) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer) + 1048576))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[(((((((((((int)threadIdx.x) + 320) >> 7) * 524288) + ((((int)blockIdx.x) >> 2) * 262144)) + (((((int)threadIdx.x) >> 3) + 8) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 262144) + ((((int)threadIdx.x) >> 3) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer) + 1572864))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[(((((((((((int)threadIdx.x) + 448) >> 7) * 524288) + ((((int)blockIdx.x) >> 2) * 262144)) + (((((int)threadIdx.x) >> 3) + 8) * 16384)) + ((((int)blockIdx.x) & 3) * 4096)) + ((((int)threadIdx.x) & 7) * 512)) + rc_outer_outer))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 128) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 64))];
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 16; ++ff_outer_inner) {
          Conv2dOutput[((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)))] = (Conv2dOutput[((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[((ff_outer_inner * 2))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 256))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 256))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 32))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 512))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 512))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 64))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 768))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 768))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 96))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 1))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 257))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 257))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 33))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 513))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 513))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 65))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 769))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 769))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)))] * placeholder_shared[(((ff_outer_inner * 2) + 97))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 32))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 32))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[((ff_outer_inner * 2))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 288))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 288))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 32))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 544))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 544))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 64))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 800))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 800))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 96))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 33))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 33))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 1))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 289))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 289))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 33))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 545))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 545))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 65))]));
          Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 801))] = (Conv2dOutput[(((((nn_outer_inner * 128) + (yy_outer_inner * 64)) + (ff_outer_inner * 2)) + 801))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (nn_outer_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + (yy_outer_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 1))] * placeholder_shared[(((ff_outer_inner * 2) + 97))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
          T_relu[(((((((((((((int)threadIdx.x) >> 5) * 262144) + (ax0_inner * 131072)) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) & 31) >> 2) * 8192)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 3) * 256)) + (ax2_inner * 128)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 128) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner))] + placeholder2[(ax3_inner)]), 0.000000e+00f);
          T_relu[((((((((((((((int)threadIdx.x) >> 5) * 262144) + (ax0_inner * 131072)) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) & 31) >> 2) * 8192)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 3) * 256)) + (ax2_inner * 128)) + ax3_inner) + 32))] = max((Conv2dOutput[((((((ax0_inner * 128) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner) + 256))] + placeholder2[((ax3_inner + 32))]), 0.000000e+00f);
          T_relu[((((((((((((((int)threadIdx.x) >> 5) * 262144) + (ax0_inner * 131072)) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) & 31) >> 2) * 8192)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 3) * 256)) + (ax2_inner * 128)) + ax3_inner) + 64))] = max((Conv2dOutput[((((((ax0_inner * 128) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner) + 512))] + placeholder2[((ax3_inner + 64))]), 0.000000e+00f);
          T_relu[((((((((((((((int)threadIdx.x) >> 5) * 262144) + (ax0_inner * 131072)) + ((((int)blockIdx.x) >> 2) * 65536)) + (((((int)threadIdx.x) & 31) >> 2) * 8192)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) & 3) * 1024)) + ((((int)threadIdx.x) & 3) * 256)) + (ax2_inner * 128)) + ax3_inner) + 96))] = max((Conv2dOutput[((((((ax0_inner * 128) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner) + 768))] + placeholder2[((ax3_inner + 96))]), 0.000000e+00f);
        }
      }
    }
  }
}


