
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[180];
  __shared__ float PaddedInput_shared[180];
  __shared__ float placeholder_shared[1024];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 3; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
        Conv2dOutput[((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 12))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 24))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 36))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 48))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 60))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 72))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 84))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 96))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 108))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 120))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 132))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 144))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 156))] = 0.000000e+00f;
        Conv2dOutput[(((((xx_outer_inner_init * 4) + (ff_outer_inner_init * 2)) + ff_inner_init) + 168))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[((((int)threadIdx.x) * 12))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 11520) + ((((int)threadIdx.x) / 5) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((int)threadIdx.x) % 5) * 384)) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((int)threadIdx.x) / 5) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((int)threadIdx.x) % 5) * 384)) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((int)threadIdx.x) / 5) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((int)threadIdx.x) % 5) * 384)) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((int)threadIdx.x) / 5) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((int)threadIdx.x) % 5) * 384)) + (rc_outer_outer * 4)) + 3))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 1) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 1) % 15) * 128)) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 1) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 1) % 15) * 128)) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 1) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 1) % 15) * 128)) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 1) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 1) % 15) * 128)) + (rc_outer_outer * 4)) + 3))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 2) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 2) % 15) * 128)) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 2) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 2) % 15) * 128)) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 2) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 2) % 15) * 128)) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 15) {
      PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] = placeholder[((((((((((int)blockIdx.x) >> 2) * 11520) + ((((((int)threadIdx.x) * 3) + 2) / 15) * 3840)) + (((((int)blockIdx.x) & 3) >> 1) * 1920)) + ((((((int)threadIdx.x) * 3) + 2) % 15) * 128)) + (rc_outer_outer * 4)) + 3))];
    }
    ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + ((((rc_outer_outer * 2048) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 256))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 2048) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + 512))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 512))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 2048) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + 1024))))[0];
    ((float4*)(placeholder_shared + (((((int)threadIdx.x) * 4) + 768))))[0] = ((float4*)(placeholder1 + (((((rc_outer_outer * 2048) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + 1536))))[0];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
          for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
            Conv2dOutput[((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner))] = (Conv2dOutput[((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner))] + (PaddedInput_shared[(((xx_outer_inner * 4) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 12))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 12))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 12))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 24))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 24))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 24))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 36))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 36))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 36))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 48))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 48))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 48))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 60))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 60))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 60))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 72))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 72))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 72))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 84))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 84))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 84))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 96))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 96))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 96))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 108))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 108))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 108))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 120))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 120))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 120))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 132))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 132))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 132))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 144))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 144))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 144))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 156))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 156))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 156))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
            Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 168))] = (Conv2dOutput[(((((xx_outer_inner * 4) + (ff_outer_inner * 2)) + ff_inner) + 168))] + (PaddedInput_shared[((((xx_outer_inner * 4) + rc_outer_inner) + 168))] * placeholder_shared[(((((rc_outer_inner * 256) + (((int)threadIdx.x) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
          }
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner))] = max(((Conv2dOutput[(((ax2_inner * 4) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 1536))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 12))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 1536))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 3072))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 24))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 3072))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 4608))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 36))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 4608))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 6144))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 6144))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 15360))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 60))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 15360))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 16896))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 72))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 16896))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 18432))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 84))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 18432))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 19968))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 96))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 19968))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 21504))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 108))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 21504))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 30720))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 120))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 30720))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 32256))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 132))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 32256))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 33792))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 144))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 33792))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 35328))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 156))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 35328))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 36864))] = max(((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 168))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + (((int)threadIdx.x) * 4)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 2) * 46080) + (((((int)blockIdx.x) & 3) >> 1) * 7680)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) * 4)) + ax3_inner) + 36864))]), 0.000000e+00f);
    }
  }
}


