
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[256];
  __shared__ float placeholder_shared[24];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 32))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 64))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 68))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 72))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 76))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 80))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 84))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 88))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 92))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 65))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 69))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 73))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 77))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 81))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 85))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 25))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 89))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 29))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 93))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 66))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 70))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 74))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 78))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 82))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 22))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 86))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 26))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 90))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 30))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 94))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 67))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 71))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 75))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 79))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 83))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 23))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 87))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 27))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 91))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 31))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 32) + 95))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 4))] = placeholder[((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 16))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 17))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 48))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 24576))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 49))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 24577))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 50))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 24592))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 51))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 24593))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 96))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 49152))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 97))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 49153))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 98))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 49168))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 99))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 49169))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 144))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 73728))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 145))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 73729))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 146))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 73744))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 147))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 73745))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 192))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 98304))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 193))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 98305))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 194))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 98320))];
    PaddedInput_shared[(((((int)threadIdx.x) * 4) + 195))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 98321))];
    if (((int)threadIdx.x) < 4) {
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 240))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 122880))];
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 241))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 122881))];
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 242))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 122896))];
      PaddedInput_shared[(((((int)threadIdx.x) * 4) + 243))] = placeholder[(((((((((int)blockIdx.x) >> 9) * 131072) + (((int)threadIdx.x) * 2048)) + (((((int)blockIdx.x) & 511) >> 3) * 32)) + (rc_outer_outer * 2)) + 122897))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 192) + ((((int)blockIdx.x) & 7) * 12)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 12))] = placeholder1[(((((rc_outer_outer * 192) + ((((int)blockIdx.x) & 7) * 12)) + ((int)threadIdx.x)) + 96))];
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
        Conv2dOutput[(((yy_outer_inner * 32) + (xx_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 32) + (xx_outer_inner * 2)))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 64))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 128))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 4))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 4))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 68))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 68))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 132))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 8))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 8))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 72))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 72))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 136))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 12))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 12))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 76))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 76))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 140))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 16))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 16))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 16))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 80))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 80))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 144))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 20))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 20))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 84))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 84))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 148))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 24))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 24))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 24))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 88))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 88))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 152))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 28))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 28))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 28))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 92))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 92))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 156))] * placeholder_shared[(((((int)threadIdx.x) % 6) * 2))]));
        Conv2dOutput[(((yy_outer_inner * 32) + (xx_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 32) + (xx_outer_inner * 2)))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 64))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 4))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 68))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 68))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 8))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 72))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 72))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 137))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 12))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 13))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 76))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 76))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 141))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 16))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 16))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 80))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 80))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 145))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 20))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 21))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 84))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 84))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 149))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 24))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 24))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 25))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 88))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 88))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 153))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 28))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 28))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 29))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 92))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 92))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 157))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 65))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 128))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 5))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 69))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 69))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 132))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 9))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 9))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 73))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 73))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 136))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 13))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 77))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 77))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 140))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 17))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 17))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 81))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 81))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 144))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 21))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 85))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 85))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 148))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 25))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 25))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 89))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 89))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 152))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 29))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 29))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 93))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 93))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 156))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 65))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 129))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 5))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 69))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 69))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 133))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 9))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 9))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 73))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 73))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 137))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 13))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 13))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 77))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 77))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 141))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 17))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 17))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 81))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 81))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 145))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 21))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 21))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 85))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 85))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 149))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 25))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 25))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 25))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 89))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 89))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 153))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 29))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 29))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 29))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 93))] = (Conv2dOutput[((((yy_outer_inner * 32) + (xx_outer_inner * 2)) + 93))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 64) + (yy_outer_inner * 32)) + (xx_outer_inner * 2)) + 157))] * placeholder_shared[((((((int)threadIdx.x) % 6) * 2) + 13))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 16; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 2; ++i3_inner) {
        compute[((((((((((((int)blockIdx.x) >> 9) * 786432) + ((((int)threadIdx.x) / 6) * 196608)) + (i1_inner * 12288)) + (((((int)blockIdx.x) & 511) >> 3) * 192)) + (i2_inner * 96)) + ((((int)blockIdx.x) & 7) * 12)) + ((((int)threadIdx.x) % 6) * 2)) + i3_inner))] = max(min((Conv2dOutput[((((i1_inner * 4) + (i2_inner * 2)) + i3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 12) + ((((int)threadIdx.x) % 6) * 2)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
        compute[(((((((((((((int)blockIdx.x) >> 9) * 786432) + ((((int)threadIdx.x) / 6) * 196608)) + (i1_inner * 12288)) + (((((int)blockIdx.x) & 511) >> 3) * 192)) + (i2_inner * 96)) + ((((int)blockIdx.x) & 7) * 12)) + ((((int)threadIdx.x) % 6) * 2)) + i3_inner) + 393216))] = max(min((Conv2dOutput[(((((i1_inner * 4) + (i2_inner * 2)) + i3_inner) + 64))] + placeholder2[(((((((int)blockIdx.x) & 7) * 12) + ((((int)threadIdx.x) % 6) * 2)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      }
    }
  }
}


