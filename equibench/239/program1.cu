
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[448];
  __shared__ float placeholder_shared[512];
  for (int yy_inner_init = 0; yy_inner_init < 4; ++yy_inner_init) {
    Conv2dOutput[((yy_inner_init * 32))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 15))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 22))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 23))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 25))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 26))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 27))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 29))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 30))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 32) + 31))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    ((float4*)(PaddedInput_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder + (((((((((int)blockIdx.x) >> 2) * 57344) + ((((int)blockIdx.x) & 3) * 3584)) + ((((int)threadIdx.x) >> 1) * 256)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)))))[0];
    ((float4*)(PaddedInput_shared + (((((int)threadIdx.x) * 4) + 112))))[0] = ((float4*)(placeholder + ((((((((((int)blockIdx.x) >> 2) * 57344) + ((((int)blockIdx.x) & 3) * 3584)) + ((((int)threadIdx.x) >> 1) * 256)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 14336))))[0];
    ((float4*)(PaddedInput_shared + (((((int)threadIdx.x) * 4) + 224))))[0] = ((float4*)(placeholder + ((((((((((int)blockIdx.x) >> 2) * 57344) + ((((int)blockIdx.x) & 3) * 3584)) + ((((int)threadIdx.x) >> 1) * 256)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 28672))))[0];
    ((float4*)(PaddedInput_shared + (((((int)threadIdx.x) * 4) + 336))))[0] = ((float4*)(placeholder + ((((((((((int)blockIdx.x) >> 2) * 57344) + ((((int)blockIdx.x) & 3) * 3584)) + ((((int)threadIdx.x) >> 1) * 256)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 43008))))[0];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 512) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 28))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 28))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 56))];
    placeholder_shared[((((int)threadIdx.x) + 84))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 84))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 112))];
    placeholder_shared[((((int)threadIdx.x) + 140))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 140))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 168))];
    placeholder_shared[((((int)threadIdx.x) + 196))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 196))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 224))];
    placeholder_shared[((((int)threadIdx.x) + 252))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 252))];
    placeholder_shared[((((int)threadIdx.x) + 280))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 280))];
    placeholder_shared[((((int)threadIdx.x) + 308))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 308))];
    placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 336))];
    placeholder_shared[((((int)threadIdx.x) + 364))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 364))];
    placeholder_shared[((((int)threadIdx.x) + 392))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 392))];
    placeholder_shared[((((int)threadIdx.x) + 420))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 420))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 448))];
    placeholder_shared[((((int)threadIdx.x) + 476))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 476))];
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[((((int)threadIdx.x) + 504))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 504))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int yy_inner = 0; yy_inner < 4; ++yy_inner) {
        Conv2dOutput[((yy_inner * 32))] = (Conv2dOutput[((yy_inner * 32))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[(((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)))]));
        Conv2dOutput[(((yy_inner * 32) + 1))] = (Conv2dOutput[(((yy_inner * 32) + 1))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 1))]));
        Conv2dOutput[(((yy_inner * 32) + 2))] = (Conv2dOutput[(((yy_inner * 32) + 2))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 2))]));
        Conv2dOutput[(((yy_inner * 32) + 3))] = (Conv2dOutput[(((yy_inner * 32) + 3))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 3))]));
        Conv2dOutput[(((yy_inner * 32) + 4))] = (Conv2dOutput[(((yy_inner * 32) + 4))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 4))]));
        Conv2dOutput[(((yy_inner * 32) + 5))] = (Conv2dOutput[(((yy_inner * 32) + 5))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 5))]));
        Conv2dOutput[(((yy_inner * 32) + 6))] = (Conv2dOutput[(((yy_inner * 32) + 6))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 6))]));
        Conv2dOutput[(((yy_inner * 32) + 7))] = (Conv2dOutput[(((yy_inner * 32) + 7))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 7))]));
        Conv2dOutput[(((yy_inner * 32) + 8))] = (Conv2dOutput[(((yy_inner * 32) + 8))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 8))]));
        Conv2dOutput[(((yy_inner * 32) + 9))] = (Conv2dOutput[(((yy_inner * 32) + 9))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 9))]));
        Conv2dOutput[(((yy_inner * 32) + 10))] = (Conv2dOutput[(((yy_inner * 32) + 10))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 10))]));
        Conv2dOutput[(((yy_inner * 32) + 11))] = (Conv2dOutput[(((yy_inner * 32) + 11))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 11))]));
        Conv2dOutput[(((yy_inner * 32) + 12))] = (Conv2dOutput[(((yy_inner * 32) + 12))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 12))]));
        Conv2dOutput[(((yy_inner * 32) + 13))] = (Conv2dOutput[(((yy_inner * 32) + 13))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 13))]));
        Conv2dOutput[(((yy_inner * 32) + 14))] = (Conv2dOutput[(((yy_inner * 32) + 14))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 14))]));
        Conv2dOutput[(((yy_inner * 32) + 15))] = (Conv2dOutput[(((yy_inner * 32) + 15))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 15))]));
        Conv2dOutput[(((yy_inner * 32) + 16))] = (Conv2dOutput[(((yy_inner * 32) + 16))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 16))]));
        Conv2dOutput[(((yy_inner * 32) + 17))] = (Conv2dOutput[(((yy_inner * 32) + 17))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 17))]));
        Conv2dOutput[(((yy_inner * 32) + 18))] = (Conv2dOutput[(((yy_inner * 32) + 18))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 18))]));
        Conv2dOutput[(((yy_inner * 32) + 19))] = (Conv2dOutput[(((yy_inner * 32) + 19))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 19))]));
        Conv2dOutput[(((yy_inner * 32) + 20))] = (Conv2dOutput[(((yy_inner * 32) + 20))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 20))]));
        Conv2dOutput[(((yy_inner * 32) + 21))] = (Conv2dOutput[(((yy_inner * 32) + 21))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 21))]));
        Conv2dOutput[(((yy_inner * 32) + 22))] = (Conv2dOutput[(((yy_inner * 32) + 22))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 22))]));
        Conv2dOutput[(((yy_inner * 32) + 23))] = (Conv2dOutput[(((yy_inner * 32) + 23))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 23))]));
        Conv2dOutput[(((yy_inner * 32) + 24))] = (Conv2dOutput[(((yy_inner * 32) + 24))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 24))]));
        Conv2dOutput[(((yy_inner * 32) + 25))] = (Conv2dOutput[(((yy_inner * 32) + 25))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 25))]));
        Conv2dOutput[(((yy_inner * 32) + 26))] = (Conv2dOutput[(((yy_inner * 32) + 26))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 26))]));
        Conv2dOutput[(((yy_inner * 32) + 27))] = (Conv2dOutput[(((yy_inner * 32) + 27))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 27))]));
        Conv2dOutput[(((yy_inner * 32) + 28))] = (Conv2dOutput[(((yy_inner * 32) + 28))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 28))]));
        Conv2dOutput[(((yy_inner * 32) + 29))] = (Conv2dOutput[(((yy_inner * 32) + 29))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 29))]));
        Conv2dOutput[(((yy_inner * 32) + 30))] = (Conv2dOutput[(((yy_inner * 32) + 30))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 30))]));
        Conv2dOutput[(((yy_inner * 32) + 31))] = (Conv2dOutput[(((yy_inner * 32) + 31))] + (PaddedInput_shared[((((yy_inner * 112) + ((((int)threadIdx.x) >> 1) * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 1) * 32)) + 31))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
      T_relu[(((((((((int)blockIdx.x) >> 2) * 14336) + (ax1_inner * 3584)) + ((((int)blockIdx.x) & 3) * 896)) + (((int)threadIdx.x) * 32)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 32) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 32) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


