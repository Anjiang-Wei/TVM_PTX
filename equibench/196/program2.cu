
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ tensor, float* __restrict__ tensor1) {
  tensor[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = (tensor1[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] * 3.906250e-03f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 16; ++rv0) {
    for (int rv1 = 0; rv1 < 16; ++rv1) {
      tensor[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = (tensor[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 672) * 172032) + (rv0 * 10752)) + (rv1 * 672)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 672)))]);
    }
  }
}


