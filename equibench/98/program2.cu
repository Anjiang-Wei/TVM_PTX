
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[6912];
  __shared__ float PaddedInput_shared[340992];
  __shared__ float placeholder_shared[147456];
  for (int dd_outer_inner_init = 0; dd_outer_inner_init < 6; ++dd_outer_inner_init) {
    for (int hh_outer_inner_init = 0; hh_outer_inner_init < 2; ++hh_outer_inner_init) {
      for (int ww_outer_inner_init = 0; ww_outer_inner_init < 2; ++ww_outer_inner_init) {
        for (int cc_outer_inner_init = 0; cc_outer_inner_init < 16; ++cc_outer_inner_init) {
          for (int hh_inner_init = 0; hh_inner_init < 3; ++hh_inner_init) {
            Conv3dOutput[((((((dd_outer_inner_init * 192) + (hh_outer_inner_init * 96)) + (hh_inner_init * 32)) + (ww_outer_inner_init * 16)) + cc_outer_inner_init))] = 0.000000e+00f;
            Conv3dOutput[(((((((dd_outer_inner_init * 192) + (hh_outer_inner_init * 96)) + (hh_inner_init * 32)) + (ww_outer_inner_init * 16)) + cc_outer_inner_init) + 1152))] = 0.000000e+00f;
            Conv3dOutput[(((((((dd_outer_inner_init * 192) + (hh_outer_inner_init * 96)) + (hh_inner_init * 32)) + (ww_outer_inner_init * 16)) + cc_outer_inner_init) + 2304))] = 0.000000e+00f;
            Conv3dOutput[(((((((dd_outer_inner_init * 192) + (hh_outer_inner_init * 96)) + (hh_inner_init * 32)) + (ww_outer_inner_init * 16)) + cc_outer_inner_init) + 3456))] = 0.000000e+00f;
            Conv3dOutput[(((((((dd_outer_inner_init * 192) + (hh_outer_inner_init * 96)) + (hh_inner_init * 32)) + (ww_outer_inner_init * 16)) + cc_outer_inner_init) + 4608))] = 0.000000e+00f;
            Conv3dOutput[(((((((dd_outer_inner_init * 192) + (hh_outer_inner_init * 96)) + (hh_inner_init * 32)) + (ww_outer_inner_init * 16)) + cc_outer_inner_init) + 5760))] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rh_outer_outer = 0; rh_outer_outer < 3; ++rh_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 10656; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)))] = (((((((4608 <= ((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x))) && (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) < 336384)) && (1 <= (((((int)blockIdx.x) * 6) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) % 4608) / 768)) + rh_outer_outer))) && ((((((int)blockIdx.x) * 6) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) % 4608) / 768)) + rh_outer_outer) < 37)) && (128 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) % 768))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) % 768) < 640)) ? placeholder[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) / 4608) * 18432) + (((int)blockIdx.x) * 3072)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) % 4608) / 768) * 512)) + (rh_outer_outer * 512)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 32) + ((int)threadIdx.x)) % 768)) - 19072))] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 < 4608; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)) / 49152) * 147456) + (rh_outer_outer * 49152)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 32) + ((int)threadIdx.x)) % 49152)))];
    }
    __syncthreads();
    for (int rd_outer_inner = 0; rd_outer_inner < 3; ++rd_outer_inner) {
      for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
        for (int dd_outer_inner = 0; dd_outer_inner < 6; ++dd_outer_inner) {
          for (int hh_outer_inner = 0; hh_outer_inner < 2; ++hh_outer_inner) {
            for (int ww_outer_inner = 0; ww_outer_inner < 2; ++ww_outer_inner) {
              for (int cc_outer_inner = 0; cc_outer_inner < 16; ++cc_outer_inner) {
                for (int rw_inner = 0; rw_inner < 3; ++rw_inner) {
                  for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
                    for (int hh_inner = 0; hh_inner < 3; ++hh_inner) {
                      Conv3dOutput[((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner))] = (Conv3dOutput[((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 3) * 27648) + (dd_outer_inner * 4608)) + (rd_outer_inner * 4608)) + (hh_outer_inner * 2304)) + (hh_inner * 768)) + (ww_outer_inner * 128)) + (rw_inner * 128)) + (rc_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((((rd_outer_inner * 49152) + (rw_inner * 16384)) + (rc_outer_inner * 1024)) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + cc_outer_inner))]));
                      Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 1152))] = (Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 1152))] + (PaddedInput_shared[((((((((((((((int)threadIdx.x) >> 3) * 27648) + (dd_outer_inner * 4608)) + (rd_outer_inner * 4608)) + (hh_outer_inner * 2304)) + (hh_inner * 768)) + (ww_outer_inner * 128)) + (rw_inner * 128)) + (rc_outer_inner * 8)) + rc_inner) + 256))] * placeholder_shared[(((((((rd_outer_inner * 49152) + (rw_inner * 16384)) + (rc_outer_inner * 1024)) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + cc_outer_inner))]));
                      Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 2304))] = (Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 2304))] + (PaddedInput_shared[((((((((((((((int)threadIdx.x) >> 3) * 27648) + (dd_outer_inner * 4608)) + (rd_outer_inner * 4608)) + (hh_outer_inner * 2304)) + (hh_inner * 768)) + (ww_outer_inner * 128)) + (rw_inner * 128)) + (rc_outer_inner * 8)) + rc_inner) + 110592))] * placeholder_shared[(((((((rd_outer_inner * 49152) + (rw_inner * 16384)) + (rc_outer_inner * 1024)) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + cc_outer_inner))]));
                      Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 3456))] = (Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 3456))] + (PaddedInput_shared[((((((((((((((int)threadIdx.x) >> 3) * 27648) + (dd_outer_inner * 4608)) + (rd_outer_inner * 4608)) + (hh_outer_inner * 2304)) + (hh_inner * 768)) + (ww_outer_inner * 128)) + (rw_inner * 128)) + (rc_outer_inner * 8)) + rc_inner) + 110848))] * placeholder_shared[(((((((rd_outer_inner * 49152) + (rw_inner * 16384)) + (rc_outer_inner * 1024)) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + cc_outer_inner))]));
                      Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 4608))] = (Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 4608))] + (PaddedInput_shared[((((((((((((((int)threadIdx.x) >> 3) * 27648) + (dd_outer_inner * 4608)) + (rd_outer_inner * 4608)) + (hh_outer_inner * 2304)) + (hh_inner * 768)) + (ww_outer_inner * 128)) + (rw_inner * 128)) + (rc_outer_inner * 8)) + rc_inner) + 221184))] * placeholder_shared[(((((((rd_outer_inner * 49152) + (rw_inner * 16384)) + (rc_outer_inner * 1024)) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + cc_outer_inner))]));
                      Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 5760))] = (Conv3dOutput[(((((((dd_outer_inner * 192) + (hh_outer_inner * 96)) + (hh_inner * 32)) + (ww_outer_inner * 16)) + cc_outer_inner) + 5760))] + (PaddedInput_shared[((((((((((((((int)threadIdx.x) >> 3) * 27648) + (dd_outer_inner * 4608)) + (rd_outer_inner * 4608)) + (hh_outer_inner * 2304)) + (hh_inner * 768)) + (ww_outer_inner * 128)) + (rw_inner * 128)) + (rc_outer_inner * 8)) + rc_inner) + 221440))] * placeholder_shared[(((((((rd_outer_inner * 49152) + (rw_inner * 16384)) + (rc_outer_inner * 1024)) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + cc_outer_inner))]));
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 6; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 6; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        for (int ax4_inner = 0; ax4_inner < 16; ++ax4_inner) {
          T_relu[(((((((((((int)threadIdx.x) >> 3) * 110592) + (ax1_inner * 18432)) + (((int)blockIdx.x) * 3072)) + (ax2_inner * 512)) + (ax3_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax4_inner))] = max(((Conv3dOutput[(((((ax1_inner * 192) + (ax2_inner * 32)) + (ax3_inner * 16)) + ax4_inner))] * placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]), 0.000000e+00f);
          T_relu[((((((((((((int)threadIdx.x) >> 3) * 110592) + (ax1_inner * 18432)) + (((int)blockIdx.x) * 3072)) + (ax2_inner * 512)) + (ax3_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax4_inner) + 256))] = max(((Conv3dOutput[((((((ax1_inner * 192) + (ax2_inner * 32)) + (ax3_inner * 16)) + ax4_inner) + 1152))] * placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]), 0.000000e+00f);
          T_relu[((((((((((((int)threadIdx.x) >> 3) * 110592) + (ax1_inner * 18432)) + (((int)blockIdx.x) * 3072)) + (ax2_inner * 512)) + (ax3_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax4_inner) + 442368))] = max(((Conv3dOutput[((((((ax1_inner * 192) + (ax2_inner * 32)) + (ax3_inner * 16)) + ax4_inner) + 2304))] * placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]), 0.000000e+00f);
          T_relu[((((((((((((int)threadIdx.x) >> 3) * 110592) + (ax1_inner * 18432)) + (((int)blockIdx.x) * 3072)) + (ax2_inner * 512)) + (ax3_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax4_inner) + 442624))] = max(((Conv3dOutput[((((((ax1_inner * 192) + (ax2_inner * 32)) + (ax3_inner * 16)) + ax4_inner) + 3456))] * placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]), 0.000000e+00f);
          T_relu[((((((((((((int)threadIdx.x) >> 3) * 110592) + (ax1_inner * 18432)) + (((int)blockIdx.x) * 3072)) + (ax2_inner * 512)) + (ax3_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax4_inner) + 884736))] = max(((Conv3dOutput[((((((ax1_inner * 192) + (ax2_inner * 32)) + (ax3_inner * 16)) + ax4_inner) + 4608))] * placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]), 0.000000e+00f);
          T_relu[((((((((((((int)threadIdx.x) >> 3) * 110592) + (ax1_inner * 18432)) + (((int)blockIdx.x) * 3072)) + (ax2_inner * 512)) + (ax3_inner * 128)) + ((((int)threadIdx.x) & 7) * 16)) + ax4_inner) + 884992))] = max(((Conv3dOutput[((((((ax1_inner * 192) + (ax2_inner * 32)) + (ax3_inner * 16)) + ax4_inner) + 5760))] * placeholder2[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 7) * 16) + ax4_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


