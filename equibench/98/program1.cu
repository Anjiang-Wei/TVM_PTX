
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[192];
  __shared__ float PaddedInput_shared[576];
  __shared__ float placeholder_shared[256];
  for (int dd_outer_inner_init = 0; dd_outer_inner_init < 2; ++dd_outer_inner_init) {
    for (int hh_outer_inner_init = 0; hh_outer_inner_init < 6; ++hh_outer_inner_init) {
      for (int cc_outer_inner_init = 0; cc_outer_inner_init < 2; ++cc_outer_inner_init) {
        for (int cc_inner_init = 0; cc_inner_init < 4; ++cc_inner_init) {
          Conv3dOutput[(((((dd_outer_inner_init * 48) + (hh_outer_inner_init * 8)) + (cc_outer_inner_init * 4)) + cc_inner_init))] = 0.000000e+00f;
          Conv3dOutput[((((((dd_outer_inner_init * 48) + (hh_outer_inner_init * 8)) + (cc_outer_inner_init * 4)) + cc_inner_init) + 96))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rd_outer_outer = 0; rd_outer_outer < 3; ++rd_outer_outer) {
    for (int rh_outer_outer = 0; rh_outer_outer < 3; ++rh_outer_outer) {
      for (int rw_outer_outer = 0; rw_outer_outer < 3; ++rw_outer_outer) {
        for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
          __syncthreads();
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) {
            if (((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) < 288) {
              if (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s) < 576) {
                if (((int)threadIdx.x) < 144) {
                  PaddedInput_shared[(((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s))] = (((((((1 <= ((((((int)blockIdx.x) / 6) * 12) + (((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) / 24)) + rd_outer_outer)) && (((((((int)blockIdx.x) / 6) * 12) + (((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) / 24)) + rd_outer_outer) < 73)) && (1 <= ((((((int)blockIdx.x) % 6) * 6) + ((((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) % 24) >> 2)) + rh_outer_outer))) && (((((((int)blockIdx.x) % 6) * 6) + ((((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) % 24) >> 2)) + rh_outer_outer) < 37)) && (1 <= (rw_outer_outer + (((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) & 3)))) && ((rw_outer_outer + (((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) & 3)) < 5)) ? placeholder[((((((((((((((int)blockIdx.x) / 6) * 221184) + ((((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) / 24) * 18432)) + (rd_outer_outer * 18432)) + ((((int)blockIdx.x) % 6) * 3072)) + (rh_outer_outer * 512)) + (rw_outer_outer * 128)) + ((((((int)threadIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s >> 1)) % 24) * 128)) + (rc_outer_outer * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_inner_s & 1)) - 19072))] : 0.000000e+00f);
                }
              }
            }
          }
          for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
            if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 192) + ((int)threadIdx.x)) < 256) {
              placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 192) + ((int)threadIdx.x)))] = placeholder1[(((((((rd_outer_outer * 147456) + (rh_outer_outer * 49152)) + (rw_outer_outer * 16384)) + (rc_outer_outer * 256)) + (ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 192)) + ((int)threadIdx.x)))];
            }
          }
          __syncthreads();
          for (int dd_outer_inner = 0; dd_outer_inner < 2; ++dd_outer_inner) {
            for (int hh_outer_inner = 0; hh_outer_inner < 6; ++hh_outer_inner) {
              for (int cc_outer_inner = 0; cc_outer_inner < 2; ++cc_outer_inner) {
                for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
                  for (int cc_inner = 0; cc_inner < 4; ++cc_inner) {
                    Conv3dOutput[(((((dd_outer_inner * 48) + (hh_outer_inner * 8)) + (cc_outer_inner * 4)) + cc_inner))] = (Conv3dOutput[(((((dd_outer_inner * 48) + (hh_outer_inner * 8)) + (cc_outer_inner * 4)) + cc_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 96) + (dd_outer_inner * 48)) + (hh_outer_inner * 8)) + (((((int)threadIdx.x) & 31) >> 4) * 2)) + rc_inner))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + (cc_outer_inner * 4)) + cc_inner))]));
                    Conv3dOutput[((((((dd_outer_inner * 48) + (hh_outer_inner * 8)) + (cc_outer_inner * 4)) + cc_inner) + 96))] = (Conv3dOutput[((((((dd_outer_inner * 48) + (hh_outer_inner * 8)) + (cc_outer_inner * 4)) + cc_inner) + 96))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 96) + (dd_outer_inner * 48)) + (hh_outer_inner * 8)) + (((((int)threadIdx.x) & 31) >> 4) * 2)) + rc_inner) + 4))] * placeholder_shared[(((((rc_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + (cc_outer_inner * 4)) + cc_inner))]));
                  }
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 6; ++ax2_inner) {
      for (int ax4_inner = 0; ax4_inner < 8; ++ax4_inner) {
        T_relu[(((((((((((int)blockIdx.x) / 6) * 221184) + ((((int)threadIdx.x) >> 5) * 36864)) + (ax1_inner * 18432)) + ((((int)blockIdx.x) % 6) * 3072)) + (ax2_inner * 512)) + ((((int)threadIdx.x) & 31) * 8)) + ax4_inner))] = max(((Conv3dOutput[((((ax1_inner * 48) + (ax2_inner * 8)) + ax4_inner))] * placeholder2[((((((int)threadIdx.x) & 15) * 8) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 15) * 8) + ax4_inner))]), 0.000000e+00f);
        T_relu[((((((((((((int)blockIdx.x) / 6) * 221184) + ((((int)threadIdx.x) >> 5) * 36864)) + (ax1_inner * 18432)) + ((((int)blockIdx.x) % 6) * 3072)) + (ax2_inner * 512)) + ((((int)threadIdx.x) & 31) * 8)) + ax4_inner) + 256))] = max(((Conv3dOutput[(((((ax1_inner * 48) + (ax2_inner * 8)) + ax4_inner) + 96))] * placeholder2[((((((int)threadIdx.x) & 15) * 8) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 15) * 8) + ax4_inner))]), 0.000000e+00f);
      }
    }
  }
}


