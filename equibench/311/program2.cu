
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_divide, float* __restrict__ T_multiply_red) {
  T_divide[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] = (T_multiply_red[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] * 9.765625e-04f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ T_multiply_red, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
  T_multiply_red[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int k2 = 0; k2 < 1024; ++k2) {
    T_multiply_red[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] = (T_multiply_red[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] + ((placeholder[((((((int)blockIdx.x) * 8192) + (((int)threadIdx.x) * 1024)) + k2))] - placeholder1[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))]) * (placeholder[((((((int)blockIdx.x) * 8192) + (((int)threadIdx.x) * 1024)) + k2))] - placeholder1[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))])));
  }
}


