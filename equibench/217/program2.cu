
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[36];
  __shared__ float PaddedInput_shared[9];
  __shared__ float placeholder_shared[128];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 608; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 9) {
      PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)blockIdx.x) / 5) * 27360) + ((((int)threadIdx.x) / 3) * 9120)) + ((((int)blockIdx.x) % 5) * 1824)) + ((((int)threadIdx.x) % 3) * 608)) + rc_outer_outer))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 128) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 32))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 64))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 96))];
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(1)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(2)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(3)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(4)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(5)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(6)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(7)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(8)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((int)blockIdx.x) / 5) * 5760) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 6) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) / 5) * 5760) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 64))] = max((Conv2dOutput[(((((ax1_inner * 6) + (ax2_inner * 2)) + ax3_inner) + 18))] + placeholder2[((((((int)threadIdx.x) * 2) + ax3_inner) + 64))]), 0.000000e+00f);
      }
    }
  }
}


