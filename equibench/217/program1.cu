
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[120];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float placeholder_shared[2048];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 15; ++xx_outer_inner_init) {
    Conv2dOutput[(xx_outer_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 15))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 30))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 45))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 60))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 75))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 90))] = 0.000000e+00f;
    Conv2dOutput[((xx_outer_inner_init + 105))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 36; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 240))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 8640))];
    PaddedInput_shared[((((int)threadIdx.x) + 480))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 17280))];
    PaddedInput_shared[((((int)threadIdx.x) + 720))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 25920))];
    PaddedInput_shared[((((int)threadIdx.x) + 960))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 34560))];
    PaddedInput_shared[((((int)threadIdx.x) + 1200))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 43200))];
    PaddedInput_shared[((((int)threadIdx.x) + 1440))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 51840))];
    PaddedInput_shared[((((int)threadIdx.x) + 1680))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 60480))];
    PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 69120))];
    PaddedInput_shared[((((int)threadIdx.x) + 2160))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 77760))];
    PaddedInput_shared[((((int)threadIdx.x) + 2400))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 86400))];
    PaddedInput_shared[((((int)threadIdx.x) + 2640))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 95040))];
    PaddedInput_shared[((((int)threadIdx.x) + 2880))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 103680))];
    PaddedInput_shared[((((int)threadIdx.x) + 3120))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 112320))];
    PaddedInput_shared[((((int)threadIdx.x) + 3360))] = placeholder[((((((((int)threadIdx.x) >> 4) * 576) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)) + 120960))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 2048) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 240))];
    placeholder_shared[((((int)threadIdx.x) + 480))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 480))];
    placeholder_shared[((((int)threadIdx.x) + 720))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 720))];
    placeholder_shared[((((int)threadIdx.x) + 960))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 960))];
    placeholder_shared[((((int)threadIdx.x) + 1200))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1200))];
    placeholder_shared[((((int)threadIdx.x) + 1440))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1440))];
    placeholder_shared[((((int)threadIdx.x) + 1680))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1680))];
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1920))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 15; ++xx_outer_inner) {
        Conv2dOutput[(xx_outer_inner)] = (Conv2dOutput[(xx_outer_inner)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)))]));
        Conv2dOutput[((xx_outer_inner + 15))] = (Conv2dOutput[((xx_outer_inner + 15))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 16))]));
        Conv2dOutput[((xx_outer_inner + 30))] = (Conv2dOutput[((xx_outer_inner + 30))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 32))]));
        Conv2dOutput[((xx_outer_inner + 45))] = (Conv2dOutput[((xx_outer_inner + 45))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 48))]));
        Conv2dOutput[((xx_outer_inner + 60))] = (Conv2dOutput[((xx_outer_inner + 60))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 64))]));
        Conv2dOutput[((xx_outer_inner + 75))] = (Conv2dOutput[((xx_outer_inner + 75))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 80))]));
        Conv2dOutput[((xx_outer_inner + 90))] = (Conv2dOutput[((xx_outer_inner + 90))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 96))]));
        Conv2dOutput[((xx_outer_inner + 105))] = (Conv2dOutput[((xx_outer_inner + 105))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 240) + (xx_outer_inner * 16)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (((int)threadIdx.x) & 15)) + 112))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
    T_relu[(((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)))] = max((Conv2dOutput[(ax2_inner)] + placeholder2[((((int)threadIdx.x) & 15))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 16))] = max((Conv2dOutput[((ax2_inner + 15))] + placeholder2[(((((int)threadIdx.x) & 15) + 16))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 32))] = max((Conv2dOutput[((ax2_inner + 30))] + placeholder2[(((((int)threadIdx.x) & 15) + 32))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 48))] = max((Conv2dOutput[((ax2_inner + 45))] + placeholder2[(((((int)threadIdx.x) & 15) + 48))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 64))] = max((Conv2dOutput[((ax2_inner + 60))] + placeholder2[(((((int)threadIdx.x) & 15) + 64))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 80))] = max((Conv2dOutput[((ax2_inner + 75))] + placeholder2[(((((int)threadIdx.x) & 15) + 80))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 96))] = max((Conv2dOutput[((ax2_inner + 90))] + placeholder2[(((((int)threadIdx.x) & 15) + 96))]), 0.000000e+00f);
    T_relu[((((((((int)threadIdx.x) >> 4) * 1920) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 112))] = max((Conv2dOutput[((ax2_inner + 105))] + placeholder2[(((((int)threadIdx.x) & 15) + 112))]), 0.000000e+00f);
  }
}


