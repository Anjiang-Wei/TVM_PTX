
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float Conv2dOutput[24];
  __shared__ float PaddedInput_shared[960];
  __shared__ float placeholder_shared[3072];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  ((float4*)(PaddedInput_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder + ((((((((int)blockIdx.x) / 20) * 9600) + ((((int)threadIdx.x) / 24) * 960)) + (((((int)blockIdx.x) % 20) >> 1) * 96)) + ((((int)threadIdx.x) % 24) * 4)))))[0];
  ((float4*)(PaddedInput_shared + (((((int)threadIdx.x) * 4) + 480))))[0] = ((float4*)(placeholder + (((((((((int)blockIdx.x) / 20) * 9600) + ((((int)threadIdx.x) / 24) * 960)) + (((((int)blockIdx.x) % 20) >> 1) * 96)) + ((((int)threadIdx.x) % 24) * 4)) + 4800))))[0];
  placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[(((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 240))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 240) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 48) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 241))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 241) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 49) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 480))] = placeholder1[((((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)) + 960))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 481))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 481) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 720))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 720) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 48) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 721))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 721) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 49) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 960))] = placeholder1[((((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)) + 1920))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 961))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 961) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1200))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1200) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 48) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1201))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1201) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 49) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1440))] = placeholder1[((((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)) + 2880))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1441))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1441) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1680))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1680) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 48) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1681))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1681) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 49) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1920))] = placeholder1[((((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)) + 3840))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 1921))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 1921) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 2160))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 2160) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 48) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 2161))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 2161) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 49) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 2400))] = placeholder1[((((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)) + 4800))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 2401))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 2401) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 2640))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 2640) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 48) % 96)))];
  placeholder_shared[(((((int)threadIdx.x) * 2) + 2641))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 2641) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 49) % 96)))];
  if (((int)threadIdx.x) < 96) {
    placeholder_shared[(((((int)threadIdx.x) * 2) + 2880))] = placeholder1[((((((((int)threadIdx.x) / 48) * 192) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) % 48) * 2)) + 5760))];
  }
  if (((int)threadIdx.x) < 96) {
    placeholder_shared[(((((int)threadIdx.x) * 2) + 2881))] = placeholder1[(((((((((int)threadIdx.x) * 2) + 2881) / 96) * 192) + ((((int)blockIdx.x) & 1) * 96)) + (((((int)threadIdx.x) * 2) + 1) % 96)))];
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
    for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 12))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 24))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 36))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 48))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 60))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 72))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 84))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 1))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 13))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 25))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 37))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 49))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 61))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 73))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 85))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 2))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 14))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 26))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 38))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 50))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 62))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 74))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 2) * 32) + (rc_outer_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 384) + (rc_inner * 96)) + ((((int)threadIdx.x) & 3) * 3)) + 86))]));
    }
  }
  for (int i3_inner = 0; i3_inner < 3; ++i3_inner) {
    compute[(((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner))] = max(min((Conv2dOutput[(i3_inner)] + placeholder2[(((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 12))] = max(min((Conv2dOutput[((i3_inner + 3))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 12))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 24))] = max(min((Conv2dOutput[((i3_inner + 6))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 24))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 36))] = max(min((Conv2dOutput[((i3_inner + 9))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 36))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 48))] = max(min((Conv2dOutput[((i3_inner + 12))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 48))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 60))] = max(min((Conv2dOutput[((i3_inner + 15))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 60))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 72))] = max(min((Conv2dOutput[((i3_inner + 18))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 72))]), 6.000000e+00f), 0.000000e+00f);
    compute[((((((((((((int)blockIdx.x) / 20) * 57600) + ((((int)threadIdx.x) / 12) * 5760)) + (((((int)blockIdx.x) % 20) >> 1) * 576)) + (((((int)threadIdx.x) % 12) >> 2) * 192)) + ((((int)blockIdx.x) & 1) * 96)) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 84))] = max(min((Conv2dOutput[((i3_inner + 21))] + placeholder2[((((((((int)blockIdx.x) & 1) * 96) + ((((int)threadIdx.x) & 3) * 3)) + i3_inner) + 84))]), 6.000000e+00f), 0.000000e+00f);
  }
}


