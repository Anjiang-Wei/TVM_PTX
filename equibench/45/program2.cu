
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[48];
  __shared__ float PaddedInput_shared[360];
  __shared__ float placeholder_shared[8];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 3; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 4; ++nn_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
          Conv2dOutput[(((((nn_inner_init * 12) + (yy_outer_inner_init * 4)) + (yy_inner_init * 2)) + ff_outer_inner_init))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 192; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 360) {
        if (((int)threadIdx.x) < 40) {
          PaddedInput_shared[(((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 90) * 172800) + ((((int)blockIdx.x) >> 3) * 34560)) + (((((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 90) / 15) * 5760)) + (((((int)blockIdx.x) & 7) >> 2) * 2880)) + ((((((int)threadIdx.x) * 9) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 15) * 192)) + rc_outer_outer))];
        }
      }
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 32) + ((((int)blockIdx.x) & 3) * 8)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 3; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int nn_inner = 0; nn_inner < 4; ++nn_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            Conv2dOutput[(((((nn_inner * 12) + (yy_outer_inner * 4)) + (yy_inner * 2)) + ff_outer_inner))] = (Conv2dOutput[(((((nn_inner * 12) + (yy_outer_inner * 4)) + (yy_inner * 2)) + ff_outer_inner))] + (PaddedInput_shared[(((((nn_inner * 90) + (yy_outer_inner * 30)) + (yy_inner * 15)) + (((int)threadIdx.x) >> 2)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 2) + ff_outer_inner))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 6; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[(((((((((ax0_inner * 28800) + ((((int)blockIdx.x) >> 3) * 5760)) + (ax1_inner * 960)) + (((((int)blockIdx.x) & 7) >> 2) * 480)) + ((((int)threadIdx.x) >> 2) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))] = (placeholder2[(((((((((ax0_inner * 28800) + ((((int)blockIdx.x) >> 3) * 5760)) + (ax1_inner * 960)) + (((((int)blockIdx.x) & 7) >> 2) * 480)) + ((((int)threadIdx.x) >> 2) * 32)) + ((((int)blockIdx.x) & 3) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))] + (Conv2dOutput[((((ax0_inner * 12) + (ax1_inner * 2)) + ax3_inner))] + placeholder3[(((((((int)blockIdx.x) & 3) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]));
      }
    }
  }
}


