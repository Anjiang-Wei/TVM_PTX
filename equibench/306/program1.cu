
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float placeholder_shared[8192];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 25))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 29))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)threadIdx.x) / 240) * 4096) + (((((int)threadIdx.x) % 240) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[(((((((((int)threadIdx.x) + 1024) / 240) * 4096) + ((((((int)threadIdx.x) >> 4) + 4) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[(((((((((int)threadIdx.x) + 2048) / 240) * 4096) + ((((((int)threadIdx.x) >> 4) + 8) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    if (((int)threadIdx.x) < 528) {
      PaddedInput_shared[((((int)threadIdx.x) + 3072))] = placeholder[(((((((((int)threadIdx.x) + 3072) / 240) * 4096) + ((((((int)threadIdx.x) >> 4) + 12) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 8192) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 1024))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 2048))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 2048))];
    placeholder_shared[((((int)threadIdx.x) + 3072))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 3072))];
    placeholder_shared[((((int)threadIdx.x) + 4096))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 4096))];
    placeholder_shared[((((int)threadIdx.x) + 5120))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 5120))];
    placeholder_shared[((((int)threadIdx.x) + 6144))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 6144))];
    placeholder_shared[((((int)threadIdx.x) + 7168))] = placeholder1[((((rc_outer_outer * 8192) + ((int)threadIdx.x)) + 7168))];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        Conv2dOutput[((xx_outer_inner * 2))] = (Conv2dOutput[((xx_outer_inner * 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 4))] = (Conv2dOutput[(((xx_outer_inner * 2) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 480))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 8))] = (Conv2dOutput[(((xx_outer_inner * 2) + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 960))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 12))] = (Conv2dOutput[(((xx_outer_inner * 2) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 1440))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 16))] = (Conv2dOutput[(((xx_outer_inner * 2) + 16))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 1920))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 20))] = (Conv2dOutput[(((xx_outer_inner * 2) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 2400))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 24))] = (Conv2dOutput[(((xx_outer_inner * 2) + 24))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 2880))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 28))] = (Conv2dOutput[(((xx_outer_inner * 2) + 28))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 3360))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 1))] = (Conv2dOutput[(((xx_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 32))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 5))] = (Conv2dOutput[(((xx_outer_inner * 2) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 512))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 9))] = (Conv2dOutput[(((xx_outer_inner * 2) + 9))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 992))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 13))] = (Conv2dOutput[(((xx_outer_inner * 2) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 1472))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 17))] = (Conv2dOutput[(((xx_outer_inner * 2) + 17))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 1952))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 21))] = (Conv2dOutput[(((xx_outer_inner * 2) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 2432))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 25))] = (Conv2dOutput[(((xx_outer_inner * 2) + 25))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 2912))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
        Conv2dOutput[(((xx_outer_inner * 2) + 29))] = (Conv2dOutput[(((xx_outer_inner * 2) + 29))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 9) * 128) + (xx_outer_inner * 64)) + rc_inner) + 3392))] * placeholder_shared[(((rc_inner * 512) + (((int)threadIdx.x) & 511)))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
    T_add[(((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)))] = (Conv2dOutput[(ax2_inner)] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 4096))] = (Conv2dOutput[((ax2_inner + 4))] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 8192))] = (Conv2dOutput[((ax2_inner + 8))] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 12288))] = (Conv2dOutput[((ax2_inner + 12))] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 16384))] = (Conv2dOutput[((ax2_inner + 16))] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 20480))] = (Conv2dOutput[((ax2_inner + 20))] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 24576))] = (Conv2dOutput[((ax2_inner + 24))] + placeholder2[((((int)threadIdx.x) & 511))]);
    T_add[((((((((int)threadIdx.x) >> 9) * 2048) + (ax2_inner * 512)) + (((int)threadIdx.x) & 511)) + 28672))] = (Conv2dOutput[((ax2_inner + 28))] + placeholder2[((((int)threadIdx.x) & 511))]);
  }
}


