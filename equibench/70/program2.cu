
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 49) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 49) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 49) + ((int)threadIdx.x)) % 672))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 49) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 49) + ((int)threadIdx.x)) % 672))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[56];
  __shared__ float PaddedInput_shared[2016];
  __shared__ float placeholder_shared[96];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 7; ++i_c_outer_inner_init) {
    DepthwiseConv2d_local[(i_c_outer_inner_init)] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 7))] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 14))] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 21))] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 28))] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 35))] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 42))] = 0.000000e+00f;
    DepthwiseConv2d_local[((i_c_outer_inner_init + 49))] = 0.000000e+00f;
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 72; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)))] = (((((1 <= (((((((int)blockIdx.x) % 84) / 42) * 7) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) / 288)) + di_outer_outer)) && ((((((((int)blockIdx.x) % 84) / 42) * 7) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) / 288)) + di_outer_outer) < 15)) && (1 <= ((((((int)blockIdx.x) % 42) / 21) * 7) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) % 288) >> 5)))) && (((((((int)blockIdx.x) % 42) / 21) * 7) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) % 288) >> 5)) < 15)) ? placeholder[((((((((((((int)blockIdx.x) / 42) * 65856) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) / 288) * 9408)) + (di_outer_outer * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) % 288) >> 5) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 28) + ((int)threadIdx.x)) & 31)) - 10080))] : 0.000000e+00f);
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((di_outer_outer * 2016) + ((((int)blockIdx.x) % 21) * 32)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 28))] = placeholder1[(((((di_outer_outer * 2016) + (((((int)threadIdx.x) + 28) >> 5) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + ((((int)threadIdx.x) + 28) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((di_outer_outer * 2016) + (((((int)threadIdx.x) + 56) >> 5) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + ((((int)threadIdx.x) + 24) & 31)))];
    if (((int)threadIdx.x) < 12) {
      placeholder_shared[((((int)threadIdx.x) + 84))] = placeholder1[(((((di_outer_outer * 2016) + (((((int)threadIdx.x) + 84) >> 5) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) + 20)))];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 7; ++i_c_outer_inner) {
      for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
        DepthwiseConv2d_local[(i_c_outer_inner)] = (DepthwiseConv2d_local[(i_c_outer_inner)] + (PaddedInput_shared[(((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)))] * placeholder_shared[(((dj_inner * 32) + (((int)threadIdx.x) & 3)))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 7))] = (DepthwiseConv2d_local[((i_c_outer_inner + 7))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 4))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 4))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 14))] = (DepthwiseConv2d_local[((i_c_outer_inner + 14))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 8))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 8))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 21))] = (DepthwiseConv2d_local[((i_c_outer_inner + 21))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 12))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 12))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 28))] = (DepthwiseConv2d_local[((i_c_outer_inner + 28))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 16))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 16))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 35))] = (DepthwiseConv2d_local[((i_c_outer_inner + 35))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 20))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 20))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 42))] = (DepthwiseConv2d_local[((i_c_outer_inner + 42))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 24))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 24))]));
        DepthwiseConv2d_local[((i_c_outer_inner + 49))] = (DepthwiseConv2d_local[((i_c_outer_inner + 49))] + (PaddedInput_shared[((((((i_c_outer_inner * 288) + ((((int)threadIdx.x) >> 2) * 32)) + (dj_inner * 32)) + (((int)threadIdx.x) & 3)) + 28))] * placeholder_shared[((((dj_inner * 32) + (((int)threadIdx.x) & 3)) + 28))]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 7; ++i_inner) {
    DepthwiseConv2d[((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)))] = DepthwiseConv2d_local[(i_inner)];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 4))] = DepthwiseConv2d_local[((i_inner + 7))];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 8))] = DepthwiseConv2d_local[((i_inner + 14))];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 12))] = DepthwiseConv2d_local[((i_inner + 21))];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 16))] = DepthwiseConv2d_local[((i_inner + 28))];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 20))] = DepthwiseConv2d_local[((i_inner + 35))];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 24))] = DepthwiseConv2d_local[((i_inner + 42))];
    DepthwiseConv2d[(((((((((((int)blockIdx.x) / 42) * 65856) + (i_inner * 9408)) + (((((int)blockIdx.x) % 42) / 21) * 4704)) + ((((int)threadIdx.x) >> 2) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + (((int)threadIdx.x) & 3)) + 28))] = DepthwiseConv2d_local[((i_inner + 49))];
  }
}


