
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
  float T_dense[16];
  __shared__ float placeholder_d_shared[16];
  __shared__ float placeholder_shared[192];
  for (int i_inner_init = 0; i_inner_init < 8; ++i_inner_init) {
    for (int j_inner_init = 0; j_inner_init < 2; ++j_inner_init) {
      T_dense[(((i_inner_init * 2) + j_inner_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 50; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_inner_s = 0; ax0_ax1_fused_inner_s < 16; ++ax0_ax1_fused_inner_s) {
      if (((((int)threadIdx.x) * 8) + (ax0_ax1_fused_inner_s >> 1)) < 8) {
        if (((((int)threadIdx.x) * 16) + ax0_ax1_fused_inner_s) < 16) {
          if (((int)threadIdx.x) < 1) {
            placeholder_d_shared[(((((int)threadIdx.x) * 16) + ax0_ax1_fused_inner_s))] = placeholder[(((((((int)threadIdx.x) * 800) + ((ax0_ax1_fused_inner_s >> 1) * 100)) + (k_outer_outer * 2)) + (ax0_ax1_fused_inner_s & 1)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_inner_s1 = 0; ax0_ax1_fused_inner_s1 < 20; ++ax0_ax1_fused_inner_s1) {
      if (((((int)threadIdx.x) * 10) + (ax0_ax1_fused_inner_s1 >> 1)) < 96) {
        if (((((int)threadIdx.x) * 20) + ax0_ax1_fused_inner_s1) < 192) {
          if (((int)threadIdx.x) < 10) {
            if ((((((int)blockIdx.x) * 96) + (((int)threadIdx.x) * 10)) + (ax0_ax1_fused_inner_s1 >> 1)) < 36864) {
              placeholder_shared[(((((int)threadIdx.x) * 20) + ax0_ax1_fused_inner_s1))] = placeholder1[((((((((int)blockIdx.x) * 9600) + (((int)threadIdx.x) * 1000)) + ((ax0_ax1_fused_inner_s1 >> 1) * 100)) + (k_outer_outer * 2)) + (ax0_ax1_fused_inner_s1 & 1)))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 2; ++k_inner) {
      for (int i_inner = 0; i_inner < 8; ++i_inner) {
        for (int j_inner = 0; j_inner < 2; ++j_inner) {
          T_dense[(((i_inner * 2) + j_inner))] = (T_dense[(((i_inner * 2) + j_inner))] + (placeholder_d_shared[(((i_inner * 2) + k_inner))] * placeholder_shared[((((((int)threadIdx.x) * 4) + (j_inner * 2)) + k_inner))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 8; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      T_relu[(((((ax0_inner * 36864) + (((int)blockIdx.x) * 96)) + (((int)threadIdx.x) * 2)) + ax1_inner))] = max(T_dense[(((ax0_inner * 2) + ax1_inner))], 0.000000e+00f);
    }
  }
}


