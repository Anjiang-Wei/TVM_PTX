
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ Conv2dOutput, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] = ((Conv2dOutput[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((int)threadIdx.x) & 15))]) * (max(min(((Conv2dOutput[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)))] + placeholder[((((int)threadIdx.x) & 15))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[32];
  __shared__ float PaddedInput_shared[1548];
  __shared__ float placeholder_shared[36];
  for (int xx_c_inner_init = 0; xx_c_inner_init < 8; ++xx_c_inner_init) {
    Conv2dOutput_local[(xx_c_inner_init)] = 0.000000e+00f;
    Conv2dOutput_local[((xx_c_inner_init + 8))] = 0.000000e+00f;
    Conv2dOutput_local[((xx_c_inner_init + 16))] = 0.000000e+00f;
    Conv2dOutput_local[((xx_c_inner_init + 24))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 49; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) < 1548) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = (((1 <= ((((((int)blockIdx.x) & 1023) >> 3) * 2) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 387) / 129))) && (1 <= ((((((int)blockIdx.x) & 7) >> 2) * 128) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 129)))) ? placeholder[((((((((((((int)blockIdx.x) >> 10) * 786432) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) / 387) * 196608)) + (((((int)blockIdx.x) & 1023) >> 3) * 1536)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 387) / 129) * 768)) + (((((int)blockIdx.x) & 7) >> 2) * 384)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 129) * 3)) + rc_outer_outer) - 771))] : 0.000000e+00f);
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 36) {
        if (((int)threadIdx.x) < 12) {
          placeholder_shared[(((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[((((((((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 2) * 48) + (rc_outer_outer * 16)) + ((((int)blockIdx.x) & 3) * 4)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)))];
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
        for (int xx_c_inner = 0; xx_c_inner < 8; ++xx_c_inner) {
          Conv2dOutput_local[(xx_c_inner)] = (Conv2dOutput_local[(xx_c_inner)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 387) + (ry_inner * 129)) + ((((int)threadIdx.x) & 7) * 16)) + (xx_c_inner * 2)) + rx_inner))] * placeholder_shared[(((ry_inner * 12) + (rx_inner * 4)))]));
          Conv2dOutput_local[((xx_c_inner + 8))] = (Conv2dOutput_local[((xx_c_inner + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 387) + (ry_inner * 129)) + ((((int)threadIdx.x) & 7) * 16)) + (xx_c_inner * 2)) + rx_inner))] * placeholder_shared[((((ry_inner * 12) + (rx_inner * 4)) + 1))]));
          Conv2dOutput_local[((xx_c_inner + 16))] = (Conv2dOutput_local[((xx_c_inner + 16))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 387) + (ry_inner * 129)) + ((((int)threadIdx.x) & 7) * 16)) + (xx_c_inner * 2)) + rx_inner))] * placeholder_shared[((((ry_inner * 12) + (rx_inner * 4)) + 2))]));
          Conv2dOutput_local[((xx_c_inner + 24))] = (Conv2dOutput_local[((xx_c_inner + 24))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 387) + (ry_inner * 129)) + ((((int)threadIdx.x) & 7) * 16)) + (xx_c_inner * 2)) + rx_inner))] * placeholder_shared[((((ry_inner * 12) + (rx_inner * 4)) + 3))]));
        }
      }
    }
  }
  for (int xx_inner = 0; xx_inner < 8; ++xx_inner) {
    Conv2dOutput[((((((((((int)blockIdx.x) >> 10) * 1048576) + ((((int)threadIdx.x) >> 3) * 262144)) + (((((int)blockIdx.x) & 1023) >> 2) * 1024)) + ((((int)threadIdx.x) & 7) * 128)) + (xx_inner * 16)) + ((((int)blockIdx.x) & 3) * 4)))] = Conv2dOutput_local[(xx_inner)];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 10) * 1048576) + ((((int)threadIdx.x) >> 3) * 262144)) + (((((int)blockIdx.x) & 1023) >> 2) * 1024)) + ((((int)threadIdx.x) & 7) * 128)) + (xx_inner * 16)) + ((((int)blockIdx.x) & 3) * 4)) + 1))] = Conv2dOutput_local[((xx_inner + 8))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 10) * 1048576) + ((((int)threadIdx.x) >> 3) * 262144)) + (((((int)blockIdx.x) & 1023) >> 2) * 1024)) + ((((int)threadIdx.x) & 7) * 128)) + (xx_inner * 16)) + ((((int)blockIdx.x) & 3) * 4)) + 2))] = Conv2dOutput_local[((xx_inner + 16))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 10) * 1048576) + ((((int)threadIdx.x) >> 3) * 262144)) + (((((int)blockIdx.x) & 1023) >> 2) * 1024)) + ((((int)threadIdx.x) & 7) * 128)) + (xx_inner * 16)) + ((((int)blockIdx.x) & 3) * 4)) + 3))] = Conv2dOutput_local[((xx_inner + 24))];
  }
}


