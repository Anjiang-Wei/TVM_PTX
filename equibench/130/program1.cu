
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[784];
  __shared__ float PaddedInput_shared[49];
  __shared__ float placeholder_shared[128];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 5))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 7))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 9))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 10))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 11))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 12))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 13))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 14))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 112) + (xx_outer_inner_init * 16)) + 15))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 8))] = placeholder[(((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer) + 4096))];
    PaddedInput_shared[((((int)threadIdx.x) + 16))] = placeholder[(((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer) + 8192))];
    PaddedInput_shared[((((int)threadIdx.x) + 24))] = placeholder[(((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer) + 12288))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[(((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer) + 16384))];
    PaddedInput_shared[((((int)threadIdx.x) + 40))] = placeholder[(((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer) + 20480))];
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[((((int)threadIdx.x) + 48))] = placeholder[(((((((int)blockIdx.x) * 25088) + (((int)threadIdx.x) * 512)) + rc_outer_outer) + 24576))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 128) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 8))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 8))];
    placeholder_shared[((((int)threadIdx.x) + 16))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 16))];
    placeholder_shared[((((int)threadIdx.x) + 24))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 24))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 32))];
    placeholder_shared[((((int)threadIdx.x) + 40))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 40))];
    placeholder_shared[((((int)threadIdx.x) + 48))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 48))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 56))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 64))];
    placeholder_shared[((((int)threadIdx.x) + 72))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 72))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 80))];
    placeholder_shared[((((int)threadIdx.x) + 88))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 88))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 96))];
    placeholder_shared[((((int)threadIdx.x) + 104))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 104))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 112))];
    placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[((((rc_outer_outer * 128) + ((int)threadIdx.x)) + 120))];
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
        Conv2dOutput[(((yy_outer_inner * 112) + (xx_outer_inner * 16)))] = (Conv2dOutput[(((yy_outer_inner * 112) + (xx_outer_inner * 16)))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[((((int)threadIdx.x) * 16))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 1))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 2))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 2))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 2))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 3))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 3))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 3))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 4))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 4))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 4))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 5))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 5))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 5))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 6))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 6))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 6))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 7))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 7))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 7))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 8))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 8))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 8))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 9))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 9))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 9))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 10))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 10))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 10))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 11))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 11))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 11))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 12))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 12))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 12))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 13))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 13))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 13))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 14))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 14))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 14))]));
        Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 15))] = (Conv2dOutput[((((yy_outer_inner * 112) + (xx_outer_inner * 16)) + 15))] + (PaddedInput_shared[(((yy_outer_inner * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) * 16) + 15))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
        T_relu[((((((((int)blockIdx.x) * 6272) + (ax1_inner * 896)) + (ax2_inner * 128)) + (((int)threadIdx.x) * 16)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 112) + (ax2_inner * 16)) + ax3_inner))] + placeholder2[(((((int)threadIdx.x) * 16) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


