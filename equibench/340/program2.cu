
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel2(float* __restrict__ T_softmax_norm, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_expsum) {
  T_softmax_norm[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = (__expf((placeholder[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] - T_softmax_maxelem[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) >> 6))])) / T_softmax_expsum[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) >> 6))]);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder) {
  T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] = -3.402823e+38f;
  for (int k = 0; k < 64; ++k) {
    T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] = max(T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))], placeholder[((((((int)blockIdx.x) * 1024) + (((int)threadIdx.x) * 64)) + k))]);
  }
}

extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_softmax_expsum, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem) {
  T_softmax_expsum[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int k = 0; k < 64; ++k) {
    T_softmax_expsum[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] = (T_softmax_expsum[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] + __expf((placeholder[((((((int)blockIdx.x) * 1024) + (((int)threadIdx.x) * 64)) + k))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))])));
  }
}


