
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[2176];
  __shared__ float placeholder_shared[8704];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 8; ++ff_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 64))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 65))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 66))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 32) + (ff_outer_inner_init * 4)) + 67))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 68; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) / 544) * 34816) + ((((int)blockIdx.x) >> 2) * 17408)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 544) / 136) * 4352)) + ((((int)blockIdx.x) & 3) * 1088)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 136) / 68) * 544)) + (rc_outer_outer * 68)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 68)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 272; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 8704) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 8; ++ff_outer_inner) {
          for (int rc_inner = 0; rc_inner < 34; ++rc_inner) {
            Conv2dOutput[(((yy_outer_inner * 32) + (ff_outer_inner * 4)))] = (Conv2dOutput[(((yy_outer_inner * 32) + (ff_outer_inner * 4)))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 64))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 64))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner) + 1088))] * placeholder_shared[(((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 1))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 65))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 65))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner) + 1088))] * placeholder_shared[((((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)) + 1))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 2))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 2))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 66))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 66))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner) + 1088))] * placeholder_shared[((((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)) + 2))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 3))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 3))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)) + 3))]));
            Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 67))] = (Conv2dOutput[((((yy_outer_inner * 32) + (ff_outer_inner * 4)) + 67))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 272) + (yy_outer_inner * 136)) + (((((int)threadIdx.x) & 7) >> 2) * 68)) + (rc_outer_inner * 34)) + rc_inner) + 1088))] * placeholder_shared[((((((rc_outer_inner * 4352) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 4)) + 3))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
      T_relu[(((((((((((int)threadIdx.x) >> 4) * 8192) + ((((int)blockIdx.x) >> 2) * 4096)) + (((((int)threadIdx.x) & 15) >> 3) * 2048)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) & 3) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 32) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 3) * 32) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 8192) + ((((int)blockIdx.x) >> 2) * 4096)) + (((((int)threadIdx.x) & 15) >> 3) * 2048)) + (ax1_inner * 1024)) + ((((int)blockIdx.x) & 3) * 256)) + ((((int)threadIdx.x) & 7) * 32)) + ax3_inner) + 16384))] = max((Conv2dOutput[((((ax1_inner * 32) + ax3_inner) + 64))] + placeholder2[((((((int)threadIdx.x) & 3) * 32) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


