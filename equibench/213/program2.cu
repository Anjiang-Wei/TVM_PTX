
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[384];
  __shared__ float PaddedInput_shared[225];
  __shared__ float placeholder_shared[128];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 3; ++xx_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 16; ++ff_inner_init) {
      Conv2dOutput[(((xx_outer_inner_init * 16) + ff_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 48))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 96))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 144))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 192))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 240))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 288))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 16) + ff_inner_init) + 336))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 256; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((int)threadIdx.x) * 256) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 75))] = placeholder[((((((int)threadIdx.x) * 256) + rc_outer_outer) + 19200))];
    PaddedInput_shared[((((int)threadIdx.x) + 150))] = placeholder[((((((int)threadIdx.x) * 256) + rc_outer_outer) + 38400))];
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[((((int)threadIdx.x) * 8))] = placeholder1[(((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 1))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 2))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 3))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 4))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 5))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 6))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 8)) + 7))];
    }
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
      for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
        Conv2dOutput[(((xx_outer_inner * 16) + ff_inner))] = (Conv2dOutput[(((xx_outer_inner * 16) + ff_inner))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[(ff_inner)]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 48))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 48))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 16))]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 96))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 96))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 32))]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 144))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 144))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 48))]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 192))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 192))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 64))]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 240))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 240))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 80))]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 288))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 288))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 96))]));
        Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 336))] = (Conv2dOutput[((((xx_outer_inner * 16) + ff_inner) + 336))] + (PaddedInput_shared[(((((int)threadIdx.x) * 3) + xx_outer_inner))] * placeholder_shared[((ff_inner + 112))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 16; ++ax3_inner) {
      T_relu[((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 16) + ax3_inner))] + placeholder2[(ax3_inner)]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 48))] + placeholder2[((ax3_inner + 16))]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 32))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 96))] + placeholder2[((ax3_inner + 32))]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 48))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 144))] + placeholder2[((ax3_inner + 48))]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 64))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 192))] + placeholder2[((ax3_inner + 64))]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 80))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 240))] + placeholder2[((ax3_inner + 80))]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 96))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 288))] + placeholder2[((ax3_inner + 96))]), 0.000000e+00f);
      T_relu[(((((((int)threadIdx.x) * 384) + (ax2_inner * 128)) + ax3_inner) + 112))] = max((Conv2dOutput[((((ax2_inner * 16) + ax3_inner) + 336))] + placeholder2[((ax3_inner + 112))]), 0.000000e+00f);
    }
  }
}


