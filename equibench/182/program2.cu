
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[4096];
  __shared__ float placeholder_shared[2048];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
          for (int ff_inner_init = 0; ff_inner_init < 16; ++ff_inner_init) {
            Conv2dOutput[((((((nn_inner_init * 128) + (yy_outer_inner_init * 64)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + ff_inner_init))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 128) + (yy_outer_inner_init * 64)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + ff_inner_init) + 256))] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 46; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      ((float2*)(PaddedInput_shared + (((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) * 2)) >> 10) * 1024) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 7) * 128)) + (((int)threadIdx.x) * 2)))))[0] = ((float2*)(placeholder + (((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) * 2)) >> 10) * 47104) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 7) * 5888)) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)))))[0];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 2048) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 64)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
          for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
            for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
              for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
                for (int ff_inner = 0; ff_inner < 16; ++ff_inner) {
                  Conv2dOutput[((((((nn_inner * 128) + (yy_outer_inner * 64)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + ff_inner))] = (Conv2dOutput[((((((nn_inner * 128) + (yy_outer_inner * 64)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + ff_inner))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 5) * 2048) + (nn_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 3) * 256)) + (yy_outer_inner * 128)) + (((((int)threadIdx.x) & 7) >> 2) * 32)) + (xx_inner * 16)) + (rc_outer_inner * 8)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 1024) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 16)) + ff_inner))]));
                  Conv2dOutput[(((((((nn_inner * 128) + (yy_outer_inner * 64)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + ff_inner) + 256))] = (Conv2dOutput[(((((((nn_inner * 128) + (yy_outer_inner * 64)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + ff_inner) + 256))] + (PaddedInput_shared[(((((((((((((int)threadIdx.x) >> 5) * 2048) + (nn_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 3) * 256)) + (yy_outer_inner * 128)) + (((((int)threadIdx.x) & 7) >> 2) * 32)) + (xx_inner * 16)) + (rc_outer_inner * 8)) + rc_inner) + 64))] * placeholder_shared[((((((rc_outer_inner * 1024) + (rc_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + (ff_outer_inner * 16)) + ff_inner))]));
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
          T_relu[((((((((((((int)threadIdx.x) >> 5) * 16384) + (ax0_inner * 8192)) + (((((int)threadIdx.x) & 31) >> 3) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 7) >> 2) * 256)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 128) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 3) * 32) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((((int)threadIdx.x) >> 5) * 16384) + (ax0_inner * 8192)) + (((((int)threadIdx.x) & 31) >> 3) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 7) >> 2) * 256)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + ax3_inner) + 512))] = max((Conv2dOutput[((((((ax0_inner * 128) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner) + 256))] + placeholder2[((((((int)threadIdx.x) & 3) * 32) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


