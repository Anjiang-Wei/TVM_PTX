
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[560];
  __shared__ float placeholder_shared[2560];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 24; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) / 40) * 6720) + (((int)blockIdx.x) * 960)) + (((((int)threadIdx.x) % 40) / 20) * 480)) + (rc_outer_outer * 20)) + (((int)threadIdx.x) % 20)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[((((((((((int)threadIdx.x) + 224) / 40) * 6720) + (((int)blockIdx.x) * 960)) + ((((((int)threadIdx.x) + 24) % 40) / 20) * 480)) + (rc_outer_outer * 20)) + ((((int)threadIdx.x) + 4) % 20)))];
    if (((int)threadIdx.x) < 112) {
      PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[((((((((((int)threadIdx.x) + 448) / 40) * 6720) + (((int)blockIdx.x) * 960)) + ((((((int)threadIdx.x) + 8) % 40) / 20) * 480)) + (rc_outer_outer * 20)) + ((((int)threadIdx.x) + 8) % 20)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 2560) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 224))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 448))];
    placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 672))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 1120))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1120))];
    placeholder_shared[((((int)threadIdx.x) + 1344))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1344))];
    placeholder_shared[((((int)threadIdx.x) + 1568))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1568))];
    placeholder_shared[((((int)threadIdx.x) + 1792))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 1792))];
    placeholder_shared[((((int)threadIdx.x) + 2016))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 2016))];
    placeholder_shared[((((int)threadIdx.x) + 2240))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 2240))];
    if (((int)threadIdx.x) < 96) {
      placeholder_shared[((((int)threadIdx.x) + 2464))] = placeholder1[((((rc_outer_outer * 2560) + ((int)threadIdx.x)) + 2464))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 20; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 1))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 2))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 3))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 1))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 2))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 3))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 4))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 5))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 6))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 7))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 4))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 5))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 6))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 5) * 80) + (((((int)threadIdx.x) & 31) >> 4) * 20)) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 15) * 8)) + 7))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[(((((((((int)threadIdx.x) >> 5) * 3584) + (ax1_inner * 1792)) + (((int)blockIdx.x) * 256)) + ((((int)threadIdx.x) & 31) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 8) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 15) * 8) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


