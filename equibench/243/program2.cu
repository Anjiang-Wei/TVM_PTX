
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[6144];
  __shared__ float placeholder_shared[3072];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 16; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 256))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 257))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 264))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 9))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 16) + (ff_outer_inner_init * 2)) + 265))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 384; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) / 96) * 12288) + ((((int)blockIdx.x) >> 1) * 384)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 96) / 48) * 192)) + (rc_outer_outer * 48)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)) % 48)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 192; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 6144) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) & 63)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 16; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 48; ++rc_inner) {
          Conv2dOutput[(((yy_outer_inner * 16) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 16) + (ff_outer_inner * 2)))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 256))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 256))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)) + 32))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 257))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 257))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)) + 33))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 8))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner) + 96))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 264))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 264))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner) + 96))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)) + 32))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 9))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 9))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner) + 96))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 265))] = (Conv2dOutput[((((yy_outer_inner * 16) + (ff_outer_inner * 2)) + 265))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 3072) + (yy_outer_inner * 192)) + (((((int)threadIdx.x) & 7) >> 2) * 48)) + rc_inner) + 96))] * placeholder_shared[(((((rc_inner * 64) + ((((int)threadIdx.x) & 3) * 8)) + (ff_outer_inner * 2)) + 33))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 32; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[(((((((((((int)threadIdx.x) >> 3) * 262144) + (ax1_inner * 8192)) + ((((int)blockIdx.x) >> 1) * 256)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 8) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 3) * 262144) + (ax1_inner * 8192)) + ((((int)blockIdx.x) >> 1) * 256)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner) + 32))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 256))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner) + 32))]), 0.000000e+00f);
    }
  }
}


