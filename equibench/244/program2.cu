
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[512];
  __shared__ float PaddedInput_shared[1536];
  __shared__ float placeholder_shared[96];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 8; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 16; ++yy_inner_init) {
      Conv2dOutput[(((yy_inner_init * 8) + ff_outer_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 8) + ff_outer_inner_init) + 128))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 8) + ff_outer_inner_init) + 256))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 8) + ff_outer_inner_init) + 384))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) / 24) * 12288) + ((((int)blockIdx.x) >> 2) * 1536)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 24) / 3) * 192)) + (rc_outer_outer * 3)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) % 3)))];
    }
    placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[(((((rc_outer_outer * 384) + ((((int)threadIdx.x) >> 4) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 15) * 2)))];
    placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((rc_outer_outer * 384) + ((((((int)threadIdx.x) * 2) + 1) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((((int)threadIdx.x) * 2) + 1) & 31)))];
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 64))] = placeholder1[(((((rc_outer_outer * 384) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) * 2)) + 256))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 65))] = placeholder1[(((((rc_outer_outer * 384) + ((((int)blockIdx.x) & 3) * 32)) + (((int)threadIdx.x) * 2)) + 257))];
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 8; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 16; ++yy_inner) {
          Conv2dOutput[(((yy_inner * 8) + ff_outer_inner))] = (Conv2dOutput[(((yy_inner * 8) + ff_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 3) * 384) + (yy_inner * 24)) + (((((int)threadIdx.x) & 7) >> 2) * 3)) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 8)) + ff_outer_inner))]));
          Conv2dOutput[((((yy_inner * 8) + ff_outer_inner) + 128))] = (Conv2dOutput[((((yy_inner * 8) + ff_outer_inner) + 128))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 384) + (yy_inner * 24)) + (((((int)threadIdx.x) & 7) >> 2) * 3)) + rc_inner) + 6))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 8)) + ff_outer_inner))]));
          Conv2dOutput[((((yy_inner * 8) + ff_outer_inner) + 256))] = (Conv2dOutput[((((yy_inner * 8) + ff_outer_inner) + 256))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 384) + (yy_inner * 24)) + (((((int)threadIdx.x) & 7) >> 2) * 3)) + rc_inner) + 12))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 8)) + ff_outer_inner))]));
          Conv2dOutput[((((yy_inner * 8) + ff_outer_inner) + 384))] = (Conv2dOutput[((((yy_inner * 8) + ff_outer_inner) + 384))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 3) * 384) + (yy_inner * 24)) + (((((int)threadIdx.x) & 7) >> 2) * 3)) + rc_inner) + 18))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 8)) + ff_outer_inner))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 16; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[(((((((((((int)threadIdx.x) >> 3) * 131072) + (ax1_inner * 8192)) + ((((int)blockIdx.x) >> 2) * 1024)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 8) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 3) * 131072) + (ax1_inner * 8192)) + ((((int)blockIdx.x) >> 2) * 1024)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner) + 256))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 128))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 3) * 131072) + (ax1_inner * 8192)) + ((((int)blockIdx.x) >> 2) * 1024)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner) + 512))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 256))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 3) * 131072) + (ax1_inner * 8192)) + ((((int)blockIdx.x) >> 2) * 1024)) + (((((int)threadIdx.x) & 7) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner) + 768))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 384))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 8)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


