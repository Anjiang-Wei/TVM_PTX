
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 28) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 28) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 28) + ((int)threadIdx.x)) % 672))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 28) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 28) + ((int)threadIdx.x)) % 672))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[4];
  __shared__ float PaddedInput_shared[1760];
  __shared__ float placeholder_shared[800];
  for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
    for (int c_c_inner_init = 0; c_c_inner_init < 2; ++c_c_inner_init) {
      DepthwiseConv2d_local[(((j_c_inner_init * 2) + c_c_inner_init))] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 55; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = (((((2 <= (((((int)blockIdx.x) / 42) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 11))) && ((((((int)blockIdx.x) / 42) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 11)) < 17)) && (2 <= ((((((int)blockIdx.x) % 42) / 21) * 8) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 11)))) && (((((((int)blockIdx.x) % 42) / 21) * 8) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 11)) < 17)) ? placeholder[(((((((((((int)blockIdx.x) / 42) * 20160) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 11) * 10080)) + (((((int)blockIdx.x) % 42) / 21) * 5376)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 11) * 672)) + ((((int)blockIdx.x) % 21) * 32)) + ((int)threadIdx.x)) - 21504))] : 0.000000e+00f);
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 25; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32) + ((int)threadIdx.x)))] = placeholder1[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 672) + ((((int)blockIdx.x) % 21) * 32)) + ((int)threadIdx.x)))];
  }
  __syncthreads();
  for (int dj_outer_inner = 0; dj_outer_inner < 5; ++dj_outer_inner) {
    for (int di_inner = 0; di_inner < 5; ++di_inner) {
      for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
        for (int c_c_inner = 0; c_c_inner < 2; ++c_c_inner) {
          DepthwiseConv2d_local[(((j_c_inner * 2) + c_c_inner))] = (DepthwiseConv2d_local[(((j_c_inner * 2) + c_c_inner))] + (PaddedInput_shared[(((((((di_inner * 352) + ((((int)threadIdx.x) >> 4) * 128)) + (j_c_inner * 64)) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + c_c_inner))] * placeholder_shared[(((((di_inner * 160) + (dj_outer_inner * 32)) + ((((int)threadIdx.x) & 15) * 2)) + c_c_inner))]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    for (int c_inner = 0; c_inner < 2; ++c_inner) {
      DepthwiseConv2d[((((((((((int)blockIdx.x) / 21) * 2688) + ((((int)threadIdx.x) >> 4) * 1344)) + (j_inner * 672)) + ((((int)blockIdx.x) % 21) * 32)) + ((((int)threadIdx.x) & 15) * 2)) + c_inner))] = DepthwiseConv2d_local[(((j_inner * 2) + c_inner))];
    }
  }
}


