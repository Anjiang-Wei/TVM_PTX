
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[720];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float placeholder_shared[128];
  for (int yy_inner_init = 0; yy_inner_init < 30; ++yy_inner_init) {
    Conv2dOutput[((yy_inner_init * 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 360))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 361))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 362))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 363))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 364))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 365))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 6))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 366))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 7))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 367))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 368))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 369))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 10))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 370))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 11))] = 0.000000e+00f;
    Conv2dOutput[(((yy_inner_init * 12) + 371))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 48; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 2))] = placeholder[(((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 320))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 15360))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 321))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 15361))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 640))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 30720))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 641))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 30721))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 960))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 46080))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 961))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 46081))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1280))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 61440))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1281))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 61441))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1600))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 76800))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1601))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 76801))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1920))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 92160))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1921))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 92161))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2240))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 107520))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2241))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 107521))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2560))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 122880))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2561))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 122881))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2880))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 138240))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 2881))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 138241))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3200))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 153600))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3201))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 153601))];
    if (((int)threadIdx.x) < 40) {
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3520))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 168960))];
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 3521))] = placeholder[((((((((int)blockIdx.x) >> 1) * 172800) + (((int)threadIdx.x) * 96)) + (rc_outer_outer * 2)) + 168961))];
    }
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 256) + ((((int)threadIdx.x) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) & 63)))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int yy_inner = 0; yy_inner < 30; ++yy_inner) {
        Conv2dOutput[((yy_inner * 12))] = (Conv2dOutput[((yy_inner * 12))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[(((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_inner * 12) + 360))] = (Conv2dOutput[(((yy_inner * 12) + 360))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
        Conv2dOutput[(((yy_inner * 12) + 1))] = (Conv2dOutput[(((yy_inner * 12) + 1))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_inner * 12) + 361))] = (Conv2dOutput[(((yy_inner * 12) + 361))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
        Conv2dOutput[(((yy_inner * 12) + 2))] = (Conv2dOutput[(((yy_inner * 12) + 2))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_inner * 12) + 362))] = (Conv2dOutput[(((yy_inner * 12) + 362))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
        Conv2dOutput[(((yy_inner * 12) + 3))] = (Conv2dOutput[(((yy_inner * 12) + 3))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_inner * 12) + 363))] = (Conv2dOutput[(((yy_inner * 12) + 363))] + (PaddedInput_shared[((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
        Conv2dOutput[(((yy_inner * 12) + 4))] = (Conv2dOutput[(((yy_inner * 12) + 4))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[(((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_inner * 12) + 364))] = (Conv2dOutput[(((yy_inner * 12) + 364))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
        Conv2dOutput[(((yy_inner * 12) + 5))] = (Conv2dOutput[(((yy_inner * 12) + 5))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_inner * 12) + 365))] = (Conv2dOutput[(((yy_inner * 12) + 365))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
        Conv2dOutput[(((yy_inner * 12) + 6))] = (Conv2dOutput[(((yy_inner * 12) + 6))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_inner * 12) + 366))] = (Conv2dOutput[(((yy_inner * 12) + 366))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
        Conv2dOutput[(((yy_inner * 12) + 7))] = (Conv2dOutput[(((yy_inner * 12) + 7))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_inner * 12) + 367))] = (Conv2dOutput[(((yy_inner * 12) + 367))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 2))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
        Conv2dOutput[(((yy_inner * 12) + 8))] = (Conv2dOutput[(((yy_inner * 12) + 8))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[(((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
        Conv2dOutput[(((yy_inner * 12) + 368))] = (Conv2dOutput[(((yy_inner * 12) + 368))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
        Conv2dOutput[(((yy_inner * 12) + 9))] = (Conv2dOutput[(((yy_inner * 12) + 9))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
        Conv2dOutput[(((yy_inner * 12) + 369))] = (Conv2dOutput[(((yy_inner * 12) + 369))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
        Conv2dOutput[(((yy_inner * 12) + 10))] = (Conv2dOutput[(((yy_inner * 12) + 10))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
        Conv2dOutput[(((yy_inner * 12) + 370))] = (Conv2dOutput[(((yy_inner * 12) + 370))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
        Conv2dOutput[(((yy_inner * 12) + 11))] = (Conv2dOutput[(((yy_inner * 12) + 11))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
        Conv2dOutput[(((yy_inner * 12) + 371))] = (Conv2dOutput[(((yy_inner * 12) + 371))] + (PaddedInput_shared[(((((yy_inner * 120) + ((((int)threadIdx.x) >> 3) * 6)) + rc_inner) + 4))] * placeholder_shared[((((rc_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 30; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[(((((((((((int)blockIdx.x) >> 1) * 230400) + (ax1_inner * 7680)) + ((((int)threadIdx.x) >> 3) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 12) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((((((int)blockIdx.x) >> 1) * 230400) + (ax1_inner * 7680)) + ((((int)threadIdx.x) >> 3) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 32))] = max((Conv2dOutput[(((((ax1_inner * 12) + (ax2_inner * 4)) + ax3_inner) + 360))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 32))]), 0.000000e+00f);
      }
    }
  }
}


