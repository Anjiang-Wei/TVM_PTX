
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[14];
  __shared__ float PaddedInput_shared[140];
  __shared__ float placeholder_shared[640];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 48; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[((((int)threadIdx.x) * 28))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + (((((int)threadIdx.x) * 28) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) * 28) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 1))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 1) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 1) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 2))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 2) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 2) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 3))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 3) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 3) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 4))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 4) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 4) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 5))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 5) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 5) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 6))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 6) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 6) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 7))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 7) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 7) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 8))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 8) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 8) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 9))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 9) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 9) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 10))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 6720) + (((((int)threadIdx.x) * 28) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) * 28) % 10)) + 480))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 11))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 11) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 1) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 12))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 12) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 2) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 13))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 13) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 3) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 14))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 14) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 4) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 15))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 15) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 5) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 16))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 16) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 6) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 17))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 17) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 7) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 18))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 18) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 8) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 19))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 19) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 9) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 20))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 6720) + (((((int)threadIdx.x) * 28) / 10) * 480)) + (rc_outer_outer * 10)) + ((((int)threadIdx.x) * 28) % 10)) + 960))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 21))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 21) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 1) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 22))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 22) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 2) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 23))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 23) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 3) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 24))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 24) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 4) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 25))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 25) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 5) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 26))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 26) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 6) % 10)))];
    }
    if (((int)threadIdx.x) < 5) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 27))] = placeholder[((((((((int)blockIdx.x) >> 1) * 6720) + ((((((int)threadIdx.x) * 28) + 27) / 10) * 480)) + (rc_outer_outer * 10)) + (((((int)threadIdx.x) * 28) + 7) % 10)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 640))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 576))] = placeholder1[(((((rc_outer_outer * 1280) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)) + 1152))];
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 70))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 5) * 70))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 10))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 20))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 20))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 30))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 30))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 40))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 40))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 50))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 50))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 60))] * placeholder_shared[(((((int)threadIdx.x) & 31) * 2))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 60))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 1))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 31))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 31))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 41))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 41))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 51))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 51))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 61))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 64))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 61))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 65))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 52))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 52))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 62))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 128))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 62))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 129))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 53))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 53))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 63))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 192))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 63))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 193))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 24))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 24))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 44))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 44))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 54))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 54))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 256))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 257))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 25))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 25))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 45))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 45))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 55))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 55))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 320))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 321))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 26))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 26))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 46))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 46))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 56))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 384))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 385))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 27))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 27))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 47))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 47))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 57))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 448))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 449))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 28))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 48))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 48))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 58))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 58))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 512))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 513))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 29))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 49))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 49))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 59))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 59))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 576))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 70) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 31) * 2) + 577))]));
  }
  for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 1) * 1792) + ((((int)threadIdx.x) >> 5) * 896)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


