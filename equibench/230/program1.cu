
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[56];
  __shared__ float PaddedInput_shared[2688];
  __shared__ float placeholder_shared[6144];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 10; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 128) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 256) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)) + 3840))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 512) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 640) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)) + 7680))];
    PaddedInput_shared[((((int)threadIdx.x) + 896))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 896) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1024))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 1024) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1152))] = placeholder[((((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)) + 11520))];
    PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 1280) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1408))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 1408) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)) + 15360))];
    PaddedInput_shared[((((int)threadIdx.x) + 1664))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 1664) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1792))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 1792) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[((((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)) + 19200))];
    PaddedInput_shared[((((int)threadIdx.x) + 2048))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 2048) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2176))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 2176) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2304))] = placeholder[((((((((int)blockIdx.x) * 26880) + ((((int)threadIdx.x) / 48) * 480)) + (rc_outer_outer * 48)) + (((int)threadIdx.x) % 48)) + 23040))];
    PaddedInput_shared[((((int)threadIdx.x) + 2432))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 2432) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 32) % 48)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[(((((((int)blockIdx.x) * 26880) + (((((int)threadIdx.x) + 2560) / 48) * 480)) + (rc_outer_outer * 48)) + ((((int)threadIdx.x) + 16) % 48)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 6144) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 640))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 768))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 1024))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 1152))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1152))];
    placeholder_shared[((((int)threadIdx.x) + 1280))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1280))];
    placeholder_shared[((((int)threadIdx.x) + 1408))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1408))];
    placeholder_shared[((((int)threadIdx.x) + 1536))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1536))];
    placeholder_shared[((((int)threadIdx.x) + 1664))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1664))];
    placeholder_shared[((((int)threadIdx.x) + 1792))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1792))];
    placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 1920))];
    placeholder_shared[((((int)threadIdx.x) + 2048))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2048))];
    placeholder_shared[((((int)threadIdx.x) + 2176))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2176))];
    placeholder_shared[((((int)threadIdx.x) + 2304))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2304))];
    placeholder_shared[((((int)threadIdx.x) + 2432))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2432))];
    placeholder_shared[((((int)threadIdx.x) + 2560))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2560))];
    placeholder_shared[((((int)threadIdx.x) + 2688))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2688))];
    placeholder_shared[((((int)threadIdx.x) + 2816))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2816))];
    placeholder_shared[((((int)threadIdx.x) + 2944))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 2944))];
    placeholder_shared[((((int)threadIdx.x) + 3072))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3072))];
    placeholder_shared[((((int)threadIdx.x) + 3200))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3200))];
    placeholder_shared[((((int)threadIdx.x) + 3328))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3328))];
    placeholder_shared[((((int)threadIdx.x) + 3456))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3456))];
    placeholder_shared[((((int)threadIdx.x) + 3584))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3584))];
    placeholder_shared[((((int)threadIdx.x) + 3712))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3712))];
    placeholder_shared[((((int)threadIdx.x) + 3840))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3840))];
    placeholder_shared[((((int)threadIdx.x) + 3968))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 3968))];
    placeholder_shared[((((int)threadIdx.x) + 4096))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4096))];
    placeholder_shared[((((int)threadIdx.x) + 4224))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4224))];
    placeholder_shared[((((int)threadIdx.x) + 4352))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4352))];
    placeholder_shared[((((int)threadIdx.x) + 4480))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4480))];
    placeholder_shared[((((int)threadIdx.x) + 4608))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4608))];
    placeholder_shared[((((int)threadIdx.x) + 4736))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4736))];
    placeholder_shared[((((int)threadIdx.x) + 4864))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4864))];
    placeholder_shared[((((int)threadIdx.x) + 4992))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 4992))];
    placeholder_shared[((((int)threadIdx.x) + 5120))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5120))];
    placeholder_shared[((((int)threadIdx.x) + 5248))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5248))];
    placeholder_shared[((((int)threadIdx.x) + 5376))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5376))];
    placeholder_shared[((((int)threadIdx.x) + 5504))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5504))];
    placeholder_shared[((((int)threadIdx.x) + 5632))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5632))];
    placeholder_shared[((((int)threadIdx.x) + 5760))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5760))];
    placeholder_shared[((((int)threadIdx.x) + 5888))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 5888))];
    placeholder_shared[((((int)threadIdx.x) + 6016))] = placeholder1[((((rc_outer_outer * 6144) + ((int)threadIdx.x)) + 6016))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 48; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 48))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 96))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 144))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 192))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 240))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 288))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 336))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 384))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 432))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 480))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 528))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 576))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 624))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 48))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 96))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 144))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 192))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 336))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 384))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 432))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 528))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 576))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 624))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 672))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 720))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 768))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 816))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 864))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 912))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 960))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1008))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1056))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1104))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1152))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1200))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1248))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1296))] * placeholder_shared[(((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 672))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 720))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 768))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 816))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 864))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 912))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 960))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1008))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1056))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1104))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1152))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1200))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1248))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 1344) + rc_outer_inner) + 1296))] * placeholder_shared[((((rc_outer_inner * 128) + ((((int)threadIdx.x) & 63) * 2)) + 1))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 28; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[((((((((int)blockIdx.x) * 7168) + ((((int)threadIdx.x) >> 6) * 3584)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 63) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 2) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 63) * 2) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


