
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[336];
  __shared__ float PaddedInput_shared[9570];
  __shared__ float placeholder_shared[18];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 2; ++i_outer_inner_init) {
    for (int c_outer_inner_init = 0; c_outer_inner_init < 6; ++c_outer_inner_init) {
      for (int j_inner_init = 0; j_inner_init < 14; ++j_inner_init) {
        DepthwiseConv2d[((((i_outer_inner_init * 84) + (j_inner_init * 6)) + c_outer_inner_init))] = 0.000000e+00f;
        DepthwiseConv2d[(((((i_outer_inner_init * 84) + (j_inner_init * 6)) + c_outer_inner_init) + 168))] = 0.000000e+00f;
      }
    }
  }
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 1368; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)) < 9570) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)))] = (((1 <= ((((((int)blockIdx.x) >> 6) * 56) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)) / 174)) + di_outer_outer)) && (1 <= ((((((int)blockIdx.x) & 63) >> 4) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)) % 174) / 6)))) ? placeholder[((((((((((((int)blockIdx.x) >> 6) * 602112) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)) / 174) * 10752)) + (di_outer_outer * 10752)) + (((((int)blockIdx.x) & 63) >> 4) * 2688)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)) % 174) / 6) * 96)) + ((((int)blockIdx.x) & 15) * 6)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + ((int)threadIdx.x)) % 6)) - 10848))] : 0.000000e+00f);
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + ((int)threadIdx.x)) < 18) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + ((int)threadIdx.x)))] = placeholder1[(((((di_outer_outer * 288) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + ((int)threadIdx.x)) / 6) * 96)) + ((((int)blockIdx.x) & 15) * 6)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + ((int)threadIdx.x)) % 6)))];
      }
    }
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
      for (int c_outer_inner = 0; c_outer_inner < 6; ++c_outer_inner) {
        for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
          for (int j_inner = 0; j_inner < 14; ++j_inner) {
            DepthwiseConv2d[((((i_outer_inner * 84) + (j_inner * 6)) + c_outer_inner))] = (DepthwiseConv2d[((((i_outer_inner * 84) + (j_inner * 6)) + c_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) * 696) + (i_outer_inner * 348)) + (j_inner * 12)) + (dj_inner * 6)) + c_outer_inner))] * placeholder_shared[(((dj_inner * 6) + c_outer_inner))]));
            DepthwiseConv2d[(((((i_outer_inner * 84) + (j_inner * 6)) + c_outer_inner) + 168))] = (DepthwiseConv2d[(((((i_outer_inner * 84) + (j_inner * 6)) + c_outer_inner) + 168))] + (PaddedInput_shared[(((((((((int)threadIdx.x) * 696) + (i_outer_inner * 348)) + (j_inner * 12)) + (dj_inner * 6)) + c_outer_inner) + 4872))] * placeholder_shared[(((dj_inner * 6) + c_outer_inner))]));
          }
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 14; ++i2_inner) {
      for (int i3_inner = 0; i3_inner < 6; ++i3_inner) {
        compute[(((((((((((int)blockIdx.x) >> 6) * 150528) + (((int)threadIdx.x) * 10752)) + (i1_inner * 5376)) + (((((int)blockIdx.x) & 63) >> 4) * 1344)) + (i2_inner * 96)) + ((((int)blockIdx.x) & 15) * 6)) + i3_inner))] = max(min((DepthwiseConv2d[((((i1_inner * 84) + (i2_inner * 6)) + i3_inner))] + placeholder2[((((((int)blockIdx.x) & 15) * 6) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
        compute[((((((((((((int)blockIdx.x) >> 6) * 150528) + (((int)threadIdx.x) * 10752)) + (i1_inner * 5376)) + (((((int)blockIdx.x) & 63) >> 4) * 1344)) + (i2_inner * 96)) + ((((int)blockIdx.x) & 15) * 6)) + i3_inner) + 75264))] = max(min((DepthwiseConv2d[(((((i1_inner * 84) + (i2_inner * 6)) + i3_inner) + 168))] + placeholder2[((((((int)blockIdx.x) & 15) * 6) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      }
    }
  }
}


