
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[32];
  __shared__ float PaddedInput_shared[8160];
  __shared__ float placeholder_shared[864];
  for (int c_outer_inner_init = 0; c_outer_inner_init < 16; ++c_outer_inner_init) {
    DepthwiseConv2d[(c_outer_inner_init)] = 0.000000e+00f;
    DepthwiseConv2d[((c_outer_inner_init + 16))] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 170; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)))] = (((1 <= (((((int)blockIdx.x) / 28) * 16) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) / 480))) && (1 <= (((((int)blockIdx.x) % 28) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) % 480) / 96)))) ? placeholder[(((((((((int)blockIdx.x) / 28) * 172032) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) / 480) * 10752)) + ((((int)blockIdx.x) % 28) * 384)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) % 480)) - 10848))] : 0.000000e+00f);
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
    if (((((int)threadIdx.x) * 48) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 864) {
      if (((int)threadIdx.x) < 18) {
        placeholder_shared[(((((int)threadIdx.x) * 48) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((int)threadIdx.x) * 48) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))];
      }
    }
  }
  __syncthreads();
  for (int c_outer_inner = 0; c_outer_inner < 16; ++c_outer_inner) {
    for (int di_inner = 0; di_inner < 3; ++di_inner) {
      DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 1920) + (di_inner * 480)) + (((((int)threadIdx.x) % 12) / 6) * 192)) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner))] * placeholder_shared[((((di_inner * 288) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner))]));
      DepthwiseConv2d[((c_outer_inner + 16))] = (DepthwiseConv2d[((c_outer_inner + 16))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 12) * 1920) + (di_inner * 480)) + (((((int)threadIdx.x) % 12) / 6) * 192)) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 960))] * placeholder_shared[((((di_inner * 288) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner))]));
      DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 12) * 1920) + (di_inner * 480)) + (((((int)threadIdx.x) % 12) / 6) * 192)) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 96))] * placeholder_shared[(((((di_inner * 288) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 96))]));
      DepthwiseConv2d[((c_outer_inner + 16))] = (DepthwiseConv2d[((c_outer_inner + 16))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 12) * 1920) + (di_inner * 480)) + (((((int)threadIdx.x) % 12) / 6) * 192)) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 1056))] * placeholder_shared[(((((di_inner * 288) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 96))]));
      DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 12) * 1920) + (di_inner * 480)) + (((((int)threadIdx.x) % 12) / 6) * 192)) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 192))] * placeholder_shared[(((((di_inner * 288) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 192))]));
      DepthwiseConv2d[((c_outer_inner + 16))] = (DepthwiseConv2d[((c_outer_inner + 16))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 12) * 1920) + (di_inner * 480)) + (((((int)threadIdx.x) % 12) / 6) * 192)) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 1152))] * placeholder_shared[(((((di_inner * 288) + ((((int)threadIdx.x) % 6) * 16)) + c_outer_inner) + 192))]));
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 16; ++i3_inner) {
      compute[((((((((((int)blockIdx.x) / 28) * 43008) + ((((int)threadIdx.x) / 12) * 10752)) + (i1_inner * 5376)) + ((((int)blockIdx.x) % 28) * 192)) + ((((int)threadIdx.x) % 12) * 16)) + i3_inner))] = max(min((DepthwiseConv2d[(((i1_inner * 16) + i3_inner))] + placeholder2[((((((int)threadIdx.x) % 6) * 16) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


