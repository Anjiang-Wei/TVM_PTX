
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[4608];
  __shared__ float placeholder_shared[32];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 4; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
      Conv2dOutput[(((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 32))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 64))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 96))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 33))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 65))] = 0.000000e+00f;
      Conv2dOutput[((((xx_outer_inner_init * 8) + (ff_outer_inner_init * 2)) + 97))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    ((float2*)(PaddedInput_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder + (((((((((int)blockIdx.x) / 5) * 46080) + ((((int)threadIdx.x) / 24) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 576))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) / 5) * 46080) + ((((int)threadIdx.x) / 24) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)) + 23040))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 1152))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) / 5) * 46080) + ((((int)threadIdx.x) / 24) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)) + 230400))))[0];
    ((float2*)(PaddedInput_shared + ((((((((int)threadIdx.x) + 864) / 576) * 1152) + (((((int)threadIdx.x) / 24) + 12) * 48)) + ((((int)threadIdx.x) % 24) * 2)))))[0] = ((float2*)(placeholder + (((((((((((int)threadIdx.x) + 864) / 576) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) / 24) + 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 2304))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) / 5) * 46080) + ((((int)threadIdx.x) / 24) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)) + 460800))))[0];
    ((float2*)(PaddedInput_shared + ((((((((int)threadIdx.x) + 1440) / 576) * 1152) + (((((int)threadIdx.x) / 24) + 12) * 48)) + ((((int)threadIdx.x) % 24) * 2)))))[0] = ((float2*)(placeholder + (((((((((((int)threadIdx.x) + 1440) / 576) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) / 24) + 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)))))[0];
    ((float2*)(PaddedInput_shared + (((((int)threadIdx.x) * 2) + 3456))))[0] = ((float2*)(placeholder + ((((((((((int)blockIdx.x) / 5) * 46080) + ((((int)threadIdx.x) / 24) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)) + 691200))))[0];
    ((float2*)(PaddedInput_shared + ((((((((int)threadIdx.x) + 2016) / 576) * 1152) + (((((int)threadIdx.x) / 24) + 12) * 48)) + ((((int)threadIdx.x) % 24) * 2)))))[0] = ((float2*)(placeholder + (((((((((((int)threadIdx.x) + 2016) / 576) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) / 24) + 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + ((((int)threadIdx.x) % 24) * 16)) + (rc_outer_outer * 2)))))[0];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 32) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 4; ++xx_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
          Conv2dOutput[(((xx_outer_inner * 8) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((xx_outer_inner * 8) + (ff_outer_inner * 2)))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 32))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 32))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner) + 288))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 64))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 64))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner) + 576))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 96))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 96))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner) + 864))] * placeholder_shared[((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner))] * placeholder_shared[(((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 1))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 33))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 33))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner) + 288))] * placeholder_shared[(((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 1))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 65))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 65))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner) + 576))] * placeholder_shared[(((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 1))]));
          Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 97))] = (Conv2dOutput[((((xx_outer_inner * 8) + (ff_outer_inner * 2)) + 97))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 72) * 1152) + (((((int)threadIdx.x) % 72) >> 1) * 8)) + (xx_outer_inner * 2)) + rc_outer_inner) + 864))] * placeholder_shared[(((((rc_outer_inner * 16) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 1))]));
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_add[((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))] = ((Conv2dOutput[(((ax2_inner * 8) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 8) + ax3_inner))]) + placeholder3[((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]);
      T_add[(((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 11520))] = ((Conv2dOutput[((((ax2_inner * 8) + ax3_inner) + 32))] + placeholder2[((((((int)threadIdx.x) & 1) * 8) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 11520))]);
      T_add[(((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 23040))] = ((Conv2dOutput[((((ax2_inner * 8) + ax3_inner) + 64))] + placeholder2[((((((int)threadIdx.x) & 1) * 8) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 23040))]);
      T_add[(((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 34560))] = ((Conv2dOutput[((((ax2_inner * 8) + ax3_inner) + 96))] + placeholder2[((((((int)threadIdx.x) & 1) * 8) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 72) * 230400) + ((((int)blockIdx.x) / 5) * 46080)) + (((((int)threadIdx.x) % 72) / 12) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (ax2_inner * 16)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 34560))]);
    }
  }
}


