
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[4];
  __shared__ float PaddedInput_shared[126];
  __shared__ float placeholder_shared[9216];
  Conv3dOutput[(0)] = 0.000000e+00f;
  Conv3dOutput[(2)] = 0.000000e+00f;
  Conv3dOutput[(1)] = 0.000000e+00f;
  Conv3dOutput[(3)] = 0.000000e+00f;
  for (int rh_outer_outer = 0; rh_outer_outer < 3; ++rh_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 256; ++rc_outer_outer) {
      __syncthreads();
      if (((int)threadIdx.x) < 21) {
        PaddedInput_shared[((((int)threadIdx.x) * 6))] = 0.000000e+00f;
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 1))] = 0.000000e+00f;
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 2))] = (((((1 <= ((((int)threadIdx.x) / 7) + ((int)blockIdx.x))) && (((((int)threadIdx.x) / 7) + ((int)blockIdx.x)) < 15)) && (1 <= (rh_outer_outer + (((int)threadIdx.x) % 7)))) && ((rh_outer_outer + (((int)threadIdx.x) % 7)) < 8)) ? placeholder[((((((((int)blockIdx.x) * 3584) + (rh_outer_outer * 512)) + (((int)threadIdx.x) * 512)) + (rc_outer_outer * 2)) - 4096))] : 0.000000e+00f);
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 3))] = (((((1 <= ((((int)threadIdx.x) / 7) + ((int)blockIdx.x))) && (((((int)threadIdx.x) / 7) + ((int)blockIdx.x)) < 15)) && (1 <= (rh_outer_outer + (((int)threadIdx.x) % 7)))) && ((rh_outer_outer + (((int)threadIdx.x) % 7)) < 8)) ? placeholder[((((((((int)blockIdx.x) * 3584) + (rh_outer_outer * 512)) + (((int)threadIdx.x) * 512)) + (rc_outer_outer * 2)) - 4095))] : 0.000000e+00f);
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 4))] = 0.000000e+00f;
        PaddedInput_shared[(((((int)threadIdx.x) * 6) + 5))] = 0.000000e+00f;
      }
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rh_outer_outer * 786432) + (rc_outer_outer * 1024)) + ((int)threadIdx.x)))];
      placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[(((((rh_outer_outer * 786432) + (((((int)threadIdx.x) + 896) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 896) & 1023)))];
      placeholder_shared[((((int)threadIdx.x) + 1792))] = placeholder1[(((((rh_outer_outer * 786432) + (((((int)threadIdx.x) + 1792) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 768) & 1023)))];
      placeholder_shared[((((int)threadIdx.x) + 2688))] = placeholder1[((((((((((int)threadIdx.x) + 2688) / 3072) * 2359296) + (rh_outer_outer * 786432)) + ((((((int)threadIdx.x) + 2688) % 3072) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 640) & 1023)))];
      placeholder_shared[((((int)threadIdx.x) + 3584))] = placeholder1[((((((((((int)threadIdx.x) + 3584) / 3072) * 2359296) + (rh_outer_outer * 786432)) + (((((int)threadIdx.x) + 512) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 512) & 1023)))];
      placeholder_shared[((((int)threadIdx.x) + 4480))] = placeholder1[((((((((((int)threadIdx.x) + 4480) / 3072) * 2359296) + (rh_outer_outer * 786432)) + (((((int)threadIdx.x) + 1408) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 384) & 1023)))];
      placeholder_shared[((((int)threadIdx.x) + 5376))] = placeholder1[((((((((((int)threadIdx.x) + 5376) / 3072) * 2359296) + (rh_outer_outer * 786432)) + ((((((int)threadIdx.x) + 2304) % 3072) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 256) & 1023)))];
      placeholder_shared[((((int)threadIdx.x) + 6272))] = placeholder1[(((((((((int)threadIdx.x) + 6272) / 3072) * 2359296) + (rh_outer_outer * 786432)) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) + 128)))];
      placeholder_shared[((((int)threadIdx.x) + 7168))] = placeholder1[((((((((((int)threadIdx.x) + 7168) / 3072) * 2359296) + (rh_outer_outer * 786432)) + (((((int)threadIdx.x) + 1024) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((int)threadIdx.x)))];
      placeholder_shared[((((int)threadIdx.x) + 8064))] = placeholder1[((((((((((int)threadIdx.x) + 8064) / 3072) * 2359296) + (rh_outer_outer * 786432)) + (((((int)threadIdx.x) + 1920) >> 10) * 262144)) + (rc_outer_outer * 1024)) + ((((int)threadIdx.x) + 896) & 1023)))];
      if (((int)threadIdx.x) < 256) {
        placeholder_shared[((((int)threadIdx.x) + 8960))] = placeholder1[((((((((((int)threadIdx.x) + 8960) / 3072) * 2359296) + (rh_outer_outer * 786432)) + (((((int)threadIdx.x) + 2816) >> 10) * 262144)) + (rc_outer_outer * 1024)) + (((int)threadIdx.x) + 768)))];
      }
      __syncthreads();
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 7) * 6))] * placeholder_shared[(((((int)threadIdx.x) & 127) * 2))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 7) * 6))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 256))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1024))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1280))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2048))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2304))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3072))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3328))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 44))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4096))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 44))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4352))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 46))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5120))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 46))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5376))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6144))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6400))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 86))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7168))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 86))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7424))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 88))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8192))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 88))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8448))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 7) * 6))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 7) * 6))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 257))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1025))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1281))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2049))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2305))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3073))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 42))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3329))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 44))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4097))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 44))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4353))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 46))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5121))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 46))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5377))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6145))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 84))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6401))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 86))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7169))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 86))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7425))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 88))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8193))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 88))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8449))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 512))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 768))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1536))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1792))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2560))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2816))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3584))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3840))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 45))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4608))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 45))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4864))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 47))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5632))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 47))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5888))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6656))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6912))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 87))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7680))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 87))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7936))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 89))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8704))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 89))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8960))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 513))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 769))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1537))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 1793))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2561))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 2817))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3585))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 43))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 3841))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 45))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4609))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 45))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 4865))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 47))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5633))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 47))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 5889))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6657))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 85))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 6913))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 87))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7681))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 87))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 7937))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 89))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8705))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 7) * 6) + 89))] * placeholder_shared[((((((int)threadIdx.x) & 127) * 2) + 8961))]));
    }
  }
  for (int ax4_inner = 0; ax4_inner < 2; ++ax4_inner) {
    T_relu[(((((((int)blockIdx.x) * 3584) + ((((int)threadIdx.x) >> 7) * 512)) + ((((int)threadIdx.x) & 127) * 2)) + ax4_inner))] = max(((Conv3dOutput[(ax4_inner)] * placeholder2[((((((int)threadIdx.x) & 127) * 2) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 127) * 2) + ax4_inner))]), 0.000000e+00f);
    T_relu[((((((((int)blockIdx.x) * 3584) + ((((int)threadIdx.x) >> 7) * 512)) + ((((int)threadIdx.x) & 127) * 2)) + ax4_inner) + 256))] = max(((Conv3dOutput[((ax4_inner + 2))] * placeholder2[(((((((int)threadIdx.x) & 127) * 2) + ax4_inner) + 256))]) + placeholder3[(((((((int)threadIdx.x) & 127) * 2) + ax4_inner) + 256))]), 0.000000e+00f);
  }
}


