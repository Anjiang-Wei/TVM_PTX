
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[28];
  __shared__ float PaddedInput_shared[539];
  __shared__ float placeholder_shared[352];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
    Conv2dOutput[(ff_outer_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 4))] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 8))] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 12))] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 16))] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 20))] = 0.000000e+00f;
    Conv2dOutput[((ff_outer_inner_init + 24))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 7))] = placeholder[((((((((int)threadIdx.x) * 7) / 11) * 704) + (rc_outer_outer * 11)) + ((((int)threadIdx.x) * 7) % 11)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 7) + 1))] = placeholder[(((((((((int)threadIdx.x) * 7) + 1) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 1) % 11)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 7) + 2))] = placeholder[(((((((((int)threadIdx.x) * 7) + 2) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 2) % 11)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 7) + 3))] = placeholder[(((((((((int)threadIdx.x) * 7) + 3) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 3) % 11)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 7) + 4))] = placeholder[(((((((((int)threadIdx.x) * 7) + 4) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 4) % 11)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 7) + 5))] = placeholder[(((((((((int)threadIdx.x) * 7) + 5) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 5) % 11)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 7) + 6))] = placeholder[(((((((((int)threadIdx.x) * 7) + 6) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 6) % 11)))];
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 392))] = placeholder[(((((((((int)threadIdx.x) * 7) + 392) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 7) % 11)))];
    }
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 393))] = placeholder[(((((((((int)threadIdx.x) * 7) + 393) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 8) % 11)))];
    }
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 394))] = placeholder[(((((((((int)threadIdx.x) * 7) + 394) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 9) % 11)))];
    }
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 395))] = placeholder[(((((((((int)threadIdx.x) * 7) + 395) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 10) % 11)))];
    }
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 396))] = placeholder[(((((((((int)threadIdx.x) * 7) / 11) * 704) + (rc_outer_outer * 11)) + ((((int)threadIdx.x) * 7) % 11)) + 25344))];
    }
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 397))] = placeholder[(((((((((int)threadIdx.x) * 7) + 397) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 1) % 11)))];
    }
    if (((int)threadIdx.x) < 21) {
      PaddedInput_shared[(((((int)threadIdx.x) * 7) + 398))] = placeholder[(((((((((int)threadIdx.x) * 7) + 398) / 11) * 704) + (rc_outer_outer * 11)) + (((((int)threadIdx.x) * 7) + 2) % 11)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 1408) + ((((int)threadIdx.x) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 56))] = placeholder1[(((((rc_outer_outer * 1408) + (((((int)threadIdx.x) + 56) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 24) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((rc_outer_outer * 1408) + (((((int)threadIdx.x) + 112) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 16) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 168))] = placeholder1[(((((rc_outer_outer * 1408) + (((((int)threadIdx.x) + 168) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 8) & 31)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((((rc_outer_outer * 1408) + ((((int)threadIdx.x) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) & 31)) + 896))];
    placeholder_shared[((((int)threadIdx.x) + 280))] = placeholder1[(((((rc_outer_outer * 1408) + (((((int)threadIdx.x) + 280) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) + 24) & 31)))];
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[((((int)threadIdx.x) + 336))] = placeholder1[(((((rc_outer_outer * 1408) + (((((int)threadIdx.x) + 336) >> 5) * 128)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) + 16)))];
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 11; ++rc_inner) {
        Conv2dOutput[(ff_outer_inner)] = (Conv2dOutput[(ff_outer_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 77) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
        Conv2dOutput[((ff_outer_inner + 4))] = (Conv2dOutput[((ff_outer_inner + 4))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 77) + rc_inner) + 11))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
        Conv2dOutput[((ff_outer_inner + 8))] = (Conv2dOutput[((ff_outer_inner + 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 77) + rc_inner) + 22))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
        Conv2dOutput[((ff_outer_inner + 12))] = (Conv2dOutput[((ff_outer_inner + 12))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 77) + rc_inner) + 33))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
        Conv2dOutput[((ff_outer_inner + 16))] = (Conv2dOutput[((ff_outer_inner + 16))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 77) + rc_inner) + 44))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
        Conv2dOutput[((ff_outer_inner + 20))] = (Conv2dOutput[((ff_outer_inner + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 77) + rc_inner) + 55))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
        Conv2dOutput[((ff_outer_inner + 24))] = (Conv2dOutput[((ff_outer_inner + 24))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 77) + rc_inner) + 66))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 7) * 4)) + ff_outer_inner))]));
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[(((((((((int)threadIdx.x) >> 3) * 896) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 4) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


