
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_tanh) {
  float conv2d_transpose_nhwc[160];
  __shared__ float PadInput_shared[3872];
  __shared__ float placeholder_shared[32];
  for (int h_outer_inner_init = 0; h_outer_inner_init < 4; ++h_outer_inner_init) {
    for (int w_outer_inner_init = 0; w_outer_inner_init < 5; ++w_outer_inner_init) {
      for (int n_inner_init = 0; n_inner_init < 2; ++n_inner_init) {
        for (int h_inner_init = 0; h_inner_init < 2; ++h_inner_init) {
          for (int w_inner_init = 0; w_inner_init < 2; ++w_inner_init) {
            conv2d_transpose_nhwc[((((((n_inner_init * 80) + (h_outer_inner_init * 20)) + (h_inner_init * 10)) + (w_outer_inner_init * 2)) + w_inner_init))] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 97; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) < 1936) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 3872) {
          PadInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)))] = (((((1 <= (((((int)blockIdx.x) / 6) * 20) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 484) / 22))) && ((((((int)blockIdx.x) / 6) * 20) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 484) / 22)) < 41)) && (1 <= ((((((int)blockIdx.x) % 6) / 3) * 20) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 22)))) && (((((((int)blockIdx.x) % 6) / 3) * 20) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 22)) < 41)) ? placeholder[((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) / 484) * 204800) + ((((int)blockIdx.x) / 6) * 102400)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 484) / 22) * 5120)) + (((((int)blockIdx.x) % 6) / 3) * 2560)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 22) * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) - 5248))] : 0.000000e+00f);
        }
      }
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((int)threadIdx.x) >> 1) * 384) + (rc_outer_outer * 6)) + ((((int)threadIdx.x) & 1) * 3)) + (((int)blockIdx.x) % 3)))];
    }
    __syncthreads();
    for (int rh_outer_inner = 0; rh_outer_inner < 2; ++rh_outer_inner) {
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int h_outer_inner = 0; h_outer_inner < 4; ++h_outer_inner) {
          for (int w_outer_inner = 0; w_outer_inner < 5; ++w_outer_inner) {
            for (int rh_inner = 0; rh_inner < 2; ++rh_inner) {
              for (int rw_inner = 0; rw_inner < 4; ++rw_inner) {
                for (int n_inner = 0; n_inner < 2; ++n_inner) {
                  for (int h_inner = 0; h_inner < 2; ++h_inner) {
                    for (int w_inner = 0; w_inner < 2; ++w_inner) {
                      conv2d_transpose_nhwc[((((((n_inner * 80) + (h_outer_inner * 20)) + (h_inner * 10)) + (w_outer_inner * 2)) + w_inner))] = (conv2d_transpose_nhwc[((((((n_inner * 80) + (h_outer_inner * 20)) + (h_inner * 10)) + (w_outer_inner * 2)) + w_inner))] + ((((((h_inner + rh_inner) % 2) == 0) && (((w_inner + rw_inner) % 2) == 0)) ? PadInput_shared[((((((((((((((int)threadIdx.x) / 20) * 1936) + (n_inner * 968)) + (((((int)threadIdx.x) % 20) >> 2) * 176)) + (((h_inner + rh_inner) >> 1) * 44)) + (h_outer_inner * 44)) + (rh_outer_inner * 44)) + ((((int)threadIdx.x) & 3) * 10)) + (((w_inner + rw_inner) >> 1) * 2)) + (w_outer_inner * 2)) + rc_outer_inner))] : 0.000000e+00f) * placeholder_shared[(((((rc_outer_inner + 30) - (rw_inner * 2)) - (rh_inner * 8)) - (rh_outer_inner * 16)))]));
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 10; ++ax2_inner) {
        T_tanh[(((((((((((((int)threadIdx.x) / 20) * 38400) + (ax0_inner * 19200)) + ((((int)blockIdx.x) / 6) * 9600)) + (((((int)threadIdx.x) % 20) >> 2) * 1920)) + (ax1_inner * 240)) + (((((int)blockIdx.x) % 6) / 3) * 120)) + ((((int)threadIdx.x) & 3) * 30)) + (ax2_inner * 3)) + (((int)blockIdx.x) % 3)))] = tanhf(conv2d_transpose_nhwc[((((ax0_inner * 80) + (ax1_inner * 10)) + ax2_inner))]);
      }
    }
  }
}


