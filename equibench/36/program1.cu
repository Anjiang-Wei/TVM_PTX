
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ tensor, float* __restrict__ tensor1) {
  tensor[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = (tensor1[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] * 2.500000e-01f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int dh = 0; dh < 2; ++dh) {
    for (int dw = 0; dw < 2; ++dw) {
      tensor[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = (tensor[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[(((((((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 3584) * 14336) + (dh * 7168)) + (((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 3584) >> 9) * 1024)) + (dw * 512)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) & 511)))]);
    }
  }
}


