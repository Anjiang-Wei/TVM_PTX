
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[36];
  __shared__ float PaddedInput_shared[36];
  __shared__ float placeholder_shared[72];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 3; ++xx_outer_inner_init) {
    for (int nn_inner_init = 0; nn_inner_init < 4; ++nn_inner_init) {
      Conv2dOutput[(((nn_inner_init * 3) + xx_outer_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 3) + xx_outer_inner_init) + 12))] = 0.000000e+00f;
      Conv2dOutput[((((nn_inner_init * 3) + xx_outer_inner_init) + 24))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 24; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 36) {
        if (((int)threadIdx.x) < 8) {
          PaddedInput_shared[(((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 9) * 86400) + ((((int)blockIdx.x) / 20) * 4320)) + (((((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 9) / 3) * 1440)) + ((((int)blockIdx.x) % 20) * 72)) + ((((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 3) * 24)) + rc_outer_outer))];
        }
      }
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 72) + ((int)threadIdx.x)))];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
      for (int nn_inner = 0; nn_inner < 4; ++nn_inner) {
        Conv2dOutput[(((nn_inner * 3) + xx_outer_inner))] = (Conv2dOutput[(((nn_inner * 3) + xx_outer_inner))] + (PaddedInput_shared[(((nn_inner * 9) + xx_outer_inner))] * placeholder_shared[(((int)threadIdx.x))]));
        Conv2dOutput[((((nn_inner * 3) + xx_outer_inner) + 12))] = (Conv2dOutput[((((nn_inner * 3) + xx_outer_inner) + 12))] + (PaddedInput_shared[((((nn_inner * 9) + xx_outer_inner) + 3))] * placeholder_shared[(((int)threadIdx.x))]));
        Conv2dOutput[((((nn_inner * 3) + xx_outer_inner) + 24))] = (Conv2dOutput[((((nn_inner * 3) + xx_outer_inner) + 24))] + (PaddedInput_shared[((((nn_inner * 9) + xx_outer_inner) + 6))] * placeholder_shared[(((int)threadIdx.x))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      T_relu[((((((ax0_inner * 259200) + ((((int)blockIdx.x) / 20) * 12960)) + ((((int)blockIdx.x) % 20) * 216)) + (ax2_inner * 72)) + ((int)threadIdx.x)))] = max((Conv2dOutput[(((ax0_inner * 3) + ax2_inner))] + placeholder2[(((int)threadIdx.x))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 259200) + ((((int)blockIdx.x) / 20) * 12960)) + ((((int)blockIdx.x) % 20) * 216)) + (ax2_inner * 72)) + ((int)threadIdx.x)) + 4320))] = max((Conv2dOutput[((((ax0_inner * 3) + ax2_inner) + 12))] + placeholder2[(((int)threadIdx.x))]), 0.000000e+00f);
      T_relu[(((((((ax0_inner * 259200) + ((((int)blockIdx.x) / 20) * 12960)) + ((((int)blockIdx.x) % 20) * 216)) + (ax2_inner * 72)) + ((int)threadIdx.x)) + 8640))] = max((Conv2dOutput[((((ax0_inner * 3) + ax2_inner) + 24))] + placeholder2[(((int)threadIdx.x))]), 0.000000e+00f);
    }
  }
}


