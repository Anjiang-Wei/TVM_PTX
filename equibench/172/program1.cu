
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[28];
  __shared__ float PaddedInput_shared[8400];
  __shared__ float placeholder_shared[192];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        Conv2dOutput[((((nn_inner_init * 14) + (yy_outer_inner_init * 2)) + ff_outer_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 49; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 49) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 8400) {
        if (((int)threadIdx.x) < 172) {
          PaddedInput_shared[(((((int)threadIdx.x) * 49) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[(((((((((((int)threadIdx.x) * 49) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 60) * 6720) + ((((int)blockIdx.x) >> 1) * 960)) + (((((((int)threadIdx.x) * 49) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 60) / 12) * 192)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 49) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 12)))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) {
      if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) < 192) {
        if (((int)threadIdx.x) < 96) {
          placeholder_shared[(((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1))] = placeholder1[(((((rc_outer_outer * 384) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) >> 4) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s1) & 15)))];
        }
      }
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 12; ++rc_inner) {
          for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
            Conv2dOutput[((((nn_inner * 14) + (yy_outer_inner * 2)) + ff_outer_inner))] = (Conv2dOutput[((((nn_inner * 14) + (yy_outer_inner * 2)) + ff_outer_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 200) * 4200) + (nn_inner * 2100)) + (((((int)threadIdx.x) % 200) / 40) * 420)) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 40) >> 3) * 12)) + rc_inner))] * placeholder_shared[((((rc_inner * 16) + ((((int)threadIdx.x) & 7) * 2)) + ff_outer_inner))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((((((((int)threadIdx.x) / 200) * 78400) + (ax0_inner * 39200)) + (((((int)threadIdx.x) % 200) / 40) * 7840)) + (ax1_inner * 1120)) + ((((int)blockIdx.x) >> 1) * 160)) + (((((int)threadIdx.x) % 40) >> 3) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 14) + (ax1_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


