
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 480))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 480))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[64];
  __shared__ float PaddedInput_shared[11520];
  __shared__ float placeholder_shared[60];
  DepthwiseConv2d_local[(0)] = 0.000000e+00f;
  DepthwiseConv2d_local[(16)] = 0.000000e+00f;
  DepthwiseConv2d_local[(32)] = 0.000000e+00f;
  DepthwiseConv2d_local[(48)] = 0.000000e+00f;
  DepthwiseConv2d_local[(1)] = 0.000000e+00f;
  DepthwiseConv2d_local[(17)] = 0.000000e+00f;
  DepthwiseConv2d_local[(33)] = 0.000000e+00f;
  DepthwiseConv2d_local[(49)] = 0.000000e+00f;
  DepthwiseConv2d_local[(2)] = 0.000000e+00f;
  DepthwiseConv2d_local[(18)] = 0.000000e+00f;
  DepthwiseConv2d_local[(34)] = 0.000000e+00f;
  DepthwiseConv2d_local[(50)] = 0.000000e+00f;
  DepthwiseConv2d_local[(3)] = 0.000000e+00f;
  DepthwiseConv2d_local[(19)] = 0.000000e+00f;
  DepthwiseConv2d_local[(35)] = 0.000000e+00f;
  DepthwiseConv2d_local[(51)] = 0.000000e+00f;
  DepthwiseConv2d_local[(4)] = 0.000000e+00f;
  DepthwiseConv2d_local[(20)] = 0.000000e+00f;
  DepthwiseConv2d_local[(36)] = 0.000000e+00f;
  DepthwiseConv2d_local[(52)] = 0.000000e+00f;
  DepthwiseConv2d_local[(5)] = 0.000000e+00f;
  DepthwiseConv2d_local[(21)] = 0.000000e+00f;
  DepthwiseConv2d_local[(37)] = 0.000000e+00f;
  DepthwiseConv2d_local[(53)] = 0.000000e+00f;
  DepthwiseConv2d_local[(6)] = 0.000000e+00f;
  DepthwiseConv2d_local[(22)] = 0.000000e+00f;
  DepthwiseConv2d_local[(38)] = 0.000000e+00f;
  DepthwiseConv2d_local[(54)] = 0.000000e+00f;
  DepthwiseConv2d_local[(7)] = 0.000000e+00f;
  DepthwiseConv2d_local[(23)] = 0.000000e+00f;
  DepthwiseConv2d_local[(39)] = 0.000000e+00f;
  DepthwiseConv2d_local[(55)] = 0.000000e+00f;
  DepthwiseConv2d_local[(8)] = 0.000000e+00f;
  DepthwiseConv2d_local[(24)] = 0.000000e+00f;
  DepthwiseConv2d_local[(40)] = 0.000000e+00f;
  DepthwiseConv2d_local[(56)] = 0.000000e+00f;
  DepthwiseConv2d_local[(9)] = 0.000000e+00f;
  DepthwiseConv2d_local[(25)] = 0.000000e+00f;
  DepthwiseConv2d_local[(41)] = 0.000000e+00f;
  DepthwiseConv2d_local[(57)] = 0.000000e+00f;
  DepthwiseConv2d_local[(10)] = 0.000000e+00f;
  DepthwiseConv2d_local[(26)] = 0.000000e+00f;
  DepthwiseConv2d_local[(42)] = 0.000000e+00f;
  DepthwiseConv2d_local[(58)] = 0.000000e+00f;
  DepthwiseConv2d_local[(11)] = 0.000000e+00f;
  DepthwiseConv2d_local[(27)] = 0.000000e+00f;
  DepthwiseConv2d_local[(43)] = 0.000000e+00f;
  DepthwiseConv2d_local[(59)] = 0.000000e+00f;
  DepthwiseConv2d_local[(12)] = 0.000000e+00f;
  DepthwiseConv2d_local[(28)] = 0.000000e+00f;
  DepthwiseConv2d_local[(44)] = 0.000000e+00f;
  DepthwiseConv2d_local[(60)] = 0.000000e+00f;
  DepthwiseConv2d_local[(13)] = 0.000000e+00f;
  DepthwiseConv2d_local[(29)] = 0.000000e+00f;
  DepthwiseConv2d_local[(45)] = 0.000000e+00f;
  DepthwiseConv2d_local[(61)] = 0.000000e+00f;
  DepthwiseConv2d_local[(14)] = 0.000000e+00f;
  DepthwiseConv2d_local[(30)] = 0.000000e+00f;
  DepthwiseConv2d_local[(46)] = 0.000000e+00f;
  DepthwiseConv2d_local[(62)] = 0.000000e+00f;
  DepthwiseConv2d_local[(15)] = 0.000000e+00f;
  DepthwiseConv2d_local[(31)] = 0.000000e+00f;
  DepthwiseConv2d_local[(47)] = 0.000000e+00f;
  DepthwiseConv2d_local[(63)] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 72; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 160) + ((int)threadIdx.x)))] = (((((1 <= ((((((int)blockIdx.x) / 24) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) % 72) / 18)) + di_outer_outer)) && (((((((int)blockIdx.x) / 24) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) % 72) / 18)) + di_outer_outer) < 17)) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) % 18))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) % 18) < 17)) ? placeholder[((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) / 72) * 122880) + ((((int)blockIdx.x) / 24) * 30720)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) % 72) / 18) * 7680)) + (di_outer_outer * 7680)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + (((int)threadIdx.x) / 20)) % 18) * 480)) + ((((int)blockIdx.x) % 24) * 20)) + (((int)threadIdx.x) % 20)) - 8160))] : 0.000000e+00f);
    }
    if (((int)threadIdx.x) < 60) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((di_outer_outer * 1440) + ((((int)threadIdx.x) / 20) * 480)) + ((((int)blockIdx.x) % 24) * 20)) + (((int)threadIdx.x) % 20)))];
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
      DepthwiseConv2d_local[((b_c_outer_inner * 4))] = (DepthwiseConv2d_local[((b_c_outer_inner * 4))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)))] * placeholder_shared[((((int)threadIdx.x) % 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 16))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 16))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 5))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 32))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 32))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 10))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 10))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 48))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 48))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 15))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 15))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 1))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 20))] * placeholder_shared[((((int)threadIdx.x) % 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 17))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 17))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 25))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 33))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 33))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 30))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 10))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 49))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 49))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 35))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 15))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 2))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 360))] * placeholder_shared[((((int)threadIdx.x) % 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 18))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 18))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 365))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 34))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 34))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 370))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 10))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 50))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 50))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 375))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 15))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 3))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 380))] * placeholder_shared[((((int)threadIdx.x) % 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 19))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 19))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 385))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 5))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 35))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 35))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 390))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 10))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 51))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 51))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 395))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 15))]));
      DepthwiseConv2d_local[((b_c_outer_inner * 4))] = (DepthwiseConv2d_local[((b_c_outer_inner * 4))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 20))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 20))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 16))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 16))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 25))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 25))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 32))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 32))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 30))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 30))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 48))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 48))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 35))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 35))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 1))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 40))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 20))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 17))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 17))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 45))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 25))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 33))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 33))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 50))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 30))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 49))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 49))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 55))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 35))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 2))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 380))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 20))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 18))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 18))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 385))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 25))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 34))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 34))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 390))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 30))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 50))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 50))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 395))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 35))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 3))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 400))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 20))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 19))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 19))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 405))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 25))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 35))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 35))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 410))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 30))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 51))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 51))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 415))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 35))]));
      DepthwiseConv2d_local[((b_c_outer_inner * 4))] = (DepthwiseConv2d_local[((b_c_outer_inner * 4))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 40))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 40))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 16))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 16))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 45))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 45))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 32))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 32))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 50))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 50))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 48))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 48))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 55))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 55))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 1))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 60))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 40))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 17))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 17))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 65))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 45))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 33))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 33))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 70))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 50))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 49))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 49))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 75))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 55))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 2))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 400))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 40))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 18))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 18))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 405))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 45))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 34))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 34))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 410))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 50))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 50))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 50))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 415))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 55))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 3))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 420))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 40))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 19))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 19))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 425))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 45))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 35))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 35))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 430))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 50))]));
      DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 51))] = (DepthwiseConv2d_local[(((b_c_outer_inner * 4) + 51))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 80) * 5760) + (b_c_outer_inner * 1440)) + (((((int)threadIdx.x) % 80) / 40) * 720)) + (((((int)threadIdx.x) % 40) / 5) * 40)) + (((int)threadIdx.x) % 5)) + 435))] * placeholder_shared[(((((int)threadIdx.x) % 5) + 55))]));
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      for (int j_inner = 0; j_inner < 2; ++j_inner) {
        DepthwiseConv2d[(((((((((((((int)threadIdx.x) / 80) * 491520) + (b_inner * 122880)) + ((((int)blockIdx.x) / 24) * 30720)) + (((((int)threadIdx.x) % 80) / 40) * 15360)) + (i_inner * 7680)) + (((((int)threadIdx.x) % 40) / 5) * 960)) + (j_inner * 480)) + ((((int)blockIdx.x) % 24) * 20)) + (((int)threadIdx.x) % 5)))] = DepthwiseConv2d_local[((((b_inner * 4) + (i_inner * 2)) + j_inner))];
        DepthwiseConv2d[((((((((((((((int)threadIdx.x) / 80) * 491520) + (b_inner * 122880)) + ((((int)blockIdx.x) / 24) * 30720)) + (((((int)threadIdx.x) % 80) / 40) * 15360)) + (i_inner * 7680)) + (((((int)threadIdx.x) % 40) / 5) * 960)) + (j_inner * 480)) + ((((int)blockIdx.x) % 24) * 20)) + (((int)threadIdx.x) % 5)) + 5))] = DepthwiseConv2d_local[(((((b_inner * 4) + (i_inner * 2)) + j_inner) + 16))];
        DepthwiseConv2d[((((((((((((((int)threadIdx.x) / 80) * 491520) + (b_inner * 122880)) + ((((int)blockIdx.x) / 24) * 30720)) + (((((int)threadIdx.x) % 80) / 40) * 15360)) + (i_inner * 7680)) + (((((int)threadIdx.x) % 40) / 5) * 960)) + (j_inner * 480)) + ((((int)blockIdx.x) % 24) * 20)) + (((int)threadIdx.x) % 5)) + 10))] = DepthwiseConv2d_local[(((((b_inner * 4) + (i_inner * 2)) + j_inner) + 32))];
        DepthwiseConv2d[((((((((((((((int)threadIdx.x) / 80) * 491520) + (b_inner * 122880)) + ((((int)blockIdx.x) / 24) * 30720)) + (((((int)threadIdx.x) % 80) / 40) * 15360)) + (i_inner * 7680)) + (((((int)threadIdx.x) % 40) / 5) * 960)) + (j_inner * 480)) + ((((int)blockIdx.x) % 24) * 20)) + (((int)threadIdx.x) % 5)) + 15))] = DepthwiseConv2d_local[(((((b_inner * 4) + (i_inner * 2)) + j_inner) + 48))];
      }
    }
  }
}


