
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[34];
  __shared__ float PaddedInput_shared[578];
  __shared__ float placeholder_shared[32];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 17; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 1))] = 0.000000e+00f;
  }
  for (int rx_outer_outer = 0; rx_outer_outer < 7; ++rx_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
      __syncthreads();
      PaddedInput_shared[(((int)threadIdx.x))] = (((3 <= (((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer)) && ((((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer) < 20)) ? placeholder[(((((((((int)threadIdx.x) >> 1) * 128) + (rx_outer_outer * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) - 384))] : 0.000000e+00f);
      PaddedInput_shared[((((int)threadIdx.x) + 136))] = (((3 <= (((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer)) && ((((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer) < 20)) ? placeholder[(((((((((int)threadIdx.x) >> 1) * 128) + (rx_outer_outer * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 8320))] : 0.000000e+00f);
      PaddedInput_shared[((((int)threadIdx.x) + 272))] = (((3 <= (((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer)) && ((((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer) < 20)) ? placeholder[(((((((((int)threadIdx.x) >> 1) * 128) + (rx_outer_outer * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 17024))] : 0.000000e+00f);
      PaddedInput_shared[((((int)threadIdx.x) + 408))] = (((3 <= (((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer)) && ((((((int)threadIdx.x) % 34) >> 1) + rx_outer_outer) < 20)) ? placeholder[(((((((((int)threadIdx.x) >> 1) * 128) + (rx_outer_outer * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 25728))] : 0.000000e+00f);
      if (((int)threadIdx.x) < 34) {
        PaddedInput_shared[((((int)threadIdx.x) + 544))] = (((3 <= ((((int)threadIdx.x) >> 1) + rx_outer_outer)) && (((((int)threadIdx.x) >> 1) + rx_outer_outer) < 20)) ? placeholder[(((((((((int)threadIdx.x) >> 1) * 128) + (rx_outer_outer * 128)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 34432))] : 0.000000e+00f);
      }
      if (((int)threadIdx.x) < 16) {
        placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[((((((rx_outer_outer * 16384) + (rc_outer_outer * 256)) + ((((int)threadIdx.x) >> 3) * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)))];
      }
      if (((int)threadIdx.x) < 16) {
        placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[((((((rx_outer_outer * 16384) + (rc_outer_outer * 256)) + ((((((int)threadIdx.x) * 2) + 1) >> 4) * 128)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 2) + 1) & 15)))];
      }
      __syncthreads();
      for (int yy_outer_inner = 0; yy_outer_inner < 17; ++yy_outer_inner) {
        Conv2dOutput[((yy_outer_inner * 2))] = (Conv2dOutput[((yy_outer_inner * 2))] + (PaddedInput_shared[(((yy_outer_inner * 34) + ((((int)threadIdx.x) >> 3) * 2)))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 2))]));
        Conv2dOutput[((yy_outer_inner * 2))] = (Conv2dOutput[((yy_outer_inner * 2))] + (PaddedInput_shared[((((yy_outer_inner * 34) + ((((int)threadIdx.x) >> 3) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 2) + 16))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 1))] = (Conv2dOutput[(((yy_outer_inner * 2) + 1))] + (PaddedInput_shared[(((yy_outer_inner * 34) + ((((int)threadIdx.x) >> 3) * 2)))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 2) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 1))] = (Conv2dOutput[(((yy_outer_inner * 2) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 34) + ((((int)threadIdx.x) >> 3) * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 2) + 17))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 17; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[((((((ax1_inner * 2176) + ((((int)threadIdx.x) >> 3) * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


