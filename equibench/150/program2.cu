
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[4800];
  __shared__ float PaddedInput_shared[1800];
  __shared__ float placeholder_shared[512];
  for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 5; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 15; ++xx_inner_init) {
        Conv2dOutput[((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 2))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 3))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 4))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 5))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 6))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 7))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 8))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 9))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 10))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 11))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 12))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 13))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 14))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 15))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 16))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 17))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 18))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 19))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 20))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 21))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 22))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 23))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 24))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 25))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 26))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 27))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 28))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 29))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 30))] = 0.000000e+00f;
        Conv2dOutput[(((((nn_inner_init * 2400) + (yy_inner_init * 480)) + (xx_inner_init * 32)) + 31))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 450) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1800) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)))] = placeholder[(((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 450) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1799) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 1))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 450) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1798) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 2))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 2))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 450) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1797) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 3))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 3))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 449) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1796) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 4))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 256))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 449) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1795) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 5))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 257))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 449) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1794) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 6))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 258))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 449) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1793) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 7))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 259))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 448) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1792) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 8))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 512))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 448) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1791) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 9))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 513))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 448) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1790) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 10))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 514))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 448) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1789) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 11))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 515))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 447) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1788) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 12))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 768))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 447) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1787) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 13))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 769))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 447) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1786) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 14))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 770))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 447) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1785) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 15))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 771))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 446) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1784) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 16))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1024))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 446) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1783) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 17))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1025))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 446) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1782) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 18))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1026))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 446) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1781) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 19))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1027))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 445) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1780) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 20))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1280))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 445) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1779) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 21))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1281))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 445) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1778) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 22))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1282))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 445) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1777) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 23))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1283))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 444) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1776) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 24))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1536))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 444) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1775) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 25))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1537))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 444) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1774) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 26))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1538))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 444) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1773) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 27))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1539))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 443) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1772) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 28))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1792))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 443) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1771) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 29))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1793))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 443) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1770) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 30))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1794))];
          }
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 96) + (((int)threadIdx.x) * 8)) < 443) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) < 1769) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 12) + ((int)threadIdx.x)) < 57) {
            PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 384) + (((int)threadIdx.x) * 32)) + 31))] = placeholder[((((((((int)blockIdx.x) * 115200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 24576)) + (((int)threadIdx.x) * 2048)) + (rc_outer_outer * 4)) + 1795))];
          }
        }
      }
    }
    placeholder_shared[((((int)threadIdx.x) * 4))] = placeholder1[(((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 3))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 48))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 48))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 49))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 49))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 50))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 50))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 51))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 51))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 96))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 96))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 97))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 97))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 98))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 98))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 99))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 99))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 144))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 144))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 145))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 145))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 146))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 146))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 147))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 147))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 192))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 192))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 193))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 193))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 194))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 194))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 195))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 195))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 240))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 240))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 241))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 241))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 242))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 242))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 243))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 243))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 288))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 288))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 289))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 289))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 290))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 290))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 291))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 291))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 336))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 336))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 337))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 337))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 338))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 338))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 339))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 339))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 384))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 384))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 385))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 385))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 386))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 386))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 387))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 387))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 432))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 432))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 433))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 433))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 434))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 434))];
    placeholder_shared[(((((int)threadIdx.x) * 4) + 435))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 435))];
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 480))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 480))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 481))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 481))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 482))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 482))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 483))] = placeholder1[((((rc_outer_outer * 512) + (((int)threadIdx.x) * 4)) + 483))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
      for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
        for (int yy_inner = 0; yy_inner < 5; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 15; ++xx_inner) {
            Conv2dOutput[((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)))] = (Conv2dOutput[((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[(((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 1))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 1))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 1))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 2))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 2))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 2))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 3))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 3))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 3))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 4))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 4))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 4))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 5))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 5))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 5))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 6))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 6))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 6))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 7))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 7))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 7))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 8))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 8))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 8))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 9))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 9))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 9))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 10))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 10))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 10))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 11))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 11))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 11))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 12))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 12))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 12))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 13))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 13))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 13))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 14))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 14))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 14))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 15))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 15))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 15))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 16))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 16))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 16))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 17))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 17))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 17))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 18))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 18))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 18))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 19))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 19))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 19))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 20))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 20))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 20))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 21))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 21))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 21))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 22))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 22))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 22))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 23))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 23))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 23))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 24))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 24))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 24))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 25))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 25))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 25))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 26))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 26))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 26))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 27))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 27))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 27))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 28))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 28))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 28))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 29))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 29))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 29))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 30))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 30))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 30))]));
            Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 31))] = (Conv2dOutput[(((((nn_inner * 2400) + (yy_inner * 480)) + (xx_inner * 32)) + 31))] + (PaddedInput_shared[((((((nn_inner * 900) + ((((int)threadIdx.x) >> 2) * 300)) + (yy_inner * 60)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + ((((int)threadIdx.x) & 3) * 32)) + 31))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
          T_relu[((((((((((int)blockIdx.x) * 57600) + (ax0_inner * 28800)) + ((((int)threadIdx.x) >> 2) * 9600)) + (ax1_inner * 1920)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 3) * 32)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 2400) + (ax1_inner * 480)) + (ax2_inner * 32)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 3) * 32) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


