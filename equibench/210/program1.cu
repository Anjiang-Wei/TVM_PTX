
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[28];
  __shared__ float PaddedInput_shared[196];
  __shared__ float placeholder_shared[16];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 14))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 2) + 15))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 608; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((int)threadIdx.x) * 608) + rc_outer_outer))];
    if (((int)threadIdx.x) < 84) {
      PaddedInput_shared[((((int)threadIdx.x) + 112))] = placeholder[((((((int)threadIdx.x) * 608) + rc_outer_outer) + 68096))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[((((int)threadIdx.x) * 2))] = placeholder1[((((rc_outer_outer * 128) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 2)))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder1[(((((rc_outer_outer * 128) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
    }
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
      Conv2dOutput[((xx_outer_inner * 2))] = (Conv2dOutput[((xx_outer_inner * 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 7) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 2))]));
      Conv2dOutput[(((xx_outer_inner * 2) + 14))] = (Conv2dOutput[(((xx_outer_inner * 2) + 14))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 7) + xx_outer_inner) + 98))] * placeholder_shared[(((((int)threadIdx.x) & 7) * 2))]));
      Conv2dOutput[(((xx_outer_inner * 2) + 1))] = (Conv2dOutput[(((xx_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 7) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 2) + 1))]));
      Conv2dOutput[(((xx_outer_inner * 2) + 15))] = (Conv2dOutput[(((xx_outer_inner * 2) + 15))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 7) + xx_outer_inner) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 2) + 1))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((((((int)threadIdx.x) >> 3) * 896) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 2) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 3) * 896) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 12544))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 14))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


