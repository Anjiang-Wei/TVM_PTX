
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[56];
  __shared__ float PaddedInput_shared[3364];
  __shared__ float placeholder_shared[9];
  for (int i_inner_init = 0; i_inner_init < 56; ++i_inner_init) {
    DepthwiseConv2d[(i_inner_init)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 61; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 3364) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = (((((58 <= ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x))) && (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 3306)) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 58))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 58) < 57)) ? placeholder[(((((((((int)blockIdx.x) / 72) * 225792) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 58) * 4032)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 58) * 72)) + (((int)blockIdx.x) % 72)) - 4104))] : 0.000000e+00f);
    }
  }
  if (((int)threadIdx.x) < 3) {
    int3 _1 = (make_int3)((((((int)threadIdx.x) * 216) + (((int)blockIdx.x) % 72)))+(72*0), (((((int)threadIdx.x) * 216) + (((int)blockIdx.x) % 72)))+(72*1), (((((int)threadIdx.x) * 216) + (((int)blockIdx.x) % 72)))+(72*2));
    ((float3*)(placeholder_shared + ((((int)threadIdx.x) * 3))))[0] = make_float3(placeholder1[_1.x],placeholder1[_1.y],placeholder1[_1.z]);
  }
  __syncthreads();
  for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
    for (int dj_inner = 0; dj_inner < 3; ++dj_inner) {
      for (int i_inner = 0; i_inner < 56; ++i_inner) {
        DepthwiseConv2d[(i_inner)] = (DepthwiseConv2d[(i_inner)] + (PaddedInput_shared[(((((i_inner * 58) + (di_outer_inner * 58)) + ((int)threadIdx.x)) + dj_inner))] * placeholder_shared[(((di_outer_inner * 3) + dj_inner))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 56; ++ax1_inner) {
    T_relu[((((((((int)blockIdx.x) / 72) * 225792) + (ax1_inner * 4032)) + (((int)threadIdx.x) * 72)) + (((int)blockIdx.x) % 72)))] = max((DepthwiseConv2d[(ax1_inner)] + placeholder2[((((int)blockIdx.x) % 72))]), 0.000000e+00f);
  }
}


