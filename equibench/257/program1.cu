
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[64];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[(((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 8192))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[(((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 16384))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[(((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 24576))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[(((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 65536))];
    PaddedInput_shared[((((int)threadIdx.x) + 160))] = placeholder[((((((((((((int)threadIdx.x) + 160) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 2) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[((((((((((((int)threadIdx.x) + 192) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[((((((((((((int)threadIdx.x) + 224) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 6) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[(((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 131072))];
    PaddedInput_shared[((((int)threadIdx.x) + 288))] = placeholder[((((((((((((int)threadIdx.x) + 288) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 2) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((((((int)threadIdx.x) + 320) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 352))] = placeholder[((((((((((((int)threadIdx.x) + 352) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 6) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[(((((((((((int)blockIdx.x) >> 10) * 32768) + ((((int)threadIdx.x) >> 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 196608))];
    PaddedInput_shared[((((int)threadIdx.x) + 416))] = placeholder[((((((((((((int)threadIdx.x) + 416) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 2) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[((((((((((((int)threadIdx.x) + 448) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 4) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 480))] = placeholder[((((((((((((int)threadIdx.x) + 480) >> 7) * 65536) + ((((int)blockIdx.x) >> 10) * 32768)) + (((((int)threadIdx.x) >> 4) + 6) * 4096)) + (((((int)blockIdx.x) & 1023) >> 7) * 512)) + (((((int)threadIdx.x) & 15) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 8192) + ((((int)threadIdx.x) >> 3) * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((((rc_outer_outer * 8192) + ((((int)threadIdx.x) >> 3) * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 4096))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 64))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 256))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 320))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 128))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 192))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 384))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 448))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 8))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 72))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 264))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 328))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 136))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 200))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 392))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 16) + rc_outer_inner) + 456))] * placeholder_shared[(((rc_outer_inner * 8) + (((int)threadIdx.x) & 7)))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      T_relu[((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)))] = max(((Conv2dOutput[(((ax0_inner * 2) + ax2_inner))] + placeholder2[((((((int)blockIdx.x) & 127) * 8) + (((int)threadIdx.x) & 7)))]) + placeholder3[((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)))]), 0.000000e+00f);
      T_relu[(((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 65536))] = max(((Conv2dOutput[((((ax0_inner * 2) + ax2_inner) + 4))] + placeholder2[((((((int)blockIdx.x) & 127) * 8) + (((int)threadIdx.x) & 7)))]) + placeholder3[(((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 65536))]), 0.000000e+00f);
      T_relu[(((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 524288))] = max(((Conv2dOutput[((((ax0_inner * 2) + ax2_inner) + 8))] + placeholder2[((((((int)blockIdx.x) & 127) * 8) + (((int)threadIdx.x) & 7)))]) + placeholder3[(((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 524288))]), 0.000000e+00f);
      T_relu[(((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 589824))] = max(((Conv2dOutput[((((ax0_inner * 2) + ax2_inner) + 12))] + placeholder2[((((((int)blockIdx.x) & 127) * 8) + (((int)threadIdx.x) & 7)))]) + placeholder3[(((((((((ax0_inner * 262144) + ((((int)blockIdx.x) >> 10) * 131072)) + ((((int)threadIdx.x) >> 3) * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (ax2_inner * 1024)) + ((((int)blockIdx.x) & 127) * 8)) + (((int)threadIdx.x) & 7)) + 589824))]), 0.000000e+00f);
    }
  }
}


