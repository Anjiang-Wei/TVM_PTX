
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[32];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) >> 3) * 16384) + ((((int)blockIdx.x) / 48) * 4096)) + (((((int)threadIdx.x) & 7) >> 2) * 2048)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 1536) + ((((int)threadIdx.x) >> 3) * 384)) + ((((int)blockIdx.x) % 48) * 8)) + (((int)threadIdx.x) & 7)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 3) * 8))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 4))] * placeholder_shared[((((int)threadIdx.x) & 7))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 1))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 5))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 8))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 2))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 16))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 6))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 16))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 3))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 24))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 8) + 7))] * placeholder_shared[(((((int)threadIdx.x) & 7) + 24))]));
  }
  for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
    T_relu[(((((((((int)threadIdx.x) >> 3) * 3072) + ((((int)blockIdx.x) / 48) * 768)) + (ax2_inner * 384)) + ((((int)blockIdx.x) % 48) * 8)) + (((int)threadIdx.x) & 7)))] = max((Conv2dOutput[(ax2_inner)] + placeholder2[((((((int)blockIdx.x) % 48) * 8) + (((int)threadIdx.x) & 7)))]), 0.000000e+00f);
  }
}


