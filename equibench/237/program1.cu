
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[56];
  __shared__ float PaddedInput_shared[4480];
  __shared__ float placeholder_shared[2048];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 224; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      int2 _1;
        int2 _2;
          int2 _3;
            int2 _4 = make_int2(((((((int)blockIdx.x) / 20) * 44800) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) / 896) * 8960)) + (((((int)blockIdx.x) % 20) >> 2) * 1792)), ((((((int)blockIdx.x) / 20) * 44800) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) / 896) * 8960)) + (((((int)blockIdx.x) % 20) >> 2) * 1792)));
            int2 _5;
              int2 _6;
                int2 _7 = (make_int2)(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) % 896))+(1*0), ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) % 896))+(1*1));
                int2 _8 = make_int2(128, 128);
                _6.x = (_7.x%_8.x);
                _6.y = (_7.y%_8.y);
              int2 _9;
                int2 _10 = (make_int2)(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) % 896))+(1*0), ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) % 896))+(1*1));
                int2 _11 = make_int2(128, 128);
                _9.x = (_10.x/_11.x);
                _9.y = (_10.y/_11.y);
              int2 _12;
              ushort2 _13;
                ushort2 _14;
                  ushort2 _15;
                    int2 _16 = make_int2(128, 128);
                    int2 _17 = make_int2(0, 0);
                    _15.x = (_16.x>=_17.x);
                    _15.y = (_16.y>=_17.y);
                  ushort2 _18;
                    int2 _19 = make_int2(0, 0);
                    _18.x = (_6.x>=_19.x);
                    _18.y = (_6.y>=_19.y);
                  _14.x = (_15.x&&_18.x);
                  _14.y = (_15.y&&_18.y);
                ushort2 _20;
                  ushort2 _21;
                    int2 _22 = make_int2(128, 128);
                    int2 _23 = make_int2(0, 0);
                    _21.x = (_22.x<_23.x);
                    _21.y = (_22.y<_23.y);
                  ushort2 _24;
                    int2 _25 = make_int2(0, 0);
                    _24.x = (_6.x<=_25.x);
                    _24.y = (_6.y<=_25.y);
                  _20.x = (_21.x&&_24.x);
                  _20.y = (_21.y&&_24.y);
                _13.x = (_14.x||_20.x);
                _13.y = (_14.y||_20.y);
              int2 _26;
                int2 _27 = make_int2(1, 1);
                _26.x = (_9.x-_27.x);
                _26.y = (_9.y-_27.y);
              _12.x = (bool(_13.x)?_9.x:_26.x);
              _12.y = (bool(_13.y)?_9.y:_26.y);
              int2 _28 = make_int2(256, 256);
              _5.x = (_12.x*_28.x);
              _5.y = (_12.y*_28.y);
            _3.x = (_4.x+_5.x);
            _3.y = (_4.y+_5.y);
          int2 _29 = make_int2((rc_outer_outer * 128), (rc_outer_outer * 128));
          _2.x = (_3.x+_29.x);
          _2.y = (_3.y+_29.y);
        int2 _30 = (make_int2)(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) & 127))+(1*0), ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)) & 127))+(1*1));
        _1.x = (_2.x+_30.x);
        _1.y = (_2.y+_30.y);
      ((float2*)(PaddedInput_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) * 2)))))[0] = make_float2(placeholder[_1.x],placeholder[_1.y]);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 205; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 10) + ((int)threadIdx.x)) < 2048) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 10) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 8192) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 10) + ((int)threadIdx.x)) >> 4) * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 10) + ((int)threadIdx.x)) & 15)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          Conv2dOutput[(xx_outer_inner)] = (Conv2dOutput[(xx_outer_inner)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)))]));
          Conv2dOutput[((xx_outer_inner + 7))] = (Conv2dOutput[((xx_outer_inner + 7))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 2))]));
          Conv2dOutput[((xx_outer_inner + 14))] = (Conv2dOutput[((xx_outer_inner + 14))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 4))]));
          Conv2dOutput[((xx_outer_inner + 21))] = (Conv2dOutput[((xx_outer_inner + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 6))]));
          Conv2dOutput[((xx_outer_inner + 28))] = (Conv2dOutput[((xx_outer_inner + 28))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 8))]));
          Conv2dOutput[((xx_outer_inner + 35))] = (Conv2dOutput[((xx_outer_inner + 35))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 10))]));
          Conv2dOutput[((xx_outer_inner + 42))] = (Conv2dOutput[((xx_outer_inner + 42))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 12))]));
          Conv2dOutput[((xx_outer_inner + 49))] = (Conv2dOutput[((xx_outer_inner + 49))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 1) * 896) + (xx_outer_inner * 128)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 16)) + (((int)threadIdx.x) & 1)) + 14))]));
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
    T_relu[((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)))] = max((Conv2dOutput[(ax2_inner)] + placeholder2[((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 2))] = max((Conv2dOutput[((ax2_inner + 7))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 2))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 4))] = max((Conv2dOutput[((ax2_inner + 14))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 4))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 6))] = max((Conv2dOutput[((ax2_inner + 21))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 6))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 8))] = max((Conv2dOutput[((ax2_inner + 28))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 8))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 10))] = max((Conv2dOutput[((ax2_inner + 35))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 10))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 12))] = max((Conv2dOutput[((ax2_inner + 42))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 12))]), 0.000000e+00f);
    T_relu[(((((((((((int)blockIdx.x) / 20) * 11200) + ((((int)threadIdx.x) >> 1) * 2240)) + (((((int)blockIdx.x) % 20) >> 2) * 448)) + (ax2_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 1)) + 14))] = max((Conv2dOutput[((ax2_inner + 49))] + placeholder2[(((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 1)) + 14))]), 0.000000e+00f);
  }
}


