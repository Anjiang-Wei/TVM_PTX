
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[64];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
          for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
            Conv2dOutput[((((((nn_outer_inner_init * 16) + (nn_inner_init * 8)) + (yy_inner_init * 4)) + (xx_inner_init * 2)) + ff_outer_inner_init))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_outer_inner_init * 16) + (nn_inner_init * 8)) + (yy_inner_init * 4)) + (xx_inner_init * 2)) + ff_outer_inner_init) + 32))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_outer_inner_init * 16) + (nn_inner_init * 8)) + (yy_inner_init * 4)) + (xx_inner_init * 2)) + ff_outer_inner_init) + 64))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_outer_inner_init * 16) + (nn_inner_init * 8)) + (yy_inner_init * 4)) + (xx_inner_init * 2)) + ff_outer_inner_init) + 96))] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 512; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 512) {
        if (((int)threadIdx.x) < 128) {
          PaddedInput_shared[(((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 7) * 524288) + ((((int)blockIdx.x) >> 2) * 131072)) + (((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 127) >> 4) * 16384)) + (((((int)blockIdx.x) & 3) >> 1) * 8192)) + ((((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15) * 512)) + rc_outer_outer))];
        }
      }
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 128) + ((((int)blockIdx.x) & 1) * 64)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
        for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
              Conv2dOutput[((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner))] = (Conv2dOutput[((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner))] + (PaddedInput_shared[(((((((nn_outer_inner * 256) + (nn_inner * 128)) + ((((int)threadIdx.x) >> 7) * 32)) + (yy_inner * 16)) + (((((int)threadIdx.x) & 127) >> 4) * 2)) + xx_inner))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
              Conv2dOutput[(((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner) + 32))] = (Conv2dOutput[(((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner) + 32))] + (PaddedInput_shared[(((((((nn_outer_inner * 256) + (nn_inner * 128)) + ((((int)threadIdx.x) >> 7) * 32)) + (yy_inner * 16)) + (((((int)threadIdx.x) & 127) >> 4) * 2)) + xx_inner))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
              Conv2dOutput[(((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner) + 64))] = (Conv2dOutput[(((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner) + 64))] + (PaddedInput_shared[((((((((nn_outer_inner * 256) + (nn_inner * 128)) + ((((int)threadIdx.x) >> 7) * 32)) + (yy_inner * 16)) + (((((int)threadIdx.x) & 127) >> 4) * 2)) + xx_inner) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner))]));
              Conv2dOutput[(((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner) + 96))] = (Conv2dOutput[(((((((nn_outer_inner * 16) + (nn_inner * 8)) + (yy_inner * 4)) + (xx_inner * 2)) + ff_outer_inner) + 96))] + (PaddedInput_shared[((((((((nn_outer_inner * 256) + (nn_inner * 128)) + ((((int)threadIdx.x) >> 7) * 32)) + (yy_inner * 16)) + (((((int)threadIdx.x) & 127) >> 4) * 2)) + xx_inner) + 64))] * placeholder_shared[(((((((int)threadIdx.x) & 15) * 2) + ff_outer_inner) + 32))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
          T_relu[(((((((((((ax0_inner * 131072) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 7) * 8192)) + (ax1_inner * 4096)) + (((((int)blockIdx.x) & 3) >> 1) * 2048)) + (((((int)threadIdx.x) & 127) >> 4) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 8) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((((((ax0_inner * 131072) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 7) * 8192)) + (ax1_inner * 4096)) + (((((int)blockIdx.x) & 3) >> 1) * 2048)) + (((((int)threadIdx.x) & 127) >> 4) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[((((((ax0_inner * 8) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner) + 32))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
          T_relu[((((((((((((ax0_inner * 131072) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 7) * 8192)) + (ax1_inner * 4096)) + (((((int)blockIdx.x) & 3) >> 1) * 2048)) + (((((int)threadIdx.x) & 127) >> 4) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 16384))] = max((Conv2dOutput[((((((ax0_inner * 8) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner) + 64))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((((((ax0_inner * 131072) + ((((int)blockIdx.x) >> 2) * 32768)) + ((((int)threadIdx.x) >> 7) * 8192)) + (ax1_inner * 4096)) + (((((int)blockIdx.x) & 3) >> 1) * 2048)) + (((((int)threadIdx.x) & 127) >> 4) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 16416))] = max((Conv2dOutput[((((((ax0_inner * 8) + (ax1_inner * 4)) + (ax2_inner * 2)) + ax3_inner) + 96))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
        }
      }
    }
  }
}


