
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[960];
  __shared__ float PaddedInput_shared[10248];
  __shared__ float placeholder_shared[576];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 3; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 5; ++yy_inner_init) {
          for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
            Conv2dOutput[((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 120))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 240))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 360))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 480))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 600))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 720))] = 0.000000e+00f;
            Conv2dOutput[(((((((nn_inner_init * 60) + (yy_inner_init * 12)) + (xx_outer_inner_init * 4)) + (ff_outer_inner_init * 2)) + ff_inner_init) + 840))] = 0.000000e+00f;
          }
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 257; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) < 5124) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 10248) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)))] = (((1 <= (((((int)blockIdx.x) >> 2) * 20) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 1281) / 61))) && (1 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 61))) ? placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) / 1281) * 230400) + ((((int)blockIdx.x) >> 2) * 76800)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 1281) / 61) * 3840)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((int)threadIdx.x) >> 1)) % 61) * 64)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) - 3904))] : 0.000000e+00f);
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 15; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)) < 576) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)))] = placeholder1[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)) >> 6) * 8192) + (rc_outer_outer * 256)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)) & 63) >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)) & 31)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
          for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
            for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
              for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
                for (int yy_inner = 0; yy_inner < 5; ++yy_inner) {
                  for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
                    Conv2dOutput[((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner))] = (Conv2dOutput[((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner))] + (PaddedInput_shared[(((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner))] * placeholder_shared[(((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 120))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 120))] + (PaddedInput_shared[(((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 8))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 240))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 240))] + (PaddedInput_shared[(((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 16))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 360))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 360))] + (PaddedInput_shared[(((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 24))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 480))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 480))] + (PaddedInput_shared[((((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner) + 5124))] * placeholder_shared[(((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 600))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 600))] + (PaddedInput_shared[((((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner) + 5124))] * placeholder_shared[((((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 8))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 720))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 720))] + (PaddedInput_shared[((((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner) + 5124))] * placeholder_shared[((((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 16))]));
                    Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 840))] = (Conv2dOutput[(((((((nn_inner * 60) + (yy_inner * 12)) + (xx_outer_inner * 4)) + (ff_outer_inner * 2)) + ff_inner) + 840))] + (PaddedInput_shared[((((((((((nn_inner * 2562) + ((((int)threadIdx.x) / 20) * 1220)) + (yy_inner * 244)) + (ry_inner * 122)) + (((((int)threadIdx.x) % 20) >> 1) * 12)) + (xx_outer_inner * 4)) + (rx_inner * 2)) + rc_outer_inner) + 5124))] * placeholder_shared[((((((((ry_inner * 192) + (rx_inner * 64)) + (rc_outer_inner * 32)) + ((((int)threadIdx.x) & 1) * 4)) + (ff_outer_inner * 2)) + ff_inner) + 24))]));
                  }
                }
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
          T_relu[((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 8))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 120))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 8))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 16))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 240))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 24))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 360))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 24))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 230400))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 480))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 230408))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 600))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 8))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 230416))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 720))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
          T_relu[(((((((((((ax0_inner * 115200) + ((((int)blockIdx.x) >> 2) * 38400)) + ((((int)threadIdx.x) / 20) * 19200)) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 20) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 230424))] = max((Conv2dOutput[((((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner) + 840))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 1) * 4)) + ax3_inner) + 24))]), 0.000000e+00f);
        }
      }
    }
  }
}


