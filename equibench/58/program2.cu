
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[3072];
  __shared__ float placeholder_shared[9216];
  for (int xx_inner_init = 0; xx_inner_init < 4; ++xx_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 4; ++ff_inner_init) {
      Conv2dOutput[(((xx_inner_init * 4) + ff_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((xx_inner_init * 4) + ff_inner_init) + 16))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 6; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)))] = (((1 <= (((((int)blockIdx.x) >> 3) * 4) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) >> 9))) && ((((((int)blockIdx.x) >> 3) * 4) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) >> 9)) < 9)) ? placeholder[(((((((((int)blockIdx.x) >> 3) * 12288) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) >> 6) * 384)) + (rc_outer_outer * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 48) + ((int)threadIdx.x)) & 63)) - 3072))] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 192; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 48) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 >> 6) * 147456) + (rc_outer_outer * 24576)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 & 63) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      for (int rc_inner = 0; rc_inner < 64; ++rc_inner) {
        for (int xx_inner = 0; xx_inner < 4; ++xx_inner) {
          for (int ff_inner = 0; ff_inner < 4; ++ff_inner) {
            Conv2dOutput[(((xx_inner * 4) + ff_inner))] = (Conv2dOutput[(((xx_inner * 4) + ff_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 12) * 512) + (ry_inner * 512)) + (xx_inner * 64)) + rc_inner))] * placeholder_shared[(((((ry_inner * 3072) + (rc_inner * 48)) + ((((int)threadIdx.x) % 12) * 4)) + ff_inner))]));
            Conv2dOutput[((((xx_inner * 4) + ff_inner) + 16))] = (Conv2dOutput[((((xx_inner * 4) + ff_inner) + 16))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 512) + (ry_inner * 512)) + (xx_inner * 64)) + rc_inner) + 256))] * placeholder_shared[(((((ry_inner * 3072) + (rc_inner * 48)) + ((((int)threadIdx.x) % 12) * 4)) + ff_inner))]));
          }
        }
      }
    }
  }
  for (int ax2_inner = 0; ax2_inner < 4; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 3072)) + (ax2_inner * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) % 12) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 4) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) % 12) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) >> 3) * 12288) + ((((int)threadIdx.x) / 12) * 3072)) + (ax2_inner * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) % 12) * 4)) + ax3_inner) + 1536))] = max((Conv2dOutput[((((ax2_inner * 4) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) % 12) * 4)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


