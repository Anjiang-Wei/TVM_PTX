
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[2432];
  __shared__ float placeholder_shared[9728];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 19; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)))] = placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)) / 608) * 119168) + ((((int)blockIdx.x) / 14) * 17024)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)) % 608) / 304) * 8512)) + (((((int)blockIdx.x) % 14) >> 1) * 1216)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)) % 304) / 152) * 608)) + (rc_outer_outer * 152)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)) % 152)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 10; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9728) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((int)threadIdx.x) >> 3)) % 152) * 64) + ((((int)threadIdx.x) & 7) * 8)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((int)threadIdx.x) >> 3)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 8)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9727) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 1) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 1) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 1) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 1) & 63)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9726) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 2) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 2) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 2) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 2) & 63)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9725) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 3) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 3) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 3) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 3) & 63)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9724) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 4) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 4) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 4) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 4) & 63)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9723) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 5) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 5) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 5) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 5) & 63)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9722) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 6) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 6) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 6) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 6) & 63)))];
        }
      }
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 8)) < 9721) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1216) {
          placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 7) >> 6)) % 152) * 64) + (((((int)threadIdx.x) * 8) + 7) & 63)))] = placeholder1[(((((rc_outer_outer * 19456) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + (((((int)threadIdx.x) * 8) + 7) >> 6)) % 152) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 8) + 7) & 63)))];
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 152; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner) + 152))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner) + 152))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner) + 304))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner) + 304))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner) + 456))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 5) * 608) + rc_outer_inner) + 456))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 31) * 2)) + 1))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((((int)threadIdx.x) >> 5) * 25088) + ((((int)blockIdx.x) / 14) * 3584)) + (ax1_inner * 1792)) + (((((int)blockIdx.x) % 14) >> 1) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 31) * 2)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


