
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[150];
  __shared__ float PaddedInput_shared[600];
  __shared__ float placeholder_shared[2048];
  for (int yy_inner_init = 0; yy_inner_init < 5; ++yy_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
      Conv2dOutput[(((yy_inner_init * 3) + xx_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 15))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 30))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 45))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 60))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 75))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 90))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 105))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 120))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 3) + xx_inner_init) + 135))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 2)) < 150) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)) < 600) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 512) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 38400) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 2)) / 15) * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + (((int)threadIdx.x) >> 2)) % 15) * 128)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      ((float2*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024) + (((int)threadIdx.x) * 2)))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 2048) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 1024)) + (((int)threadIdx.x) * 2)))))[0];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 5; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
            Conv2dOutput[(((yy_inner * 3) + xx_inner))] = (Conv2dOutput[(((yy_inner * 3) + xx_inner))] + (PaddedInput_shared[(((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 15))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 15))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 12))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 30))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 30))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 24))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 45))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 45))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 36))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 60))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 60))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 48))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 75))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 75))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 90))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 90))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 312))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 105))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 105))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 324))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 120))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 120))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 336))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
            Conv2dOutput[((((yy_inner * 3) + xx_inner) + 135))] = (Conv2dOutput[((((yy_inner * 3) + xx_inner) + 135))] + (PaddedInput_shared[((((((yy_inner * 60) + (xx_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 348))] * placeholder_shared[((((rc_outer_inner * 1024) + (rc_inner * 512)) + ((int)threadIdx.x)))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      T_relu[(((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)))] = max(((Conv2dOutput[(((ax1_inner * 3) + ax2_inner))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[(((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 1536))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 15))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 1536))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 3072))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 30))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 3072))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 4608))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 45))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 4608))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 6144))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 60))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 6144))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 76800))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 75))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 76800))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 78336))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 90))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 78336))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 79872))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 105))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 79872))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 81408))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 120))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 81408))]), 0.000000e+00f);
      T_relu[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 82944))] = max(((Conv2dOutput[((((ax1_inner * 3) + ax2_inner) + 135))] + placeholder2[(((int)threadIdx.x))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 153600) + (ax1_inner * 15360)) + ((((int)blockIdx.x) & 1) * 7680)) + (ax2_inner * 512)) + ((int)threadIdx.x)) + 82944))]), 0.000000e+00f);
    }
  }
}


