
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2048];
  __shared__ float PaddedInput_shared[8192];
  __shared__ float placeholder_shared[1024];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 8; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
        for (int ff_inner_init = 0; ff_inner_init < 32; ++ff_inner_init) {
          Conv2dOutput[(((((nn_outer_inner_init * 512) + (yy_inner_init * 64)) + (xx_inner_init * 32)) + ff_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 512) + (yy_inner_init * 64)) + (xx_inner_init * 32)) + ff_inner_init) + 1024))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 44; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 512; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = placeholder[((((((((int)blockIdx.x) >> 1) * 360448) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 704)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 16) + ((int)threadIdx.x)) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 8; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
              for (int ff_inner = 0; ff_inner < 32; ++ff_inner) {
                Conv2dOutput[(((((nn_outer_inner * 512) + (yy_inner * 64)) + (xx_inner * 32)) + ff_inner))] = (Conv2dOutput[(((((nn_outer_inner * 512) + (yy_inner * 64)) + (xx_inner * 32)) + ff_inner))] + (PaddedInput_shared[((((((((nn_outer_inner * 4096) + ((((int)threadIdx.x) >> 3) * 2048)) + (yy_inner * 256)) + (((((int)threadIdx.x) & 7) >> 1) * 32)) + (xx_inner * 16)) + (rc_outer_inner * 8)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 512) + (rc_inner * 64)) + ((((int)threadIdx.x) & 1) * 32)) + ff_inner))]));
                Conv2dOutput[((((((nn_outer_inner * 512) + (yy_inner * 64)) + (xx_inner * 32)) + ff_inner) + 1024))] = (Conv2dOutput[((((((nn_outer_inner * 512) + (yy_inner * 64)) + (xx_inner * 32)) + ff_inner) + 1024))] + (PaddedInput_shared[(((((((((nn_outer_inner * 4096) + ((((int)threadIdx.x) >> 3) * 2048)) + (yy_inner * 256)) + (((((int)threadIdx.x) & 7) >> 1) * 32)) + (xx_inner * 16)) + (rc_outer_inner * 8)) + rc_inner) + 128))] * placeholder_shared[(((((rc_outer_inner * 512) + (rc_inner * 64)) + ((((int)threadIdx.x) & 1) * 32)) + ff_inner))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
          T_relu[(((((((((((((int)blockIdx.x) >> 1) * 65536) + (ax0_inner * 32768)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 7) >> 1) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 512) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 1) * 32)) + ax3_inner))]), 0.000000e+00f);
          T_relu[((((((((((((((int)blockIdx.x) >> 1) * 65536) + (ax0_inner * 32768)) + ((((int)threadIdx.x) >> 3) * 16384)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 7) >> 1) * 256)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 1) * 32)) + ax3_inner) + 1024))] = max((Conv2dOutput[((((((ax0_inner * 512) + (ax1_inner * 64)) + (ax2_inner * 32)) + ax3_inner) + 1024))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 1) * 32)) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


