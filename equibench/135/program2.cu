
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[28];
  __shared__ float placeholder_shared[16];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 448; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[((((int)threadIdx.x) * 28))] = placeholder[((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 1))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 2))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 6272))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 3))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 6273))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 4))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 12544))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 5))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 12545))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 6))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 18816))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 7))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 18817))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 8))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 25088))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 9))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 25089))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 10))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 31360))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 11))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 31361))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 12))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 37632))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 13))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 37633))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 14))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 43904))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 15))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 43905))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 16))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 50176))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 17))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 50177))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 18))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 56448))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 19))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 56449))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 20))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 62720))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 21))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 62721))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 22))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 68992))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 23))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 68993))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 24))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 75264))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 25))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 75265))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 26))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 81536))];
    }
    if (((int)threadIdx.x) < 1) {
      PaddedInput_shared[(((((int)threadIdx.x) * 28) + 27))] = placeholder[(((((((int)threadIdx.x) * 87808) + ((((int)blockIdx.x) >> 4) * 896)) + (rc_outer_outer * 2)) + 81537))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 256) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((int)threadIdx.x) & 7)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 2) * 2))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 2))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 2) * 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 2) + 1))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 2) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 2) + 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 2) * 2) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 2) + 9))]));
  }
  for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
    T_relu[((((((((int)threadIdx.x) >> 2) * 896) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]), 0.000000e+00f);
  }
}


