
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[1920];
  __shared__ float PaddedInput_shared[4096];
  __shared__ float placeholder_shared[240];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 4; ++yy_outer_inner_init) {
      for (int xx_outer_inner_init = 0; xx_outer_inner_init < 2; ++xx_outer_inner_init) {
        for (int ff_outer_inner_init = 0; ff_outer_inner_init < 15; ++ff_outer_inner_init) {
          Conv2dOutput[(((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 30))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 31))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 60))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 61))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 90))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 960) + (yy_outer_inner_init * 240)) + (xx_outer_inner_init * 120)) + (ff_outer_inner_init * 2)) + 91))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 5; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 512; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer >> 6) * 40960) + ((((int)blockIdx.x) >> 3) * 5120)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 63) >> 4) * 1280)) + (((((int)blockIdx.x) & 7) >> 2) * 640)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 15) * 40)) + (rc_outer_outer * 8)) + ((int)threadIdx.x)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 192) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 240) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) < 10) {
            placeholder_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 192) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[(((((rc_outer_outer * 960) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 192) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 30) * 120)) + ((((int)blockIdx.x) & 3) * 30)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 192) + (((int)threadIdx.x) * 24)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 30)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 4; ++yy_outer_inner) {
          for (int xx_outer_inner = 0; xx_outer_inner < 2; ++xx_outer_inner) {
            for (int ff_outer_inner = 0; ff_outer_inner < 15; ++ff_outer_inner) {
              Conv2dOutput[(((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)))] = (Conv2dOutput[(((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 30) + (ff_outer_inner * 2)))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 1))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 30) + (ff_outer_inner * 2)) + 1))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 30))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 30))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner) + 8))] * placeholder_shared[(((rc_outer_inner * 30) + (ff_outer_inner * 2)))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 31))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 31))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner) + 8))] * placeholder_shared[((((rc_outer_inner * 30) + (ff_outer_inner * 2)) + 1))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 60))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 60))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner) + 16))] * placeholder_shared[(((rc_outer_inner * 30) + (ff_outer_inner * 2)))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 61))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 61))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner) + 16))] * placeholder_shared[((((rc_outer_inner * 30) + (ff_outer_inner * 2)) + 1))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 90))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 90))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner) + 24))] * placeholder_shared[(((rc_outer_inner * 30) + (ff_outer_inner * 2)))]));
              Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 91))] = (Conv2dOutput[((((((nn_outer_inner * 960) + (yy_outer_inner * 240)) + (xx_outer_inner * 120)) + (ff_outer_inner * 2)) + 91))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) >> 1) * 1024) + (nn_outer_inner * 512)) + (yy_outer_inner * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (xx_outer_inner * 32)) + rc_outer_inner) + 24))] * placeholder_shared[((((rc_outer_inner * 30) + (ff_outer_inner * 2)) + 1))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 30; ++ax3_inner) {
          T_relu[(((((((((((((int)threadIdx.x) >> 1) * 245760) + (ax0_inner * 122880)) + ((((int)blockIdx.x) >> 3) * 15360)) + (ax1_inner * 3840)) + (((((int)blockIdx.x) & 7) >> 2) * 1920)) + ((((int)threadIdx.x) & 1) * 960)) + (ax2_inner * 120)) + ((((int)blockIdx.x) & 3) * 30)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 960) + (ax1_inner * 240)) + (ax2_inner * 30)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) & 3) * 30) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


