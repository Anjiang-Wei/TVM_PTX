
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[48];
  __shared__ float PaddedInput_shared[4680];
  __shared__ float placeholder_shared[6656];
  for (int yy_inner_init = 0; yy_inner_init < 3; ++yy_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
      Conv2dOutput[(((yy_inner_init * 2) + ff_inner_init))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 12))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 18))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 24))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 30))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 36))] = 0.000000e+00f;
      Conv2dOutput[((((yy_inner_init * 2) + ff_inner_init) + 42))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 20; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 240) + ((int)threadIdx.x)) < 4680) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 240) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) / 5) * 374400) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 240) + ((int)threadIdx.x)) / 156) * 12480)) + ((((int)blockIdx.x) % 5) * 2496)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 240) + ((int)threadIdx.x)) % 156) / 52) * 832)) + (rc_outer_outer * 52)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 240) + ((int)threadIdx.x)) % 52)))];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 28; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 240) + ((int)threadIdx.x)) < 6656) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 240) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 6656) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 240)) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 26; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 3; ++yy_inner) {
          for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
            Conv2dOutput[(((yy_inner * 2) + ff_inner))] = (Conv2dOutput[(((yy_inner * 2) + ff_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 6))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 6))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 16))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 12))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 12))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 32))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 18))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 18))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 48))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 24))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 24))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 64))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 30))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 30))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 80))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 36))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 36))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 96))]));
            Conv2dOutput[((((yy_inner * 2) + ff_inner) + 42))] = (Conv2dOutput[((((yy_inner * 2) + ff_inner) + 42))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 24) * 468) + (yy_inner * 156)) + (((((int)threadIdx.x) % 24) >> 3) * 52)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 256) + (rc_inner * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner) + 112))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 7) * 2) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 6))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 16))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 12))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 48))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 18))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 48))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 64))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 24))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 64))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 80))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 30))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 80))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 96))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 36))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 96))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 5) * 57600) + ((((int)threadIdx.x) / 24) * 5760)) + (ax1_inner * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (((((int)threadIdx.x) % 24) >> 3) * 128)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 112))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 42))] + placeholder2[(((((((int)threadIdx.x) & 7) * 2) + ax3_inner) + 112))]), 0.000000e+00f);
    }
  }
}


