
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_multiply, float* __restrict__ DepthwiseConv2d, float* __restrict__ placeholder) {
  T_multiply[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = ((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 184))]) * (max(min(((DepthwiseConv2d[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] + placeholder[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 184))]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f));
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ DepthwiseConv2d) {
  float DepthwiseConv2d_local[120];
  __shared__ float PaddedInput_shared[33120];
  __shared__ float placeholder_shared[552];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 3; ++j_c_outer_inner_init) {
    for (int c_c_outer_inner_init = 0; c_c_outer_inner_init < 2; ++c_c_outer_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 5; ++j_c_inner_init) {
        DepthwiseConv2d_local[((((j_c_outer_inner_init * 10) + (j_c_inner_init * 2)) + c_c_outer_inner_init))] = 0.000000e+00f;
        DepthwiseConv2d_local[(((((j_c_outer_inner_init * 10) + (j_c_inner_init * 2)) + c_c_outer_inner_init) + 30))] = 0.000000e+00f;
        DepthwiseConv2d_local[(((((j_c_outer_inner_init * 10) + (j_c_inner_init * 2)) + c_c_outer_inner_init) + 60))] = 0.000000e+00f;
        DepthwiseConv2d_local[(((((j_c_outer_inner_init * 10) + (j_c_inner_init * 2)) + c_c_outer_inner_init) + 90))] = 0.000000e+00f;
      }
    }
  }
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 360; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)))] = (((((1 <= (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)) % 8280) / 2760) + (((int)blockIdx.x) % 15))) && ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)) % 8280) / 2760) + (((int)blockIdx.x) % 15)) < 16)) && (1 <= (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)) % 2760) / 184) + dj_outer_outer))) && ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)) % 2760) / 184) + dj_outer_outer) < 16)) ? placeholder[((((((((((int)blockIdx.x) / 15) * 165600) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)) / 8280) * 41400)) + ((((int)blockIdx.x) % 15) * 2760)) + (dj_outer_outer * 184)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 92) + ((int)threadIdx.x)) % 8280)) - 2944))] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 92) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 92) + ((int)threadIdx.x)) / 184) * 552) + (dj_outer_outer * 184)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 92) + ((int)threadIdx.x)) % 184)))];
    }
    __syncthreads();
    for (int di_outer_inner = 0; di_outer_inner < 3; ++di_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 3; ++j_c_outer_inner) {
        for (int c_c_outer_inner = 0; c_c_outer_inner < 2; ++c_c_outer_inner) {
          for (int j_c_inner = 0; j_c_inner < 5; ++j_c_inner) {
            DepthwiseConv2d_local[((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner))] = (DepthwiseConv2d_local[((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 23) * 8280) + (di_outer_inner * 2760)) + (j_c_outer_inner * 920)) + (j_c_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner))] * placeholder_shared[((((di_outer_inner * 184) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner))]));
            DepthwiseConv2d_local[(((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner) + 30))] = (DepthwiseConv2d_local[(((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner) + 30))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 23) * 8280) + (di_outer_inner * 2760)) + (j_c_outer_inner * 920)) + (j_c_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner) + 46))] * placeholder_shared[(((((di_outer_inner * 184) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner) + 46))]));
            DepthwiseConv2d_local[(((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner) + 60))] = (DepthwiseConv2d_local[(((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner) + 60))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 23) * 8280) + (di_outer_inner * 2760)) + (j_c_outer_inner * 920)) + (j_c_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner) + 92))] * placeholder_shared[(((((di_outer_inner * 184) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner) + 92))]));
            DepthwiseConv2d_local[(((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner) + 90))] = (DepthwiseConv2d_local[(((((j_c_outer_inner * 10) + (j_c_inner * 2)) + c_c_outer_inner) + 90))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 23) * 8280) + (di_outer_inner * 2760)) + (j_c_outer_inner * 920)) + (j_c_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner) + 138))] * placeholder_shared[(((((di_outer_inner * 184) + ((((int)threadIdx.x) % 23) * 2)) + c_c_outer_inner) + 138))]));
          }
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 15; ++j_inner) {
    for (int c_inner = 0; c_inner < 2; ++c_inner) {
      DepthwiseConv2d[((((((((((int)blockIdx.x) / 15) * 165600) + ((((int)threadIdx.x) / 23) * 41400)) + ((((int)blockIdx.x) % 15) * 2760)) + (j_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_inner))] = DepthwiseConv2d_local[(((j_inner * 2) + c_inner))];
      DepthwiseConv2d[(((((((((((int)blockIdx.x) / 15) * 165600) + ((((int)threadIdx.x) / 23) * 41400)) + ((((int)blockIdx.x) % 15) * 2760)) + (j_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_inner) + 46))] = DepthwiseConv2d_local[((((j_inner * 2) + c_inner) + 30))];
      DepthwiseConv2d[(((((((((((int)blockIdx.x) / 15) * 165600) + ((((int)threadIdx.x) / 23) * 41400)) + ((((int)blockIdx.x) % 15) * 2760)) + (j_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_inner) + 92))] = DepthwiseConv2d_local[((((j_inner * 2) + c_inner) + 60))];
      DepthwiseConv2d[(((((((((((int)blockIdx.x) / 15) * 165600) + ((((int)threadIdx.x) / 23) * 41400)) + ((((int)blockIdx.x) % 15) * 2760)) + (j_inner * 184)) + ((((int)threadIdx.x) % 23) * 2)) + c_inner) + 138))] = DepthwiseConv2d_local[((((j_inner * 2) + c_inner) + 90))];
    }
  }
}


