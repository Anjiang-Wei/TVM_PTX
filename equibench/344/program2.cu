
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel2(float* __restrict__ T_softmax_norm, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_expsum) {
  T_softmax_norm[(((int)blockIdx.x))] = (__expf((placeholder[(((int)blockIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) >> 8))])) / T_softmax_expsum[((((int)blockIdx.x) >> 8))]);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder) {
  T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] = -3.402823e+38f;
  for (int k = 0; k < 256; ++k) {
    T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))] = max(T_softmax_maxelem[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)))], placeholder[((((((int)blockIdx.x) * 4096) + (((int)threadIdx.x) * 256)) + k))]);
  }
}

extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_softmax_expsum, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem) {
  T_softmax_expsum[(((int)blockIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 256; ++k) {
    T_softmax_expsum[(((int)blockIdx.x))] = (T_softmax_expsum[(((int)blockIdx.x))] + __expf((placeholder[(((((int)blockIdx.x) * 256) + k))] - T_softmax_maxelem[(((int)blockIdx.x))])));
  }
}


