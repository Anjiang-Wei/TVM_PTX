
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel2(float* __restrict__ T_softmax_norm, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_expsum) {
  T_softmax_norm[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = (__expf((placeholder[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] - T_softmax_maxelem[((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) >> 8))])) / T_softmax_expsum[((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) >> 8))]);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem) {
  float normal_reduce_temp0[1];
  __shared__ float red_buf0[4];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[((((((int)blockIdx.x) * 256) + (k_outer * 4)) + ((int)threadIdx.x)))]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = normal_reduce_temp0[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 2) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = max(((volatile float*)red_buf0)[(((int)threadIdx.x))], ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = max(((volatile float*)red_buf0)[(((int)threadIdx.x))], ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  T_softmax_maxelem[(((int)blockIdx.x))] = ((volatile float*)red_buf0)[(0)];
}

extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_softmax_expsum, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem) {
  T_softmax_expsum[(((int)blockIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 256; ++k) {
    T_softmax_expsum[(((int)blockIdx.x))] = (T_softmax_expsum[(((int)blockIdx.x))] + __expf((placeholder[(((((int)blockIdx.x) * 256) + k))] - T_softmax_maxelem[(((int)blockIdx.x))])));
  }
}


