
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[120];
  __shared__ float PaddedInput_shared[4800];
  __shared__ float placeholder_shared[2048];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int yy_outer_inner_init = 0; yy_outer_inner_init < 5; ++yy_outer_inner_init) {
      Conv2dOutput[(((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 4))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 5))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 6))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 7))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 8))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 9))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 10))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (yy_outer_inner_init * 12)) + 11))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 38400) + ((((int)threadIdx.x) / 240) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + (((((int)threadIdx.x) % 240) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 38400) + (((((int)threadIdx.x) + 320) / 240) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 5) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 38400) + (((((int)threadIdx.x) + 640) / 240) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 10) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 960))] = placeholder[((((((((((((int)threadIdx.x) + 960) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + (((((((int)threadIdx.x) >> 4) + 60) % 75) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + (((((int)threadIdx.x) % 240) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[((((((((((((int)threadIdx.x) + 1280) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 5) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 5) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1600))] = placeholder[((((((((((((int)threadIdx.x) + 1600) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 25) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 10) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[((((((((((((int)threadIdx.x) + 1920) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 45) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + (((((int)threadIdx.x) % 240) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2240))] = placeholder[((((((((((((int)threadIdx.x) + 2240) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + (((((((int)threadIdx.x) >> 4) + 65) % 75) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 5) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[((((((((((((int)threadIdx.x) + 2560) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 10) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 10) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2880))] = placeholder[((((((((((((int)threadIdx.x) + 2880) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 30) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + (((((int)threadIdx.x) % 240) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3200))] = placeholder[((((((((((((int)threadIdx.x) + 3200) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 50) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 5) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3520))] = placeholder[((((((((((((int)threadIdx.x) + 3520) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + (((((((int)threadIdx.x) >> 4) + 70) % 75) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 10) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3840))] = placeholder[((((((((((((int)threadIdx.x) + 3840) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 15) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + (((((int)threadIdx.x) % 240) >> 4) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4160))] = placeholder[((((((((((((int)threadIdx.x) + 4160) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 35) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 5) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4480))] = placeholder[((((((((((((int)threadIdx.x) + 4480) / 1200) * 230400) + ((((int)blockIdx.x) >> 1) * 38400)) + ((((((int)threadIdx.x) >> 4) + 55) / 15) * 7680)) + ((((int)blockIdx.x) & 1) * 3840)) + ((((((int)threadIdx.x) >> 4) + 10) % 15) * 256)) + (rc_outer_outer * 16)) + (((int)threadIdx.x) & 15)))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 2048) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 320))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 640))];
    placeholder_shared[((((int)threadIdx.x) + 960))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 960))];
    placeholder_shared[((((int)threadIdx.x) + 1280))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1280))];
    placeholder_shared[((((int)threadIdx.x) + 1600))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1600))];
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[((((int)threadIdx.x) + 1920))] = placeholder1[((((rc_outer_outer * 2048) + ((int)threadIdx.x)) + 1920))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 5; ++yy_outer_inner) {
          for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
            Conv2dOutput[((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)))] = (Conv2dOutput[((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)))]));
            Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 1))] = (Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 1))]));
            Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 2))] = (Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 2))]));
            Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 3))] = (Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 3))]));
            Conv2dOutput[((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)))] = (Conv2dOutput[((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 128))]));
            Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 1))] = (Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 1))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 129))]));
            Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 2))] = (Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 2))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 130))]));
            Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 3))] = (Conv2dOutput[(((((nn_outer_inner * 60) + (yy_outer_inner * 12)) + (xx_outer_inner * 4)) + 3))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 160) * 2400) + (nn_outer_inner * 1200)) + (yy_outer_inner * 240)) + (((((int)threadIdx.x) % 160) >> 5) * 48)) + (xx_outer_inner * 16)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 31) * 4)) + 131))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
          T_relu[(((((((((((((int)threadIdx.x) / 160) * 230400) + (ax0_inner * 115200)) + ((((int)blockIdx.x) >> 1) * 19200)) + (ax1_inner * 3840)) + ((((int)blockIdx.x) & 1) * 1920)) + (((((int)threadIdx.x) % 160) >> 5) * 384)) + (ax2_inner * 128)) + ((((int)threadIdx.x) & 31) * 4)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 60) + (ax1_inner * 12)) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 31) * 4) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


