
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[10];
  __shared__ float PaddedInput_shared[7500];
  __shared__ float placeholder_shared[100];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 25))] = placeholder[((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 1))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 2))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 2))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 3))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 3))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 4))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 4))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 5))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 5))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 6))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 6))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 7))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 7))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 8))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 8))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 9))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 9))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 10))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 10))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 11))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 11))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 12))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 12))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 13))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 13))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 14))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 14))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 15))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 15))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 16))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 16))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 17))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 17))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 18))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 18))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 19))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 19))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 20))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 20))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 21))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 21))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 22))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 22))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 23))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 23))];
    PaddedInput_shared[(((((int)threadIdx.x) * 25) + 24))] = placeholder[(((((((((int)threadIdx.x) / 5) * 3000) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 24))];
    PaddedInput_shared[((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)))] = placeholder[((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 1))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 1))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 2))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 2))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 3))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 3))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 4))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 4))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 5))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 5))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 6))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 6))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 7))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 7))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 8))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 8))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 9))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 9))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 10))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 10))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 11))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 11))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 12))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 12))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 13))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 13))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 14))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 14))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 15))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 15))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 16))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 16))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 17))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 17))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 18))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 18))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 19))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 19))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 20))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 20))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 21))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 21))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 22))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 22))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 23))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 23))];
    PaddedInput_shared[(((((((((int)threadIdx.x) + 120) / 75) * 1875) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 24))] = placeholder[(((((((((((int)threadIdx.x) + 120) / 75) * 45000) + ((((((int)threadIdx.x) / 5) + 9) % 15) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 24))];
    if (((int)threadIdx.x) < 60) {
      PaddedInput_shared[((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)))] = placeholder[((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 1))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 1))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 2))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 2))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 3))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 3))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 4))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 4))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 5))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 5))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 6))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 6))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 7))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 7))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 8))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 8))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 9))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 9))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 10))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 10))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 11))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 11))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 12))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 12))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 13))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 13))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 14))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 14))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 15))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 15))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 16))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 16))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 17))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 17))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 18))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 18))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 19))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 19))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 20))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 20))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 21))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 21))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 22))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 22))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 23))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 23))];
      PaddedInput_shared[(((((((((int)threadIdx.x) + 240) / 75) * 1875) + (((((int)threadIdx.x) / 5) + 3) * 125)) + ((((int)threadIdx.x) % 5) * 25)) + 24))] = placeholder[(((((((((((int)threadIdx.x) + 240) / 75) * 45000) + (((((int)threadIdx.x) / 5) + 3) * 3000)) + ((((int)blockIdx.x) / 20) * 1000)) + ((((int)threadIdx.x) % 5) * 200)) + (rc_outer_outer * 25)) + 24))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[((((int)threadIdx.x) * 25))] = placeholder1[(((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 1))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 1) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 1) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 2))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 2) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 2) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 3))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 3) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 3) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 4))] = placeholder1[((((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)) + 80))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 5))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 5) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 1) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 6))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 6) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 2) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 7))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 7) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 3) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 8))] = placeholder1[((((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)) + 160))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 9))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 9) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 1) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 10))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 10) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 2) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 11))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 11) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 3) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 12))] = placeholder1[((((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)) + 240))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 13))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 13) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 1) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 14))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 14) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 2) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 15))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 15) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 3) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 16))] = placeholder1[((((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)) + 320))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 17))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 17) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 1) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 18))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 18) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 2) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 19))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 19) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 3) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 20))] = placeholder1[((((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)) + 400))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 21))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 21) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 1) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 22))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 22) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 2) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 23))] = placeholder1[(((((rc_outer_outer * 2000) + ((((((int)threadIdx.x) * 25) + 23) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + (((((int)threadIdx.x) * 25) + 3) & 3)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 25) + 24))] = placeholder1[((((((rc_outer_outer * 2000) + (((((int)threadIdx.x) * 25) >> 2) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) * 25) & 3)) + 480))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 2))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 1))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 4))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 5))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 9))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 12))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 13))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 16))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 17))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 125))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 125))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 126))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 4))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 126))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 5))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 127))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 8))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 127))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 9))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 12))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 13))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 16))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 17))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 250))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 2))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 250))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 1))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 251))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 4))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 251))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 5))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 252))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 8))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 252))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 9))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 253))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 12))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 253))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 13))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 254))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 16))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 254))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 17))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 375))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 2))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 375))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 1))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 376))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 4))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 376))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 5))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 377))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 8))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 377))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 9))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 378))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 12))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 378))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 13))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 379))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 16))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 379))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 17))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 500))] * placeholder_shared[(((((int)threadIdx.x) & 1) * 2))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 500))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 1))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 501))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 4))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 501))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 5))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 502))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 8))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 502))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 9))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 503))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 12))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 503))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 13))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 504))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 16))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 504))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 17))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 20))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 21))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 24))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 25))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 28))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 29))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 32))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 8))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 33))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 36))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 9))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 37))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 20))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 21))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 24))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 25))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 28))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 29))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 32))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 33))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 36))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 37))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 255))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 20))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 255))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 21))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 256))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 24))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 256))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 25))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 257))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 28))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 257))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 29))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 258))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 32))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 258))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 33))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 259))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 36))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 259))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 37))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 380))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 20))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 380))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 21))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 381))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 24))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 381))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 25))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 382))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 28))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 382))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 29))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 383))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 32))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 383))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 33))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 384))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 36))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 384))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 37))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 505))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 20))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 505))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 21))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 506))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 24))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 506))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 25))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 507))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 28))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 507))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 29))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 508))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 32))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 508))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 33))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 509))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 36))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 509))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 37))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 40))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 10))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 41))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 44))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 11))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 45))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 48))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 12))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 49))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 52))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 13))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 53))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 56))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 14))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 57))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 40))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 41))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 136))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 44))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 136))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 45))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 137))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 48))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 137))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 49))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 138))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 52))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 138))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 53))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 139))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 56))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 139))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 57))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 260))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 40))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 260))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 41))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 261))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 44))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 261))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 45))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 262))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 48))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 262))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 49))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 263))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 52))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 263))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 53))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 264))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 56))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 264))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 57))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 385))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 40))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 385))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 41))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 386))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 44))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 386))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 45))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 387))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 48))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 387))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 49))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 388))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 52))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 388))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 53))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 389))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 56))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 389))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 57))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 510))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 40))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 510))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 41))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 511))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 44))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 511))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 45))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 512))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 48))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 512))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 49))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 513))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 52))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 513))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 53))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 514))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 56))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 514))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 57))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 60))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 15))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 61))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 64))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 65))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 68))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 17))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 69))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 72))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 18))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 73))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 76))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 19))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 77))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 140))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 60))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 140))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 61))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 141))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 64))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 141))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 65))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 142))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 68))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 142))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 69))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 143))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 72))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 143))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 73))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 144))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 76))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 144))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 77))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 265))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 60))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 265))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 61))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 266))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 64))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 266))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 65))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 267))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 68))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 267))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 69))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 268))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 72))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 268))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 73))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 269))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 76))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 269))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 77))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 390))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 60))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 390))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 61))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 391))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 64))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 391))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 65))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 392))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 68))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 392))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 69))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 393))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 72))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 393))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 73))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 394))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 76))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 394))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 77))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 515))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 60))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 515))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 61))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 516))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 64))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 516))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 65))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 517))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 68))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 517))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 69))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 518))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 72))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 518))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 73))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 519))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 76))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 519))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 77))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 20))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 80))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 20))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 81))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 84))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 21))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 85))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 88))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 22))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 89))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 92))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 23))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 93))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 24))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 96))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 24))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 97))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 145))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 80))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 145))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 81))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 146))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 84))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 146))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 85))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 147))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 88))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 147))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 89))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 148))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 92))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 148))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 93))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 149))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 96))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 149))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 97))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 270))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 80))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 270))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 81))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 271))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 84))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 271))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 85))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 272))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 88))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 272))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 89))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 273))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 92))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 273))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 93))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 274))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 96))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 274))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 97))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 395))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 80))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 395))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 81))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 396))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 84))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 396))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 85))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 397))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 88))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 397))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 89))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 398))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 92))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 398))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 93))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 399))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 96))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 399))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 97))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 520))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 80))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 520))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 81))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 521))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 84))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 521))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 85))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 522))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 88))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 522))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 89))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 523))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 92))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 523))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 93))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 524))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 96))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 10) * 625) + (((((int)threadIdx.x) % 10) >> 1) * 25)) + 524))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 2) + 97))]));
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_add[(((((((((((int)threadIdx.x) / 10) * 6000) + (ax1_inner * 1200)) + ((((int)blockIdx.x) / 20) * 400)) + (((((int)threadIdx.x) % 10) >> 1) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))] = ((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) % 20) * 4) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))]) + placeholder3[(((((((((((int)threadIdx.x) / 10) * 6000) + (ax1_inner * 1200)) + ((((int)blockIdx.x) / 20) * 400)) + (((((int)threadIdx.x) % 10) >> 1) * 80)) + ((((int)blockIdx.x) % 20) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax3_inner))]);
    }
  }
}


