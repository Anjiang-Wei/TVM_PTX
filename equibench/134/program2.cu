
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[28];
  __shared__ float PaddedInput_shared[2156];
  __shared__ float placeholder_shared[352];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 7; ++xx_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 14) + (xx_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 39; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 2156) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 22) * 704) + (rc_outer_outer * 22)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 22)))];
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 112) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 352) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 176) {
            placeholder_shared[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 4)) % 22) * 16) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15)))] = placeholder1[(((((rc_outer_outer * 2816) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 7) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 4)) % 22) * 128)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 11; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int xx_inner = 0; xx_inner < 7; ++xx_inner) {
            for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
              Conv2dOutput[((((nn_outer_inner * 14) + (xx_inner * 2)) + ff_inner))] = (Conv2dOutput[((((nn_outer_inner * 14) + (xx_inner * 2)) + ff_inner))] + (PaddedInput_shared[((((((nn_outer_inner * 1078) + ((((int)threadIdx.x) >> 3) * 154)) + (xx_inner * 22)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 32) + (rc_inner * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ff_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((ax0_inner * 6272) + ((((int)threadIdx.x) >> 3) * 896)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 14) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


