
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[1200];
  __shared__ float placeholder_shared[36];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 64) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 1200) {
        if (((int)threadIdx.x) < 19) {
          PaddedInput_shared[(((((int)threadIdx.x) * 64) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder[((((((((((int)blockIdx.x) / 24) * 115200) + ((((((int)threadIdx.x) * 64) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 60) * 5760)) + (((((int)blockIdx.x) % 24) >> 2) * 960)) + (((((((int)threadIdx.x) * 64) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 60) / 6) * 96)) + (rc_outer_outer * 6)) + (((((int)threadIdx.x) * 64) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 6)))];
        }
      }
    }
    if (((int)threadIdx.x) < 36) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 144) + ((((int)threadIdx.x) / 6) * 24)) + ((((int)blockIdx.x) & 3) * 6)) + (((int)threadIdx.x) % 6)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
        Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) / 6) * 6) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 12) + (rc_inner * 6)) + (((int)threadIdx.x) % 6)))]));
        Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 6) * 6) + (rc_outer_inner * 2)) + rc_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 12) + (rc_inner * 6)) + (((int)threadIdx.x) % 6)))]));
      }
    }
  }
  T_add[((((((((((int)blockIdx.x) / 24) * 28800) + ((((int)threadIdx.x) / 60) * 1440)) + (((((int)blockIdx.x) % 24) >> 2) * 240)) + (((((int)threadIdx.x) % 60) / 6) * 24)) + ((((int)blockIdx.x) & 3) * 6)) + (((int)threadIdx.x) % 6)))] = (Conv2dOutput[(0)] + placeholder2[((((((int)blockIdx.x) & 3) * 6) + (((int)threadIdx.x) % 6)))]);
  T_add[(((((((((((int)blockIdx.x) / 24) * 28800) + ((((int)threadIdx.x) / 60) * 1440)) + (((((int)blockIdx.x) % 24) >> 2) * 240)) + (((((int)threadIdx.x) % 60) / 6) * 24)) + ((((int)blockIdx.x) & 3) * 6)) + (((int)threadIdx.x) % 6)) + 14400))] = (Conv2dOutput[(1)] + placeholder2[((((((int)blockIdx.x) & 3) * 6) + (((int)threadIdx.x) % 6)))]);
}


