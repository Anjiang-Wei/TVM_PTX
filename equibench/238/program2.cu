
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[256];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
    Conv2dOutput[((ff_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((ff_outer_inner_init * 2) + 25))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    ((float4*)(PaddedInput_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder + (((((((((int)blockIdx.x) / 14) * 28672) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((int)threadIdx.x) >> 1) * 256)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)))))[0];
    ((float4*)(PaddedInput_shared + (((((int)threadIdx.x) * 4) + 64))))[0] = ((float4*)(placeholder + ((((((((((int)blockIdx.x) / 14) * 28672) + (((((int)blockIdx.x) % 14) >> 1) * 2048)) + ((((int)threadIdx.x) >> 1) * 256)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 14336))))[0];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 16))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 16))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 64))];
    placeholder_shared[((((int)threadIdx.x) + 48))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 48) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 80) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 192))];
    placeholder_shared[((((int)threadIdx.x) + 112))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 112) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 144))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 144) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 320))];
    placeholder_shared[((((int)threadIdx.x) + 176))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 176) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 208))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 208) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)blockIdx.x) & 1) * 32)) + ((int)threadIdx.x)) + 448))];
    placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[(((((rc_outer_outer * 512) + (((((int)threadIdx.x) + 240) >> 5) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + (((int)threadIdx.x) + 16)))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
        Conv2dOutput[((ff_outer_inner * 2))] = (Conv2dOutput[((ff_outer_inner * 2))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 16))] = (Conv2dOutput[(((ff_outer_inner * 2) + 16))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 16))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 1))] = (Conv2dOutput[(((ff_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 17))] = (Conv2dOutput[(((ff_outer_inner * 2) + 17))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 17))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 8))] = (Conv2dOutput[(((ff_outer_inner * 2) + 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 24))] = (Conv2dOutput[(((ff_outer_inner * 2) + 24))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 16))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 9))] = (Conv2dOutput[(((ff_outer_inner * 2) + 9))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 1))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 25))] = (Conv2dOutput[(((ff_outer_inner * 2) + 25))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 17))]));
        Conv2dOutput[((ff_outer_inner * 2))] = (Conv2dOutput[((ff_outer_inner * 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 32))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 16))] = (Conv2dOutput[(((ff_outer_inner * 2) + 16))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 48))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 1))] = (Conv2dOutput[(((ff_outer_inner * 2) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 33))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 17))] = (Conv2dOutput[(((ff_outer_inner * 2) + 17))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 49))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 8))] = (Conv2dOutput[(((ff_outer_inner * 2) + 8))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 32))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 24))] = (Conv2dOutput[(((ff_outer_inner * 2) + 24))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 48))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 9))] = (Conv2dOutput[(((ff_outer_inner * 2) + 9))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 33))]));
        Conv2dOutput[(((ff_outer_inner * 2) + 25))] = (Conv2dOutput[(((ff_outer_inner * 2) + 25))] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 1) * 8) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[(((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 1) * 8)) + (ff_outer_inner * 2)) + 49))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) / 14) * 7168) + (ax1_inner * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + ((((int)threadIdx.x) >> 1) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 8) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 32) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) / 14) * 7168) + (ax1_inner * 3584)) + (((((int)blockIdx.x) % 14) >> 1) * 512)) + ((((int)threadIdx.x) >> 1) * 64)) + ((((int)blockIdx.x) & 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax1_inner * 8) + ax3_inner) + 16))] + placeholder2[((((((((int)blockIdx.x) & 1) * 32) + ((((int)threadIdx.x) & 1) * 8)) + ax3_inner) + 16))]), 0.000000e+00f);
    }
  }
}


