
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[6272];
  __shared__ float PaddedInput_shared[392];
  __shared__ float placeholder_shared[64];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
      for (int nn_inner_init = 0; nn_inner_init < 2; ++nn_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 14; ++xx_inner_init) {
          Conv2dOutput[(((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 1))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 2))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 3))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 4))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 5))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 6))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 7))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 8))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 9))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 10))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 11))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 12))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 13))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 14))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_inner_init * 3136) + (yy_outer_inner_init * 448)) + (xx_inner_init * 32)) + (ff_outer_inner_init * 16)) + 15))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 192; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 392; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer)] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 196) * 75264) + ((((int)blockIdx.x) >> 2) * 37632)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 196) >> 1) * 384)) + (rc_outer_outer * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer & 1)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 64; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1)] = placeholder1[(((((rc_outer_outer * 256) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 >> 5) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 & 31)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
          for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
            for (int xx_inner = 0; xx_inner < 14; ++xx_inner) {
              Conv2dOutput[(((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)))] = (Conv2dOutput[(((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 32) + (ff_outer_inner * 16)))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 1))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 1))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 1))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 2))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 2))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 2))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 3))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 3))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 3))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 4))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 4))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 4))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 5))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 5))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 5))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 6))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 6))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 6))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 7))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 7))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 7))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 8))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 8))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 8))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 9))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 9))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 9))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 10))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 10))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 10))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 11))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 11))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 11))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 12))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 12))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 12))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 13))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 13))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 13))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 14))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 14))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 14))]));
              Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 15))] = (Conv2dOutput[((((((nn_inner * 3136) + (yy_outer_inner * 448)) + (xx_inner * 32)) + (ff_outer_inner * 16)) + 15))] + (PaddedInput_shared[(((((nn_inner * 196) + (yy_outer_inner * 28)) + (xx_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 32) + (ff_outer_inner * 16)) + 15))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 32; ++ax3_inner) {
          T_relu[(((((((ax0_inner * 25088) + ((((int)blockIdx.x) >> 2) * 12544)) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ax3_inner))] = max((Conv2dOutput[(((((ax0_inner * 3136) + (ax1_inner * 448)) + (ax2_inner * 32)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) & 3) * 32) + ax3_inner))]), 0.000000e+00f);
        }
      }
    }
  }
}


