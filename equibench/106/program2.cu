
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[24];
  __shared__ float PaddedInput_shared[18];
  __shared__ float placeholder_shared[128];
  for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
    Conv2dOutput[(xx_inner_init)] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 3))] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 6))] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 9))] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 12))] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 15))] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 18))] = 0.000000e+00f;
    Conv2dOutput[((xx_inner_init + 21))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 736; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)threadIdx.x) / 9) * 165600) + ((((int)blockIdx.x) / 5) * 33120)) + (((((int)threadIdx.x) % 9) / 3) * 11040)) + ((((int)blockIdx.x) % 5) * 2208)) + ((((int)threadIdx.x) % 3) * 736)) + rc_outer_outer))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 128) {
        if (((int)threadIdx.x) < 64) {
          placeholder_shared[(((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = placeholder1[((((rc_outer_outer * 128) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))];
        }
      }
    }
    __syncthreads();
    for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
      Conv2dOutput[(xx_inner)] = (Conv2dOutput[(xx_inner)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[((((int)threadIdx.x) & 15))]));
      Conv2dOutput[((xx_inner + 3))] = (Conv2dOutput[((xx_inner + 3))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 16))]));
      Conv2dOutput[((xx_inner + 6))] = (Conv2dOutput[((xx_inner + 6))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 32))]));
      Conv2dOutput[((xx_inner + 9))] = (Conv2dOutput[((xx_inner + 9))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 48))]));
      Conv2dOutput[((xx_inner + 12))] = (Conv2dOutput[((xx_inner + 12))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 64))]));
      Conv2dOutput[((xx_inner + 15))] = (Conv2dOutput[((xx_inner + 15))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 80))]));
      Conv2dOutput[((xx_inner + 18))] = (Conv2dOutput[((xx_inner + 18))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 96))]));
      Conv2dOutput[((xx_inner + 21))] = (Conv2dOutput[((xx_inner + 21))] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 3) + xx_inner))] * placeholder_shared[(((((int)threadIdx.x) & 15) + 112))]));
    }
  }
  for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
    T_relu[((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)))] = max((Conv2dOutput[(ax2_inner)] + placeholder2[((((int)threadIdx.x) & 15))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 16))] = max((Conv2dOutput[((ax2_inner + 3))] + placeholder2[(((((int)threadIdx.x) & 15) + 16))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 32))] = max((Conv2dOutput[((ax2_inner + 6))] + placeholder2[(((((int)threadIdx.x) & 15) + 32))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 48))] = max((Conv2dOutput[((ax2_inner + 9))] + placeholder2[(((((int)threadIdx.x) & 15) + 48))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 64))] = max((Conv2dOutput[((ax2_inner + 12))] + placeholder2[(((((int)threadIdx.x) & 15) + 64))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 80))] = max((Conv2dOutput[((ax2_inner + 15))] + placeholder2[(((((int)threadIdx.x) & 15) + 80))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 96))] = max((Conv2dOutput[((ax2_inner + 18))] + placeholder2[(((((int)threadIdx.x) & 15) + 96))]), 0.000000e+00f);
    T_relu[(((((((((((int)threadIdx.x) / 48) * 28800) + ((((int)blockIdx.x) / 5) * 5760)) + (((((int)threadIdx.x) % 48) >> 4) * 1920)) + ((((int)blockIdx.x) % 5) * 384)) + (ax2_inner * 128)) + (((int)threadIdx.x) & 15)) + 112))] = max((Conv2dOutput[((ax2_inner + 21))] + placeholder2[(((((int)threadIdx.x) & 15) + 112))]), 0.000000e+00f);
  }
}


