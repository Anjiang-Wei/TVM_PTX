
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[240];
  __shared__ float PaddedInput_shared[450];
  __shared__ float placeholder_shared[32];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 15; ++xx_outer_inner_init) {
      Conv2dOutput[(((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 120))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 121))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 122))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 60) + (xx_outer_inner_init * 4)) + 123))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 736; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((int)threadIdx.x) * 736) + rc_outer_outer))];
    PaddedInput_shared[((((int)threadIdx.x) + 60))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 44160))];
    PaddedInput_shared[((((int)threadIdx.x) + 120))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 88320))];
    PaddedInput_shared[((((int)threadIdx.x) + 180))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 132480))];
    PaddedInput_shared[((((int)threadIdx.x) + 240))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 176640))];
    PaddedInput_shared[((((int)threadIdx.x) + 300))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 220800))];
    PaddedInput_shared[((((int)threadIdx.x) + 360))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 264960))];
    if (((int)threadIdx.x) < 30) {
      PaddedInput_shared[((((int)threadIdx.x) + 420))] = placeholder[((((((int)threadIdx.x) * 736) + rc_outer_outer) + 309120))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((rc_outer_outer * 128) + (((int)blockIdx.x) * 32)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 15; ++xx_outer_inner) {
        Conv2dOutput[(((nn_outer_inner * 60) + (xx_outer_inner * 4)))] = (Conv2dOutput[(((nn_outer_inner * 60) + (xx_outer_inner * 4)))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 120))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 120))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 1))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 1))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 121))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 121))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 2))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 2))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 122))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 122))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 3))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 3))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
        Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 123))] = (Conv2dOutput[((((nn_outer_inner * 60) + (xx_outer_inner * 4)) + 123))] + (PaddedInput_shared[((((nn_outer_inner * 225) + ((((int)threadIdx.x) >> 2) * 15)) + xx_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[(((((((ax0_inner * 28800) + ((((int)threadIdx.x) >> 2) * 1920)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 60) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((ax0_inner * 28800) + ((((int)threadIdx.x) >> 2) * 1920)) + (ax2_inner * 128)) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))] = max((Conv2dOutput[(((((ax0_inner * 60) + (ax2_inner * 4)) + ax3_inner) + 120))] + placeholder2[(((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
      }
    }
  }
}


