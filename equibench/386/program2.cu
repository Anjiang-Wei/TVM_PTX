
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[8];
  __shared__ float PaddedInput_shared[384];
  __shared__ float placeholder_shared[96];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 80; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 10))] = placeholder[((((((((((int)threadIdx.x) * 10) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + ((((((int)threadIdx.x) * 10) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + ((((int)threadIdx.x) * 10) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 1))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 1) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 1) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 1) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 2))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 2) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 2) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 2) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 3))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 3) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 3) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 3) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 4))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 4) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 4) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 4) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 5))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 5) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 5) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 5) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 6))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 6) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 6) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 6) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 7))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 7) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 7) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 7) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 8))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 8) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 8) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 8) % 12)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 10) + 9))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 9) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 9) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 9) % 12)))];
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 320))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 320) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 32) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 8) % 12)))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 321))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 321) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 33) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 9) % 12)))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 322))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 322) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 34) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 10) % 12)))];
    }
    if (((int)threadIdx.x) < 7) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 323))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 323) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 35) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 11) % 12)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 324))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 324) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 36) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + ((((int)threadIdx.x) * 10) % 12)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 325))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 325) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 37) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 1) % 12)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 326))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 326) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 38) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 2) % 12)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 327))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 327) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 39) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 3) % 12)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 328))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 328) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 40) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 4) % 12)))];
    }
    if (((int)threadIdx.x) < 6) {
      PaddedInput_shared[(((((int)threadIdx.x) * 10) + 329))] = placeholder[(((((((((((int)threadIdx.x) * 10) + 329) / 48) * 7680) + ((((int)blockIdx.x) / 20) * 3840)) + (((((((int)threadIdx.x) * 10) + 41) % 48) / 12) * 960)) + (rc_outer_outer * 12)) + (((((int)threadIdx.x) * 10) + 5) % 12)))];
    }
    if (((int)threadIdx.x) < 20) {
      placeholder_shared[((((int)threadIdx.x) * 5))] = placeholder1[(((((rc_outer_outer * 1920) + (((((int)threadIdx.x) * 5) >> 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + ((((int)threadIdx.x) * 5) & 7)))];
    }
    if (((int)threadIdx.x) < 19) {
      placeholder_shared[(((((int)threadIdx.x) * 5) + 1))] = placeholder1[(((((rc_outer_outer * 1920) + ((((((int)threadIdx.x) * 5) + 1) >> 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + (((((int)threadIdx.x) * 5) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 19) {
      placeholder_shared[(((((int)threadIdx.x) * 5) + 2))] = placeholder1[(((((rc_outer_outer * 1920) + ((((((int)threadIdx.x) * 5) + 2) >> 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + (((((int)threadIdx.x) * 5) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 19) {
      placeholder_shared[(((((int)threadIdx.x) * 5) + 3))] = placeholder1[(((((rc_outer_outer * 1920) + ((((((int)threadIdx.x) * 5) + 3) >> 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + (((((int)threadIdx.x) * 5) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 19) {
      placeholder_shared[(((((int)threadIdx.x) * 5) + 4))] = placeholder1[(((((rc_outer_outer * 1920) + ((((((int)threadIdx.x) * 5) + 4) >> 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + (((((int)threadIdx.x) * 5) + 4) & 7)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(0)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(1)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(2)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(3)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(4)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(5)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(6)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((int)threadIdx.x) * 12))] * placeholder_shared[(7)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(8)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(9)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(10)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(11)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(12)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(13)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(14)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 1))] * placeholder_shared[(15)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(16)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(17)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(18)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(19)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(20)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(21)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(22)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 2))] * placeholder_shared[(23)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(24)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(25)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(26)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(27)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(28)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(29)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(30)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 3))] * placeholder_shared[(31)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(32)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(33)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(34)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(35)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(36)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(37)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(38)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 4))] * placeholder_shared[(39)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(40)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(41)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(42)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(43)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(44)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(45)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(46)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 5))] * placeholder_shared[(47)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(48)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(49)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(50)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(51)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(52)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(53)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(54)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 6))] * placeholder_shared[(55)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(56)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(57)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(58)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(59)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(60)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(61)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(62)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 7))] * placeholder_shared[(63)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(64)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(65)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(66)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(67)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(68)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(69)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(70)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 8))] * placeholder_shared[(71)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(72)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(73)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(74)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(75)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(76)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(77)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(78)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 9))] * placeholder_shared[(79)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(80)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(81)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(82)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(83)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(84)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(85)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(86)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 10))] * placeholder_shared[(87)]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(88)]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(89)]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(90)]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(91)]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(92)]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(93)]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(94)]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((int)threadIdx.x) * 12) + 11))] * placeholder_shared[(95)]));
  }
  for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
    T_add[(((((((((int)threadIdx.x) >> 2) * 1280) + ((((int)blockIdx.x) / 20) * 640)) + ((((int)threadIdx.x) & 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + ax3_inner))] = (placeholder2[(((((((((int)threadIdx.x) >> 2) * 1280) + ((((int)blockIdx.x) / 20) * 640)) + ((((int)threadIdx.x) & 3) * 160)) + ((((int)blockIdx.x) % 20) * 8)) + ax3_inner))] + (Conv2dOutput[(ax3_inner)] + placeholder3[((((((int)blockIdx.x) % 20) * 8) + ax3_inner))]));
  }
}


