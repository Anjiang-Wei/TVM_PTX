
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[15];
  __shared__ float placeholder_shared[120];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((rc_outer_outer * 15) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 4))] = placeholder[((((rc_outer_outer * 15) + ((int)threadIdx.x)) + 4))];
    PaddedInput_shared[((((int)threadIdx.x) + 8))] = placeholder[((((rc_outer_outer * 15) + ((int)threadIdx.x)) + 8))];
    if (((int)threadIdx.x) < 3) {
      PaddedInput_shared[((((int)threadIdx.x) + 12))] = placeholder[((((rc_outer_outer * 15) + ((int)threadIdx.x)) + 12))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[((((int)threadIdx.x) * 60))] = placeholder1[(((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 1))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 1) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 2))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 2) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 3))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 3) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 4))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 4) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 5))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 5) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 6))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 6) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 7))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 7) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 8))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 32))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 9))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 9) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 10))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 10) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 11))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 11) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 12))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 12) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 13))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 13) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 14))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 14) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 15))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 15) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 16))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 64))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 17))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 17) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 18))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 18) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 19))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 19) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 20))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 20) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 21))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 21) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 22))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 22) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 23))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 23) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 24))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 96))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 25))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 25) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 26))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 26) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 27))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 27) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 28))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 28) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 29))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 29) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 30))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 30) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 31))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 31) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 32))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 128))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 33))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 33) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 34))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 34) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 35))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 35) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 36))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 36) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 37))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 37) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 38))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 38) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 39))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 39) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 40))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 160))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 41))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 41) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 42))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 42) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 43))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 43) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 44))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 44) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 45))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 45) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 46))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 46) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 47))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 47) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 48))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 192))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 49))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 49) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 50))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 50) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 51))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 51) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 52))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 52) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 4) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 53))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 53) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 5) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 54))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 54) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 6) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 55))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 55) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 7) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 56))] = placeholder1[((((((rc_outer_outer * 480) + (((((int)threadIdx.x) * 60) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) * 60) & 7)) + 224))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 57))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 57) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 1) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 58))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 58) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 2) & 7)))];
    }
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[(((((int)threadIdx.x) * 60) + 59))] = placeholder1[(((((rc_outer_outer * 480) + ((((((int)threadIdx.x) * 60) + 59) >> 3) * 32)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) * 60) + 3) & 7)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 9))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 16))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 17))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 24))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 25))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 32))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 33))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 40))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 41))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 48))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 49))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 56))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 57))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 72))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 73))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 80))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 81))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 88))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 89))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 96))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 97))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 104))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 105))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 112))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 113))]));
  }
  for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
    T_relu[((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
  }
}


