
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[2];
  __shared__ float PaddedInput_shared[120];
  __shared__ float placeholder_shared[480];
  for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
    Conv2dOutput[(ff_inner_init)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 60; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)))] = placeholder[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)))];
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 240; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
    placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)))] = placeholder1[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)) >> 2) * 32) + (((int)blockIdx.x) * 4)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)) & 3)))];
  }
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 60; ++rc_outer_inner) {
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
        Conv2dOutput[(ff_inner)] = (Conv2dOutput[(ff_inner)] + (PaddedInput_shared[(((rc_outer_inner * 2) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 8) + (rc_inner * 4)) + (((int)threadIdx.x) * 2)) + ff_inner))]));
      }
    }
  }
  for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
    T_relu[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[(ax3_inner)] + placeholder2[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
  }
}


