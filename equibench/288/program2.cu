
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[10];
  __shared__ float placeholder_d_shared[16];
  __shared__ float placeholder_shared[400];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  T_dense[(4)] = 0.000000e+00f;
  T_dense[(5)] = 0.000000e+00f;
  T_dense[(6)] = 0.000000e+00f;
  T_dense[(7)] = 0.000000e+00f;
  T_dense[(8)] = 0.000000e+00f;
  T_dense[(9)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) >> 3) * 1024) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    if (((int)threadIdx.x) < 6) {
      placeholder_d_shared[((((int)threadIdx.x) + 10))] = placeholder[((((((((int)threadIdx.x) + 10) >> 3) * 1024) + (k_outer_outer * 8)) + (((int)threadIdx.x) + 2)))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 20))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 20) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 40))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 5120))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 60))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 60) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 80))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 10240))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 100))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 100) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 120))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 15360))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 140))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 140) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 160))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 20480))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 180))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 180) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 200))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 25600))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 220))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 220) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 240))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 30720))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 260))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 260) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 280))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 35840))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 300))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 300) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 320))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 40960))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 340))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 340) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 360))))[0] = ((float2*)(placeholder1 + ((((((((int)blockIdx.x) * 51200) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 46080))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 380))))[0] = ((float2*)(placeholder1 + (((((((int)blockIdx.x) * 51200) + ((((((int)threadIdx.x) * 2) + 380) >> 3) * 1024)) + (k_outer_outer * 8)) + (((((int)threadIdx.x) * 2) + 4) & 7)))))[0];
    __syncthreads();
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 40))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 8))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 16))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 24))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 32))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(8)] * placeholder_shared[((((int)threadIdx.x) * 40))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 8))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 16))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 24))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 32))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 1))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 9))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 17))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 25))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 33))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 1))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 9))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 17))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 25))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 33))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 2))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 10))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 18))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 26))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 34))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 2))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 10))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 18))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 26))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 34))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 3))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 11))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 19))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 27))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 35))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 3))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 11))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 19))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 27))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 35))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 4))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 12))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 20))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 28))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 36))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 4))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 12))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 20))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 28))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 36))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 5))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 13))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 21))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 29))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 37))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 5))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 13))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 21))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 29))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 37))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 6))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 14))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 22))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 30))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 38))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 6))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 14))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 22))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 30))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 38))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 7))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 15))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 23))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 31))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 39))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 7))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 15))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 23))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 31))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 40) + 39))]));
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
      T_add[(((((ax0_inner * 1000) + (((int)blockIdx.x) * 50)) + (((int)threadIdx.x) * 5)) + ax1_inner))] = (T_dense[(((ax0_inner * 5) + ax1_inner))] + placeholder2[((((((int)blockIdx.x) * 50) + (((int)threadIdx.x) * 5)) + ax1_inner))]);
    }
  }
}


