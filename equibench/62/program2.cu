
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ T_divide, float* __restrict__ placeholder_red) {
  T_divide[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] = (placeholder_red[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)))] * 1.953125e-03f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder_red, float* __restrict__ placeholder) {
  placeholder_red[(((int)blockIdx.x))] = 0.000000e+00f;
  for (int k2 = 0; k2 < 512; ++k2) {
    placeholder_red[(((int)blockIdx.x))] = (placeholder_red[(((int)blockIdx.x))] + placeholder[(((((int)blockIdx.x) * 512) + k2))]);
  }
}


