
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[16];
  __shared__ float PaddedInput_shared[1792];
  __shared__ float placeholder_shared[48];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(8)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(9)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(10)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(11)] = 0.000000e+00f;
  DepthwiseConv2d[(4)] = 0.000000e+00f;
  DepthwiseConv2d[(12)] = 0.000000e+00f;
  DepthwiseConv2d[(5)] = 0.000000e+00f;
  DepthwiseConv2d[(13)] = 0.000000e+00f;
  DepthwiseConv2d[(6)] = 0.000000e+00f;
  DepthwiseConv2d[(14)] = 0.000000e+00f;
  DepthwiseConv2d[(7)] = 0.000000e+00f;
  DepthwiseConv2d[(15)] = 0.000000e+00f;
  for (int di_outer_outer = 0; di_outer_outer < 3; ++di_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
      if (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 1792) {
        if (((int)threadIdx.x) < 75) {
          PaddedInput_shared[(((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s))] = (((((1 <= ((((int)blockIdx.x) / 20) + di_outer_outer)) && (((((int)blockIdx.x) / 20) + di_outer_outer) < 121)) && (1 <= ((((((int)blockIdx.x) % 20) >> 1) * 12) + ((((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224) >> 4)))) && (((((((int)blockIdx.x) % 20) >> 1) * 12) + ((((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224) >> 4)) < 121)) ? placeholder[(((((((((((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 224) * 460800) + (di_outer_outer * 3840)) + ((((int)blockIdx.x) >> 1) * 384)) + (((((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) % 224) >> 4) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((((int)threadIdx.x) * 24) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 15)) - 3872))] : 0.000000e+00f);
        }
      }
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[((((int)threadIdx.x) * 3))] = placeholder1[(((((di_outer_outer * 96) + (((((int)threadIdx.x) * 3) >> 4) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) * 3) & 15)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder1[(((((di_outer_outer * 96) + ((((((int)threadIdx.x) * 3) + 1) >> 4) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((((int)threadIdx.x) * 3) + 1) & 15)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder1[(((((di_outer_outer * 96) + ((((((int)threadIdx.x) * 3) + 2) >> 4) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + (((((int)threadIdx.x) * 3) + 2) & 15)))];
    }
    __syncthreads();
    for (int c_outer_inner = 0; c_outer_inner < 8; ++c_outer_inner) {
      DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[((((((((int)threadIdx.x) / 12) * 224) + (((((int)threadIdx.x) % 12) >> 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + c_outer_inner))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + c_outer_inner))]));
      DepthwiseConv2d[((c_outer_inner + 8))] = (DepthwiseConv2d[((c_outer_inner + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 224) + (((((int)threadIdx.x) % 12) >> 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + c_outer_inner) + 16))] * placeholder_shared[((((((int)threadIdx.x) & 1) * 8) + c_outer_inner))]));
      DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 224) + (((((int)threadIdx.x) % 12) >> 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + c_outer_inner) + 16))] * placeholder_shared[(((((((int)threadIdx.x) & 1) * 8) + c_outer_inner) + 16))]));
      DepthwiseConv2d[((c_outer_inner + 8))] = (DepthwiseConv2d[((c_outer_inner + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 224) + (((((int)threadIdx.x) % 12) >> 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + c_outer_inner) + 32))] * placeholder_shared[(((((((int)threadIdx.x) & 1) * 8) + c_outer_inner) + 16))]));
      DepthwiseConv2d[(c_outer_inner)] = (DepthwiseConv2d[(c_outer_inner)] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 224) + (((((int)threadIdx.x) % 12) >> 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + c_outer_inner) + 32))] * placeholder_shared[(((((((int)threadIdx.x) & 1) * 8) + c_outer_inner) + 32))]));
      DepthwiseConv2d[((c_outer_inner + 8))] = (DepthwiseConv2d[((c_outer_inner + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 12) * 224) + (((((int)threadIdx.x) % 12) >> 1) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + c_outer_inner) + 48))] * placeholder_shared[(((((((int)threadIdx.x) & 1) * 8) + c_outer_inner) + 32))]));
    }
  }
  for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
    for (int i3_inner = 0; i3_inner < 8; ++i3_inner) {
      compute[(((((((((((int)threadIdx.x) / 12) * 460800) + ((((int)blockIdx.x) >> 1) * 384)) + (((((int)threadIdx.x) % 12) >> 1) * 64)) + (i2_inner * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + i3_inner))] = max(min((DepthwiseConv2d[(((i2_inner * 8) + i3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 16) + ((((int)threadIdx.x) & 1) * 8)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


