
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[14];
  __shared__ float PaddedInput_shared[406];
  __shared__ float placeholder_shared[3712];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) >> 1) * 928) + (rc_outer_outer * 58)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[(((((((((int)threadIdx.x) + 32) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 32) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[(((((((((int)threadIdx.x) + 64) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 6)))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[(((((((((int)threadIdx.x) + 96) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 38) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[(((((((((int)threadIdx.x) + 128) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 12)))];
    PaddedInput_shared[((((int)threadIdx.x) + 160))] = placeholder[(((((((((int)threadIdx.x) + 160) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 44) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[(((((((((int)threadIdx.x) + 192) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 18)))];
    PaddedInput_shared[((((int)threadIdx.x) + 224))] = placeholder[(((((((((int)threadIdx.x) + 224) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 50) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[(((((((((int)threadIdx.x) + 256) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 288))] = placeholder[(((((((((int)threadIdx.x) + 288) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 56) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[(((((((((int)threadIdx.x) + 320) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + ((((int)threadIdx.x) + 30) % 58)))];
    PaddedInput_shared[((((int)threadIdx.x) + 352))] = placeholder[(((((((((int)threadIdx.x) + 352) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 4)))];
    if (((int)threadIdx.x) < 22) {
      PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[(((((((((int)threadIdx.x) + 384) / 58) * 6496) + ((((int)blockIdx.x) >> 1) * 928)) + (rc_outer_outer * 58)) + (((int)threadIdx.x) + 36)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 116; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 7424) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 32) + ((int)threadIdx.x)) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int rc_inner = 0; rc_inner < 29; ++rc_inner) {
        Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((rc_outer_inner * 29) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((rc_outer_inner * 29) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
        Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 58))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 58))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
        Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 116))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 116))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
        Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 174))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 174))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
        Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 232))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 232))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
        Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 290))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 290))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
        Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 348))] * placeholder_shared[((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)))]));
        Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[((((rc_outer_inner * 29) + rc_inner) + 348))] * placeholder_shared[(((((rc_outer_inner * 1856) + (rc_inner * 64)) + (((int)threadIdx.x) * 2)) + 1))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((ax1_inner * 896) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


