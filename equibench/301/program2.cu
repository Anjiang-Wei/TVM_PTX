
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[480];
  __shared__ float PaddedInput_shared[236];
  __shared__ float placeholder_shared[256];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 10; ++yy_inner_init) {
      Conv2dOutput[(((nn_outer_inner_init * 20) + (yy_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 80))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 160))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 240))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 320))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 400))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 81))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 161))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 241))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 321))] = 0.000000e+00f;
      Conv2dOutput[((((nn_outer_inner_init * 20) + (yy_inner_init * 2)) + 401))] = 0.000000e+00f;
    }
  }
  for (int ry_outer_outer = 0; ry_outer_outer < 3; ++ry_outer_outer) {
    for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
      for (int rc_outer_outer = 0; rc_outer_outer < 256; ++rc_outer_outer) {
        __syncthreads();
        PaddedInput_shared[((((int)threadIdx.x) * 2))] = (((1 <= (ry_outer_outer + ((((int)threadIdx.x) * 2) % 59))) && (1 <= ((((int)blockIdx.x) * 2) + rx_outer_outer))) ? placeholder[((((((((((((int)threadIdx.x) * 2) / 59) * 921600) + (ry_outer_outer * 15360)) + (((((int)threadIdx.x) * 2) % 59) * 15360)) + (((int)blockIdx.x) * 512)) + (rx_outer_outer * 256)) + rc_outer_outer) - 15616))] : 0.000000e+00f);
        PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] = (((1 <= (ry_outer_outer + (((((int)threadIdx.x) * 2) + 1) % 59))) && (1 <= ((((int)blockIdx.x) * 2) + rx_outer_outer))) ? placeholder[(((((((((((((int)threadIdx.x) * 2) + 1) / 59) * 921600) + (ry_outer_outer * 15360)) + ((((((int)threadIdx.x) * 2) + 1) % 59) * 15360)) + (((int)blockIdx.x) * 512)) + (rx_outer_outer * 256)) + rc_outer_outer) - 15616))] : 0.000000e+00f);
        if (((int)threadIdx.x) < 54) {
          PaddedInput_shared[(((((int)threadIdx.x) * 2) + 128))] = (((1 <= (ry_outer_outer + (((((int)threadIdx.x) * 2) + 10) % 59))) && (1 <= ((((int)blockIdx.x) * 2) + rx_outer_outer))) ? placeholder[(((((((((((((int)threadIdx.x) * 2) + 128) / 59) * 921600) + (ry_outer_outer * 15360)) + ((((((int)threadIdx.x) * 2) + 10) % 59) * 15360)) + (((int)blockIdx.x) * 512)) + (rx_outer_outer * 256)) + rc_outer_outer) - 15616))] : 0.000000e+00f);
        }
        if (((int)threadIdx.x) < 54) {
          PaddedInput_shared[(((((int)threadIdx.x) * 2) + 129))] = (((1 <= (ry_outer_outer + (((((int)threadIdx.x) * 2) + 11) % 59))) && (1 <= ((((int)blockIdx.x) * 2) + rx_outer_outer))) ? placeholder[(((((((((((((int)threadIdx.x) * 2) + 129) / 59) * 921600) + (ry_outer_outer * 15360)) + ((((((int)threadIdx.x) * 2) + 11) % 59) * 15360)) + (((int)blockIdx.x) * 512)) + (rx_outer_outer * 256)) + rc_outer_outer) - 15616))] : 0.000000e+00f);
        }
        placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((ry_outer_outer * 196608) + (rx_outer_outer * 65536)) + (rc_outer_outer * 256)) + ((int)threadIdx.x)))];
        placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((((ry_outer_outer * 196608) + (rx_outer_outer * 65536)) + (rc_outer_outer * 256)) + ((int)threadIdx.x)) + 64))];
        placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((((ry_outer_outer * 196608) + (rx_outer_outer * 65536)) + (rc_outer_outer * 256)) + ((int)threadIdx.x)) + 128))];
        placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((((ry_outer_outer * 196608) + (rx_outer_outer * 65536)) + (rc_outer_outer * 256)) + ((int)threadIdx.x)) + 192))];
        __syncthreads();
        for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
          for (int yy_inner = 0; yy_inner < 10; ++yy_inner) {
            Conv2dOutput[(((nn_outer_inner * 20) + (yy_inner * 2)))] = (Conv2dOutput[(((nn_outer_inner * 20) + (yy_inner * 2)))] + (PaddedInput_shared[(((nn_outer_inner * 59) + (yy_inner * 2)))] * placeholder_shared[((((int)threadIdx.x) * 2))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 80))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 80))] + (PaddedInput_shared[(((nn_outer_inner * 59) + (yy_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 160))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 160))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 20))] * placeholder_shared[((((int)threadIdx.x) * 2))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 240))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 240))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 20))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 320))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 320))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 40))] * placeholder_shared[((((int)threadIdx.x) * 2))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 400))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 400))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 40))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 1))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 1))] + (PaddedInput_shared[(((nn_outer_inner * 59) + (yy_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 81))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 81))] + (PaddedInput_shared[(((nn_outer_inner * 59) + (yy_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 161))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 161))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 20))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 241))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 241))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 20))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 321))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 321))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 40))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
            Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 401))] = (Conv2dOutput[((((nn_outer_inner * 20) + (yy_inner * 2)) + 401))] + (PaddedInput_shared[((((nn_outer_inner * 59) + (yy_inner * 2)) + 40))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 10; ++ax1_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((ax0_inner * 230400) + (ax1_inner * 7680)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 20) + (ax1_inner * 2)) + ax3_inner))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 230400) + (ax1_inner * 7680)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 128))] = max((Conv2dOutput[(((((ax0_inner * 20) + (ax1_inner * 2)) + ax3_inner) + 80))] + placeholder2[((((((int)threadIdx.x) * 2) + ax3_inner) + 128))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 230400) + (ax1_inner * 7680)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 76800))] = max((Conv2dOutput[(((((ax0_inner * 20) + (ax1_inner * 2)) + ax3_inner) + 160))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 230400) + (ax1_inner * 7680)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 76928))] = max((Conv2dOutput[(((((ax0_inner * 20) + (ax1_inner * 2)) + ax3_inner) + 240))] + placeholder2[((((((int)threadIdx.x) * 2) + ax3_inner) + 128))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 230400) + (ax1_inner * 7680)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 153600))] = max((Conv2dOutput[(((((ax0_inner * 20) + (ax1_inner * 2)) + ax3_inner) + 320))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((ax0_inner * 230400) + (ax1_inner * 7680)) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 153728))] = max((Conv2dOutput[(((((ax0_inner * 20) + (ax1_inner * 2)) + ax3_inner) + 400))] + placeholder2[((((((int)threadIdx.x) * 2) + ax3_inner) + 128))]), 0.000000e+00f);
      }
    }
  }
}


