
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[600];
  __shared__ float PaddedInput_shared[3600];
  __shared__ float placeholder_shared[512];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 5; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 300))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 10))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 310))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 20))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 320))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 30))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 330))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 40))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 340))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 50))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 350))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 301))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 11))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 311))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 21))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 321))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 31))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 331))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 41))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 341))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 51))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 60) + (xx_outer_inner_init * 2)) + 351))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) / 30) * 7680) + ((((int)blockIdx.x) >> 1) * 3840)) + (((((int)threadIdx.x) % 30) >> 1) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((((int)threadIdx.x) + 768) / 30) * 7680) + ((((int)blockIdx.x) >> 1) * 3840)) + ((((((int)threadIdx.x) >> 1) + 9) % 15) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((((int)threadIdx.x) + 1536) / 30) * 7680) + ((((int)blockIdx.x) >> 1) * 3840)) + ((((((int)threadIdx.x) >> 1) + 3) % 15) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2304))] = placeholder[((((((((((int)threadIdx.x) + 2304) / 30) * 7680) + ((((int)blockIdx.x) >> 1) * 3840)) + ((((((int)threadIdx.x) >> 1) + 12) % 15) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    if (((int)threadIdx.x) < 528) {
      PaddedInput_shared[((((int)threadIdx.x) + 3072))] = placeholder[((((((((((int)threadIdx.x) + 3072) / 30) * 7680) + ((((int)blockIdx.x) >> 1) * 3840)) + ((((((int)threadIdx.x) >> 1) + 6) % 15) * 256)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    if (((int)threadIdx.x) < 512) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 1024) + ((((int)threadIdx.x) >> 8) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (((int)threadIdx.x) & 255)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 5; ++yy_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
          Conv2dOutput[(((yy_outer_inner * 60) + (xx_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 60) + (xx_outer_inner * 2)))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 300))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 300))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1800))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 10))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 10))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 30))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 310))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 310))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1830))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 20))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 20))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 60))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 320))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 320))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1860))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 30))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 30))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 90))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 330))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 330))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1890))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 40))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 40))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 120))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 340))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 340))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1920))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 50))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 50))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 150))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 350))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 350))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1950))] * placeholder_shared[(((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 301))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 301))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1800))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 11))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 11))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 30))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 311))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 311))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1830))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 21))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 21))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 321))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 321))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1860))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 31))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 31))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 90))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 331))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 331))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1890))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 41))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 41))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 341))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 341))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1920))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 51))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 51))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 150))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
          Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 351))] = (Conv2dOutput[((((yy_outer_inner * 60) + (xx_outer_inner * 2)) + 351))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 384) * 900) + (yy_outer_inner * 180)) + (((((int)threadIdx.x) % 384) >> 7) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner) + 1950))] * placeholder_shared[((((rc_outer_inner * 256) + ((((int)threadIdx.x) & 127) * 2)) + 1))]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 30; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((((int)threadIdx.x) / 384) * 460800) + (ax1_inner * 15360)) + ((((int)blockIdx.x) >> 1) * 7680)) + (((((int)threadIdx.x) % 384) >> 7) * 2560)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner))] = max(((Conv2dOutput[((((ax1_inner * 10) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner))]) + placeholder3[((((((((((((int)threadIdx.x) / 384) * 460800) + (ax1_inner * 15360)) + ((((int)blockIdx.x) >> 1) * 7680)) + (((((int)threadIdx.x) % 384) >> 7) * 2560)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)threadIdx.x) / 384) * 460800) + (ax1_inner * 15360)) + ((((int)blockIdx.x) >> 1) * 7680)) + (((((int)threadIdx.x) % 384) >> 7) * 2560)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner) + 921600))] = max(((Conv2dOutput[(((((ax1_inner * 10) + (ax2_inner * 2)) + ax3_inner) + 300))] + placeholder2[(((((((int)blockIdx.x) & 1) * 256) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner))]) + placeholder3[(((((((((((((int)threadIdx.x) / 384) * 460800) + (ax1_inner * 15360)) + ((((int)blockIdx.x) >> 1) * 7680)) + (((((int)threadIdx.x) % 384) >> 7) * 2560)) + (ax2_inner * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 127) * 2)) + ax3_inner) + 921600))]), 0.000000e+00f);
      }
    }
  }
}


