
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[80];
  __shared__ float PaddedInput_shared[2880];
  __shared__ float placeholder_shared[48];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 4; ++nn_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 20) + (xx_outer_inner_init * 4)) + ff_outer_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 72; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 216) + (((int)threadIdx.x) >> 1)) < 1440) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 432) + ((int)threadIdx.x)) < 2880) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 432) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 216) + (((int)threadIdx.x) >> 1)) / 360) * 518400) + (((int)blockIdx.x) * 51840)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 216) + (((int)threadIdx.x) >> 1)) % 360) * 144)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
        }
      }
    }
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 48) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 4; ++nn_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
            Conv2dOutput[((((nn_outer_inner * 20) + (xx_outer_inner * 4)) + ff_outer_inner))] = (Conv2dOutput[((((nn_outer_inner * 20) + (xx_outer_inner * 4)) + ff_outer_inner))] + (PaddedInput_shared[(((((nn_outer_inner * 720) + ((((int)threadIdx.x) / 6) * 10)) + (xx_outer_inner * 2)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 24) + ((((int)threadIdx.x) % 6) * 4)) + ff_outer_inner))]));
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_add[(((((((ax0_inner * 86400) + (((int)blockIdx.x) * 8640)) + ((((int)threadIdx.x) / 6) * 120)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 6) * 4)) + ax3_inner))] = (placeholder2[(((((((ax0_inner * 86400) + (((int)blockIdx.x) * 8640)) + ((((int)threadIdx.x) / 6) * 120)) + (ax2_inner * 24)) + ((((int)threadIdx.x) % 6) * 4)) + ax3_inner))] + (Conv2dOutput[((((ax0_inner * 20) + (ax2_inner * 4)) + ax3_inner))] + placeholder3[((((((int)threadIdx.x) % 6) * 4) + ax3_inner))]));
      }
    }
  }
}


