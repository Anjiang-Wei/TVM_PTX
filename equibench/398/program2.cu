
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[60];
  __shared__ float PaddedInput_shared[9000];
  __shared__ float placeholder_shared[800];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 10; ++ff_inner_init) {
        Conv2dOutput[((((nn_outer_inner_init * 30) + (xx_inner_init * 10)) + ff_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 10; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 30; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 300) + ((int)threadIdx.x)))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 90000) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 3000)) + ((((int)threadIdx.x) / 20) * 200)) + (rc_outer_outer * 20)) + (((int)threadIdx.x) % 20)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 300) + ((int)threadIdx.x)) < 800) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 300) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 1600) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 300) + ((int)threadIdx.x)) / 40) * 80)) + ((((int)blockIdx.x) & 1) * 40)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 300) + ((int)threadIdx.x)) % 40)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int rc_inner = 0; rc_inner < 5; ++rc_inner) {
          for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
            for (int ff_inner = 0; ff_inner < 10; ++ff_inner) {
              Conv2dOutput[((((nn_outer_inner * 30) + (xx_inner * 10)) + ff_inner))] = (Conv2dOutput[((((nn_outer_inner * 30) + (xx_inner * 10)) + ff_inner))] + (PaddedInput_shared[((((((nn_outer_inner * 4500) + ((((int)threadIdx.x) >> 2) * 60)) + (xx_inner * 20)) + (rc_outer_inner * 5)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 200) + (rc_inner * 40)) + ((((int)threadIdx.x) & 3) * 10)) + ff_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 10; ++ax3_inner) {
        T_add[(((((((((((int)blockIdx.x) >> 1) * 36000) + (ax0_inner * 18000)) + ((((int)threadIdx.x) >> 2) * 240)) + (ax2_inner * 80)) + ((((int)blockIdx.x) & 1) * 40)) + ((((int)threadIdx.x) & 3) * 10)) + ax3_inner))] = ((Conv2dOutput[((((ax0_inner * 30) + (ax2_inner * 10)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 40) + ((((int)threadIdx.x) & 3) * 10)) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 1) * 36000) + (ax0_inner * 18000)) + ((((int)threadIdx.x) >> 2) * 240)) + (ax2_inner * 80)) + ((((int)blockIdx.x) & 1) * 40)) + ((((int)threadIdx.x) & 3) * 10)) + ax3_inner))]);
      }
    }
  }
}


