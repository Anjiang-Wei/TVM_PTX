
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[28];
  __shared__ float PaddedInput_shared[5184];
  __shared__ float placeholder_shared[1008];
  for (int dd_outer_inner_init = 0; dd_outer_inner_init < 7; ++dd_outer_inner_init) {
    for (int hh_outer_inner_init = 0; hh_outer_inner_init < 2; ++hh_outer_inner_init) {
      for (int dd_inner_init = 0; dd_inner_init < 2; ++dd_inner_init) {
        Conv3dOutput[((((dd_outer_inner_init * 4) + (dd_inner_init * 2)) + hh_outer_inner_init))] = 0.000000e+00f;
      }
    }
  }
  for (int rw_outer_outer = 0; rw_outer_outer < 7; ++rw_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 41; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) < 5184) {
        PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)))] = (((((((1 <= (((((int)blockIdx.x) / 448) * 14) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 1296) / 81))) && ((((((int)blockIdx.x) / 448) * 14) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 1296) / 81)) < 113)) && (3 <= ((((((int)blockIdx.x) % 448) >> 4) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 81) / 9)))) && (((((((int)blockIdx.x) % 448) >> 4) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 81) / 9)) < 115)) && (3 <= (((((((int)blockIdx.x) & 15) >> 2) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 9) / 3)) + rw_outer_outer))) && ((((((((int)blockIdx.x) & 15) >> 2) * 4) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 9) / 3)) + rw_outer_outer) < 19)) ? placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) / 1296) * 602112) + ((((int)blockIdx.x) / 448) * 75264)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 1296) / 81) * 5376)) + (((((int)blockIdx.x) % 448) >> 4) * 192)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 81) / 9) * 48)) + (((((int)blockIdx.x) & 15) >> 2) * 12)) + (rw_outer_outer * 3)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + ((int)threadIdx.x)) % 9)) - 5529))] : 0.000000e+00f);
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 8) + (((int)threadIdx.x) >> 4)) < 63) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 128) + ((int)threadIdx.x)) < 1008) {
          placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 128) + ((int)threadIdx.x)))] = placeholder1[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 8) + (((int)threadIdx.x) >> 4)) / 3) * 1344) + (rw_outer_outer * 192)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer1 * 8) + (((int)threadIdx.x) >> 4)) % 3) * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)))];
        }
      }
    }
    __syncthreads();
    for (int rd_outer_inner = 0; rd_outer_inner < 3; ++rd_outer_inner) {
      for (int rh_outer_inner = 0; rh_outer_inner < 7; ++rh_outer_inner) {
        for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
          for (int dd_outer_inner = 0; dd_outer_inner < 7; ++dd_outer_inner) {
            for (int hh_outer_inner = 0; hh_outer_inner < 2; ++hh_outer_inner) {
              for (int dd_inner = 0; dd_inner < 2; ++dd_inner) {
                Conv3dOutput[((((dd_outer_inner * 4) + (dd_inner * 2)) + hh_outer_inner))] = (Conv3dOutput[((((dd_outer_inner * 4) + (dd_inner * 2)) + hh_outer_inner))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) >> 5) * 1296) + (dd_outer_inner * 162)) + (dd_inner * 81)) + (rd_outer_inner * 81)) + (hh_outer_inner * 18)) + (rh_outer_inner * 9)) + (((((int)threadIdx.x) & 31) >> 4) * 6)) + rc_outer_inner))] * placeholder_shared[(((((rd_outer_inner * 336) + (rh_outer_inner * 48)) + (rc_outer_inner * 16)) + (((int)threadIdx.x) & 15)))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 14; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      T_relu[(((((((((((((int)threadIdx.x) >> 5) * 3211264) + ((((int)blockIdx.x) / 448) * 401408)) + (ax1_inner * 28672)) + (((((int)blockIdx.x) % 448) >> 4) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 15) >> 2) * 128)) + (((((int)threadIdx.x) & 31) >> 4) * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) & 15)))] = max(((Conv3dOutput[(((ax1_inner * 2) + ax2_inner))] * placeholder2[((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)))]) + placeholder3[((((((int)blockIdx.x) & 3) * 16) + (((int)threadIdx.x) & 15)))]), 0.000000e+00f);
    }
  }
}


