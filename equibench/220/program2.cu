
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[100];
  __shared__ float PaddedInput_shared[1575];
  __shared__ float placeholder_shared[1344];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(70)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(80)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(90)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(71)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(81)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(91)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(72)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(82)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(92)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(73)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(83)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(93)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(64)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(74)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(84)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(94)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(65)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(75)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(85)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(95)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(66)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(76)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(86)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(96)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(67)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(77)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(87)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(97)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(68)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(78)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(88)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(98)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(69)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(79)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(89)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(99)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 1) * 3360) + ((((int)threadIdx.x) / 21) * 672)) + (rc_outer_outer * 21)) + (((int)threadIdx.x) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 48))] = placeholder[((((((((int)blockIdx.x) >> 1) * 3360) + (((((int)threadIdx.x) + 48) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 6) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 96))] = placeholder[((((((((((int)threadIdx.x) + 96) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 96) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 12) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 144))] = placeholder[((((((((((int)threadIdx.x) + 144) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 39) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 18) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[((((((((((int)threadIdx.x) + 192) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 87) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 3) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 240))] = placeholder[((((((((((int)threadIdx.x) + 240) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 30) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 9) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 288))] = placeholder[((((((((((int)threadIdx.x) + 288) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 78) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 15) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 336))] = placeholder[((((((((((int)threadIdx.x) + 336) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 21) / 21) * 672)) + (rc_outer_outer * 21)) + (((int)threadIdx.x) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[((((((((((int)threadIdx.x) + 384) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 69) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 6) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 432))] = placeholder[((((((((((int)threadIdx.x) + 432) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 12) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 12) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 480))] = placeholder[((((((((((int)threadIdx.x) + 480) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 60) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 18) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 528))] = placeholder[((((((((((int)threadIdx.x) + 528) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 3) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 3) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 576))] = placeholder[((((((((((int)threadIdx.x) + 576) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 51) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 9) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 624))] = placeholder[((((((((((int)threadIdx.x) + 624) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 99) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 15) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 672))] = placeholder[((((((((((int)threadIdx.x) + 672) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 42) / 21) * 672)) + (rc_outer_outer * 21)) + (((int)threadIdx.x) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 720))] = placeholder[((((((((((int)threadIdx.x) + 720) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 90) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 6) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[((((((((((int)threadIdx.x) + 768) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 33) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 12) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 816))] = placeholder[((((((((((int)threadIdx.x) + 816) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 81) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 18) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 864))] = placeholder[((((((((((int)threadIdx.x) + 864) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 24) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 3) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 912))] = placeholder[((((((((((int)threadIdx.x) + 912) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 72) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 9) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 960))] = placeholder[((((((((((int)threadIdx.x) + 960) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 15) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 15) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1008))] = placeholder[((((((((((int)threadIdx.x) + 1008) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 63) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + (((int)threadIdx.x) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1056))] = placeholder[((((((((((int)threadIdx.x) + 1056) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 6) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 6) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1104))] = placeholder[((((((((((int)threadIdx.x) + 1104) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 54) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 12) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1152))] = placeholder[((((((((((int)threadIdx.x) + 1152) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 102) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 18) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1200))] = placeholder[((((((((((int)threadIdx.x) + 1200) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 45) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 3) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1248))] = placeholder[((((((((((int)threadIdx.x) + 1248) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 93) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 9) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1296))] = placeholder[((((((((((int)threadIdx.x) + 1296) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 36) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 15) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1344))] = placeholder[((((((((((int)threadIdx.x) + 1344) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 84) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + (((int)threadIdx.x) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1392))] = placeholder[((((((((((int)threadIdx.x) + 1392) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 27) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 6) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1440))] = placeholder[((((((((((int)threadIdx.x) + 1440) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + ((((((int)threadIdx.x) + 75) % 105) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 12) % 21)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1488))] = placeholder[((((((((((int)threadIdx.x) + 1488) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 18) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 18) % 21)))];
    if (((int)threadIdx.x) < 39) {
      PaddedInput_shared[((((int)threadIdx.x) + 1536))] = placeholder[((((((((((int)threadIdx.x) + 1536) / 105) * 10080) + ((((int)blockIdx.x) >> 1) * 3360)) + (((((int)threadIdx.x) + 66) / 21) * 672)) + (rc_outer_outer * 21)) + ((((int)threadIdx.x) + 3) % 21)))];
    }
    placeholder_shared[((((int)threadIdx.x) * 21))] = placeholder1[(((((rc_outer_outer * 2688) + (((((int)threadIdx.x) * 21) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) * 21) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 1))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 1) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 2))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 2) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 2) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 3))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 3) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 3) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 4))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 4) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 4) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 5))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 5) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 5) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 6))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 6) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 6) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 7))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 7) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 7) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 8))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 8) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 8) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 9))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 9) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 9) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 10))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 10) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 10) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 11))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 11) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 11) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 12))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 12) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 12) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 13))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 13) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 13) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 14))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 14) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 14) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 15))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 15) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 15) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 16))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 16) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 16) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 17))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 17) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 17) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 18))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 18) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 18) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 19))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 19) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 19) & 63)))];
    placeholder_shared[(((((int)threadIdx.x) * 21) + 20))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 20) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 20) & 63)))];
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1008))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1008) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 48) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1009))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1009) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 49) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1010))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1010) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 50) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1011))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1011) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 51) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1012))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1012) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 52) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1013))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1013) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 53) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1014))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1014) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 54) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1015))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1015) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 55) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1016))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1016) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 56) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1017))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1017) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 57) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1018))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1018) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 58) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1019))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1019) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 59) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1020))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1020) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 60) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1021))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1021) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 61) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1022))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1022) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 62) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1023))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1023) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 63) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1024))] = placeholder1[((((((rc_outer_outer * 2688) + (((((int)threadIdx.x) * 21) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) * 21) & 63)) + 2048))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1025))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1025) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 1) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1026))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1026) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 2) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1027))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1027) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 3) & 63)))];
    }
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[(((((int)threadIdx.x) * 21) + 1028))] = placeholder1[(((((rc_outer_outer * 2688) + ((((((int)threadIdx.x) * 21) + 1028) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 21) + 4) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 21; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 105))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 105))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 210))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 210))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 315))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 315))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 420))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 105))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 105))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 210))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 210))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 315))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 315))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 21))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 21))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 126))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 126))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 231))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 231))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 336))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 336))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 441))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 441))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 21))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 21))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 126))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 126))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 231))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 231))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 336))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 336))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 441))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 441))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 42))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 42))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 147))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 147))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 252))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 252))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 357))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 357))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 462))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 462))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 42))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 42))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 147))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 147))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 252))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 252))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 357))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 357))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 462))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 462))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 63))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 63))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 168))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 168))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 273))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 273))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 378))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 378))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 483))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 483))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 63))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 63))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 168))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 168))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 273))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 273))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 378))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 378))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 483))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 483))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 84))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 84))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 189))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 189))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 294))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 294))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 399))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 399))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 504))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 504))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 32))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 84))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 84))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 189))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 189))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 294))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 294))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 399))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 399))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 504))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 1))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 525) + rc_outer_inner) + 504))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 15) * 2)) + 33))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[(((((((((((int)threadIdx.x) >> 4) * 9600) + (ax1_inner * 1920)) + ((((int)blockIdx.x) >> 1) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 10) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[((((((((((((int)threadIdx.x) >> 4) * 9600) + (ax1_inner * 1920)) + ((((int)blockIdx.x) >> 1) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[(((((ax1_inner * 10) + (ax2_inner * 2)) + ax3_inner) + 50))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 15) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
      }
    }
  }
}


