
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[16];
  __shared__ float PaddedInput_shared[64];
  __shared__ float placeholder_shared[128];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 192; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 64) {
      PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 98304) + ((((int)threadIdx.x) >> 2) * 6144)) + ((((int)blockIdx.x) & 7) * 768)) + ((((int)threadIdx.x) & 3) * 192)) + rc_outer_outer))];
    }
    if (((int)threadIdx.x) < 128) {
      placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 128) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 8))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 1))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 3))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 4))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 5))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 6))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 7))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 8))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 1))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 2))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 3))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 4))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 5))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 6))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 6) * 8) + ((((int)threadIdx.x) & 63) >> 4)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 8) + 7))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 8; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 6) * 8192)) + (ax1_inner * 4096)) + ((((int)blockIdx.x) & 7) * 512)) + ((((int)threadIdx.x) & 63) * 8)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 8) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 15) * 8) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


