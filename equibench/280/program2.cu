
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[112];
  __shared__ float PaddedInput_shared[702];
  __shared__ float placeholder_shared[512];
  Conv3dOutput[(0)] = 0.000000e+00f;
  Conv3dOutput[(14)] = 0.000000e+00f;
  Conv3dOutput[(28)] = 0.000000e+00f;
  Conv3dOutput[(42)] = 0.000000e+00f;
  Conv3dOutput[(56)] = 0.000000e+00f;
  Conv3dOutput[(70)] = 0.000000e+00f;
  Conv3dOutput[(84)] = 0.000000e+00f;
  Conv3dOutput[(98)] = 0.000000e+00f;
  Conv3dOutput[(1)] = 0.000000e+00f;
  Conv3dOutput[(15)] = 0.000000e+00f;
  Conv3dOutput[(29)] = 0.000000e+00f;
  Conv3dOutput[(43)] = 0.000000e+00f;
  Conv3dOutput[(57)] = 0.000000e+00f;
  Conv3dOutput[(71)] = 0.000000e+00f;
  Conv3dOutput[(85)] = 0.000000e+00f;
  Conv3dOutput[(99)] = 0.000000e+00f;
  Conv3dOutput[(2)] = 0.000000e+00f;
  Conv3dOutput[(16)] = 0.000000e+00f;
  Conv3dOutput[(30)] = 0.000000e+00f;
  Conv3dOutput[(44)] = 0.000000e+00f;
  Conv3dOutput[(58)] = 0.000000e+00f;
  Conv3dOutput[(72)] = 0.000000e+00f;
  Conv3dOutput[(86)] = 0.000000e+00f;
  Conv3dOutput[(100)] = 0.000000e+00f;
  Conv3dOutput[(3)] = 0.000000e+00f;
  Conv3dOutput[(17)] = 0.000000e+00f;
  Conv3dOutput[(31)] = 0.000000e+00f;
  Conv3dOutput[(45)] = 0.000000e+00f;
  Conv3dOutput[(59)] = 0.000000e+00f;
  Conv3dOutput[(73)] = 0.000000e+00f;
  Conv3dOutput[(87)] = 0.000000e+00f;
  Conv3dOutput[(101)] = 0.000000e+00f;
  Conv3dOutput[(4)] = 0.000000e+00f;
  Conv3dOutput[(18)] = 0.000000e+00f;
  Conv3dOutput[(32)] = 0.000000e+00f;
  Conv3dOutput[(46)] = 0.000000e+00f;
  Conv3dOutput[(60)] = 0.000000e+00f;
  Conv3dOutput[(74)] = 0.000000e+00f;
  Conv3dOutput[(88)] = 0.000000e+00f;
  Conv3dOutput[(102)] = 0.000000e+00f;
  Conv3dOutput[(5)] = 0.000000e+00f;
  Conv3dOutput[(19)] = 0.000000e+00f;
  Conv3dOutput[(33)] = 0.000000e+00f;
  Conv3dOutput[(47)] = 0.000000e+00f;
  Conv3dOutput[(61)] = 0.000000e+00f;
  Conv3dOutput[(75)] = 0.000000e+00f;
  Conv3dOutput[(89)] = 0.000000e+00f;
  Conv3dOutput[(103)] = 0.000000e+00f;
  Conv3dOutput[(6)] = 0.000000e+00f;
  Conv3dOutput[(20)] = 0.000000e+00f;
  Conv3dOutput[(34)] = 0.000000e+00f;
  Conv3dOutput[(48)] = 0.000000e+00f;
  Conv3dOutput[(62)] = 0.000000e+00f;
  Conv3dOutput[(76)] = 0.000000e+00f;
  Conv3dOutput[(90)] = 0.000000e+00f;
  Conv3dOutput[(104)] = 0.000000e+00f;
  Conv3dOutput[(7)] = 0.000000e+00f;
  Conv3dOutput[(21)] = 0.000000e+00f;
  Conv3dOutput[(35)] = 0.000000e+00f;
  Conv3dOutput[(49)] = 0.000000e+00f;
  Conv3dOutput[(63)] = 0.000000e+00f;
  Conv3dOutput[(77)] = 0.000000e+00f;
  Conv3dOutput[(91)] = 0.000000e+00f;
  Conv3dOutput[(105)] = 0.000000e+00f;
  Conv3dOutput[(8)] = 0.000000e+00f;
  Conv3dOutput[(22)] = 0.000000e+00f;
  Conv3dOutput[(36)] = 0.000000e+00f;
  Conv3dOutput[(50)] = 0.000000e+00f;
  Conv3dOutput[(64)] = 0.000000e+00f;
  Conv3dOutput[(78)] = 0.000000e+00f;
  Conv3dOutput[(92)] = 0.000000e+00f;
  Conv3dOutput[(106)] = 0.000000e+00f;
  Conv3dOutput[(9)] = 0.000000e+00f;
  Conv3dOutput[(23)] = 0.000000e+00f;
  Conv3dOutput[(37)] = 0.000000e+00f;
  Conv3dOutput[(51)] = 0.000000e+00f;
  Conv3dOutput[(65)] = 0.000000e+00f;
  Conv3dOutput[(79)] = 0.000000e+00f;
  Conv3dOutput[(93)] = 0.000000e+00f;
  Conv3dOutput[(107)] = 0.000000e+00f;
  Conv3dOutput[(10)] = 0.000000e+00f;
  Conv3dOutput[(24)] = 0.000000e+00f;
  Conv3dOutput[(38)] = 0.000000e+00f;
  Conv3dOutput[(52)] = 0.000000e+00f;
  Conv3dOutput[(66)] = 0.000000e+00f;
  Conv3dOutput[(80)] = 0.000000e+00f;
  Conv3dOutput[(94)] = 0.000000e+00f;
  Conv3dOutput[(108)] = 0.000000e+00f;
  Conv3dOutput[(11)] = 0.000000e+00f;
  Conv3dOutput[(25)] = 0.000000e+00f;
  Conv3dOutput[(39)] = 0.000000e+00f;
  Conv3dOutput[(53)] = 0.000000e+00f;
  Conv3dOutput[(67)] = 0.000000e+00f;
  Conv3dOutput[(81)] = 0.000000e+00f;
  Conv3dOutput[(95)] = 0.000000e+00f;
  Conv3dOutput[(109)] = 0.000000e+00f;
  Conv3dOutput[(12)] = 0.000000e+00f;
  Conv3dOutput[(26)] = 0.000000e+00f;
  Conv3dOutput[(40)] = 0.000000e+00f;
  Conv3dOutput[(54)] = 0.000000e+00f;
  Conv3dOutput[(68)] = 0.000000e+00f;
  Conv3dOutput[(82)] = 0.000000e+00f;
  Conv3dOutput[(96)] = 0.000000e+00f;
  Conv3dOutput[(110)] = 0.000000e+00f;
  Conv3dOutput[(13)] = 0.000000e+00f;
  Conv3dOutput[(27)] = 0.000000e+00f;
  Conv3dOutput[(41)] = 0.000000e+00f;
  Conv3dOutput[(55)] = 0.000000e+00f;
  Conv3dOutput[(69)] = 0.000000e+00f;
  Conv3dOutput[(83)] = 0.000000e+00f;
  Conv3dOutput[(97)] = 0.000000e+00f;
  Conv3dOutput[(111)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[((((int)threadIdx.x) * 52))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + (((((int)threadIdx.x) * 26) / 27) * 14336)) + (((((int)threadIdx.x) * 26) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 1))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + (((((int)threadIdx.x) * 26) / 27) * 14336)) + (((((int)threadIdx.x) * 26) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 2))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 1) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 1) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 3))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 1) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 1) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 4))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 2) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 2) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 5))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 2) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 2) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 6))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 3) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 3) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 7))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 3) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 3) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 8))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 4) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 4) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 9))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 4) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 4) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 10))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 5) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 5) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 11))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 5) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 5) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 12))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 6) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 6) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 13))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 6) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 6) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 14))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 7) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 7) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 15))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 7) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 7) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 16))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 8) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 8) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 17))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 8) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 8) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 18))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 9) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 9) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 19))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 9) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 9) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 20))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 10) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 10) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 21))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 10) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 10) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 22))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 11) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 11) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 23))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 11) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 11) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 24))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 12) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 12) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 14) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 25))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 12) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 12) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 26))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 13) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 13) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 27))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 13) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 13) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 28))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 14) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 14) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 29))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 14) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 14) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 30))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 15) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 15) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 31))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 15) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 15) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 32))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 16) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 16) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 33))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 16) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 16) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 34))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 17) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 17) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 35))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 17) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 17) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 36))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 18) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 18) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 37))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 18) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 18) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 38))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 19) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 19) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 39))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 19) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 19) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 40))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 20) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 20) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 41))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 20) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 20) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 42))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 21) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 21) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 43))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 21) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 21) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 44))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 22) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 22) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 45))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 22) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 22) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 46))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 23) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 23) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 47))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 23) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 23) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 48))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 24) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 24) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 49))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 24) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 24) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 50))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 25) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 25) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 13) {
      PaddedInput_shared[(((((int)threadIdx.x) * 52) + 51))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 200704) + ((((((int)threadIdx.x) * 26) + 25) / 27) * 14336)) + ((((((int)threadIdx.x) * 26) + 25) % 27) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + (rc_outer_outer * 2)) + 1))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 512) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 224))];
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 448))];
    }
    __syncthreads();
    Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(14)] = (Conv3dOutput[(14)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(28)] = (Conv3dOutput[(28)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(42)] = (Conv3dOutput[(42)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(56)] = (Conv3dOutput[(56)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(70)] = (Conv3dOutput[(70)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(84)] = (Conv3dOutput[(84)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(98)] = (Conv3dOutput[(98)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(15)] = (Conv3dOutput[(15)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(29)] = (Conv3dOutput[(29)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(43)] = (Conv3dOutput[(43)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(57)] = (Conv3dOutput[(57)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(71)] = (Conv3dOutput[(71)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(85)] = (Conv3dOutput[(85)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(99)] = (Conv3dOutput[(99)] + (PaddedInput_shared[(((((int)threadIdx.x) >> 4) * 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(16)] = (Conv3dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(30)] = (Conv3dOutput[(30)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(44)] = (Conv3dOutput[(44)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(58)] = (Conv3dOutput[(58)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(72)] = (Conv3dOutput[(72)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(86)] = (Conv3dOutput[(86)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(100)] = (Conv3dOutput[(100)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(17)] = (Conv3dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(31)] = (Conv3dOutput[(31)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(45)] = (Conv3dOutput[(45)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(59)] = (Conv3dOutput[(59)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(73)] = (Conv3dOutput[(73)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(87)] = (Conv3dOutput[(87)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(101)] = (Conv3dOutput[(101)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 108))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(4)] = (Conv3dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(18)] = (Conv3dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(32)] = (Conv3dOutput[(32)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(46)] = (Conv3dOutput[(46)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(60)] = (Conv3dOutput[(60)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(74)] = (Conv3dOutput[(74)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(88)] = (Conv3dOutput[(88)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(102)] = (Conv3dOutput[(102)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(5)] = (Conv3dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(19)] = (Conv3dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(33)] = (Conv3dOutput[(33)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(47)] = (Conv3dOutput[(47)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(61)] = (Conv3dOutput[(61)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(75)] = (Conv3dOutput[(75)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(89)] = (Conv3dOutput[(89)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(103)] = (Conv3dOutput[(103)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 216))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(6)] = (Conv3dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(20)] = (Conv3dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(34)] = (Conv3dOutput[(34)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(48)] = (Conv3dOutput[(48)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(62)] = (Conv3dOutput[(62)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(76)] = (Conv3dOutput[(76)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(90)] = (Conv3dOutput[(90)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(104)] = (Conv3dOutput[(104)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(7)] = (Conv3dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(21)] = (Conv3dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(35)] = (Conv3dOutput[(35)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(49)] = (Conv3dOutput[(49)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(63)] = (Conv3dOutput[(63)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(77)] = (Conv3dOutput[(77)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(91)] = (Conv3dOutput[(91)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(105)] = (Conv3dOutput[(105)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 324))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(8)] = (Conv3dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(22)] = (Conv3dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(36)] = (Conv3dOutput[(36)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(50)] = (Conv3dOutput[(50)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(64)] = (Conv3dOutput[(64)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(78)] = (Conv3dOutput[(78)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(92)] = (Conv3dOutput[(92)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(106)] = (Conv3dOutput[(106)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(9)] = (Conv3dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(23)] = (Conv3dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(37)] = (Conv3dOutput[(37)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(51)] = (Conv3dOutput[(51)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(65)] = (Conv3dOutput[(65)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(79)] = (Conv3dOutput[(79)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(93)] = (Conv3dOutput[(93)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(107)] = (Conv3dOutput[(107)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 432))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(10)] = (Conv3dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(24)] = (Conv3dOutput[(24)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(38)] = (Conv3dOutput[(38)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(52)] = (Conv3dOutput[(52)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(66)] = (Conv3dOutput[(66)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(80)] = (Conv3dOutput[(80)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(94)] = (Conv3dOutput[(94)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(108)] = (Conv3dOutput[(108)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(11)] = (Conv3dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(25)] = (Conv3dOutput[(25)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(39)] = (Conv3dOutput[(39)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(53)] = (Conv3dOutput[(53)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(67)] = (Conv3dOutput[(67)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(81)] = (Conv3dOutput[(81)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(95)] = (Conv3dOutput[(95)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(109)] = (Conv3dOutput[(109)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 540))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(12)] = (Conv3dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 2))]));
    Conv3dOutput[(26)] = (Conv3dOutput[(26)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 32))]));
    Conv3dOutput[(40)] = (Conv3dOutput[(40)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 64))]));
    Conv3dOutput[(54)] = (Conv3dOutput[(54)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 96))]));
    Conv3dOutput[(68)] = (Conv3dOutput[(68)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 128))]));
    Conv3dOutput[(82)] = (Conv3dOutput[(82)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 160))]));
    Conv3dOutput[(96)] = (Conv3dOutput[(96)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 192))]));
    Conv3dOutput[(110)] = (Conv3dOutput[(110)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 224))]));
    Conv3dOutput[(13)] = (Conv3dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 1))]));
    Conv3dOutput[(27)] = (Conv3dOutput[(27)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 33))]));
    Conv3dOutput[(41)] = (Conv3dOutput[(41)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 65))]));
    Conv3dOutput[(55)] = (Conv3dOutput[(55)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 97))]));
    Conv3dOutput[(69)] = (Conv3dOutput[(69)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 129))]));
    Conv3dOutput[(83)] = (Conv3dOutput[(83)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 161))]));
    Conv3dOutput[(97)] = (Conv3dOutput[(97)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 193))]));
    Conv3dOutput[(111)] = (Conv3dOutput[(111)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 648))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 225))]));
    Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(14)] = (Conv3dOutput[(14)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(28)] = (Conv3dOutput[(28)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(42)] = (Conv3dOutput[(42)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(56)] = (Conv3dOutput[(56)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(70)] = (Conv3dOutput[(70)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(84)] = (Conv3dOutput[(84)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(98)] = (Conv3dOutput[(98)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(15)] = (Conv3dOutput[(15)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(29)] = (Conv3dOutput[(29)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(43)] = (Conv3dOutput[(43)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(57)] = (Conv3dOutput[(57)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(71)] = (Conv3dOutput[(71)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(85)] = (Conv3dOutput[(85)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(99)] = (Conv3dOutput[(99)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(16)] = (Conv3dOutput[(16)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(30)] = (Conv3dOutput[(30)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(44)] = (Conv3dOutput[(44)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(58)] = (Conv3dOutput[(58)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(72)] = (Conv3dOutput[(72)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(86)] = (Conv3dOutput[(86)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(100)] = (Conv3dOutput[(100)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(17)] = (Conv3dOutput[(17)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(31)] = (Conv3dOutput[(31)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(45)] = (Conv3dOutput[(45)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(59)] = (Conv3dOutput[(59)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(73)] = (Conv3dOutput[(73)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(87)] = (Conv3dOutput[(87)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(101)] = (Conv3dOutput[(101)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 109))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv3dOutput[(4)] = (Conv3dOutput[(4)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(18)] = (Conv3dOutput[(18)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(32)] = (Conv3dOutput[(32)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(46)] = (Conv3dOutput[(46)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(60)] = (Conv3dOutput[(60)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(74)] = (Conv3dOutput[(74)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(88)] = (Conv3dOutput[(88)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(102)] = (Conv3dOutput[(102)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(5)] = (Conv3dOutput[(5)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(19)] = (Conv3dOutput[(19)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(33)] = (Conv3dOutput[(33)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(47)] = (Conv3dOutput[(47)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(61)] = (Conv3dOutput[(61)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(75)] = (Conv3dOutput[(75)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(89)] = (Conv3dOutput[(89)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(103)] = (Conv3dOutput[(103)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 217))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv3dOutput[(6)] = (Conv3dOutput[(6)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(20)] = (Conv3dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(34)] = (Conv3dOutput[(34)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(48)] = (Conv3dOutput[(48)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(62)] = (Conv3dOutput[(62)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(76)] = (Conv3dOutput[(76)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(90)] = (Conv3dOutput[(90)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(104)] = (Conv3dOutput[(104)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(7)] = (Conv3dOutput[(7)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(21)] = (Conv3dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(35)] = (Conv3dOutput[(35)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(49)] = (Conv3dOutput[(49)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(63)] = (Conv3dOutput[(63)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(77)] = (Conv3dOutput[(77)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(91)] = (Conv3dOutput[(91)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(105)] = (Conv3dOutput[(105)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 325))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv3dOutput[(8)] = (Conv3dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(22)] = (Conv3dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(36)] = (Conv3dOutput[(36)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(50)] = (Conv3dOutput[(50)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(64)] = (Conv3dOutput[(64)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(78)] = (Conv3dOutput[(78)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(92)] = (Conv3dOutput[(92)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(106)] = (Conv3dOutput[(106)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(9)] = (Conv3dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(23)] = (Conv3dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(37)] = (Conv3dOutput[(37)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(51)] = (Conv3dOutput[(51)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(65)] = (Conv3dOutput[(65)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(79)] = (Conv3dOutput[(79)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(93)] = (Conv3dOutput[(93)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(107)] = (Conv3dOutput[(107)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 433))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv3dOutput[(10)] = (Conv3dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(24)] = (Conv3dOutput[(24)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(38)] = (Conv3dOutput[(38)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(52)] = (Conv3dOutput[(52)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(66)] = (Conv3dOutput[(66)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(80)] = (Conv3dOutput[(80)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(94)] = (Conv3dOutput[(94)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(108)] = (Conv3dOutput[(108)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(11)] = (Conv3dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(25)] = (Conv3dOutput[(25)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(39)] = (Conv3dOutput[(39)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(53)] = (Conv3dOutput[(53)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(67)] = (Conv3dOutput[(67)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(81)] = (Conv3dOutput[(81)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(95)] = (Conv3dOutput[(95)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(109)] = (Conv3dOutput[(109)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 541))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
    Conv3dOutput[(12)] = (Conv3dOutput[(12)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 256))]));
    Conv3dOutput[(26)] = (Conv3dOutput[(26)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 288))]));
    Conv3dOutput[(40)] = (Conv3dOutput[(40)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 320))]));
    Conv3dOutput[(54)] = (Conv3dOutput[(54)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 352))]));
    Conv3dOutput[(68)] = (Conv3dOutput[(68)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 384))]));
    Conv3dOutput[(82)] = (Conv3dOutput[(82)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 416))]));
    Conv3dOutput[(96)] = (Conv3dOutput[(96)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 448))]));
    Conv3dOutput[(110)] = (Conv3dOutput[(110)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 480))]));
    Conv3dOutput[(13)] = (Conv3dOutput[(13)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 257))]));
    Conv3dOutput[(27)] = (Conv3dOutput[(27)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 289))]));
    Conv3dOutput[(41)] = (Conv3dOutput[(41)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 321))]));
    Conv3dOutput[(55)] = (Conv3dOutput[(55)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 353))]));
    Conv3dOutput[(69)] = (Conv3dOutput[(69)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 385))]));
    Conv3dOutput[(83)] = (Conv3dOutput[(83)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 417))]));
    Conv3dOutput[(97)] = (Conv3dOutput[(97)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 449))]));
    Conv3dOutput[(111)] = (Conv3dOutput[(111)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 4) + 649))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 2) + 481))]));
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax4_inner = 0; ax4_inner < 2; ++ax4_inner) {
      T_add[((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner))] = ((Conv3dOutput[(((ax1_inner * 2) + ax4_inner))] * placeholder2[((((((int)threadIdx.x) & 15) * 2) + ax4_inner))]) + placeholder3[((((((int)threadIdx.x) & 15) * 2) + ax4_inner))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 32))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 14))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 32))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 32))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 64))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 28))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 64))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 64))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 96))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 42))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 96))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 96))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 128))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 56))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 128))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 128))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 160))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 70))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 160))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 160))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 192))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 84))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 192))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 192))]);
      T_add[(((((((((((int)blockIdx.x) >> 1) * 50176) + (ax1_inner * 7168)) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ((((int)threadIdx.x) & 15) * 2)) + ax4_inner) + 224))] = ((Conv3dOutput[((((ax1_inner * 2) + ax4_inner) + 98))] * placeholder2[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 224))]) + placeholder3[(((((((int)threadIdx.x) & 15) * 2) + ax4_inner) + 224))]);
    }
  }
}


