
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[1568];
  __shared__ float PaddedInput_shared[3136];
  __shared__ float placeholder_shared[320];
  for (int nn_outer_inner_init = 0; nn_outer_inner_init < 2; ++nn_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 7; ++xx_outer_inner_init) {
      for (int ff_outer_inner_init = 0; ff_outer_inner_init < 4; ++ff_outer_inner_init) {
        for (int yy_inner_init = 0; yy_inner_init < 7; ++yy_inner_init) {
          Conv2dOutput[(((((nn_outer_inner_init * 196) + (yy_inner_init * 28)) + (xx_outer_inner_init * 4)) + ff_outer_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 196) + (yy_inner_init * 28)) + (xx_outer_inner_init * 4)) + ff_outer_inner_init) + 392))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 196) + (yy_inner_init * 28)) + (xx_outer_inner_init * 4)) + ff_outer_inner_init) + 784))] = 0.000000e+00f;
          Conv2dOutput[((((((nn_outer_inner_init * 196) + (yy_inner_init * 28)) + (xx_outer_inner_init * 4)) + ff_outer_inner_init) + 1176))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 46; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 40; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) {
        if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) < 3136) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) < 1568) {
            PaddedInput_shared[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 56) * 56) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 2)) % 14) * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) / 56) * 2576) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 20) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) >> 2)) % 14) * 184)) + (rc_outer_outer * 4)) + (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_s) & 3)))];
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 320) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
      for (int nn_outer_inner = 0; nn_outer_inner < 2; ++nn_outer_inner) {
        for (int xx_outer_inner = 0; xx_outer_inner < 7; ++xx_outer_inner) {
          for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
            for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
              for (int yy_inner = 0; yy_inner < 7; ++yy_inner) {
                Conv2dOutput[(((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner))] = (Conv2dOutput[(((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 20) * 1568) + (nn_outer_inner * 784)) + (yy_inner * 56)) + (((((int)threadIdx.x) % 20) / 10) * 28)) + (xx_outer_inner * 4)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 160) + (rc_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ff_outer_inner))]));
                Conv2dOutput[((((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner) + 392))] = (Conv2dOutput[((((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner) + 392))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 20) * 1568) + (nn_outer_inner * 784)) + (yy_inner * 56)) + (((((int)threadIdx.x) % 20) / 10) * 28)) + (xx_outer_inner * 4)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 160) + (rc_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ff_outer_inner) + 40))]));
                Conv2dOutput[((((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner) + 784))] = (Conv2dOutput[((((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner) + 784))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 20) * 1568) + (nn_outer_inner * 784)) + (yy_inner * 56)) + (((((int)threadIdx.x) % 20) / 10) * 28)) + (xx_outer_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 392))] * placeholder_shared[(((((rc_outer_inner * 160) + (rc_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ff_outer_inner))]));
                Conv2dOutput[((((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner) + 1176))] = (Conv2dOutput[((((((nn_outer_inner * 196) + (yy_inner * 28)) + (xx_outer_inner * 4)) + ff_outer_inner) + 1176))] + (PaddedInput_shared[((((((((((((int)threadIdx.x) / 20) * 1568) + (nn_outer_inner * 784)) + (yy_inner * 56)) + (((((int)threadIdx.x) % 20) / 10) * 28)) + (xx_outer_inner * 4)) + (rc_outer_inner * 2)) + rc_inner) + 392))] * placeholder_shared[((((((rc_outer_inner * 160) + (rc_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ff_outer_inner) + 40))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 7; ++ax2_inner) {
        for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
          T_add[(((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner))] = ((Conv2dOutput[(((((ax0_inner * 196) + (ax1_inner * 28)) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) % 10) * 4) + ax3_inner))]) + placeholder3[(((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner))]);
          T_add[((((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner) + 40))] = ((Conv2dOutput[((((((ax0_inner * 196) + (ax1_inner * 28)) + (ax2_inner * 4)) + ax3_inner) + 392))] + placeholder2[(((((((int)threadIdx.x) % 10) * 4) + ax3_inner) + 40))]) + placeholder3[((((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner) + 40))]);
          T_add[((((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner) + 7840))] = ((Conv2dOutput[((((((ax0_inner * 196) + (ax1_inner * 28)) + (ax2_inner * 4)) + ax3_inner) + 784))] + placeholder2[((((((int)threadIdx.x) % 10) * 4) + ax3_inner))]) + placeholder3[((((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner) + 7840))]);
          T_add[((((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner) + 7880))] = ((Conv2dOutput[((((((ax0_inner * 196) + (ax1_inner * 28)) + (ax2_inner * 4)) + ax3_inner) + 1176))] + placeholder2[(((((((int)threadIdx.x) % 10) * 4) + ax3_inner) + 40))]) + placeholder3[((((((((((((int)threadIdx.x) / 20) * 31360) + (ax0_inner * 15680)) + (ax1_inner * 1120)) + (((((int)threadIdx.x) % 20) / 10) * 560)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 10) * 4)) + ax3_inner) + 7880))]);
        }
      }
    }
  }
}


