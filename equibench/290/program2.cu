
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[4];
  __shared__ float placeholder_d_shared[8];
  __shared__ float placeholder_shared[800];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 256; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[((((int)threadIdx.x) * 8))] = placeholder[(((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 1))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 512))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 513))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 1024))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 1025))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 1536))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder[((((((int)threadIdx.x) * 2048) + (k_outer_outer * 2)) + 1537))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((int)threadIdx.x) >> 1) * 512) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    placeholder_shared[((((int)threadIdx.x) + 400))] = placeholder1[((((((((int)threadIdx.x) >> 1) * 512) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 102400))];
    __syncthreads();
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((((int)threadIdx.x) / 200) * 4))] * placeholder_shared[(((((int)threadIdx.x) % 200) * 4))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 2))] * placeholder_shared[(((((int)threadIdx.x) % 200) * 4))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 1))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 1))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(((((int)threadIdx.x) / 200) * 4))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 2))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 2))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 3))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 3))]));
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      T_add[((((((((int)threadIdx.x) / 200) * 800) + (ax0_inner * 400)) + ((((int)threadIdx.x) % 200) * 2)) + ax1_inner))] = (T_dense[(((ax0_inner * 2) + ax1_inner))] + placeholder2[((((((int)threadIdx.x) % 200) * 2) + ax1_inner))]);
    }
  }
}


