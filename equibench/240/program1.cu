
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[3840];
  __shared__ float PaddedInput_shared[9600];
  __shared__ float placeholder_shared[1024];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 3; ++xx_outer_inner_init) {
    for (int ff_outer_inner_init = 0; ff_outer_inner_init < 32; ++ff_outer_inner_init) {
      for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
        for (int xx_inner_init = 0; xx_inner_init < 5; ++xx_inner_init) {
          Conv2dOutput[(((((yy_inner_init * 1920) + (xx_outer_inner_init * 640)) + (xx_inner_init * 128)) + (ff_outer_inner_init * 4)))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 1920) + (xx_outer_inner_init * 640)) + (xx_inner_init * 128)) + (ff_outer_inner_init * 4)) + 1))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 1920) + (xx_outer_inner_init * 640)) + (xx_inner_init * 128)) + (ff_outer_inner_init * 4)) + 2))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 1920) + (xx_outer_inner_init * 640)) + (xx_inner_init * 128)) + (ff_outer_inner_init * 4)) + 3))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 240; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)))] = placeholder[((((((((int)blockIdx.x) * 307200) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 1280)) + ((((int)threadIdx.x) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 26; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)) < 1024) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 1024) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40)) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 32; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 5; ++xx_inner) {
              Conv2dOutput[(((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)))] = (Conv2dOutput[(((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 960) + (yy_inner * 480)) + ((((int)threadIdx.x) & 3) * 120)) + (xx_outer_inner * 40)) + (xx_inner * 8)) + rc_inner))] * placeholder_shared[(((rc_inner * 128) + (ff_outer_inner * 4)))]));
              Conv2dOutput[((((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)) + 1))] = (Conv2dOutput[((((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 960) + (yy_inner * 480)) + ((((int)threadIdx.x) & 3) * 120)) + (xx_outer_inner * 40)) + (xx_inner * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (ff_outer_inner * 4)) + 1))]));
              Conv2dOutput[((((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)) + 2))] = (Conv2dOutput[((((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)) + 2))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 960) + (yy_inner * 480)) + ((((int)threadIdx.x) & 3) * 120)) + (xx_outer_inner * 40)) + (xx_inner * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (ff_outer_inner * 4)) + 2))]));
              Conv2dOutput[((((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)) + 3))] = (Conv2dOutput[((((((yy_inner * 1920) + (xx_outer_inner * 640)) + (xx_inner * 128)) + (ff_outer_inner * 4)) + 3))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 2) * 960) + (yy_inner * 480)) + ((((int)threadIdx.x) & 3) * 120)) + (xx_outer_inner * 40)) + (xx_inner * 8)) + rc_inner))] * placeholder_shared[((((rc_inner * 128) + (ff_outer_inner * 4)) + 3))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 128; ++ax3_inner) {
        T_relu[(((((((((int)blockIdx.x) * 153600) + ((((int)threadIdx.x) >> 2) * 15360)) + (ax1_inner * 7680)) + ((((int)threadIdx.x) & 3) * 1920)) + (ax2_inner * 128)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 1920) + (ax2_inner * 128)) + ax3_inner))] + placeholder2[(ax3_inner)]), 0.000000e+00f);
      }
    }
  }
}


