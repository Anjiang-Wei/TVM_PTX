
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[240];
  __shared__ float PaddedInput_shared[960];
  __shared__ float placeholder_shared[1024];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 3; ++xx_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 5; ++xx_inner_init) {
        Conv2dOutput[((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 60))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 120))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 180))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 1))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 61))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 121))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 30) + (xx_outer_inner_init * 10)) + (xx_inner_init * 2)) + 181))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 16; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 60; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 16) + ((int)threadIdx.x)))] = placeholder[((((((((((int)blockIdx.x) >> 3) * 61440) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer / 15) * 15360)) + (((((int)blockIdx.x) & 7) >> 1) * 3840)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer % 15) * 256)) + (rc_outer_outer * 16)) + ((int)threadIdx.x)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      ((float2*)(placeholder_shared + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32) + (((int)threadIdx.x) * 2)))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 2048) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32) + (((int)threadIdx.x) * 2)) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 32) + (((int)threadIdx.x) * 2)) & 63)))))[0];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 3; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 5; ++xx_inner) {
              Conv2dOutput[((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)))] = (Conv2dOutput[((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 60))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 60))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 120))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 120))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 32))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 180))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 180))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 48))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 1))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 1))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 61))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 61))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 121))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 121))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 33))]));
              Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 181))] = (Conv2dOutput[(((((yy_inner * 30) + (xx_outer_inner * 10)) + (xx_inner * 2)) + 181))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 3) * 480) + (yy_inner * 240)) + (xx_outer_inner * 80)) + (xx_inner * 16)) + (rc_outer_inner * 2)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 128) + (rc_inner * 64)) + ((((int)threadIdx.x) & 7) * 2)) + 49))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 15; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((((int)blockIdx.x) >> 3) * 30720) + ((((int)threadIdx.x) >> 3) * 15360)) + (ax1_inner * 7680)) + (((((int)blockIdx.x) & 7) >> 1) * 1920)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 30) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 3) * 30720) + ((((int)threadIdx.x) >> 3) * 15360)) + (ax1_inner * 7680)) + (((((int)blockIdx.x) & 7) >> 1) * 1920)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 16))] = max((Conv2dOutput[(((((ax1_inner * 30) + (ax2_inner * 2)) + ax3_inner) + 60))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 16))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 3) * 30720) + ((((int)threadIdx.x) >> 3) * 15360)) + (ax1_inner * 7680)) + (((((int)blockIdx.x) & 7) >> 1) * 1920)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[(((((ax1_inner * 30) + (ax2_inner * 2)) + ax3_inner) + 120))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
        T_relu[(((((((((((((int)blockIdx.x) >> 3) * 30720) + ((((int)threadIdx.x) >> 3) * 15360)) + (ax1_inner * 7680)) + (((((int)blockIdx.x) & 7) >> 1) * 1920)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 48))] = max((Conv2dOutput[(((((ax1_inner * 30) + (ax2_inner * 2)) + ax3_inner) + 180))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner) + 48))]), 0.000000e+00f);
      }
    }
  }
}


