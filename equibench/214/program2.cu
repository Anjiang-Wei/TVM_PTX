
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[18];
  __shared__ float PaddedInput_shared[72];
  __shared__ float placeholder_shared[512];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 32; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) / 10) * 11520) + ((((int)threadIdx.x) / 24) * 3840)) + (((((int)blockIdx.x) % 10) >> 1) * 768)) + (((((int)threadIdx.x) % 24) >> 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 32))] = placeholder[((((((((((int)blockIdx.x) / 10) * 11520) + (((((int)threadIdx.x) + 32) / 24) * 3840)) + (((((int)blockIdx.x) % 10) >> 1) * 768)) + ((((((int)threadIdx.x) >> 3) + 1) % 3) * 256)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[(((((((((int)blockIdx.x) / 10) * 11520) + (((((int)blockIdx.x) % 10) >> 1) * 768)) + (rc_outer_outer * 8)) + ((int)threadIdx.x)) + 8192))];
    }
    ((float2*)(placeholder_shared + ((((int)threadIdx.x) * 2))))[0] = ((float2*)(placeholder1 + ((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 64))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 128))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 128))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 256))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 192))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 384))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 256))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 512))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 320))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 640))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 384))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 768))))[0];
    ((float2*)(placeholder_shared + (((((int)threadIdx.x) * 2) + 448))))[0] = ((float2*)(placeholder1 + (((((rc_outer_outer * 1024) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + 896))))[0];
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(24)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(48)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(25)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(49)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(26)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(50)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(27)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(51)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(28)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(52)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(29)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(53)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(30)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(54)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(31)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(55)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(24)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(48)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(25)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(49)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(2)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(26)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(50)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(3)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(27)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(51)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(4)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(28)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(52)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(5)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(29)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(53)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(6)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(30)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(54)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(7)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(31)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(55)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(8)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(32)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(56)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(33)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(57)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(34)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(58)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(35)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(59)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(36)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(60)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(37)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(61)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(38)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(62)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(39)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(63)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(8)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(32)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(56)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(9)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(33)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(57)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(10)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(34)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(58)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(11)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(35)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(59)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(12)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(36)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(60)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(13)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(37)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(61)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(14)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(38)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(62)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(15)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(39)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(63)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(16)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(40)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(64)] * placeholder_shared[((((int)threadIdx.x) * 2))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(17)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(41)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(65)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 64))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(18)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(42)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(66)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(19)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(43)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(67)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 192))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(20)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(44)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(68)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(21)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(45)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(69)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 320))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(22)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(46)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(70)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(23)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(47)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(71)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 448))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(16)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(40)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(64)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(17)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(41)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(65)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 65))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(18)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(42)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(66)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(19)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(43)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(67)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 193))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(20)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(44)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(68)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(21)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(45)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(69)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 321))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(22)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(46)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(70)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(23)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(47)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(71)] * placeholder_shared[(((((int)threadIdx.x) * 2) + 449))]));
  }
  for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[((((((((((int)blockIdx.x) / 10) * 5760) + (((((int)blockIdx.x) % 10) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax2_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) / 10) * 5760) + (((((int)blockIdx.x) % 10) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 1920))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 6))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((((int)blockIdx.x) / 10) * 5760) + (((((int)blockIdx.x) % 10) >> 1) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 3840))] = max((Conv2dOutput[((((ax2_inner * 2) + ax3_inner) + 12))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
    }
  }
}


