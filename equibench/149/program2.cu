
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[24];
  __shared__ float PaddedInput_shared[3840];
  __shared__ float placeholder_shared[1024];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((int)blockIdx.x) >> 3) * 3840) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 40))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 40) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 80))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 80) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 120))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 120) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 160))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 160) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 200))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 200) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 240))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 240) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 280))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 280) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 1280))];
    PaddedInput_shared[((((int)threadIdx.x) + 360))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 360) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 400))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 400) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 440))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 440) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 480))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 480) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 520))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 520) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 560))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 560) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 600))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 600) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 2560))];
    PaddedInput_shared[((((int)threadIdx.x) + 680))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 680) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 720))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 720) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 760))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 760) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 800))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 800) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 840))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 840) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 880))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 880) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 920))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (((((int)threadIdx.x) + 920) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 960))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 57600))];
    PaddedInput_shared[((((int)threadIdx.x) + 1000))] = placeholder[((((((((((int)threadIdx.x) + 1000) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 40) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1040))] = placeholder[((((((((((int)threadIdx.x) + 1040) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 80) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1080))] = placeholder[((((((((((int)threadIdx.x) + 1080) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 120) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1120))] = placeholder[((((((((((int)threadIdx.x) + 1120) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 160) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1160))] = placeholder[((((((((((int)threadIdx.x) + 1160) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 200) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1200))] = placeholder[((((((((((int)threadIdx.x) + 1200) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 240) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1240))] = placeholder[((((((((((int)threadIdx.x) + 1240) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 280) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1280))] = placeholder[((((((((((int)threadIdx.x) + 1280) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 320) >> 6) * 256)) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1320))] = placeholder[((((((((((int)threadIdx.x) + 1320) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 360) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1360))] = placeholder[((((((((((int)threadIdx.x) + 1360) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 400) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1400))] = placeholder[((((((((((int)threadIdx.x) + 1400) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 440) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1440))] = placeholder[((((((((((int)threadIdx.x) + 1440) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 480) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1480))] = placeholder[((((((((((int)threadIdx.x) + 1480) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 520) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1520))] = placeholder[((((((((((int)threadIdx.x) + 1520) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 560) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1560))] = placeholder[((((((((((int)threadIdx.x) + 1560) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 600) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1600))] = placeholder[((((((((((int)threadIdx.x) + 1600) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 640) >> 6) * 256)) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1640))] = placeholder[((((((((((int)threadIdx.x) + 1640) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 680) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1680))] = placeholder[((((((((((int)threadIdx.x) + 1680) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 720) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1720))] = placeholder[((((((((((int)threadIdx.x) + 1720) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 760) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1760))] = placeholder[((((((((((int)threadIdx.x) + 1760) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 800) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1800))] = placeholder[((((((((((int)threadIdx.x) + 1800) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 840) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1840))] = placeholder[((((((((((int)threadIdx.x) + 1840) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 880) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1880))] = placeholder[((((((((((int)threadIdx.x) + 1880) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 920) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1920))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 115200))];
    PaddedInput_shared[((((int)threadIdx.x) + 1960))] = placeholder[((((((((((int)threadIdx.x) + 1960) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 40) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2000))] = placeholder[((((((((((int)threadIdx.x) + 2000) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 80) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2040))] = placeholder[((((((((((int)threadIdx.x) + 2040) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 120) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2080))] = placeholder[((((((((((int)threadIdx.x) + 2080) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 160) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2120))] = placeholder[((((((((((int)threadIdx.x) + 2120) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 200) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2160))] = placeholder[((((((((((int)threadIdx.x) + 2160) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 240) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2200))] = placeholder[((((((((((int)threadIdx.x) + 2200) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 280) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2240))] = placeholder[((((((((((int)threadIdx.x) + 2240) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 320) >> 6) * 256)) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2280))] = placeholder[((((((((((int)threadIdx.x) + 2280) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 360) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2320))] = placeholder[((((((((((int)threadIdx.x) + 2320) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 400) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2360))] = placeholder[((((((((((int)threadIdx.x) + 2360) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 440) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2400))] = placeholder[((((((((((int)threadIdx.x) + 2400) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 480) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2440))] = placeholder[((((((((((int)threadIdx.x) + 2440) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 520) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2480))] = placeholder[((((((((((int)threadIdx.x) + 2480) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 560) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2520))] = placeholder[((((((((((int)threadIdx.x) + 2520) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 600) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2560))] = placeholder[((((((((((int)threadIdx.x) + 2560) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 640) >> 6) * 256)) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2600))] = placeholder[((((((((((int)threadIdx.x) + 2600) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 680) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2640))] = placeholder[((((((((((int)threadIdx.x) + 2640) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 720) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2680))] = placeholder[((((((((((int)threadIdx.x) + 2680) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 760) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2720))] = placeholder[((((((((((int)threadIdx.x) + 2720) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 800) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2760))] = placeholder[((((((((((int)threadIdx.x) + 2760) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 840) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2800))] = placeholder[((((((((((int)threadIdx.x) + 2800) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 880) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2840))] = placeholder[((((((((((int)threadIdx.x) + 2840) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 920) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2880))] = placeholder[((((((((int)blockIdx.x) >> 3) * 3840) + (rc_outer_outer * 64)) + ((int)threadIdx.x)) + 172800))];
    PaddedInput_shared[((((int)threadIdx.x) + 2920))] = placeholder[((((((((((int)threadIdx.x) + 2920) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 40) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2960))] = placeholder[((((((((((int)threadIdx.x) + 2960) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 80) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3000))] = placeholder[((((((((((int)threadIdx.x) + 3000) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 120) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3040))] = placeholder[((((((((((int)threadIdx.x) + 3040) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 160) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3080))] = placeholder[((((((((((int)threadIdx.x) + 3080) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 200) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3120))] = placeholder[((((((((((int)threadIdx.x) + 3120) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 240) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3160))] = placeholder[((((((((((int)threadIdx.x) + 3160) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 280) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3200))] = placeholder[((((((((((int)threadIdx.x) + 3200) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 320) >> 6) * 256)) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3240))] = placeholder[((((((((((int)threadIdx.x) + 3240) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 360) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3280))] = placeholder[((((((((((int)threadIdx.x) + 3280) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 400) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3320))] = placeholder[((((((((((int)threadIdx.x) + 3320) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 440) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3360))] = placeholder[((((((((((int)threadIdx.x) + 3360) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 480) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3400))] = placeholder[((((((((((int)threadIdx.x) + 3400) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 520) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3440))] = placeholder[((((((((((int)threadIdx.x) + 3440) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 560) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3480))] = placeholder[((((((((((int)threadIdx.x) + 3480) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 600) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3520))] = placeholder[((((((((((int)threadIdx.x) + 3520) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 640) >> 6) * 256)) + (rc_outer_outer * 64)) + ((int)threadIdx.x)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3560))] = placeholder[((((((((((int)threadIdx.x) + 3560) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 680) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 40) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3600))] = placeholder[((((((((((int)threadIdx.x) + 3600) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 720) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 16)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3640))] = placeholder[((((((((((int)threadIdx.x) + 3640) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 760) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 56) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3680))] = placeholder[((((((((((int)threadIdx.x) + 3680) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 800) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 32) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3720))] = placeholder[((((((((((int)threadIdx.x) + 3720) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 840) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 8)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3760))] = placeholder[((((((((((int)threadIdx.x) + 3760) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 880) >> 6) * 256)) + (rc_outer_outer * 64)) + ((((int)threadIdx.x) + 48) & 63)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3800))] = placeholder[((((((((((int)threadIdx.x) + 3800) / 960) * 57600) + ((((int)blockIdx.x) >> 3) * 3840)) + (((((int)threadIdx.x) + 920) >> 6) * 256)) + (rc_outer_outer * 64)) + (((int)threadIdx.x) + 24)))];
    placeholder_shared[((((int)threadIdx.x) * 8))] = placeholder1[(((((rc_outer_outer * 8192) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) & 1) * 8)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 1) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 1) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 2) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 2) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 3) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 3) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 4) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 4) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 5) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 5) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 6) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 6) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder1[(((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 7) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 7) & 15)))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 320))] = placeholder1[((((((rc_outer_outer * 8192) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 321))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 1) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 1) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 322))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 2) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 2) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 323))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 3) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 3) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 324))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 4) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 4) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 325))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 5) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 5) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 326))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 6) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 6) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 327))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 7) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 7) & 15)) + 2560))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 640))] = placeholder1[((((((rc_outer_outer * 8192) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 641))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 1) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 1) & 15)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 642))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 2) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 2) & 15)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 643))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 3) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 3) & 15)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 644))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 4) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 4) & 15)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 645))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 5) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 5) & 15)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 646))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 6) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 6) & 15)) + 5120))];
    placeholder_shared[(((((int)threadIdx.x) * 8) + 647))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 7) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 7) & 15)) + 5120))];
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 960))] = placeholder1[((((((rc_outer_outer * 8192) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 961))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 1) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 1) & 15)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 962))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 2) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 2) & 15)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 963))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 3) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 3) & 15)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 964))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 4) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 4) & 15)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 965))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 5) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 5) & 15)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 966))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 6) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 6) & 15)) + 7680))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_shared[(((((int)threadIdx.x) * 8) + 967))] = placeholder1[((((((rc_outer_outer * 8192) + ((((((int)threadIdx.x) * 8) + 7) >> 4) * 128)) + ((((int)blockIdx.x) & 7) * 16)) + (((((int)threadIdx.x) * 8) + 7) & 15)) + 7680))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 32; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 960))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 960))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 961))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 961))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 64))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1024))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1024))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 65))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1025))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1025))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 128))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 128))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1088))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1088))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 129))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 129))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1089))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1089))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1920))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1920))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2880))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2880))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1921))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1921))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2881))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2881))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1984))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1984))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2944))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2944))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1985))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 1985))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2945))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2945))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2048))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2048))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 3008))] * placeholder_shared[(((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 3008))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 1))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2049))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 2049))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 3009))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 16))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 192) + (rc_outer_inner * 2)) + 3009))] * placeholder_shared[((((rc_outer_inner * 32) + ((((int)threadIdx.x) & 7) * 2)) + 17))]));
    }
  }
  for (int ax0_inner = 0; ax0_inner < 4; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((ax0_inner * 28800) + ((((int)blockIdx.x) >> 3) * 1920)) + ((((int)threadIdx.x) >> 3) * 384)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 7) * 16)) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 6) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 7) * 16) + ((((int)threadIdx.x) & 7) * 2)) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


