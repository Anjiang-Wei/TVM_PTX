
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[5760];
  __shared__ float PaddedInput_shared[230400];
  __shared__ float placeholder_shared[65536];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 16; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 3; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 3; ++xx_inner_init) {
        for (int ff_inner_init = 0; ff_inner_init < 8; ++ff_inner_init) {
          Conv2dOutput[(((((yy_inner_init * 384) + (xx_inner_init * 128)) + (ff_outer_inner_init * 8)) + ff_inner_init))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 384) + (xx_inner_init * 128)) + (ff_outer_inner_init * 8)) + ff_inner_init) + 1152))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 384) + (xx_inner_init * 128)) + (ff_outer_inner_init * 8)) + ff_inner_init) + 2304))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 384) + (xx_inner_init * 128)) + (ff_outer_inner_init * 8)) + ff_inner_init) + 3456))] = 0.000000e+00f;
          Conv2dOutput[((((((yy_inner_init * 384) + (xx_inner_init * 128)) + (ff_outer_inner_init * 8)) + ff_inner_init) + 4608))] = 0.000000e+00f;
        }
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 5760; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)))] = placeholder[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) >> 8) * 1024) + (rc_outer_outer * 256)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 40) + ((int)threadIdx.x)) & 255)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 1639; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)) < 65536) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer_outer * 65536) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 40)) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 16; ++rc_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 16; ++ff_outer_inner) {
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 3; ++yy_inner) {
            for (int xx_inner = 0; xx_inner < 3; ++xx_inner) {
              for (int ff_inner = 0; ff_inner < 8; ++ff_inner) {
                Conv2dOutput[(((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner))] = (Conv2dOutput[(((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner))] + (PaddedInput_shared[((((((((((int)threadIdx.x) / 10) * 57600) + (yy_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (xx_inner * 256)) + (rc_outer_inner * 16)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 4096) + (rc_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + (ff_outer_inner * 8)) + ff_inner))]));
                Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 1152))] = (Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 1152))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 10) * 57600) + (yy_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (xx_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 11520))] * placeholder_shared[((((((rc_outer_inner * 4096) + (rc_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + (ff_outer_inner * 8)) + ff_inner))]));
                Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 2304))] = (Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 2304))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 10) * 57600) + (yy_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (xx_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 23040))] * placeholder_shared[((((((rc_outer_inner * 4096) + (rc_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + (ff_outer_inner * 8)) + ff_inner))]));
                Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 3456))] = (Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 3456))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 10) * 57600) + (yy_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (xx_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 34560))] * placeholder_shared[((((((rc_outer_inner * 4096) + (rc_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + (ff_outer_inner * 8)) + ff_inner))]));
                Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 4608))] = (Conv2dOutput[((((((yy_inner * 384) + (xx_inner * 128)) + (ff_outer_inner * 8)) + ff_inner) + 4608))] + (PaddedInput_shared[(((((((((((int)threadIdx.x) / 10) * 57600) + (yy_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (xx_inner * 256)) + (rc_outer_inner * 16)) + rc_inner) + 46080))] * placeholder_shared[((((((rc_outer_inner * 4096) + (rc_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + (ff_outer_inner * 8)) + ff_inner))]));
              }
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 3; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 3; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 128; ++ax3_inner) {
        T_relu[((((((((((int)threadIdx.x) / 10) * 57600) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 384) + (ax2_inner * 128)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 1) * 128) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)threadIdx.x) / 10) * 57600) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ax3_inner) + 11520))] = max((Conv2dOutput[(((((ax1_inner * 384) + (ax2_inner * 128)) + ax3_inner) + 1152))] + placeholder2[((((((int)threadIdx.x) & 1) * 128) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)threadIdx.x) / 10) * 57600) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ax3_inner) + 23040))] = max((Conv2dOutput[(((((ax1_inner * 384) + (ax2_inner * 128)) + ax3_inner) + 2304))] + placeholder2[((((((int)threadIdx.x) & 1) * 128) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)threadIdx.x) / 10) * 57600) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ax3_inner) + 34560))] = max((Conv2dOutput[(((((ax1_inner * 384) + (ax2_inner * 128)) + ax3_inner) + 3456))] + placeholder2[((((((int)threadIdx.x) & 1) * 128) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)threadIdx.x) / 10) * 57600) + (ax1_inner * 3840)) + (((((int)threadIdx.x) % 10) >> 1) * 768)) + (ax2_inner * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ax3_inner) + 46080))] = max((Conv2dOutput[(((((ax1_inner * 384) + (ax2_inner * 128)) + ax3_inner) + 4608))] + placeholder2[((((((int)threadIdx.x) & 1) * 128) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


