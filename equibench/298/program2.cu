
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[64];
  __shared__ float PaddedInput_shared[1024];
  __shared__ float placeholder_shared[128];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      for (int xx_inner_init = 0; xx_inner_init < 2; ++xx_inner_init) {
        Conv2dOutput[((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 8))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 16))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 24))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 32))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 40))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 48))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 4) + (xx_inner_init * 2)) + ff_outer_inner_init) + 56))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 36; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 128) + ((int)threadIdx.x)))] = placeholder[((((((((int)blockIdx.x) * 36864) + (ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 4608)) + ((((int)threadIdx.x) >> 2) * 144)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 128) + ((int)threadIdx.x)))];
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
        for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
          for (int xx_inner = 0; xx_inner < 2; ++xx_inner) {
            Conv2dOutput[((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner))] = (Conv2dOutput[((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 8))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 8))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner) + 8))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 16))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 16))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner) + 16))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 24))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 24))] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner))] * placeholder_shared[(((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner) + 24))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 32))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 32))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner) + 64))] * placeholder_shared[((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 40))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 40))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner) + 64))] * placeholder_shared[(((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner) + 8))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 48))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 48))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner) + 64))] * placeholder_shared[(((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner) + 16))]));
            Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 56))] = (Conv2dOutput[(((((yy_inner * 4) + (xx_inner * 2)) + ff_outer_inner) + 56))] + (PaddedInput_shared[((((((((((int)threadIdx.x) >> 5) * 256) + (yy_inner * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 8)) + (xx_inner * 4)) + rc_inner) + 64))] * placeholder_shared[(((((rc_inner * 32) + ((((int)threadIdx.x) & 3) * 2)) + ff_outer_inner) + 24))]));
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_add[((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))] = (Conv2dOutput[((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 3) * 2) + ax3_inner))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 8))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 8))] + placeholder2[(((((((int)threadIdx.x) & 3) * 2) + ax3_inner) + 8))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 16))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 16))] + placeholder2[(((((((int)threadIdx.x) & 3) * 2) + ax3_inner) + 16))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 24))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 24))] + placeholder2[(((((((int)threadIdx.x) & 3) * 2) + ax3_inner) + 24))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 512))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 32))] + placeholder2[((((((int)threadIdx.x) & 3) * 2) + ax3_inner))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 520))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 40))] + placeholder2[(((((((int)threadIdx.x) & 3) * 2) + ax3_inner) + 8))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 528))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 48))] + placeholder2[(((((((int)threadIdx.x) & 3) * 2) + ax3_inner) + 16))]);
        T_add[(((((((((((int)blockIdx.x) * 8192) + ((((int)threadIdx.x) >> 5) * 2048)) + (ax1_inner * 1024)) + (((((int)threadIdx.x) & 31) >> 2) * 64)) + (ax2_inner * 32)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 536))] = (Conv2dOutput[(((((ax1_inner * 4) + (ax2_inner * 2)) + ax3_inner) + 56))] + placeholder2[(((((((int)threadIdx.x) & 3) * 2) + ax3_inner) + 24))]);
      }
    }
  }
}


