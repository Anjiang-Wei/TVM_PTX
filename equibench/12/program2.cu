
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float Conv2dOutput[36];
  __shared__ float PaddedInput_shared[1800];
  __shared__ float placeholder_shared[192];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 6; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 3))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 3) + 18))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 3) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 3) + 19))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 3) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 3) + 20))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 8; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[((((int)threadIdx.x) * 8))] = placeholder[(((((((((int)threadIdx.x) * 2) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + (((((int)threadIdx.x) * 2) % 15) * 32)) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 1))] = placeholder[((((((((((int)threadIdx.x) * 2) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + (((((int)threadIdx.x) * 2) % 15) * 32)) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 2))] = placeholder[((((((((((int)threadIdx.x) * 2) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + (((((int)threadIdx.x) * 2) % 15) * 32)) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 3))] = placeholder[((((((((((int)threadIdx.x) * 2) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + (((((int)threadIdx.x) * 2) % 15) * 32)) + (rc_outer_outer * 4)) + 3))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 4))] = placeholder[((((((((((int)threadIdx.x) * 2) + 1) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + ((((((int)threadIdx.x) * 2) + 1) % 15) * 32)) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 5))] = placeholder[(((((((((((int)threadIdx.x) * 2) + 1) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + ((((((int)threadIdx.x) * 2) + 1) % 15) * 32)) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 6))] = placeholder[(((((((((((int)threadIdx.x) * 2) + 1) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + ((((((int)threadIdx.x) * 2) + 1) % 15) * 32)) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 225) {
      PaddedInput_shared[(((((int)threadIdx.x) * 8) + 7))] = placeholder[(((((((((((int)threadIdx.x) * 2) + 1) / 15) * 960) + ((((int)blockIdx.x) >> 2) * 480)) + ((((((int)threadIdx.x) * 2) + 1) % 15) * 32)) + (rc_outer_outer * 4)) + 3))];
    }
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[((((int)threadIdx.x) * 4))] = placeholder1[(((((rc_outer_outer * 768) + ((((int)threadIdx.x) / 12) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + ((((int)threadIdx.x) % 12) * 4)))];
    }
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder1[(((((rc_outer_outer * 768) + ((((((int)threadIdx.x) * 4) + 1) / 48) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + (((((int)threadIdx.x) * 4) + 1) % 48)))];
    }
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder1[(((((rc_outer_outer * 768) + ((((((int)threadIdx.x) * 4) + 2) / 48) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + (((((int)threadIdx.x) * 4) + 2) % 48)))];
    }
    if (((int)threadIdx.x) < 48) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder1[(((((rc_outer_outer * 768) + ((((((int)threadIdx.x) * 4) + 3) / 48) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + (((((int)threadIdx.x) * 4) + 3) % 48)))];
    }
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 6; ++yy_outer_inner) {
      for (int ff_outer_inner = 0; ff_outer_inner < 3; ++ff_outer_inner) {
        Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)))] * placeholder_shared[((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner))]));
        Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] = (Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 24))]));
        Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 48))]));
        Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] = (Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)) + 1))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 72))]));
        Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)) + 2))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 96))]));
        Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] = (Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)) + 2))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 120))]));
        Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] = (Conv2dOutput[(((yy_outer_inner * 3) + ff_outer_inner))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)) + 3))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 144))]));
        Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] = (Conv2dOutput[((((yy_outer_inner * 3) + ff_outer_inner) + 18))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 120) * 360) + (yy_outer_inner * 60)) + (((((int)threadIdx.x) % 120) >> 3) * 4)) + 3))] * placeholder_shared[(((((((int)threadIdx.x) & 7) * 3) + ff_outer_inner) + 168))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 6; ++i1_inner) {
    for (int i3_inner = 0; i3_inner < 3; ++i3_inner) {
      compute[(((((((((((int)threadIdx.x) / 120) * 34560) + (i1_inner * 5760)) + ((((int)blockIdx.x) >> 2) * 2880)) + (((((int)threadIdx.x) % 120) >> 3) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + ((((int)threadIdx.x) & 7) * 3)) + i3_inner))] = max(min((Conv2dOutput[(((i1_inner * 3) + i3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 48) + ((((int)threadIdx.x) & 7) * 3)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
      compute[((((((((((((int)threadIdx.x) / 120) * 34560) + (i1_inner * 5760)) + ((((int)blockIdx.x) >> 2) * 2880)) + (((((int)threadIdx.x) % 120) >> 3) * 192)) + ((((int)blockIdx.x) & 3) * 48)) + ((((int)threadIdx.x) & 7) * 3)) + i3_inner) + 24))] = max(min((Conv2dOutput[((((i1_inner * 3) + i3_inner) + 18))] + placeholder2[((((((((int)blockIdx.x) & 3) * 48) + ((((int)threadIdx.x) & 7) * 3)) + i3_inner) + 24))]), 6.000000e+00f), 0.000000e+00f);
    }
  }
}


