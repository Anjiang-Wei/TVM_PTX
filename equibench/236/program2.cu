
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[280];
  __shared__ float PaddedInput_shared[700];
  __shared__ float placeholder_shared[256];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(80)] = 0.000000e+00f;
  Conv2dOutput[(100)] = 0.000000e+00f;
  Conv2dOutput[(120)] = 0.000000e+00f;
  Conv2dOutput[(140)] = 0.000000e+00f;
  Conv2dOutput[(160)] = 0.000000e+00f;
  Conv2dOutput[(180)] = 0.000000e+00f;
  Conv2dOutput[(200)] = 0.000000e+00f;
  Conv2dOutput[(220)] = 0.000000e+00f;
  Conv2dOutput[(240)] = 0.000000e+00f;
  Conv2dOutput[(260)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(81)] = 0.000000e+00f;
  Conv2dOutput[(101)] = 0.000000e+00f;
  Conv2dOutput[(121)] = 0.000000e+00f;
  Conv2dOutput[(141)] = 0.000000e+00f;
  Conv2dOutput[(161)] = 0.000000e+00f;
  Conv2dOutput[(181)] = 0.000000e+00f;
  Conv2dOutput[(201)] = 0.000000e+00f;
  Conv2dOutput[(221)] = 0.000000e+00f;
  Conv2dOutput[(241)] = 0.000000e+00f;
  Conv2dOutput[(261)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(82)] = 0.000000e+00f;
  Conv2dOutput[(102)] = 0.000000e+00f;
  Conv2dOutput[(122)] = 0.000000e+00f;
  Conv2dOutput[(142)] = 0.000000e+00f;
  Conv2dOutput[(162)] = 0.000000e+00f;
  Conv2dOutput[(182)] = 0.000000e+00f;
  Conv2dOutput[(202)] = 0.000000e+00f;
  Conv2dOutput[(222)] = 0.000000e+00f;
  Conv2dOutput[(242)] = 0.000000e+00f;
  Conv2dOutput[(262)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  Conv2dOutput[(83)] = 0.000000e+00f;
  Conv2dOutput[(103)] = 0.000000e+00f;
  Conv2dOutput[(123)] = 0.000000e+00f;
  Conv2dOutput[(143)] = 0.000000e+00f;
  Conv2dOutput[(163)] = 0.000000e+00f;
  Conv2dOutput[(183)] = 0.000000e+00f;
  Conv2dOutput[(203)] = 0.000000e+00f;
  Conv2dOutput[(223)] = 0.000000e+00f;
  Conv2dOutput[(243)] = 0.000000e+00f;
  Conv2dOutput[(263)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(64)] = 0.000000e+00f;
  Conv2dOutput[(84)] = 0.000000e+00f;
  Conv2dOutput[(104)] = 0.000000e+00f;
  Conv2dOutput[(124)] = 0.000000e+00f;
  Conv2dOutput[(144)] = 0.000000e+00f;
  Conv2dOutput[(164)] = 0.000000e+00f;
  Conv2dOutput[(184)] = 0.000000e+00f;
  Conv2dOutput[(204)] = 0.000000e+00f;
  Conv2dOutput[(224)] = 0.000000e+00f;
  Conv2dOutput[(244)] = 0.000000e+00f;
  Conv2dOutput[(264)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(65)] = 0.000000e+00f;
  Conv2dOutput[(85)] = 0.000000e+00f;
  Conv2dOutput[(105)] = 0.000000e+00f;
  Conv2dOutput[(125)] = 0.000000e+00f;
  Conv2dOutput[(145)] = 0.000000e+00f;
  Conv2dOutput[(165)] = 0.000000e+00f;
  Conv2dOutput[(185)] = 0.000000e+00f;
  Conv2dOutput[(205)] = 0.000000e+00f;
  Conv2dOutput[(225)] = 0.000000e+00f;
  Conv2dOutput[(245)] = 0.000000e+00f;
  Conv2dOutput[(265)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(66)] = 0.000000e+00f;
  Conv2dOutput[(86)] = 0.000000e+00f;
  Conv2dOutput[(106)] = 0.000000e+00f;
  Conv2dOutput[(126)] = 0.000000e+00f;
  Conv2dOutput[(146)] = 0.000000e+00f;
  Conv2dOutput[(166)] = 0.000000e+00f;
  Conv2dOutput[(186)] = 0.000000e+00f;
  Conv2dOutput[(206)] = 0.000000e+00f;
  Conv2dOutput[(226)] = 0.000000e+00f;
  Conv2dOutput[(246)] = 0.000000e+00f;
  Conv2dOutput[(266)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(67)] = 0.000000e+00f;
  Conv2dOutput[(87)] = 0.000000e+00f;
  Conv2dOutput[(107)] = 0.000000e+00f;
  Conv2dOutput[(127)] = 0.000000e+00f;
  Conv2dOutput[(147)] = 0.000000e+00f;
  Conv2dOutput[(167)] = 0.000000e+00f;
  Conv2dOutput[(187)] = 0.000000e+00f;
  Conv2dOutput[(207)] = 0.000000e+00f;
  Conv2dOutput[(227)] = 0.000000e+00f;
  Conv2dOutput[(247)] = 0.000000e+00f;
  Conv2dOutput[(267)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(68)] = 0.000000e+00f;
  Conv2dOutput[(88)] = 0.000000e+00f;
  Conv2dOutput[(108)] = 0.000000e+00f;
  Conv2dOutput[(128)] = 0.000000e+00f;
  Conv2dOutput[(148)] = 0.000000e+00f;
  Conv2dOutput[(168)] = 0.000000e+00f;
  Conv2dOutput[(188)] = 0.000000e+00f;
  Conv2dOutput[(208)] = 0.000000e+00f;
  Conv2dOutput[(228)] = 0.000000e+00f;
  Conv2dOutput[(248)] = 0.000000e+00f;
  Conv2dOutput[(268)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(69)] = 0.000000e+00f;
  Conv2dOutput[(89)] = 0.000000e+00f;
  Conv2dOutput[(109)] = 0.000000e+00f;
  Conv2dOutput[(129)] = 0.000000e+00f;
  Conv2dOutput[(149)] = 0.000000e+00f;
  Conv2dOutput[(169)] = 0.000000e+00f;
  Conv2dOutput[(189)] = 0.000000e+00f;
  Conv2dOutput[(209)] = 0.000000e+00f;
  Conv2dOutput[(229)] = 0.000000e+00f;
  Conv2dOutput[(249)] = 0.000000e+00f;
  Conv2dOutput[(269)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(70)] = 0.000000e+00f;
  Conv2dOutput[(90)] = 0.000000e+00f;
  Conv2dOutput[(110)] = 0.000000e+00f;
  Conv2dOutput[(130)] = 0.000000e+00f;
  Conv2dOutput[(150)] = 0.000000e+00f;
  Conv2dOutput[(170)] = 0.000000e+00f;
  Conv2dOutput[(190)] = 0.000000e+00f;
  Conv2dOutput[(210)] = 0.000000e+00f;
  Conv2dOutput[(230)] = 0.000000e+00f;
  Conv2dOutput[(250)] = 0.000000e+00f;
  Conv2dOutput[(270)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(71)] = 0.000000e+00f;
  Conv2dOutput[(91)] = 0.000000e+00f;
  Conv2dOutput[(111)] = 0.000000e+00f;
  Conv2dOutput[(131)] = 0.000000e+00f;
  Conv2dOutput[(151)] = 0.000000e+00f;
  Conv2dOutput[(171)] = 0.000000e+00f;
  Conv2dOutput[(191)] = 0.000000e+00f;
  Conv2dOutput[(211)] = 0.000000e+00f;
  Conv2dOutput[(231)] = 0.000000e+00f;
  Conv2dOutput[(251)] = 0.000000e+00f;
  Conv2dOutput[(271)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(72)] = 0.000000e+00f;
  Conv2dOutput[(92)] = 0.000000e+00f;
  Conv2dOutput[(112)] = 0.000000e+00f;
  Conv2dOutput[(132)] = 0.000000e+00f;
  Conv2dOutput[(152)] = 0.000000e+00f;
  Conv2dOutput[(172)] = 0.000000e+00f;
  Conv2dOutput[(192)] = 0.000000e+00f;
  Conv2dOutput[(212)] = 0.000000e+00f;
  Conv2dOutput[(232)] = 0.000000e+00f;
  Conv2dOutput[(252)] = 0.000000e+00f;
  Conv2dOutput[(272)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(73)] = 0.000000e+00f;
  Conv2dOutput[(93)] = 0.000000e+00f;
  Conv2dOutput[(113)] = 0.000000e+00f;
  Conv2dOutput[(133)] = 0.000000e+00f;
  Conv2dOutput[(153)] = 0.000000e+00f;
  Conv2dOutput[(173)] = 0.000000e+00f;
  Conv2dOutput[(193)] = 0.000000e+00f;
  Conv2dOutput[(213)] = 0.000000e+00f;
  Conv2dOutput[(233)] = 0.000000e+00f;
  Conv2dOutput[(253)] = 0.000000e+00f;
  Conv2dOutput[(273)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(74)] = 0.000000e+00f;
  Conv2dOutput[(94)] = 0.000000e+00f;
  Conv2dOutput[(114)] = 0.000000e+00f;
  Conv2dOutput[(134)] = 0.000000e+00f;
  Conv2dOutput[(154)] = 0.000000e+00f;
  Conv2dOutput[(174)] = 0.000000e+00f;
  Conv2dOutput[(194)] = 0.000000e+00f;
  Conv2dOutput[(214)] = 0.000000e+00f;
  Conv2dOutput[(234)] = 0.000000e+00f;
  Conv2dOutput[(254)] = 0.000000e+00f;
  Conv2dOutput[(274)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(75)] = 0.000000e+00f;
  Conv2dOutput[(95)] = 0.000000e+00f;
  Conv2dOutput[(115)] = 0.000000e+00f;
  Conv2dOutput[(135)] = 0.000000e+00f;
  Conv2dOutput[(155)] = 0.000000e+00f;
  Conv2dOutput[(175)] = 0.000000e+00f;
  Conv2dOutput[(195)] = 0.000000e+00f;
  Conv2dOutput[(215)] = 0.000000e+00f;
  Conv2dOutput[(235)] = 0.000000e+00f;
  Conv2dOutput[(255)] = 0.000000e+00f;
  Conv2dOutput[(275)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(76)] = 0.000000e+00f;
  Conv2dOutput[(96)] = 0.000000e+00f;
  Conv2dOutput[(116)] = 0.000000e+00f;
  Conv2dOutput[(136)] = 0.000000e+00f;
  Conv2dOutput[(156)] = 0.000000e+00f;
  Conv2dOutput[(176)] = 0.000000e+00f;
  Conv2dOutput[(196)] = 0.000000e+00f;
  Conv2dOutput[(216)] = 0.000000e+00f;
  Conv2dOutput[(236)] = 0.000000e+00f;
  Conv2dOutput[(256)] = 0.000000e+00f;
  Conv2dOutput[(276)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(77)] = 0.000000e+00f;
  Conv2dOutput[(97)] = 0.000000e+00f;
  Conv2dOutput[(117)] = 0.000000e+00f;
  Conv2dOutput[(137)] = 0.000000e+00f;
  Conv2dOutput[(157)] = 0.000000e+00f;
  Conv2dOutput[(177)] = 0.000000e+00f;
  Conv2dOutput[(197)] = 0.000000e+00f;
  Conv2dOutput[(217)] = 0.000000e+00f;
  Conv2dOutput[(237)] = 0.000000e+00f;
  Conv2dOutput[(257)] = 0.000000e+00f;
  Conv2dOutput[(277)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(78)] = 0.000000e+00f;
  Conv2dOutput[(98)] = 0.000000e+00f;
  Conv2dOutput[(118)] = 0.000000e+00f;
  Conv2dOutput[(138)] = 0.000000e+00f;
  Conv2dOutput[(158)] = 0.000000e+00f;
  Conv2dOutput[(178)] = 0.000000e+00f;
  Conv2dOutput[(198)] = 0.000000e+00f;
  Conv2dOutput[(218)] = 0.000000e+00f;
  Conv2dOutput[(238)] = 0.000000e+00f;
  Conv2dOutput[(258)] = 0.000000e+00f;
  Conv2dOutput[(278)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(79)] = 0.000000e+00f;
  Conv2dOutput[(99)] = 0.000000e+00f;
  Conv2dOutput[(119)] = 0.000000e+00f;
  Conv2dOutput[(139)] = 0.000000e+00f;
  Conv2dOutput[(159)] = 0.000000e+00f;
  Conv2dOutput[(179)] = 0.000000e+00f;
  Conv2dOutput[(199)] = 0.000000e+00f;
  Conv2dOutput[(219)] = 0.000000e+00f;
  Conv2dOutput[(239)] = 0.000000e+00f;
  Conv2dOutput[(259)] = 0.000000e+00f;
  Conv2dOutput[(279)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[((((int)threadIdx.x) * 40))] = placeholder[((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 1))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 2))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 3))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 3))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 4))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 256))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 5))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 257))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 6))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 258))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 7))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 259))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 8))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 512))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 9))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 513))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 10))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 514))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 11))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 515))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 12))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 768))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 13))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 769))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 14))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 770))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 15))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 771))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 16))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1024))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 17))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1025))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 18))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1026))];
    }
    if (((int)threadIdx.x) < 18) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 19))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1027))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 20))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1280))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 21))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1281))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 22))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1282))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 23))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1283))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 24))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1536))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 25))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1537))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 26))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1538))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 27))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1539))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 28))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1792))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 29))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1793))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 30))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1794))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 31))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 1795))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 32))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2048))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 33))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2049))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 34))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2050))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 35))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2051))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 36))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2304))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 37))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2305))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 38))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2306))];
    }
    if (((int)threadIdx.x) < 17) {
      PaddedInput_shared[(((((int)threadIdx.x) * 40) + 39))] = placeholder[(((((((int)blockIdx.x) * 44800) + (((int)threadIdx.x) * 2560)) + (rc_outer_outer * 4)) + 2307))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 256) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 40))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 40))];
    placeholder_shared[((((int)threadIdx.x) + 80))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 80))];
    placeholder_shared[((((int)threadIdx.x) + 120))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 120))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 160))];
    placeholder_shared[((((int)threadIdx.x) + 200))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 200))];
    if (((int)threadIdx.x) < 16) {
      placeholder_shared[((((int)threadIdx.x) + 240))] = placeholder1[((((rc_outer_outer * 256) + ((int)threadIdx.x)) + 240))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(80)] = (Conv2dOutput[(80)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(100)] = (Conv2dOutput[(100)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(120)] = (Conv2dOutput[(120)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(140)] = (Conv2dOutput[(140)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(160)] = (Conv2dOutput[(160)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(180)] = (Conv2dOutput[(180)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(200)] = (Conv2dOutput[(200)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(220)] = (Conv2dOutput[(220)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(240)] = (Conv2dOutput[(240)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(260)] = (Conv2dOutput[(260)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(81)] = (Conv2dOutput[(81)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(101)] = (Conv2dOutput[(101)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(121)] = (Conv2dOutput[(121)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(141)] = (Conv2dOutput[(141)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(161)] = (Conv2dOutput[(161)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(181)] = (Conv2dOutput[(181)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(201)] = (Conv2dOutput[(201)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(221)] = (Conv2dOutput[(221)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(241)] = (Conv2dOutput[(241)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(261)] = (Conv2dOutput[(261)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(82)] = (Conv2dOutput[(82)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(102)] = (Conv2dOutput[(102)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(122)] = (Conv2dOutput[(122)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(142)] = (Conv2dOutput[(142)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(162)] = (Conv2dOutput[(162)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(182)] = (Conv2dOutput[(182)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(202)] = (Conv2dOutput[(202)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(222)] = (Conv2dOutput[(222)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(242)] = (Conv2dOutput[(242)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(262)] = (Conv2dOutput[(262)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 20))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(83)] = (Conv2dOutput[(83)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(103)] = (Conv2dOutput[(103)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 40))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(123)] = (Conv2dOutput[(123)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(143)] = (Conv2dOutput[(143)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 60))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(163)] = (Conv2dOutput[(163)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(183)] = (Conv2dOutput[(183)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 80))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(203)] = (Conv2dOutput[(203)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(223)] = (Conv2dOutput[(223)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 100))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(243)] = (Conv2dOutput[(243)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(263)] = (Conv2dOutput[(263)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 120))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(64)] = (Conv2dOutput[(64)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(84)] = (Conv2dOutput[(84)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(104)] = (Conv2dOutput[(104)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(124)] = (Conv2dOutput[(124)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(144)] = (Conv2dOutput[(144)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(164)] = (Conv2dOutput[(164)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(184)] = (Conv2dOutput[(184)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(204)] = (Conv2dOutput[(204)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(224)] = (Conv2dOutput[(224)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(244)] = (Conv2dOutput[(244)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(264)] = (Conv2dOutput[(264)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(65)] = (Conv2dOutput[(65)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(85)] = (Conv2dOutput[(85)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(105)] = (Conv2dOutput[(105)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(125)] = (Conv2dOutput[(125)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(145)] = (Conv2dOutput[(145)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(165)] = (Conv2dOutput[(165)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(185)] = (Conv2dOutput[(185)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(205)] = (Conv2dOutput[(205)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(225)] = (Conv2dOutput[(225)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(245)] = (Conv2dOutput[(245)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(265)] = (Conv2dOutput[(265)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(66)] = (Conv2dOutput[(66)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(86)] = (Conv2dOutput[(86)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(106)] = (Conv2dOutput[(106)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(126)] = (Conv2dOutput[(126)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(146)] = (Conv2dOutput[(146)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(166)] = (Conv2dOutput[(166)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(186)] = (Conv2dOutput[(186)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(206)] = (Conv2dOutput[(206)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(226)] = (Conv2dOutput[(226)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(246)] = (Conv2dOutput[(246)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(266)] = (Conv2dOutput[(266)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 140))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(67)] = (Conv2dOutput[(67)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 160))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(87)] = (Conv2dOutput[(87)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(107)] = (Conv2dOutput[(107)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 180))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(127)] = (Conv2dOutput[(127)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(147)] = (Conv2dOutput[(147)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 200))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(167)] = (Conv2dOutput[(167)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(187)] = (Conv2dOutput[(187)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 220))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(207)] = (Conv2dOutput[(207)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(227)] = (Conv2dOutput[(227)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 240))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(247)] = (Conv2dOutput[(247)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(267)] = (Conv2dOutput[(267)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 260))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(68)] = (Conv2dOutput[(68)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(88)] = (Conv2dOutput[(88)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(108)] = (Conv2dOutput[(108)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(128)] = (Conv2dOutput[(128)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(148)] = (Conv2dOutput[(148)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(168)] = (Conv2dOutput[(168)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(188)] = (Conv2dOutput[(188)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(208)] = (Conv2dOutput[(208)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(228)] = (Conv2dOutput[(228)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(248)] = (Conv2dOutput[(248)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(268)] = (Conv2dOutput[(268)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(69)] = (Conv2dOutput[(69)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(89)] = (Conv2dOutput[(89)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(109)] = (Conv2dOutput[(109)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(129)] = (Conv2dOutput[(129)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(149)] = (Conv2dOutput[(149)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(169)] = (Conv2dOutput[(169)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(189)] = (Conv2dOutput[(189)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(209)] = (Conv2dOutput[(209)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(229)] = (Conv2dOutput[(229)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(249)] = (Conv2dOutput[(249)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(269)] = (Conv2dOutput[(269)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(70)] = (Conv2dOutput[(70)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(90)] = (Conv2dOutput[(90)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(110)] = (Conv2dOutput[(110)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(130)] = (Conv2dOutput[(130)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(150)] = (Conv2dOutput[(150)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(170)] = (Conv2dOutput[(170)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(190)] = (Conv2dOutput[(190)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(210)] = (Conv2dOutput[(210)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(230)] = (Conv2dOutput[(230)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(250)] = (Conv2dOutput[(250)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(270)] = (Conv2dOutput[(270)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 280))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(71)] = (Conv2dOutput[(71)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 300))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(91)] = (Conv2dOutput[(91)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(111)] = (Conv2dOutput[(111)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 320))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(131)] = (Conv2dOutput[(131)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(151)] = (Conv2dOutput[(151)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 340))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(171)] = (Conv2dOutput[(171)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(191)] = (Conv2dOutput[(191)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 360))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(211)] = (Conv2dOutput[(211)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(231)] = (Conv2dOutput[(231)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 380))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(251)] = (Conv2dOutput[(251)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(271)] = (Conv2dOutput[(271)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 400))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(72)] = (Conv2dOutput[(72)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(92)] = (Conv2dOutput[(92)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(112)] = (Conv2dOutput[(112)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(132)] = (Conv2dOutput[(132)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(152)] = (Conv2dOutput[(152)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(172)] = (Conv2dOutput[(172)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(192)] = (Conv2dOutput[(192)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(212)] = (Conv2dOutput[(212)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(232)] = (Conv2dOutput[(232)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(252)] = (Conv2dOutput[(252)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(272)] = (Conv2dOutput[(272)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(73)] = (Conv2dOutput[(73)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(93)] = (Conv2dOutput[(93)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(113)] = (Conv2dOutput[(113)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(133)] = (Conv2dOutput[(133)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(153)] = (Conv2dOutput[(153)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(173)] = (Conv2dOutput[(173)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(193)] = (Conv2dOutput[(193)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(213)] = (Conv2dOutput[(213)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(233)] = (Conv2dOutput[(233)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(253)] = (Conv2dOutput[(253)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(273)] = (Conv2dOutput[(273)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(74)] = (Conv2dOutput[(74)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(94)] = (Conv2dOutput[(94)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(114)] = (Conv2dOutput[(114)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(134)] = (Conv2dOutput[(134)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(154)] = (Conv2dOutput[(154)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(174)] = (Conv2dOutput[(174)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(194)] = (Conv2dOutput[(194)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(214)] = (Conv2dOutput[(214)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(234)] = (Conv2dOutput[(234)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(254)] = (Conv2dOutput[(254)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(274)] = (Conv2dOutput[(274)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 420))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(75)] = (Conv2dOutput[(75)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 440))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(95)] = (Conv2dOutput[(95)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(115)] = (Conv2dOutput[(115)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 460))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(135)] = (Conv2dOutput[(135)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(155)] = (Conv2dOutput[(155)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 480))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(175)] = (Conv2dOutput[(175)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(195)] = (Conv2dOutput[(195)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 500))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(215)] = (Conv2dOutput[(215)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(235)] = (Conv2dOutput[(235)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 520))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(255)] = (Conv2dOutput[(255)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(275)] = (Conv2dOutput[(275)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 540))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(76)] = (Conv2dOutput[(76)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(96)] = (Conv2dOutput[(96)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(116)] = (Conv2dOutput[(116)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(136)] = (Conv2dOutput[(136)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(156)] = (Conv2dOutput[(156)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(176)] = (Conv2dOutput[(176)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(196)] = (Conv2dOutput[(196)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(216)] = (Conv2dOutput[(216)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(236)] = (Conv2dOutput[(236)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(256)] = (Conv2dOutput[(256)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)))]));
      Conv2dOutput[(276)] = (Conv2dOutput[(276)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 32))]));
      Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(77)] = (Conv2dOutput[(77)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(97)] = (Conv2dOutput[(97)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(117)] = (Conv2dOutput[(117)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(137)] = (Conv2dOutput[(137)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(157)] = (Conv2dOutput[(157)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(177)] = (Conv2dOutput[(177)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(197)] = (Conv2dOutput[(197)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(217)] = (Conv2dOutput[(217)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(237)] = (Conv2dOutput[(237)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(257)] = (Conv2dOutput[(257)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 1))]));
      Conv2dOutput[(277)] = (Conv2dOutput[(277)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 33))]));
      Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(78)] = (Conv2dOutput[(78)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(98)] = (Conv2dOutput[(98)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(118)] = (Conv2dOutput[(118)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(138)] = (Conv2dOutput[(138)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(158)] = (Conv2dOutput[(158)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(178)] = (Conv2dOutput[(178)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(198)] = (Conv2dOutput[(198)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(218)] = (Conv2dOutput[(218)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(238)] = (Conv2dOutput[(238)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(258)] = (Conv2dOutput[(258)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 2))]));
      Conv2dOutput[(278)] = (Conv2dOutput[(278)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 34))]));
      Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 560))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(79)] = (Conv2dOutput[(79)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 580))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(99)] = (Conv2dOutput[(99)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(119)] = (Conv2dOutput[(119)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 600))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(139)] = (Conv2dOutput[(139)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(159)] = (Conv2dOutput[(159)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 620))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(179)] = (Conv2dOutput[(179)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(199)] = (Conv2dOutput[(199)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 640))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(219)] = (Conv2dOutput[(219)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(239)] = (Conv2dOutput[(239)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 660))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
      Conv2dOutput[(259)] = (Conv2dOutput[(259)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 3))]));
      Conv2dOutput[(279)] = (Conv2dOutput[(279)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 3) * 4) + rc_outer_inner) + 680))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 7) * 4)) + 35))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 4) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 32))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 20))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 320))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 40))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 352))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 60))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 640))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 80))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 672))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 100))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 960))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 120))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 992))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 140))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 1280))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 160))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 1312))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 180))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 1600))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 200))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 1632))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 220))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 1920))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 240))] + placeholder2[((((((int)threadIdx.x) & 7) * 4) + ax3_inner))]), 0.000000e+00f);
      T_relu[(((((((((int)blockIdx.x) * 11200) + (ax1_inner * 2240)) + ((((int)threadIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 7) * 4)) + ax3_inner) + 1952))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 260))] + placeholder2[(((((((int)threadIdx.x) & 7) * 4) + ax3_inner) + 32))]), 0.000000e+00f);
    }
  }
}


