
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute, float* __restrict__ placeholder2) {
  float DepthwiseConv2d[84];
  __shared__ float PaddedInput_shared[5376];
  __shared__ float placeholder_shared[144];
  for (int c_outer_inner_init = 0; c_outer_inner_init < 4; ++c_outer_inner_init) {
    for (int c_inner_init = 0; c_inner_init < 3; ++c_inner_init) {
      DepthwiseConv2d[(((c_outer_inner_init * 3) + c_inner_init))] = 0.000000e+00f;
      DepthwiseConv2d[((((c_outer_inner_init * 3) + c_inner_init) + 12))] = 0.000000e+00f;
      DepthwiseConv2d[((((c_outer_inner_init * 3) + c_inner_init) + 24))] = 0.000000e+00f;
      DepthwiseConv2d[((((c_outer_inner_init * 3) + c_inner_init) + 36))] = 0.000000e+00f;
      DepthwiseConv2d[((((c_outer_inner_init * 3) + c_inner_init) + 48))] = 0.000000e+00f;
      DepthwiseConv2d[((((c_outer_inner_init * 3) + c_inner_init) + 60))] = 0.000000e+00f;
      DepthwiseConv2d[((((c_outer_inner_init * 3) + c_inner_init) + 72))] = 0.000000e+00f;
    }
  }
  for (int dj_outer_outer = 0; dj_outer_outer < 3; ++dj_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 96; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)))] = (((((336 <= ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x))) && (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) < 5040)) && (1 <= ((((((int)blockIdx.x) >> 3) * 7) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 336) / 48)) + dj_outer_outer))) && (((((((int)blockIdx.x) >> 3) * 7) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 336) / 48)) + dj_outer_outer) < 15)) ? placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) / 336) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 336) / 48) * 384)) + (dj_outer_outer * 384)) + ((((int)blockIdx.x) & 7) * 48)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 56) + ((int)threadIdx.x)) % 48)) - 5760))] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) < 144) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)))] = placeholder1[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) / 48) * 1152) + (dj_outer_outer * 384)) + ((((int)blockIdx.x) & 7) * 48)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 56) + ((int)threadIdx.x)) % 48)))];
      }
    }
    __syncthreads();
    for (int c_outer_inner = 0; c_outer_inner < 4; ++c_outer_inner) {
      for (int di_inner = 0; di_inner < 3; ++di_inner) {
        for (int c_inner = 0; c_inner < 3; ++c_inner) {
          DepthwiseConv2d[(((c_outer_inner * 3) + c_inner))] = (DepthwiseConv2d[(((c_outer_inner * 3) + c_inner))] + (PaddedInput_shared[(((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
          DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 12))] = (DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 12))] + (PaddedInput_shared[((((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner) + 672))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
          DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 24))] = (DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 24))] + (PaddedInput_shared[((((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner) + 1344))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
          DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 36))] = (DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 36))] + (PaddedInput_shared[((((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner) + 2016))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
          DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 48))] = (DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 48))] + (PaddedInput_shared[((((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner) + 2688))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
          DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 60))] = (DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 60))] + (PaddedInput_shared[((((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner) + 3360))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
          DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 72))] = (DepthwiseConv2d[((((c_outer_inner * 3) + c_inner) + 72))] + (PaddedInput_shared[((((((di_inner * 336) + (((int)threadIdx.x) * 12)) + (c_outer_inner * 3)) + c_inner) + 4032))] * placeholder_shared[(((((di_inner * 48) + ((((int)threadIdx.x) & 3) * 12)) + (c_outer_inner * 3)) + c_inner))]));
        }
      }
    }
  }
  for (int i3_inner = 0; i3_inner < 12; ++i3_inner) {
    compute[((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))] = max(min((DepthwiseConv2d[(i3_inner)] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner) + 10752))] = max(min((DepthwiseConv2d[((i3_inner + 12))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner) + 21504))] = max(min((DepthwiseConv2d[((i3_inner + 24))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner) + 32256))] = max(min((DepthwiseConv2d[((i3_inner + 36))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner) + 43008))] = max(min((DepthwiseConv2d[((i3_inner + 48))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner) + 53760))] = max(min((DepthwiseConv2d[((i3_inner + 60))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
    compute[(((((((((((int)threadIdx.x) / 28) * 5376) + ((((int)blockIdx.x) >> 3) * 2688)) + (((((int)threadIdx.x) % 28) >> 2) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner) + 64512))] = max(min((DepthwiseConv2d[((i3_inner + 72))] + placeholder2[(((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) & 3) * 12)) + i3_inner))]), 6.000000e+00f), 0.000000e+00f);
  }
}


