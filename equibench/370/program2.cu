
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float Conv2dOutput[4];
  __shared__ float PaddedInput_shared[1568];
  __shared__ float placeholder_shared[2560];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 21; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 784) {
      PaddedInput_shared[((((int)threadIdx.x) * 2))] = placeholder[((((((((int)blockIdx.x) >> 1) * 32928) + ((((int)threadIdx.x) >> 4) * 672)) + (rc_outer_outer * 32)) + ((((int)threadIdx.x) & 15) * 2)))];
    }
    if (((int)threadIdx.x) < 784) {
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder[((((((((int)blockIdx.x) >> 1) * 32928) + ((((((int)threadIdx.x) * 2) + 1) >> 5) * 672)) + (rc_outer_outer * 32)) + (((((int)threadIdx.x) * 2) + 1) & 31)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 5120) + ((((int)threadIdx.x) / 80) * 160)) + ((((int)blockIdx.x) & 1) * 80)) + (((int)threadIdx.x) % 80)))];
    placeholder_shared[((((int)threadIdx.x) + 980))] = placeholder1[(((((rc_outer_outer * 5120) + (((((int)threadIdx.x) + 980) / 80) * 160)) + ((((int)blockIdx.x) & 1) * 80)) + ((((int)threadIdx.x) + 20) % 80)))];
    if (((int)threadIdx.x) < 600) {
      placeholder_shared[((((int)threadIdx.x) + 1960))] = placeholder1[(((((rc_outer_outer * 5120) + (((((int)threadIdx.x) + 1960) / 80) * 160)) + ((((int)blockIdx.x) & 1) * 80)) + ((((int)threadIdx.x) + 40) % 80)))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((int)threadIdx.x) / 20) * 32))] * placeholder_shared[(((((int)threadIdx.x) % 20) * 4))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((int)threadIdx.x) / 20) * 32))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 80))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 81))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 160))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 161))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 240))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 241))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 320))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 321))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 400))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 401))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 480))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 481))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 560))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 561))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 640))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 641))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 720))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 721))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 10))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 800))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 10))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 801))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 11))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 880))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 11))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 881))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 960))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 961))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 13))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1040))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 13))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1041))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 14))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1120))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 14))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1121))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 15))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1200))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 15))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1201))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1280))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1281))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1360))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1361))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 18))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1440))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 18))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1441))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 19))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1520))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 19))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1521))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1600))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1601))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 21))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1680))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 21))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1681))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 22))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1760))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 22))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1761))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 23))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1840))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 23))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1841))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1920))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1921))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 25))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2000))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 25))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2001))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 26))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2080))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 26))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2081))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 27))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2160))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 27))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2161))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2240))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2241))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 29))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2320))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 29))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2321))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 30))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2400))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 30))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2401))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 31))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2480))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 31))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2481))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((int)threadIdx.x) / 20) * 32))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((int)threadIdx.x) / 20) * 32))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 3))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 82))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 83))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 162))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 163))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 242))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 243))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 322))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 4))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 323))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 402))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 5))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 403))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 482))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 6))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 483))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 562))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 7))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 563))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 642))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 8))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 643))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 722))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 723))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 10))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 802))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 10))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 803))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 11))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 882))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 11))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 883))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 962))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 12))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 963))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 13))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1042))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 13))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1043))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 14))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1122))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 14))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1123))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 15))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1202))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 15))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1203))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1282))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1283))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1362))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1363))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 18))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1442))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 18))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1443))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 19))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1522))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 19))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1523))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1602))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 20))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1603))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 21))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1682))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 21))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1683))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 22))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1762))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 22))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1763))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 23))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1842))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 23))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1843))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1922))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 24))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1923))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 25))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2002))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 25))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2003))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 26))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2082))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 26))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2083))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 27))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2162))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 27))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2163))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2242))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 28))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2243))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 29))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2322))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 29))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2323))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 30))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2402))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 30))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2403))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 31))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2482))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + 31))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2483))]));
  }
  for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
    T_add[(((((((((int)blockIdx.x) >> 1) * 7840) + ((((int)threadIdx.x) / 20) * 160)) + ((((int)blockIdx.x) & 1) * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax3_inner))] = (Conv2dOutput[(ax3_inner)] + placeholder2[(((((((int)blockIdx.x) & 1) * 80) + ((((int)threadIdx.x) % 20) * 4)) + ax3_inner))]);
  }
}


