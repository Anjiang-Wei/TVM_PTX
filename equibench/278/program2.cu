
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv3dOutput[32];
  __shared__ float PaddedInput_shared[10080];
  __shared__ float placeholder_shared[32];
  Conv3dOutput[(0)] = 0.000000e+00f;
  Conv3dOutput[(16)] = 0.000000e+00f;
  Conv3dOutput[(2)] = 0.000000e+00f;
  Conv3dOutput[(18)] = 0.000000e+00f;
  Conv3dOutput[(4)] = 0.000000e+00f;
  Conv3dOutput[(20)] = 0.000000e+00f;
  Conv3dOutput[(6)] = 0.000000e+00f;
  Conv3dOutput[(22)] = 0.000000e+00f;
  Conv3dOutput[(8)] = 0.000000e+00f;
  Conv3dOutput[(24)] = 0.000000e+00f;
  Conv3dOutput[(10)] = 0.000000e+00f;
  Conv3dOutput[(26)] = 0.000000e+00f;
  Conv3dOutput[(12)] = 0.000000e+00f;
  Conv3dOutput[(28)] = 0.000000e+00f;
  Conv3dOutput[(14)] = 0.000000e+00f;
  Conv3dOutput[(30)] = 0.000000e+00f;
  Conv3dOutput[(1)] = 0.000000e+00f;
  Conv3dOutput[(17)] = 0.000000e+00f;
  Conv3dOutput[(3)] = 0.000000e+00f;
  Conv3dOutput[(19)] = 0.000000e+00f;
  Conv3dOutput[(5)] = 0.000000e+00f;
  Conv3dOutput[(21)] = 0.000000e+00f;
  Conv3dOutput[(7)] = 0.000000e+00f;
  Conv3dOutput[(23)] = 0.000000e+00f;
  Conv3dOutput[(9)] = 0.000000e+00f;
  Conv3dOutput[(25)] = 0.000000e+00f;
  Conv3dOutput[(11)] = 0.000000e+00f;
  Conv3dOutput[(27)] = 0.000000e+00f;
  Conv3dOutput[(13)] = 0.000000e+00f;
  Conv3dOutput[(29)] = 0.000000e+00f;
  Conv3dOutput[(15)] = 0.000000e+00f;
  Conv3dOutput[(31)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer < 79; ++ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) < 630) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10080) {
          PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)))] = placeholder[((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10079) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 1))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 1))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10078) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 2))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 2))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10077) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 3))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 3))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10076) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 4))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 4))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10075) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 5))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 5))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10074) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 6))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 6))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10073) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 7))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 7))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10072) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 8))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 8))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10071) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 9))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 9))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10070) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 10))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 10))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10069) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 11))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 11))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10068) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 12))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 12))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10067) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 13))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 13))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10066) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 14))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 14))];
        }
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) < 10065) {
          PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 128) + (((int)threadIdx.x) * 16)) + 15))] = placeholder[(((((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) / 315) * 4194304) + ((((int)blockIdx.x) >> 10) * 524288)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 315) / 21) * 32768)) + (((((int)blockIdx.x) & 1023) >> 7) * 4096)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 21) / 3) * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_ax4_fused_outer_outer * 8) + ((int)threadIdx.x)) % 3) * 64)) + (rc_outer_outer * 16)) + 15))];
        }
      }
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 63) * 2)) + (((int)threadIdx.x) & 1)))];
    placeholder_shared[((((int)threadIdx.x) + 8))] = placeholder1[((((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 63) * 2)) + (((int)threadIdx.x) & 1)) + 512))];
    placeholder_shared[((((int)threadIdx.x) + 16))] = placeholder1[((((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 63) * 2)) + (((int)threadIdx.x) & 1)) + 1024))];
    placeholder_shared[((((int)threadIdx.x) + 24))] = placeholder1[((((((rc_outer_outer * 2048) + ((((int)threadIdx.x) >> 1) * 128)) + ((((int)blockIdx.x) & 63) * 2)) + (((int)threadIdx.x) & 1)) + 1536))];
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 8; ++rc_outer_inner) {
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(16)] = (Conv3dOutput[(16)] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 672))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(18)] = (Conv3dOutput[(18)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 672))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(4)] = (Conv3dOutput[(4)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1344))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(20)] = (Conv3dOutput[(20)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1344))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(6)] = (Conv3dOutput[(6)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2016))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(22)] = (Conv3dOutput[(22)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2016))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(8)] = (Conv3dOutput[(8)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2688))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(24)] = (Conv3dOutput[(24)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2688))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(10)] = (Conv3dOutput[(10)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3360))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(26)] = (Conv3dOutput[(26)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3360))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(12)] = (Conv3dOutput[(12)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4032))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(28)] = (Conv3dOutput[(28)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4032))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(14)] = (Conv3dOutput[(14)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4704))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(30)] = (Conv3dOutput[(30)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4704))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(0)] = (Conv3dOutput[(0)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(16)] = (Conv3dOutput[(16)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(2)] = (Conv3dOutput[(2)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 673))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(18)] = (Conv3dOutput[(18)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 673))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(4)] = (Conv3dOutput[(4)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1345))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(20)] = (Conv3dOutput[(20)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1345))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(6)] = (Conv3dOutput[(6)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2017))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(22)] = (Conv3dOutput[(22)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2017))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(8)] = (Conv3dOutput[(8)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2689))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(24)] = (Conv3dOutput[(24)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2689))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(10)] = (Conv3dOutput[(10)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3361))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(26)] = (Conv3dOutput[(26)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3361))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(12)] = (Conv3dOutput[(12)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4033))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(28)] = (Conv3dOutput[(28)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4033))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(14)] = (Conv3dOutput[(14)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4705))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(30)] = (Conv3dOutput[(30)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4705))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 96))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(17)] = (Conv3dOutput[(17)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 96))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 768))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(19)] = (Conv3dOutput[(19)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 768))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(5)] = (Conv3dOutput[(5)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1440))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(21)] = (Conv3dOutput[(21)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1440))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(7)] = (Conv3dOutput[(7)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2112))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(23)] = (Conv3dOutput[(23)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2112))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(9)] = (Conv3dOutput[(9)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2784))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(25)] = (Conv3dOutput[(25)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2784))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(11)] = (Conv3dOutput[(11)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3456))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(27)] = (Conv3dOutput[(27)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3456))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(13)] = (Conv3dOutput[(13)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4128))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(29)] = (Conv3dOutput[(29)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4128))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(15)] = (Conv3dOutput[(15)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4800))] * placeholder_shared[((rc_outer_inner * 4))]));
      Conv3dOutput[(31)] = (Conv3dOutput[(31)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4800))] * placeholder_shared[(((rc_outer_inner * 4) + 1))]));
      Conv3dOutput[(1)] = (Conv3dOutput[(1)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 97))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(17)] = (Conv3dOutput[(17)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 97))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(3)] = (Conv3dOutput[(3)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 769))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(19)] = (Conv3dOutput[(19)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 769))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(5)] = (Conv3dOutput[(5)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1441))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(21)] = (Conv3dOutput[(21)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 1441))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(7)] = (Conv3dOutput[(7)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2113))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(23)] = (Conv3dOutput[(23)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2113))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(9)] = (Conv3dOutput[(9)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2785))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(25)] = (Conv3dOutput[(25)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 2785))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(11)] = (Conv3dOutput[(11)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3457))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(27)] = (Conv3dOutput[(27)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 3457))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(13)] = (Conv3dOutput[(13)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4129))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(29)] = (Conv3dOutput[(29)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4129))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
      Conv3dOutput[(15)] = (Conv3dOutput[(15)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4801))] * placeholder_shared[(((rc_outer_inner * 4) + 2))]));
      Conv3dOutput[(31)] = (Conv3dOutput[(31)] + (PaddedInput_shared[(((((((((int)threadIdx.x) >> 2) * 5040) + (((((int)threadIdx.x) & 3) >> 1) * 192)) + ((((int)threadIdx.x) & 1) * 32)) + (rc_outer_inner * 2)) + 4801))] * placeholder_shared[(((rc_outer_inner * 4) + 3))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 2; ++ax2_inner) {
      T_add[(((((((((((((int)threadIdx.x) >> 2) * 1048576) + ((((int)blockIdx.x) >> 10) * 131072)) + (ax1_inner * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ((((int)blockIdx.x) & 63) * 2)))] = ((Conv3dOutput[(((ax1_inner * 2) + ax2_inner))] * placeholder2[(((((int)blockIdx.x) & 63) * 2))]) + placeholder3[(((((int)blockIdx.x) & 63) * 2))]);
      T_add[((((((((((((((int)threadIdx.x) >> 2) * 1048576) + ((((int)blockIdx.x) >> 10) * 131072)) + (ax1_inner * 16384)) + (((((int)blockIdx.x) & 1023) >> 7) * 2048)) + (((((int)threadIdx.x) & 3) >> 1) * 1024)) + (ax2_inner * 512)) + (((((int)blockIdx.x) & 127) >> 6) * 256)) + ((((int)threadIdx.x) & 1) * 128)) + ((((int)blockIdx.x) & 63) * 2)) + 1))] = ((Conv3dOutput[((((ax1_inner * 2) + ax2_inner) + 16))] * placeholder2[((((((int)blockIdx.x) & 63) * 2) + 1))]) + placeholder3[((((((int)blockIdx.x) & 63) * 2) + 1))]);
    }
  }
}


