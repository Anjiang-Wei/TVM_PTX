
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[896];
  __shared__ float PaddedInput_shared[448];
  __shared__ float placeholder_shared[512];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 8; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 14; ++xx_outer_inner_init) {
      Conv2dOutput[(((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 224))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 448))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 672))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 1))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 225))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 449))] = 0.000000e+00f;
      Conv2dOutput[((((yy_outer_inner_init * 28) + (xx_outer_inner_init * 2)) + 673))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 64; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[((((int)threadIdx.x) * 2))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 57344) + ((((int)threadIdx.x) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + ((((int)threadIdx.x) % 28) * 128)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 57344) + ((((int)threadIdx.x) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + ((((int)threadIdx.x) % 28) * 128)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 128))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 57344) + (((((int)threadIdx.x) + 64) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + (((((int)threadIdx.x) + 8) % 28) * 128)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 129))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 57344) + (((((int)threadIdx.x) + 64) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + (((((int)threadIdx.x) + 8) % 28) * 128)) + (rc_outer_outer * 2)) + 1))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 256))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 57344) + (((((int)threadIdx.x) + 128) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + (((((int)threadIdx.x) + 16) % 28) * 128)) + (rc_outer_outer * 2)))];
    PaddedInput_shared[(((((int)threadIdx.x) * 2) + 257))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 57344) + (((((int)threadIdx.x) + 128) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + (((((int)threadIdx.x) + 16) % 28) * 128)) + (rc_outer_outer * 2)) + 1))];
    if (((int)threadIdx.x) < 32) {
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 384))] = placeholder[(((((((((int)blockIdx.x) >> 1) * 57344) + (((((int)threadIdx.x) + 192) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + (((((int)threadIdx.x) + 24) % 28) * 128)) + (rc_outer_outer * 2)))];
      PaddedInput_shared[(((((int)threadIdx.x) * 2) + 385))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 57344) + (((((int)threadIdx.x) + 192) / 28) * 7168)) + ((((int)blockIdx.x) & 1) * 3584)) + (((((int)threadIdx.x) + 24) % 28) * 128)) + (rc_outer_outer * 2)) + 1))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((rc_outer_outer * 512) + ((int)threadIdx.x)))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 64))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 128))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 192))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 256))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 320))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 384))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((rc_outer_outer * 512) + ((int)threadIdx.x)) + 448))];
    __syncthreads();
    for (int yy_outer_inner = 0; yy_outer_inner < 8; ++yy_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 14; ++xx_outer_inner) {
        Conv2dOutput[(((yy_outer_inner * 28) + (xx_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 28) + (xx_outer_inner * 2)))] + (PaddedInput_shared[(((yy_outer_inner * 56) + (xx_outer_inner * 2)))] * placeholder_shared[((((int)threadIdx.x) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 224))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 224))] + (PaddedInput_shared[(((yy_outer_inner * 56) + (xx_outer_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 448))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 448))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 28))] * placeholder_shared[((((int)threadIdx.x) * 2))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 672))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 672))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 28))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 128))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 1))] + (PaddedInput_shared[(((yy_outer_inner * 56) + (xx_outer_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 225))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 225))] + (PaddedInput_shared[(((yy_outer_inner * 56) + (xx_outer_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 449))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 449))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 28))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 1))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 673))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 673))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 28))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 129))]));
        Conv2dOutput[(((yy_outer_inner * 28) + (xx_outer_inner * 2)))] = (Conv2dOutput[(((yy_outer_inner * 28) + (xx_outer_inner * 2)))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 224))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 224))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 448))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 448))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 29))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 256))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 672))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 672))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 29))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 384))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 1))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 1))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 225))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 225))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 449))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 449))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 29))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 257))]));
        Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 673))] = (Conv2dOutput[((((yy_outer_inner * 28) + (xx_outer_inner * 2)) + 673))] + (PaddedInput_shared[((((yy_outer_inner * 56) + (xx_outer_inner * 2)) + 29))] * placeholder_shared[(((((int)threadIdx.x) * 2) + 385))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max(((Conv2dOutput[((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]) + placeholder3[((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 128))] = max(((Conv2dOutput[(((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner) + 224))] + placeholder2[((((((int)threadIdx.x) * 2) + ax3_inner) + 128))]) + placeholder3[(((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 128))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 3584))] = max(((Conv2dOutput[(((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner) + 448))] + placeholder2[(((((int)threadIdx.x) * 2) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 3584))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 3712))] = max(((Conv2dOutput[(((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner) + 672))] + placeholder2[((((((int)threadIdx.x) * 2) + ax3_inner) + 128))]) + placeholder3[(((((((((((int)blockIdx.x) >> 1) * 114688) + (ax1_inner * 14336)) + ((((int)blockIdx.x) & 1) * 7168)) + (ax2_inner * 256)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 3712))]), 0.000000e+00f);
      }
    }
  }
}


