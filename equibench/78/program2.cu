
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2, float* __restrict__ placeholder3) {
  float Conv2dOutput[128];
  __shared__ float PaddedInput_shared[128];
  __shared__ float placeholder_shared[160];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 8; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 4))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 64))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 32))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 96))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 65))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 33))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 97))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 66))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 34))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 98))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 67))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 35))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 99))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 92; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 23552) + ((((int)threadIdx.x) >> 4) * 2944)) + ((((int)blockIdx.x) & 1) * 1472)) + (((((int)threadIdx.x) & 15) >> 1) * 184)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 40))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 23552) + (((((int)threadIdx.x) + 40) >> 4) * 2944)) + ((((int)blockIdx.x) & 1) * 1472)) + ((((((int)threadIdx.x) >> 1) + 4) & 7) * 184)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    PaddedInput_shared[((((int)threadIdx.x) + 80))] = placeholder[(((((((((((int)blockIdx.x) >> 1) * 23552) + ((((int)threadIdx.x) >> 4) * 2944)) + ((((int)blockIdx.x) & 1) * 1472)) + (((((int)threadIdx.x) & 15) >> 1) * 184)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)) + 14720))];
    if (((int)threadIdx.x) < 8) {
      PaddedInput_shared[((((int)threadIdx.x) + 120))] = placeholder[((((((((((int)blockIdx.x) >> 1) * 23552) + (((((int)threadIdx.x) + 120) >> 4) * 2944)) + ((((int)blockIdx.x) & 1) * 1472)) + (((((int)threadIdx.x) >> 1) + 4) * 184)) + (rc_outer_outer * 2)) + (((int)threadIdx.x) & 1)))];
    }
    ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + (((rc_outer_outer * 160) + (((int)threadIdx.x) * 4)))))[0];
    __syncthreads();
    for (int xx_outer_inner = 0; xx_outer_inner < 8; ++xx_outer_inner) {
      Conv2dOutput[((xx_outer_inner * 4))] = (Conv2dOutput[((xx_outer_inner * 4))] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)))] * placeholder_shared[(((((int)threadIdx.x) % 20) * 4))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 64))] = (Conv2dOutput[(((xx_outer_inner * 4) + 64))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 64))] * placeholder_shared[(((((int)threadIdx.x) % 20) * 4))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 32))] = (Conv2dOutput[(((xx_outer_inner * 4) + 32))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 16))] * placeholder_shared[(((((int)threadIdx.x) % 20) * 4))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 96))] = (Conv2dOutput[(((xx_outer_inner * 4) + 96))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 80))] * placeholder_shared[(((((int)threadIdx.x) % 20) * 4))]));
      Conv2dOutput[((xx_outer_inner * 4))] = (Conv2dOutput[((xx_outer_inner * 4))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 80))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 64))] = (Conv2dOutput[(((xx_outer_inner * 4) + 64))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 80))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 32))] = (Conv2dOutput[(((xx_outer_inner * 4) + 32))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 80))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 96))] = (Conv2dOutput[(((xx_outer_inner * 4) + 96))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 81))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 80))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 1))] = (Conv2dOutput[(((xx_outer_inner * 4) + 1))] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 65))] = (Conv2dOutput[(((xx_outer_inner * 4) + 65))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 33))] = (Conv2dOutput[(((xx_outer_inner * 4) + 33))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 97))] = (Conv2dOutput[(((xx_outer_inner * 4) + 97))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 80))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 1))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 1))] = (Conv2dOutput[(((xx_outer_inner * 4) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 81))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 65))] = (Conv2dOutput[(((xx_outer_inner * 4) + 65))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 81))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 33))] = (Conv2dOutput[(((xx_outer_inner * 4) + 33))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 81))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 97))] = (Conv2dOutput[(((xx_outer_inner * 4) + 97))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 81))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 81))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 2))] = (Conv2dOutput[(((xx_outer_inner * 4) + 2))] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 66))] = (Conv2dOutput[(((xx_outer_inner * 4) + 66))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 34))] = (Conv2dOutput[(((xx_outer_inner * 4) + 34))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 98))] = (Conv2dOutput[(((xx_outer_inner * 4) + 98))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 80))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 2))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 2))] = (Conv2dOutput[(((xx_outer_inner * 4) + 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 82))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 66))] = (Conv2dOutput[(((xx_outer_inner * 4) + 66))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 82))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 34))] = (Conv2dOutput[(((xx_outer_inner * 4) + 34))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 82))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 98))] = (Conv2dOutput[(((xx_outer_inner * 4) + 98))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 81))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 82))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 3))] = (Conv2dOutput[(((xx_outer_inner * 4) + 3))] + (PaddedInput_shared[((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 3))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 67))] = (Conv2dOutput[(((xx_outer_inner * 4) + 67))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 64))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 3))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 35))] = (Conv2dOutput[(((xx_outer_inner * 4) + 35))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 16))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 3))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 99))] = (Conv2dOutput[(((xx_outer_inner * 4) + 99))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 80))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 3))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 3))] = (Conv2dOutput[(((xx_outer_inner * 4) + 3))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 83))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 67))] = (Conv2dOutput[(((xx_outer_inner * 4) + 67))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 65))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 83))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 35))] = (Conv2dOutput[(((xx_outer_inner * 4) + 35))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 17))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 83))]));
      Conv2dOutput[(((xx_outer_inner * 4) + 99))] = (Conv2dOutput[(((xx_outer_inner * 4) + 99))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 20) * 32) + (xx_outer_inner * 2)) + 81))] * placeholder_shared[((((((int)threadIdx.x) % 20) * 4) + 83))]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 8; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_add[(((((((((((int)blockIdx.x) >> 1) * 10240) + ((((int)threadIdx.x) / 20) * 2560)) + (ax1_inner * 1280)) + ((((int)blockIdx.x) & 1) * 640)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax3_inner))] = ((Conv2dOutput[((((ax1_inner * 32) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)threadIdx.x) % 20) * 4) + ax3_inner))]) + placeholder3[(((((((((((int)blockIdx.x) >> 1) * 10240) + ((((int)threadIdx.x) / 20) * 2560)) + (ax1_inner * 1280)) + ((((int)blockIdx.x) & 1) * 640)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax3_inner))]);
        T_add[((((((((((((int)blockIdx.x) >> 1) * 10240) + ((((int)threadIdx.x) / 20) * 2560)) + (ax1_inner * 1280)) + ((((int)blockIdx.x) & 1) * 640)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax3_inner) + 5120))] = ((Conv2dOutput[(((((ax1_inner * 32) + (ax2_inner * 4)) + ax3_inner) + 64))] + placeholder2[((((((int)threadIdx.x) % 20) * 4) + ax3_inner))]) + placeholder3[((((((((((((int)blockIdx.x) >> 1) * 10240) + ((((int)threadIdx.x) / 20) * 2560)) + (ax1_inner * 1280)) + ((((int)blockIdx.x) & 1) * 640)) + (ax2_inner * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax3_inner) + 5120))]);
      }
    }
  }
}


