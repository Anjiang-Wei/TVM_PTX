
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_divide, float* __restrict__ placeholder2) {
  float Conv2dOutput[120];
  __shared__ float PaddedInput_shared[16];
  __shared__ float placeholder_shared[1920];
  for (int ff_outer_inner_init = 0; ff_outer_inner_init < 2; ++ff_outer_inner_init) {
    for (int ff_inner_init = 0; ff_inner_init < 60; ++ff_inner_init) {
      Conv2dOutput[(((ff_outer_inner_init * 60) + ff_inner_init))] = 0.000000e+00f;
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer)] = placeholder[(((rc_outer_outer * 16) + ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 1920; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1)] = placeholder1[(((rc_outer_outer * 1920) + ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1))];
    }
    __syncthreads();
    for (int ff_outer_inner = 0; ff_outer_inner < 2; ++ff_outer_inner) {
      for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
        for (int ff_inner = 0; ff_inner < 60; ++ff_inner) {
          Conv2dOutput[(((ff_outer_inner * 60) + ff_inner))] = (Conv2dOutput[(((ff_outer_inner * 60) + ff_inner))] + (PaddedInput_shared[(rc_inner)] * placeholder_shared[((((rc_inner * 120) + (ff_outer_inner * 60)) + ff_inner))]));
        }
      }
    }
  }
  for (int ax3_inner = 0; ax3_inner < 120; ++ax3_inner) {
    T_divide[(ax3_inner)] = (max(min(((Conv2dOutput[(ax3_inner)] + placeholder2[(ax3_inner)]) + 3.000000e+00f), 6.000000e+00f), 0.000000e+00f) * 1.666667e-01f);
  }
}


