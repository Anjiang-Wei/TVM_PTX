
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[2];
  __shared__ float placeholder_d_shared[16];
  __shared__ float placeholder_shared[800];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 16) {
      placeholder_d_shared[(((int)threadIdx.x))] = placeholder[(((((((int)threadIdx.x) >> 2) * 512) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    }
    if (((int)threadIdx.x) < 200) {
      ((float4*)(placeholder_shared + ((((int)threadIdx.x) * 4))))[0] = ((float4*)(placeholder1 + ((((((int)blockIdx.x) * 102400) + (((int)threadIdx.x) * 512)) + (k_outer_outer * 4)))))[0];
    }
    __syncthreads();
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(((((int)threadIdx.x) / 200) * 4))] * placeholder_shared[(((((int)threadIdx.x) % 200) * 4))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 8))] * placeholder_shared[(((((int)threadIdx.x) % 200) * 4))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 1))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 1))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 9))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 1))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 2))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 2))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 10))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 2))]));
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 3))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 3))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) / 200) * 4) + 11))] * placeholder_shared[((((((int)threadIdx.x) % 200) * 4) + 3))]));
  }
  T_add[(((((((int)threadIdx.x) / 200) * 400) + (((int)blockIdx.x) * 200)) + (((int)threadIdx.x) % 200)))] = (T_dense[(0)] + placeholder2[(((((int)blockIdx.x) * 200) + (((int)threadIdx.x) % 200)))]);
  T_add[((((((((int)threadIdx.x) / 200) * 400) + (((int)blockIdx.x) * 200)) + (((int)threadIdx.x) % 200)) + 800))] = (T_dense[(1)] + placeholder2[(((((int)blockIdx.x) * 200) + (((int)threadIdx.x) % 200)))]);
}


