
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
  float T_dense[64];
  __shared__ float placeholder_d_shared[2];
  __shared__ float placeholder_shared[1024];
  T_dense[(0)] = 0.000000e+00f;
  T_dense[(1)] = 0.000000e+00f;
  T_dense[(2)] = 0.000000e+00f;
  T_dense[(3)] = 0.000000e+00f;
  T_dense[(4)] = 0.000000e+00f;
  T_dense[(5)] = 0.000000e+00f;
  T_dense[(6)] = 0.000000e+00f;
  T_dense[(7)] = 0.000000e+00f;
  T_dense[(32)] = 0.000000e+00f;
  T_dense[(33)] = 0.000000e+00f;
  T_dense[(34)] = 0.000000e+00f;
  T_dense[(35)] = 0.000000e+00f;
  T_dense[(36)] = 0.000000e+00f;
  T_dense[(37)] = 0.000000e+00f;
  T_dense[(38)] = 0.000000e+00f;
  T_dense[(39)] = 0.000000e+00f;
  T_dense[(8)] = 0.000000e+00f;
  T_dense[(9)] = 0.000000e+00f;
  T_dense[(10)] = 0.000000e+00f;
  T_dense[(11)] = 0.000000e+00f;
  T_dense[(12)] = 0.000000e+00f;
  T_dense[(13)] = 0.000000e+00f;
  T_dense[(14)] = 0.000000e+00f;
  T_dense[(15)] = 0.000000e+00f;
  T_dense[(40)] = 0.000000e+00f;
  T_dense[(41)] = 0.000000e+00f;
  T_dense[(42)] = 0.000000e+00f;
  T_dense[(43)] = 0.000000e+00f;
  T_dense[(44)] = 0.000000e+00f;
  T_dense[(45)] = 0.000000e+00f;
  T_dense[(46)] = 0.000000e+00f;
  T_dense[(47)] = 0.000000e+00f;
  T_dense[(16)] = 0.000000e+00f;
  T_dense[(17)] = 0.000000e+00f;
  T_dense[(18)] = 0.000000e+00f;
  T_dense[(19)] = 0.000000e+00f;
  T_dense[(20)] = 0.000000e+00f;
  T_dense[(21)] = 0.000000e+00f;
  T_dense[(22)] = 0.000000e+00f;
  T_dense[(23)] = 0.000000e+00f;
  T_dense[(48)] = 0.000000e+00f;
  T_dense[(49)] = 0.000000e+00f;
  T_dense[(50)] = 0.000000e+00f;
  T_dense[(51)] = 0.000000e+00f;
  T_dense[(52)] = 0.000000e+00f;
  T_dense[(53)] = 0.000000e+00f;
  T_dense[(54)] = 0.000000e+00f;
  T_dense[(55)] = 0.000000e+00f;
  T_dense[(24)] = 0.000000e+00f;
  T_dense[(25)] = 0.000000e+00f;
  T_dense[(26)] = 0.000000e+00f;
  T_dense[(27)] = 0.000000e+00f;
  T_dense[(28)] = 0.000000e+00f;
  T_dense[(29)] = 0.000000e+00f;
  T_dense[(30)] = 0.000000e+00f;
  T_dense[(31)] = 0.000000e+00f;
  T_dense[(56)] = 0.000000e+00f;
  T_dense[(57)] = 0.000000e+00f;
  T_dense[(58)] = 0.000000e+00f;
  T_dense[(59)] = 0.000000e+00f;
  T_dense[(60)] = 0.000000e+00f;
  T_dense[(61)] = 0.000000e+00f;
  T_dense[(62)] = 0.000000e+00f;
  T_dense[(63)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 100; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[((((int)threadIdx.x) * 2))] = placeholder[(((((((int)blockIdx.x) / 25) * 200) + (((int)threadIdx.x) * 200)) + k_outer_outer))];
    }
    if (((int)threadIdx.x) < 1) {
      placeholder_d_shared[(((((int)threadIdx.x) * 2) + 1))] = placeholder[((((((((int)blockIdx.x) / 25) * 200) + (((int)threadIdx.x) * 200)) + k_outer_outer) + 100))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer))];
    placeholder_shared[((((int)threadIdx.x) + 32))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 3200))];
    placeholder_shared[((((int)threadIdx.x) + 64))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 6400))];
    placeholder_shared[((((int)threadIdx.x) + 96))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 9600))];
    placeholder_shared[((((int)threadIdx.x) + 128))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 12800))];
    placeholder_shared[((((int)threadIdx.x) + 160))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 16000))];
    placeholder_shared[((((int)threadIdx.x) + 192))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 19200))];
    placeholder_shared[((((int)threadIdx.x) + 224))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 22400))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 25600))];
    placeholder_shared[((((int)threadIdx.x) + 288))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 28800))];
    placeholder_shared[((((int)threadIdx.x) + 320))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 32000))];
    placeholder_shared[((((int)threadIdx.x) + 352))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 35200))];
    placeholder_shared[((((int)threadIdx.x) + 384))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 38400))];
    placeholder_shared[((((int)threadIdx.x) + 416))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 41600))];
    placeholder_shared[((((int)threadIdx.x) + 448))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 44800))];
    placeholder_shared[((((int)threadIdx.x) + 480))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 48000))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 51200))];
    placeholder_shared[((((int)threadIdx.x) + 544))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 54400))];
    placeholder_shared[((((int)threadIdx.x) + 576))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 57600))];
    placeholder_shared[((((int)threadIdx.x) + 608))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 60800))];
    placeholder_shared[((((int)threadIdx.x) + 640))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 64000))];
    placeholder_shared[((((int)threadIdx.x) + 672))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 67200))];
    placeholder_shared[((((int)threadIdx.x) + 704))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 70400))];
    placeholder_shared[((((int)threadIdx.x) + 736))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 73600))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 76800))];
    placeholder_shared[((((int)threadIdx.x) + 800))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 80000))];
    placeholder_shared[((((int)threadIdx.x) + 832))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 83200))];
    placeholder_shared[((((int)threadIdx.x) + 864))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 86400))];
    placeholder_shared[((((int)threadIdx.x) + 896))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 89600))];
    placeholder_shared[((((int)threadIdx.x) + 928))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 92800))];
    placeholder_shared[((((int)threadIdx.x) + 960))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 96000))];
    placeholder_shared[((((int)threadIdx.x) + 992))] = placeholder1[((((((((int)blockIdx.x) % 25) * 102400) + (((int)threadIdx.x) * 100)) + k_outer_outer) + 99200))];
    __syncthreads();
    T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(0)] * placeholder_shared[((((int)threadIdx.x) * 32))]));
    T_dense[(1)] = (T_dense[(1)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 1))]));
    T_dense[(2)] = (T_dense[(2)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 2))]));
    T_dense[(3)] = (T_dense[(3)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 3))]));
    T_dense[(4)] = (T_dense[(4)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 4))]));
    T_dense[(5)] = (T_dense[(5)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 5))]));
    T_dense[(6)] = (T_dense[(6)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 6))]));
    T_dense[(7)] = (T_dense[(7)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 7))]));
    T_dense[(32)] = (T_dense[(32)] + (placeholder_d_shared[(1)] * placeholder_shared[((((int)threadIdx.x) * 32))]));
    T_dense[(33)] = (T_dense[(33)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 1))]));
    T_dense[(34)] = (T_dense[(34)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 2))]));
    T_dense[(35)] = (T_dense[(35)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 3))]));
    T_dense[(36)] = (T_dense[(36)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 4))]));
    T_dense[(37)] = (T_dense[(37)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 5))]));
    T_dense[(38)] = (T_dense[(38)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 6))]));
    T_dense[(39)] = (T_dense[(39)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 7))]));
    T_dense[(8)] = (T_dense[(8)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 8))]));
    T_dense[(9)] = (T_dense[(9)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 9))]));
    T_dense[(10)] = (T_dense[(10)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 10))]));
    T_dense[(11)] = (T_dense[(11)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 11))]));
    T_dense[(12)] = (T_dense[(12)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 12))]));
    T_dense[(13)] = (T_dense[(13)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 13))]));
    T_dense[(14)] = (T_dense[(14)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 14))]));
    T_dense[(15)] = (T_dense[(15)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 15))]));
    T_dense[(40)] = (T_dense[(40)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 8))]));
    T_dense[(41)] = (T_dense[(41)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 9))]));
    T_dense[(42)] = (T_dense[(42)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 10))]));
    T_dense[(43)] = (T_dense[(43)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 11))]));
    T_dense[(44)] = (T_dense[(44)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 12))]));
    T_dense[(45)] = (T_dense[(45)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 13))]));
    T_dense[(46)] = (T_dense[(46)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 14))]));
    T_dense[(47)] = (T_dense[(47)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 15))]));
    T_dense[(16)] = (T_dense[(16)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 16))]));
    T_dense[(17)] = (T_dense[(17)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 17))]));
    T_dense[(18)] = (T_dense[(18)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 18))]));
    T_dense[(19)] = (T_dense[(19)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 19))]));
    T_dense[(20)] = (T_dense[(20)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 20))]));
    T_dense[(21)] = (T_dense[(21)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 21))]));
    T_dense[(22)] = (T_dense[(22)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 22))]));
    T_dense[(23)] = (T_dense[(23)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 23))]));
    T_dense[(48)] = (T_dense[(48)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 16))]));
    T_dense[(49)] = (T_dense[(49)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 17))]));
    T_dense[(50)] = (T_dense[(50)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 18))]));
    T_dense[(51)] = (T_dense[(51)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 19))]));
    T_dense[(52)] = (T_dense[(52)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 20))]));
    T_dense[(53)] = (T_dense[(53)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 21))]));
    T_dense[(54)] = (T_dense[(54)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 22))]));
    T_dense[(55)] = (T_dense[(55)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 23))]));
    T_dense[(24)] = (T_dense[(24)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 24))]));
    T_dense[(25)] = (T_dense[(25)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 25))]));
    T_dense[(26)] = (T_dense[(26)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 26))]));
    T_dense[(27)] = (T_dense[(27)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 27))]));
    T_dense[(28)] = (T_dense[(28)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 28))]));
    T_dense[(29)] = (T_dense[(29)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 29))]));
    T_dense[(30)] = (T_dense[(30)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 30))]));
    T_dense[(31)] = (T_dense[(31)] + (placeholder_d_shared[(0)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 31))]));
    T_dense[(56)] = (T_dense[(56)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 24))]));
    T_dense[(57)] = (T_dense[(57)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 25))]));
    T_dense[(58)] = (T_dense[(58)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 26))]));
    T_dense[(59)] = (T_dense[(59)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 27))]));
    T_dense[(60)] = (T_dense[(60)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 28))]));
    T_dense[(61)] = (T_dense[(61)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 29))]));
    T_dense[(62)] = (T_dense[(62)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 30))]));
    T_dense[(63)] = (T_dense[(63)] + (placeholder_d_shared[(1)] * placeholder_shared[(((((int)threadIdx.x) * 32) + 31))]));
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 32; ++ax1_inner) {
      T_relu[(((((((((int)blockIdx.x) / 25) * 51200) + (ax0_inner * 25600)) + ((((int)blockIdx.x) % 25) * 1024)) + (((int)threadIdx.x) * 32)) + ax1_inner))] = max(T_dense[(((ax0_inner * 32) + ax1_inner))], 0.000000e+00f);
    }
  }
}


