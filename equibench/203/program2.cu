
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[392];
  __shared__ float pad_temp_shared[3132];
  __shared__ float placeholder_shared[192];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 7; ++yy_outer_inner_init) {
    for (int xx_outer_inner_init = 0; xx_outer_inner_init < 14; ++xx_outer_inner_init) {
      compute[(((yy_outer_inner_init * 14) + xx_outer_inner_init))] = 0.000000e+00f;
      compute[((((yy_outer_inner_init * 14) + xx_outer_inner_init) + 98))] = 0.000000e+00f;
      compute[((((yy_outer_inner_init * 14) + xx_outer_inner_init) + 196))] = 0.000000e+00f;
      compute[((((yy_outer_inner_init * 14) + xx_outer_inner_init) + 294))] = 0.000000e+00f;
    }
  }
  for (int rx_outer_outer = 0; rx_outer_outer < 3; ++rx_outer_outer) {
    for (int rc_outer_outer = 0; rc_outer_outer < 4; ++rc_outer_outer) {
      __syncthreads();
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 392; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
        if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 2)) < 783) {
          if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)) < 3132) {
            pad_temp_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 8) + ((int)threadIdx.x)))] = (((27 <= ((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 2))) && (1 <= (rx_outer_outer + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 2)) % 27)))) ? placeholder[(((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 2)) / 27) * 14336) + (rx_outer_outer * 512)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + (((int)threadIdx.x) >> 2)) % 27) * 512)) + (((int)blockIdx.x) * 16)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) - 14848))] : 0.000000e+00f);
          }
        }
      }
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 24; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
        placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)))] = placeholder1[((((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) >> 6) * 24576) + (rx_outer_outer * 8192)) + (rc_outer_outer * 2048)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) & 63) >> 4) * 512)) + (((int)blockIdx.x) * 16)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 8) + ((int)threadIdx.x)) & 15)))];
      }
      __syncthreads();
      for (int rc_outer_inner = 0; rc_outer_inner < 2; ++rc_outer_inner) {
        for (int yy_outer_inner = 0; yy_outer_inner < 7; ++yy_outer_inner) {
          for (int xx_outer_inner = 0; xx_outer_inner < 14; ++xx_outer_inner) {
            for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
              compute[(((yy_outer_inner * 14) + xx_outer_inner))] = (compute[(((yy_outer_inner * 14) + xx_outer_inner))] + (pad_temp_shared[(((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)))] * placeholder_shared[((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)))]));
              compute[((((yy_outer_inner * 14) + xx_outer_inner) + 98))] = (compute[((((yy_outer_inner * 14) + xx_outer_inner) + 98))] + (pad_temp_shared[(((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 4))]));
              compute[((((yy_outer_inner * 14) + xx_outer_inner) + 196))] = (compute[((((yy_outer_inner * 14) + xx_outer_inner) + 196))] + (pad_temp_shared[(((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 8))]));
              compute[((((yy_outer_inner * 14) + xx_outer_inner) + 294))] = (compute[((((yy_outer_inner * 14) + xx_outer_inner) + 294))] + (pad_temp_shared[(((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 12))]));
              compute[(((yy_outer_inner * 14) + xx_outer_inner))] = (compute[(((yy_outer_inner * 14) + xx_outer_inner))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 16))]));
              compute[((((yy_outer_inner * 14) + xx_outer_inner) + 98))] = (compute[((((yy_outer_inner * 14) + xx_outer_inner) + 98))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 20))]));
              compute[((((yy_outer_inner * 14) + xx_outer_inner) + 196))] = (compute[((((yy_outer_inner * 14) + xx_outer_inner) + 196))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 24))]));
              compute[((((yy_outer_inner * 14) + xx_outer_inner) + 294))] = (compute[((((yy_outer_inner * 14) + xx_outer_inner) + 294))] + (pad_temp_shared[((((((((((int)threadIdx.x) >> 2) * 1512) + (yy_outer_inner * 216)) + (ry_inner * 108)) + (xx_outer_inner * 8)) + (rc_outer_inner * 2)) + 1))] * placeholder_shared[(((((ry_inner * 64) + (rc_outer_inner * 32)) + (((int)threadIdx.x) & 3)) + 28))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 7; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
      T_relu[(((((((((int)threadIdx.x) >> 2) * 50176) + (ax1_inner * 7168)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 3)))] = max((compute[(((ax1_inner * 14) + ax2_inner))] + placeholder2[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 3)))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 2) * 50176) + (ax1_inner * 7168)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 3)) + 4))] = max((compute[((((ax1_inner * 14) + ax2_inner) + 98))] + placeholder2[((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 3)) + 4))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 2) * 50176) + (ax1_inner * 7168)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 3)) + 8))] = max((compute[((((ax1_inner * 14) + ax2_inner) + 196))] + placeholder2[((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 3)) + 8))]), 0.000000e+00f);
      T_relu[((((((((((int)threadIdx.x) >> 2) * 50176) + (ax1_inner * 7168)) + (ax2_inner * 512)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) & 3)) + 12))] = max((compute[((((ax1_inner * 14) + ax2_inner) + 294))] + placeholder2[((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) & 3)) + 12))]), 0.000000e+00f);
    }
  }
}


