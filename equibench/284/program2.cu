
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense[1];
  __shared__ float placeholder_d_shared[40];
  __shared__ float placeholder_shared[8000];
  T_dense[(0)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 32; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 8) {
      placeholder_d_shared[((((int)threadIdx.x) * 5))] = placeholder[(((k_outer_outer * 40) + (((int)threadIdx.x) * 5)))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_d_shared[(((((int)threadIdx.x) * 5) + 1))] = placeholder[((((k_outer_outer * 40) + (((int)threadIdx.x) * 5)) + 1))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_d_shared[(((((int)threadIdx.x) * 5) + 2))] = placeholder[((((k_outer_outer * 40) + (((int)threadIdx.x) * 5)) + 2))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_d_shared[(((((int)threadIdx.x) * 5) + 3))] = placeholder[((((k_outer_outer * 40) + (((int)threadIdx.x) * 5)) + 3))];
    }
    if (((int)threadIdx.x) < 8) {
      placeholder_d_shared[(((((int)threadIdx.x) * 5) + 4))] = placeholder[((((k_outer_outer * 40) + (((int)threadIdx.x) * 5)) + 4))];
    }
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 40; ++ax0_ax1_fused_outer_outer) {
      placeholder_shared[(((ax0_ax1_fused_outer_outer * 200) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.x) * 256000) + (ax0_ax1_fused_outer_outer * 6400)) + ((((int)threadIdx.x) / 40) * 1280)) + (k_outer_outer * 40)) + (((int)threadIdx.x) % 40)))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 40; ++k_outer_inner) {
      T_dense[(0)] = (T_dense[(0)] + (placeholder_d_shared[(k_outer_inner)] * placeholder_shared[(((((int)threadIdx.x) * 40) + k_outer_inner))]));
    }
  }
  T_add[(((((int)blockIdx.x) * 200) + ((int)threadIdx.x)))] = (T_dense[(0)] + placeholder2[(((((int)blockIdx.x) * 200) + ((int)threadIdx.x)))]);
}


