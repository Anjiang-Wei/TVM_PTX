
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[40];
  __shared__ float PaddedInput_shared[2880];
  __shared__ float placeholder_shared[128];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 5; ++xx_outer_inner_init) {
    Conv2dOutput[((xx_outer_inner_init * 4))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 2))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 22))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 3))] = 0.000000e+00f;
    Conv2dOutput[(((xx_outer_inner_init * 4) + 23))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 27; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 320; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + ((int)threadIdx.x)) / 1440) * 194400) + ((((int)blockIdx.x) >> 5) * 38880)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + ((int)threadIdx.x)) % 1440) >> 5) * 864)) + (rc_outer_outer * 32)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 9) + ((int)threadIdx.x)) & 31)))];
    }
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 9))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 9) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 1) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 18))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 18) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 27))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 27) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 3) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 36))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)) + 1152))];
    placeholder_shared[((((int)threadIdx.x) + 45))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 45) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 1) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 54))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 54) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 63))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 63) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 3) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 72))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)) + 2304))];
    placeholder_shared[((((int)threadIdx.x) + 81))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 81) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 1) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 90))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 90) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 2) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 99))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 99) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 3) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 108))] = placeholder1[((((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) & 3)) + 3456))];
    placeholder_shared[((((int)threadIdx.x) + 117))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 117) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ((((int)threadIdx.x) + 1) & 3)))];
    if (((int)threadIdx.x) < 2) {
      placeholder_shared[((((int)threadIdx.x) + 126))] = placeholder1[(((((rc_outer_outer * 4096) + (((((int)threadIdx.x) + 126) >> 2) * 128)) + ((((int)blockIdx.x) & 31) * 4)) + (((int)threadIdx.x) + 2)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 5; ++xx_outer_inner) {
        for (int ff_outer_inner = 0; ff_outer_inner < 4; ++ff_outer_inner) {
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)))] * placeholder_shared[(((rc_outer_inner * 32) + ff_outer_inner))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1440))] * placeholder_shared[(((rc_outer_inner * 32) + ff_outer_inner))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 4))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1441))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 4))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 2))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 8))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1442))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 8))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 3))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 12))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1443))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 12))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 4))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 16))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1444))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 16))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 5))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 20))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1445))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 20))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 6))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 24))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1446))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 24))]));
          Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] = (Conv2dOutput[(((xx_outer_inner * 4) + ff_outer_inner))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 7))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 28))]));
          Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] = (Conv2dOutput[((((xx_outer_inner * 4) + ff_outer_inner) + 20))] + (PaddedInput_shared[(((((((int)threadIdx.x) * 160) + (xx_outer_inner * 32)) + (rc_outer_inner * 8)) + 1447))] * placeholder_shared[((((rc_outer_inner * 32) + ff_outer_inner) + 28))]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax2_inner = 0; ax2_inner < 5; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
        T_relu[(((((((ax0_inner * 28800) + ((((int)blockIdx.x) >> 5) * 5760)) + (((int)threadIdx.x) * 640)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 31) * 4)) + ax3_inner))] = max((Conv2dOutput[((((ax0_inner * 20) + (ax2_inner * 4)) + ax3_inner))] + placeholder2[((((((int)blockIdx.x) & 31) * 4) + ax3_inner))]), 0.000000e+00f);
      }
    }
  }
}


