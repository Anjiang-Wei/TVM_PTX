
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel1(float* __restrict__ tensor, float* __restrict__ tensor1) {
  tensor[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = (tensor1[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] * 1.562500e-02f);
}

extern "C" __global__ void my_kernel_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 8; ++rv0) {
    for (int rv1 = 0; rv1 < 8; ++rv1) {
      tensor[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] = (tensor[(((((int)blockIdx.x) * 2) + ((int)threadIdx.x)))] + placeholder[((((((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) >> 9) * 32768) + (rv0 * 4096)) + (rv1 * 512)) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) & 511)))]);
    }
  }
}


