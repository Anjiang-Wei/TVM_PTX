
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[64];
  __shared__ float PaddedInput_shared[1024];
  __shared__ float placeholder_shared[256];
  Conv2dOutput[(0)] = 0.000000e+00f;
  Conv2dOutput[(8)] = 0.000000e+00f;
  Conv2dOutput[(16)] = 0.000000e+00f;
  Conv2dOutput[(24)] = 0.000000e+00f;
  Conv2dOutput[(32)] = 0.000000e+00f;
  Conv2dOutput[(40)] = 0.000000e+00f;
  Conv2dOutput[(48)] = 0.000000e+00f;
  Conv2dOutput[(56)] = 0.000000e+00f;
  Conv2dOutput[(4)] = 0.000000e+00f;
  Conv2dOutput[(12)] = 0.000000e+00f;
  Conv2dOutput[(20)] = 0.000000e+00f;
  Conv2dOutput[(28)] = 0.000000e+00f;
  Conv2dOutput[(36)] = 0.000000e+00f;
  Conv2dOutput[(44)] = 0.000000e+00f;
  Conv2dOutput[(52)] = 0.000000e+00f;
  Conv2dOutput[(60)] = 0.000000e+00f;
  Conv2dOutput[(1)] = 0.000000e+00f;
  Conv2dOutput[(9)] = 0.000000e+00f;
  Conv2dOutput[(17)] = 0.000000e+00f;
  Conv2dOutput[(25)] = 0.000000e+00f;
  Conv2dOutput[(33)] = 0.000000e+00f;
  Conv2dOutput[(41)] = 0.000000e+00f;
  Conv2dOutput[(49)] = 0.000000e+00f;
  Conv2dOutput[(57)] = 0.000000e+00f;
  Conv2dOutput[(5)] = 0.000000e+00f;
  Conv2dOutput[(13)] = 0.000000e+00f;
  Conv2dOutput[(21)] = 0.000000e+00f;
  Conv2dOutput[(29)] = 0.000000e+00f;
  Conv2dOutput[(37)] = 0.000000e+00f;
  Conv2dOutput[(45)] = 0.000000e+00f;
  Conv2dOutput[(53)] = 0.000000e+00f;
  Conv2dOutput[(61)] = 0.000000e+00f;
  Conv2dOutput[(2)] = 0.000000e+00f;
  Conv2dOutput[(10)] = 0.000000e+00f;
  Conv2dOutput[(18)] = 0.000000e+00f;
  Conv2dOutput[(26)] = 0.000000e+00f;
  Conv2dOutput[(34)] = 0.000000e+00f;
  Conv2dOutput[(42)] = 0.000000e+00f;
  Conv2dOutput[(50)] = 0.000000e+00f;
  Conv2dOutput[(58)] = 0.000000e+00f;
  Conv2dOutput[(6)] = 0.000000e+00f;
  Conv2dOutput[(14)] = 0.000000e+00f;
  Conv2dOutput[(22)] = 0.000000e+00f;
  Conv2dOutput[(30)] = 0.000000e+00f;
  Conv2dOutput[(38)] = 0.000000e+00f;
  Conv2dOutput[(46)] = 0.000000e+00f;
  Conv2dOutput[(54)] = 0.000000e+00f;
  Conv2dOutput[(62)] = 0.000000e+00f;
  Conv2dOutput[(3)] = 0.000000e+00f;
  Conv2dOutput[(11)] = 0.000000e+00f;
  Conv2dOutput[(19)] = 0.000000e+00f;
  Conv2dOutput[(27)] = 0.000000e+00f;
  Conv2dOutput[(35)] = 0.000000e+00f;
  Conv2dOutput[(43)] = 0.000000e+00f;
  Conv2dOutput[(51)] = 0.000000e+00f;
  Conv2dOutput[(59)] = 0.000000e+00f;
  Conv2dOutput[(7)] = 0.000000e+00f;
  Conv2dOutput[(15)] = 0.000000e+00f;
  Conv2dOutput[(23)] = 0.000000e+00f;
  Conv2dOutput[(31)] = 0.000000e+00f;
  Conv2dOutput[(39)] = 0.000000e+00f;
  Conv2dOutput[(47)] = 0.000000e+00f;
  Conv2dOutput[(55)] = 0.000000e+00f;
  Conv2dOutput[(63)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 92; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 64))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 5888))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 11776))];
    PaddedInput_shared[((((int)threadIdx.x) + 192))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 17664))];
    PaddedInput_shared[((((int)threadIdx.x) + 256))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 23552))];
    PaddedInput_shared[((((int)threadIdx.x) + 320))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 29440))];
    PaddedInput_shared[((((int)threadIdx.x) + 384))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 35328))];
    PaddedInput_shared[((((int)threadIdx.x) + 448))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 41216))];
    PaddedInput_shared[((((int)threadIdx.x) + 512))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 47104))];
    PaddedInput_shared[((((int)threadIdx.x) + 576))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 52992))];
    PaddedInput_shared[((((int)threadIdx.x) + 640))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 58880))];
    PaddedInput_shared[((((int)threadIdx.x) + 704))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 64768))];
    PaddedInput_shared[((((int)threadIdx.x) + 768))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 70656))];
    PaddedInput_shared[((((int)threadIdx.x) + 832))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 76544))];
    PaddedInput_shared[((((int)threadIdx.x) + 896))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 82432))];
    PaddedInput_shared[((((int)threadIdx.x) + 960))] = placeholder[(((((((((int)blockIdx.x) >> 2) * 94208) + ((((int)threadIdx.x) >> 3) * 736)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 88320))];
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[((((int)threadIdx.x) * 64))] = placeholder1[((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 1))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 1))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 2))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 2))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 3))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 3))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 4))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 4))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 5))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 5))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 6))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 6))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 7))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 7))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 8))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 8))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 9))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 9))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 10))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 10))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 11))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 11))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 12))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 12))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 13))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 13))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 14))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 14))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 15))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 15))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 16))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 16))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 17))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 17))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 18))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 18))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 19))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 19))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 20))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 20))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 21))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 21))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 22))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 22))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 23))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 23))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 24))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 24))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 25))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 25))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 26))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 26))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 27))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 27))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 28))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 28))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 29))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 29))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 30))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 30))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 31))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 31))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 32))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 128))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 33))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 129))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 34))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 130))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 35))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 131))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 36))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 132))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 37))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 133))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 38))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 134))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 39))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 135))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 40))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 136))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 41))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 137))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 42))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 138))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 43))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 139))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 44))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 140))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 45))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 141))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 46))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 142))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 47))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 143))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 48))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 144))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 49))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 145))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 50))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 146))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 51))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 147))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 52))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 148))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 53))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 149))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 54))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 150))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 55))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 151))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 56))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 152))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 57))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 153))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 58))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 154))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 59))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 155))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 60))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 156))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 61))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 157))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 62))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 158))];
    }
    if (((int)threadIdx.x) < 4) {
      placeholder_shared[(((((int)threadIdx.x) * 64) + 63))] = placeholder1[(((((rc_outer_outer * 1024) + (((int)threadIdx.x) * 256)) + ((((int)blockIdx.x) & 3) * 32)) + 159))];
    }
    __syncthreads();
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[(((((int)threadIdx.x) & 3) * 4))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 16))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 32))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 48))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 64))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 80))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 96))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 112))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 128))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 144))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 160))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 176))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 192))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 208))]));
    Conv2dOutput[(0)] = (Conv2dOutput[(0)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(8)] = (Conv2dOutput[(8)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(16)] = (Conv2dOutput[(16)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(24)] = (Conv2dOutput[(24)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(32)] = (Conv2dOutput[(32)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(40)] = (Conv2dOutput[(40)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(48)] = (Conv2dOutput[(48)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(56)] = (Conv2dOutput[(56)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(4)] = (Conv2dOutput[(4)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(12)] = (Conv2dOutput[(12)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(20)] = (Conv2dOutput[(20)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(28)] = (Conv2dOutput[(28)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(36)] = (Conv2dOutput[(36)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(44)] = (Conv2dOutput[(44)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(52)] = (Conv2dOutput[(52)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 224))]));
    Conv2dOutput[(60)] = (Conv2dOutput[(60)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 240))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 1))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 17))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 33))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 49))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 65))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 81))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 97))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 113))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 129))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 145))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 161))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 177))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 193))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 209))]));
    Conv2dOutput[(1)] = (Conv2dOutput[(1)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(9)] = (Conv2dOutput[(9)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(17)] = (Conv2dOutput[(17)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(25)] = (Conv2dOutput[(25)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(33)] = (Conv2dOutput[(33)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(41)] = (Conv2dOutput[(41)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(49)] = (Conv2dOutput[(49)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(57)] = (Conv2dOutput[(57)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(5)] = (Conv2dOutput[(5)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(13)] = (Conv2dOutput[(13)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(21)] = (Conv2dOutput[(21)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(29)] = (Conv2dOutput[(29)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(37)] = (Conv2dOutput[(37)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(45)] = (Conv2dOutput[(45)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(53)] = (Conv2dOutput[(53)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 225))]));
    Conv2dOutput[(61)] = (Conv2dOutput[(61)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 241))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 2))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 18))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 34))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 50))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 66))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 82))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 98))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 114))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 130))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 146))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 162))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 178))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 194))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 210))]));
    Conv2dOutput[(2)] = (Conv2dOutput[(2)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(10)] = (Conv2dOutput[(10)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(18)] = (Conv2dOutput[(18)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(26)] = (Conv2dOutput[(26)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(34)] = (Conv2dOutput[(34)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(42)] = (Conv2dOutput[(42)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(50)] = (Conv2dOutput[(50)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(58)] = (Conv2dOutput[(58)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(6)] = (Conv2dOutput[(6)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(14)] = (Conv2dOutput[(14)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(22)] = (Conv2dOutput[(22)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(30)] = (Conv2dOutput[(30)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(38)] = (Conv2dOutput[(38)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(46)] = (Conv2dOutput[(46)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(54)] = (Conv2dOutput[(54)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 226))]));
    Conv2dOutput[(62)] = (Conv2dOutput[(62)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 242))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 32))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 64))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 96))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 160))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 192))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 3))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 224))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 19))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 33))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 65))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 97))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 161))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 193))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 35))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 225))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 51))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 34))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 66))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 98))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 162))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 194))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 67))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 226))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 83))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 35))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 67))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 99))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 163))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 195))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 99))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 227))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 115))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 36))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 68))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 100))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 164))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 196))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 131))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 228))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 147))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 37))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 69))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 101))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 165))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 197))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 163))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 229))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 179))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 38))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 70))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 102))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 166))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 198))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 195))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 230))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 211))]));
    Conv2dOutput[(3)] = (Conv2dOutput[(3)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(11)] = (Conv2dOutput[(11)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(19)] = (Conv2dOutput[(19)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(27)] = (Conv2dOutput[(27)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 39))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(35)] = (Conv2dOutput[(35)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(43)] = (Conv2dOutput[(43)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 71))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(51)] = (Conv2dOutput[(51)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(59)] = (Conv2dOutput[(59)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 103))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(7)] = (Conv2dOutput[(7)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(15)] = (Conv2dOutput[(15)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(23)] = (Conv2dOutput[(23)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(31)] = (Conv2dOutput[(31)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 167))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(39)] = (Conv2dOutput[(39)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(47)] = (Conv2dOutput[(47)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 199))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
    Conv2dOutput[(55)] = (Conv2dOutput[(55)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 227))]));
    Conv2dOutput[(63)] = (Conv2dOutput[(63)] + (PaddedInput_shared[(((((((int)threadIdx.x) >> 4) * 256) + (((((int)threadIdx.x) & 15) >> 2) * 8)) + 231))] * placeholder_shared[((((((int)threadIdx.x) & 3) * 4) + 243))]));
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 4; ++ax3_inner) {
      T_relu[(((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 4) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 8))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 512))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 16))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 528))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 24))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 1024))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 32))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 1040))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 40))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 1536))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 48))] + placeholder2[(((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)blockIdx.x) >> 2) * 16384) + ((((int)threadIdx.x) >> 4) * 4096)) + (ax1_inner * 2048)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 3) * 32)) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 1552))] = max((Conv2dOutput[((((ax1_inner * 4) + ax3_inner) + 56))] + placeholder2[((((((((int)blockIdx.x) & 3) * 32) + ((((int)threadIdx.x) & 3) * 4)) + ax3_inner) + 16))]), 0.000000e+00f);
    }
  }
}


