
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[32];
  __shared__ float PaddedInput_shared[256];
  __shared__ float placeholder_shared[256];
  for (int yy_outer_inner_init = 0; yy_outer_inner_init < 2; ++yy_outer_inner_init) {
    Conv2dOutput[((yy_outer_inner_init * 2))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 4))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 8))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 12))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 16))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 20))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 24))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 28))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 1))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 5))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 9))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 13))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 17))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 21))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 25))] = 0.000000e+00f;
    Conv2dOutput[(((yy_outer_inner_init * 2) + 29))] = 0.000000e+00f;
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 184; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[(((((((((int)threadIdx.x) >> 4) * 11776) + ((((int)blockIdx.x) >> 1) * 2944)) + (((((int)threadIdx.x) & 15) >> 2) * 736)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    PaddedInput_shared[((((int)threadIdx.x) + 128))] = placeholder[((((((((((int)threadIdx.x) >> 4) * 11776) + ((((int)blockIdx.x) >> 1) * 2944)) + (((((int)threadIdx.x) & 15) >> 2) * 736)) + (rc_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 94208))];
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[((((int)threadIdx.x) * 4))] = placeholder1[(((((rc_outer_outer * 512) + ((((int)threadIdx.x) >> 4) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 15) * 4)))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 1))] = placeholder1[(((((rc_outer_outer * 512) + ((((((int)threadIdx.x) * 4) + 1) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 4) + 1) & 63)))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 2))] = placeholder1[(((((rc_outer_outer * 512) + ((((((int)threadIdx.x) * 4) + 2) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 4) + 2) & 63)))];
    }
    if (((int)threadIdx.x) < 64) {
      placeholder_shared[(((((int)threadIdx.x) * 4) + 3))] = placeholder1[(((((rc_outer_outer * 512) + ((((((int)threadIdx.x) * 4) + 3) >> 6) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + (((((int)threadIdx.x) * 4) + 3) & 63)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int yy_outer_inner = 0; yy_outer_inner < 2; ++yy_outer_inner) {
        Conv2dOutput[((yy_outer_inner * 2))] = (Conv2dOutput[((yy_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[(((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 4))] = (Conv2dOutput[(((yy_outer_inner * 2) + 4))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 8))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 8))] = (Conv2dOutput[(((yy_outer_inner * 2) + 8))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 16))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 12))] = (Conv2dOutput[(((yy_outer_inner * 2) + 12))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 24))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 16))] = (Conv2dOutput[(((yy_outer_inner * 2) + 16))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 32))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 20))] = (Conv2dOutput[(((yy_outer_inner * 2) + 20))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 40))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 24))] = (Conv2dOutput[(((yy_outer_inner * 2) + 24))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 48))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 28))] = (Conv2dOutput[(((yy_outer_inner * 2) + 28))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 56))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 1))] = (Conv2dOutput[(((yy_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 1))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 5))] = (Conv2dOutput[(((yy_outer_inner * 2) + 5))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 9))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 9))] = (Conv2dOutput[(((yy_outer_inner * 2) + 9))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 17))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 13))] = (Conv2dOutput[(((yy_outer_inner * 2) + 13))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 25))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 17))] = (Conv2dOutput[(((yy_outer_inner * 2) + 17))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 33))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 21))] = (Conv2dOutput[(((yy_outer_inner * 2) + 21))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 41))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 25))] = (Conv2dOutput[(((yy_outer_inner * 2) + 25))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 49))]));
        Conv2dOutput[(((yy_outer_inner * 2) + 29))] = (Conv2dOutput[(((yy_outer_inner * 2) + 29))] + (PaddedInput_shared[((((((((int)threadIdx.x) >> 4) * 32) + (yy_outer_inner * 16)) + (((((int)threadIdx.x) & 15) >> 2) * 4)) + rc_outer_inner))] * placeholder_shared[((((rc_outer_inner * 64) + ((((int)threadIdx.x) & 3) * 2)) + 57))]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
      T_relu[(((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))] = max((Conv2dOutput[(((ax1_inner * 2) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 8))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 4))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 8))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 16))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 8))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 16))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 24))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 12))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 24))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 32))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 16))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 32))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 40))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 20))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 40))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 48))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 24))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 48))]), 0.000000e+00f);
      T_relu[((((((((((((int)threadIdx.x) >> 4) * 4096) + (ax1_inner * 2048)) + ((((int)blockIdx.x) >> 1) * 512)) + (((((int)threadIdx.x) & 15) >> 2) * 128)) + ((((int)blockIdx.x) & 1) * 64)) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 56))] = max((Conv2dOutput[((((ax1_inner * 2) + ax3_inner) + 28))] + placeholder2[((((((((int)blockIdx.x) & 1) * 64) + ((((int)threadIdx.x) & 3) * 2)) + ax3_inner) + 56))]), 0.000000e+00f);
    }
  }
}


