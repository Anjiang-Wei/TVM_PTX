
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ Conv2dOutput) {
  float Conv2dOutput_local[256];
  __shared__ float PaddedInput_shared[3136];
  __shared__ float placeholder_shared[512];
  Conv2dOutput_local[(0)] = 0.000000e+00f;
  Conv2dOutput_local[(32)] = 0.000000e+00f;
  Conv2dOutput_local[(64)] = 0.000000e+00f;
  Conv2dOutput_local[(96)] = 0.000000e+00f;
  Conv2dOutput_local[(128)] = 0.000000e+00f;
  Conv2dOutput_local[(160)] = 0.000000e+00f;
  Conv2dOutput_local[(192)] = 0.000000e+00f;
  Conv2dOutput_local[(224)] = 0.000000e+00f;
  Conv2dOutput_local[(1)] = 0.000000e+00f;
  Conv2dOutput_local[(33)] = 0.000000e+00f;
  Conv2dOutput_local[(65)] = 0.000000e+00f;
  Conv2dOutput_local[(97)] = 0.000000e+00f;
  Conv2dOutput_local[(129)] = 0.000000e+00f;
  Conv2dOutput_local[(161)] = 0.000000e+00f;
  Conv2dOutput_local[(193)] = 0.000000e+00f;
  Conv2dOutput_local[(225)] = 0.000000e+00f;
  Conv2dOutput_local[(2)] = 0.000000e+00f;
  Conv2dOutput_local[(34)] = 0.000000e+00f;
  Conv2dOutput_local[(66)] = 0.000000e+00f;
  Conv2dOutput_local[(98)] = 0.000000e+00f;
  Conv2dOutput_local[(130)] = 0.000000e+00f;
  Conv2dOutput_local[(162)] = 0.000000e+00f;
  Conv2dOutput_local[(194)] = 0.000000e+00f;
  Conv2dOutput_local[(226)] = 0.000000e+00f;
  Conv2dOutput_local[(3)] = 0.000000e+00f;
  Conv2dOutput_local[(35)] = 0.000000e+00f;
  Conv2dOutput_local[(67)] = 0.000000e+00f;
  Conv2dOutput_local[(99)] = 0.000000e+00f;
  Conv2dOutput_local[(131)] = 0.000000e+00f;
  Conv2dOutput_local[(163)] = 0.000000e+00f;
  Conv2dOutput_local[(195)] = 0.000000e+00f;
  Conv2dOutput_local[(227)] = 0.000000e+00f;
  Conv2dOutput_local[(4)] = 0.000000e+00f;
  Conv2dOutput_local[(36)] = 0.000000e+00f;
  Conv2dOutput_local[(68)] = 0.000000e+00f;
  Conv2dOutput_local[(100)] = 0.000000e+00f;
  Conv2dOutput_local[(132)] = 0.000000e+00f;
  Conv2dOutput_local[(164)] = 0.000000e+00f;
  Conv2dOutput_local[(196)] = 0.000000e+00f;
  Conv2dOutput_local[(228)] = 0.000000e+00f;
  Conv2dOutput_local[(5)] = 0.000000e+00f;
  Conv2dOutput_local[(37)] = 0.000000e+00f;
  Conv2dOutput_local[(69)] = 0.000000e+00f;
  Conv2dOutput_local[(101)] = 0.000000e+00f;
  Conv2dOutput_local[(133)] = 0.000000e+00f;
  Conv2dOutput_local[(165)] = 0.000000e+00f;
  Conv2dOutput_local[(197)] = 0.000000e+00f;
  Conv2dOutput_local[(229)] = 0.000000e+00f;
  Conv2dOutput_local[(6)] = 0.000000e+00f;
  Conv2dOutput_local[(38)] = 0.000000e+00f;
  Conv2dOutput_local[(70)] = 0.000000e+00f;
  Conv2dOutput_local[(102)] = 0.000000e+00f;
  Conv2dOutput_local[(134)] = 0.000000e+00f;
  Conv2dOutput_local[(166)] = 0.000000e+00f;
  Conv2dOutput_local[(198)] = 0.000000e+00f;
  Conv2dOutput_local[(230)] = 0.000000e+00f;
  Conv2dOutput_local[(7)] = 0.000000e+00f;
  Conv2dOutput_local[(39)] = 0.000000e+00f;
  Conv2dOutput_local[(71)] = 0.000000e+00f;
  Conv2dOutput_local[(103)] = 0.000000e+00f;
  Conv2dOutput_local[(135)] = 0.000000e+00f;
  Conv2dOutput_local[(167)] = 0.000000e+00f;
  Conv2dOutput_local[(199)] = 0.000000e+00f;
  Conv2dOutput_local[(231)] = 0.000000e+00f;
  Conv2dOutput_local[(8)] = 0.000000e+00f;
  Conv2dOutput_local[(40)] = 0.000000e+00f;
  Conv2dOutput_local[(72)] = 0.000000e+00f;
  Conv2dOutput_local[(104)] = 0.000000e+00f;
  Conv2dOutput_local[(136)] = 0.000000e+00f;
  Conv2dOutput_local[(168)] = 0.000000e+00f;
  Conv2dOutput_local[(200)] = 0.000000e+00f;
  Conv2dOutput_local[(232)] = 0.000000e+00f;
  Conv2dOutput_local[(9)] = 0.000000e+00f;
  Conv2dOutput_local[(41)] = 0.000000e+00f;
  Conv2dOutput_local[(73)] = 0.000000e+00f;
  Conv2dOutput_local[(105)] = 0.000000e+00f;
  Conv2dOutput_local[(137)] = 0.000000e+00f;
  Conv2dOutput_local[(169)] = 0.000000e+00f;
  Conv2dOutput_local[(201)] = 0.000000e+00f;
  Conv2dOutput_local[(233)] = 0.000000e+00f;
  Conv2dOutput_local[(10)] = 0.000000e+00f;
  Conv2dOutput_local[(42)] = 0.000000e+00f;
  Conv2dOutput_local[(74)] = 0.000000e+00f;
  Conv2dOutput_local[(106)] = 0.000000e+00f;
  Conv2dOutput_local[(138)] = 0.000000e+00f;
  Conv2dOutput_local[(170)] = 0.000000e+00f;
  Conv2dOutput_local[(202)] = 0.000000e+00f;
  Conv2dOutput_local[(234)] = 0.000000e+00f;
  Conv2dOutput_local[(11)] = 0.000000e+00f;
  Conv2dOutput_local[(43)] = 0.000000e+00f;
  Conv2dOutput_local[(75)] = 0.000000e+00f;
  Conv2dOutput_local[(107)] = 0.000000e+00f;
  Conv2dOutput_local[(139)] = 0.000000e+00f;
  Conv2dOutput_local[(171)] = 0.000000e+00f;
  Conv2dOutput_local[(203)] = 0.000000e+00f;
  Conv2dOutput_local[(235)] = 0.000000e+00f;
  Conv2dOutput_local[(12)] = 0.000000e+00f;
  Conv2dOutput_local[(44)] = 0.000000e+00f;
  Conv2dOutput_local[(76)] = 0.000000e+00f;
  Conv2dOutput_local[(108)] = 0.000000e+00f;
  Conv2dOutput_local[(140)] = 0.000000e+00f;
  Conv2dOutput_local[(172)] = 0.000000e+00f;
  Conv2dOutput_local[(204)] = 0.000000e+00f;
  Conv2dOutput_local[(236)] = 0.000000e+00f;
  Conv2dOutput_local[(13)] = 0.000000e+00f;
  Conv2dOutput_local[(45)] = 0.000000e+00f;
  Conv2dOutput_local[(77)] = 0.000000e+00f;
  Conv2dOutput_local[(109)] = 0.000000e+00f;
  Conv2dOutput_local[(141)] = 0.000000e+00f;
  Conv2dOutput_local[(173)] = 0.000000e+00f;
  Conv2dOutput_local[(205)] = 0.000000e+00f;
  Conv2dOutput_local[(237)] = 0.000000e+00f;
  Conv2dOutput_local[(14)] = 0.000000e+00f;
  Conv2dOutput_local[(46)] = 0.000000e+00f;
  Conv2dOutput_local[(78)] = 0.000000e+00f;
  Conv2dOutput_local[(110)] = 0.000000e+00f;
  Conv2dOutput_local[(142)] = 0.000000e+00f;
  Conv2dOutput_local[(174)] = 0.000000e+00f;
  Conv2dOutput_local[(206)] = 0.000000e+00f;
  Conv2dOutput_local[(238)] = 0.000000e+00f;
  Conv2dOutput_local[(15)] = 0.000000e+00f;
  Conv2dOutput_local[(47)] = 0.000000e+00f;
  Conv2dOutput_local[(79)] = 0.000000e+00f;
  Conv2dOutput_local[(111)] = 0.000000e+00f;
  Conv2dOutput_local[(143)] = 0.000000e+00f;
  Conv2dOutput_local[(175)] = 0.000000e+00f;
  Conv2dOutput_local[(207)] = 0.000000e+00f;
  Conv2dOutput_local[(239)] = 0.000000e+00f;
  Conv2dOutput_local[(16)] = 0.000000e+00f;
  Conv2dOutput_local[(48)] = 0.000000e+00f;
  Conv2dOutput_local[(80)] = 0.000000e+00f;
  Conv2dOutput_local[(112)] = 0.000000e+00f;
  Conv2dOutput_local[(144)] = 0.000000e+00f;
  Conv2dOutput_local[(176)] = 0.000000e+00f;
  Conv2dOutput_local[(208)] = 0.000000e+00f;
  Conv2dOutput_local[(240)] = 0.000000e+00f;
  Conv2dOutput_local[(17)] = 0.000000e+00f;
  Conv2dOutput_local[(49)] = 0.000000e+00f;
  Conv2dOutput_local[(81)] = 0.000000e+00f;
  Conv2dOutput_local[(113)] = 0.000000e+00f;
  Conv2dOutput_local[(145)] = 0.000000e+00f;
  Conv2dOutput_local[(177)] = 0.000000e+00f;
  Conv2dOutput_local[(209)] = 0.000000e+00f;
  Conv2dOutput_local[(241)] = 0.000000e+00f;
  Conv2dOutput_local[(18)] = 0.000000e+00f;
  Conv2dOutput_local[(50)] = 0.000000e+00f;
  Conv2dOutput_local[(82)] = 0.000000e+00f;
  Conv2dOutput_local[(114)] = 0.000000e+00f;
  Conv2dOutput_local[(146)] = 0.000000e+00f;
  Conv2dOutput_local[(178)] = 0.000000e+00f;
  Conv2dOutput_local[(210)] = 0.000000e+00f;
  Conv2dOutput_local[(242)] = 0.000000e+00f;
  Conv2dOutput_local[(19)] = 0.000000e+00f;
  Conv2dOutput_local[(51)] = 0.000000e+00f;
  Conv2dOutput_local[(83)] = 0.000000e+00f;
  Conv2dOutput_local[(115)] = 0.000000e+00f;
  Conv2dOutput_local[(147)] = 0.000000e+00f;
  Conv2dOutput_local[(179)] = 0.000000e+00f;
  Conv2dOutput_local[(211)] = 0.000000e+00f;
  Conv2dOutput_local[(243)] = 0.000000e+00f;
  Conv2dOutput_local[(20)] = 0.000000e+00f;
  Conv2dOutput_local[(52)] = 0.000000e+00f;
  Conv2dOutput_local[(84)] = 0.000000e+00f;
  Conv2dOutput_local[(116)] = 0.000000e+00f;
  Conv2dOutput_local[(148)] = 0.000000e+00f;
  Conv2dOutput_local[(180)] = 0.000000e+00f;
  Conv2dOutput_local[(212)] = 0.000000e+00f;
  Conv2dOutput_local[(244)] = 0.000000e+00f;
  Conv2dOutput_local[(21)] = 0.000000e+00f;
  Conv2dOutput_local[(53)] = 0.000000e+00f;
  Conv2dOutput_local[(85)] = 0.000000e+00f;
  Conv2dOutput_local[(117)] = 0.000000e+00f;
  Conv2dOutput_local[(149)] = 0.000000e+00f;
  Conv2dOutput_local[(181)] = 0.000000e+00f;
  Conv2dOutput_local[(213)] = 0.000000e+00f;
  Conv2dOutput_local[(245)] = 0.000000e+00f;
  Conv2dOutput_local[(22)] = 0.000000e+00f;
  Conv2dOutput_local[(54)] = 0.000000e+00f;
  Conv2dOutput_local[(86)] = 0.000000e+00f;
  Conv2dOutput_local[(118)] = 0.000000e+00f;
  Conv2dOutput_local[(150)] = 0.000000e+00f;
  Conv2dOutput_local[(182)] = 0.000000e+00f;
  Conv2dOutput_local[(214)] = 0.000000e+00f;
  Conv2dOutput_local[(246)] = 0.000000e+00f;
  Conv2dOutput_local[(23)] = 0.000000e+00f;
  Conv2dOutput_local[(55)] = 0.000000e+00f;
  Conv2dOutput_local[(87)] = 0.000000e+00f;
  Conv2dOutput_local[(119)] = 0.000000e+00f;
  Conv2dOutput_local[(151)] = 0.000000e+00f;
  Conv2dOutput_local[(183)] = 0.000000e+00f;
  Conv2dOutput_local[(215)] = 0.000000e+00f;
  Conv2dOutput_local[(247)] = 0.000000e+00f;
  Conv2dOutput_local[(24)] = 0.000000e+00f;
  Conv2dOutput_local[(56)] = 0.000000e+00f;
  Conv2dOutput_local[(88)] = 0.000000e+00f;
  Conv2dOutput_local[(120)] = 0.000000e+00f;
  Conv2dOutput_local[(152)] = 0.000000e+00f;
  Conv2dOutput_local[(184)] = 0.000000e+00f;
  Conv2dOutput_local[(216)] = 0.000000e+00f;
  Conv2dOutput_local[(248)] = 0.000000e+00f;
  Conv2dOutput_local[(25)] = 0.000000e+00f;
  Conv2dOutput_local[(57)] = 0.000000e+00f;
  Conv2dOutput_local[(89)] = 0.000000e+00f;
  Conv2dOutput_local[(121)] = 0.000000e+00f;
  Conv2dOutput_local[(153)] = 0.000000e+00f;
  Conv2dOutput_local[(185)] = 0.000000e+00f;
  Conv2dOutput_local[(217)] = 0.000000e+00f;
  Conv2dOutput_local[(249)] = 0.000000e+00f;
  Conv2dOutput_local[(26)] = 0.000000e+00f;
  Conv2dOutput_local[(58)] = 0.000000e+00f;
  Conv2dOutput_local[(90)] = 0.000000e+00f;
  Conv2dOutput_local[(122)] = 0.000000e+00f;
  Conv2dOutput_local[(154)] = 0.000000e+00f;
  Conv2dOutput_local[(186)] = 0.000000e+00f;
  Conv2dOutput_local[(218)] = 0.000000e+00f;
  Conv2dOutput_local[(250)] = 0.000000e+00f;
  Conv2dOutput_local[(27)] = 0.000000e+00f;
  Conv2dOutput_local[(59)] = 0.000000e+00f;
  Conv2dOutput_local[(91)] = 0.000000e+00f;
  Conv2dOutput_local[(123)] = 0.000000e+00f;
  Conv2dOutput_local[(155)] = 0.000000e+00f;
  Conv2dOutput_local[(187)] = 0.000000e+00f;
  Conv2dOutput_local[(219)] = 0.000000e+00f;
  Conv2dOutput_local[(251)] = 0.000000e+00f;
  Conv2dOutput_local[(28)] = 0.000000e+00f;
  Conv2dOutput_local[(60)] = 0.000000e+00f;
  Conv2dOutput_local[(92)] = 0.000000e+00f;
  Conv2dOutput_local[(124)] = 0.000000e+00f;
  Conv2dOutput_local[(156)] = 0.000000e+00f;
  Conv2dOutput_local[(188)] = 0.000000e+00f;
  Conv2dOutput_local[(220)] = 0.000000e+00f;
  Conv2dOutput_local[(252)] = 0.000000e+00f;
  Conv2dOutput_local[(29)] = 0.000000e+00f;
  Conv2dOutput_local[(61)] = 0.000000e+00f;
  Conv2dOutput_local[(93)] = 0.000000e+00f;
  Conv2dOutput_local[(125)] = 0.000000e+00f;
  Conv2dOutput_local[(157)] = 0.000000e+00f;
  Conv2dOutput_local[(189)] = 0.000000e+00f;
  Conv2dOutput_local[(221)] = 0.000000e+00f;
  Conv2dOutput_local[(253)] = 0.000000e+00f;
  Conv2dOutput_local[(30)] = 0.000000e+00f;
  Conv2dOutput_local[(62)] = 0.000000e+00f;
  Conv2dOutput_local[(94)] = 0.000000e+00f;
  Conv2dOutput_local[(126)] = 0.000000e+00f;
  Conv2dOutput_local[(158)] = 0.000000e+00f;
  Conv2dOutput_local[(190)] = 0.000000e+00f;
  Conv2dOutput_local[(222)] = 0.000000e+00f;
  Conv2dOutput_local[(254)] = 0.000000e+00f;
  Conv2dOutput_local[(31)] = 0.000000e+00f;
  Conv2dOutput_local[(63)] = 0.000000e+00f;
  Conv2dOutput_local[(95)] = 0.000000e+00f;
  Conv2dOutput_local[(127)] = 0.000000e+00f;
  Conv2dOutput_local[(159)] = 0.000000e+00f;
  Conv2dOutput_local[(191)] = 0.000000e+00f;
  Conv2dOutput_local[(223)] = 0.000000e+00f;
  Conv2dOutput_local[(255)] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 128; ++rc_outer_outer) {
    __syncthreads();
    PaddedInput_shared[(((int)threadIdx.x))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 98))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 98) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 196))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 196) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 294))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 294) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 392))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 50176))];
    PaddedInput_shared[((((int)threadIdx.x) + 490))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 490) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 588))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 588) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 686))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 686) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 784))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 100352))];
    PaddedInput_shared[((((int)threadIdx.x) + 882))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 882) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 980))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 980) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1078))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1078) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1176))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 150528))];
    PaddedInput_shared[((((int)threadIdx.x) + 1274))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1274) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1372))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1372) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1470))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1470) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1568))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 200704))];
    PaddedInput_shared[((((int)threadIdx.x) + 1666))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1666) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1764))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1764) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1862))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 1862) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 1960))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 250880))];
    PaddedInput_shared[((((int)threadIdx.x) + 2058))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2058) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2156))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2156) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2254))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2254) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2352))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 301056))];
    PaddedInput_shared[((((int)threadIdx.x) + 2450))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2450) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2548))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2548) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2646))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2646) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2744))] = placeholder[(((((((((int)blockIdx.x) >> 3) * 401408) + ((((int)threadIdx.x) >> 3) * 1024)) + (rc_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 351232))];
    PaddedInput_shared[((((int)threadIdx.x) + 2842))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2842) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 2) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 2940))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 2940) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 4) & 7)))];
    PaddedInput_shared[((((int)threadIdx.x) + 3038))] = placeholder[((((((((int)blockIdx.x) >> 3) * 401408) + (((((int)threadIdx.x) + 3038) >> 3) * 1024)) + (rc_outer_outer * 8)) + ((((int)threadIdx.x) + 6) & 7)))];
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[((((int)threadIdx.x) * 16))] = placeholder1[(((((rc_outer_outer * 4096) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ((((int)threadIdx.x) & 3) * 16)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 1))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 1) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 1) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 2))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 2) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 2) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 3))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 3) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 3) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 4))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 4) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 4) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 5))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 5) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 5) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 6))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 6) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 6) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 7))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 7) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 7) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 8))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 8) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 8) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 9))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 9) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 9) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 10))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 10) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 10) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 11))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 11) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 11) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 12))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 12) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 12) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 13))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 13) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 13) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 14))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 14) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 14) & 63)))];
    }
    if (((int)threadIdx.x) < 32) {
      placeholder_shared[(((((int)threadIdx.x) * 16) + 15))] = placeholder1[(((((rc_outer_outer * 4096) + ((((((int)threadIdx.x) * 16) + 15) >> 6) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + (((((int)threadIdx.x) * 16) + 15) & 63)))];
    }
    __syncthreads();
    for (int ff_c_outer_inner = 0; ff_c_outer_inner < 16; ++ff_c_outer_inner) {
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)))] * placeholder_shared[((ff_c_outer_inner * 2))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)))] * placeholder_shared[(((ff_c_outer_inner * 2) + 32))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 56))] * placeholder_shared[((ff_c_outer_inner * 2))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 56))] * placeholder_shared[(((ff_c_outer_inner * 2) + 32))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 784))] * placeholder_shared[((ff_c_outer_inner * 2))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 784))] * placeholder_shared[(((ff_c_outer_inner * 2) + 32))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 840))] * placeholder_shared[((ff_c_outer_inner * 2))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 840))] * placeholder_shared[(((ff_c_outer_inner * 2) + 32))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)))] * placeholder_shared[(((ff_c_outer_inner * 2) + 1))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[(((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)))] * placeholder_shared[(((ff_c_outer_inner * 2) + 33))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 56))] * placeholder_shared[(((ff_c_outer_inner * 2) + 1))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 56))] * placeholder_shared[(((ff_c_outer_inner * 2) + 33))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 784))] * placeholder_shared[(((ff_c_outer_inner * 2) + 1))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 784))] * placeholder_shared[(((ff_c_outer_inner * 2) + 33))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 840))] * placeholder_shared[(((ff_c_outer_inner * 2) + 1))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 840))] * placeholder_shared[(((ff_c_outer_inner * 2) + 33))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 1))] * placeholder_shared[(((ff_c_outer_inner * 2) + 64))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 1))] * placeholder_shared[(((ff_c_outer_inner * 2) + 96))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 57))] * placeholder_shared[(((ff_c_outer_inner * 2) + 64))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 57))] * placeholder_shared[(((ff_c_outer_inner * 2) + 96))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 785))] * placeholder_shared[(((ff_c_outer_inner * 2) + 64))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 785))] * placeholder_shared[(((ff_c_outer_inner * 2) + 96))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 841))] * placeholder_shared[(((ff_c_outer_inner * 2) + 64))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 841))] * placeholder_shared[(((ff_c_outer_inner * 2) + 96))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 1))] * placeholder_shared[(((ff_c_outer_inner * 2) + 65))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 1))] * placeholder_shared[(((ff_c_outer_inner * 2) + 97))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 57))] * placeholder_shared[(((ff_c_outer_inner * 2) + 65))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 57))] * placeholder_shared[(((ff_c_outer_inner * 2) + 97))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 785))] * placeholder_shared[(((ff_c_outer_inner * 2) + 65))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 785))] * placeholder_shared[(((ff_c_outer_inner * 2) + 97))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 841))] * placeholder_shared[(((ff_c_outer_inner * 2) + 65))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 841))] * placeholder_shared[(((ff_c_outer_inner * 2) + 97))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 2))] * placeholder_shared[(((ff_c_outer_inner * 2) + 128))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 2))] * placeholder_shared[(((ff_c_outer_inner * 2) + 160))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 58))] * placeholder_shared[(((ff_c_outer_inner * 2) + 128))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 58))] * placeholder_shared[(((ff_c_outer_inner * 2) + 160))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 786))] * placeholder_shared[(((ff_c_outer_inner * 2) + 128))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 786))] * placeholder_shared[(((ff_c_outer_inner * 2) + 160))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 842))] * placeholder_shared[(((ff_c_outer_inner * 2) + 128))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 842))] * placeholder_shared[(((ff_c_outer_inner * 2) + 160))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 2))] * placeholder_shared[(((ff_c_outer_inner * 2) + 129))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 2))] * placeholder_shared[(((ff_c_outer_inner * 2) + 161))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 58))] * placeholder_shared[(((ff_c_outer_inner * 2) + 129))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 58))] * placeholder_shared[(((ff_c_outer_inner * 2) + 161))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 786))] * placeholder_shared[(((ff_c_outer_inner * 2) + 129))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 786))] * placeholder_shared[(((ff_c_outer_inner * 2) + 161))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 842))] * placeholder_shared[(((ff_c_outer_inner * 2) + 129))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 842))] * placeholder_shared[(((ff_c_outer_inner * 2) + 161))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 3))] * placeholder_shared[(((ff_c_outer_inner * 2) + 192))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 3))] * placeholder_shared[(((ff_c_outer_inner * 2) + 224))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 59))] * placeholder_shared[(((ff_c_outer_inner * 2) + 192))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 59))] * placeholder_shared[(((ff_c_outer_inner * 2) + 224))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 787))] * placeholder_shared[(((ff_c_outer_inner * 2) + 192))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 787))] * placeholder_shared[(((ff_c_outer_inner * 2) + 224))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 843))] * placeholder_shared[(((ff_c_outer_inner * 2) + 192))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 843))] * placeholder_shared[(((ff_c_outer_inner * 2) + 224))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 3))] * placeholder_shared[(((ff_c_outer_inner * 2) + 193))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 3))] * placeholder_shared[(((ff_c_outer_inner * 2) + 225))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 59))] * placeholder_shared[(((ff_c_outer_inner * 2) + 193))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 59))] * placeholder_shared[(((ff_c_outer_inner * 2) + 225))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 787))] * placeholder_shared[(((ff_c_outer_inner * 2) + 193))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 787))] * placeholder_shared[(((ff_c_outer_inner * 2) + 225))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 843))] * placeholder_shared[(((ff_c_outer_inner * 2) + 193))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 843))] * placeholder_shared[(((ff_c_outer_inner * 2) + 225))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 4))] * placeholder_shared[(((ff_c_outer_inner * 2) + 256))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 4))] * placeholder_shared[(((ff_c_outer_inner * 2) + 288))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 60))] * placeholder_shared[(((ff_c_outer_inner * 2) + 256))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 60))] * placeholder_shared[(((ff_c_outer_inner * 2) + 288))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 788))] * placeholder_shared[(((ff_c_outer_inner * 2) + 256))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 788))] * placeholder_shared[(((ff_c_outer_inner * 2) + 288))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 844))] * placeholder_shared[(((ff_c_outer_inner * 2) + 256))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 844))] * placeholder_shared[(((ff_c_outer_inner * 2) + 288))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 4))] * placeholder_shared[(((ff_c_outer_inner * 2) + 257))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 4))] * placeholder_shared[(((ff_c_outer_inner * 2) + 289))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 60))] * placeholder_shared[(((ff_c_outer_inner * 2) + 257))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 60))] * placeholder_shared[(((ff_c_outer_inner * 2) + 289))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 788))] * placeholder_shared[(((ff_c_outer_inner * 2) + 257))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 788))] * placeholder_shared[(((ff_c_outer_inner * 2) + 289))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 844))] * placeholder_shared[(((ff_c_outer_inner * 2) + 257))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 844))] * placeholder_shared[(((ff_c_outer_inner * 2) + 289))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 5))] * placeholder_shared[(((ff_c_outer_inner * 2) + 320))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 5))] * placeholder_shared[(((ff_c_outer_inner * 2) + 352))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 61))] * placeholder_shared[(((ff_c_outer_inner * 2) + 320))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 61))] * placeholder_shared[(((ff_c_outer_inner * 2) + 352))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 789))] * placeholder_shared[(((ff_c_outer_inner * 2) + 320))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 789))] * placeholder_shared[(((ff_c_outer_inner * 2) + 352))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 845))] * placeholder_shared[(((ff_c_outer_inner * 2) + 320))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 845))] * placeholder_shared[(((ff_c_outer_inner * 2) + 352))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 5))] * placeholder_shared[(((ff_c_outer_inner * 2) + 321))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 5))] * placeholder_shared[(((ff_c_outer_inner * 2) + 353))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 61))] * placeholder_shared[(((ff_c_outer_inner * 2) + 321))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 61))] * placeholder_shared[(((ff_c_outer_inner * 2) + 353))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 789))] * placeholder_shared[(((ff_c_outer_inner * 2) + 321))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 789))] * placeholder_shared[(((ff_c_outer_inner * 2) + 353))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 845))] * placeholder_shared[(((ff_c_outer_inner * 2) + 321))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 845))] * placeholder_shared[(((ff_c_outer_inner * 2) + 353))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 6))] * placeholder_shared[(((ff_c_outer_inner * 2) + 384))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 6))] * placeholder_shared[(((ff_c_outer_inner * 2) + 416))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 62))] * placeholder_shared[(((ff_c_outer_inner * 2) + 384))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 62))] * placeholder_shared[(((ff_c_outer_inner * 2) + 416))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 790))] * placeholder_shared[(((ff_c_outer_inner * 2) + 384))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 790))] * placeholder_shared[(((ff_c_outer_inner * 2) + 416))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 846))] * placeholder_shared[(((ff_c_outer_inner * 2) + 384))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 846))] * placeholder_shared[(((ff_c_outer_inner * 2) + 416))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 6))] * placeholder_shared[(((ff_c_outer_inner * 2) + 385))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 6))] * placeholder_shared[(((ff_c_outer_inner * 2) + 417))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 62))] * placeholder_shared[(((ff_c_outer_inner * 2) + 385))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 62))] * placeholder_shared[(((ff_c_outer_inner * 2) + 417))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 790))] * placeholder_shared[(((ff_c_outer_inner * 2) + 385))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 790))] * placeholder_shared[(((ff_c_outer_inner * 2) + 417))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 846))] * placeholder_shared[(((ff_c_outer_inner * 2) + 385))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 846))] * placeholder_shared[(((ff_c_outer_inner * 2) + 417))]));
      Conv2dOutput_local[((ff_c_outer_inner * 2))] = (Conv2dOutput_local[((ff_c_outer_inner * 2))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 7))] * placeholder_shared[(((ff_c_outer_inner * 2) + 448))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 32))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 7))] * placeholder_shared[(((ff_c_outer_inner * 2) + 480))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 64))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 63))] * placeholder_shared[(((ff_c_outer_inner * 2) + 448))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 96))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 63))] * placeholder_shared[(((ff_c_outer_inner * 2) + 480))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 128))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 791))] * placeholder_shared[(((ff_c_outer_inner * 2) + 448))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 160))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 791))] * placeholder_shared[(((ff_c_outer_inner * 2) + 480))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 192))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 847))] * placeholder_shared[(((ff_c_outer_inner * 2) + 448))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 224))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 847))] * placeholder_shared[(((ff_c_outer_inner * 2) + 480))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 1))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 7))] * placeholder_shared[(((ff_c_outer_inner * 2) + 449))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 33))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 7))] * placeholder_shared[(((ff_c_outer_inner * 2) + 481))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 65))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 63))] * placeholder_shared[(((ff_c_outer_inner * 2) + 449))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 97))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 63))] * placeholder_shared[(((ff_c_outer_inner * 2) + 481))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 129))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 791))] * placeholder_shared[(((ff_c_outer_inner * 2) + 449))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 161))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 791))] * placeholder_shared[(((ff_c_outer_inner * 2) + 481))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 193))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 847))] * placeholder_shared[(((ff_c_outer_inner * 2) + 449))]));
      Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] = (Conv2dOutput_local[(((ff_c_outer_inner * 2) + 225))] + (PaddedInput_shared[((((((((int)threadIdx.x) / 49) * 1568) + (((((int)threadIdx.x) % 49) / 7) * 112)) + ((((int)threadIdx.x) % 7) * 8)) + 847))] * placeholder_shared[(((ff_c_outer_inner * 2) + 481))]));
    }
  }
  for (int ff_inner = 0; ff_inner < 32; ++ff_inner) {
    Conv2dOutput[((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner))] = Conv2dOutput_local[(ff_inner)];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 32))] = Conv2dOutput_local[((ff_inner + 32))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 3584))] = Conv2dOutput_local[((ff_inner + 64))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 3616))] = Conv2dOutput_local[((ff_inner + 96))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 50176))] = Conv2dOutput_local[((ff_inner + 128))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 50208))] = Conv2dOutput_local[((ff_inner + 160))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 53760))] = Conv2dOutput_local[((ff_inner + 192))];
    Conv2dOutput[(((((((((((int)blockIdx.x) >> 3) * 200704) + ((((int)threadIdx.x) / 49) * 100352)) + (((((int)threadIdx.x) % 49) / 7) * 7168)) + ((((int)threadIdx.x) % 7) * 512)) + ((((int)blockIdx.x) & 7) * 64)) + ff_inner) + 53792))] = Conv2dOutput_local[((ff_inner + 224))];
  }
}


