
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float placeholder_d_shared[512];
  __shared__ float placeholder_shared[1024];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(16)] = 0.000000e+00f;
  compute_local[(24)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(17)] = 0.000000e+00f;
  compute_local[(25)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(18)] = 0.000000e+00f;
  compute_local[(26)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(19)] = 0.000000e+00f;
  compute_local[(27)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(20)] = 0.000000e+00f;
  compute_local[(28)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(21)] = 0.000000e+00f;
  compute_local[(29)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(22)] = 0.000000e+00f;
  compute_local[(30)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  compute_local[(23)] = 0.000000e+00f;
  compute_local[(31)] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    placeholder_d_shared[(((int)threadIdx.x))] = placeholder[((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 7) * 65536)) + ((((int)blockIdx.x) & 7) * 8192)) + (((((int)threadIdx.x) & 127) >> 2) * 256)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    placeholder_d_shared[((((int)threadIdx.x) + 256))] = placeholder[(((((((((((int)blockIdx.x) >> 3) * 262144) + ((((int)threadIdx.x) >> 7) * 65536)) + ((((int)blockIdx.x) & 7) * 8192)) + (((((int)threadIdx.x) & 127) >> 2) * 256)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 131072))];
    placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 2) * 256)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)))];
    placeholder_shared[((((int)threadIdx.x) + 256))] = placeholder1[(((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 2) * 256)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 16384))];
    placeholder_shared[((((int)threadIdx.x) + 512))] = placeholder1[(((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 2) * 256)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 32768))];
    placeholder_shared[((((int)threadIdx.x) + 768))] = placeholder1[(((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)threadIdx.x) >> 2) * 256)) + (k_outer_outer * 4)) + (((int)threadIdx.x) & 3)) + 49152))];
    __syncthreads();
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[(((((int)threadIdx.x) >> 4) * 8))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 16))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 256))]));
    compute_local[(16)] = (compute_local[(16)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 256))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 512))]));
    compute_local[(24)] = (compute_local[(24)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 384))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 768))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[(((((int)threadIdx.x) >> 4) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 4))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 260))]));
    compute_local[(17)] = (compute_local[(17)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 256))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 516))]));
    compute_local[(25)] = (compute_local[(25)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 384))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 772))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[(((((int)threadIdx.x) >> 4) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 8))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 264))]));
    compute_local[(18)] = (compute_local[(18)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 256))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 520))]));
    compute_local[(26)] = (compute_local[(26)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 384))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 776))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[(((((int)threadIdx.x) >> 4) * 8))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 12))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 128))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 268))]));
    compute_local[(19)] = (compute_local[(19)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 256))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 524))]));
    compute_local[(27)] = (compute_local[(27)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 384))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 780))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 1))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 257))]));
    compute_local[(16)] = (compute_local[(16)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 257))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 513))]));
    compute_local[(24)] = (compute_local[(24)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 385))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 769))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 5))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 261))]));
    compute_local[(17)] = (compute_local[(17)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 257))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 517))]));
    compute_local[(25)] = (compute_local[(25)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 385))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 773))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 9))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 265))]));
    compute_local[(18)] = (compute_local[(18)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 257))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 521))]));
    compute_local[(26)] = (compute_local[(26)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 385))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 777))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 1))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 13))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 129))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 269))]));
    compute_local[(19)] = (compute_local[(19)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 257))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 525))]));
    compute_local[(27)] = (compute_local[(27)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 385))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 781))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 2))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 258))]));
    compute_local[(16)] = (compute_local[(16)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 258))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 514))]));
    compute_local[(24)] = (compute_local[(24)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 386))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 770))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 6))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 262))]));
    compute_local[(17)] = (compute_local[(17)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 258))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 518))]));
    compute_local[(25)] = (compute_local[(25)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 386))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 774))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 10))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 266))]));
    compute_local[(18)] = (compute_local[(18)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 258))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 522))]));
    compute_local[(26)] = (compute_local[(26)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 386))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 778))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 2))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 14))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 130))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 270))]));
    compute_local[(19)] = (compute_local[(19)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 258))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 526))]));
    compute_local[(27)] = (compute_local[(27)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 386))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 782))]));
    compute_local[(0)] = (compute_local[(0)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 3))]));
    compute_local[(8)] = (compute_local[(8)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 259))]));
    compute_local[(16)] = (compute_local[(16)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 259))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 515))]));
    compute_local[(24)] = (compute_local[(24)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 387))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 771))]));
    compute_local[(1)] = (compute_local[(1)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 7))]));
    compute_local[(9)] = (compute_local[(9)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 263))]));
    compute_local[(17)] = (compute_local[(17)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 259))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 519))]));
    compute_local[(25)] = (compute_local[(25)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 387))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 775))]));
    compute_local[(2)] = (compute_local[(2)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 11))]));
    compute_local[(10)] = (compute_local[(10)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 267))]));
    compute_local[(18)] = (compute_local[(18)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 259))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 523))]));
    compute_local[(26)] = (compute_local[(26)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 387))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 779))]));
    compute_local[(3)] = (compute_local[(3)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 3))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 15))]));
    compute_local[(11)] = (compute_local[(11)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 131))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 271))]));
    compute_local[(19)] = (compute_local[(19)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 259))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 527))]));
    compute_local[(27)] = (compute_local[(27)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 387))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 783))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 4))] * placeholder_shared[(((((int)threadIdx.x) & 15) * 16))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 256))]));
    compute_local[(20)] = (compute_local[(20)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 260))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 512))]));
    compute_local[(28)] = (compute_local[(28)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 388))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 768))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 4))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 260))]));
    compute_local[(21)] = (compute_local[(21)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 260))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 516))]));
    compute_local[(29)] = (compute_local[(29)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 388))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 772))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 8))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 264))]));
    compute_local[(22)] = (compute_local[(22)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 260))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 520))]));
    compute_local[(30)] = (compute_local[(30)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 388))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 776))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 4))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 12))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 132))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 268))]));
    compute_local[(23)] = (compute_local[(23)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 260))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 524))]));
    compute_local[(31)] = (compute_local[(31)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 388))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 780))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 1))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 257))]));
    compute_local[(20)] = (compute_local[(20)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 261))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 513))]));
    compute_local[(28)] = (compute_local[(28)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 389))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 769))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 5))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 261))]));
    compute_local[(21)] = (compute_local[(21)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 261))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 517))]));
    compute_local[(29)] = (compute_local[(29)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 389))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 773))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 9))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 265))]));
    compute_local[(22)] = (compute_local[(22)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 261))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 521))]));
    compute_local[(30)] = (compute_local[(30)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 389))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 777))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 5))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 13))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 133))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 269))]));
    compute_local[(23)] = (compute_local[(23)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 261))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 525))]));
    compute_local[(31)] = (compute_local[(31)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 389))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 781))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 2))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 258))]));
    compute_local[(20)] = (compute_local[(20)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 262))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 514))]));
    compute_local[(28)] = (compute_local[(28)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 390))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 770))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 6))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 262))]));
    compute_local[(21)] = (compute_local[(21)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 262))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 518))]));
    compute_local[(29)] = (compute_local[(29)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 390))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 774))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 10))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 266))]));
    compute_local[(22)] = (compute_local[(22)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 262))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 522))]));
    compute_local[(30)] = (compute_local[(30)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 390))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 778))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 6))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 14))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 134))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 270))]));
    compute_local[(23)] = (compute_local[(23)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 262))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 526))]));
    compute_local[(31)] = (compute_local[(31)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 390))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 782))]));
    compute_local[(4)] = (compute_local[(4)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 3))]));
    compute_local[(12)] = (compute_local[(12)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 259))]));
    compute_local[(20)] = (compute_local[(20)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 263))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 515))]));
    compute_local[(28)] = (compute_local[(28)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 391))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 771))]));
    compute_local[(5)] = (compute_local[(5)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 7))]));
    compute_local[(13)] = (compute_local[(13)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 263))]));
    compute_local[(21)] = (compute_local[(21)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 263))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 519))]));
    compute_local[(29)] = (compute_local[(29)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 391))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 775))]));
    compute_local[(6)] = (compute_local[(6)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 11))]));
    compute_local[(14)] = (compute_local[(14)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 267))]));
    compute_local[(22)] = (compute_local[(22)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 263))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 523))]));
    compute_local[(30)] = (compute_local[(30)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 391))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 779))]));
    compute_local[(7)] = (compute_local[(7)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 7))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 15))]));
    compute_local[(15)] = (compute_local[(15)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 135))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 271))]));
    compute_local[(23)] = (compute_local[(23)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 263))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 527))]));
    compute_local[(31)] = (compute_local[(31)] + (placeholder_d_shared[((((((int)threadIdx.x) >> 4) * 8) + 391))] * placeholder_shared[((((((int)threadIdx.x) & 15) * 16) + 783))]));
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    for (int j_inner = 0; j_inner < 4; ++j_inner) {
      compute[((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (i_inner * 64)) + ((((int)threadIdx.x) & 15) * 4)) + j_inner))] = compute_local[(((i_inner * 4) + j_inner))];
      compute[(((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (i_inner * 64)) + ((((int)threadIdx.x) & 15) * 4)) + j_inner) + 16384))] = compute_local[((((i_inner * 4) + j_inner) + 8))];
      compute[(((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (i_inner * 64)) + ((((int)threadIdx.x) & 15) * 4)) + j_inner) + 32768))] = compute_local[((((i_inner * 4) + j_inner) + 16))];
      compute[(((((((((((int)blockIdx.x) >> 3) * 65536) + ((((int)blockIdx.x) & 7) * 2048)) + ((((int)threadIdx.x) >> 4) * 128)) + (i_inner * 64)) + ((((int)threadIdx.x) & 15) * 4)) + j_inner) + 49152))] = compute_local[((((i_inner * 4) + j_inner) + 24))];
    }
  }
}


