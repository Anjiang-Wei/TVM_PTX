
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void my_kernel_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float Conv2dOutput[224];
  __shared__ float PaddedInput_shared[8960];
  __shared__ float placeholder_shared[1280];
  for (int xx_outer_inner_init = 0; xx_outer_inner_init < 14; ++xx_outer_inner_init) {
    for (int yy_inner_init = 0; yy_inner_init < 2; ++yy_inner_init) {
      for (int ff_inner_init = 0; ff_inner_init < 2; ++ff_inner_init) {
        Conv2dOutput[((((yy_inner_init * 28) + (xx_outer_inner_init * 2)) + ff_inner_init))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 28) + (xx_outer_inner_init * 2)) + ff_inner_init) + 56))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 28) + (xx_outer_inner_init * 2)) + ff_inner_init) + 112))] = 0.000000e+00f;
        Conv2dOutput[(((((yy_inner_init * 28) + (xx_outer_inner_init * 2)) + ff_inner_init) + 168))] = 0.000000e+00f;
      }
    }
  }
  for (int rc_outer_outer = 0; rc_outer_outer < 2; ++rc_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 4480; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
      PaddedInput_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)) / 4480) * 62720) + ((((int)blockIdx.x) >> 4) * 8960)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)) % 4480) / 160) * 320)) + (rc_outer_outer * 160)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 2) + ((int)threadIdx.x)) % 160)))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 < 640; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1) {
      placeholder_shared[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)))] = placeholder1[(((((rc_outer_outer * 20480) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)) >> 3) * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer1 * 2) + ((int)threadIdx.x)) & 7)))];
    }
    __syncthreads();
    for (int rc_outer_inner = 0; rc_outer_inner < 4; ++rc_outer_inner) {
      for (int xx_outer_inner = 0; xx_outer_inner < 14; ++xx_outer_inner) {
        for (int rc_inner = 0; rc_inner < 40; ++rc_inner) {
          for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
            for (int ff_inner = 0; ff_inner < 2; ++ff_inner) {
              Conv2dOutput[((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner))] = (Conv2dOutput[((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner))] + (PaddedInput_shared[(((((yy_inner * 2240) + (xx_outer_inner * 160)) + (rc_outer_inner * 40)) + rc_inner))] * placeholder_shared[(((((rc_outer_inner * 320) + (rc_inner * 8)) + (((int)threadIdx.x) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner) + 56))] = (Conv2dOutput[(((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner) + 56))] + (PaddedInput_shared[(((((yy_inner * 2240) + (xx_outer_inner * 160)) + (rc_outer_inner * 40)) + rc_inner))] * placeholder_shared[((((((rc_outer_inner * 320) + (rc_inner * 8)) + (((int)threadIdx.x) * 2)) + ff_inner) + 4))]));
              Conv2dOutput[(((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner) + 112))] = (Conv2dOutput[(((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner) + 112))] + (PaddedInput_shared[((((((yy_inner * 2240) + (xx_outer_inner * 160)) + (rc_outer_inner * 40)) + rc_inner) + 4480))] * placeholder_shared[(((((rc_outer_inner * 320) + (rc_inner * 8)) + (((int)threadIdx.x) * 2)) + ff_inner))]));
              Conv2dOutput[(((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner) + 168))] = (Conv2dOutput[(((((yy_inner * 28) + (xx_outer_inner * 2)) + ff_inner) + 168))] + (PaddedInput_shared[((((((yy_inner * 2240) + (xx_outer_inner * 160)) + (rc_outer_inner * 40)) + rc_inner) + 4480))] * placeholder_shared[((((((rc_outer_inner * 320) + (rc_inner * 8)) + (((int)threadIdx.x) * 2)) + ff_inner) + 4))]));
            }
          }
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    for (int ax2_inner = 0; ax2_inner < 14; ++ax2_inner) {
      for (int ax3_inner = 0; ax3_inner < 2; ++ax3_inner) {
        T_relu[((((((((((int)blockIdx.x) >> 4) * 3584) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((int)threadIdx.x) * 2)) + ax3_inner))] = max((Conv2dOutput[((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner))] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) >> 4) * 3584) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 4))] = max((Conv2dOutput[(((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner) + 56))] + placeholder2[((((((((int)blockIdx.x) & 15) * 8) + (((int)threadIdx.x) * 2)) + ax3_inner) + 4))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) >> 4) * 3584) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 25088))] = max((Conv2dOutput[(((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner) + 112))] + placeholder2[(((((((int)blockIdx.x) & 15) * 8) + (((int)threadIdx.x) * 2)) + ax3_inner))]), 0.000000e+00f);
        T_relu[(((((((((((int)blockIdx.x) >> 4) * 3584) + (ax1_inner * 1792)) + (ax2_inner * 128)) + ((((int)blockIdx.x) & 15) * 8)) + (((int)threadIdx.x) * 2)) + ax3_inner) + 25092))] = max((Conv2dOutput[(((((ax1_inner * 28) + (ax2_inner * 2)) + ax3_inner) + 168))] + placeholder2[((((((((int)blockIdx.x) & 15) * 8) + (((int)threadIdx.x) * 2)) + ax3_inner) + 4))]), 0.000000e+00f);
      }
    }
  }
}


